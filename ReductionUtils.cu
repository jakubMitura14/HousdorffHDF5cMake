
#include <hip/hip_runtime.h>
//
//#include "cuda_runtime.h"
//#include <cstdint>
//#include <cooperative_groups.h>
//#include <cooperative_groups/reduce.h>
//namespace cg = cooperative_groups;
///// The following example accepts input in *A and outputs a result into *sum
///// It spreads the data within the block, one element per thread
//template <typename TYU>
//inline __device__ void block_reduce(const int* A, int* sum, thread_block cta, thread_block_tile<32> tile) {
//	__shared__ int reduction_s[32];
//	//cg::thread_block cta = cg::this_thread_block();
//	//cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cta);
//	const int tid = cta.thread_rank();
//	// reduce across the tile
//	// cg::plus<int> allows cg::reduce() to know it can use hardware acceleration
//
//	reduction_s[tid] = cg::reduce(tile, beta, cg::plus<int>());
//	// synchronize the block so all data is ready
//	cg::sync(cta);
//	// single leader accumulates the result
//	if (cta.thread_rank() == 0) {
//		beta = 0;
//		for (int i = 0; i < blocksz; i += tile.num_threads()) {
//			beta += reduction_s[i];
//		}
//
//		sum[blockIdx.x] = beta;
//	}
//}
