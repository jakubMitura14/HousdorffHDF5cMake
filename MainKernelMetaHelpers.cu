/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, uint16_t*& resultListPointer) {
    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes.arrP[0][0][7] + cpuArgs.metaData.minMaxes.arrP[0][0][8];

    size_t size = sizeof(uint16_t) * 5 * fpPlusFn + 1;
    hipMallocAsync(&resultListPointer, size, 0);
    gpuArgs.metaData.resultList = resultListPointer;


    // hipFreeAsync(gpuArgs.metaData.resultList, 0);

     //hipFree(resultListPointer);


};




#pragma once
template <typename ZZR>
inline void allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs) {
    ////reduced arrays
    array3dWithDimsGPU reducedGold;
    array3dWithDimsGPU reducedSegm;

    array3dWithDimsGPU reducedGoldRef;
    array3dWithDimsGPU reducedSegmRef;


    array3dWithDimsGPU reducedGoldPrev;
    array3dWithDimsGPU reducedSegmPrev;


    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes.arrP[0][0][1] - cpuArgs.metaData.minMaxes.arrP[0][0][2];
    unsigned int yRange = cpuArgs.metaData.minMaxes.arrP[0][0][3] - cpuArgs.metaData.minMaxes.arrP[0][0][4];
    unsigned int zRange = cpuArgs.metaData.minMaxes.arrP[0][0][5] - cpuArgs.metaData.minMaxes.arrP[0][0][6];

    //allocating needed memory
    reducedGold = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    reducedSegm = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    reducedGoldRef = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    reducedSegmRef = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    reducedGoldPrev = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    reducedSegmPrev = getArrGpu<uint32_t>(xRange * cpuArgs.dbXLength, yRange * cpuArgs.dbYLength, zRange * cpuArgs.dbZLength);
    allocateMetaDataOnGPU(xRange, yRange, zRange);
    //unsigned int fpPlusFn = fFArgs.metaData.minMaxes.arrP[0][0][7] + fFArgs.metaData.minMaxes.arrP[0][0][8];
    //uint16_t* resultListPointer;
    //size_t size = sizeof(uint16_t) * 5 * fpPlusFn + 1;
    //hipMallocAsync(&resultListPointer, size, 0);
    //fbArgs.metaData.resultList = resultListPointer;


};



//
//#pragma once
//template <typename ZZR>
//inline void calculateOccupancy(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs) {
//   
//
//    int numBlocks; // Occupancy in terms of active blocks
//    int blockSize = 32;
//    // These variables are used to convert occupancy to warps
//    int device;
//    hipDeviceProp_t prop;
//    int activeWarps;
//    int maxWarps;
//    hipGetDevice(&device);
//    hipGetDeviceProperties(&prop, device);
//
//    hipOccupancyMaxActiveBlocksPerMultiprocessor(
//        &numBlocks,
//        MyKernel,
//        blockSize,
//        0);
//    activeWarps = numBlocks * blockSize / prop.warpSize;
//    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
//    std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" <<
//        std::endl;
//
//
//
//
//
//
//};


