#include "hip/hip_runtime.h"
/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, uint32_t*& resultListPointer) {
    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];

    size = sizeof(uint32_t) * 5 * fpPlusFn + 1;
    hipMallocAsync(&resultListPointer, size, 0);
    gpuArgs.metaData.resultList = resultListPointer;


    // hipFreeAsync(gpuArgs.metaData.resultList, 0);

     //hipFree(resultListPointer);


};




#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs
            , uint32_t*& mainArr, uint32_t*& workQueue, unsigned int* minMaxes, MetaDataGPU metaData
) {
    ////reduced arrays


    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy( cpuArgs.metaData.minMaxes, minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2]+1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4]+1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6]+1;
    unsigned int totalMetaLength = xRange* yRange* zRange;

 
    //updating size informations
    metaData.metaXLength = xRange;
    metaData.MetaYLength = yRange;
    metaData.MetaZLength = zRange;
    metaData.totalMetaLength = totalMetaLength;

    cpuArgs.metaData.metaXLength = xRange;
    cpuArgs.metaData.MetaYLength = yRange;
    cpuArgs.metaData.MetaZLength = zRange;
    cpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    metaData.maxX = cpuArgs.metaData.minMaxes[1];
    metaData.minX = cpuArgs.metaData.minMaxes[2];
    metaData.maxY = cpuArgs.metaData.minMaxes[3];
    metaData.minY = cpuArgs.metaData.minMaxes[4];
    metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    metaData.minZ = cpuArgs.metaData.minMaxes[6];

    //allocating needed memory
    // main array
    unsigned int mainArrXLength = cpuArgs.dbXLength * cpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 6) + 18;
    metaData.mainArrXLength = mainArrXLength;
    metaData.mainArrSectionLength = mainArrSectionLength;
    metaData.metaDataOffset = (mainArrXLength * 6);
    
    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);
    std::cout <<"totalMetaLength  ";
    std::cout << totalMetaLength;
    std::cout << "\n";


    //std::cout << "xRange  ";
    //std::cout << xRange;
    //std::cout << "\n";

    //std::cout << "yRange  ";
    //std::cout << yRange;
    //std::cout << "\n";

    //std::cout << "zRange  ";
    //std::cout << zRange;
    //std::cout << "\n";


    hipMallocAsync(&mainArr, sizeB, 0);
    //workqueue

    size_t sizeC = (totalMetaLength * sizeof(uint32_t));
   //hipMallocAsync(&workQueue, size, 0);
   hipMalloc(&workQueue, size);

   return metaData;
};




#pragma once
template <typename ZZR>
inline void printForDebug(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, uint32_t* resultListPointer
    , uint32_t* mainArrPointer, uint32_t* workQueuePointer, MetaDataGPU metaData) {
    // getting arrays allocated on  cpu to be able to print and test them easier
    size_t size = sizeof(uint32_t) * metaData.totalMetaLength* metaData.mainArrSectionLength;
    uint32_t* mainArrCPU = (uint32_t*)malloc(size);


    for (int linIdexMeta = 0; linIdexMeta < metaData.totalMetaLength; linIdexMeta++) {
        

    }

};


