#include "hip/hip_runtime.h"
/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline int allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, 
    uint32_t*& resultListPointerMeta
    ,uint32_t*& resultListPointerLocal
    ,uint32_t*& resultListPointerIterNumb,
    uint32_t*& origArrsPointer,
    uint32_t*& mainArrAPointer,
    uint32_t*& mainArrBPointer, MetaDataGPU metaData,array3dWithDimsGPU goldArr, array3dWithDimsGPU segmArr) {
    
    //free no longer needed arrays
    hipFreeAsync(goldArr.arrPStr.ptr, 0);
    hipFreeAsync(segmArr.arrPStr.ptr, 0);
    
    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];


    size = sizeof(uint32_t)* (fpPlusFn + 50);



    //hipMalloc(&resultListPointerLocal, size);
    //hipMalloc(&resultListPointerIterNumb, size);
    //hipMalloc(&resultListPointerMeta, size);

    hipMallocAsync(&resultListPointerLocal, size, 0);
    hipMallocAsync(&resultListPointerIterNumb, size, 0);
    hipMallocAsync(&resultListPointerMeta, size, 0);


   auto xRange  = metaData.metaXLength ;
   auto yRange =  metaData.MetaYLength ;
   auto zRange = metaData.MetaZLength ;
    
    

    
    size_t sizeB = metaData.totalMetaLength * metaData.mainArrSectionLength * sizeof(uint32_t);

    hipMallocAsync(&mainArrAPointer, sizeB, 0);
    hipMemcpyAsync(mainArrAPointer, origArrsPointer, sizeB, hipMemcpyDeviceToDevice,0);

    
    hipMallocAsync(&mainArrBPointer, sizeB, 0);
    hipMemcpyAsync(mainArrBPointer, origArrsPointer, sizeB, hipMemcpyDeviceToDevice,0);

    
   // size_t sizeorigArr = totalMetaLength * (mainArrXLength * 2) * sizeof(uint32_t);
    
   // metaData.resultList = resultListPointer;


    // hipFreeAsync(gpuArgs.metaData.resultList, 0);

     //hipFree(resultListPointer);

    return fpPlusFn;
};




#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs,
             uint32_t*& workQueue, unsigned int* minMaxes, MetaDataGPU metaData, uint32_t*& origArr
    , uint32_t*& metaDataArr) {
    ////reduced arrays


    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy( cpuArgs.metaData.minMaxes, minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2]+1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4]+1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6]+1;
    unsigned int totalMetaLength = xRange* yRange* zRange;

 
    //updating size informations
    metaData.metaXLength = xRange;
    metaData.MetaYLength = yRange;
    metaData.MetaZLength = zRange;
    metaData.totalMetaLength = totalMetaLength;

    cpuArgs.metaData.metaXLength = xRange;
    cpuArgs.metaData.MetaYLength = yRange;
    cpuArgs.metaData.MetaZLength = zRange;
    cpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    metaData.maxX = cpuArgs.metaData.minMaxes[1];
    metaData.minX = cpuArgs.metaData.minMaxes[2];
    metaData.maxY = cpuArgs.metaData.minMaxes[3];
    metaData.minY = cpuArgs.metaData.minMaxes[4];
    metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    metaData.minZ = cpuArgs.metaData.minMaxes[6];

    //allocating needed memory
    // main array
    unsigned int mainArrXLength = gpuArgs.dbXLength * gpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 2);
    metaData.mainArrXLength = mainArrXLength;
    metaData.mainArrSectionLength = mainArrSectionLength;
    
    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);


    //hipMallocAsync(&mainArr, sizeB, 0);
    size_t sizeorigArr = totalMetaLength * (mainArrXLength * 2) * sizeof(uint32_t);
    hipMallocAsync(&origArr, sizeorigArr, 0);
    size_t sizemetaDataArr = totalMetaLength * (20) * sizeof(uint32_t);
    hipMallocAsync(&metaDataArr, sizemetaDataArr, 0);

    
    size_t sizeC = (totalMetaLength * sizeof(uint32_t));
   //hipMallocAsync(&workQueue, size, 0);
    hipMallocAsync(&workQueue, sizeC,0);

   return metaData;
};



