#include "hip/hip_runtime.h"
/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, uint32_t*& resultListPointer) {
    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];

    size = sizeof(uint32_t) * 5 * fpPlusFn + 1;
    hipMallocAsync(&resultListPointer, size, 0);
    gpuArgs.metaData.resultList = resultListPointer;


    // hipFreeAsync(gpuArgs.metaData.resultList, 0);

     //hipFree(resultListPointer);


};




#pragma once
template <typename ZZR>
inline void allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs
            , uint32_t*& mainArr, uint32_t*& workQueue, unsigned int* minMaxes
) {
    ////reduced arrays


    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy( cpuArgs.metaData.minMaxes, minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2];
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4];
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6];
    unsigned int totalMetaLength = xRange* yRange* zRange;

 
    //updating size informations
    gpuArgs.metaData.metaXLength = xRange;
    gpuArgs.metaData.MetaYLength = yRange;
    gpuArgs.metaData.MetaZLength = zRange;
    gpuArgs.metaData.totalMetaLength = totalMetaLength;

    cpuArgs.metaData.metaXLength = xRange;
    cpuArgs.metaData.MetaYLength = yRange;
    cpuArgs.metaData.MetaZLength = zRange;
    cpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    gpuArgs.maxX = cpuArgs.metaData.minMaxes[1];
    gpuArgs.minX = cpuArgs.metaData.minMaxes[2];
    gpuArgs.maxY = cpuArgs.metaData.minMaxes[3];
    gpuArgs.minY = cpuArgs.metaData.minMaxes[4];
    gpuArgs.maxZ = cpuArgs.metaData.minMaxes[5];
    gpuArgs.minZ = cpuArgs.metaData.minMaxes[6];

    //allocating needed memory
    // main array
    unsigned int mainArrXLength = cpuArgs.dbXLength * cpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 6) + 18;
    gpuArgs.mainArrXLength = mainArrXLength;
    gpuArgs.mainArrSectionLength = mainArrSectionLength;
    gpuArgs.metaDataOffset = (mainArrXLength * 6);
    
    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);
    std::cout <<"size  ";
    std::cout << (totalMetaLength * mainArrSectionLength * sizeof(uint32_t))/1000000000;
    std::cout << "\n";
    hipMallocAsync(&mainArr, sizeB, 0);
    //workqueue

    size_t sizeC = (totalMetaLength * sizeof(uint32_t));
   //hipMallocAsync(&workQueue, size, 0);
   hipMalloc(&workQueue, size);


};




#pragma once
template <typename ZZR>
inline void printForDebug(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs) {
    // getting arrays allocated on  cpu to be able to print and test them easier
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);


};


