#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include "Structs.cu"


#pragma once
template <typename EEY>
array3dWithDimsCPU<EEY>  get3dArrCPU(EEY* arrP, int Nx, int Ny, int Nz) {
    array3dWithDimsCPU<EEY> res;
    res.Nx = Nx;
    res.Ny = Ny;
    res.Nz = Nz;
    res.arrP = arrP;

    return res;
}



template <typename T >
array3dWithDimsGPU<T> allocateMainArray(T*& gpuArrPointer, T*& cpuArrPointer, const int WIDTH, const int HEIGHT, const int DEPTH, hipStream_t stream) {
    size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    array3dWithDimsGPU<T> res;

    hipMallocAsync(&gpuArrPointer, sizeMainArr, stream);
    hipMemcpyAsync(gpuArrPointer, cpuArrPointer, sizeMainArr, hipMemcpyHostToDevice, stream);
    res.arrP = gpuArrPointer;
    res.Nx = WIDTH;
    res.Ny = HEIGHT;
    res.Nz = DEPTH;
    return res;
}




/*
given appropriate hipPitchedPtr and ForFullBoolPrepArgs will return ForBoolKernelArgs
*/
#pragma once
template <typename TCC>
inline ForBoolKernelArgs<TCC> getArgsForKernel(ForFullBoolPrepArgs<TCC>& mainFunArgs
    , int& warpsNumbForMainPass, int& blockForMainPass
    , const int xLen, const int yLen, const int zLen, hipStream_t stream
) {

    //main arrays allocations
    TCC* goldArrPointer;
    TCC* segmArrPointer;
    //size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    size_t sizeMainArr = (sizeof(TCC) * xLen * yLen * zLen);
    array3dWithDimsGPU<TCC> goldArr = allocateMainArray(goldArrPointer, mainFunArgs.goldArr.arrP, xLen, yLen, zLen, stream);
    array3dWithDimsGPU<TCC> segmArr = allocateMainArray(segmArrPointer, mainFunArgs.segmArr.arrP, xLen, yLen, zLen, stream);
    unsigned int* minMaxes;
    size_t sizeminMaxes = sizeof(unsigned int) * 20;
    hipMallocAsync(&minMaxes, sizeminMaxes, stream);
    ForBoolKernelArgs<TCC> res;
    res.metaData = allocateMetaDataOnGPU(mainFunArgs.metaData, minMaxes);
    res.metaData.minMaxes = minMaxes;
    res.minMaxes = minMaxes;
    res.numberToLookFor = mainFunArgs.numberToLookFor;
    res.dbXLength = 32;
    res.dbYLength = warpsNumbForMainPass;
    res.dbZLength = 32;

    //printf("in setting bool args ylen %d dbYlen %d calculated meta %d  \n ", yLen, res.dbYLength, int(ceil(yLen / res.dbYLength)));
    res.metaData.metaXLength = int(ceil(xLen / res.dbXLength));
    res.metaData.MetaYLength = int(ceil(yLen / res.dbYLength));;
    res.metaData.MetaZLength = int(ceil(zLen / res.dbZLength));;
    res.metaData.minX = 0;
    res.metaData.minY = 0;
    res.metaData.minZ = 0;
    res.metaData.maxX = res.metaData.metaXLength;
    res.metaData.maxY = res.metaData.MetaYLength;
    res.metaData.maxZ = res.metaData.MetaZLength;

    res.metaData.totalMetaLength = res.metaData.metaXLength * res.metaData.MetaYLength * res.metaData.MetaZLength;
    res.goldArr = goldArr;
    res.segmArr = segmArr;


    return res;
}







#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs, hipStream_t stream) {
    ////reduced arrays
    uint32_t* origArr;
    uint32_t* metaDataArr;
    uint32_t* workQueue;
    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpyAsync(cpuArgs.metaData.minMaxes, gpuArgs.minMaxes, size, hipMemcpyDeviceToHost, stream);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2] + 1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4] + 1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6] + 1;
    unsigned int totalMetaLength = (xRange) * (yRange) * (zRange);
    //updating size informations
    gpuArgs.metaData.metaXLength = xRange;
    gpuArgs.metaData.MetaYLength = yRange;
    gpuArgs.metaData.MetaZLength = zRange;
    gpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    gpuArgs.metaData.maxX = cpuArgs.metaData.minMaxes[1];
    gpuArgs.metaData.minX = cpuArgs.metaData.minMaxes[2];
    gpuArgs.metaData.maxY = cpuArgs.metaData.minMaxes[3];
    gpuArgs.metaData.minY = cpuArgs.metaData.minMaxes[4];
    gpuArgs.metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    gpuArgs.metaData.minZ = cpuArgs.metaData.minMaxes[6];

    //allocating needed memory
    // main array
    unsigned int mainArrXLength = gpuArgs.dbXLength * gpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 2);
    gpuArgs.metaData.mainArrXLength = mainArrXLength;
    gpuArgs.metaData.mainArrSectionLength = mainArrSectionLength;

    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);
    //hipMallocAsync(&mainArr, sizeB, 0);
    size_t sizeorigArr = totalMetaLength * (mainArrXLength * 2) * sizeof(uint32_t);
    hipMallocAsync(&origArr, sizeorigArr, stream);
    size_t sizemetaDataArr = totalMetaLength * (20) * sizeof(uint32_t) + 100;
    hipMallocAsync(&metaDataArr, sizemetaDataArr, stream);
    size_t sizeC = (totalMetaLength * 2 * sizeof(uint32_t) + 50);
    hipMallocAsync(&workQueue, sizeC, stream);
    gpuArgs.origArrsPointer = origArr;
    gpuArgs.metaDataArrPointer = metaDataArr;
    gpuArgs.workQueuePointer = workQueue;
    return gpuArgs.metaData;
};




/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline int allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs, hipStream_t stream) {

    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;
    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;
    //free no longer needed arrays
    hipFreeAsync(gpuArgs.goldArr.arrP, stream);
    hipFreeAsync(gpuArgs.segmArr.arrP, stream);

    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpyAsync(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost, stream);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];
    size = sizeof(uint32_t) * (fpPlusFn + 50);


    hipMallocAsync(&resultListPointerLocal, size, stream);
    hipMallocAsync(&resultListPointerIterNumb, size, stream);
    hipMallocAsync(&resultListPointerMeta, size, stream);

    auto xRange = gpuArgs.metaData.metaXLength;
    auto yRange = gpuArgs.metaData.MetaYLength;
    auto zRange = gpuArgs.metaData.MetaZLength;


    size_t sizeB = gpuArgs.metaData.totalMetaLength * gpuArgs.metaData.mainArrSectionLength * sizeof(uint32_t);

    //printf("size of reduced main arr %d total meta len %d mainArrSectionLen %d  \n", sizeB, metaData.totalMetaLength, metaData.mainArrSectionLength);

    hipMallocAsync(&mainArrAPointer, sizeB, 0);
    hipMemcpyAsync(mainArrAPointer, gpuArgs.origArrsPointer, sizeB, hipMemcpyDeviceToDevice, stream);


    hipMallocAsync(&mainArrBPointer, sizeB, 0);
    hipMemcpyAsync(mainArrBPointer, gpuArgs.origArrsPointer, sizeB, hipMemcpyDeviceToDevice, stream);

    //just in order set it to 0
    uint32_t* resultListPointerMetaCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    hipMemcpyAsync(resultListPointerMeta, resultListPointerMetaCPU, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(resultListPointerIterNumb, resultListPointerMetaCPU, size, hipMemcpyHostToDevice, stream);
    free(resultListPointerMetaCPU);

    gpuArgs.resultListPointerMeta = resultListPointerMeta;
    gpuArgs.resultListPointerLocal = resultListPointerLocal;
    gpuArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    //fbArgs.origArrsPointer = origArrsPointer;
    gpuArgs.mainArrAPointer = mainArrAPointer;
    gpuArgs.mainArrBPointer = mainArrBPointer;


    return fpPlusFn;
};








#pragma once
template <typename T>
inline void  copyResultstoCPU(ForBoolKernelArgs<T>& gpuArgs, ForFullBoolPrepArgs<T>& cpuArgs, hipStream_t stream) {


    ////copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpyAsync(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost, stream);
    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];
    size = sizeof(uint32_t) * (fpPlusFn + 50);

    //uint32_t* resultListPointerMeta = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    //uint32_t* resultListPointerLocal = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    //uint32_t* resultListPointerIterNumb = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));

    cpuArgs.resultListPointerMeta = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));;
    cpuArgs.resultListPointerLocalCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));;
    cpuArgs.resultListPointerIterNumb = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));;

    hipMemcpyAsync(cpuArgs.resultListPointerMeta, gpuArgs.resultListPointerMeta, size, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(cpuArgs.resultListPointerLocalCPU, gpuArgs.resultListPointerLocal, size, hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(cpuArgs.resultListPointerIterNumb, gpuArgs.resultListPointerIterNumb, size, hipMemcpyDeviceToHost, stream);


};
