#include "hip/hip_runtime.h"
/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, 
    uint32_t*& resultListPointerMeta,uint16_t*& resultListPointerLocal,uint16_t*& resultListPointerIterNumb) {
    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];

    size = sizeof(uint32_t)* fpPlusFn + 1;
    hipMallocAsync(&resultListPointerMeta, size, 0);

    size = sizeof(uint16_t) * fpPlusFn + 1;
    hipMallocAsync(&resultListPointerLocal, size, 0);
    hipMallocAsync(&resultListPointerIterNumb, size, 0);

   // metaData.resultList = resultListPointer;


    // hipFreeAsync(gpuArgs.metaData.resultList, 0);

     //hipFree(resultListPointer);


};




#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs
            , uint32_t*& mainArr, uint32_t*& workQueue, unsigned int* minMaxes, MetaDataGPU metaData
) {
    ////reduced arrays


    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy( cpuArgs.metaData.minMaxes, minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2]+1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4]+1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6]+1;
    unsigned int totalMetaLength = xRange* yRange* zRange;

 
    //updating size informations
    metaData.metaXLength = xRange;
    metaData.MetaYLength = yRange;
    metaData.MetaZLength = zRange;
    metaData.totalMetaLength = totalMetaLength;

    cpuArgs.metaData.metaXLength = xRange;
    cpuArgs.metaData.MetaYLength = yRange;
    cpuArgs.metaData.MetaZLength = zRange;
    cpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    metaData.maxX = cpuArgs.metaData.minMaxes[1];
    metaData.minX = cpuArgs.metaData.minMaxes[2];
    metaData.maxY = cpuArgs.metaData.minMaxes[3];
    metaData.minY = cpuArgs.metaData.minMaxes[4];
    metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    metaData.minZ = cpuArgs.metaData.minMaxes[6];

    //allocating needed memory
    // main array
    unsigned int mainArrXLength = cpuArgs.dbXLength * cpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 6) + 19;
    metaData.mainArrXLength = mainArrXLength;
    metaData.mainArrSectionLength = mainArrSectionLength;
    metaData.metaDataOffset = (mainArrXLength * 6);
    
    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);
    std::cout <<"totalMetaLength  ";
    std::cout << totalMetaLength;
    std::cout << "\n";


    //std::cout << "xRange  ";
    //std::cout << xRange;
    //std::cout << "\n";

    //std::cout << "yRange  ";
    //std::cout << yRange;
    //std::cout << "\n";

    //std::cout << "zRange  ";
    //std::cout << zRange;
    //std::cout << "\n";


    uint32_t* mainArrCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));

    //hipMallocAsync(&mainArr, sizeB, 0);
    hipMallocAsync(&mainArr, sizeB, 0);
    hipMemcpy(mainArr, mainArrCPU, sizeB, hipMemcpyHostToDevice);
    free(mainArrCPU);
    //workqueue

    size_t sizeC = (totalMetaLength * sizeof(uint32_t));
   //hipMallocAsync(&workQueue, size, 0);
   hipMalloc(&workQueue, size);

   return metaData;
};




#pragma once
template <typename ZZR>
inline void printForDebug(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, uint32_t* resultListPointer
    , uint32_t* mainArrPointer, uint32_t* workQueuePointer, MetaDataGPU metaData) {
    // getting arrays allocated on  cpu to be able to print and test them easier
    size_t size = sizeof(uint32_t) * metaData.totalMetaLength * metaData.mainArrSectionLength;
    //size_t size = sizeof(uint32_t) * metaData.totalMetaLength * metaData.mainArrSectionLength;
    uint32_t* mainArrCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(mainArrCPU, mainArr, size, hipMemcpyDeviceToHost);

    uint32_t column = mainArrCPU[33];
    printf("column\n ");
    std::cout<<column;
    //in kernel x 33 y 1 z 71 linearLocal 33 linIdexMeta 0
    //    in kernel x 75 y 20 z 70 linearLocal 267 linIdexMeta 3

    for (int linIdexMeta = 0; linIdexMeta < metaData.totalMetaLength; linIdexMeta++) {
        uint8_t xMeta = linIdexMeta % metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (metaData.metaXLength * metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength));

        for (int threadIdxX = 0; threadIdxX < 32; threadIdxX++) {
         
                    for (int threadIdxY = 0; threadIdxY < 18; threadIdxY++) {

                        uint8_t xLoc = threadIdxX;
                        uint16_t x = (xMeta + metaData.minX) * gpuArgs.dbXLength + xLoc;//absolute position
                            uint8_t yLoc = threadIdxY;
                                uint16_t  y = (yMeta + metaData.minY) * gpuArgs.dbYLength + yLoc;//absolute position

                                uint32_t columnGold = mainArrCPU[linIdexMeta * metaData.mainArrSectionLength + gpuArgs.dbXLength * threadIdxY + threadIdxX];
                                if(columnGold >0){
                                    printf("found set at x %d y%d columnGold %d \n", x, y,  columnGold);
                                        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                                            uint16_t z = (zMeta + metaData.minZ) * gpuArgs.dbZLength + bitPos;//absolute position
                                            //if any bit here is set it means it should be added to result list 
                                            if (isBitAtCPU(columnGold, bitPos)) {
                                               // printf("found set at x %d y%d z %d  \n",x,y,z);

                                            
                                        }
                                    }

                                }


            }
        }
    }

        
        
        
        
        
        
        //gold pass
        
        //segm pass
       // mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.mainArrXLength]



    }



