#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include "Structs.cu"

#pragma once
template <typename EEY>
array3dWithDimsCPU<EEY>  get3dArrCPU(EEY* arrP, int Nx, int Ny, int Nz) {
    array3dWithDimsCPU<EEY> res;
    res.Nx = Nx;
    res.Ny = Ny;
    res.Nz = Nz;
    res.arrP = arrP;

    return res;
}

template <typename T >
array3dWithDimsGPU<T> allocateMainArray(T*& gpuArrPointer, T*& cpuArrPointer, const int WIDTH, const int HEIGHT, const int DEPTH) {
    size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    array3dWithDimsGPU<T> res;

    hipMallocAsync(&gpuArrPointer, sizeMainArr, 0);
    hipMemcpyAsync(gpuArrPointer, cpuArrPointer, sizeMainArr, hipMemcpyHostToDevice, 0);
    res.arrP = gpuArrPointer;
    res.Nx = WIDTH;
    res.Ny = HEIGHT;
    res.Nz = DEPTH;
    return res;
}




/*
given appropriate hipPitchedPtr and ForFullBoolPrepArgs will return ForBoolKernelArgs
*/
#pragma once
template <typename TCC>
inline ForBoolKernelArgs<TCC> getArgsForKernel(ForFullBoolPrepArgs<TCC>& mainFunArgs
    , int& warpsNumbForMainPass, int& blockForMainPass
    , const int xLen, const int yLen, const int zLen
) {

    //main arrays allocations
    TCC* goldArrPointer;
    TCC* segmArrPointer;

    //size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    size_t sizeMainArr = (sizeof(TCC) * xLen * yLen * zLen);


    array3dWithDimsGPU<TCC> goldArr = allocateMainArray(goldArrPointer, mainFunArgs.goldArr.arrP, xLen, yLen, zLen);
    array3dWithDimsGPU<TCC> segmArr = allocateMainArray(segmArrPointer, mainFunArgs.segmArr.arrP, xLen, yLen, zLen);


    unsigned int* minMaxes;
    size_t sizeminMaxes = sizeof(unsigned int) * 20;
    hipMallocAsync(&minMaxes, sizeminMaxes, 0);

    ForBoolKernelArgs<TCC> res;
    res.metaData = allocateMetaDataOnGPU(mainFunArgs.metaData, minMaxes);

    res.metaData.minMaxes = minMaxes;
    res.minMaxes = minMaxes;




    res.numberToLookFor = mainFunArgs.numberToLookFor;
    res.dbXLength = 32;
    res.dbYLength = warpsNumbForMainPass;
    res.dbZLength = 32;

    //printf("in setting bool args ylen %d dbYlen %d calculated meta %d  \n ", yLen, res.dbYLength, int(ceil(yLen / res.dbYLength)));
    res.metaData.metaXLength = int(ceil(xLen / res.dbXLength));
    res.metaData.MetaYLength = int(ceil(yLen / res.dbYLength));;
    res.metaData.MetaZLength = int(ceil(zLen / res.dbZLength));;
    res.metaData.minX = 0;
    res.metaData.minY = 0;
    res.metaData.minZ = 0;
    res.metaData.maxX = res.metaData.metaXLength;
    res.metaData.maxY = res.metaData.MetaYLength;
    res.metaData.maxZ = res.metaData.MetaZLength;


    res.metaData.totalMetaLength = res.metaData.metaXLength * res.metaData.MetaYLength * res.metaData.MetaZLength;


    res.goldArr = goldArr;
    res.segmArr = segmArr;


    return res;
}







#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs) {
    ////reduced arrays


    uint32_t* origArr;

    uint32_t* metaDataArr;

    uint32_t* workQueue;




    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2] + 1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4] + 1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6] + 1;
    unsigned int totalMetaLength = (xRange) * (yRange) * (zRange);

    //updating size informations
    gpuArgs.metaData.metaXLength = xRange;
    gpuArgs.metaData.MetaYLength = yRange;
    gpuArgs.metaData.MetaZLength = zRange;
    gpuArgs.metaData.totalMetaLength = totalMetaLength;



    //saving min maxes
    gpuArgs.metaData.maxX = cpuArgs.metaData.minMaxes[1];
    gpuArgs.metaData.minX = cpuArgs.metaData.minMaxes[2];
    gpuArgs.metaData.maxY = cpuArgs.metaData.minMaxes[3];
    gpuArgs.metaData.minY = cpuArgs.metaData.minMaxes[4];
    gpuArgs.metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    gpuArgs.metaData.minZ = cpuArgs.metaData.minMaxes[6];



    //allocating needed memory
    // main array
    unsigned int mainArrXLength = gpuArgs.dbXLength * gpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 2);
    gpuArgs.metaData.mainArrXLength = mainArrXLength;
    gpuArgs.metaData.mainArrSectionLength = mainArrSectionLength;

    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);


    //hipMallocAsync(&mainArr, sizeB, 0);
    size_t sizeorigArr = totalMetaLength * (mainArrXLength * 2) * sizeof(uint32_t);
    hipMallocAsync(&origArr, sizeorigArr, 0);
    size_t sizemetaDataArr = totalMetaLength * (20) * sizeof(uint32_t) + 100;
    hipMallocAsync(&metaDataArr, sizemetaDataArr, 0);


    size_t sizeC = (totalMetaLength * 2 * sizeof(uint32_t) + 50);
    hipMallocAsync(&workQueue, sizeC, 0);


    gpuArgs.origArrsPointer = origArr;
    gpuArgs.metaDataArrPointer = metaDataArr;
    gpuArgs.workQueuePointer = workQueue;


    return gpuArgs.metaData;
};




/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline int allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs) {


    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;

    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;

    //free no longer needed arrays
    hipFreeAsync(gpuArgs.goldArr.arrP, 0);
    hipFreeAsync(gpuArgs.segmArr.arrP, 0);

    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];


    size = sizeof(uint32_t) * (fpPlusFn + 50);


    hipMallocAsync(&resultListPointerLocal, size, 0);
    hipMallocAsync(&resultListPointerIterNumb, size, 0);
    hipMallocAsync(&resultListPointerMeta, size, 0);


    auto xRange = gpuArgs.metaData.metaXLength;
    auto yRange = gpuArgs.metaData.MetaYLength;
    auto zRange = gpuArgs.metaData.MetaZLength;




    size_t sizeB = gpuArgs.metaData.totalMetaLength * gpuArgs.metaData.mainArrSectionLength * sizeof(uint32_t);

    //printf("size of reduced main arr %d total meta len %d mainArrSectionLen %d  \n", sizeB, metaData.totalMetaLength, metaData.mainArrSectionLength);

    hipMallocAsync(&mainArrAPointer, sizeB, 0);
    hipMemcpyAsync(mainArrAPointer, gpuArgs.origArrsPointer, sizeB, hipMemcpyDeviceToDevice, 0);


    hipMallocAsync(&mainArrBPointer, sizeB, 0);
    hipMemcpyAsync(mainArrBPointer, gpuArgs.origArrsPointer, sizeB, hipMemcpyDeviceToDevice, 0);

    //just in order set it to 0
    uint32_t* resultListPointerMetaCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    hipMemcpyAsync(resultListPointerMeta, resultListPointerMetaCPU, size, hipMemcpyHostToDevice, 0);
    free(resultListPointerMetaCPU);

    gpuArgs.resultListPointerMeta = resultListPointerMeta;
    gpuArgs.resultListPointerLocal = resultListPointerLocal;
    gpuArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    //fbArgs.origArrsPointer = origArrsPointer;
    gpuArgs.mainArrAPointer = mainArrAPointer;
    gpuArgs.mainArrBPointer = mainArrBPointer;


    return fpPlusFn;
};































////////////////// with pipeline ofr barrier

/*
initial cleaning  and initializations of dilatation kernel

*/
#pragma once
inline __device__  void dilBlockInitialClean(thread_block_tile<32>& tile,
    const  bool isPaddingPass, int(&iterationNumb)[1],
    unsigned int(&localWorkQueueCounter)[1], unsigned int(&blockFpConter)[1],
    unsigned int(&blockFnConter)[1], unsigned int(&localFpConter)[1],
    unsigned int(&localFnConter)[1], bool(&isBlockFull)[2],
    unsigned int(&fpFnLocCounter)[1],
    unsigned int(&localTotalLenthOfWorkQueue)[1], unsigned int(&globalWorkQueueOffset)[1]
    , unsigned int(&worQueueStep)[1], unsigned int*& minMaxes, unsigned int(&localMinMaxes)[5], uint32_t(&lastI)[1])
{

    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] = true;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 1) {
        isBlockFull[1] = true;
    };

    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        lastI[0] = UINT32_MAX;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }
}



/*
load work que from global memory
*/
#pragma once
inline __device__  void loadWorkQueue(thread_block& cta, uint32_t(&mainShmem)[lengthOfMainShmem], uint32_t*& workQueue
    , bool(&isGoldForLocQueue)[localWorkQueLength], uint32_t& bigloop, unsigned int(&worQueueStep)[1]) {

    //to do change into barrier

    //cuda::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]), pipeline);

    for (uint16_t ii = cta.thread_rank(); ii < worQueueStep[0]; ii += cta.size()) {
        mainShmem[startOfLocalWorkQ + ii] = workQueue[bigloop + ii];
        isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
        mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];

    }
}


/*
loads metadata of given block to meta data
*/
#pragma once
inline __device__  void loadMetaDataToShmem(thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, const uint8_t toAdd, uint32_t& ii) {

    //cuda::memcpy_async(cta, (&localBlockMetaData[(ii&1)*20]),
    //    (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd+ii])
    //        * metaData.metaDataSectionLength])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

    cuda::memcpy_async(cta, (&localBlockMetaData[((ii + 1) & 1) * 20]),
        (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd + ii])
            * metaData.metaDataSectionLength])
        , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


}





////////////////////MAin
/*
loading data about this block to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    //auto inMainLineMeta = mainShmem[startOfLocalWorkQ + i] ;
    //auto inMainFullIndex = inMainLineMeta * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i]);
    //printf("inMain load full index %d \n ", inMainFullIndex);

    //cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[inMainFullIndex],
    //    cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    //pipeline.producer_commit();


    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block
*/
#pragma once
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isBlockFull)[2]) {


    pipeline.consumer_wait();

    if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) < 32) {
        isBlockFull[i & 1] = false;
    }

    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
    //marking weather block is already full and no more dilatations are possible 


    pipeline.consumer_release();


}

////////////////TOP
/*
loading data about block above to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem, i);

    pipeline.consumer_release();

}

/////BOTTOM
#pragma once
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14]
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}
#pragma once
template <typename TXPI>
inline __device__  void processBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
        , 0, 31
        , begSecRegShmem, i);

    pipeline.consumer_release();

}






///////////// right
#pragma once
template <typename TXPI>
inline __device__  void loadRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.x == (fbArgs.dbXLength - 1)),
        3, (1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 0
        , 16, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);

    pipeline.consumer_release();
}



///////////// left
#pragma once
template <typename TXPI>
inline __device__  void loadLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (mainShmem[startOfLocalWorkQ + i] > 0) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.x == 0),
        2, (-1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 31
        , 15, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);

    pipeline.consumer_release();
}

///////////// anterior
#pragma once
template <typename TXPI>
inline __device__  void loadAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {

        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.y == (fbArgs.dbYLength - 1)), 4
        , (0), (1), mainShmem, isAnythingInPadding
        , 0, threadIdx.x
        , 17, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);
    pipeline.consumer_release();
}

///////////// posterior
#pragma once
template <typename TXPI>
inline __device__  void loadPosterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {


        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}





//////////// last load 

/*
load reference if needed or data for next iteration if there is such
*/
#pragma once
template <typename TXPI>
inline __device__  void lastLoad(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , uint32_t*& origArrs, unsigned int(&worQueueStep)[1]) {

    pipeline.producer_acquire();

    //if block should be validated we load data for validation
    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);

    }
    else {//if we are not validating we immidiately start loading data for next loop
        if (i + 1 < worQueueStep[0]) {
            cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1 + i])
                    * metaData.metaDataSectionLength])
                , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


        }
    }


    pipeline.producer_commit();
}
#pragma once
template <typename TXPI>
inline __device__  void processPosteriorAndSaveResShmem(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta
    , uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6],
    bool(&isBlockFull)[2]) {

    pipeline.consumer_wait();
    //dilatate posterior 
    dilatateHelperForTransverse(fbArgs, (threadIdx.y == 0), 5
        , (0), (-1), mainShmem, isAnythingInPadding
        , fbArgs.dbYLength - 1, threadIdx.x // we add offset depending on y dimension
        , 18, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);
    //now all data should be properly dilatated we save it to global memory
    //try save target reduced via mempcy async ...

    getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
        + threadIdx.x + threadIdx.y * 32]
        = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];



    pipeline.consumer_release();
}



//////////// validation
#pragma once
template <typename TXPI>
inline __device__  void validate(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , bool(&isBlockFull)[2]
    , unsigned int(&localFpConter)[1], unsigned int(&localFnConter)[1]
    , uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb

) {

    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
        > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
            //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 
            //    ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) 
            //        & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



            //we now look for bits prasent in both reference arrays and current one
           // mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

            // now we look through bits and when some is set we call it a result 
#pragma unroll
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                ) {

                //just re
                mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = 0;
                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (isGoldForLocQueue[i]) {
                    mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3]);
                }
                else {
                    mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4]);
                    //    printf("local fn counter add \n");

                };
                //   add results to global memory    
                //we add one gere jjust to distinguish it from empty result
                resultListPointerMeta[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                resultListPointerLocal[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                resultListPointerIterNumb[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(iterationNumb[0]);



            }

        };

    }
}


