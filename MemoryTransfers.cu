#include "hip/hip_runtime.h"
#include ""
#include <iostream>
//#include <hip/hip_runtime_api.h>
#include <cmath>
#include "Structs.cu"
#include <cstdint>



#pragma once
template <typename TADD>
inline void copyDeviceToHost3d(array3dWithDimsGPU arrGPU, array3dWithDimsCPU<TADD> arrCPU) {
    hipMemcpy3DParms cpyB = { 0 };
    cpyB.srcPtr = arrGPU.arrPStr;
    cpyB.dstPtr = make_hipPitchedPtr(arrCPU.arrP[0][0], arrCPU.Nx * sizeof(TADD), arrCPU.Nx, arrCPU.Ny);
    cpyB.extent = make_hipExtent(arrCPU.Nx * sizeof(TADD), arrCPU.Ny, arrCPU.Nz);
    cpyB.kind = hipMemcpyDeviceToHost;
    hipMemcpy3DAsync(&cpyB);
};

//
//#pragma once
//template <typename TADY>
//inline void copyDeviceToDevice(array3dWithDimsGPU arrGPUSource, array3dWithDimsGPU arrGPUTarget) {
//    hipMemcpy3DParms cpyB = { 0 };
//    cpyB.srcPtr = arrGPUSource.arrPStr;
//    cpyB.dstPtr = arrGPUTarget.arrPStr;
//    cpyB.extent = make_hipExtent(arrGPUSource.Nx * sizeof(TADY), arrGPUSource.Ny, arrGPUSource.Nz);
//    cpyB.kind = hipMemcpyDeviceToDevice;
//    hipMemcpy3DAsync(&cpyB);
//};

#pragma once
template <typename TAL>
inline void copyHostToDevice(array3dWithDimsGPU arrGPU, array3dWithDimsCPU<TAL> arrCPU) {
    hipMemcpy3DParms cpy = { 0 };
    cpy.srcPtr = make_hipPitchedPtr(arrCPU.arrP[0][0], arrCPU.Nx * sizeof(TAL), arrCPU.Nx, arrCPU.Ny);
    cpy.dstPtr = arrGPU.arrPStr;
    cpy.extent = make_hipExtent(arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz);
    cpy.kind = hipMemcpyHostToDevice;

    hipMemcpy3DAsync(&cpy);
};


#pragma once
template <typename TAL>
inline array3dWithDimsGPU allocate3dInGPU(array3dWithDimsCPU<TAL> arrCPU) {
    array3dWithDimsGPU res;
    struct hipPitchedPtr resStrPointer;
    hipMalloc3D(&resStrPointer, make_hipExtent(arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz));
    //hipMalloc3D(&resStrPointer, make_hipExtent(8 * 4, 9, 10));
    res.arrPStr = resStrPointer;
    //!!!!!!!!!!!!!!! intentionally swithing x and z dimensions to make iterations possible ...
    res.Nz = arrCPU.Nx;
    res.Ny = arrCPU.Ny;
    res.Nx = arrCPU.Nz;


    copyHostToDevice(res, arrCPU);
    //hipMemcpy3DParms cpy = { 0 };
    //cpy.srcPtr = make_hipPitchedPtr(arrCPU.arrP[0][0], arrCPU.Nx * sizeof(TAL), arrCPU.Nx, arrCPU.Ny);
    //cpy.dstPtr = res.arrPStr;
    //cpy.extent = make_hipExtent(arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz);
    //cpy.kind = hipMemcpyHostToDevice;

    //hipMemcpy3DAsync(&cpy);


    //array3dWithDimsGPU res;
    //struct hipPitchedPtr resStrPointer;
    //hipMalloc3D(&resStrPointer, make_hipExtent(arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz));
    ////hipMalloc3D(&resStrPointer, make_hipExtent(8 * 4, 9, 10));
    //res.arrPStr = resStrPointer;
    //res.Nx = arrCPU.Nx;
    //res.Ny = arrCPU.Ny;
    //res.Nz = arrCPU.Nz;  
    //
    //hipMemcpy3DParms cpy = { 0 };
    //cpy.srcPtr = make_hipPitchedPtr(arrCPU.arrP[0][0], arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz);
    //cpy.dstPtr = resStrPointer;
    //cpy.extent = make_hipExtent(arrCPU.Nx * sizeof(TAL), arrCPU.Ny, arrCPU.Nz);
    //cpy.kind = hipMemcpyHostToDevice;
    //hipMemcpy3D(&cpy);


    return res;
};



template <typename TALGG>
inline hipPitchedPtr allocate3dInGPUSimple(TALGG*** cpuArr, int Nx, int Ny, int Nz) {
    struct hipPitchedPtr res;
    hipMalloc3D(&res, make_hipExtent(Nx * sizeof(TALGG), Ny, Nz));
    copyDeviceToHost3dSimple(cpuArr, res, Nx, Ny, Nz);
    return res;
};

template <typename TADHDF>
inline void copyDeviceToHost3dSimple(TADHDF*** hostTensor, hipPitchedPtr deviceTarget, int Nx, int Ny, int Nz) {
    hipMemcpy3DParms cpy = { 0 };
    cpy.srcPtr = make_hipPitchedPtr(hostTensor[0][0], Nx * sizeof(TADHDF), Nx, Ny);
    cpy.dstPtr = deviceTarget;
    cpy.extent = make_hipExtent(Nx * sizeof(TADHDF), Ny, Nz);
    cpy.kind = hipMemcpyHostToDevice;

    hipMemcpy3DAsync(&cpy);
};


#pragma once
template <typename ZZ>
inline void setArrCPU(array3dWithDimsCPU<ZZ> arrCPU, int x, int y, int z, ZZ value, bool toPrint = true) {
    if (toPrint) {
      //  printf(" set imn meta gold %d  %d  %d \n", x, y, z);
    }
    arrCPU.arrP[z][y][x] = value;
};




