#include "hip/hip_runtime.h"
#include "Structs.cu"
#include "hip/hip_runtime.h"
#include <iostream>     // std::cout
#pragma once
void printFromReduced(ForBoolKernelArgs<int> fbArgs, uint32_t* arrsCPU) {
	for (uint32_t linIdexMeta = 0; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
		//we get from linear index  the coordinates of the metadata block of intrest
		uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
		uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
		uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));

		for (int locPos = 0; locPos < 32 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						printf("point gold set at x %d y %d z %d  \n"
							, locPos % 32 + xMeta * fbArgs.dbXLength
							, int(floor((float)(locPos / 32)) + yMeta * fbArgs.dbYLength)
							, bitPos + zMeta * fbArgs.dbZLength
						);
					}
				}
			}
		}


		for (int locPos = 32 * fbArgs.dbYLength; locPos < 32 * 2 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						int locPosB = locPos - 32 * fbArgs.dbYLength;
						printf("point segm  set at x %d y %d z %d  \n"
							, locPosB % 32 + xMeta * fbArgs.dbXLength
							, int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength)
							, bitPos + zMeta * fbArgs.dbZLength
						);
					}
				}
			}
		}
	}
}















#pragma once
inline forTestPointStruct getTestPoint(int x, int y, int z,
	bool isGold, int xMeta, int yMeta, int zMeta
	, int dbXLength, int dbYLength, int  dbZLength
	, int& pointsNumberRef
	, bool isGoldAndSegm = false
	, bool shouldBeInResAfterOneDil = false
	, bool shouldBeInResAfterTwoDil = false
) {
	pointsNumberRef += 1;
	forTestPointStruct res;

	res.x = xMeta * dbXLength + x;
	res.y = yMeta * dbYLength + y;
	res.z = zMeta * dbZLength + z;
	res.isGoldAndSegm = isGoldAndSegm;
	res.isGold = isGold;

	res.xMeta = xMeta;
	res.yMeta = yMeta;
	res.zMeta = zMeta;


	res.shouldBeInResAfterOneDil = shouldBeInResAfterOneDil;
	res.shouldBeInResAfterTwoDil = shouldBeInResAfterTwoDil;

	return res;

}




#pragma once
inline forTestMetaDataStruct getMetdataTestStruct(
	int& metasNumberRef,
	int xMeta,
	int yMeta,
	int zMeta,

	int fpCount = 0,
	int fnCount = 0,

	bool isToBeActiveAtStart = true,
	bool isToBeActiveAfterOneIter = true,
	bool isToBeActiveAfterTwoIter = true,

	bool isToBeFullAfterOneIter = false,
	bool isToBeFullAfterTwoIter = false,

	bool isToBeValidatedFpAfterOneIter = false,
	bool isToBeValidatedFpAfterTwoIter = false,

	bool isToBeValidatedFnAfterOneIter = false,
	bool isToBeValidatedFnAfterTwoIter = false,

	int fpConterAfterOneDil = 0,
	int fpConterAfterTwoDil = 0,

	int fnConterAfterOneDil = 0,
	int fnConterAfterTwoDil = 0) {


	forTestMetaDataStruct res;
	metasNumberRef += 1;
	res.xMeta = xMeta;
	res.yMeta = yMeta;
	res.zMeta = zMeta;


	res.isToBeActiveAtStart = (fpCount+ fnCount)>0;
	res.isToBeActiveAfterOneIter = isToBeActiveAfterOneIter;
	res.isToBeActiveAfterTwoIter = isToBeActiveAfterTwoIter;

	res.isToBeFullAfterOneIter = isToBeFullAfterOneIter;
	res.isToBeFullAfterTwoIter = isToBeFullAfterTwoIter;

	res.fpCount = fpCount;
	res.fnCount = fnCount;

	res.requiredspaceInFpResultList = fpCount;
	res.requiredspaceInFnResultList = fnCount;

	res.isToBeValidatedFpAfterOneIter = fpCount > 0;
	res.isToBeValidatedFpAfterTwoIter = fpCount > 0;

	res.isToBeValidatedFnAfterOneIter = fnCount > 0;
	res.isToBeValidatedFnAfterTwoIter = fnCount > 0;


	res.fpConterAfterOneDil = fpConterAfterOneDil;
	res.fpConterAfterTwoDil = fpConterAfterTwoDil;

	res.fnConterAfterOneDil = fnConterAfterOneDil;
	res.fnConterAfterTwoDil = fnConterAfterTwoDil;
	return res;
}