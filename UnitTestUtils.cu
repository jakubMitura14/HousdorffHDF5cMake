
#include <hip/hip_runtime.h>
//#include "Structs.cu"
//#include "cuda_runtime.h"
//#include <iostream>     // std::cout
//
//
//
//#pragma once
//inline forTestPointStruct getTestPoint(int x, int y, int z,
//	bool isGold, int xMeta, int yMeta, int zMeta
//	, int dbXLength, int dbYLength, int  dbZLength
//	, int& pointsNumberRef
//	, bool isGoldAndSegm = false
//	, bool shouldBeInResAfterOneDil = false
//	, bool shouldBeInResAfterTwoDil = false
//) {
//	pointsNumberRef += 1;
//	forTestPointStruct res;
//
//	res.x = xMeta * dbXLength + x;
//	res.y = yMeta * dbYLength + y;
//	res.z = zMeta * dbZLength + z;
//	res.isGoldAndSegm = isGoldAndSegm;
//	res.isGold = isGold;
//
//	res.xMeta = xMeta;
//	res.yMeta = yMeta;
//	res.zMeta = zMeta;
//
//
//	res.shouldBeInResAfterOneDil = shouldBeInResAfterOneDil;
//	res.shouldBeInResAfterTwoDil = shouldBeInResAfterTwoDil;
//
//	return res;
//
//}
//
//
//
//
//#pragma once
//inline forTestMetaDataStruct getMetdataTestStruct(
//	int& metasNumberRef,
//	int xMeta,
//	int yMeta,
//	int zMeta,
//
//	int fpCount = 0,
//	int fnCount = 0,
//
//	bool isToBeActiveAtStart = true,
//	bool isToBeActiveAfterOneIter = true,
//	bool isToBeActiveAfterTwoIter = true,
//
//	bool isToBeFullAfterOneIter = false,
//	bool isToBeFullAfterTwoIter = false,
//
//	bool isToBeValidatedFpAfterOneIter = false,
//	bool isToBeValidatedFpAfterTwoIter = false,
//
//	bool isToBeValidatedFnAfterOneIter = false,
//	bool isToBeValidatedFnAfterTwoIter = false,
//
//	int fpConterAfterOneDil = 0,
//	int fpConterAfterTwoDil = 0,
//
//	int fnConterAfterOneDil = 0,
//	int fnConterAfterTwoDil = 0) {
//
//
//	forTestMetaDataStruct res;
//	metasNumberRef += 1;
//	res.xMeta = xMeta;
//	res.yMeta = yMeta;
//	res.zMeta = zMeta;
//
//
//	res.isToBeActiveAtStart = (fpCount+ fnCount)>0;
//	res.isToBeActiveAfterOneIter = isToBeActiveAfterOneIter;
//	res.isToBeActiveAfterTwoIter = isToBeActiveAfterTwoIter;
//
//	res.isToBeFullAfterOneIter = isToBeFullAfterOneIter;
//	res.isToBeFullAfterTwoIter = isToBeFullAfterTwoIter;
//
//	res.fpCount = fpCount;
//	res.fnCount = fnCount;
//
//	res.requiredspaceInFpResultList = fpCount;
//	res.requiredspaceInFnResultList = fnCount;
//
//	res.isToBeValidatedFpAfterOneIter = fpCount > 0;
//	res.isToBeValidatedFpAfterTwoIter = fpCount > 0;
//
//	res.isToBeValidatedFnAfterOneIter = fnCount > 0;
//	res.isToBeValidatedFnAfterTwoIter = fnCount > 0;
//
//
//	res.fpConterAfterOneDil = fpConterAfterOneDil;
//	res.fpConterAfterTwoDil = fpConterAfterTwoDil;
//
//	res.fnConterAfterOneDil = fnConterAfterOneDil;
//	res.fnConterAfterTwoDil = fnConterAfterTwoDil;
//	return res;
//}
//
//
//
//
//
//
//
//
//////////// for boolkernel tests
//
////1) is reduced arrs are they should be - all of them - are there entries in correct spots
////2) do number of fp and fn fo the begining works
////3) do we have min and maxes aset correctly
//#pragma once	
//inline void forBoolKernelTestUnitTests(ForFullBoolPrepArgs<int> fbArgs, forTestPointStruct allPointsA[], forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber
//	,int dbXLength, int dbYLength, int dbZLength) {
//	
//
//	//1) is reduced arrs are they should be - all of them - are there entries in correct spots
//	for (int i = 0; i < pointsNumber; i++) {
//		bool isInReducedRef=false;
//		bool isInReduced = false;
//		bool isInReducedPrev = false;
//
//		forTestPointStruct currPoint = allPointsA[i];
//		int bitPos = currPoint.z - currPoint.zMeta * dbZLength;
//
//		//printf("point %d %d %d \n  ", currPoint.x, currPoint.y, currPoint.z);
//
//		if (currPoint.isGold) {  
//			isInReducedRef = (fbArgs.reducedGoldRef.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));
//			isInReduced = (fbArgs.reducedGold.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));
//			isInReducedPrev = (fbArgs.reducedGoldPrev.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));		
//		}
//		else { 
//			isInReducedRef = (fbArgs.reducedSegmRef.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));
//			isInReduced = (fbArgs.reducedSegm.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));
//			isInReducedPrev = (fbArgs.reducedSegmPrev.arrP[currPoint.zMeta][currPoint.y][currPoint.x] & (1 << (bitPos)));
//		
//		}
//
//		if (!isInReducedRef) {
//			printf("nnnnnnnnnnn  not found point %d %d %d in referenca reduced \n ", currPoint.x, currPoint.y, currPoint.z);
//		}
//		if (!isInReduced) {
//			printf("nnnnnnnnnnn  not found point %d %d %d in  reduced \n ", currPoint.x, currPoint.y, currPoint.z);
//
//		}
//		if (!isInReducedPrev) {
//			printf("nnnnnnnnnnn  not found point %d %d %d in referenca prev \n ", currPoint.x, currPoint.y, currPoint.z);
//
//		}
//
//
//
//		//else {
//		//	printf("ffffffffffffft found point %d %d %d in referenca reduced \n ", currPoint.x, currPoint.y, currPoint.z);
//
//		//}
//
////		
//	}
//	printf("metasNumber %d \n ", metasNumber);
//	//2) do number of fp and fn fo the begining works
//	for (int i = 0; i < metasNumber; i++) {
//		forTestMetaDataStruct locMeta = allMetas[i];
//
//		//printf("block  %d %d %d fp count %d fncount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, locMeta.fpCount, locMeta.fnCount);
//
//
//		bool isFpOk = fbArgs.metaData.fpCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta] == locMeta.fpCount;
//		bool isFnOk = fbArgs.metaData.fnCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta] == locMeta.fnCount;
//
//		if (!isFpOk) {
//			printf("nnnnnnnnnnn  not correct fp number in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta
//				, fbArgs.metaData.fpCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta], locMeta.fpCount);
//		}
//		else {
//		//	printf("tttttt  correct fp number in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta
//			//	, fbArgs.metaData.fpCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta], locMeta.fpCount);
//		}
//		if (!isFnOk) {
//			printf("nnnnnnnnnnn  not correct fn number in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta
//				, fbArgs.metaData.fnCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta], locMeta.fnCount);
//		}
//		else {
//		//	printf("tttttt  correct fn number in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta
//			//	, fbArgs.metaData.fnCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta], locMeta.fnCount);
//		}
//
//
//	}
//
//
//
////	for (i = 0; i < mainXLength; i++) {
////	for (j = 0; j < mainYLength; j++) {
////		for (k = 0; k < MetaZLength; k++) {
////			 k = 5;
////		if (reducedSegm[k][j][i] > 0) {
////			for (int tt = 0; tt < 32; tt++) {
////				if ((reducedSegm[k][j][i] & (1 << (tt)))) {
////					printf("found in reduced segm  [%d][%d][%d]\n", i, j, k * 32 + tt);
////				}
////			}
////		}
////		}
////	}
////}
//
//
//}
//
//
//
//
//
///// first meta pass
////1) do all blocks have enough space defined by offsets	
////2) doues all blocks marked as active are in the work queue
////3) are block that supposed to be actie are 
//#pragma once
//inline void firstMetaPassKernelTestUnitTests(ForFullBoolPrepArgs<int> fbArgs, forTestPointStruct allPointsA[], forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber
//	, int dbXLength, int dbYLength, int dbZLength) {
//
//	int totalFp = 0;
//	int totalFn = 0;
//
//	bool isSetArr[100000];
//	for (int i = 0; i < metasNumber; i++) {
//		isSetArr[i] = false;
//	}
//
//
//	for (int i = 0; i < metasNumber; i++) {
//		forTestMetaDataStruct locMeta = allMetas[i];
//		int fpOffset = fbArgs.metaData.fpOffset.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fnOffset = fbArgs.metaData.fnOffset.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//
//		isSetArr[fpOffset]=true ;
//		isSetArr[fnOffset] =true;
//
//
//		
//	};
//
//	for (int i = 0; i < metasNumber; i++) {
//
//		//printf("block  %d %d %d fp count %d fncount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, locMeta.fpCount, locMeta.fnCount);
//		//1) do all blocks have enough space defined by offsets	
//		forTestMetaDataStruct locMeta = allMetas[i];
//
//
//		int fpOffset = fbArgs.metaData.fpOffset.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fnOffset = fbArgs.metaData.fnOffset.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fpCount = fbArgs.metaData.fpCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fnCount = fbArgs.metaData.fnCount.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//
//
//		totalFp += fpCount;
//		totalFn+= fnCount;
//
//
//		//printf("block  %d %d %d fp count %d fncount %d   \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, locMeta.fpCount, locMeta.fnCount);
//
//		if (fpOffset > 0) {
//				for (int jj = 1; jj < 100000; jj++) {
//					if (isSetArr[fpOffset + jj]==true) {
//					//	printf("fpOffset %d  jj %d \n ", fpOffset, jj);
//
//						if (jj<fpCount) {
//							printf("nnnnnnnnnnn  not correct fp offset  in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, jj, fpCount);
//
//						}
//						else {
//						//	printf("tttttttt  correct fp offset  in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta,  jj, fpCount);
//
//						}
//						break;
//					}
//				}
//			};
//
//		if (fnOffset > 0) {
//			for (int jj = 1; jj < 100000; jj++) {
//				if (isSetArr[fnOffset + jj] == true) {
//					//	printf("fpOffset %d  jj %d \n ", fpOffset, jj);
//
//					if (jj < fnCount) {
//						printf("nnnnnnnnnnn  not correct fn offset  in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, jj, fnCount);
//
//					}
//					else {
//						//printf("tttttttt  correct fn offset  in block  %d %d %d is %d should be %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, jj, fnCount);
//
//					}
//					break;
//				}
//			}
//		};
//		////checking is block active
//
//		int fpActive = fbArgs.metaData.isActiveGold.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fnActie = fbArgs.metaData.isActiveSegm.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//
//
//		////if (fpActive) {
//		////	if (fnCount > 0) {
//		////		//printf("tttt  fp correct is active as should be   in block  %d %d %d fpActive  %d fpCount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fpActive, fpCount);
//
//		////	}
//		////	else {
//		////		printf("nnnnnnnnnnn fp is not active and should be   in block  %d %d %d fpActive %d fpCount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fpActive, fpCount);
//
//
//		////	};
//		////}
//		////if (fnActie) {
//		////	if (fpCount > 0) {
//		////		//printf("tttt fn correct is active as should be   in block  %d %d %d fnActie %d fnCount%d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fnActie, fnCount);
//
//		////	}
//		////	else {
//		////		printf("nnnnnnnnnnn  fn is not active and should be   in block  %d %d %d fnActie %d  fnCount %d  \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fnActie, fnCount);
//
//
//		////	};
//		////}
//
//		/// checking work queue
//		bool isInWorkQueue=false;
//		int numb = 0;
//		for (int ji = 0; ji < 10000;ji++) {
//		bool boolX=	fbArgs.metaData.workQueue.arrP[0][0][ji] == locMeta.xMeta;
//		bool boolY=	fbArgs.metaData.workQueue.arrP[0][1][ji] == locMeta.yMeta;
//		bool boolZ=	fbArgs.metaData.workQueue.arrP[0][2][ji] == locMeta.zMeta;
//		//bool boolIsGold=	fbArgs.metaData.workQueue.arrP[0][3][ji] == fpActive;
//		isInWorkQueue = (boolX && boolY && boolZ );
//			if (isInWorkQueue) {
//				numb = ji;
//				break;
//			}
//
//		//bool booliG=	fbArgs.metaData.workQueue.arrP[0][3][ji] == locMeta.is;
//		}
//
//		if (fpActive || fnActie){
//			if (isInWorkQueue) {
//		//	printf("tttt  correct is in work queue  in block  %d %d %d isGold %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fbArgs.metaData.workQueue.arrP[0][3][numb]);
//
//			}
//			else {
//				printf("nnnnnnnnnnn   not in work queue and should be  in block  %d %d %d isGold %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fbArgs.metaData.workQueue.arrP[0][3][numb]);
//
//
//			};
//		}
//		
//
//	}
//
//
//	if (fbArgs.metaData.minMaxes.arrP[0][0][7] == totalFp) {
//			printf("tttt  correct number of fp global  \n ");
//	}
//	else {
//		printf("nnnnnnnnnnn   incorrect fp global  is  %d  should be  %d  \n ", (fbArgs.metaData.minMaxes.arrP[0][0][7]), totalFp);
//
//
//	};
//
//	//totalFp += fpCount;
//	//totalFn += fnCount;
//
//
//	//printf("global FP count %d  [%d]\n", minMaxes.arrP[0][0][7], i);
//	//printf("global FN count %d  [%d]\n", minMaxes.arrP[0][0][8], i);
//
//
//}
//
////////////////////////////////main pass single
//
//inline void isDilatatedSingle(int xChange,int yChange, int zChange, int x, int y, int z, uint32_t*** arr,int dbZLength, int zMeta, int maxZmeta, int maxy, int maxx, int xMeta, int yMeta) {
//		int newZmeta = zMeta;
//		int bitPos = z - zMeta * dbZLength;
//		int newBitPos = bitPos + zChange;
//		//correcting for z fringes
//		if (newBitPos== dbZLength) {
//			newZmeta += 1;
//				newBitPos = 0;
//		
//		}
//		if (newBitPos < 0) {
//			newZmeta -= 1;
//			newBitPos = (dbZLength-1) ;
//
//		}
//		int newX = x+ xChange;
//		int newY = y+ yChange;
//		auto str = "";
//		if (xChange == 1) { str =  "look right "; };
//		if (xChange == -1) { str =  "look left "; };
//		if (yChange == 1) { str =  "look anterior "; };
//		if (yChange == -1) { str =  "look posterior "; };
//		if (zChange == 1) { str =  "look down "; };
//		if (zChange == -1) { str =  "look up "; };
//
//
//
//		if (newZmeta>0 && newX>0 && newY>0
//			&& newZmeta < maxZmeta && newX < maxx && newY <maxy) {
//
//			bool newVal = (arr[newZmeta][newY][newX] & (1 << (newBitPos)));
//			if (newVal) {
//				// printf("ttt  found %s dil orig point %d %d %d new point %d %d %d     xMeta %d yMeta %d zMeta %d  \n ", str, x,y,z, newX,newY, newZmeta* dbZLength + newBitPos, xMeta, yMeta, zMeta);
//
//			}
//			else {
//				printf("ffff not found %s  dil orig point %d %d %d new point %d %d %d      xMeta %d yMeta %d zMeta %d  \n ", str, x, y, z, newX, newY, newZmeta * dbZLength + newBitPos, xMeta, yMeta, zMeta);
//
//			}
//
//		}
//
//
//
//	
//}
//
//
//inline void isDilatatedAll(uint32_t*** arr, int x, int y , int z , int dbZLength, int zMeta, int maxZmeta, int maxy, int maxx, uint32_t*** reference, int xMeta, int yMeta) {
//
//
//	//printf("point %d %d %d   in result here %d and in reference %d \n  ", x, y, z, arr[zMeta][y][x], reference[zMeta][y][x]);
//
//	isDilatatedSingle(1, 0, 0, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//	isDilatatedSingle(-1, 0, 0, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//
//	isDilatatedSingle(0, 1, 0, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//	isDilatatedSingle(0, -1, 0, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//
//	isDilatatedSingle(0, 0, 1, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//	isDilatatedSingle(0, 0, -1, x, y, z, arr, dbZLength, zMeta, maxZmeta, maxy, maxx, xMeta, yMeta);
//}
//
//
////1) do we have a correct dilatation for points inside the block
////2) do we have correct dilatations for points on the fringes of the blocks
////3) did metadatablock counters changed as they should
////4) are results that should be added are
////5) does result counter is ok?
////6) are block that should be marked as full are
//  //7) do blocks that should be marked as to be activated are 
//#pragma once
//inline void mainPassKernelTestUnitTests(ForFullBoolPrepArgs<int> fbArgs, forTestPointStruct allPointsA[], forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber
//	, int dbXLength, int dbYLength, int dbZLength, int maxZmeta, int maxy, int maxx) {
//
//	//1) do we have a correct dilatation for points inside the block
//	//2) do we have correct dilatations for points on the fringes of the blocks
//	for (int i = 0; i < pointsNumber; i++) {
//
//		forTestPointStruct currPoint = allPointsA[i];
//		int bitPos = currPoint.z - currPoint.zMeta * dbZLength;
//
//		//printf("point %d %d %d \n  ", currPoint.x, currPoint.y, currPoint.z);
//		
//		if (currPoint.isGold) {
//			isDilatatedAll(fbArgs.reducedGold.arrP, currPoint.x, currPoint.y, currPoint.z, dbZLength, currPoint.zMeta, maxZmeta, maxy, maxx, fbArgs.reducedGoldRef.arrP, currPoint.xMeta, currPoint.yMeta);
//		}
//		else {
//			isDilatatedAll(fbArgs.reducedSegm.arrP, currPoint.x, currPoint.y, currPoint.z, dbZLength, currPoint.zMeta, maxZmeta, maxy, maxx, fbArgs.reducedSegmRef.arrP, currPoint.xMeta, currPoint.yMeta);
//
//		}	}
//
//
//
//
//	////4) are results that should be added are
//	//for (int i = 0; i < pointsNumber; i++) {
//
//	//	forTestPointStruct currPoint = allPointsA[i];
//
//	//	if (currPoint.shouldBeInResAfterOneDil){
//	//		//printf("point %d %d %d   \n  ", currPoint.x, currPoint.y, currPoint.z);
//
//	//		bool isInRes = false;
//	//		int numb = 0;
//	//		for (int ji = 0; ji < 10000; ji++) {
//	//			bool boolX = fbArgs.metaData.resultList.arrP[0][0][ji] == currPoint.x;
//	//			bool boolY = fbArgs.metaData.resultList.arrP[0][1][ji] == currPoint.y;
//	//			bool boolZ = fbArgs.metaData.resultList.arrP[0][2][ji] == currPoint.z;
//	//			/*	bool boolG = fbArgs.metaData.resultList.arrP[0][3][ji] == currPoint.isGold;
//	//			bool boolI = fbArgs.metaData.resultList.arrP[0][4][ji] == 0;*/
//	//			//bool boolIsGold=	fbArgs.metaData.resultList.arrP[0][3][ji] == fpActive;
//	//			isInRes = (boolX && boolY && boolZ);// && boolG && boolI
//	//			if (isInRes) {
//	//				numb = ji;
//	//				break;
//	//			}
//
//	//			//bool booliG=	fbArgs.metaData.resultList.arrP[0][3][ji] == locMeta.is;
//	//		}
//	//		if (isInRes) {
//	//			printf("tttt  found correct result  in point  %d %d %d isGold %d iteration %d \n ", fbArgs.metaData.resultList.arrP[0][0][numb]
//	//				, fbArgs.metaData.resultList.arrP[0][1][numb], fbArgs.metaData.resultList.arrP[0][2][numb], fbArgs.metaData.resultList.arrP[0][3][numb], fbArgs.metaData.resultList.arrP[0][4][numb]);
//
//	//		}
//	//		else {
//	//		/*	printf("fffffffff  not found result  in point  %d %d %d isGold %d iteration %d \n ", fbArgs.metaData.resultList.arrP[0][0][numb]
//	//				, fbArgs.metaData.resultList.arrP[0][1][numb], fbArgs.metaData.resultList.arrP[0][2][numb], fbArgs.metaData.resultList.arrP[0][3][numb], fbArgs.metaData.resultList.arrP[0][4][numb]);*/
//
//
//	//		};
//	//	}
//	//
//
//
//	//}
//
//	//5) does result counter is ok?
////6) are block that should be marked as full are
//  //7) do blocks that should be marked as to be activated are 
//	for (int i = 0; i < metasNumber; i++) {
//
//		//printf("block  %d %d %d fp count %d fncount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, locMeta.fpCount, locMeta.fnCount);
//		forTestMetaDataStruct locMeta = allMetas[i];
//
//		int fpCounter = fbArgs.metaData.fpCounter.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int fnCounter = fbArgs.metaData.fnCounter.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//
//		if (fpCounter == locMeta.fpConterAfterOneDil) {
//			//printf("tttt correct fp counter in block  %d %d %d is %d should be  %d \n ",
//			//	locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fpCounter, locMeta.fpConterAfterOneDil);
//
//		}
//		else {
//			printf("fff incorrect fp counter in block  %d %d %d is %d should be  %d \n ",
//				locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fpCounter, locMeta.fpConterAfterOneDil);
//		};
//
//		if (fnCounter == locMeta.fnConterAfterOneDil) {
//				//printf("tttt correct fn counter in block  %d %d %d is %d should be  %d \n ",
//				//	locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fnCounter, locMeta.fnConterAfterOneDil);
//
//		}
//		else {
//			printf("fff in correct fn counter in block  %d %d %d is %d should be  %d \n ",
//				locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, fnCounter, locMeta.fnConterAfterOneDil);
//		};
//
//	}
//
//
//	/*for (int i = 0; i < metasNumber; i++) {
//
//		printf("block  %d %d %d fp count %d fncount %d \n ", locMeta.xMeta, locMeta.yMeta, locMeta.zMeta, locMeta.fpCount, locMeta.fnCount);
//		forTestMetaDataStruct locMeta = allMetas[i];
//
//		bool isActivatedGold = fbArgs.metaData.isToBeValidatedFp.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//		int isActivatedSegm = fbArgs.metaData.isToBeValidatedFn.arrP[locMeta.zMeta][locMeta.yMeta][locMeta.xMeta];
//	
//		
//		if (locMeta.isToBeValidatedFpAfterOneIter) {
//			if (isActivatedGold) {
//				printf("tttt correct gold is to be validated  as should be in block  %d %d %d  \n ",
//					locMeta.xMeta, locMeta.yMeta, locMeta.zMeta);
//
//			}
//			else {
//				printf("fff incorrect gold  is to be validated   in block  %d %d %d \n ",
//					locMeta.xMeta, locMeta.yMeta, locMeta.zMeta);
//			};
//		};
//		if (isActivatedSegm) {
//			if (locMeta.isToBeValidatedFnAfterOneIter) {
//				printf("tttt segm is to be validated  as should be in block  %d %d %d  \n ",
//					locMeta.xMeta, locMeta.yMeta, locMeta.zMeta);
//
//			}
//			else {
//				printf("fff  segm is not   to be validated  as should be in block  %d %d %d\n ",
//					locMeta.xMeta, locMeta.yMeta, locMeta.zMeta);
//			};
//		};
//
//	}*/
//
//
//	//printf("\n result aaa \n ");
//
//	//for (int ji = 0; ji < 30; ji++) {
//	//	if (fbArgs.metaData.resultList.arrP[0][0][ji]>0) {
//	//		printf("result  in point  %d %d %d isGold %d iteration %d \n ", fbArgs.metaData.resultList.arrP[0][0][ji]
//	//			, fbArgs.metaData.resultList.arrP[0][1][ji], fbArgs.metaData.resultList.arrP[0][2][ji], fbArgs.metaData.resultList.arrP[0][3][ji], fbArgs.metaData.resultList.arrP[0][4][ji]);
//	//	}
//	//}
//
//
//
//}
//
//#pragma once
//inline void checkAfterSecondDil(ForFullBoolPrepArgs<int> fbArgs, forTestPointStruct allPointsA[], forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber
//	, int dbXLength, int dbYLength, int dbZLength) {
//
//
//
//	//4) are results that should be added are
//	for (int i = 0; i < pointsNumber; i++) {
//
//		forTestPointStruct currPoint = allPointsA[i];
//
//		if (currPoint.shouldBeInResAfterTwoDil) {
//			//printf("point %d %d %d   \n  ", currPoint.x, currPoint.y, currPoint.z);
//
//			bool isInRes = false;
//			int numb = 0;
//			for (int ji = 0; ji < 10000; ji++) {
//				bool boolX = fbArgs.metaData.resultList.arrP[0][0][ji] == currPoint.x;
//				bool boolY = fbArgs.metaData.resultList.arrP[0][1][ji] == currPoint.y;
//				bool boolZ = fbArgs.metaData.resultList.arrP[0][2][ji] == currPoint.z;
//				/*	bool boolG = fbArgs.metaData.resultList.arrP[0][3][ji] == currPoint.isGold;
//				bool boolI = fbArgs.metaData.resultList.arrP[0][4][ji] == 0;*/
//				//bool boolIsGold=	fbArgs.metaData.resultList.arrP[0][3][ji] == fpActive;
//				isInRes = (boolX && boolY && boolZ);// && boolG && boolI
//				if (isInRes) {
//					numb = ji;
//					break;
//				}
//
//				//bool booliG=	fbArgs.metaData.resultList.arrP[0][3][ji] == locMeta.is;
//			}
//			if (isInRes) {
//				printf("tttt  found correct result after second dil  in point  %d %d %d isGold %d iteration %d  and in point  x %d y %d z %d \n ", fbArgs.metaData.resultList.arrP[0][0][numb]
//					, fbArgs.metaData.resultList.arrP[0][1][numb], fbArgs.metaData.resultList.arrP[0][2][numb], fbArgs.metaData.resultList.arrP[0][3][numb], fbArgs.metaData.resultList.arrP[0][4][numb]
//				, currPoint.x, currPoint.y, currPoint.z);
//
//			}
//			else {
//				printf("fffffffff  not found result  in point  after second dil   %d %d %d isGold %d iteration %d   and in point  x %d y %d z %d  \n ", fbArgs.metaData.resultList.arrP[0][0][numb]
//					, fbArgs.metaData.resultList.arrP[0][1][numb], fbArgs.metaData.resultList.arrP[0][2][numb], fbArgs.metaData.resultList.arrP[0][3][numb], fbArgs.metaData.resultList.arrP[0][4][numb], currPoint.x, currPoint.y, currPoint.z);
//
//
//			};
//		}
//
//
//
//	}
//
//
//}
//
//
//
//
/////// till the end
//
////1) does final iteration number matches max distance
//
////2) do we have all points that do not match in both arrays in result list and with correct coordinates
//#pragma once
//inline void finalCheckTestUnitTests(ForFullBoolPrepArgs<int> fbArgs, forTestPointStruct allPointsA[], forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber
//	, int dbXLength, int dbYLength, int dbZLength) {
//
//
//	//for (int i = 0; i < pointsNumber; i++) {
//
//	//	forTestPointStruct currPoint = allPointsA[i];
//	//	int bitPos = currPoint.z - currPoint.zMeta * dbZLength;
//
//	//	//printf("point %d %d %d \n  ", currPoint.x, currPoint.y, currPoint.z);
//
//	//	if (currPoint.isGold) {
//	//		isDilatatedAll(fbArgs.reducedGold.arrP, currPoint.x, currPoint.y, currPoint.z, dbZLength, currPoint.zMeta, maxZmeta, maxy, maxx, fbArgs.reducedGoldRef.arrP, currPoint.xMeta, currPoint.yMeta);
//	//	}
//	//	else {
//	//		isDilatatedAll(fbArgs.reducedSegm.arrP, currPoint.x, currPoint.y, currPoint.z, dbZLength, currPoint.zMeta, maxZmeta, maxy, maxx, fbArgs.reducedSegmRef.arrP, currPoint.xMeta, currPoint.yMeta);
//
//	//	}
//	//}
//
//
//
//
//
//}
//
//
//
//
//
//
