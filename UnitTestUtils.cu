#include "hip/hip_runtime.h"
#include "Structs.cu"
#include "hip/hip_runtime.h"
#include <iostream>     // std::cout
#pragma once
void printFromReduced(ForBoolKernelArgs<int> fbArgs, uint32_t* arrsCPU) {
	for (uint32_t linIdexMeta = 0; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
		//we get from linear index  the coordinates of the metadata block of intrest
		uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
		uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
		uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));

		for (int locPos = 0; locPos < 32 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						printf("point gold set at x %d y %d z %d  \n"
							, locPos % 32 + xMeta * fbArgs.dbXLength
							, int(floor((float)(locPos / 32)) + yMeta * fbArgs.dbYLength)
							, bitPos + zMeta * fbArgs.dbZLength
						);
					}
				}
			}
		}


		for (int locPos = 32 * fbArgs.dbYLength; locPos < 32 * 2 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						int locPosB = locPos - 32 * fbArgs.dbYLength;
						printf("point segm  set at x %d y %d z %d  \n"
							, locPosB % 32 + xMeta * fbArgs.dbXLength
							, int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength)
							, bitPos + zMeta * fbArgs.dbZLength
						);
					}
				}
			}
		}
	}
}




#pragma once
void printIsBlockActiveEtc(ForBoolKernelArgs<int> fbArgs,  uint32_t* metaDataArr,MetaDataGPU metaData) {
//testing dilatations
	printf("\n********* printIsBlockActiveEtc %d ***************\n", fbArgs.metaData.totalMetaLength);

	for (uint32_t linIdexMeta = 0; linIdexMeta <= fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
		//we get from linear index  the coordinates of the metadata block of intrest
		uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
		uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
		uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
// 7 :isActiveGold
// 8 :isFullGold
// 9 :isActiveSegm
// 10 :isFullSegm
// 11 :isToBeActivatedGold
// 12 :isToBeActivatedSegm
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7] >0 ){
		printf("is active gold linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8] ==1 ){
		printf("is full gold linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9] ==1 ){
		printf("is active segm linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10] ==1 ){
		printf("is full segm linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11] ==1 ){
		printf("is to be activated gold linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}	
	if( metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12] ==1 ){
		printf("is to be activated segm linMeta %d xMeta %d yMeta %d zMeta %d   \n", linIdexMeta, xMeta, yMeta, zMeta);
	}
	
	}

	printf("\n********* end ***************\n");


}



void testDilsSinglePoint(forTestPointStruct*& points, int x, int y, int z, int& pointsNumberRef, bool isGold) {
	for (int i = 0; i < pointsNumberRef; i++) {
		forTestPointStruct point = points[i];
		if ((point.isGold && isGold) || (!point.isGold && !isGold)) {
			//this point
			if (point.x == (x) && point.y == (y) && point.z == (z)) {
				point.isFoundAndDilatated = true;
			}
			//top
			if (point.x == (x) && point.y == (y) && point.z - 1 == (z)) {
				point.isFoundDilTop = true;
			}
			//bottom
			if (point.x == (x) && point.y == (y) && point.z + 1 == (z)) {
				point.isFoundDilBottom = true;
			}


			//anterior
			if (point.x == (x) && point.y - 1 == (y) && point.z == (z)) {
				point.isFoundDilAnterior = true;
			}
			//posterior
			if (point.x == (x) && point.y + 1 == (y) && point.z == (z)) {
				point.isFoundDilPosterior = true;
			}


			//left
			if (point.x + 1 == (x) && point.y == (y) && point.z == (z)) {
				point.isFoundDilLeft = true;
			}
			//right
			if (point.x - 1 == (x) && point.y == (y) && point.z == (z)) {
				point.isFoundDilRight = true;
			}
		}
	}



}




#pragma once
void testDilatations (ForBoolKernelArgs<int> fbArgs, uint32_t* arrsCPU,forTestPointStruct* pointsList, int& pointsNumberRef) {
//testing dilatations
	
	for (uint32_t linIdexMeta = 0; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
		//we get from linear index  the coordinates of the metadata block of intrest
		uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
		uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
		uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));

		for (int locPos = 0; locPos < 32 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						int locPosB = locPos - 32 * fbArgs.dbYLength;

						int x = locPosB % 32 + xMeta * fbArgs.dbXLength;
						int y = int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength);
						int z = bitPos + zMeta * fbArgs.dbZLength;
							testDilsSinglePoint(pointsList, x, y, z, pointsNumberRef, true);
						
					}
				}
			}
		}


		for (int locPos = 32 * fbArgs.dbYLength; locPos < 32 * 2 * fbArgs.dbYLength; locPos++) {
			auto col = arrsCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
			if (col > 0) {
				for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
					if (isBitAtCPU(col, bitPos)) {
						int locPosB = locPos - 32 * fbArgs.dbYLength;
						int x = locPosB % 32 + xMeta * fbArgs.dbXLength;
						int y = int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength);
						int z = bitPos + zMeta * fbArgs.dbZLength;
						testDilsSinglePoint(pointsList, x,y,z,pointsNumberRef,false );
					}
				}
			}
		}
	}
	
	//by now we checked wheather a point is present in dilatation array now we need to also establish is dilatation that we are looking for is in range
	for(int i =0; i<pointsNumberRef;i++ ){
		forTestPointStruct point= pointsList[i];
		//this point
		if( !point.isFoundAndDilatated){
			printf(" point %d %d %d not found at all \n", point.x, point.y, point.z);
			}
		//top
		if(!point.isFoundDilTop  &&  point.z>0 ){
			printf(" point %d %d %d not found TOP dilatation \n", point.x, point.y, point.z);
		}	
		//bottom
		if(!point.isFoundDilBottom && point.z< (fbArgs.metaData.MetaZLength*32 -1) ){
			printf(" point %d %d %d not found BOTTOM dilatation \n", point.x, point.y, point.z);
		}	
		
		
		//anterior
		if(!point.isFoundDilAnterior && point.y <(fbArgs.metaData.MetaYLength*fbArgs.dbYLength -1)  ){
			printf(" point %d %d %d not found ANTERIOR dilatation \n", point.x, point.y, point.z);
		}	
		//posterior
		if(!point.isFoundDilPosterior && point.y>0){
			printf(" point %d %d %d not found POSTERIOR dilatation \n", point.x, point.y, point.z);
		}	
		
				
		//left
		if( !point.isFoundDilLeft && point.x>0 ){
			printf(" point %d %d %d not found LEFT dilatation \n", point.x, point.y, point.z);
		}	
		//right
		if( !point.isFoundDilRight && point.x< (fbArgs.metaData.metaXLength*32 -1)  ){
			printf(" point %d %d %d not found RIGHT dilatation \n", point.x, point.y, point.z);
		}	
	}
		
	
}




/*
so we will look through result list and  given the information about wheather it 
is first or second dilatation we would establish weather all results that should be present here are 
additionally we will collect the fp counts and fncounts and in the end of the function establish weather fp and fn counts 
of the block in metadata are correct
*/
#pragma once
void testResultsAndCounters(ForBoolKernelArgs<int> fbArgs, uint32_t* arrsCPU,forTestPointStruct* points, int& pointsNumberRef
,uint32_t*& resultListPointerMetaCPU, uint32_t*& resultListPointerLocalCPU, uint32_t*& resultListPointerIterNumb
, int numberOfResults, uint32_t* metaDataArrPointerCPU, MetaDataGPU metaData) {
		
		int* fpCounts = (int*)calloc(fbArgs.metaData.totalMetaLength, sizeof(int));
		int* fnCounts = (int*)calloc(fbArgs.metaData.totalMetaLength, sizeof(int));



	
		for(int iRes=0;iRes< numberOfResults; iRes++){
		if(resultListPointerMetaCPU[iRes]>0){
			uint32_t linIdexMeta = resultListPointerMetaCPU[iRes] - (isGoldOffset * (resultListPointerMetaCPU[iRes] > isGoldOffset))-1;
			uint32_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
			uint32_t zMeta = uint32_t(floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength))));
			uint32_t yMeta = uint32_t(floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength)));
			
			uint32_t linLocal = resultListPointerLocalCPU[iRes];
			uint32_t xLoc = linLocal % fbArgs.dbXLength;
			uint32_t zLoc = uint32_t(floor((float)(linLocal / (32 * fbArgs.dbYLength))));
			uint32_t yLoc = uint32_t(floor((float)((linLocal - ((zLoc * 32 * fbArgs.dbYLength) + xLoc)) / 32)));


			uint32_t x = xMeta * 32 + xLoc;
			uint32_t y= yMeta * fbArgs.dbYLength + yLoc;
			uint32_t z = zMeta * 32 + zLoc;
			
			
			uint32_t iterNumb = resultListPointerIterNumb[iRes];
   // bool shouldBeInResAfterOneDil;
    //bool shouldBeInResAfterTwoDil;
			if((resultListPointerMetaCPU[iRes] > isGoldOffset)){
				fpCounts[linIdexMeta]+=1;
			}
			else{
				fnCounts[linIdexMeta]+=1;
			}
			
				for(int i =0; i<pointsNumberRef;i++ ){
						forTestPointStruct point= points[i];
						if( (point.shouldBeInResAfterOneDil && iterNumb==0)  
							||(point.shouldBeInResAfterTwoDil &&  iterNumb==1)      ){
							point.isFoundInResult=true;
						}
					}
			
		
		}
		}
		for (int i = 0; i < pointsNumberRef; i++) {
			forTestPointStruct point = points[i];
			if ((point.shouldBeInResAfterOneDil || point.shouldBeInResAfterTwoDil) && !point.isFoundInResult) {
				printf("fff point %d %d %d not found in result \n", point.x, point.y, point.z);

			}
			else {
				printf("ttt  point %d %d %d found in result \n", point.x, point.y, point.z);

			}

		};
//checking weather counters in metadata match the count that should be present 		
for(int linIdexMeta=0;linIdexMeta< fbArgs.metaData.totalMetaLength; linIdexMeta++){
if(fpCounts[linIdexMeta]>0){
	if(metaDataArrPointerCPU[ linIdexMeta* metaData.metaDataSectionLength + 3] = fpCounts[linIdexMeta] ){
		printf(" correct fp count %d in linMeta %d  "
		,fpCounts[linIdexMeta]
		,linIdexMeta
		);

	}else{
		printf("fff incorrect fp count %d is in meta %d in linMeta %d  "
		,fpCounts[linIdexMeta]
		,metaDataArrPointerCPU[ linIdexMeta* metaData.metaDataSectionLength + 3]
		,linIdexMeta
		);
	}
}
if(fnCounts[linIdexMeta]>0){
	if(metaDataArrPointerCPU[ linIdexMeta* metaData.metaDataSectionLength + 4] = fnCounts[linIdexMeta] ){
		printf(" correct fn count %d in linMeta %d  "
		,fnCounts[linIdexMeta]
		,linIdexMeta
		);

	}else{
		printf("fff incorrect fn count %d is in meta %d in linMeta %d  "
		,fnCounts[linIdexMeta]
		,metaDataArrPointerCPU[ linIdexMeta* metaData.metaDataSectionLength + 4]
		,linIdexMeta
		);
	}
}



}
		

}






#pragma once
inline forTestPointStruct getTestPoint(int x, int y, int z,
	bool isGold, int xMeta, int yMeta, int zMeta
	, int dbXLength, int dbYLength, int  dbZLength
	, int& pointsNumberRef
	, bool isGoldAndSegm = false
	, bool shouldBeInResAfterOneDil = false
	, bool shouldBeInResAfterTwoDil = false
) {
	pointsNumberRef += 1;
	forTestPointStruct res;

	res.x = xMeta * dbXLength + x;
	res.y = yMeta * dbYLength + y;
	res.z = zMeta * dbZLength + z;
	res.isGoldAndSegm = isGoldAndSegm;
	res.isGold = isGold;

	res.xMeta = xMeta;
	res.yMeta = yMeta;
	res.zMeta = zMeta;


	res.shouldBeInResAfterOneDil = shouldBeInResAfterOneDil;
	res.shouldBeInResAfterTwoDil = shouldBeInResAfterTwoDil;

	return res;

}




#pragma once
inline forTestMetaDataStruct getMetdataTestStruct(
	int& metasNumberRef,
	int xMeta,
	int yMeta,
	int zMeta,

	int fpCount = 0,
	int fnCount = 0,

	bool isToBeActiveAtStart = true,
	bool isToBeActiveAfterOneIter = true,
	bool isToBeActiveAfterTwoIter = true,

	bool isToBeFullAfterOneIter = false,
	bool isToBeFullAfterTwoIter = false,

	bool isToBeValidatedFpAfterOneIter = false,
	bool isToBeValidatedFpAfterTwoIter = false,

	bool isToBeValidatedFnAfterOneIter = false,
	bool isToBeValidatedFnAfterTwoIter = false,

	int fpConterAfterOneDil = 0,
	int fpConterAfterTwoDil = 0,

	int fnConterAfterOneDil = 0,
	int fnConterAfterTwoDil = 0) {


	forTestMetaDataStruct res;
	metasNumberRef += 1;
	res.xMeta = xMeta;
	res.yMeta = yMeta;
	res.zMeta = zMeta;


	res.isToBeActiveAtStart = (fpCount+ fnCount)>0;
	res.isToBeActiveAfterOneIter = isToBeActiveAfterOneIter;
	res.isToBeActiveAfterTwoIter = isToBeActiveAfterTwoIter;

	res.isToBeFullAfterOneIter = isToBeFullAfterOneIter;
	res.isToBeFullAfterTwoIter = isToBeFullAfterTwoIter;

	res.fpCount = fpCount;
	res.fnCount = fnCount;

	res.requiredspaceInFpResultList = fpCount;
	res.requiredspaceInFnResultList = fnCount;

	res.isToBeValidatedFpAfterOneIter = fpCount > 0;
	res.isToBeValidatedFpAfterTwoIter = fpCount > 0;

	res.isToBeValidatedFnAfterOneIter = fnCount > 0;
	res.isToBeValidatedFnAfterTwoIter = fnCount > 0;


	res.fpConterAfterOneDil = fpConterAfterOneDil;
	res.fpConterAfterTwoDil = fpConterAfterTwoDil;

	res.fnConterAfterOneDil = fnConterAfterOneDil;
	res.fnConterAfterTwoDil = fnConterAfterTwoDil;
	return res;
}
