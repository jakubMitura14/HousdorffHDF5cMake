#include "hip/hip_runtime.h"
#include <cstdint>
#include "Structs.cu"
#include "MemoryTransfers.cu"

/*
copy from host to device
*/
#pragma once
inline MetaDataGPU allocateMetaDataOnGPU(MetaDataCPU metaDataCPU) {
	MetaDataGPU res;
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 1, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 2, 0, 0, 1000, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 3, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 4, 0, 0, 1000, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 5, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 6, 0, 0, 1000, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 7, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 8, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 9, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 10, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 11, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 12, 0, 0, 1, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 13, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 14, 0, 0, 0, false);
	setArrCPU<unsigned int>(metaDataCPU.minMaxes, 15, 0, 0, 0, false);




	//!! x and z intentionally mixed !!
	res.fpCount = allocate3dInGPU(metaDataCPU.fpCount);
	res.fnCount = allocate3dInGPU(metaDataCPU.fnCount);
	res.minMaxes = allocate3dInGPU(metaDataCPU.minMaxes);

	res.fpCount = allocate3dInGPU(metaDataCPU.fpCount);
	res.fnCount = allocate3dInGPU(metaDataCPU.fnCount);
	res.fpCounter = allocate3dInGPU(metaDataCPU.fpCounter);
	res.fnCounter = allocate3dInGPU(metaDataCPU.fnCounter);
	res.fpOffset = allocate3dInGPU(metaDataCPU.fpOffset);
	res.fnOffset = allocate3dInGPU(metaDataCPU.fnOffset);

	res.isActiveGold = allocate3dInGPU(metaDataCPU.isActiveGold);
	res.isFullGold = allocate3dInGPU(metaDataCPU.isFullGold);
	res.isActiveSegm = allocate3dInGPU(metaDataCPU.isActiveSegm);
	res.isFullSegm = allocate3dInGPU(metaDataCPU.isFullSegm);

	res.isToBeActivatedGold = allocate3dInGPU(metaDataCPU.isToBeActivatedGold);
	res.isToBeActivatedSegm = allocate3dInGPU(metaDataCPU.isToBeActivatedSegm);



	res.isToBeValidatedFp = allocate3dInGPU(metaDataCPU.isToBeValidatedFp);
	res.isToBeValidatedFn = allocate3dInGPU(metaDataCPU.isToBeValidatedFn);

	res.workQueue = allocate3dInGPU(metaDataCPU.workQueue);
	res.resultList = allocate3dInGPU(metaDataCPU.resultList);

	res.metaXLength = res.fpCount.Nx;
	res.MetaYLength = res.fpCount.Ny;
	res.MetaZLength = res.fpCount.Nz;

	res.totalMetaLength = metaDataCPU.totalMetaLength;
	//allocating on GPU and copying  cpu data onto GPU

	return res;

}

/*
copy from device to host
*/
#pragma once
inline void copyMetaDataToCPU(MetaDataCPU metaDataCPU, MetaDataGPU metaDataGPU) {
	copyDeviceToHost3d(metaDataGPU.fpCount, metaDataCPU.fpCount);
	copyDeviceToHost3d(metaDataGPU.fnCount, metaDataCPU.fnCount);

	copyDeviceToHost3d(metaDataGPU.minMaxes, metaDataCPU.minMaxes);

	copyDeviceToHost3d(metaDataGPU.fpCounter, metaDataCPU.fpCounter);
	copyDeviceToHost3d(metaDataGPU.fnCounter, metaDataCPU.fnCounter);
	copyDeviceToHost3d(metaDataGPU.fpOffset, metaDataCPU.fpOffset);
	copyDeviceToHost3d(metaDataGPU.fnOffset, metaDataCPU.fnOffset);

	copyDeviceToHost3d(metaDataGPU.isActiveGold, metaDataCPU.isActiveGold);
	copyDeviceToHost3d(metaDataGPU.isFullGold, metaDataCPU.isFullGold);
	copyDeviceToHost3d(metaDataGPU.isActiveSegm, metaDataCPU.isActiveSegm);
	copyDeviceToHost3d(metaDataGPU.isFullSegm, metaDataCPU.isFullSegm);

	copyDeviceToHost3d(metaDataGPU.isToBeActivatedGold, metaDataCPU.isToBeActivatedGold);
	copyDeviceToHost3d(metaDataGPU.isToBeActivatedSegm, metaDataCPU.isToBeActivatedSegm);

	copyDeviceToHost3d(metaDataGPU.workQueue, metaDataCPU.workQueue);
	copyDeviceToHost3d(metaDataGPU.resultList, metaDataCPU.resultList);

	copyDeviceToHost3d(metaDataGPU.isToBeValidatedFp, metaDataCPU.isToBeValidatedFp);
	copyDeviceToHost3d(metaDataGPU.isToBeValidatedFn, metaDataCPU.isToBeValidatedFn);




}


/*
free metadata
*/
#pragma once
inline void freeMetaDataGPU(MetaDataGPU metaDataGPU) {
	hipFree(metaDataGPU.fpCount.arrPStr.ptr);
	hipFree(metaDataGPU.fnCount.arrPStr.ptr);
	hipFree(metaDataGPU.minMaxes.arrPStr.ptr);

	hipFree(metaDataGPU.fpCounter.arrPStr.ptr);
	hipFree(metaDataGPU.fnCounter.arrPStr.ptr);
	hipFree(metaDataGPU.fpOffset.arrPStr.ptr);
	hipFree(metaDataGPU.fnOffset.arrPStr.ptr);

	hipFree(metaDataGPU.isActiveGold.arrPStr.ptr);
	hipFree(metaDataGPU.isFullGold.arrPStr.ptr);
	hipFree(metaDataGPU.isActiveSegm.arrPStr.ptr);
	hipFree(metaDataGPU.isFullSegm.arrPStr.ptr);

	hipFree(metaDataGPU.isToBeActivatedGold.arrPStr.ptr);
	hipFree(metaDataGPU.isToBeActivatedSegm.arrPStr.ptr);

	hipFree(metaDataGPU.workQueue.arrPStr.ptr);

	hipFree(metaDataGPU.resultList.arrPStr.ptr);

	hipFree(metaDataGPU.isToBeValidatedFp.arrPStr.ptr);
	hipFree(metaDataGPU.isToBeValidatedFn.arrPStr.ptr);




}


///*
//free metadata
//*/
//inline void freeMetaDataCPU(MetaDataCPU metaDataCPU) {
//	free(metaDataCPU.fpCount);
//	free(metaDataCPU.fnCount);
//}