#include "hip/hip_runtime.h"
#include <cstdint>
#include "Structs.cu"

/*
copy from host to device
*/
#pragma once
inline MetaDataGPU allocateMetaDataOnGPU(MetaDataCPU metaDataCPU, unsigned int*& minMaxes) {
	MetaDataGPU res;

	metaDataCPU.minMaxes[1] = 0;
	metaDataCPU.minMaxes[2] = 1000;
	metaDataCPU.minMaxes[3] = 0;
	metaDataCPU.minMaxes[4] = 1000;
	metaDataCPU.minMaxes[5] = 0;
	metaDataCPU.minMaxes[6] = 1000;
	metaDataCPU.minMaxes[7] = 0;
	metaDataCPU.minMaxes[8] = 0;
	metaDataCPU.minMaxes[9] = 0;
	metaDataCPU.minMaxes[10] = 0;
	metaDataCPU.minMaxes[11] = 0;
	metaDataCPU.minMaxes[12] = 0;
	metaDataCPU.minMaxes[13] = 0;
	metaDataCPU.minMaxes[14] = 0;
	metaDataCPU.minMaxes[15] = 0;
	metaDataCPU.minMaxes[16] = 0;
	metaDataCPU.minMaxes[17] = 0;
	metaDataCPU.minMaxes[18] = 0;
	metaDataCPU.minMaxes[19] = 0;
	metaDataCPU.minMaxes[20] = 0;

	size_t size = sizeof(unsigned int) * 20;
	hipMemcpy(minMaxes, metaDataCPU.minMaxes, size, hipMemcpyHostToDevice);

	//res.resultList = allocate3dInGPU(metaDataCPU.resultList);

	//res.metaXLength = metaDataCPU.metaXLength;
	//res.MetaYLength = metaDataCPU.MetaYLength;
	//res.MetaZLength = metaDataCPU.MetaZLength;

	//res.totalMetaLength = metaDataCPU.totalMetaLength;
	//allocating on GPU and copying  cpu data onto GPU

	return res;

}

/*
copy from device to host
*/
#pragma once
inline void copyMetaDataToCPU(MetaDataCPU metaDataCPU, MetaDataGPU metaDataGPU) {
	//copyDeviceToHost3d(metaDataGPU.fpCount, metaDataCPU.fpCount);
	//copyDeviceToHost3d(metaDataGPU.fnCount, metaDataCPU.fnCount);
	size_t size = sizeof(unsigned int) * 20;

	hipMemcpy(metaDataCPU.minMaxes, metaDataGPU.minMaxes, size, hipMemcpyDeviceToHost);







}


///*
//free metadata
//*/
//#pragma once
//inline void freeMetaDataGPU(MetaDataGPU metaDataGPU) {
//	hipFree(metaDataGPU.fpCount.arrPStr.ptr);
//	hipFree(metaDataGPU.fnCount.arrPStr.ptr);
//	hipFree(metaDataGPU.minMaxes);
//
//	hipFree(metaDataGPU.fpCounter.arrPStr.ptr);
//	hipFree(metaDataGPU.fnCounter.arrPStr.ptr);
//	hipFree(metaDataGPU.fpOffset.arrPStr.ptr);
//	hipFree(metaDataGPU.fnOffset.arrPStr.ptr);
//
//	hipFree(metaDataGPU.isActiveGold.arrPStr.ptr);
//	hipFree(metaDataGPU.isFullGold.arrPStr.ptr);
//	hipFree(metaDataGPU.isActiveSegm.arrPStr.ptr);
//	hipFree(metaDataGPU.isFullSegm.arrPStr.ptr);
//
//	hipFree(metaDataGPU.isToBeActivatedGold.arrPStr.ptr);
//	hipFree(metaDataGPU.isToBeActivatedSegm.arrPStr.ptr);
//
//	hipFree(metaDataGPU.workQueue.arrPStr.ptr);
//
//
//	//hipFree(metaDataGPU.resultList);
//
//	//hipFreeAsync(metaDataGPU.resultList,0);
//	//hipFree(metaDataGPU.resultList.arrPStr.ptr);
//
//	hipFree(metaDataGPU.isToBeValidatedFp.arrPStr.ptr);
//	hipFree(metaDataGPU.isToBeValidatedFn.arrPStr.ptr);
//
//
//
//
//}


///*
//free metadata
//*/
//inline void freeMetaDataCPU(MetaDataCPU metaDataCPU) {
//	free(metaDataCPU.fpCount);
//	free(metaDataCPU.fnCount);
//}