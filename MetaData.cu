#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cstdint>
#include "Structs.cu"
#include "MemoryTransfers.cu"

/*
copy from host to device
*/
#pragma once
inline MetaDataGPU allocateMetaDataOnGPU(unsigned int Nx, unsigned int Ny, unsigned int Nz) {
	MetaDataGPU res;
	//!! x and z intentionally mixed !!
	res.fpCount = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fnCount = getArrGpu<unsigned int>(Nx, Ny, Nz);
	//res.minMaxes = allocate3dInGPU(metaDataCPU.minMaxes);

	res.fpCount = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fnCount = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fpCounter = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fnCounter = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fpOffset = getArrGpu<unsigned int>(Nx, Ny, Nz);
	res.fnOffset = getArrGpu<unsigned int>(Nx, Ny, Nz);

	res.isActiveGold = getArrGpu<bool>(Nx, Ny, Nz);
	res.isFullGold = getArrGpu<bool>(Nx, Ny, Nz);
	res.isActiveSegm = getArrGpu<bool>(Nx, Ny, Nz);
	res.isFullSegm = getArrGpu<bool>(Nx, Ny, Nz);

	res.isToBeActivatedGold = getArrGpu<bool>(Nx, Ny, Nz);
	res.isToBeActivatedSegm = getArrGpu<bool>(Nx, Ny, Nz);


	uint16_t* workQueue;
	size_t size = (Nx * Ny * Nz) * 4 + 5;
	hipMallocAsync(&workQueue, size, 0);
	//res.workQueue = workQueue;

	//res.isToBeValidatedFp = allocate3dInGPU(metaDataCPU.isToBeValidatedFp);
	//res.isToBeValidatedFn = allocate3dInGPU(metaDataCPU.isToBeValidatedFn);

	//res.workQueue = allocate3dInGPU(metaDataCPU.workQueue);
	//res.resultList = allocate3dInGPU(metaDataCPU.resultList);

	res.metaXLength = res.fpCount.Nx;
	res.MetaYLength = res.fpCount.Ny;
	res.MetaZLength = res.fpCount.Nz;

	res.totalMetaLength = (Nx * Ny * Nz);
	//allocating on GPU and copying  cpu data onto GPU

	return res;

}

/*
copy from device to host
*/
#pragma once
inline void copyMetaDataToCPU(MetaDataCPU metaDataCPU, MetaDataGPU metaDataGPU) {
	copyDeviceToHost3d(metaDataGPU.fpCount, metaDataCPU.fpCount);
	copyDeviceToHost3d(metaDataGPU.fnCount, metaDataCPU.fnCount);

	copyDeviceToHost3d(metaDataGPU.minMaxes, metaDataCPU.minMaxes);

	copyDeviceToHost3d(metaDataGPU.fpCounter, metaDataCPU.fpCounter);
	copyDeviceToHost3d(metaDataGPU.fnCounter, metaDataCPU.fnCounter);
	copyDeviceToHost3d(metaDataGPU.fpOffset, metaDataCPU.fpOffset);
	copyDeviceToHost3d(metaDataGPU.fnOffset, metaDataCPU.fnOffset);

	copyDeviceToHost3d(metaDataGPU.isActiveGold, metaDataCPU.isActiveGold);
	copyDeviceToHost3d(metaDataGPU.isFullGold, metaDataCPU.isFullGold);
	copyDeviceToHost3d(metaDataGPU.isActiveSegm, metaDataCPU.isActiveSegm);
	copyDeviceToHost3d(metaDataGPU.isFullSegm, metaDataCPU.isFullSegm);

	copyDeviceToHost3d(metaDataGPU.isToBeActivatedGold, metaDataCPU.isToBeActivatedGold);
	copyDeviceToHost3d(metaDataGPU.isToBeActivatedSegm, metaDataCPU.isToBeActivatedSegm);

	copyDeviceToHost3d(metaDataGPU.workQueue, metaDataCPU.workQueue);
	//copyDeviceToHost3d(metaDataGPU.resultList, metaDataCPU.resultList);

	copyDeviceToHost3d(metaDataGPU.isToBeValidatedFp, metaDataCPU.isToBeValidatedFp);
	copyDeviceToHost3d(metaDataGPU.isToBeValidatedFn, metaDataCPU.isToBeValidatedFn);




}


/*
free metadata
*/
#pragma once
inline void freeMetaDataGPU(MetaDataGPU metaDataGPU) {
	hipFree(metaDataGPU.fpCount.arrPStr.ptr);
	hipFree(metaDataGPU.fnCount.arrPStr.ptr);
	hipFree(metaDataGPU.minMaxes.arrPStr.ptr);

	hipFree(metaDataGPU.fpCounter.arrPStr.ptr);
	hipFree(metaDataGPU.fnCounter.arrPStr.ptr);
	hipFree(metaDataGPU.fpOffset.arrPStr.ptr);
	hipFree(metaDataGPU.fnOffset.arrPStr.ptr);

	hipFree(metaDataGPU.isActiveGold.arrPStr.ptr);
	hipFree(metaDataGPU.isFullGold.arrPStr.ptr);
	hipFree(metaDataGPU.isActiveSegm.arrPStr.ptr);
	hipFree(metaDataGPU.isFullSegm.arrPStr.ptr);

	hipFree(metaDataGPU.isToBeActivatedGold.arrPStr.ptr);
	hipFree(metaDataGPU.isToBeActivatedSegm.arrPStr.ptr);

	hipFree(metaDataGPU.workQueue.arrPStr.ptr);
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD


=======
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
	
	
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
	//hipFree(metaDataGPU.resultList);

	//hipFreeAsync(metaDataGPU.resultList,0);
	//hipFree(metaDataGPU.resultList.arrPStr.ptr);

	hipFree(metaDataGPU.isToBeValidatedFp.arrPStr.ptr);
	hipFree(metaDataGPU.isToBeValidatedFn.arrPStr.ptr);




}


///*
//free metadata
//*/
//inline void freeMetaDataCPU(MetaDataCPU metaDataCPU) {
//	free(metaDataCPU.fpCount);
//	free(metaDataCPU.fnCount);
//}