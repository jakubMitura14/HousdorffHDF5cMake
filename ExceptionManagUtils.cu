#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <cmath>
#include <cstdint>
#include <assert.h>


#pragma once
inline hipError_t checkCuda(hipError_t result, std::string description)
{
    if (result != hipSuccess) {
        printf("%d", description);
        fprintf(stderr, "CUDA Runtime Error in %d : %s\n", description, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}
