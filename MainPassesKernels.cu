#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include "DilatationKernels.cu"
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include "BiggerMainFunctions.cu"
#include <cooperative_groups/memcpy_async.h>

using namespace cooperative_groups;


/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/





/*
we need to
Data
- shared memory
    -for uploaded data from reduced arrays
    -for dilatation results
    -for result paddings
0) load data about what metadata blocks should be analyzed from work queue
1) load data from given reduced arr into shared memory
2) perform bit  dilatations in 6 directions
    and save to result to result shared memory - additionally dilatations into its own shared memory
3) given the block is to be validated (in case it is first main pass - all needs to be) we check  if
    - if there is set bit (voxel) in res shmem but not in source shmem
        - we establish is there anything of intrest in the primary given array of other type (so for gold we check segm and for segm gold - but original ones)
        - if so we add this to the result list in a spot we established from offsets of metadata
            - we set metadata's fp and fn result counters - so later we will be able to establish wheather block should be validated at all
            - we also increment local counters of fp and fn - those will be used for later
4) we save data from result shmem into reduced arrays and from paddings into padding store (both in global memory)

*/






template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {

    //inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs, uint32_t * mainArr, MetaDataGPU metaData
    //    , unsigned int* minMaxes, uint32_t * workQueue
    //    , uint32_t * resultListPointerMeta, uint32_t * resultListPointerLocal, uint32_t * resultListPointerIterNumb, uint32_t * origArrs, uint32_t * metaDataArr) {

    //if (threadIdx.x == 0 && threadIdx.y == 0) {
    //    printf("in metadataPass totalMetaLength  %d   \n", fbArgs.metaData.totalMetaLength);

    //};

    thread_block cta = cooperative_groups::this_thread_block();

    thread_block_tile<32> tile = tiled_partition<32>(cta);
    grid_group grid = cooperative_groups::this_grid();

    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t mainShmem[lengthOfMainShmem];



    constexpr size_t stages_count = 2; // Pipeline stages number

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;

    //cuda::pipeline<cuda::thread_scope_thread>  pipeline = cuda::make_pipeline(cta, &shared_state);
    cuda::pipeline<cuda::thread_scope_block>  pipeline = cuda::make_pipeline(cta, &shared_state);



    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[1];

    __shared__ uint32_t lastI[1];


    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];


    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[40];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX

 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior

    */


    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) { iterationNumb[0] = -1; };
    if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        isSegmPassToContinue[0] = true;
    };


    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number




    do{
        if (threadIdx.x == 2 && threadIdx.y == 0) {
    if (blockIdx.x == 0) {
     //   printf("iter nuumb %d \n", iterationNumb[0]);
      //  fbArgs.metaData.minMaxes[13] = iterationNumb[0];
    }
};

        mainDilatation(false, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
            , fbArgs.workQueuePointer
            , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
            , cta, tile, grid, mainShmem
            , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset
            , globalWorkQueueCounter
            , localWorkQueueCounter
            , localTotalLenthOfWorkQueue
            , localFpConter
            , localFnConter, blockFpConter
            , blockFnConter
            , resultfpOffset
            , resultfnOffset, worQueueStep, localMinMaxes
            , localBlockMetaData, fpFnLocCounter
            , isGoldPassToContinue, isSegmPassToContinue
            , fbArgs.origArrsPointer
            , fbArgs.metaDataArrPointer, isGoldForLocQueue
            , lastI, pipeline

        );

        grid.sync();

        ///////////// loading work queue for padding dilatations
        metadataPass(fbArgs, true, 11, 7, 8,
            12, 9, 10
            , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
            , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
            , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
            , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);




        //////////// padding dilatations
        grid.sync();
        mainDilatation(true, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
            , fbArgs.workQueuePointer
            , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
            , cta, tile, grid, mainShmem
            , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset
            , globalWorkQueueCounter
            , localWorkQueueCounter
            , localTotalLenthOfWorkQueue
            , localFpConter
            , localFnConter, blockFpConter
            , blockFnConter
            , resultfpOffset
            , resultfnOffset, worQueueStep, localMinMaxes
            , localBlockMetaData, fpFnLocCounter
            , isGoldPassToContinue, isSegmPassToContinue
            , fbArgs.origArrsPointer
            , fbArgs.metaDataArrPointer, isGoldForLocQueue
            , lastI, pipeline

        );


        grid.sync();
        ////////////////////////main metadata pass
        metadataPass(fbArgs, false, 7, 8, 8,
            9, 10, 8
            , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
            , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
            , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
            , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);
        grid.sync();
        //if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        //    printf("  isGoldPassToContinue %d isSegmPassToContinue %d \n ", isGoldPassToContinue[0], isSegmPassToContinue[0]);
        //};
    
    } while (isGoldPassToContinue[0] || isSegmPassToContinue[0]);

    //grid.sync();

    ////for final result
    //if (threadIdx.x == 2 && threadIdx.y == 0) {
    //    if (blockIdx.x == 0) {

    //      //  fbArgs.metaData.minMaxes[13] = iterationNumb[0];
    //    }
    //};
    

    //grid.sync();


    //if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
    //    printf("  isGoldPassToContinue %d isSegmPassToContinue %d \n ", isGoldPassToContinue[0], isSegmPassToContinue[0]);
    //};

//  }// end while

  //setting global iteration number to local one 

}





#pragma once
template <typename T>
ForBoolKernelArgs<T> mainKernelsRun(ForFullBoolPrepArgs<T> fFArgs, uint32_t*& reducedResCPU
    , uint32_t*& resultListPointerMetaCPU
    ,uint32_t*& resultListPointerLocalCPU
    ,uint32_t*& resultListPointerIterNumbCPU
    ,uint32_t*& metaDataArrPointerCPU
    ,uint32_t*& workQueuePointerCPU
    ,uint32_t*& origArrsCPU
    , const int WIDTH, const int HEIGHT, const int DEPTH
) {

    //hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;
    int device = 0;
    unsigned int cpuIterNumb = -1;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<T>,
        0);
    int warpsNumbForMinMax = blockSize / 32;
    int blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    int warpsNumbForboolPrepareKernel = blockSize / 32;
    int blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    int theadsForFirstMetaPass = blockSize;
    int blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)mainPassKernel<T>,
        0);
    int warpsNumbForMainPass = blockSize / 32;
    int blockForMainPass = minGridSize;
        printf("warpsNumbForMainPass %d blockForMainPass %d  ", warpsNumbForMainPass, blockForMainPass);


   // warpsNumbForMainPass = 5;
  //  blockForMainPass = 1;

        




    //pointers ...
    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;

    uint32_t* origArrsPointer;
    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;
    uint32_t* metaDataArrPointer;

    uint32_t* workQueuePointer;

    
    
    //main arrays allocations
    T* goldArrPointer;
    T* segmArrPointer;
    //size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);

    hipMallocAsync(&goldArrPointer, sizeMainArr,0);
    hipMallocAsync(&segmArrPointer, sizeMainArr,0);

    hipMemcpyAsync(goldArrPointer, fFArgs.goldArr.arrP, sizeMainArr, hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(segmArrPointer, fFArgs.segmArr.arrP, sizeMainArr, hipMemcpyHostToDevice, 0);


    array3dWithDimsGPU<T> goldArr;
    array3dWithDimsGPU<T> segmArr;

    goldArr.arrP = goldArrPointer;
    goldArr.Nx = WIDTH;
    goldArr.Ny = HEIGHT;
    goldArr.Nz = DEPTH;



    segmArr.arrP = segmArrPointer;
    segmArr.Nx = WIDTH;
    segmArr.Ny = HEIGHT;
    segmArr.Nz = DEPTH;
    checkCuda(hipDeviceSynchronize(), "a0a");

    unsigned int* minMaxes;
    size_t sizeminMaxes = sizeof(unsigned int) * 20;
    hipMallocAsync(&minMaxes, sizeminMaxes,0);




    checkCuda(hipDeviceSynchronize(), "a0b");
    ForBoolKernelArgs<T> fbArgs = getArgsForKernel<T>(fFArgs, goldArrPointer, segmArrPointer, minMaxes, warpsNumbForMainPass, blockForMainPass, WIDTH,HEIGHT, DEPTH);
    MetaDataGPU metaData = fbArgs.metaData;
    fbArgs.metaData.minMaxes = minMaxes;
    fbArgs.minMaxes = minMaxes;


    fbArgs.goldArr = goldArr;
    fbArgs.segmArr = segmArr;


    ////preparation kernel

    // initialize, then launch

    checkCuda(hipDeviceSynchronize(), "a1");


    //getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > ( minMaxes);
    getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > (fbArgs, minMaxes, goldArrPointer, segmArrPointer);

    checkCuda(hipDeviceSynchronize(), "a1b");


    checkCuda(hipDeviceSynchronize(), "a2a");

    metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs, workQueuePointer, minMaxes, metaData, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a2b");

   boolPrepareKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, metaData, origArrsPointer, metaDataArrPointer, goldArrPointer, segmArrPointer, minMaxes);
  //  //uint32_t* origArrs, uint32_t* metaDataArr     metaDataArr[linIdexMeta * metaData.metaDataSectionLength     metaDataOffset

   checkCuda(hipDeviceSynchronize(), "a3");



  int fpPlusFn =  allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb, origArrsPointer, mainArrAPointer, mainArrBPointer, metaData,goldArr,segmArr);




    checkCuda(hipDeviceSynchronize(), "a4");

    //hipFreeAsync(goldArrPointer, 0);
    //hipFreeAsync(segmArrPointer, 0);

    firstMetaPrepareKernel << <blockForFirstMetaPass, theadsForFirstMetaPass >> > (fbArgs, metaData, minMaxes, workQueuePointer, origArrsPointer, metaDataArrPointer);

   checkCuda(hipDeviceSynchronize(), "a5");
    //void* kernel_args[] = { &fbArgs, mainArrPointer,&metaData,minMaxes, workQueuePointer,resultListPointerMeta,resultListPointerLocal, resultListPointerIterNumb };
    
    
    
    //fbArgs.goldArr = goldArr;
    //fbArgs.segmArr = segmArr;
    fbArgs.metaData = metaData;

    fbArgs.resultListPointerMeta = resultListPointerMeta;
    fbArgs.resultListPointerLocal = resultListPointerLocal;
    fbArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    fbArgs.origArrsPointer = origArrsPointer;
    fbArgs.mainArrAPointer = mainArrAPointer;
    fbArgs.mainArrBPointer = mainArrBPointer;


    fbArgs.metaDataArrPointer = metaDataArrPointer;
    fbArgs.workQueuePointer = workQueuePointer;
    fbArgs.minMaxes = minMaxes;
    void* kernel_args[] = { &fbArgs };


    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), blockForMainPass, dim3(32, warpsNumbForMainPass), kernel_args);



    checkCuda(hipDeviceSynchronize(), "a6");



    size_t sizeMinnMax  = sizeof(unsigned int) * 20;

    hipMemcpy(fFArgs.metaData.minMaxes, minMaxes, sizeMinnMax, hipMemcpyDeviceToHost);

    //copy to CPU
    size_t sizeCPU = metaData.totalMetaLength * metaData.mainArrSectionLength * sizeof(uint32_t);
    reducedResCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(reducedResCPU, mainArrAPointer, sizeCPU, hipMemcpyDeviceToHost);

    origArrsCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(origArrsCPU, origArrsPointer, sizeCPU, hipMemcpyDeviceToHost);


    size_t sizeRes = sizeof(uint32_t) * (fpPlusFn + 50);


  resultListPointerMetaCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    resultListPointerLocalCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
   resultListPointerIterNumbCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
   hipMemcpy(resultListPointerMetaCPU, resultListPointerMeta, sizeRes, hipMemcpyDeviceToHost);

   hipMemcpy(resultListPointerLocalCPU, resultListPointerLocal, sizeRes, hipMemcpyDeviceToHost);

   hipMemcpy(resultListPointerIterNumbCPU, resultListPointerIterNumb, sizeRes, hipMemcpyDeviceToHost);

   size_t sizemetaDataArr = metaData.totalMetaLength * (20) * sizeof(uint32_t);
   metaDataArrPointerCPU = (uint32_t*)calloc(metaData.totalMetaLength * (20), sizeof(uint32_t));
   hipMemcpy(metaDataArrPointerCPU, metaDataArrPointer, sizemetaDataArr, hipMemcpyDeviceToHost);

   size_t sizeC = (metaData.totalMetaLength * sizeof(uint32_t));

   workQueuePointerCPU = (uint32_t*)calloc(metaData.totalMetaLength, sizeof(uint32_t));
   hipMemcpy(workQueuePointerCPU, workQueuePointer, sizeC, hipMemcpyDeviceToHost);



   checkCuda(hipDeviceSynchronize(), "a7");






  //  //hipLaunchCooperativeKernel((void*)mainPassKernel<int>, deviceProp.multiProcessorCount, fFArgs.threadsMainPass, fbArgs);




  //  ////copyDeviceToHost3d(goldArr, fFArgs.goldArr);
  //  ////copyDeviceToHost3d(segmArr, fFArgs.segmArr);
  //  //// getting arrays allocated on  cpu to 


  //  //copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);

  //  //// printForDebug(fbArgs, fFArgs, resultListPointer, mainArrPointer, workQueuePointer, metaData);


  //  checkCuda(hipDeviceSynchronize(), "just after copy device to host");
  //  //hipGetLastError();

  //hipFreeAsync(goldArrPointer, 0);
  //hipFreeAsync(segmArrPointer, 0);


    hipFreeAsync(resultListPointerMeta, 0);
    hipFreeAsync(resultListPointerLocal, 0);
    hipFreeAsync(resultListPointerIterNumb, 0);
    hipFreeAsync(workQueuePointer, 0);
    hipFreeAsync(origArrsPointer, 0);
    hipFreeAsync(metaDataArrPointer, 0);
    hipFreeAsync(mainArrAPointer, 0);
    hipFreeAsync(mainArrBPointer, 0);



    checkCuda(hipDeviceSynchronize(), "last ");

/////////// error handling 
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    ForBoolKernelArgs<T> res;
    return res;
   // return fbArgs;
}













