#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include "DilatationKernels.cu"
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include "BiggerMainFunctions.cu"
#include <cooperative_groups/memcpy_async.h>

using namespace cooperative_groups;


/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/





/*
we need to
Data
- shared memory
    -for uploaded data from reduced arrays
    -for dilatation results
    -for result paddings
0) load data about what metadata blocks should be analyzed from work queue
1) load data from given reduced arr into shared memory
2) perform bit  dilatations in 6 directions
    and save to result to result shared memory - additionally dilatations into its own shared memory
3) given the block is to be validated (in case it is first main pass - all needs to be) we check  if
    - if there is set bit (voxel) in res shmem but not in source shmem
        - we establish is there anything of intrest in the primary given array of other type (so for gold we check segm and for segm gold - but original ones)
        - if so we add this to the result list in a spot we established from offsets of metadata
            - we set metadata's fp and fn result counters - so later we will be able to establish wheather block should be validated at all
            - we also increment local counters of fp and fn - those will be used for later
4) we save data from result shmem into reduced arrays and from paddings into padding store (both in global memory)

*/




/**
CPU part of the loop - where we copy data required to know wheather next loop should be executed and to increment the iteration number
*/
template <typename TKKI>
inline bool runAfterOneLoop(ForBoolKernelArgs<TKKI> gpuArgs, ForFullBoolPrepArgs<TKKI> cpuArgs, unsigned int& cpuIterNumb) {
    cpuIterNumb += 1;

    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    cpuArgs.metaData.minMaxes.arrP[0][0][13] = cpuIterNumb;
    //copy back on gpu
    copyHostToDevice(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    // returning true - so signal that we need to loop on only when we did not reach yet the required percent of covered voxels
    return ((ceil(cpuArgs.metaData.minMaxes.arrP[0][0][7] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][10])
        || (ceil(cpuArgs.metaData.minMaxes.arrP[0][0][8] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][11]));

}












template <typename TKKI>
inline __global__ void testKernel(ForBoolKernelArgs<TKKI> fbArgs, unsigned int* minMaxes, uint32_t* mainArr, MetaDataGPU metaData, uint32_t* workQueue, uint32_t* origArr) {
    thread_block cta = this_thread_block();

    //work queue !!
    //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    //    for (uint32_t ii = blockIdx.x; ii < 7; ii += gridDim.x) {
    //        if (workQueue[ii] > 0) {
    //            if (workQueue[ii] > (isGoldOffset-1)) {
    //                printf("in gold workqueue elment %d  \n", (workQueue[ii] - isGoldOffset));
    //            }
    //            else {
    //                printf("in segm workqueue elment %d  \n", (workQueue[ii]));

    //            }

    //        }

    //    }
    //}
    // 
        //results  !!
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        for (uint32_t ii = blockIdx.x; ii < 10; ii += gridDim.x) {
            if (fbArgs.resultListPointerMeta[ii] > 0) {
                printf("in TEST kernel  result lin meta %d ii  \n", fbArgs.resultListPointerMeta[ii]);

            }

        }
    }



    sync(cta);
    char* tensorslice;


    for (uint32_t linIdexMeta = blockIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (metaData.metaXLength * metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength));

        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint32_t x = (xMeta + metaData.minX) * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint32_t  y = (yMeta + metaData.minY) * fbArgs.dbYLength + yLoc;//absolute position
                for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {

                    uint32_t z = (zMeta + metaData.minZ) * fbArgs.dbZLength + zLoc;//absolute position
                    uint8_t ww = 0;
                    //uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength)*ww];//
                    uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww];//
                    //uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength)];




                    //rrrrresult meta 1 isGold 1 old 0 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655351  localInd 24544

                    //if (linIdexMeta== 1 ) {
                    //    if (  (fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 24544) {
                    //            printf("res in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
                    //  ,  x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);

                    //    }
                    //
                    //}
                    ////    rrrrresult meta 2 isGold 1 old 1 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655352  localInd 23839

                    //if (linIdexMeta == 2) {
                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 23839) {
                    //        printf( "res in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);

                    //    }

                    //}
                    ////    rrrrresult meta 4 isGold 1 old 2 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655354  localInd 767

                    //if (linIdexMeta == 4) {
                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 767) {
                    //        printf("res in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);

                    //    }

                    //}
                    ////    rrrrresult meta 0 isGold 0 old 3 localFpConter 0 localFnConter 1 fpOffset 3 fnOffset 1 linIndUpdated 0  localInd 24575

                    //if (linIdexMeta == 0) {
                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 24575) {
                    //        printf("res in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);

                    //    }

                    //}



                 //if (x==33 && y==1 && z==71) {
                 //    printf("in 33 1 71 TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d column %d looking in %d \n"
                 //        , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
                 //        , column, linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww);
                 //}




                    if (isBitAt(column, zLoc) && column > 0) {


                        printf("in TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d looking in %d    \n"
                                    , xMeta, yMeta, zMeta,x,y,z,  (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
                                , column , linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww, fbArgs.dbYLength);
                    }

                    ww = 1;
                    // uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww];//
                    column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww];//


                    //if (x == 33 && y == 1 && z == 71) {
                    //    printf("in 33 1 71 TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d column %d looking in %d \n"
                    //        , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
                    //        , column, linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww);
                    //}

                    if (isBitAt(column, zLoc) && column > 0) {

                           printf("in TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d looking in %d   \n"
                               , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
                               , column, linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww, fbArgs.dbYLength);
                    }

                }
            }
        }

        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    auto count = fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 1];
        //    if (count > 0) {
        //        printf("in TEST kernel looking fp count  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d counter %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count, fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 3]);
        //    }
        //}
        //if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        //    auto count = fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 2];
        //    if (count > 0) {
        //        printf("in TEST kernel looking fn count   xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d counter %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count, fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 4]);
        //    }
        //}





        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength+ metaData.metaDataOffset + 7];
        //    if (count ==1) {
        //        printf("in TEST kernel looking active gold  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
        //    }
        //}
        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 9];
        //    if (count == 1) {
        //        printf("in TEST kernel looking active segm  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
        //    }
        //}
        ///// testing  calculation of surrounding blocks linear indicies
        // block 1,1,1
        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    if (xMeta==1 && yMeta==1 && zMeta==1) {
        //        printf("linear indicies from metadata  top %d bottom %d left %d right %d anterior %d posterior %d  linIdexMeta current %d \n    "
        //            ,mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 13]
        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 14]

        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 15]
        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 16]

        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 17]
        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 18]
        //            , linIdexMeta
        //        );
        //    }
        //    if (xMeta ==  1&& yMeta == 1 && zMeta == 0) {
        //        printf("linear index top linIdexMeta %d \n    ", linIdexMeta);
        //    
        //    }
        //    if (xMeta ==1 && yMeta == 1 && zMeta == 2) {
        //        printf("linear index bottom linIdexMeta %d \n    ", linIdexMeta);

        //    }
        //    if (xMeta == 1&& yMeta == 2 && zMeta == 1) {
        //        printf("linear index anterior linIdexMeta %d \n    ", linIdexMeta);

        //    }
        //    if (xMeta == 1&& yMeta == 0 && zMeta == 1) {
        //        printf("linear index posterior linIdexMeta %d \n    ", linIdexMeta);

        //    }

        //    if (xMeta ==2 && yMeta == 1 && zMeta == 1) {
        //        printf("linear index right linIdexMeta %d \n    ", linIdexMeta);

        //    }
        //    if (xMeta == 0&& yMeta == 1 && zMeta == 1) {
        //        printf("linear index left linIdexMeta %d \n    ", linIdexMeta);

        //    }

        //}

//// checking weather on edges it shows UINT32_MAX
        //   if ((threadIdx.x == 0) && (threadIdx.y == 0)) {

        //    if (xMeta ==  1&& yMeta == 1 && zMeta == 0) {
        //        printf("linear index top linIdexMeta %d  and max is %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 13], UINT32_MAX);
        //    
        //    }
        //    if (xMeta ==1 && yMeta == 1 && zMeta == 3) {
        //        printf("linear index bottom linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 14]);

        //    }
        //    if (xMeta == 1&& yMeta == 5 && zMeta == 1) {
        //        printf("linear index anterior linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 17]);

        //    }
        //    if (xMeta == 1&& yMeta == 0 && zMeta == 1) {
        //        printf("linear index posterior linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 18]);

        //    }

        //    if (xMeta ==2 && yMeta == 1 && zMeta == 1) {
        //        printf("linear index right linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 16]);

        //    }
        //    if (xMeta == 0&& yMeta == 1 && zMeta == 1) {
        //        printf("linear index left linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 15]);

        //    }

        //}

        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 5];
        //    if (count >0) {
        //        printf("in TEST kernel offset fp  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
        //    }
        //}
        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 6];
        //    if (count > 0) {
        //        printf("in TEST kernel offset fn  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
        //    }
        //}

    }




    //for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < 80; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {


    // /*   if (fbArgs.metaData.resultList[linIdexMeta * 5 + 4] != 131 && fbArgs.metaData.resultList[linIdexMeta * 5] > 0) {

    //        printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d \n ",
    //            fbArgs.metaData.resultList[linIdexMeta * 5]
    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 1]
    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 2]
    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 3]
    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 4]
    //            , linIdexMeta


    //        );
    //    }
    //    else {
    //        printf(" *** ");
    //        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[17]), 1);

    //    }*/
    //}
}





template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {

    //inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs, uint32_t * mainArr, MetaDataGPU metaData
    //    , unsigned int* minMaxes, uint32_t * workQueue
    //    , uint32_t * resultListPointerMeta, uint32_t * resultListPointerLocal, uint32_t * resultListPointerIterNumb, uint32_t * origArrs, uint32_t * metaDataArr) {



    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);
    grid_group grid = cooperative_groups::this_grid();


    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t mainShmem[lengthOfMainShmem];
    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[1];
    //marks wheather there can be any result of intest there
    __shared__ bool isBlockToBeValidated[1];
    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];

    __shared__ uint32_t isGold[1];
    __shared__ uint32_t currLinIndM[1];


    __shared__ uint32_t oldIsGold[1];
    __shared__ uint32_t oldLinIndM[1];

    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[60];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX

 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior

    */


    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) { iterationNumb[0] = -1; };
    if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        isSegmPassToContinue[0] = true;
    };

    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number





    //while (isGoldPassToContinue[0] || isSegmPassToContinue[0]) {



    mainDilatation(false, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
        , fbArgs.workQueuePointer
        , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
        , cta, tile, grid, mainShmem
        , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset,
        globalWorkQueueCounter, localWorkQueueCounter, localTotalLenthOfWorkQueue, localFpConter,
        localFnConter, blockFpConter, blockFnConter, resultfpOffset,
        resultfnOffset, worQueueStep, isGold, currLinIndM, localMinMaxes
        , localBlockMetaData, fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, fbArgs.origArrsPointer
        , fbArgs.metaDataArrPointer, oldIsGold, oldLinIndM,  isGoldForLocQueue, isBlockToBeValidated);





    // grid.sync();

     //  krowa predicates must be lambdas probablu now they will not compute well as we do not have for example linIdexMeta ...
    /////////////// loading work queue for padding dilatations
    metadataPass(fbArgs, true, 11, 7, 8,
        12, 9, 10
        , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
        , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
        , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
        , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);
    //////////// padding dilatations






//     grid.sync();
     ////////////////////////main metadata pass
        //  krowa predicates must be lambdas probablu now they will not compute well as we do not have for example linIdexMeta ...

     //metadataPass(false,(isGoldPassToContinue[0] &&  mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 7]
     //         && !mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 8]),
     //         (isSegmPassToContinue[0] && mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 9]
     //             && !mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 10]),
     //         , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
     //         , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
     //         , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
     //         , mainArr, metaData, minMaxes, workQueue,metaDataArr);
     // 

//  }// end while

  //setting global iteration number to local one 

}



#pragma once
extern "C" inline bool mainKernelsRun(ForFullBoolPrepArgs<int> fFArgs) {

    hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;
    int device = 0;
    unsigned int cpuIterNumb = -1;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<int>,
        0);
    int warpsNumbForMinMax = blockSize / 32;
    int blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<int>,
        0);
    int warpsNumbForboolPrepareKernel = blockSize / 32;
    int blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<int>,
        0);
    int theadsForFirstMetaPass = blockSize;
    int blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)mainPassKernel<int>,
        0);
    int warpsNumbForMainPass = blockSize / 32;
    int blockForMainPass = minGridSize;

    printf("warpsNumbForMainPass %d blockForMainPass %d  ", warpsNumbForMainPass, blockForMainPass);






    //for debugging
    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
    //main arrays allocations
    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);

    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
    //pointers ...
    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;

    uint32_t* origArrsPointer;
    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;
    uint32_t* metaDataArrPointer;

    uint32_t* workQueuePointer;
    unsigned int* minMaxes;
    size_t size = sizeof(unsigned int) * 20;
    hipMalloc(&minMaxes, size);


    checkCuda(hipDeviceSynchronize(), "a0");
    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, minMaxes, warpsNumbForMainPass, blockForMainPass);
    MetaDataGPU metaData = fbArgs.metaData;
    fbArgs.metaData.minMaxes = minMaxes;


    //3086


    ////preparation kernel

    // initialize, then launch

    checkCuda(hipDeviceSynchronize(), "a1");


    getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > (fbArgs, minMaxes);

    checkCuda(hipDeviceSynchronize(), "a1");


    checkCuda(hipDeviceSynchronize(), "a2");

    metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs, workQueuePointer, minMaxes, metaData, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a2");

    boolPrepareKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, metaData, origArrsPointer, metaDataArrPointer);
    //uint32_t* origArrs, uint32_t* metaDataArr     metaDataArr[linIdexMeta * metaData.metaDataSectionLength     metaDataOffset

    checkCuda(hipDeviceSynchronize(), "a3");


    allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb, origArrsPointer, mainArrAPointer, mainArrBPointer, metaData, goldArr, segmArr);

    checkCuda(hipDeviceSynchronize(), "a4");

    firstMetaPrepareKernel << <blockForFirstMetaPass, theadsForFirstMetaPass >> > (fbArgs, metaData, minMaxes, workQueuePointer, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a5");
    //void* kernel_args[] = { &fbArgs, mainArrPointer,&metaData,minMaxes, workQueuePointer,resultListPointerMeta,resultListPointerLocal, resultListPointerIterNumb };
    fbArgs.forDebugArr = forDebug;
    fbArgs.goldArr = goldArr;
    fbArgs.segmArr = segmArr;
    fbArgs.metaData = metaData;

    fbArgs.resultListPointerMeta = resultListPointerMeta;
    fbArgs.resultListPointerLocal = resultListPointerLocal;
    fbArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    fbArgs.origArrsPointer = origArrsPointer;
    fbArgs.mainArrAPointer = mainArrAPointer;
    fbArgs.mainArrBPointer = mainArrBPointer;


    fbArgs.metaDataArrPointer = metaDataArrPointer;
    fbArgs.workQueuePointer = workQueuePointer;
    fbArgs.minMaxes = minMaxes;
    void* kernel_args[] = { &fbArgs };


   // hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), blockForMainPass, dim3(32, warpsNumbForMainPass), kernel_args);
    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), 10, dim3(32, warpsNumbForMainPass), kernel_args);



    checkCuda(hipDeviceSynchronize(), "a6");


    //hipLaunchCooperativeKernel((void*)mainPassKernel<int>, deviceProp.multiProcessorCount, fFArgs.threadsMainPass, fbArgs);



  //  checkCuda(hipDeviceSynchronize(), "cc");




  //  ////mainPassKernel << <fFArgs.blocksMainPass, fFArgs.threadsMainPass >> > (fbArgs);

    testKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, minMaxes, mainArrBPointer, metaData, workQueuePointer, origArrsPointer);

    //  testKernel << <10, 512 >> > (fbArgs, minMaxes);


      ////sync
    checkCuda(hipDeviceSynchronize(), "cc");




    //deviceTohost



    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);


    //copyDeviceToHost3d(goldArr, fFArgs.goldArr);
    //copyDeviceToHost3d(segmArr, fFArgs.segmArr);
    // getting arrays allocated on  cpu to 


    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);

    // printForDebug(fbArgs, fFArgs, resultListPointer, mainArrPointer, workQueuePointer, metaData);


    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
    //hipGetLastError();

    hipFreeAsync(forDebug.arrPStr.ptr, 0);
    //hipFreeAsync(goldArr.arrPStr.ptr, 0);
    //hipFreeAsync(segmArr.arrPStr.ptr, 0);


    hipFreeAsync(resultListPointerMeta, 0);
    hipFreeAsync(resultListPointerLocal, 0);
    hipFreeAsync(resultListPointerIterNumb, 0);
    hipFreeAsync(workQueuePointer, 0);
    hipFreeAsync(origArrsPointer, 0);
    hipFreeAsync(metaDataArrPointer, 0);

    checkCuda(hipDeviceSynchronize(), "last ");

    /*   hipFree(reducedGold.arrPStr.ptr);
       hipFree(reducedSegm.arrPStr.ptr);
       hipFree(reducedGoldPrev.arrPStr.ptr);
       hipFree(reducedSegmPrev.arrPStr.ptr);*/

       //    hipFreeAsync(resultListPointer, 0);

       //    freeMetaDataGPU(fbArgs.metaData);


           /*
        * Catch errors for both the kernel launch above and any
        * errors that occur during the asynchronous `doubleElements`
        * kernel execution.
        */

    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    return true;
}



