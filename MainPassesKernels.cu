#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include "DilatationKernels.cu"
#include "MinMaxesKernel.cu"

using namespace cooperative_groups;


/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/





/*
we need to
Data
- shared memory
    -for uploaded data from reduced arrays
    -for dilatation results
    -for result paddings
0) load data about what metadata blocks should be analyzed from work queue
1) load data from given reduced arr into shared memory
2) perform bit  dilatations in 6 directions
    and save to result to result shared memory - additionally dilatations into its own shared memory
3) given the block is to be validated (in case it is first main pass - all needs to be) we check  if
    - if there is set bit (voxel) in res shmem but not in source shmem
        - we establish is there anything of intrest in the primary given array of other type (so for gold we check segm and for segm gold - but original ones)
        - if so we add this to the result list in a spot we established from offsets of metadata
            - we set metadata's fp and fn result counters - so later we will be able to establish wheather block should be validated at all
            - we also increment local counters of fp and fn - those will be used for later
4) we save data from result shmem into reduced arrays and from paddings into padding store (both in global memory)

*/




/**
CPU part of the loop - where we copy data required to know wheather next loop should be executed and to increment the iteration number
*/
template <typename TKKI>
inline bool runAfterOneLoop(ForBoolKernelArgs<TKKI> gpuArgs, ForFullBoolPrepArgs<TKKI> cpuArgs, unsigned int& cpuIterNumb) {
    cpuIterNumb += 1;

    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    cpuArgs.metaData.minMaxes.arrP[0][0][13] = cpuIterNumb;
    //copy back on gpu
    copyHostToDevice(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    // returning true - so signal that we need to loop on only when we did not reach yet the required percent of covered voxels
    return ((ceil(cpuArgs.metaData.minMaxes.arrP[0][0][7] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][10])
        || (ceil(cpuArgs.metaData.minMaxes.arrP[0][0][8] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][11]));

}

template <typename TKKI>
inline __global__ void testKernel(ForBoolKernelArgs<TKKI> fbArgs) {
    char* tensorslice;
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < 80; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
        if (fbArgs.metaData.resultList[linIdexMeta *5+4] !=131 && fbArgs.metaData.resultList[linIdexMeta * 5 ]>0) {

        printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d \n ",
            fbArgs.metaData.resultList[linIdexMeta * 5 ]
            ,fbArgs.metaData.resultList[linIdexMeta * 5 + 1]
            ,fbArgs.metaData.resultList[linIdexMeta * 5 + 2]
            ,fbArgs.metaData.resultList[linIdexMeta * 5 + 3]
            ,fbArgs.metaData.resultList[linIdexMeta * 5 + 4]
            , linIdexMeta


        );
    }
    else {
        printf(" *** ");
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[17]), 1);

    }
    }
}

/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling 
*/
#pragma once
template <typename ZZR>
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, void* resultListPointer) {
=======
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, void*& resultListPointer) {
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, void*& resultListPointer) {
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, void*& resultListPointer) {
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
inline void allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs, void*& resultListPointer) {
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
   unsigned int fpPlusFn=  cpuArgs.metaData.minMaxes.arrP[0][0][7] + cpuArgs.metaData.minMaxes.arrP[0][0][8];

    size_t size = sizeof(uint16_t)*5*fpPlusFn+1;
    hipMallocAsync(&resultListPointer, size,0);
    gpuArgs.metaData.resultList = resultListPointer;

<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

   // hipFreeAsync(gpuArgs.metaData.resultList, 0);

    //hipFree(resultListPointer);


};
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)




#pragma once
template <typename ZZR>
inline void allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs,  array3dWithDimsGPU reducedGold
   , array3dWithDimsGPU& reducedSegm
    , array3dWithDimsGPU& reducedGoldRef
    , array3dWithDimsGPU& reducedSegmRef
    , array3dWithDimsGPU& reducedGoldPrev
    , array3dWithDimsGPU& reducedSegmPrev) {
    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes.arrP[0][0][1] - cpuArgs.metaData.minMaxes.arrP[0][0][2];
    unsigned int yRange = cpuArgs.metaData.minMaxes.arrP[0][0][3] - cpuArgs.metaData.minMaxes.arrP[0][0][4];
    unsigned int zRange = cpuArgs.metaData.minMaxes.arrP[0][0][5] - cpuArgs.metaData.minMaxes.arrP[0][0][6];

    //allocating needed memory
    reducedGold = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegm = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedGoldRef = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegmRef = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedGoldPrev = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegmPrev = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    allocateMetaDataOnGPU(xRange, yRange, zRange);
    //unsigned int fpPlusFn = fFArgs.metaData.minMaxes.arrP[0][0][7] + fFArgs.metaData.minMaxes.arrP[0][0][8];
    //uint16_t* resultListPointer;
    //size_t size = sizeof(uint16_t) * 5 * fpPlusFn + 1;
    //hipMallocAsync(&resultListPointer, size, 0);
    //fbArgs.metaData.resultList = resultListPointer;


};
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)




#pragma once
template <typename ZZR>
inline void allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR> gpuArgs, ForFullBoolPrepArgs<ZZR> cpuArgs,  array3dWithDimsGPU reducedGold
   , array3dWithDimsGPU& reducedSegm
    , array3dWithDimsGPU& reducedGoldRef
    , array3dWithDimsGPU& reducedSegmRef
    , array3dWithDimsGPU& reducedGoldPrev
    , array3dWithDimsGPU& reducedSegmPrev) {
    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes.arrP[0][0][1] - cpuArgs.metaData.minMaxes.arrP[0][0][2];
    unsigned int yRange = cpuArgs.metaData.minMaxes.arrP[0][0][3] - cpuArgs.metaData.minMaxes.arrP[0][0][4];
    unsigned int zRange = cpuArgs.metaData.minMaxes.arrP[0][0][5] - cpuArgs.metaData.minMaxes.arrP[0][0][6];

    //allocating needed memory
    reducedGold = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegm = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedGoldRef = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegmRef = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedGoldPrev = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    reducedSegmPrev = getArrGpu<uint32_t>(xRange* cpuArgs.dbXLength, yRange* cpuArgs.dbYLength, zRange*cpuArgs.dbZLength);
    allocateMetaDataOnGPU(xRange, yRange, zRange);
    //unsigned int fpPlusFn = fFArgs.metaData.minMaxes.arrP[0][0][7] + fFArgs.metaData.minMaxes.arrP[0][0][8];
    //uint16_t* resultListPointer;
    //size_t size = sizeof(uint16_t) * 5 * fpPlusFn + 1;
    //hipMallocAsync(&resultListPointer, size, 0);
    //fbArgs.metaData.resultList = resultListPointer;


};
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

   // hipFreeAsync(gpuArgs.metaData.resultList, 0);

    //hipFree(resultListPointer);


};


#pragma once
extern "C" inline bool mainKernelsRun(ForFullBoolPrepArgs<int> fFArgs) {


    hipError_t syncErr;
    hipError_t asyncErr;
    int device = 0;
    unsigned int cpuIterNumb = -1;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);



    //for debugging
    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
    //main arrays allocations
    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);

    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
    ////reduced arrays
    array3dWithDimsGPU reducedGold ;
    array3dWithDimsGPU reducedSegm;
<<<<<<< HEAD

    array3dWithDimsGPU reducedGoldRef;
    array3dWithDimsGPU reducedSegmRef ;
<<<<<<< HEAD
<<<<<<< HEAD
=======
=======

    array3dWithDimsGPU reducedGoldRef;
    array3dWithDimsGPU reducedSegmRef ;


    array3dWithDimsGPU reducedGoldPrev ;
    array3dWithDimsGPU reducedSegmPrev;
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======

>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

    array3dWithDimsGPU reducedGoldPrev ;
    array3dWithDimsGPU reducedSegmPrev;

    array3dWithDimsGPU reducedGoldPrev ;
    array3dWithDimsGPU reducedSegmPrev;


<<<<<<< HEAD
=======
    uint16_t* resultListPointer;
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, reducedGoldRef, reducedSegmRef, reducedGoldPrev, reducedSegmPrev);

    ////preparation kernel

    // initialize, then launch


    checkCuda(hipDeviceSynchronize(), "bb");

    void* kernel_args[] = { &fbArgs };
    
    getMinMaxes << <deviceProp.multiProcessorCount, fFArgs.threadsMainPass >> > (fbArgs);

    , reducedGold, reducedSegm, reducedGoldRef, reducedSegmRef, reducedGoldPrev, reducedSegmPrev

<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD

    //hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);

=======
    //hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    //hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)

    unsigned int fpPlusFn = fFArgs.metaData.minMaxes.arrP[0][0][7] + fFArgs.metaData.minMaxes.arrP[0][0][8];
    uint16_t* resultListPointer;
    size_t size = sizeof(uint16_t) * 5 * fpPlusFn + 1;
    hipMallocAsync(&resultListPointer, size, 0);
    fbArgs.metaData.resultList = resultListPointer;

    //allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointer);
=======
    //hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);



<<<<<<< HEAD
<<<<<<< HEAD

    allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointer);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    //hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);




    allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointer);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointer);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointer);
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
    
    //hipLaunchCooperativeKernel((void*)(firstMetaPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threadsFirstMetaDataPass, kernel_args);


    //hipLaunchCooperativeKernel((void*)(firstMetaPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threadsFirstMetaDataPass, kernel_args);

    checkCuda(hipDeviceSynchronize(), "bb");


    //hipLaunchCooperativeKernel((void*)mainPassKernel<int>, deviceProp.multiProcessorCount, fFArgs.threadsMainPass, fbArgs);

  // // for (int i = 0; i < 205; i++) {
  //  while(runAfterOneLoop(fbArgs, fFArgs, cpuIterNumb)){
  //     // runAfterOneLoop(fbArgs, fFArgs, cpuIterNumb);

  //    /*  checkCuda(hipDeviceSynchronize(), "bb");
  //      printf("mainDilatation %d  \n", cpuIterNumb);*/

  //      //hipLaunchCooperativeKernel((void*)(mainDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
  //      mainDilatation << <deviceProp.multiProcessorCount, fFArgs.threadsMainPass >> > (fbArgs);

  //    /*  syncErr = hipGetLastError();
  //      asyncErr = hipDeviceSynchronize();
  //      if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
  //      if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));*/


  //      //hipLaunchCooperativeKernel((void*)(getWorkQueeueFromIsToBeActivated<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
  //      getWorkQueeueFromIsToBeActivated << <deviceProp.multiProcessorCount, fFArgs.threadsMainPass >> > (fbArgs);


  //     /* checkCuda(hipDeviceSynchronize(), "bb");
  //      printf("getWorkQueeueFromIsToBeActivated %d  \n", cpuIterNumb);
  //      syncErr = hipGetLastError();
  //      asyncErr = hipDeviceSynchronize();
  //      if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
  //      if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));*/

  //      paddingDilatation << <deviceProp.multiProcessorCount, fFArgs.threadsMainPass >> > (fbArgs);

  //      //hipLaunchCooperativeKernel((void*)(paddingDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
  //      checkCuda(hipDeviceSynchronize(), "bb");


  //      /*checkCuda(hipDeviceSynchronize(), "bb");
  //      printf("paddingDilatation %d  \n", cpuIterNumb);
  //      syncErr = hipGetLastError();
  //      asyncErr = hipDeviceSynchronize();
  //      if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
  //      if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));*/

  //      //hipLaunchCooperativeKernel((void*)(getWorkQueeueFromActive_mainPass<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
  //      getWorkQueeueFromActive_mainPass << <deviceProp.multiProcessorCount, fFArgs.threadsMainPass >> > (fbArgs);


  ///*      checkCuda(hipDeviceSynchronize(), "bb");
  //      printf("getWorkQueeueFromActive_mainPass %d  \n", cpuIterNumb);
  //      syncErr = hipGetLastError();
  //      asyncErr = hipDeviceSynchronize();
  //      if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
  //      if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));*/
  // }
  //  checkCuda(hipDeviceSynchronize(), "cc");




  //  ////mainPassKernel << <fFArgs.blocksMainPass, fFArgs.threadsMainPass >> > (fbArgs);

  //  testKernel << <10,512>> > (fbArgs);
 


    ////sync
    checkCuda(hipDeviceSynchronize(), "cc");




    //deviceTohost

    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);


    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
    copyDeviceToHost3d(segmArr, fFArgs.segmArr);

    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);

    copyDeviceToHost3d(reducedGoldPrev, fFArgs.reducedGoldPrev);
    copyDeviceToHost3d(reducedSegmPrev, fFArgs.reducedSegmPrev);


    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);



    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
    //hipGetLastError();

    hipFree(forDebug.arrPStr.ptr);
    hipFree(goldArr.arrPStr.ptr);
    hipFree(segmArr.arrPStr.ptr);
    hipFree(reducedGold.arrPStr.ptr);
    hipFree(reducedSegm.arrPStr.ptr);
    hipFree(reducedGoldPrev.arrPStr.ptr);
    hipFree(reducedSegmPrev.arrPStr.ptr);

    hipFreeAsync(resultListPointer, 0);

    freeMetaDataGPU(fbArgs.metaData);


       /*
    * Catch errors for both the kernel launch above and any
    * errors that occur during the asynchronous `doubleElements`
    * kernel execution.
    */

       syncErr = hipGetLastError();
       asyncErr = hipDeviceSynchronize();
       if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
       if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));



    return true;
}













/*

template <typename TKKI>
__global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);

    char* tensorslice;
    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
    unsigned int old = 0;
    uint16_t i = 0;
    uint8_t j = 0;
    uint8_t bigloop = 0;
    uint8_t bitPos = 0;
    // some references using as aliases
    unsigned int& oldRef = old;
    uint16_t& linIdexMeta = i;
    uint8_t& xMeta = j;
    uint8_t& yMeta = bigloop;
    uint8_t& zMeta = bitPos;
    bool& isToBeActivated = isBlockFull;


    // main shared memory spaces
    __shared__ uint32_t sourceShared[32][32];
    __shared__ uint32_t resShared[32][32];
    // holding data about paddings


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];
    //variables needed for all threads
    __shared__ unsigned int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    __shared__ bool isBlockToBeValidated[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];


    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];

    // we will load here multiple entries from workqueue
    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
    //initializations and loading
    auto active = coalesced_threads();
    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number

    if (isToBeExecutedOnActive(active, 3)) {
        localWorkQueueCounter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 4)) {
        isGoldPassToContinue[0] = true;
    };
    if (isToBeExecutedOnActive(active, 5)) {
        isSegmPassToContinue[0] = true;
    };

    if (isToBeExecutedOnActive(active, 6)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 7)) {
        localFnConter[0] = 0;
    };




    if (isToBeExecutedOnActive(active, 1)) {
        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    sync(cta);
    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
    /// load work QueueData into shared memory

    //TODO change looping so it will access contigous memory
    for (bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle
        ///////////// loading to work queue
        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep,j);

        sync(cta);// now local work queue is populated

            //now all of the threads in the block needs to have the same i value so we will increment by 1
        for (i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
                // so we do not need iteration loop

                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                /////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, bitPos, oldRef, blockFpConter, blockFnConter);

                ////on the basis of isAnythingInPadding we will mark  the neighbouring block as to be activated if there is and if such neighbouring block exists
                auto activeC = coalesced_threads();

                if (localWorkQueue[i][3] == 1) {//gold
                    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                };
                if (localWorkQueue[i][3] == 0) {//segm
                    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                };
                // marking blocks as full

                if (localWorkQueue[i][3] == 1) {//gold
                    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                };
                if (localWorkQueue[i][3] == 0) {//segm
                    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                };
                sync(cta);// all results that should be saved to result list are saved

                //we need to clear isAnythingInPadding to 0
                clearisAnythingInPadding(isAnythingInPadding);
            }
        }
    }
    sync(cta);
    //     updating global counters
    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);


    grid.sync();
    auto activeE = coalesced_threads();
    if (isToBeExecutedOnActive(activeE, 0)) {
        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9] = 0;
    };

    grid.sync();
    // checking global count and counters
    checkIsToBeDilatated(fbArgs, tensorslice, isGoldPassToContinue, isSegmPassToContinue);

    sync(cta);




    auto activeO = coalesced_threads();
    //if (isToBeExecutedOnActive(activeO, 0)) {
    //    printf("\n ****************************** \n");
    //};

    ///////// now we need to look through blocks that we just  activated
    for (linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //gold pass

        isToBeActivated = isGoldPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedGold, fbArgs.metaData.isToBeActivatedGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullGold, fbArgs.metaData.isFullGold.Ny, yMeta, zMeta)[xMeta]);

        addToQueueOtherPasses(fbArgs,oldRef, tensorslice, xMeta, yMeta, zMeta ,1  ,  localWorkQueue, localWorkQueueCounter , sourceShared, resShared, isToBeActivated);
        if (isToBeActivated) {
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedGold, fbArgs.metaData.isToBeActivatedGold.Ny, yMeta, zMeta)[xMeta] = false;
        }
        //segmPass
        isToBeActivated = isSegmPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedSegm, fbArgs.metaData.isToBeActivatedSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]  );

            addToQueueOtherPasses(fbArgs, oldRef, tensorslice, xMeta, yMeta, zMeta, 0, localWorkQueue, localWorkQueueCounter, sourceShared, resShared, isToBeActivated);
        if (isToBeActivated) {
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedSegm, fbArgs.metaData.isToBeActivatedSegm.Ny, yMeta, zMeta)[xMeta] = false;

            //printf("\n found to be actvated xMeta %d yMeta %d zMeta %d isGold  %d isSegmPassToContinue[0] %d  isActive %d isFull %d \n ", xMeta, yMeta, zMeta, 0, isSegmPassToContinue[0], getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm
            //    , fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta], getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]);
        }
    }

    sync(cta);
    auto activeF = coalesced_threads();

     if(isToBeExecutedOnActive(activeF, 0)) {
        globalWorkQueueCounter[0] = atomicAdd(&(getTensorRow<int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), (localWorkQueueCounter[0]));
    }

     sync(cta);
     // pushing work queue to global memory
    fromShmemToGlobalWorkQueue(fbArgs, oldRef, i, sourceShared, resShared, localWorkQueue, globalWorkQueueCounter, tensorslice, localWorkQueueCounter);
    grid.sync();



    sync(cta);
    clearShmemBeforeDilatation(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);
    if (isToBeExecutedOnActive(active, 1)) {
        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };

   sync(cta);

    ////// now we do the dilatations and validations of blocks that were just activated

    //TODO change looping so it will access contigous memory
    for (bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle
        ///////////// loading to work queue
        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep,j);

        sync(cta);// now local work queue is populated

            //now all of the threads in the block needs to have the same i value so we will increment by 1
        for (i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {



                //if (isToBeExecutedOnActive(activeJF, 0)) {
                //    printf("\n local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d  i %d workQueLength %d workQueueStep %d globalWorkQueueOffset %d bigloop %d blockIdx.x %d"
                //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3], i
                //        , localTotalLenthOfWorkQueue[0], worQueueStep[0], globalWorkQueueOffset[0], bigloop, blockIdx.x);
                //}


                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
                // so we do not need iteration loop

               // loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                 //   isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                /////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
              //  validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
              //      isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, bitPos, oldRef, blockFpConter, blockFnConter);


    //first we load data to source shmem
                loadDataToShmem(fbArgs, tensorslice, sourceShared, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb), localWorkQueue, i);





            }
        }
    }
    //sync(cta);
    ////we need to clear isAnythingInPadding to 0
    //clearisAnythingInPadding(isAnythingInPadding);
    ////     updating global counters
    //updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);

    //grid.sync();
    //// checking global count and counters
    //checkIsToBeDilatated(fbArgs, tensorslice, isGoldPassToContinue, isSegmPassToContinue);

    //sync(cta);







    ///////// now we need to look through all  blocks - for next dilatation pass ...
    for (linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //gold pass

        isToBeActivated = isGoldPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullGold, fbArgs.metaData.isFullGold.Ny, yMeta, zMeta)[xMeta]);

        addToQueueOtherPasses(fbArgs, oldRef, tensorslice, xMeta, yMeta, zMeta, 1, localWorkQueue, localWorkQueueCounter, sourceShared, resShared, isToBeActivated);

        //segmPass
        isToBeActivated = isSegmPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]);

        addToQueueOtherPasses(fbArgs, oldRef, tensorslice, xMeta, yMeta, zMeta, 0, localWorkQueue, localWorkQueueCounter, sourceShared, resShared, isToBeActivated);

    }

    sync(cta);
    auto activeG = coalesced_threads();

    if (isToBeExecutedOnActive(activeG, 0)) {
        globalWorkQueueCounter[0] = atomicAdd(&(getTensorRow<int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), (localWorkQueueCounter[0]));
    }

    sync(cta);
    // pushing work queue to global memory
    fromShmemToGlobalWorkQueue(fbArgs, oldRef, i, sourceShared, resShared, localWorkQueue, globalWorkQueueCounter, tensorslice, localWorkQueueCounter);




    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf



}


*/


// runAfterOneLoop(fbArgs, fFArgs, cpuIterNumb);// cpu part



//#pragma once
//extern "C" inline bool mainKernelsTestRun(ForFullBoolPrepArgs<int> fFArgs, forTestPointStruct allPointsA[]
//    , forTestMetaDataStruct allMetas[], int pointsNumber, int metasNumber) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//    unsigned int cpuIterNumb = -1;
//    int device = 0;
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, device);
//
//
//    for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    //reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//    array3dWithDimsGPU reducedGoldRef = allocate3dInGPU(fFArgs.reducedGoldRef);
//    array3dWithDimsGPU reducedSegmRef = allocate3dInGPU(fFArgs.reducedSegmRef);
//
//
//    array3dWithDimsGPU reducedGoldPrev = allocate3dInGPU(fFArgs.reducedGoldPrev);
//    array3dWithDimsGPU reducedSegmPrev = allocate3dInGPU(fFArgs.reducedSegmPrev);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, reducedGoldRef, reducedSegmRef, reducedGoldPrev, reducedSegmPrev);
//    void* kernel_args[] = { &fbArgs };
//
//    //preparation kernel
//    hipLaunchCooperativeKernel((void*)(boolPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threads, kernel_args);
//    //sync
//    checkCuda(hipDeviceSynchronize(), "aa");
//
//     bool test
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//    copyDeviceToHost3d(reducedGoldRef, fFArgs.reducedGoldRef);
//    copyDeviceToHost3d(reducedSegmRef, fFArgs.reducedSegmRef);
//    copyDeviceToHost3d(reducedGoldPrev, fFArgs.reducedGoldPrev);
//    copyDeviceToHost3d(reducedSegmPrev, fFArgs.reducedSegmPrev);
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//    checkCuda(hipDeviceSynchronize(), "aa");
//    forBoolKernelTestUnitTests(fFArgs, allPointsA, allMetas, pointsNumber, metasNumber, fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength);
//    checkCuda(hipDeviceSynchronize(), "aa");
//
//       //here threads one dimensionsonal !!
//       //TODO() reallocate memory - make reduced arrs and metadata smaller - allocate work queue, padding store, result list ...
//
//    hipLaunchCooperativeKernel((void*)(firstMetaPrepareKernel<int>), deviceProp.multiProcessorCount, fFArgs.threadsFirstMetaDataPass, kernel_args);
//
//    checkCuda(hipDeviceSynchronize(), "aa");
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//    copyDeviceToHost3d(reducedGoldRef, fFArgs.reducedGoldRef);
//    copyDeviceToHost3d(reducedSegmRef, fFArgs.reducedSegmRef);
//    copyDeviceToHost3d(reducedGoldPrev, fFArgs.reducedGoldPrev);
//    copyDeviceToHost3d(reducedSegmPrev, fFArgs.reducedSegmPrev);
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//    firstMetaPassKernelTestUnitTests(fFArgs, allPointsA, allMetas, pointsNumber, metasNumber, fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength);
//
//
//
//    runAfterOneLoop(fbArgs, fFArgs, cpuIterNumb);// cpu part
//
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(mainDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(getWorkQueeueFromIsToBeActivated<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(paddingDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(getWorkQueeueFromActive_mainPass<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "bb");
//
//    deviceTohost
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//    copyDeviceToHost3d(reducedGoldRef, fFArgs.reducedGoldRef);
//    copyDeviceToHost3d(reducedSegmRef, fFArgs.reducedSegmRef);
//    copyDeviceToHost3d(reducedGoldPrev, fFArgs.reducedGoldPrev);
//    copyDeviceToHost3d(reducedSegmPrev, fFArgs.reducedSegmPrev);
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//    mainPassKernelTestUnitTests(fFArgs, allPointsA, allMetas, pointsNumber, metasNumber
//        , fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength, fFArgs.metaData.MetaZLength, goldArr.Ny, goldArr.Nx);
//
//
//
//    runAfterOneLoop(fbArgs, fFArgs, cpuIterNumb);// cpu part
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(mainDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(getWorkQueeueFromIsToBeActivated<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(paddingDilatation<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//    hipLaunchCooperativeKernel((void*)(getWorkQueeueFromActive_mainPass<int>), deviceProp.multiProcessorCount, fFArgs.threadsMainPass, kernel_args);
//    checkCuda(hipDeviceSynchronize(), "bb");
//
//
//
//    deviceTohost
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//    copyDeviceToHost3d(reducedGoldRef, fFArgs.reducedGoldRef);
//    copyDeviceToHost3d(reducedSegmRef, fFArgs.reducedSegmRef);
//    copyDeviceToHost3d(reducedGoldPrev, fFArgs.reducedGoldPrev);
//    copyDeviceToHost3d(reducedSegmPrev, fFArgs.reducedSegmPrev);
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//    checkCuda(hipDeviceSynchronize(), "bb");
//
//    checkAfterSecondDil(fFArgs, allPointsA, allMetas, pointsNumber, metasNumber, fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength);
//
//
//
//    
//
//
//
//    sync
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//    hipFree(reducedGoldPrev.arrPStr.ptr);
//    hipFree(reducedSegmPrev.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//       /*
//    * Catch errors for both the kernel launch above and any
//    * errors that occur during the asynchronous `doubleElements`
//    * kernel execution.
//    */
//
//       syncErr = hipGetLastError();
//       asyncErr = hipDeviceSynchronize();
//
//       /*
//        * Print errors should they exist.
//        */
//
//       if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//       if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
//

