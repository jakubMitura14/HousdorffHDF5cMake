#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include "DilatationKernels.cu"
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include "BiggerMainFunctions.cu"
#include <cooperative_groups/memcpy_async.h>

using namespace cooperative_groups;


/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/





/*
we need to
Data
- shared memory
    -for uploaded data from reduced arrays
    -for dilatation results
    -for result paddings
0) load data about what metadata blocks should be analyzed from work queue
1) load data from given reduced arr into shared memory
2) perform bit  dilatations in 6 directions
    and save to result to result shared memory - additionally dilatations into its own shared memory
3) given the block is to be validated (in case it is first main pass - all needs to be) we check  if
    - if there is set bit (voxel) in res shmem but not in source shmem
        - we establish is there anything of intrest in the primary given array of other type (so for gold we check segm and for segm gold - but original ones)
        - if so we add this to the result list in a spot we established from offsets of metadata
            - we set metadata's fp and fn result counters - so later we will be able to establish wheather block should be validated at all
            - we also increment local counters of fp and fn - those will be used for later
4) we save data from result shmem into reduced arrays and from paddings into padding store (both in global memory)

*/




/**
CPU part of the loop - where we copy data required to know wheather next loop should be executed and to increment the iteration number
*/
template <typename TKKI>
inline bool runAfterOneLoop(ForBoolKernelArgs<TKKI> gpuArgs, ForFullBoolPrepArgs<TKKI> cpuArgs, unsigned int& cpuIterNumb) {
    cpuIterNumb += 1;

    //copy on cpu
    copyDeviceToHost3d(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    //read an modify
    cpuArgs.metaData.minMaxes.arrP[0][0][13] = cpuIterNumb;
    //copy back on gpu
    copyHostToDevice(gpuArgs.metaData.minMaxes, cpuArgs.metaData.minMaxes);
    // returning true - so signal that we need to loop on only when we did not reach yet the required percent of covered voxels
    return ((ceil(cpuArgs.metaData.minMaxes.arrP[0][0][7] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][10])
        || (ceil(cpuArgs.metaData.minMaxes.arrP[0][0][8] * cpuArgs.robustnessPercent) > cpuArgs.metaData.minMaxes.arrP[0][0][11]));

}








template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {

    //inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs, uint32_t * mainArr, MetaDataGPU metaData
    //    , unsigned int* minMaxes, uint32_t * workQueue
    //    , uint32_t * resultListPointerMeta, uint32_t * resultListPointerLocal, uint32_t * resultListPointerIterNumb, uint32_t * origArrs, uint32_t * metaDataArr) {



    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);
    grid_group grid = cooperative_groups::this_grid();


    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t mainShmem[lengthOfMainShmem];
    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[1];
    //marks wheather there can be any result of intest there
    __shared__ bool isBlockToBeValidated[1];
    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];

    __shared__ uint32_t isGold[1];
    __shared__ uint32_t currLinIndM[1];


    __shared__ uint32_t oldIsGold[1];
    __shared__ uint32_t oldLinIndM[1];

    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[20];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX

 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior

    */

    __shared__ uint32_t localBlockMetaDataOld[20];

    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) { iterationNumb[0] = -1; };
    if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        isSegmPassToContinue[0] = true;
    };

    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number





    //while (isGoldPassToContinue[0] || isSegmPassToContinue[0]) {



    mainDilatation(false, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
        , fbArgs.workQueuePointer
        , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
        , cta, tile, grid, mainShmem
        , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset,
        globalWorkQueueCounter, localWorkQueueCounter, localTotalLenthOfWorkQueue, localFpConter,
        localFnConter, blockFpConter, blockFnConter, resultfpOffset,
        resultfnOffset, worQueueStep, isGold, currLinIndM, localMinMaxes
        , localBlockMetaData, fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, fbArgs.origArrsPointer
        , fbArgs.metaDataArrPointer, oldIsGold, oldLinIndM, localBlockMetaDataOld, isGoldForLocQueue, isBlockToBeValidated);





    // grid.sync();

     //  krowa predicates must be lambdas probablu now they will not compute well as we do not have for example linIdexMeta ...
    /////////////// loading work queue for padding dilatations
    metadataPass(fbArgs, true, 11, 7, 8,
        12, 9, 10
        , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
        , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
        , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
        , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);
    //////////// padding dilatations






//     grid.sync();
     ////////////////////////main metadata pass
        //  krowa predicates must be lambdas probablu now they will not compute well as we do not have for example linIdexMeta ...

     //metadataPass(false,(isGoldPassToContinue[0] &&  mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 7]
     //         && !mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 8]),
     //         (isSegmPassToContinue[0] && mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 9]
     //             && !mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 10]),
     //         , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
     //         , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
     //         , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
     //         , mainArr, metaData, minMaxes, workQueue,metaDataArr);
     // 

//  }// end while

  //setting global iteration number to local one 

}













#pragma once
ForBoolKernelArgs<int> mainKernelsRun(ForFullBoolPrepArgs<int> fFArgs, uint32_t*& reducedResCPU
    , uint32_t*& resultListPointerMetaCPU
    ,uint32_t*& resultListPointerLocalCPU
    ,uint32_t*& resultListPointerIterNumbCPU
    ,uint32_t*& metaDataArrPointerCPU
    ,uint32_t*& workQueuePointerCPU
    ,uint32_t*& origArrsCPU
) {

    hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;
    int device = 0;
    unsigned int cpuIterNumb = -1;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<int>,
        0);
    int warpsNumbForMinMax = blockSize / 32;
    int blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<int>,
        0);
    int warpsNumbForboolPrepareKernel = blockSize / 32;
    int blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<int>,
        0);
    int theadsForFirstMetaPass = blockSize;
    int blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    //hipOccupancyMaxPotentialBlockSize(
    //    &minGridSize,
    //    &blockSize,
    //    (void*)mainPassKernel<int>,
    //    0);
    //int warpsNumbForMainPass = blockSize / 32;
    //int blockForMainPass = minGridSize;
    //    printf("warpsNumbForMainPass %d blockForMainPass %d  ", warpsNumbForMainPass, blockForMainPass);


    int warpsNumbForMainPass = 10;
    int blockForMainPass = 4;







    //for debugging
    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
    //main arrays allocations
    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);

    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
    //pointers ...
    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;

    uint32_t* origArrsPointer;
    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;
    uint32_t* metaDataArrPointer;

    uint32_t* workQueuePointer;
    unsigned int* minMaxes;
    size_t size = sizeof(unsigned int) * 20;
    hipMalloc(&minMaxes, size);


    checkCuda(hipDeviceSynchronize(), "a0");
    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, minMaxes, warpsNumbForMainPass, blockForMainPass);
    MetaDataGPU metaData = fbArgs.metaData;
    fbArgs.metaData.minMaxes = minMaxes;


    //3086


    ////preparation kernel

    // initialize, then launch

    checkCuda(hipDeviceSynchronize(), "a1");


    getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > (fbArgs, minMaxes);

    checkCuda(hipDeviceSynchronize(), "a1");


    checkCuda(hipDeviceSynchronize(), "a2");

    metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs, workQueuePointer, minMaxes, metaData, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a2");

    boolPrepareKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, metaData, origArrsPointer, metaDataArrPointer);
    //uint32_t* origArrs, uint32_t* metaDataArr     metaDataArr[linIdexMeta * metaData.metaDataSectionLength     metaDataOffset

    checkCuda(hipDeviceSynchronize(), "a3");


   int fpPlusFn =  allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb, origArrsPointer, mainArrAPointer, mainArrBPointer, metaData, goldArr, segmArr);

    checkCuda(hipDeviceSynchronize(), "a4");

    firstMetaPrepareKernel << <blockForFirstMetaPass, theadsForFirstMetaPass >> > (fbArgs, metaData, minMaxes, workQueuePointer, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a5");
    //void* kernel_args[] = { &fbArgs, mainArrPointer,&metaData,minMaxes, workQueuePointer,resultListPointerMeta,resultListPointerLocal, resultListPointerIterNumb };
    
    
    
    fbArgs.forDebugArr = forDebug;
    fbArgs.goldArr = goldArr;
    fbArgs.segmArr = segmArr;
    fbArgs.metaData = metaData;

    fbArgs.resultListPointerMeta = resultListPointerMeta;
    fbArgs.resultListPointerLocal = resultListPointerLocal;
    fbArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    fbArgs.origArrsPointer = origArrsPointer;
    fbArgs.mainArrAPointer = mainArrAPointer;
    fbArgs.mainArrBPointer = mainArrBPointer;


    fbArgs.metaDataArrPointer = metaDataArrPointer;
    fbArgs.workQueuePointer = workQueuePointer;
    fbArgs.minMaxes = minMaxes;
    void* kernel_args[] = { &fbArgs };


    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), blockForMainPass, dim3(32, warpsNumbForMainPass), kernel_args);



    checkCuda(hipDeviceSynchronize(), "a6");

    //copy to CPU
    size_t sizeCPU = metaData.totalMetaLength * metaData.mainArrSectionLength * sizeof(uint32_t);
    reducedResCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(reducedResCPU, mainArrBPointer, sizeCPU, hipMemcpyDeviceToHost);


    origArrsCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(origArrsCPU, origArrsPointer, sizeCPU, hipMemcpyDeviceToHost);


    size_t sizeRes = sizeof(uint32_t) * (fpPlusFn + 50);


  resultListPointerMetaCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    resultListPointerLocalCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
   resultListPointerIterNumbCPU= (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
   hipMemcpy(resultListPointerMetaCPU, resultListPointerMeta, sizeRes, hipMemcpyDeviceToHost);
   hipMemcpy(resultListPointerLocalCPU, resultListPointerLocal, sizeRes, hipMemcpyDeviceToHost);
   hipMemcpy(resultListPointerIterNumbCPU, resultListPointerIterNumb, sizeRes, hipMemcpyDeviceToHost);

   size_t sizemetaDataArr = metaData.totalMetaLength * (20) * sizeof(uint32_t);
   metaDataArrPointerCPU = (uint32_t*)calloc(metaData.totalMetaLength * (20), sizeof(uint32_t));
   hipMemcpy(metaDataArrPointerCPU, metaDataArrPointer, sizeRes, hipMemcpyDeviceToHost);

   size_t sizeC = (metaData.totalMetaLength * sizeof(uint32_t));

   workQueuePointerCPU = (uint32_t*)calloc(metaData.totalMetaLength, sizeof(uint32_t));
   hipMemcpy(workQueuePointerCPU, workQueuePointer, sizeC, hipMemcpyDeviceToHost);









    //hipLaunchCooperativeKernel((void*)mainPassKernel<int>, deviceProp.multiProcessorCount, fFArgs.threadsMainPass, fbArgs);




  //  ////mainPassKernel << <fFArgs.blocksMainPass, fFArgs.threadsMainPass >> > (fbArgs);

    //testKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, minMaxes, mainArrBPointer, metaData, workQueuePointer, origArrsPointer);

    //  testKernel << <10, 512 >> > (fbArgs, minMaxes);


      ////sync
    checkCuda(hipDeviceSynchronize(), "cc");




    //deviceTohost



    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);


    //copyDeviceToHost3d(goldArr, fFArgs.goldArr);
    //copyDeviceToHost3d(segmArr, fFArgs.segmArr);
    // getting arrays allocated on  cpu to 


    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);

    // printForDebug(fbArgs, fFArgs, resultListPointer, mainArrPointer, workQueuePointer, metaData);


    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
    //hipGetLastError();

    hipFreeAsync(forDebug.arrPStr.ptr, 0);
    //hipFreeAsync(goldArr.arrPStr.ptr, 0);
    //hipFreeAsync(segmArr.arrPStr.ptr, 0);


    hipFreeAsync(resultListPointerMeta, 0);
    hipFreeAsync(resultListPointerLocal, 0);
    hipFreeAsync(resultListPointerIterNumb, 0);
    hipFreeAsync(workQueuePointer, 0);
    hipFreeAsync(origArrsPointer, 0);
    hipFreeAsync(metaDataArrPointer, 0);

    checkCuda(hipDeviceSynchronize(), "last ");

    /*   hipFree(reducedGold.arrPStr.ptr);
       hipFree(reducedSegm.arrPStr.ptr);
       hipFree(reducedGoldPrev.arrPStr.ptr);
       hipFree(reducedSegmPrev.arrPStr.ptr);*/

       //    hipFreeAsync(resultListPointer, 0);

       //    freeMetaDataGPU(fbArgs.metaData);


           /*
        * Catch errors for both the kernel launch above and any
        * errors that occur during the asynchronous `doubleElements`
        * kernel execution.
        */

    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    return fbArgs;
}













