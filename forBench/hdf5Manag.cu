#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

#include <algorithm>    // std::min
#include <cmath>
#include <math.h>
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <chrono>

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
#include "Volume.h"
#include "HausdorffDistance.cuh"
#include "HausdorffDistance.cu"



using namespace H5;








void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {
    /*
     * Open the specified file and the specified dataset in the file.
     */
    H5File file(FILE_NAME, H5F_ACC_RDONLY);
    DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();
    
    
    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    cout << "Datasize: " << dims[0] << endl; // this is the correct number of values

    // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    DataSpace memspace(1, dimsm);


    
   data = (bool*)calloc(dims[0], sizeof(bool));




    dset.read(data, PredType::NATIVE_HBOOL, memspace, dspace); 


    //int sum = 0;
    //for (int i = 0; i < dims[0]; i++) {
    //    sum += data[i];
    //}
    //printf("suuum %d \n  ", sum);


    file.close();

}



/*
benchmark for original code from  https://github.com/Oyatsumi/HausdorffDistanceComparison
*/
void benchmarkOliviera(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {
    Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    for (int x = 0; x < WIDTH; x++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int z = 0; z < DEPTH; z++) {
                img1.setVoxelValue(onlyLungsBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
                img2.setVoxelValue(onlyBladderBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
            }
        }
    }



    auto begin = std::chrono::high_resolution_clock::now();


    HausdorffDistance* hd = new HausdorffDistance();
    int dist = (*hd).computeDistance(&img1, &img2);



    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;

    printf("HD: %d \n", dist);

    //freeing memory
    img1.dispose(); img2.dispose();

//Datasize: 216530944
//Datasize : 216530944
//Total elapsed time : 2.62191s
//HD : 234

}




void loadHDF() {
    const int WIDTH = 512;
    const int HEIGHT = 512;
    const int DEPTH = 826;

    const H5std_string FILE_NAMEonlyLungsBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlat, DATASET_NAMEonlyLungsBoolFlat, onlyLungsBoolFlat);

    const H5std_string FILE_NAMEonlyBladderBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlat, onlyBladderBoolFlat);







}


