#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

#include <algorithm>    // std::min
#include <cmath>
#include <math.h>
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <chrono>

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
#include "Volume.h"
#include "HausdorffDistance.cuh"
#include "HausdorffDistance.cu"



using namespace H5;








void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {
    /*
     * Open the specified file and the specified dataset in the file.
     */
    H5File file(FILE_NAME, H5F_ACC_RDONLY);
    DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();
    
    
    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    cout << "Datasize: " << dims[0] << endl; // this is the correct number of values

    // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    DataSpace memspace(1, dimsm);


    
   data = (bool*)calloc(dims[0], sizeof(bool));




    dset.read(data, PredType::NATIVE_HBOOL, memspace, dspace); 


    //int sum = 0;
    //for (int i = 0; i < dims[0]; i++) {
    //    sum += data[i];
    //}
    //printf("suuum %d \n  ", sum);


    file.close();

}



/*
benchmark for original code from  https://github.com/Oyatsumi/HausdorffDistanceComparison
*/
void benchmarkOliviera(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {
    Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    for (int x = 0; x < WIDTH; x++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int z = 0; z < DEPTH; z++) {
                img1.setVoxelValue(onlyLungsBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
                img2.setVoxelValue(onlyBladderBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
            }
        }
    }



    auto begin = std::chrono::high_resolution_clock::now();


    HausdorffDistance* hd = new HausdorffDistance();
    int dist = (*hd).computeDistance(&img1, &img2);



    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;

    printf("HD: %d \n", dist);

    //freeing memory
    img1.dispose(); img2.dispose();

//Datasize: 216530944
//Datasize : 216530944
//Total elapsed time : 2.62191s
//HD : 234

}





void benchmarkMitura(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {
    
    //// some preparations and configuring
    MetaDataCPU metaData;
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = true;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(onlyBladderBoolFlat, WIDTH, HEIGHT, DEPTH);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);
    /// for debugging
    uint32_t* resultListPointerMetaCPU;
    uint32_t* resultListPointerLocalCPU;
    uint32_t* resultListPointerIterNumbCPU;
    uint32_t* metaDataArrPointerCPU;
    uint32_t* workQueuePointerCPU;
    uint32_t* reducedResCPU;
    uint32_t* origArrsCPU;


    //function invocation
    auto begin = std::chrono::high_resolution_clock::now();

    ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
        , resultListPointerLocalCPU, resultListPointerIterNumbCPU
        , metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU, WIDTH, HEIGHT, DEPTH
    );
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;


    size_t sizeMinMax = sizeof(unsigned int) * 20;
    hipMemcpy(minMaxesCPU, fbArgs.metaData.minMaxes, sizeMinMax, hipMemcpyDeviceToHost);

    printf("HD: %d \n", minMaxesCPU[13]);


    // freeee
    free(onlyBladderBoolFlat);
    free(onlyLungsBoolFlat);


    free(resultListPointerMetaCPU);
    free(resultListPointerLocalCPU);
    free(resultListPointerIterNumbCPU);
    free(metaDataArrPointerCPU);
    free(workQueuePointerCPU);

    free(reducedResCPU);
    free(origArrsCPU);

}



void loadHDF() {
    const int WIDTH = 512;
    const int HEIGHT = 512;
    const int DEPTH = 826;

    const H5std_string FILE_NAMEonlyLungsBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlat, DATASET_NAMEonlyLungsBoolFlat, onlyLungsBoolFlat);

    const H5std_string FILE_NAMEonlyBladderBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlat, onlyBladderBoolFlat);

  //  benchmarkOliviera(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);
    benchmarkMitura(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);





}


