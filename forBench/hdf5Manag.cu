#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

#include <algorithm>    // std::min
#include <cmath>
#include <math.h>
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <chrono>

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
#include "Volume.h"
#include "HausdorffDistance.cuh"
#include "HausdorffDistance.cu"



using namespace H5;








void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {
    /*
     * Open the specified file and the specified dataset in the file.
     */
    H5File file(FILE_NAME, H5F_ACC_RDONLY);
    DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();
    
    
    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    cout << "Datasize: " << dims[0] << endl; // this is the correct number of values

    // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    DataSpace memspace(1, dimsm);


    
   data = (bool*)calloc(dims[0], sizeof(bool));




    dset.read(data, PredType::NATIVE_HBOOL, memspace, dspace); 


    int sum = 0;
    for (int i = 0; i < dims[0]; i++) {
        sum += data[i];
    }
    printf("suuum %d \n  ", sum);


    file.close();

}



void loadHDF() {

    const H5std_string FILE_NAME("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAME("onlyLungsBoolFlat");
    // create a vector the same size as the dataset
    bool* data;
    loadHDFIntoBoolArr(FILE_NAME, DATASET_NAME, data);
}