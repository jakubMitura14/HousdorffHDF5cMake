#include "hip/hip_runtime.h"
#include <cstdint>

#pragma once
constexpr auto localWorkQueLength = 32;
constexpr auto localWorkQueLengthDiv32 = 1;
// includes localWorkQueLength and source and res shmem
constexpr auto totalCombinedShmemWorkQueue = (8 * 32) + localWorkQueLength;

/**
In order to be able to use cuda malloc 3d we will implemnt it as a series
of 3d arrays
*/



#pragma once
template <typename TFPP>
struct array3dWithDimsCPU {
    TFPP*** arrP;
    int Nx;
    int Ny;
    int Nz;
};


#pragma once
struct array3dWithDimsGPU {
    hipPitchedPtr arrPStr;
    int Nx;
    int Ny;
    int Nz;
};


#pragma once
extern "C" struct MetaDataCPU {
    int metaXLength;
    int MetaYLength;
    int MetaZLength;
    int totalMetaLength;


    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ - minimal and maximum coordinates of blocks with some entries of intrest
    //7)global FP count; 8)global FN count  9) workQueueCounter 10)resultFP globalCounter 11) resultFn globalCounter 
     //12) global FPandFn offset 13)globalIterationNumb
    //array3dWithDimsCPU<unsigned int> minMaxes;
    unsigned int* minMaxes;

    ////// counts of false positive and false negatives in given metadata blocks

    ///// sizes of array below will be established on the basis of fp and fn values known after boolKernel finished execution

    //work queue -  workqueue counter already present in minMaxes as entry 9 
    uint32_t* workQueue;
    //in practice it is matrix of length the same as FP+FN global count +1 and width of 5
         //1) xMeta; 2)yMeta 3)zMeta 4)isGold 5)iteration number  
    //we use one single long rewsult list - in order to avoid overwriting each block each block has established offset where it would write it's results 
    uint32_t* resultList;

};

#pragma once
extern "C" struct MetaDataGPU {
    int metaXLength;
    int MetaYLength;
    int MetaZLength;
    int totalMetaLength;

    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ - minimal and maximum coordinates of blocks with some entries of intrest
    //7)global FP count; 8)global FN count 9) workQueueCounter 10)resultFP globalCounter 11) resultFn globalCounter
    //12) global FPandFn offset 13)globalIterationNumb

    unsigned int* minMaxes;

    uint32_t* workQueue;
    uint32_t* resultList;

    //represents x from description of main Arr
    unsigned int mainArrXLength;
    //have length 6x+18
    unsigned int mainArrSectionLength;
    //have length 6x 
    unsigned int metaDataOffset;
    // now we will store here also calculated by min maxes kernel values of minimum and maximumvalues 
        //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ 
    unsigned int maxX;
    unsigned int minX;
    unsigned int maxY;
    unsigned int minY;
    unsigned int maxZ;
    unsigned int minZ;
};


/*
* Basically holding the arguments for master functions controlling full preparation to get all for Housedorff kernel
*/
#pragma once
template <typename TFF>
struct ForFullBoolPrepArgs {



    //metadata struct
    MetaDataCPU metaData;
    //pointer to the array used to debug
    array3dWithDimsCPU<int> forDebugArr;
    // dimensions of data block
    int dbXLength;
    int dbYLength;
    int dbZLength;
    // gold standard and segmentation output array
    array3dWithDimsCPU<TFF> goldArr;
    array3dWithDimsCPU<TFF> segmArr;
    TFF numberToLookFor;// what we will look for in arrays

    //number and dimensionality of threads and blocks required to lounch bool kernel
    dim3 threads;
    int blocks;
    //threads and blocks for first metadata pass kernel
    int threadsFirstMetaDataPass;
    int blocksFirstMetaDataPass;
    //threads and blocks for main pass 
    dim3 threadsMainPass;
    int blocksMainPass;
    //threads and blocks for padding pass 
    dim3 threadsPaddingPass;
    int blocksPaddingPass;
    //threads and blocks for non first metadata passes 
    int threadsOtherMetaDataPasses;
    int blocksOtherMetaDataPasses;
    // will establish how many points we want to include in dilatation and how many we can ignore so typically set to 95% - so we will ignore only 5% most distant
    float robustnessPercent = 0.95;

};





/*
* Basically holding the arguments for main kernel in the FullBoolPrep
*/
#pragma once
template <typename TFB>
struct ForBoolKernelArgs {
    //matadata struct
    MetaDataGPU metaData;
    //pointer to the array used to debug
    array3dWithDimsGPU forDebugArr;

    // dimensions of data block
    int dbXLength;
    int dbYLength;
    int dbZLength;
    // gold standard and segmentation output array
    array3dWithDimsGPU goldArr;
    array3dWithDimsGPU segmArr;
    TFB numberToLookFor;


    /*
main array with all required data  organized in sections for each metadata block
x-  is block dimx times block dimy
now what occupies what positions
0-x : reducedGold
(x+1) - 2x : reducedSegm
(2x+1) - 3x : reducedGoldRef
(3x+1) - 4x : reducedSegmRef
(4x+1) - 5x : reducedGoldPrev
(5x+1) - 6x : reducedSegmPrev
6x+1 :fpCount
6x+2 :fnCount
6x+3 :fpCounter
6x+4 :fnCounter
6x+5 :fpOffset
6x+6 :fnOffset
6x+7 :isActiveGold
6x+8 :isFullGold
6x+9 :isActiveSegm
6x+10 :isFullSegm
6x+11 :isToBeActivatedGold
6x+12 :isToBeActivatedSegm
6x+12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
6x+13 : top
6x+14 : bottom
6x+15 : left
6x+16 : right
6x+17 : anterior
6x+18 : posterior
*/
    uint32_t* mainArr;





    float robustnessPercent = 0.95;

};





//just utility for unit testing - set some data bout points
#pragma once
extern "C"  struct forTestPointStruct {
    int x;
    int y;
    int z;

    bool isGold;
    bool isGoldAndSegm;

    int xMeta;
    int yMeta;
    int zMeta;



    bool shouldBeInResAfterOneDil;
    bool shouldBeInResAfterTwoDil;


};


#pragma once
extern "C" struct forTestMetaDataStruct {

    int xMeta;
    int yMeta;
    int zMeta;

    int requiredspaceInFpResultList;
    int requiredspaceInFnResultList;

    bool isToBeActiveAtStart;
    bool isToBeActiveAfterOneIter;
    bool isToBeActiveAfterTwoIter;

    bool isToBeValidatedFpAfterOneIter;
    bool isToBeValidatedFpAfterTwoIter;

    bool isToBeValidatedFnAfterOneIter;
    bool isToBeValidatedFnAfterTwoIter;


    bool isToBeFullAfterOneIter;
    bool isToBeFullAfterTwoIter;

    int fpCount;
    int fnCount;

    int fpConterAfterOneDil;
    int fpConterAfterTwoDil;

    int fnConterAfterOneDil;
    int fnConterAfterTwoDil;


};