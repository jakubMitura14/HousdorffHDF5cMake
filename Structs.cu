#include "hip/hip_runtime.h"
#include <cstdint>

#pragma once
constexpr auto localWorkQueLength = 32;
constexpr auto localWorkQueLengthDiv32 = 1;
// includes localWorkQueLength and source and res shmem
constexpr auto totalCombinedShmemWorkQueue = (8 * 32) + localWorkQueLength;

/**
In order to be able to use cuda malloc 3d we will implemnt it as a series
of 3d arrays
*/



#pragma once
template <typename TFPP>
struct array3dWithDimsCPU {
    TFPP*** arrP;
    int Nx;
    int Ny;
    int Nz;
};


#pragma once
struct array3dWithDimsGPU {
    hipPitchedPtr arrPStr;
    int Nx;
    int Ny;
    int Nz;
};


#pragma once
extern "C" struct MetaDataCPU {
    int metaXLength;
    int MetaYLength;
    int MetaZLength;
    int totalMetaLength;


    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ - minimal and maximum coordinates of blocks with some entries of intrest
    //7)global FP count; 8)global FN count  9) workQueueCounter 10)resultFP globalCounter 11) resultFn globalCounter 
     //12) global FPandFn offset 13)globalIterationNumb
    array3dWithDimsCPU<unsigned int> minMaxes;
    ////// counts of false positive and false negatives in given metadata blocks

    array3dWithDimsCPU<unsigned int> fpCount;
    array3dWithDimsCPU<unsigned int> fnCount;
    //variables needed to add result to correct spot and keep information about it
    //counts how many fps or fns had been already covered in this data block
    array3dWithDimsCPU<unsigned int> fpCounter;
    array3dWithDimsCPU<unsigned int> fnCounter;
    //tells  what is the offset in result list where space for this data block is given
    array3dWithDimsCPU<unsigned int> fpOffset;
    array3dWithDimsCPU<unsigned int> fnOffset;

    // variables neded to establish is block should be put into workqueue
    array3dWithDimsCPU<bool> isActiveGold;
    array3dWithDimsCPU<bool> isFullGold;

    array3dWithDimsCPU<bool> isActiveSegm;
    array3dWithDimsCPU<bool> isFullSegm;

    array3dWithDimsCPU<bool> isToBeActivatedGold;
    array3dWithDimsCPU<bool> isToBeActivatedSegm;


    array3dWithDimsCPU<bool> isToBeValidatedFp;
    array3dWithDimsCPU<bool> isToBeValidatedFn;

    ///// sizes of array below will be established on the basis of fp and fn values known after boolKernel finished execution

    //work queue -  workqueue counter already present in minMaxes as entry 9 
    //in practice it is matrix of length the same as FP+FN global count +1 and width of 4 
        //1) xMeta; 2)yMeta 3)zMeta 4)isGold
    array3dWithDimsCPU<uint16_t> workQueue;
    //in practice it is matrix of length the same as FP+FN global count +1 and width of 5
         //1) xMeta; 2)yMeta 3)zMeta 4)isGold 5)iteration number  
    //we use one single long rewsult list - in order to avoid overwriting each block each block has established offset where it would write it's results 
    uint16_t* resultList;

};

#pragma once
extern "C" struct MetaDataGPU {
    int metaXLength;
    int MetaYLength;
    int MetaZLength;
    int totalMetaLength;

    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ - minimal and maximum coordinates of blocks with some entries of intrest
    //7)global FP count; 8)global FN count 9) workQueueCounter 10)resultFP globalCounter 11) resultFn globalCounter
    //12) global FPandFn offset 13)globalIterationNumb

    array3dWithDimsGPU minMaxes;

    array3dWithDimsGPU fpCount;
    array3dWithDimsGPU fnCount;

    array3dWithDimsGPU fpCounter;
    array3dWithDimsGPU fnCounter;

    array3dWithDimsGPU fpOffset;
    array3dWithDimsGPU fnOffset;


    array3dWithDimsGPU isActiveGold;
    array3dWithDimsGPU isFullGold;
    array3dWithDimsGPU isActiveSegm;
    array3dWithDimsGPU isFullSegm;

    array3dWithDimsGPU isToBeActivatedGold;
    array3dWithDimsGPU isToBeActivatedSegm;


    array3dWithDimsGPU isToBeValidatedFp;
    array3dWithDimsGPU isToBeValidatedFn;

    array3dWithDimsGPU workQueue;
    uint16_t* resultList;


};


/*
* Basically holding the arguments for master functions controlling full preparation to get all for Housedorff kernel
*/
#pragma once
template <typename TFF>
struct ForFullBoolPrepArgs {
    //pointer to reduced arrays holders will be used for dilatation
    array3dWithDimsCPU<uint32_t> reducedGold;
    array3dWithDimsCPU<uint32_t> reducedSegm;
    //will be used as reference - will not be dilatated
    array3dWithDimsCPU<uint32_t> reducedGoldRef;
    array3dWithDimsCPU<uint32_t> reducedSegmRef;
    // space in global memory where one can store padding information
    array3dWithDimsCPU<uint32_t> reducedGoldPrev;
    array3dWithDimsCPU<uint32_t> reducedSegmPrev;
    int reducedArrsZdim;// x and y dimensions are like normal arrays but z dimension gets reduced
    //metadata struct
    MetaDataCPU metaData;
    //pointer to the array used to debug
    array3dWithDimsCPU<int> forDebugArr;
    // dimensions of data block
    int dbXLength;
    int dbYLength;
    int dbZLength;
    // gold standard and segmentation output array
    array3dWithDimsCPU<TFF> goldArr;
    array3dWithDimsCPU<TFF> segmArr;
    TFF numberToLookFor;// what we will look for in arrays

    //number and dimensionality of threads and blocks required to lounch bool kernel
    dim3 threads;
    int blocks;
    //threads and blocks for first metadata pass kernel
    int threadsFirstMetaDataPass;
    int blocksFirstMetaDataPass;
    //threads and blocks for main pass 
    dim3 threadsMainPass;
    int blocksMainPass;
    //threads and blocks for padding pass 
    dim3 threadsPaddingPass;
    int blocksPaddingPass;
    //threads and blocks for non first metadata passes 
    int threadsOtherMetaDataPasses;
    int blocksOtherMetaDataPasses;
    // will establish how many points we want to include in dilatation and how many we can ignore so typically set to 95% - so we will ignore only 5% most distant
    float robustnessPercent = 0.95;

};





/*
* Basically holding the arguments for main kernel in the FullBoolPrep
*/
#pragma once
template <typename TFB>
struct ForBoolKernelArgs {
    //matadata struct
    MetaDataGPU metaData;
    //pointer to the array used to debug
    array3dWithDimsGPU forDebugArr;

    // dimensions of data block
    int dbXLength;
    int dbYLength;
    int dbZLength;
    // gold standard and segmentation output array
    array3dWithDimsGPU goldArr;
    array3dWithDimsGPU segmArr;
    TFB numberToLookFor;

    //pointer to reduced arrays holders
    array3dWithDimsGPU reducedGold;
    array3dWithDimsGPU reducedSegm;

    array3dWithDimsGPU reducedGoldRef;
    array3dWithDimsGPU reducedSegmRef;

    array3dWithDimsGPU reducedGoldPrev;
    array3dWithDimsGPU reducedSegmPrev;
    float robustnessPercent = 0.95;

};





//just utility for unit testing - set some data bout points
#pragma once
extern "C"  struct forTestPointStruct {
    int x;
    int y;
    int z;

    bool isGold;
    bool isGoldAndSegm;

    int xMeta;
    int yMeta;
    int zMeta;



    bool shouldBeInResAfterOneDil;
    bool shouldBeInResAfterTwoDil;


};


#pragma once
extern "C" struct forTestMetaDataStruct {

    int xMeta;
    int yMeta;
    int zMeta;

    int requiredspaceInFpResultList;
    int requiredspaceInFnResultList;

    bool isToBeActiveAtStart;
    bool isToBeActiveAfterOneIter;
    bool isToBeActiveAfterTwoIter;

    bool isToBeValidatedFpAfterOneIter;
    bool isToBeValidatedFpAfterTwoIter;

    bool isToBeValidatedFnAfterOneIter;
    bool isToBeValidatedFnAfterTwoIter;


    bool isToBeFullAfterOneIter;
    bool isToBeFullAfterTwoIter;

    int fpCount;
    int fnCount;

    int fpConterAfterOneDil;
    int fpConterAfterTwoDil;

    int fnConterAfterOneDil;
    int fnConterAfterTwoDil;


};