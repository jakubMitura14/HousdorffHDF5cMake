#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "Structs.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"

using namespace cooperative_groups;

/*
  First we need to be sure that we start from global workqueue to be 0 but it is hard to  get without additional grid sync 
  so we will use 2 counters for odd and even iteration number and here below we will zero the old one on one thread of first thread block
  
  We need to populate the worqueue
   We need to get count of the total FP, FN so we will know wheather we should start loop anew 
*/




inline __device__ bool getPredGoldPass(const bool isPaddingPass
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , MetaDataGPU& metaData
   , uint32_t*& metaDataArr, uint32_t& linIdexMeta

){
    if (isPaddingPass) {


        return (isGoldPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8]);


    }
    else {
        return (isGoldPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8]);


    }

}


inline __device__ bool getPredSegmPass(const bool isPaddingPass
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , MetaDataGPU& metaData
   , uint32_t*& metaDataArr, uint32_t& linIdexMeta

) {
    if (isPaddingPass) {
        return (isSegmPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10]);

    }
    else {
        return (isSegmPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10]);
    }


}



/*
as we have limited space in work queue we will use also the resShmem and source shmem in order to keep calculations easy 
we will divide all shared memory in  blocks of 32 length what will enable us using fast shift operators 
- then on the basis of the result spot we will deide to which shared memory array to put the data locally
0) we are supplied with a spot obtain from atomic addition to local counter where we want to put our data
1) we divide by shifting 5 times so we will know to which shared memory space to put our data we will need to use if operators
2) using sutractions and getting remainder will give us spot in 32 subblock where to put the data 
https://stackoverflow.com/questions/13548172/bitshifts-to-obtain-remainder
*/




#pragma once
template <typename TKKI>
inline __device__ void metadataPass(ForBoolKernelArgs<TKKI> fbArgs, const bool isPaddingPass
    , const uint8_t predicateAa, const uint8_t predicateAb, const uint8_t predicateAc
    , const uint8_t predicateBa, const uint8_t predicateBb, const uint8_t predicateBc
    ,uint32_t (&mainShmem)[lengthOfMainShmem], unsigned int(&globalWorkQueueOffset)[1], unsigned int(&globalWorkQueueCounter)[1]
    , unsigned int(&localWorkQueueCounter)[1], unsigned int(&localTotalLenthOfWorkQueue)[1], unsigned int(&localMinMaxes)[5]
    , unsigned int(&fpFnLocCounter)[1], bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , thread_block& cta, thread_block_tile<32>& tile
    , MetaDataGPU& metaData
    , unsigned int*& minMaxes, uint32_t*& workQueue, uint32_t*& metaDataArr

) {
  // preparation loads
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    fpFnLocCounter[0] = 0;
}
if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
    //printf(" workCounter at start %d ", minMaxes[9] );

}

/*
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
     */
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { 

  
    isGoldPassToContinue[0] 
= (  (minMaxes[7] * fbArgs.robustnessPercent) > minMaxes[10]); 



    //if (blockIdx.x == 0) {

    //    printf("in meta pass fp count %d  ceiled %f fp counter %d isTo be continued %d \n "
    //        , minMaxes[7]
    //        , minMaxes[7] * fbArgs.robustnessPercent
    //        , minMaxes[10]
    //        , isGoldPassToContinue[0]
    //    );
    //}

};

if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { 

    isSegmPassToContinue[0] 
        = ((minMaxes[8] * fbArgs.robustnessPercent) > minMaxes[11]); 
   
    //if (blockIdx.x == 0) {

    //    printf("in meta pass fn count %d  ceiled %f fn counter %d isTo be continued %d \n "
    //        , minMaxes[8]
    //        , minMaxes[8] * fbArgs.robustnessPercent
    //        , minMaxes[11]
    //        , isSegmPassToContinue[0]
    //    );
    //}

};




sync(cta);

//iterations 
for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta <= metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
    
    //if (metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11]) {
    //    printf("in meta pass gold  linIdexMeta %d to be activated  1  isActiveGold %d  isFullGold %d \n"
    //        , linIdexMeta
    //        , metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7]
    //    , metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8]);

    //
    //}
    //
    //
    //if (metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12] ) {
    //    printf("in meta pass segm  linIdexMeta %d to be activated  1  isActive %d  isFull %d \n"
    //        , linIdexMeta
    //        , metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9]
    //        , metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10]);

    //}


    //goldpass
    if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue    , metaData, metaDataArr, linIdexMeta)) {

        //printf("in meta pass gold linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);

        auto old = atomicAdd_block(&localWorkQueueCounter[0], 1) ;
        if (old < lengthOfMainShmem) {
            mainShmem[old] = linIdexMeta + (isGoldOffset);
        }
        else {
            old = atomicAdd(&(minMaxes[9]), 1);
            workQueue[old] = linIdexMeta + (isGoldOffset) ;
        }
        if (isPaddingPass) {
            //setting to be activated to 0 
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11] = 0;
            //setting active to 1
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7] = 1;
        }
    }
    //segm pass
    if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue  , metaData, metaDataArr, linIdexMeta)) {

        //printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);

        auto old = atomicAdd_block(&localWorkQueueCounter[0], 1);
        if (old < lengthOfMainShmem) {
            mainShmem[old] = linIdexMeta;
        }
        else {
            old = atomicAdd(&(minMaxes[9]), 1);
            workQueue[old] = linIdexMeta;
        }
        if (isPaddingPass) {
            //setting to be activated to 0 
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12] = 0;
            //setting active to 1
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9] = 1;
        }
    }

}
//getting begining where we would copy local queue to global one 
sync(cta);
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    if (localWorkQueueCounter[0] > 0) {
        //printf("local work Counter in meta pass %d  \n"
        //    , localWorkQueueCounter[0]
        //    );
        globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
    }
}
sync(cta);
for (uint32_t linI =threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y ) {
  workQueue[globalWorkQueueCounter[0]+linI]=mainShmem[linI];
}

//cooperative_groups::memcpy_async(cta, (&workQueue[globalWorkQueueCounter[0]]), (&mainArr[0]), (sizeof(uint32_t) * localWorkQueueCounter[0]));
}




