#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "Structs.cu"
 
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"

using namespace cooperative_groups;

/*
  First we need to be sure that we start from global workqueue to be 0 but it is hard to  get without additional grid sync 
  so we will use 2 counters for odd and even iteration number and here below we will zero the old one on one thread of first thread block
  
  We need to populate the worqueue
   We need to get count of the total FP, FN so we will know wheather we should start loop anew 
*/




inline __device__ bool getPredGoldPass(const bool isPaddingPass
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , MetaDataGPU& metaData
   , uint32_t*& metaDataArr, uint32_t& linIdexMeta

){
    if (isPaddingPass) {


        return (isGoldPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8]);


    }
    else {
        return (isGoldPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 8]);


    }

}


inline __device__ bool getPredSegmPass(const bool isPaddingPass
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , MetaDataGPU& metaData
   , uint32_t*& metaDataArr, uint32_t& linIdexMeta

) {
    if (isPaddingPass) {
        return (isSegmPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10]);

    }
    else {
        return (isSegmPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9]
            && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 10]);
    }


}


inline __device__ void modifyMetaDataInPaddingPass(const bool isPaddingPass, uint32_t*& metaDataArr, uint32_t& linIdexMeta, MetaDataGPU& metaData, const uint8_t toBeActivated, const uint8_t isActiveNumb) {
    if (isPaddingPass) {
        //setting to be activated to 0 
        metaDataArr[linIdexMeta * metaData.metaDataSectionLength + toBeActivated] = 0;
        //setting active to 1
        metaDataArr[linIdexMeta * metaData.metaDataSectionLength + isActiveNumb] = 1;
    }
}


inline __device__ void saveWorkQueueToGlobal(thread_block& cta, thread_block_tile<32>& tile, unsigned int(&localWorkQueueCounter)[1]
    , unsigned int*& minMaxes, uint32_t*& workQueue, unsigned int(&globalWorkQueueCounter)[1]
    , uint32_t(&mainShmem)[lengthOfMainShmem]) {
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        if (localWorkQueueCounter[0] > 0) {
            //printf("local work Counter in meta pass %d  \n"
            //    , localWorkQueueCounter[0]
            //    );
            globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
        }
    }
    __syncthreads();
    for (uint32_t linI = threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y) {
        workQueue[globalWorkQueueCounter[0] + linI] = mainShmem[linI];
    }
}



/*
as we have limited space in work queue we will use also the resShmem and source shmem in order to keep calculations easy 
we will divide all shared memory in  blocks of 32 length what will enable us using fast shift operators 
- then on the basis of the result spot we will deide to which shared memory array to put the data locally
0) we are supplied with a spot obtain from atomic addition to local counter where we want to put our data
1) we divide by shifting 5 times so we will know to which shared memory space to put our data we will need to use if operators
2) using sutractions and getting remainder will give us spot in 32 subblock where to put the data 
https://stackoverflow.com/questions/13548172/bitshifts-to-obtain-remainder
*/

#pragma once
template <typename TKKI>
inline __device__ void metadataPass(ForBoolKernelArgs<TKKI> fbArgs, const bool isPaddingPass
    , const uint8_t predicateAa, const uint8_t predicateAb, const uint8_t predicateAc
    , const uint8_t predicateBa, const uint8_t predicateBb, const uint8_t predicateBc
    ,uint32_t (&mainShmem)[lengthOfMainShmem], unsigned int(&globalWorkQueueOffset)[1], unsigned int(&globalWorkQueueCounter)[1]
    , unsigned int(&localWorkQueueCounter)[1], unsigned int(&localTotalLenthOfWorkQueue)[1], unsigned int(&localMinMaxes)[5]
    , unsigned int(&fpFnLocCounter)[1], bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , thread_block& cta, thread_block_tile<32>& tile
    , MetaDataGPU& metaData
    , unsigned int*& minMaxes, uint32_t*& workQueue, uint32_t*& metaDataArr

) {
  // preparation loads
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    fpFnLocCounter[0] = 0;
}
if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
    //printf(" workCounter at start %d ", minMaxes[9] );

}

/*
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
     */
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { 

  
    isGoldPassToContinue[0] 
= (  (minMaxes[7] * fbArgs.robustnessPercent) > minMaxes[10]); 



};

if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { 

    isSegmPassToContinue[0] 
        = ((minMaxes[8] * fbArgs.robustnessPercent) > minMaxes[11]); 


};






//for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x
//        ; linIdexMeta <= fbArgs.metaData.totalMetaLength  // we add in order to 
//        ; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
//    //TODO() consider doing it warp centric way  
//    //  we need to be sure that amount of blocks in local work queue do not exceed lengthOfMainShmem probably the most optimal would be to divide work queue to sections where each warp would be responsible for 
//    //  then if number in warp queue will exceed the size of available shared memory it will write it to global memory ... this way we will avoid  thread divergence and keep local work queue in available shared memory space
//    //if (localWorkQueueCounter[0] < (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//      //goldpass
//    if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
//    }
//    //segm pass
//    if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
//    }
//    //sync(cta);
//    //if (localWorkQueueCounter[0] > (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//    //   saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
//    //}
//    //sync(cta);
//    //localWorkQueueCounter[0] = 0;
//    //sync(cta);
//
//
//};




__syncthreads();
 



for (uint8_t outer = 0; outer <= ceilf(fbArgs.metaData.totalMetaLength / (blockDim.x * blockDim.y * gridDim.x)); outer++) {
    uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x + (blockDim.x * blockDim.y * gridDim.x) * outer;
    bool isResFound = false;
    if (linIdexMeta <= fbArgs.metaData.totalMetaLength) {

        //TODO() consider doing it warp centric way  
    //  we need to be sure that amount of blocks in local work queue do not exceed lengthOfMainShmem probably the most optimal would be to divide work queue to sections where each warp would be responsible for 
    //  then if number in warp queue will exceed the size of available shared memory it will write it to global memory ... this way we will avoid  thread divergence and keep local work queue in available shared memory space
    //if (localWorkQueueCounter[0] < (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
      //goldpass
        if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
            //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
            // localWorkQueueCounter[0] += 1;
            isResFound = true;
            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
           // mainShmem[atomicAdd(localWorkQueueCounter, 1)] = linIdexMeta + (isGoldOffset);

            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
        }

    }




}


__syncthreads();
if (localWorkQueueCounter[0]>0) {
    saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
}
__syncthreads();
localWorkQueueCounter[0] = 0;
__syncthreads();

for (uint8_t outer = 0; outer <= ceilf(fbArgs.metaData.totalMetaLength / (blockDim.x * blockDim.y * gridDim.x)); outer++) {
    uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x + (blockDim.x * blockDim.y * gridDim.x) * outer;

    if (linIdexMeta <= fbArgs.metaData.totalMetaLength) {

        //segm pass
        if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
            //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
           // localWorkQueueCounter[0] += 1;
            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
            //mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
           // mainShmem[atomicAdd(localWorkQueueCounter, 1)] = linIdexMeta;


            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
        }
    }
 
}


sync(cta);

if (localWorkQueueCounter[0] > 0) {
    saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
}





//
//
////getting begining where we would copy local queue to global one 
//sync(cta);
//if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
//    if (localWorkQueueCounter[0] > 0) {
//        //printf("local work Counter in meta pass %d  \n"
//        //    , localWorkQueueCounter[0]
//        //    );
//        globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
//    }
//}
//sync(cta);
//for (uint32_t linI =threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y ) {
//  workQueue[globalWorkQueueCounter[0]+linI]=mainShmem[linI];
//}

//cooperative_groups::memcpy_async(cta, (&workQueue[globalWorkQueueCounter[0]]), (&mainArr[0]), (sizeof(uint32_t) * localWorkQueueCounter[0]));
}




//
//if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
//    if (localWorkQueueCounter[0] > 0) {
//        //printf("local work Counter in meta pass %d  \n"
//        //    , localWorkQueueCounter[0]
//        //    );
//        globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
//    }
//}
//sync(cta);
//for (uint32_t linI = threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y) {
//    workQueue[globalWorkQueueCounter[0] + linI] = mainShmem[linI];
//}






//uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
//bool isLoopToContinue = (linIdexMeta <= fbArgs.metaData.totalMetaLength);
//isLoopToContinue = __syncthreads_or(isLoopToContinue);
//
//while (isLoopToContinue) {
//        //goldpass
//        if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//            //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
//            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
//            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
//        }
//        //segm pass
//        if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//            //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
//            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
//            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
//        }
//        // this sway of stopping a loop avoids thread diverging and stalling during synchronization ...
//        isLoopToContinue = (linIdexMeta <= fbArgs.metaData.totalMetaLength );
//        isLoopToContinue = __syncthreads_or(isLoopToContinue);
//
//
//        if (localWorkQueueCounter[0] > (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//            saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
//           sync(cta);
//           localWorkQueueCounter[0] = 0;
//           sync(cta);
//             }
//      //  sync(cta);
//       // localWorkQueueCounter[0] = 0;
//        //sync(cta);
//        linIdexMeta += (blockDim.x * blockDim.y * gridDim.x);
//
//}



//for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x
//        ; linIdexMeta <= fbArgs.metaData.totalMetaLength  // we add in order to 
//        ; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
//    //TODO() consider doing it warp centric way  
//    //  we need to be sure that amount of blocks in local work queue do not exceed lengthOfMainShmem probably the most optimal would be to divide work queue to sections where each warp would be responsible for 
//    //  then if number in warp queue will exceed the size of available shared memory it will write it to global memory ... this way we will avoid  thread divergence and keep local work queue in available shared memory space
//    //if (localWorkQueueCounter[0] < (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//      //goldpass
//    if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
//    }
//    //segm pass
//    if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
//    }
//    //sync(cta);
//    //if (localWorkQueueCounter[0] > (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//    //   saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
//    //}
//    //sync(cta);
//    //localWorkQueueCounter[0] = 0;
//    //sync(cta);
//
//
//};


//for (uint8_t outer = 0; outer < 1; outer++) {
//    uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x + (blockDim.x * blockDim.y * gridDim.x) * outer;
//    if (linIdexMeta <= fbArgs.metaData.totalMetaLength) {
//
//        //TODO() consider doing it warp centric way  
//    //  we need to be sure that amount of blocks in local work queue do not exceed lengthOfMainShmem probably the most optimal would be to divide work queue to sections where each warp would be responsible for 
//    //  then if number in warp queue will exceed the size of available shared memory it will write it to global memory ... this way we will avoid  thread divergence and keep local work queue in available shared memory space
//    //if (localWorkQueueCounter[0] < (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//      //goldpass
//        if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//            //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
//            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
//            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
//        }
//        //segm pass
//        if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//            //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
//            mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
//            modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
//        }
//    }
//}
//
//


//
//if (linIdexMeta <= fbArgs.metaData.totalMetaLength) {
//
//    //TODO() consider doing it warp centric way  
////  we need to be sure that amount of blocks in local work queue do not exceed lengthOfMainShmem probably the most optimal would be to divide work queue to sections where each warp would be responsible for 
////  then if number in warp queue will exceed the size of available shared memory it will write it to global memory ... this way we will avoid  thread divergence and keep local work queue in available shared memory space
////if (localWorkQueueCounter[0] < (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
//  //goldpass
//    if (getPredGoldPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //    printf("in meta pass gold linIdexMeta %d isPaddingPass %d  total meta %d \n", linIdexMeta, isPaddingPass, fbArgs.metaData.totalMetaLength);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 11, 7);
//    }
//    //segm pass
//    if (getPredSegmPass(isPaddingPass, isGoldPassToContinue, isSegmPassToContinue, metaData, metaDataArr, linIdexMeta)) {
//        //  printf("in meta pass segm linIdexMeta %d isPaddingPass %d \n", linIdexMeta, isPaddingPass);
//        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
//        modifyMetaDataInPaddingPass(isPaddingPass, metaDataArr, linIdexMeta, metaData, 12, 9);
//    }
//}
////    sync(cta);
////if (localWorkQueueCounter[0] > (lengthOfMainShmem - (blockDim.x * blockDim.y))) {
////   //saveWorkQueueToGlobal(cta, tile, localWorkQueueCounter, minMaxes, workQueue, globalWorkQueueCounter, mainShmem);
////    sync(cta);
////    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) {
////    //    localWorkQueueCounter[0] = 0;
////    }
////    sync(cta);
////}
//
//
//}
