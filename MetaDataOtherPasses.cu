#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"

using namespace cooperative_groups;

/*
  First we need to be sure that we start from global workqueue to be 0 but it is hard to  get without additional grid sync 
  so we will use 2 counters for odd and even iteration number and here below we will zero the old one on one thread of first thread block
  
  We need to populate the worqueue
   We need to get count of the total FP, FN so we will know wheather we should start loop anew 
*/



/*
as we have limited space in work queue we will use also the resShmem and source shmem in order to keep calculations easy 
we will divide all shared memory in  blocks of 32 length what will enable us using fast shift operators 
- then on the basis of the result spot we will deide to which shared memory array to put the data locally
0) we are supplied with a spot obtain from atomic addition to local counter where we want to put our data
1) we divide by shifting 5 times so we will know to which shared memory space to put our data we will need to use if operators
2) using sutractions and getting remainder will give us spot in 32 subblock where to put the data 
https://stackoverflow.com/questions/13548172/bitshifts-to-obtain-remainder


*/








//
//
//#pragma once
//template <typename POYO>
//__device__ void addToQueueOtherPasses(ForBoolKernelArgs<POYO> fbArgs, 
//    char* tensorslice, uint8_t xMeta, uint8_t yMeta, uint8_t zMeta, uint8_t isGold, unsigned int fpFnLocCounter[1]
//    , uint8_t localWorkAndOffsetQueue[3000][4], unsigned int localWorkQueueCounter[1], bool metaDataPredicate
//) {
//
//    //given fp is non zero we need to  add this to local queue
//    if (metaDataPredicate) {
//        //we need to establish where to put the entry in the local queue
//        old = atomicAdd(&localWorkQueueCounter[0], 1);
//        //we check weather we still have space in shared memory
//        if (old < 2990) {// so we still have space in shared memory
//            localWorkAndOffsetQueue[old][0] = xMeta;
//            localWorkAndOffsetQueue[old][1] = yMeta;
//            localWorkAndOffsetQueue[old][2] = zMeta;
//            localWorkAndOffsetQueue[old][3] = isGold;// marking it is about gold pass - FP
//        }
//        else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
//            unsigned int old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
//            getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
//            getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
//            getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
//            getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = isGold;
//        };
//
//        if (isGold == 1) {
//            //so we check is counter smaller than total count
//            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
//                = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
//                    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);
//
//        }
//        //FN pass
//        else {
//            //so we check is counter smaller than total count
//            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
//                = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
//                    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
//        };
//
//
//    }
//}







#pragma once
template <typename PYO>
inline __global__ void getWorkQueeueFromIsToBeActivated(ForBoolKernelArgs<PYO> fbArgs) {

    //////initializations
    thread_block cta = this_thread_block();
    char* tensorslice;// needed for iterations over 3d arrays
    unsigned int count = 0;// local variable

    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];
    //local offset counters  for fp and fn's
    __shared__ unsigned int fpFnLocCounter[1];
    // used to store the start position in global memory for whole block
    __shared__ unsigned int globalOffsetForBlock[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    //used as local work queue counter
    __shared__ unsigned int localWorkQueueCounter[1];
    //according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes so it will give us 1600 length of shared memory
    //so here we will store locally the calculated offsets and coordinates of meta data block of intrest marking also wheather we are  talking about gold or segmentation pass (fp or fn )
    __shared__ uint8_t localWorkAndOffsetQueue[2000][4];
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        fpFnLocCounter[0] = 0;
    }
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        localWorkQueueCounter[0] = 0;
    }

        checkIsToBeDilatated(fbArgs, tensorslice, isGoldPassToContinue, isSegmPassToContinue);
    sync(cta);


    ///////// now we need to look through blocks that we just  activated 
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //gold pass

        bool isToBeActivated = isGoldPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedGold, fbArgs.metaData.isToBeActivatedGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullGold, fbArgs.metaData.isFullGold.Ny, yMeta, zMeta)[xMeta]);


        //given fp is non zero we need to  add this to local queue
        if (isToBeActivated) {
                   //     printf("to be activated pass putting to work queue xMeta %d yMeta %d zMeta %d isGold %d \n", xMeta,yMeta, zMeta, 0 );

            //we need to establish where to put the entry in the local queue
            unsigned int old = atomicAdd(&localWorkQueueCounter[0], 1);
            //we check weather we still have space in shared memory
            if (old < 1990) {// so we still have space in shared memory
      /*          printf( "\naaaa adding to shmem to be activated xMeta %d yMeta %d zMeta %d localWorkQueueCounter %d     \n"
                , xMeta, yMeta, zMeta, localWorkQueueCounter[0] );
*/

                localWorkAndOffsetQueue[old][0] = xMeta;
                localWorkAndOffsetQueue[old][1] = yMeta;
                localWorkAndOffsetQueue[old][2] = zMeta;
                localWorkAndOffsetQueue[old][3] = 1;// marking it is about gold pass - FP
            }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = 1;
            };
            //printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);

            //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);

        }
        //sync(cta);
        //if (isToBeActivated) {


        //         //so we check is counter smaller than total count
        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);

        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);


        //}



            
        if (isToBeActivated) {
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedGold, fbArgs.metaData.isToBeActivatedGold.Ny, yMeta, zMeta)[xMeta] = false;
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta] = true;

        }
        //segmPass
        isToBeActivated = isSegmPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedSegm, fbArgs.metaData.isToBeActivatedSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]);
       
        //given fp is non zero we need to  add this to local queue
        if (isToBeActivated) {
          //  printf("to be activated pass putting to work queue xMeta %d yMeta %d zMeta %d isGold %d \n", xMeta, yMeta, zMeta, 0);

            //we need to establish where to put the entry in the local queue
            unsigned int old = atomicAdd(&localWorkQueueCounter[0], 1);
            //we check weather we still have space in shared memory
            if (old < 1990) {// so we still have space in shared memory
                localWorkAndOffsetQueue[old][0] = xMeta;
                localWorkAndOffsetQueue[old][1] = yMeta;
                localWorkAndOffsetQueue[old][2] = zMeta;
                localWorkAndOffsetQueue[old][3] = 0;

               // printf("\naaaa adding to shmem to be activated xMeta %d yMeta %d zMeta %d localWorkQueueCounter %d     \n"
            //        , xMeta, yMeta, zMeta, localWorkQueueCounter[0]);

            }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
               old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = 0;
            };

            //printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //, xMeta, yMeta, zMeta);

            //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);
        }
        //sync(cta);
        //if (isToBeActivated) {

        //        //so we check is counter smaller than total count
        //        getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
        //            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
        //                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);

        //        getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
        //            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);

        //}



        if (isToBeActivated) {
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeActivatedSegm, fbArgs.metaData.isToBeActivatedSegm.Ny, yMeta, zMeta)[xMeta] = false;
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta] = true;

            //printf("\n found to be actvated xMeta %d yMeta %d zMeta %d isGold  %d isSegmPassToContinue[0] %d  isActive %d isFull %d \n ", xMeta, yMeta, zMeta, 0, isSegmPassToContinue[0], getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm
            //    , fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta], getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]);
        }
    }
    sync(cta);
    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        if (localWorkQueueCounter[0] > 0) {
            globalWorkQueueCounter[0] = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), (localWorkQueueCounter[0]));
           // printf(" \n globalWorkQueueCounter in looking for padding blocks %d   \n ", globalWorkQueueCounter[0]);
        }
    }
    sync(cta);
    //grid stride loop for pushing value from local memory to global 


    for (uint16_t i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {

        // printf("addTo %d global Queue xMeta [%d] yMeta [%d] zMeta [%d] isGold %d \n", globalWorkQueueCounter[0] + i, localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);
         //TODO() instead of copying memory manually better would be to use mempcyasync ...
        // printf("\n saving to local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d   ", localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);

        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][0];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][1];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][2];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][3];
        //and offset 

    }


}

#pragma once
template <typename PYO>
inline __global__ void getWorkQueeueFromActive_mainPass(ForBoolKernelArgs<PYO> fbArgs) {
    //////initializations
    thread_block cta = this_thread_block();
    char* tensorslice;// needed for iterations over 3d arrays
    unsigned int count = 0;// local variable

    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];
    //local offset counters  for fp and fn's
    __shared__ unsigned int fpFnLocCounter[1];
    // used to store the start position in global memory for whole block
    __shared__ unsigned int globalOffsetForBlock[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    //used as local work queue counter
    __shared__ unsigned int localWorkQueueCounter[1];
    //according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes so it will give us 1600 length of shared memory
    //so here we will store locally the calculated offsets and coordinates of meta data block of intrest marking also wheather we are  talking about gold or segmentation pass (fp or fn )
    __shared__ uint8_t localWorkAndOffsetQueue[2000][4];
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        fpFnLocCounter[0] = 0;
    }
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        localWorkQueueCounter[0] = 0;
    }

    checkIsToBeDilatated(fbArgs, tensorslice, isGoldPassToContinue, isSegmPassToContinue);
    sync(cta);


    ///////// now we need to look through blocks that we just  activated 
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //gold pass

        bool isToBeActivated = isGoldPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullGold, fbArgs.metaData.isFullGold.Ny, yMeta, zMeta)[xMeta]);




        //given fp is non zero we need to  add this to local queue
        if (isToBeActivated) {
            //we need to establish where to put the entry in the local queue
            unsigned int old = atomicAdd(&localWorkQueueCounter[0], 1);
            //printf("main pass putting to work queue xMeta %d yMeta %d zMeta %d isGold %d \n", xMeta, yMeta, zMeta, 1);
            //we check weather we still have space in shared memory
            if (old < 1990) {// so we still have space in shared memory
                localWorkAndOffsetQueue[old][0] = xMeta;
                localWorkAndOffsetQueue[old][1] = yMeta;
                localWorkAndOffsetQueue[old][2] = zMeta;
                localWorkAndOffsetQueue[old][3] = 1;// marking it is about gold pass - FP
            }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = 1;
            };
            //printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);

            //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);
        }
        //sync(cta);
        //if (isToBeActivated) {

        //    //so we check is counter smaller than total count
        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);

        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);


        //}

        //segmPass
        isToBeActivated = isSegmPassToContinue[0] && (getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta]
            && !getTensorRow<bool>(tensorslice, fbArgs.metaData.isFullSegm, fbArgs.metaData.isFullSegm.Ny, yMeta, zMeta)[xMeta]);

        //given fp is non zero we need to  add this to local queue
        if (isToBeActivated) {
            //we need to establish where to put the entry in the local queue
            unsigned int old = atomicAdd(&localWorkQueueCounter[0], 1);
           // printf("main pass putting to work queue xMeta %d yMeta %d zMeta %d isGold %d \n", xMeta,yMeta, zMeta, 0 );
            //we check weather we still have space in shared memory
            if (old < 1990) {// so we still have space in shared memory
                localWorkAndOffsetQueue[old][0] = xMeta;
                localWorkAndOffsetQueue[old][1] = yMeta;
                localWorkAndOffsetQueue[old][2] = zMeta;
                localWorkAndOffsetQueue[old][3] = 0;
            }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = 0;
            };

            //printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            //, xMeta, yMeta, zMeta);

            //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
            //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
            //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            //    , xMeta, yMeta, zMeta);

        }
        //sync(cta);
        //if (isToBeActivated) {

        //    //so we check is counter smaller than total count
        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);

        //    getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
        //        = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);

        //}


    }
    sync(cta);
    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        if (localWorkQueueCounter[0] > 0) {
            globalWorkQueueCounter[0] = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), (localWorkQueueCounter[0]));

        }
    }
    sync(cta);
    //grid stride loop for pushing value from local memory to global 


    for (uint16_t i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {

        // printf("addTo %d global Queue xMeta [%d] yMeta [%d] zMeta [%d] isGold %d \n", globalWorkQueueCounter[0] + i, localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);
         //TODO() instead of copying memory manually better would be to use mempcyasync ...
        // printf("\n saving to local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d   ", localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);

        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][0];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][1];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][2];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][3];
        //and offset 

    }






}
























/*

#pragma once
template <typename PYOPP>
inline __device__ void getValueOfLocalWorQ(ForBoolKernelArgs<PYOPP> fbArgs, uint8_t  subSpot,  uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t& i, unsigned int globalWorkQueueCounter[1], char* tensorslice){
  if( (i>>5)==0){ // using local work queue
     // remainder div 16 + is oddd times 16 ...
      getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, subSpot, 0)[globalWorkQueueCounter[0] + i]= localWorkQueue[((i & (15)) + 16 * (((i >> 5) & 1)))][subSpot];
  }else if((i>>5)< 5){// using source shmem
      getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, subSpot, 0)[globalWorkQueueCounter[0] + i]=  sourceShared[((i & (15)) + 16 * (((i >> 5) & 1)))][subSpot + ((i >> 5) - 1) * 4];
  }else{// using resshmem
      getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, subSpot, 0)[globalWorkQueueCounter[0] + i]= resShared[((i & (15)) + 16 * (((i >> 5) & 1)))][subSpot + ((i >> 5) - 5) * 4] ;
  }

};
#pragma once
inline __device__ void setValueOfLocalWorQ(unsigned int spot, uint8_t  subSpot, uint8_t value, uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , uint16_t localWorkQueue[localWorkQueLength][4]  ) {
    if ((spot >> 5) == 0) { // using local work queue
       // remainder div 16 + is oddd times 16 ...
        localWorkQueue[((spot & (15)) + 16 * (((spot >> 5) & 1)))][subSpot] = value;
    }
    else if ((spot >> 5) < 5) {// using source shmem
        sourceShared[((spot & (15)) + 16 * (((spot >> 5) & 1)))][subSpot + ((spot >> 5) - 1) * 4] = value;
    }
    else {// using resshmem
        resShared[((spot & (15)) + 16 * (((spot >> 5) & 1)))][subSpot + ((spot >> 5) - 5) * 4] = value;
    }

};


*/




/*#pragma once
template <typename PYO>
inline __device__ void addToQueueOtherPasses(ForBoolKernelArgs<PYO> fbArgs
, unsigned int& old, char* tensorslice
    , uint8_t& xMeta, uint8_t& yMeta, uint8_t& zMeta    , uint8_t isGold
      , uint16_t localWorkQueue[30][4], unsigned int localWorkQueueCounter[1], uint32_t sourceShared[32][32], uint32_t resShared[32][32],
    bool metaDataPredicate
) {

        if (metaDataPredicate) {
            //we need to establish where to put the entry in the local queue
            old = atomicAdd(&localWorkQueueCounter[0], 1);
           // printf("\n saving to shmem xMeta %d yMeta %d zMeta %d  isGold %d \n" , xMeta, yMeta, zMeta, isGold);
            //we check weather we still have space in shared memory
              if (old < totalCombinedShmemWorkQueue) {// so we still have space in shared memory
                  setValueOfLocalWorQ(old, 0, xMeta, sourceShared, resShared, localWorkQueue);
                  setValueOfLocalWorQ(old, 1, yMeta, sourceShared, resShared, localWorkQueue);
                  setValueOfLocalWorQ(old, 2, zMeta, sourceShared, resShared, localWorkQueue);
                  setValueOfLocalWorQ(old, 3, isGold, sourceShared, resShared, localWorkQueue);
                }
                else {// so we do not have any space more in the shared memory  -
                old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = isGold;
            };

       if (isGold == 1) {
           //so we check is counter smaller than total count
            getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
                = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta] 
                    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);
            
        }
        //FN pass
        else {
           //so we check is counter smaller than total count
           getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
               = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
                   < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
        };




        }
}







#pragma once
template <typename PKKYO>
inline __device__ void fromShmemToGlobalWorkQueue(ForBoolKernelArgs<PKKYO> fbArgs , unsigned int& old, uint16_t& i, uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , uint16_t localWorkQueue[localWorkQueLength][4], unsigned int globalWorkQueueCounter[1], char* tensorslice, unsigned int localWorkQueueCounter[1]) {
    for (i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {
       // printf("\n loading from shmem xMeta %d yMeta %d zMeta %d  isGold %d \n", localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
        getValueOfLocalWorQ(fbArgs, 0, sourceShared, resShared, localWorkQueue, i, globalWorkQueueCounter, tensorslice);
        getValueOfLocalWorQ(fbArgs, 1, sourceShared, resShared, localWorkQueue, i, globalWorkQueueCounter, tensorslice);
        getValueOfLocalWorQ(fbArgs, 2, sourceShared, resShared, localWorkQueue, i, globalWorkQueueCounter, tensorslice);
        getValueOfLocalWorQ(fbArgs, 3, sourceShared, resShared, localWorkQueue, i, globalWorkQueueCounter, tensorslice);

    }
}




*/