#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "Structs.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"

using namespace cooperative_groups;

/*
  First we need to be sure that we start from global workqueue to be 0 but it is hard to  get without additional grid sync 
  so we will use 2 counters for odd and even iteration number and here below we will zero the old one on one thread of first thread block
  
  We need to populate the worqueue
   We need to get count of the total FP, FN so we will know wheather we should start loop anew 
*/



/*
as we have limited space in work queue we will use also the resShmem and source shmem in order to keep calculations easy 
we will divide all shared memory in  blocks of 32 length what will enable us using fast shift operators 
- then on the basis of the result spot we will deide to which shared memory array to put the data locally
0) we are supplied with a spot obtain from atomic addition to local counter where we want to put our data
1) we divide by shifting 5 times so we will know to which shared memory space to put our data we will need to use if operators
2) using sutractions and getting remainder will give us spot in 32 subblock where to put the data 
https://stackoverflow.com/questions/13548172/bitshifts-to-obtain-remainder
*/




#pragma once
template <typename TKKI>
inline __device__ void metadataPass(ForBoolKernelArgs<TKKI> fbArgs, bool isPaddingPass
    , uint8_t predicateAa, uint8_t predicateAb, uint8_t predicateAc
    , uint8_t predicateBa, uint8_t predicateBb, uint8_t predicateBc
    ,uint32_t mainShmem[], unsigned int globalWorkQueueOffset[1], unsigned int globalWorkQueueCounter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localTotalLenthOfWorkQueue[1], unsigned int localMinMaxes[5]
    , unsigned int fpFnLocCounter[1], bool isGoldPassToContinue[1], bool isSegmPassToContinue[1], thread_block cta, thread_block_tile<32> tile
    , MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue, uint16_t* metaDataArr

) {
  // preparation loads
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    fpFnLocCounter[0] = 0;
}
if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
    localWorkQueueCounter[0] = 0;
}
if (tile.meta_group_rank() == 1) {
    cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
}
tile.sync();
/*
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
     */
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { isGoldPassToContinue[0] = ((localMinMaxes[0] * fbArgs.robustnessPercent) > localMinMaxes[3]); };
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 1) { isGoldPassToContinue[0] = ((localMinMaxes[1] * fbArgs.robustnessPercent) > localMinMaxes[4]); };
sync(cta);

//iterations 
for (uint16_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
    //goldpass


    if (isGoldPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateAa]
        && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateAb]
        && (isPaddingPass &&  !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateAc])) {

        auto old = atomicAdd_block(&localWorkQueueCounter[0], 1) - 1;
        if (old < lengthOfMainShmem) {
            mainShmem[old] = uint32_t(linIdexMeta + (isGoldOffset) );
        }
        else {
            old = atomicAdd(&(minMaxes[9]), 1);
            workQueue[old] = uint32_t(linIdexMeta + (isGoldOffset) );
        }
        if (isPaddingPass) {
            //setting to be activated to 0 
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 11] = 0;
            //setting active to 1
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 7] = 1;
        }
    }
    //segm pass
    if (isSegmPassToContinue[0] && metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateBa]
        && !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateBb]
        && (isPaddingPass &&  !metaDataArr[linIdexMeta * metaData.metaDataSectionLength + predicateBc]) ) {

        auto old = atomicAdd_block(&localWorkQueueCounter[0], 1) - 1;
        if (old < lengthOfMainShmem) {
            mainShmem[old] = uint32_t(linIdexMeta);
        }
        else {
            old = atomicAdd(&(minMaxes[9]), 1);
            workQueue[old] = uint32_t(linIdexMeta);
        }
        if (isPaddingPass) {
            //setting to be activated to 0 
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 12] = 0;
            //setting active to 1
            metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 9] = 1;
        }
    }

}
//getting begining where we would copy local queue to global one 
sync(cta);
if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    if (localWorkQueueCounter[0] > 0) {
        globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
    }
}
sync(cta);
for (uint16_t linI =threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y ) {
  workQueue[globalWorkQueueCounter[0]+linI]=mainShmem[linI];
}

//cooperative_groups::memcpy_async(cta, (&workQueue[globalWorkQueueCounter[0]]), (&mainArr[0]), (sizeof(uint32_t) * localWorkQueueCounter[0]));
}




