#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




template <typename TKKI>
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArrAPointer,
    uint32_t* mainArrBPointer, MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint32_t* resultListPointerLocal, uint32_t* resultListPointerIterNumb,
    thread_block cta, thread_block_tile<32> tile, grid_group grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6]  , bool isBlockFull[1], int iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint32_t localBlockMetaData[20], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint32_t* metaDataArr, uint32_t oldIsGold[1], uint32_t oldLinIndM[1], uint32_t localBlockMetaDataOld[6]
) {




    auto pipeline = cuda::make_pipeline();
    auto bigShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength));
    auto thirdRegShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (32));


    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0  && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] =true;
    };
    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };

    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        // if it will be still of such value it mean that no block was processed
        currLinIndM[0] = UINT16_MAX;
    };

    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }




    sync(cta);
    /// load work QueueData into shared memory 
    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        
        cuda::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop]), cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]));
        //to do change into barrier
        sync(cta);
        //setting main indexes to make later work easier
        if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
            isGold[0] = (mainShmem[startOfLocalWorkQ] >= isGoldOffset);

        }
        if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block

            currLinIndM[0] = mainShmem[startOfLocalWorkQ] - isGoldOffset * (mainShmem[startOfLocalWorkQ] >= isGoldOffset);
            //  printf("in pipeline  linear index meta %d block %d \n ", currLinIndM[0], blockIdx.x);
        }        
        sync(cta);
        //now all of the threads in the block needs to have the same i value so we will increment by 1
        // we are preloading to the pipeline block metaData
////##### pipeline Step 0
        pipeline.producer_acquire();

        cuda::memcpy_async(cta, (&localBlockMetaData[0]),
            (&metaDataArr[(mainShmem[startOfLocalWorkQ] - isGoldOffset * (mainShmem[startOfLocalWorkQ] >= isGoldOffset))
                * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

        pipeline.producer_commit();
        
        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                ///#### pipeline step 1) now we load data for next step (to mainly sourceshmem and left-right if apply) and process data loaded in previous step
                pipeline.producer_acquire();

                //load data of intrest from block to the right
                if (currLinIndM[0]<(metaData.totalMetaLength-1)) {
                    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemB + tile.meta_group_rank()]),
                        &getSourceReduced(fbArgs, iterationNumb)[
                            (currLinIndM[0] + 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGold[0])
                            + tile.meta_group_rank() * 32], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                        cuda::aligned_size_t<4>(sizeof(uint32_t))
                        , pipeline);
                }
                //load data of interst form block to the left
                if (currLinIndM[0] > 0) {
                    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemA + tile.meta_group_rank()]),
                        &getSourceReduced(fbArgs, iterationNumb)[
                            (currLinIndM[0] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGold[0])
                                //we look for indicies 31,63... up to metaData.mainArrXLength
                            + (tile.meta_group_rank() * 32) + 31])
                        , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
                }

                //load data about this block
                cuda::memcpy_async(cta, &mainShmem[begSourceShmem],   &getSourceReduced(fbArgs, iterationNumb)[
                        currLinIndM[0] *metaData.mainArrSectionLength+metaData.mainArrXLength * (1 - isGold[0])],
                    , bigShape   , pipeline);


                pipeline.producer_commit();


                //        ////we need to do the cleaning after previous block .. compute first we load data about calculated linear index meta and information is it gold iteration ...
                pipeline.consumer_wait();

           
                afterBlockClean(cta, worQueueStep, localBlockMetaDataOld, mainShmem, i,
                    metaData, tile, localFpConter, localFnConter
                    , blockFpConter, blockFnConter
                    , metaDataArr, oldLinIndM, oldIsGold
                    , isAnythingInPadding, isBlockFull, isPaddingPass);



                pipeline.consumer_release();

 ////////#### pipeline step 2)  load block from top and process center that is in source shmem; and both smallRegShmems
                //load for next step - so we load block to the top
                if (localBlockMetaData[13] < UINT16_MAX) {
                    pipeline.producer_acquire();

                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 13)], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);

                    pipeline.producer_commit();
                }

                //compute - now we have data in source shmem about this block and left and right padding and we need to process it 
                pipeline.consumer_wait();
                // first we perform up and down dilatations inside the block
                
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
                
                
                //we also do the left and right dilatations
                ////left
                
                //dilatateHelperForTransverse((threadIdx.x == 0),
                //    2, (-1), (0), mainShmem, isAnythingInPadding
                //    , 0, threadIdx.y + (32 - fbArgs.dbYLength) // we add offset depending on y dimension
                //    , 15, begSMallRegShmemA, localBlockMetaData);

                ////right
                //dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
                //    3, (1), (0), mainShmem, isAnythingInPadding
                //    , 0, threadIdx.y
                //    , 16, begSMallRegShmemB, localBlockMetaData);





                //dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
                //    3, (1), (0), mainShmem, isAnythingInPadding
                //    , 0,threadIdx.y
                //    , 16, begSMallRegShmemA, localBlockMetaData);

                ///////////saving old
                //additionally we save previous copies of data so refreshing will keep easier
                if (tile.thread_rank() < 6 && tile.meta_group_rank() == 0) {
                    localBlockMetaDataOld[tile.thread_rank()] = localBlockMetaData[13 + tile.thread_rank()];
                }
                if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
                    oldIsGold[0] == isGold[0];
                }
                if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0) {
                    oldLinIndM[0] == currLinIndM[0];
                }

                pipeline.consumer_release();
////////#### pipeline step 3) we load bottom, anterior and posterior and we process top
                      //load anterior and posterior and bottom
                pipeline.producer_acquire();
                //block to anterior 
                if (localBlockMetaData[17] < UINT16_MAX && tile.meta_group_rank() == 0) {

                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemA], &getSourceReduced(fbArgs, iterationNumb)[
                        (localBlockMetaData[17]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGold[0])],
                        , thirdRegShape, pipeline);

                }
                // block to posterior
                if (localBlockMetaData[18] < UINT16_MAX && tile.meta_group_rank() == 1) {
                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemB], &getSourceReduced(fbArgs, iterationNumb)[
                        (localBlockMetaData[18]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGold[0])
                        +(blockDim.y-1)*32// we need last 32 length entry of the posterior block 
                    ],
                        , thirdRegShape, pipeline);
                }


                pipeline.producer_commit();


                //    compute - now we have data in source shmem about block to the top
                pipeline.consumer_wait();
                dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
                    , 1// represent a uint32 number that has a bit of intrest in this block set and all others 0 here first bit is set
                    , 2147483648
                    , begfirstRegShmem);
                pipeline.consumer_release();
////////#### pipeline step 5) if block is to be validated we load reference data and we process bottom, left and right
                //load reference data if block is to be validated otherwise if it is not the last step in the loop we load data for next loop
                pipeline.producer_acquire();
                if (localBlockMetaData[((1 - isGold[0]) + 1)] //fp for gold and fn count for not gold
                > localBlockMetaData[((1 - isGold[0]) + 1)]) {// so count is bigger than counter so we should validate
            //now we load data from reference arrays 
                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &origArrs[metaData.mainArrXLength * (isGold[0])], //we look for 
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);

                }
                else {//if we are not validating we immidiately start loading data for next loop
                    if (i + 1 <= worQueueStep[0]) {
                        cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + i] - isGoldOffset * (mainShmem[startOfLocalWorkQ + i] >= isGoldOffset))
                                * metaData.metaDataSectionLength])
                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
                    }
                }
                //    compute - now we have data in source shmem about block to the bottom, left and right

                pipeline.producer_commit();
                pipeline.consumer_wait();
                //bottom
                dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
                    , 2147483648// represent a uint32 number that has a bit of intrest in this block set and all others 0 here last bit is set
                    , 1
                    , begSecRegShmem);
                //posterior
                
                //dilatateHelperForTransverse((threadIdx.y == 0), 5
                //    , (0), (-1), mainShmem, isAnythingInPadding
                //    , 0, threadIdx.x // we add offset depending on y dimension
                //    , 18, begSMallRegShmemB, localBlockMetaData);
                ////anterior
                //dilatateHelperForTransverse((threadIdx.y == (fbArgs.dbYLength - 1)), 4
                //    , (0), (1), mainShmem, isAnythingInPadding
                //    , 0, threadIdx.x
                //    , 17, begSMallRegShmemA, localBlockMetaData);
                
               // now all of the data is processed we need to save it into global memory
               // TODO try to use mempcy async here
                getTargetReduced(fbArgs, iterationNumb)[getIndexForSaveResShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData) + threadIdx.x + threadIdx.y * 32] = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
               // setting information about is block full


                if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] != UINT32_MAX) {
                    isBlockFull[0] = false;
                }
                //setting indexes for next iteration
                if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0 && i + 1 <= worQueueStep[0]) {// this is how it is encoded wheather it is gold or segm block
                    isGold[0] = (mainShmem[startOfLocalWorkQ + i + 1] >= isGoldOffset);

                }
                if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0 && i + 1 <= worQueueStep[0]) {// this is how it is encoded wheather it is gold or segm block

                    currLinIndM[0] = mainShmem[startOfLocalWorkQ + i + 1] - isGoldOffset * (mainShmem[startOfLocalWorkQ + i] >= isGoldOffset);
                }


                pipeline.consumer_release();


//////////#### pipeline step 6) if block is to be validated we process the res and reference data and start loading data for begining of the next loop

                if (localBlockMetaData[((1 - isGold[0]) + 1)] //fp for gold and fn count for not gold
                > localBlockMetaData[((1 - isGold[0]) + 1)]) {// so count is bigger than counter so we should validate
    ////load data for next iteration
        pipeline.producer_acquire();
            if (i + 1 <= worQueueStep[0]) {
                cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                    (&metaDataArr[(mainShmem[startOfLocalWorkQ+i] - isGoldOffset * (mainShmem[startOfLocalWorkQ+i] >= isGoldOffset))
                        * metaData.metaDataSectionLength])
                    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
            }
            
        pipeline.producer_commit();

        //process check is there any new result (we have reference in begfirstRegShmem)
                 //now first we need to check for bits that are true now after dilatation but were not in source we will save it in res shmem becouse we will no longer need it
        pipeline.consumer_wait();
            mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);


            //we now look for bits prasent in both reference arrays and current one
            mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

            // now we look through bits and when some is set we call it a result 
            #pragma unroll
            for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                //if any bit here is set it means it should be added to result list 
                if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                    //first we add to the resList
                    //TODO consider first passing it into shared memory and then async mempcy ...
                    //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                    unsigned int old = 0;
                    ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                    if (isGold[0]) { old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[6]; }
                    else { old = atomicAdd_block(&(localFnConter[0]), 1)-1 + localBlockMetaData[5]; };
                    //   add results to global memory    
                    resultListPointerMeta[old] = oldLinIndM[0] + isGoldOffset * oldIsGold[0];
                    resultListPointerLocal[old] = (fbArgs.dbYLength * 32 * bitPos + threadIdx.y * 32 + threadIdx.x);
                    resultListPointerIterNumb[old] = iterationNumb[0];
                }
            }
         pipeline.consumer_release();

                };
            }
  }  
    }

    //here we are after all of the blocks planned to be processed by this block are
    sync(cta);

    //updating local counters of last local block (normally it is done at the bagining of the next block)
    //but we need to check weather any block was processed at all
    if (currLinIndM[0]!= UINT16_MAX) {
        afterBlockClean(cta, worQueueStep, localBlockMetaDataOld, mainShmem,2,//2 is completely arbitrary important it is bigger than 0
            metaData, tile, localFpConter, localFnConter
            , blockFpConter, blockFnConter
            , metaDataArr, oldLinIndM, oldIsGold
            , isAnythingInPadding, isBlockFull, isPaddingPass);


    }



    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        if (blockFpConter[0] > 0) {
            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
        }
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        if (blockFnConter[0] > 0) {
            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
        }
    };
    // in first thread block we zero work queue counter
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        if (blockIdx.x==0) {
            minMaxes[9] = 0;
        }
    };


}
