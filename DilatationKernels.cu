#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




template <typename TKKI>
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArr, MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint16_t* resultListPointerLocal, uint16_t* resultListPointerIterNumb,
    thread_block cta, thread_block_tile<32> tile, grid_group grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6]  , bool isBlockFull[1], uint32_t iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint16_t localBlockMetaData[20], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint16_t* metaDataArr
) {
    auto pipeline = cuda::make_pipeline();
    auto bigShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength));
    auto thirdRegShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (32));
    thread_block_tile<1> miniTile = tiled_partition<1>(block);


    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0  && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] =true;
    };
    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }

    sync(cta);
    /// load work QueueData into shared memory 
    for (uint16_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        
        cooperative_groups::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop]), cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]));
        sync(cta);
        //now all of the threads in the block needs to have the same i value so we will increment by 1
        // we are preloading to the pipeline block metaData
        ////##### pipeline Step 0
        pipeline.producer_acquire();

        cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&metaDataArr[(mainShmem[startOfLocalWorkQ] - UINT16_MAX * (mainShmem[startOfLocalWorkQ] >= UINT16_MAX)) * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint16_t) * 20), pipeline);



        pipeline.producer_commit();
        
        for (uint16_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                 ///#### pipeline step 1) now we load data for next step (to mainly sourceshmem and left-right if apply) and process data loaded in previous step
                    pipeline.producer_acquire();

                    cuda::memcpy_async(cta, (&mainShmem[(((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) > 0) * (-32)) + begSourceShmem]), // we check weather there is anything to the left - not on left border if so we need place for left 32 entries
                        &mainArr[((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) *(-32)) // we check weather there is anything to the left - not on left border if so we load left 32 entries
                        +  getIndexForSourceShmem(metaData, mainShmem, iterationNumb,i )] , 
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * //below we check weather we have block to the left and right if so we increase number of copied entries
                            (metaData.mainArrXLength+32*(((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) > 0)
                                + ((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) <(metaData.totalMetaLength -1)))   ))
                        , pipeline);


                    pipeline.producer_commit();

        //        ////compute first we load data about calculated linear index meta and information is it gold iteration ...
                   pipeline.consumer_wait();
                       if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                           isGold[0] = uint32_t(mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX);
                           if (isGold[0]) {
                               //removing info about wheather it is gold or not pass so we will be able to use it as linear metadata index
                               currLinIndM[0] = mainShmem[startOfLocalWorkQ + i] - UINT16_MAX;
                           }
                       };
                      if (tile.thread_rank() <6 && tile.meta_group_rank() == 1) {// this is how it is encoded wheather it is gold or segm block
                            isAnythingInPadding[0] = false;                       
                       };
                   pipeline.consumer_release();


               ////////#### pipeline step 2) 
               //load for next step - so we load posterior of anterior block  and anterior of posterior block given they exist
                   //anterior and posterior
                   if (localBlockMetaData[17] < UINT16_MAX   || (localBlockMetaData[18] < UINT16_MAX) {
                       pipeline.producer_acquire();
                           //posterior of the block to anterior we load it using single threads and multple mempcy async becouse memory is non aligned
                           if (localBlockMetaData[17] < UINT16_MAX  && miniTile.meta_group_rank()< fbArgs.dbYLength) {
                               cooperative_groups::memcpy_async(miniTile, (&mainShmem[begfirstRegShmem+32+ miniTile.meta_group_rank()]),
                                   (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 17)] //basic offset
                                       //we look for indicies 0,32,64... up to metaData.mainArrXLength
                                       + miniTile.meta_group_rank()*32
                                       )
                                   , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
                           }
                           //anterior of the block to posterior
                           if (localBlockMetaData[18] < UINT16_MAX&& miniTile.meta_group_rank()< fbArgs.dbYLength*2) {
                               cooperative_groups::memcpy_async(miniTile, (&mainShmem[begfirstRegShmem+64+ miniTile.meta_group_rank() ]),
                                   (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 18)
                                       //we look for indicies 31,63... up to metaData.mainArrXLength
                                       + (miniTile.meta_group_rank() * 32)+31
                                   ])
                                   , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
                           }
                       pipeline.producer_commit();
                   }
                     //compute - now we have data in source shmem about this block and left and right padding
                       pipeline.consumer_wait();
                           // first we perform up and down dilatations inside the block
                           mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = bitDilatate(mainShmem[begSourceShmem+threadIdx.x + threadIdx.y * 32]);
                           //we also do the left and right dilatations
                           if (localBlockMetaData[17] < UINT16_MAX) {

                           };
                           if (localBlockMetaData[18] < UINT16_MAX) {

                           };

                       pipeline.consumer_release();


        //         ////////#### pipeline step 3) process anterior block data and load posterior
        //        loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
        //        localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
        //        , metaData,mainArr, //pointers to arrays with data
        //        //now some variables needed to load data  
        //            18 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begSecRegShmem //offset defined in shared memory used to load data into 
        //            , bigShape // shape and alignment of data in load - inludes length of data
        //        //now variables needed for dilatations we dilatate to anterior
        //            17 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begfirstRegShmem//offset defined in shared memory used to process  data from 
        //        ,(threadIdx.y == (fbArgs.dbYLength - 1) // defining when our thread is a corner case and need to load data from outside of the block
        //        , 4,// needed to know wheather block in given direction should be marked as to be activated
        //        (0), (1)// x and y changes
        //        , 0, threadIdx.x// coordinates in new block

        //         ////////#### pipeline step 4) process posterior block data and load right
        //        loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
        //        localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
        //        , metaData,mainArr, //pointers to arrays with data
        //        //now some variables needed to load data  
        //            16 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begfirstRegShmem //offset defined in shared memory used to load data into 
        //            , thirdRegShape // shape and alignment of data in load - inludes length of data
        //        //now variables needed for dilatations we dilatate to anterior
        //            18 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begSecRegShmem//offset defined in shared memory used to process  data from 
        //        ,(threadIdx.y == 0) // defining when our thread is a corner case and need to load data from outside of the block
        //        , 5,// needed to know wheather block in given direction should be marked as to be activated
        //        (0), (-1)// x and y changes
        //        , (fbArgs.dbYLength - 1), threadIdx.x)// coordinates in new block


        //         ////////#### pipeline step 5) process right block data and load left
        //        loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
        //        localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
        //        , metaData,mainArr, //pointers to arrays with data
        //        //now some variables needed to load data  
        //            15 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begSecRegShmem //offset defined in shared memory used to load data into 
        //            , bigShape // shape and alignment of data in load - inludes length of data
        //        //now variables needed for dilatations we dilatate to anterior
        //            16 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begfirstRegShmem//offset defined in shared memory used to process  data from 
        //        ,(threadIdx.x == (fbArgs.dbXLength - 1) // defining when our thread is a corner case and need to load data from outside of the block
        //        , 3,// needed to know wheather block in given direction should be marked as to be activated
        //        (1), (0)// x and y changes
        //        , threadIdx.y, 0// coordinates in new block


        //         ////////#### pipeline step 6) process left block data and load top
        //        loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
        //        localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
        //        , metaData,mainArr, //pointers to arrays with data
        //        //now some variables needed to load data  
        //            13 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begfirstRegShmem //offset defined in shared memory used to load data into 
        //            , bigShape // shape and alignment of data in load - inludes length of data
        //        //now variables needed for dilatations we dilatate to anterior
        //            15 // where is the index describing linear index of the neighbour in direction of intrest
        //            ,begSecRegShmem //offset defined in shared memory used to process  data from 
        //        ,(threadIdx.x == 0) // defining when our thread is a corner case and need to load data from outside of the block
        //        , 2,// needed to know wheather block in given direction should be marked as to be activated
        //        (-1), (0)// x and y changes
        //        , threadIdx.y, (fbArgs.dbXLength - 1))// coordinates in new block

        //         ////////#### pipeline step 7) process top block data and load bottom
        //            if (localBlockMetaData[14]<UINT16_MAX) {
        //                pipeline.producer_acquire();
        //                   cooperative_groups::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
        //                    (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData,14 )])
        //                    , bigShape, pipeline);
        //                pipeline.producer_commit();
        //              }
        //        //compute
        //        pipeline.consumer_wait();
        //        dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, pipeline, localBlockMetaData, 13,
        //            , 1// represent a uint32 number that has a bit of intrest in this block set and all others 0 here first bit is set
        //            , 2147483648
        //            , begfirstRegShmem);
        //            pipeline.consumer_release();    
        //         ////////#### pipeline step 8) process bottom block data  - do final operations for a block and load reference data if block is to be validated
        //        // now we need to establish weather this block should be validated so weahter the counter in metadata is smaller than metadata count
        //        //load

        //        if( localBlockMetaData[((1-isGold[0])+1)] //fp for gold and fn count for not gold
        //            > localBlockMetaData[((1-isGold[0])+1)]   ){// so count is bigger than counter so we should validate
        //        //now we load data from referenca arrays 
    

        //        }else{//if we are not validating we immidiately start loading data for next loop
        //            lastLoad(pipeline,cta,worQueueStep, localBlockMetaData, mainArr, mainShmem, i, metaData
        //        )
        //        }

        //        //compute bottom block data
        //        pipeline.consumer_wait();

        //        dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, pipeline,localBlockMetaData,14, 
        //                , 2147483648// represent a uint32 number that has a bit of intrest in this block set and all others 0 here last bit is set
        //                , 1
        //                ,begfirstRegShmem)

        //        
        //         krowa additionally we need to establish and save information is block full and mark neighbouring blocks as to be activated if it is not a padding pass       
        //                we also need to save results of res shmem into dilatation array
        //
        //
        //        pipeline.consumer_release();    
        //         ////////#### pipeline step 9 ) this step exists only  if block is to be validated 
        //        if( localBlockMetaData[((1-isGold[0])+1)] //fp for gold and fn count for not gold
        //            > localBlockMetaData[((1-isGold[0])+1)]   ){// so count is bigger than counter so we should validate
        //            lastLoad(pipeline,cta/
        //        worQueueStep, localBlockMetaData, mainArr, mainShmem, i, metaData)
        //            //here we are establishing weather we have any results if so we save it to global memory
    
    
        //        }












                sync(cta);
                // now we have metadata linear coordinate and information is it gold or segm pass ...

                //loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                //validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep,  oldRef, blockFpConter, blockFnConter);

                //////on the basis of isAnythingInPadding we will mark  the neighbouring block as to be activated if there is and if such neighbouring block exists
                //auto activeC = coalesced_threads();

                //if (localWorkQueue[i][3] == 1) {//gold
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                //};
                //// marking blocks as full 

                //if (localWorkQueue[i][3] == 1) {//gold
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                //};
                //sync(cta);// all results that should be saved to result list are saved                        

                ////we need to clear isAnythingInPadding to 0
                //clearisAnythingInPadding(isAnythingInPadding);
            }
        }
    }
    sync(cta);
    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
         atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
    };
    // in first thread block we zero work queue counter
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        if (blockIdx.x==0) {
            minMaxes[9] = 0;
        }
    };





}
//
//
//template <typename TKKI>
//inline __global__ void paddingDilatation(ForBoolKernelArgs<TKKI> fbArgs) {
//
//
//
//    thread_block cta = this_thread_block();
//    thread_block_tile<32> tile = tiled_partition<32>(cta);
//
//    char* tensorslice;
//    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
//    unsigned int old = 0;
//
//    // some references using as aliases
//    unsigned int& oldRef = old;
//
//
//
//    // main shared memory spaces 
//    __shared__ uint32_t sourceShared[32][32];
//    __shared__ uint32_t resShared[32][32];
//    // holding data about paddings 
//
//
//    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
//    __shared__ bool isAnythingInPadding[6];
//    //variables needed for all threads
//    __shared__ unsigned int iterationNumb[1];
//    __shared__ unsigned int globalWorkQueueOffset[1];
//    __shared__ unsigned int globalWorkQueueCounter[1];
//    __shared__ unsigned int localWorkQueueCounter[1];
//    __shared__ bool isBlockToBeValidated[1];
//    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters
//
//    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
//    //counters for per block number of results added in this iteration
//    __shared__ unsigned int localFpConter[1];
//    __shared__ unsigned int localFnConter[1];
//
//    __shared__ unsigned int blockFpConter[1];
//    __shared__ unsigned int blockFnConter[1];
//
//    //result list offset - needed to know where to write a result in a result list
//    __shared__ unsigned int resultfpOffset[1];
//    __shared__ unsigned int resultfnOffset[1];
//
//    __shared__ unsigned int worQueueStep[1];
//
//    // we will load here multiple entries from workqueue
//    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
//    //initializations and loading    
//    auto active = coalesced_threads();
//    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
//    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
//    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number
//
//    if (isToBeExecutedOnActive(active, 3)) {
//        localWorkQueueCounter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 4)) {
//        blockFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 5)) {
//        blockFnConter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 6)) {
//        localFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 7)) {
//        localFnConter[0] = 0;
//    };
//
//
//
//    if (isToBeExecutedOnActive(active, 1)) {
//        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
//        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
//        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
//    };
//    sync(cta);
//    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
//    /// load work QueueData into shared memory 
//
//    //TODO change looping so it will access contigous memory
//    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
//        // grid stride loop - sadly most of threads will be idle 
//        ///////////// loading to work queue
//        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);
//
//        sync(cta);// now local work queue is populated 
//
//            //now all of the threads in the block needs to have the same i value so we will increment by 1
//        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
//            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
//
//
//
//                //TODO() remove
//       /*         auto activee = coalesced_threads();
//                if (isToBeExecutedOnActive(activee, 3)) {
//                    printf("\n in padding looping  xMeta %d yMeta %d zMeta %d isGold %d \n"
//                        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
//                };*/
//
//
//
//                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
//                // so we do not need iteration loop 
//
//                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);
//
//                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
//                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, oldRef, blockFpConter, blockFnConter);
//
//                sync(cta);
//            }
//        }
//    }
//    sync(cta);
//    //     updating global counters
//    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);
//
//
//    //KROWA!!!
//    //remember to zero out the global work queue counter
//    //and inccrement iterationNumb[1]
//}
