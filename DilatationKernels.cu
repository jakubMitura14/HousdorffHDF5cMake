#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




//template <typename TKKI, typename forPipeline >
template <typename TKKI >
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI>& fbArgs, uint32_t* mainArrAPointer,
    uint32_t* mainArrBPointer, MetaDataGPU& metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint32_t* resultListPointerLocal, uint32_t* resultListPointerIterNumb,
    thread_block& cta, thread_block_tile<32>& tile, grid_group& grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6], bool isBlockFull[], int iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint32_t localBlockMetaData[40], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint32_t* metaDataArr, bool iasAnyProcessed[1],
    bool isGoldForLocQueue[localWorkQueLength], bool isBlockToBeValidated[1]
    , cuda::pipeline<cuda::thread_scope_block>& pipeline, cuda::aligned_size_t<128Ui64>& bigShape
    , cuda::aligned_size_t<128Ui64>& thirdRegShape
) {

    //initial cleaning  and initializations include loading min maxes
    dilBlockInitialClean(tile, isPaddingPass, iterationNumb, localWorkQueueCounter, blockFpConter,
        blockFnConter, localFpConter, localFnConter, isBlockFull, fpFnLocCounter,
        iasAnyProcessed, localTotalLenthOfWorkQueue, globalWorkQueueOffset
        , worQueueStep, minMaxes, localMinMaxes);
    sync(cta);
    /// load work QueueData into shared memory 
    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        loadWorkQueue(mainShmem, workQueue, isGoldForLocQueue, bigloop, worQueueStep);

        //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
////##### pipeline Step 0



        sync(cta);
        //loading metadata
        pipeline.producer_acquire();

        cuda::memcpy_async(cta, (&localBlockMetaData[20]),
            (&metaDataArr[(mainShmem[startOfLocalWorkQ ])
                * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

        cuda::memcpy_async(cta, (&localBlockMetaData[0]),
            (&metaDataArr[(mainShmem[startOfLocalWorkQ])
                * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

        //loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 0, 0);

        pipeline.producer_commit();


        //if (tile.thread_rank() < 20 && tile.meta_group_rank() == 2) {
        //    localBlockMetaData[(i & 1) * 20+tile.thread_rank()] =
        //        metaDataArr[(mainShmem[startOfLocalWorkQ ])
        //        * metaData.metaDataSectionLength + tile.thread_rank()];
        //};




        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                //if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {                      
                //    printf("linMeta beg %d is gold %d\n ", mainShmem[startOfLocalWorkQ + i], isGoldForLocQueue[i]);
                //};

//////////////// step 0  load main data and final processing of previous block
               //loading main data for first dilatation
                loadMain(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);
                
                
                pipeline.consumer_wait();
               // mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);



                pipeline.consumer_release();
///////// step 1 load top and process main data 
               //load top 
                loadTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);
                //process main
                processMain(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);                
///////// step 2 load bottom and process top 
                loadBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                //process top
                processTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);                     
///////// step 3 load right  process bottom  
                loadRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                //process bototm
                processBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 4 load left process right  
                loadLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 5 load anterior process left 
                loadAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 6 load posterior process anterior 
                loadPosterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 7 
                //load reference if needed or data for next iteration if there is such 
                //process posterior, save data from res shmem to global memory also we mark weather block is full
                lastLoad(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, origArrs, worQueueStep);
                processPosteriorAndSaveResShmem(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, isBlockFull);

 ////////// step 8 basically in order to complete here anyting the count need to be bigger than counter
                //loading for next block if block is not to be validated it was already done earlier
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                  > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                    if (i + 1 <= worQueueStep[0]) {
                        loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 1, i);
                    }
                }
                pipeline.producer_commit();
                


                pipeline.consumer_wait();


                //dilatateHelperForTransverse((threadIdx.y == 0), 5
                //    , (0), (-1), mainShmem, isAnythingInPadding
                //    , 0, threadIdx.x // we add offset depending on y dimension
                //    , 18, begSecRegShmem, localBlockMetaData);

                // //TODO remove
                // getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                //     + threadIdx.x + threadIdx.y * 32]
                //     = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                pipeline.consumer_release();
//////////

      



              //  ////////#### pipeline step 6) if block is to be validated we process the res and reference data and start loading data for begining of the next loop
              // sync(cta);
              //  ////load data for next iteration
              //  if (i + 1 <= worQueueStep[0]) {
              //      pipeline.producer_acquire();
              //      cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
              //          mainShmem[startOfLocalWorkQ + i+1] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i+1])],
              //          bigShape, pipeline);
              //      pipeline.producer_commit();
              //  
              //   }


              //  // now all of the data is processed we need to save it into global memory
              //  // TODO try to use mempcy async here
              //  //if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]>0) {
              //  getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
              //      + threadIdx.x + threadIdx.y * 32]
              //      = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
              //  //}


              //  if (localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
              //          > localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
              //      mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



              //      //we now look for bits prasent in both reference arrays and current one
              //      mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);


              //      // now we look through bits and when some is set we call it a result 
              //      #pragma unroll
              //      for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
              //          //if any bit here is set it means it should be added to result list 
              //          if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
              //              //first we add to the resList
              //              //TODO consider first passing it into shared memory and then async mempcy ...
              //              //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
              //              unsigned int old = 0;
              //              ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
              //              if (isGoldForLocQueue[i]) {
              //                  old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20+6] + localBlockMetaData[(i & 1) * 20+4];
              //              }
              //              else {
              //                  old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20+5] + localBlockMetaData[(i & 1) * 20+3];
              //              };
              //              //   add results to global memory    
              //              //we add one gere jjust to distinguish it from empty result
              //              resultListPointerMeta[old] = uint32_t(mainShmem[startOfLocalWorkQ + i] +(isGoldOffset * isGoldForLocQueue[i])+1);
              //              resultListPointerLocal[old] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x) );
              //              resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);

              //           /*   printf("rrrrresult i %d  meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d  xLoc %d yLoc %d zLoc %d \n"
              //                  ,i
              //                  ,mainShmem[startOfLocalWorkQ + i]
              //                  , isGoldForLocQueue[i]
              //                  , old
              //                  , localFpConter[0]
              //                  , localFnConter[0]
              //                  , localBlockMetaData[(i & 1) * 20+ 5]
              //                  , localBlockMetaData[(i & 1) * 20+6]
              //                  , uint32_t(mainShmem[startOfLocalWorkQ + i] + isGoldOffset * isGoldForLocQueue[i])
              //                  , uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x))
              //                  , threadIdx.x
              //                  , threadIdx.y
              //                  , bitPos
              //              );*/

              //          }

              //      };
              //sync(cta);
              //  }
              //      //loading metadaa for next loop 
              //      if (i + 1 <= worQueueStep[0]) {
              //          if (tile.thread_rank() < 20 && tile.meta_group_rank() == 2) {
              //                  localBlockMetaData[(i & 1) * 20+tile.thread_rank()] = 
              //                  metaDataArr[(mainShmem[startOfLocalWorkQ + i + 1])
              //                      * metaData.metaDataSectionLength + tile.thread_rank()];
              //          };
              //      }

              //      //finilizing
              //      afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i,
              //          metaData, tile, localFpConter, localFnConter
              //          , blockFpConter, blockFnConter
              //          , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue);


              //  sync(cta);


            }
        }
    }

    //here we are after all of the blocks planned to be processed by this block are
    sync(cta);

    //updating local counters of last local block (normally it is done at the bagining of the next block)
    //but we need to check weather any block was processed at all
    //if (iasAnyProcessed[0]) {
    //    afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, 1,
    //        metaData, tile, localFpConter, localFnConter
    //        , blockFpConter, blockFnConter
    //        , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue);
    //}

    ////     updating global counters
    //if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
    //    if (blockFpConter[0] > 0) {
    //        atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
    //    }
    //};
    //if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
    //    if (blockFnConter[0] > 0) {
    //        atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
    //    }
    //};
    //// in first thread block we zero work queue counter
    //if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
    //    if (blockIdx.x == 0) {
    //        minMaxes[9] = 0;
    //    }
    //};


}
