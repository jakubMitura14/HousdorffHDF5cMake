#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




template <typename TKKI>
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArrAPointer,
    uint32_t* mainArrBPointer, MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint16_t* resultListPointerLocal, uint32_t* resultListPointerIterNumb,
    thread_block cta, thread_block_tile<32> tile, grid_group grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6]  , bool isBlockFull[1], uint32_t iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint16_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint16_t localBlockMetaData[20], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint16_t* metaDataArr, uint32_t oldIsGold[1], uint16_t oldLinIndM[1], uint16_t localBlockMetaDataOld[6]
) {




    auto pipeline = cuda::make_pipeline();
    auto bigShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength));
    auto thirdRegShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (32));
    thread_block_tile<1> miniTile = tiled_partition<1>(cta);


    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0  && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] =true;
    };
    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }




    sync(cta);
    /// load work QueueData into shared memory 
    for (uint16_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        
        cooperative_groups::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop]), cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]));
        sync(cta);
        //now all of the threads in the block needs to have the same i value so we will increment by 1
        // we are preloading to the pipeline block metaData
        ////##### pipeline Step 0
        pipeline.producer_acquire();

        cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&metaDataArr[(mainShmem[startOfLocalWorkQ] - UINT16_MAX * (mainShmem[startOfLocalWorkQ] >= UINT16_MAX)) * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint16_t) * 20), pipeline);

        pipeline.producer_commit();
        
        for (uint16_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                 ///#### pipeline step 1) now we load data for next step (to mainly sourceshmem and left-right if apply) and process data loaded in previous step
                    pipeline.producer_acquire();

                    cuda::memcpy_async(cta, (&mainShmem[(((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) > 0) * (-32)) + begSourceShmem]), // we check weather there is anything to the left - not on left border if so we need place for left 32 entries
                        &getSourceReduced(fbArgs,iterationNumb )[((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) *(-32)) // we check weather there is anything to the left - not on left border if so we load left 32 entries
                        +  getIndexForSourceShmem(metaData, mainShmem, iterationNumb,i )] , 
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * //below we check weather we have block to the left and right if so we increase number of copied entries
                            (metaData.mainArrXLength+32*(((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) > 0)
                                + ((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) <(metaData.totalMetaLength -1)))   ))
                        , pipeline);


        //        ////we need to do the cleaning after previous block .. compute first we load data about calculated linear index meta and information is it gold iteration ...
                   pipeline.consumer_wait();
                
                   if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                       isGold[0] = (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX);

                   }
                   if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                       
                       currLinIndM[0] = mainShmem[startOfLocalWorkQ + i] - UINT16_MAX* (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX);

                   }

                   afterBlockClean(cta, worQueueStep, localBlockMetaDataOld, mainShmem, i, metaData
                       , tile, localFpConter, localFnConter
                       , blockFpConter, blockFnConter, metaDataArr, oldLinIndM, oldIsGold
                       , isAnythingInPadding, isBlockFull)


                   pipeline.consumer_release();

                   ////////#### pipeline step 2)  load block from top and process center that is in source shmem; and both smallRegShmems
                   //load for next step - so we load block to the top
                               if (localBlockMetaData[13] < UINT16_MAX ) {
                                   pipeline.producer_acquire();

                                   cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                       &getSourceReduced(fbArgs,iterationNumb )[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 13 )], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                                           cuda::aligned_size_t<128>(sizeof(uint32_t)* metaData.mainArrXLength)
                                       , pipeline);

                                   pipeline.producer_commit();
                               }
                       
                         //compute - now we have data in source shmem about this block and left and right padding and we need to process it 
                           pipeline.consumer_wait();
                               // first we perform up and down dilatations inside the block
                               mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = bitDilatate(mainShmem[begSourceShmem+threadIdx.x + threadIdx.y * 32]);
                               //we also do the left and right dilatations
                               //left
                               dilatateHelperForTransverse((threadIdx.x == 0),
                                       2, (-1), (0), mainShmem, isAnythingInPadding
                                       , 0, threadIdx.y + (32 - fbArgs.dbYLength) // we add offset depending on y dimension
                                       , 15, begSMallRegShmemA, localBlockMetaData);
                                       
                               //right
                               dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
                                       3, (1), (0), mainShmem, isAnythingInPadding
                                       , 0, threadIdx.y
                                       , 16, begSMallRegShmemB, localBlockMetaData);

                               ///////////saving old
                               //additionally we save previous copies of data so refreshing will keep easier
                               if (tile.thread_rank() <6 && tile.meta_group_rank() == 0) {
                                   localBlockMetaDataOld[tile.thread_rank()] = localBlockMetaData[13+ tile.thread_rank()];
                               }
                               if (tile.thread_rank() ==6 && tile.meta_group_rank() == 0) {
                                   oldIsGold[0] == isGold[0];
                               }
                               if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0) {
                                   oldLinIndM[0] == currLinIndM[0];
                               }

                           pipeline.consumer_release();
            ////////#### pipeline step 3) we load bottom, anterior and posterior and we process top
                  //load anterior and posterior and bottom
                      pipeline.producer_acquire();
                          //posterior of the block to anterior we load it using single threads and multple mempcy async becouse memory is non aligned
                          if (localBlockMetaData[17] < UINT16_MAX  && miniTile.meta_group_rank()< fbArgs.dbXLength) {

                              cuda::memcpy_async(miniTile, (&mainShmem[begSMallRegShmemA + miniTile.meta_group_rank()]), 
                                  &getSourceReduced(fbArgs,iterationNumb )[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 17) //basic offset
                                  + miniTile.meta_group_rank() * 32 ], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                                  cuda::aligned_size_t<4>(sizeof(uint32_t))
                                  , pipeline);

                          }
                         // anterior of the block to posterior
                          if (localBlockMetaData[18] < UINT16_MAX && miniTile.meta_group_rank()>= fbArgs.dbXLength    && miniTile.meta_group_rank()< fbArgs.dbXLength*2) {
                             cooperative_groups::memcpy_async(miniTile, (&mainShmem[begSMallRegShmemB+ miniTile.meta_group_rank() ]),
                                 (&getSourceReduced(fbArgs,iterationNumb )[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 18)
                                     //we look for indicies 31,63... up to metaData.mainArrXLength
                                     + (miniTile.meta_group_rank() * 32)+31  ])
                                 , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
                          }
                          //bottom  block
                          if (localBlockMetaData[14] < UINT16_MAX){
                          cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                              &getSourceReduced(fbArgs,iterationNumb )[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 14 )], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                                 cuda::aligned_size_t<128>(sizeof(uint32_t)* metaData.mainArrXLength)
                                       , pipeline); 
                           }            
                                       
                      pipeline.producer_commit();
                                     
                  
                //    compute - now we have data in source shmem about block to the top
               pipeline.consumer_wait();
               dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13,
                   , 1// represent a uint32 number that has a bit of intrest in this block set and all others 0 here first bit is set
                   , 2147483648
                   , begfirstRegShmem);
              pipeline.consumer_release(); 
            ////////#### pipeline step 5) if block is to be validated we load reference data and we process bottom, left and right
            //load reference data if block is to be validated otherwise if it is not the last step in the loop we load data for next loop
            pipeline.producer_acquire();
              if( localBlockMetaData[((1-isGold[0])+1)] //fp for gold and fn count for not gold
                   > localBlockMetaData[((1-isGold[0])+1)]   ){// so count is bigger than counter so we should validate
               //now we load data from reference arrays 
                  cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                     &origArrs[metaData.mainArrXLength*(isGold[0]) ], //we look for 
                      cuda::aligned_size_t<128>(sizeof(uint32_t)* metaData.mainArrXLength)
                       , pipeline);    }                       

               }else{//if we are not validating we immidiately start loading data for next loop
                   lastLoad(cta,worQueueStep, localBlockMetaData, mainArr, mainShmem, i, metaData);
               }
             //    compute - now we have data in source shmem about block to the bottom, left and right
         
            pipeline.producer_commit();
               pipeline.consumer_wait();
               //bottom
               dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, pipeline,localBlockMetaData,14, 
                       , 2147483648// represent a uint32 number that has a bit of intrest in this block set and all others 0 here last bit is set
                       , 1
                       ,begSecRegShmem)
               //posterior
                 dilatateHelperForTransverse((threadIdx.y == 0), 5
                 , (0), (-1), mainShmem, isAnythingInPadding
                       , 0, threadIdx.x // we add offset depending on y dimension
                        , 18, begSMallRegShmemB, localBlockMetaData);
                                       
             //anterior
             dilatateHelperForTransverse((threadIdx.y == (fbArgs.dbYLength - 1)), 4
             , (0), (1), mainShmem, isAnythingInPadding
               , 0, threadIdx.x
               , 17, begSMallRegShmemA, localBlockMetaData);         
             //now all of the data is processed we need to save it into global memory
             //TODO try to use mempcy async here
              getTargetReduced(fbArgs,iterationNumb )[getIndexForSaveResShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData)+threadIdx.x+threadIdx.y*32]= mainShmem[begResShmem+threadIdx.x+threadIdx.y*32];
              //setting information about is block full
              if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] != UINT32_MAX) {
                  isBlockFull = false
              }
              pipeline.consumer_release(); 


 ////////#### pipeline step 6) if block is to be validated we process the res and reference data and start loading data for begining of the next loop

              if (localBlockMetaData[((1 - isGold[0]) + 1)] //fp for gold and fn count for not gold
                          > localBlockMetaData[((1 - isGold[0]) + 1)]) {// so count is bigger than counter so we should validate
              //load data for next iteration
                  pipeline.producer_acquire();
                    lastLoad(cta, worQueueStep, localBlockMetaData, mainArr, mainShmem, i, metaData);
                  pipeline.producer_commit();

                  //process check is there any new result (we have reference in begfirstRegShmem)
                           //now first we need to check for bits that are true now after dilatation but were not in source we will save it in res shmem becouse we will no longer need it
                  pipeline.consumer_wait();
                      mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);


                      //we now look for bits prasent in both reference arrays and current one
                      mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

                      // now we look through bits and when some is set we call it a result 
                      for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                          //if any bit here is set it means it should be added to result list 
                          if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                              //first we add to the resList
                              //TODO consider first passing it into shared memory and then async mempcy ...
                              //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                              unsigned int old = 0;
                              ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                              if (isGold[0]) { old = atomicAdd(&(localFpConter[0]), 1) + localBlockMetaData[6]; }
                              else { old = atomicAdd(&(localFnConter[0]), 1) + localBlockMetaData[5]; };
                              //   add results to global memory    
                              resultListPointerMeta = currLinIndM[0] + UINT16_MAX * isGold[0];
                              resultListPointerLocal = (fbArgs.dbYLength * 32 * bitPos + threadIdx.y * 32 + threadIdx.x);
                              resultListPointerIterNumb = iterationNumb[0];
                          }
                      }
                   pipeline.consumer_release();

              };

 //here we are in a spot where all block from 

 //if (localWorkQueue[i][3] == 1) {//gold
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                //};
                //// marking blocks as full 

                //if (localWorkQueue[i][3] == 1) {//gold
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                //};
                //sync(cta);// all results that should be saved to result list are saved                        

                ////we need to clear isAnythingInPadding to 0
                //clearisAnythingInPadding(isAnythingInPadding);
            
  }


        
    }

    //here we are after all of the blocks planned to be processed by this block are
    sync(cta);

    //updating local counters of last local block (normally it is done at the bagining of the next block)
    afterBlockClean(cta,worQueueStep, localBlockMetaDataOld   , mainShmem, globalWorkQueueOffset[0] , metaData
        , tile , localFpConter,localFnConter
        ,  blockFpConter, blockFnConter ,metaDataArr, oldLinIndM, oldIsGold
        , isAnythingInPadding, isBlockFull)




    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
         atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
    };
    // in first thread block we zero work queue counter
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        if (blockIdx.x==0) {
            minMaxes[9] = 0;
        }
    };





}
//
//
//template <typename TKKI>
//inline __global__ void paddingDilatation(ForBoolKernelArgs<TKKI> fbArgs) {
//
//
//
//    thread_block cta = this_thread_block();
//    thread_block_tile<32> tile = tiled_partition<32>(cta);
//
//    char* tensorslice;
//    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
//    unsigned int old = 0;
//
//    // some references using as aliases
//    unsigned int& oldRef = old;
//
//
//
//    // main shared memory spaces 
//    __shared__ uint32_t sourceShared[32][32];
//    __shared__ uint32_t resShared[32][32];
//    // holding data about paddings 
//
//
//    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
//    __shared__ bool isAnythingInPadding[6];
//    //variables needed for all threads
//    __shared__ unsigned int iterationNumb[1];
//    __shared__ unsigned int globalWorkQueueOffset[1];
//    __shared__ unsigned int globalWorkQueueCounter[1];
//    __shared__ unsigned int localWorkQueueCounter[1];
//    __shared__ bool isBlockToBeValidated[1];
//    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters
//
//    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
//    //counters for per block number of results added in this iteration
//    __shared__ unsigned int localFpConter[1];
//    __shared__ unsigned int localFnConter[1];
//
//    __shared__ unsigned int blockFpConter[1];
//    __shared__ unsigned int blockFnConter[1];
//
//    //result list offset - needed to know where to write a result in a result list
//    __shared__ unsigned int resultfpOffset[1];
//    __shared__ unsigned int resultfnOffset[1];
//
//    __shared__ unsigned int worQueueStep[1];
//
//    // we will load here multiple entries from workqueue
//    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
//    //initializations and loading    
//    auto active = coalesced_threads();
//    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
//    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
//    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number
//
//    if (isToBeExecutedOnActive(active, 3)) {
//        localWorkQueueCounter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 4)) {
//        blockFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 5)) {
//        blockFnConter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 6)) {
//        localFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 7)) {
//        localFnConter[0] = 0;
//    };
//
//
//
//    if (isToBeExecutedOnActive(active, 1)) {
//        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
//        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
//        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
//    };
//    sync(cta);
//    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
//    /// load work QueueData into shared memory 
//
//    //TODO change looping so it will access contigous memory
//    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
//        // grid stride loop - sadly most of threads will be idle 
//        ///////////// loading to work queue
//        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);
//
//        sync(cta);// now local work queue is populated 
//
//            //now all of the threads in the block needs to have the same i value so we will increment by 1
//        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
//            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
//
//
//
//                //TODO() remove
//       /*         auto activee = coalesced_threads();
//                if (isToBeExecutedOnActive(activee, 3)) {
//                    printf("\n in padding looping  xMeta %d yMeta %d zMeta %d isGold %d \n"
//                        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
//                };*/
//
//
//
//                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
//                // so we do not need iteration loop 
//
//                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);
//
//                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
//                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, oldRef, blockFpConter, blockFnConter);
//
//                sync(cta);
//            }
//        }
//    }
//    sync(cta);
//    //     updating global counters
//    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);
//
//
//    //KROWA!!!
//    //remember to zero out the global work queue counter
//    //and inccrement iterationNumb[1]
//}
