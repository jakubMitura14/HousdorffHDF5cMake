#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"

using namespace cooperative_groups;

template <typename TKKI>
inline __global__ void mainDilatation(ForBoolKernelArgs<TKKI> fbArgs) {


    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);

    char* tensorslice;
    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
    unsigned int old = 0;

    // some references using as aliases
    unsigned int& oldRef = old;



    // main shared memory spaces 
    __shared__ uint32_t sourceShared[32][32];
    __shared__ uint32_t resShared[32][32];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];
    //variables needed for all threads
    __shared__ unsigned int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    __shared__ bool isBlockToBeValidated[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];

    // we will load here multiple entries from workqueue
    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
    //initializations and loading    
    auto active = coalesced_threads();
    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number

    if (isToBeExecutedOnActive(active, 3)) {
        localWorkQueueCounter[0] = 0;
    };

    if (isToBeExecutedOnActive(active, 4)) {
        blockFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 5)) {
        blockFnConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 6)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 7)) {
        localFnConter[0] = 0;
    };




    if (isToBeExecutedOnActive(active, 1)) {
        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    sync(cta);
    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
    /// load work QueueData into shared memory 

    //TODO change looping so it will access contigous memory
    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        ///////////// loading to work queue
        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);

        sync(cta);// now local work queue is populated 

            //now all of the threads in the block needs to have the same i value so we will increment by 1
        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
                // so we do not need iteration loop 

                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                /////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep,  oldRef, blockFpConter, blockFnConter);

                ////on the basis of isAnythingInPadding we will mark  the neighbouring block as to be activated if there is and if such neighbouring block exists
                auto activeC = coalesced_threads();

                if (localWorkQueue[i][3] == 1) {//gold
                    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                };
                if (localWorkQueue[i][3] == 0) {//segm
                    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                };
                // marking blocks as full 

                if (localWorkQueue[i][3] == 1) {//gold
                    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                };
                if (localWorkQueue[i][3] == 0) {//segm
                    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                };
                sync(cta);// all results that should be saved to result list are saved                        

                //we need to clear isAnythingInPadding to 0
                clearisAnythingInPadding(isAnythingInPadding);
            }
        }
    }
    sync(cta);
    //     updating global counters
    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);


}




template <typename TKKI>
inline __global__ void paddingDilatation(ForBoolKernelArgs<TKKI> fbArgs) {



    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);

    char* tensorslice;
    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
    unsigned int old = 0;

    // some references using as aliases
    unsigned int& oldRef = old;



    // main shared memory spaces 
    __shared__ uint32_t sourceShared[32][32];
    __shared__ uint32_t resShared[32][32];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];
    //variables needed for all threads
    __shared__ unsigned int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    __shared__ bool isBlockToBeValidated[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];

    // we will load here multiple entries from workqueue
    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
    //initializations and loading    
    auto active = coalesced_threads();
    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number

    if (isToBeExecutedOnActive(active, 3)) {
        localWorkQueueCounter[0] = 0;
    };

    if (isToBeExecutedOnActive(active, 4)) {
        blockFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 5)) {
        blockFnConter[0] = 0;
    };

    if (isToBeExecutedOnActive(active, 6)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(active, 7)) {
        localFnConter[0] = 0;
    };



    if (isToBeExecutedOnActive(active, 1)) {
        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    sync(cta);
    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
    /// load work QueueData into shared memory 

    //TODO change looping so it will access contigous memory
    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        ///////////// loading to work queue
        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);

        sync(cta);// now local work queue is populated 

            //now all of the threads in the block needs to have the same i value so we will increment by 1
        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {



                //TODO() remove
       /*         auto activee = coalesced_threads();
                if (isToBeExecutedOnActive(activee, 3)) {
                    printf("\n in padding looping  xMeta %d yMeta %d zMeta %d isGold %d \n"
                        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
                };*/



                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
                // so we do not need iteration loop 

                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, oldRef, blockFpConter, blockFnConter);

                sync(cta);
            }
        }
    }
    sync(cta);
    //     updating global counters
    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);



}