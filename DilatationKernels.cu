#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;

constexpr auto startOfLocalWorkQ = 372;


template <typename TKKI>
inline __global__ void mainDilatation(ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArr, MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint16_t* resultListPointerLocal, uint16_t* resultListPointerIterNumb) {


    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);
    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;

    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces 
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-4468 (372 length) : place for local work queue in dilatation kernels
    */ 
    __shared__ uint32_t mainShmem[4468];
    // holding data about paddings 
        

    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];
    //variables needed for all threads
    __shared__ uint32_t iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    __shared__ bool isBlockToBeValidated[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];
    __shared__ uint32_t isGold[1];

    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count; 
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index 
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */
    __shared__ unsigned int localBlockMetaData[19];
    
    //initializations and loading    
    if (tile.thread_rank() == 5 && tile.meta_group_rank() == 0) { iterationNumb[0] = 0; };
    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 5 && tile.meta_group_rank() == 0) {
        init(&barrier, blockDim.x * blockDim.y);
    };







     if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
     /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
     if (tile.meta_group_rank() == 1) {
         cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
     }

    sync(cta);
    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
    /// load work QueueData into shared memory 

    //TODO change looping so it will access contigous memory
    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        ///////////// loading to work queue
        cooperative_groups::memcpy_async(cta, (&mainShmem[4096]), (&workQueue[bigloop]), cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]));
        sync(cta);

            //now all of the threads in the block needs to have the same i value so we will increment by 1
        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                //preparations fo block 
                if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                    isGold[0] = uint32_t(mainShmem[startOfLocalWorkQ+i] >= UINT16_MAX);
                    if (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX) {
                        //removing info about wheather it is gold or not pass so we will be able to use it as linear metadata index
                        mainShmem[startOfLocalWorkQ + i] = mainShmem[startOfLocalWorkQ + i] - UINT16_MAX;
                    }
                };
                tile.sync();
                //load meta data of a block to shared memory
                if (tile.meta_group_rank() == 1) {
                    //mainShmem[startOfLocalWorkQ + i] is linindexMeta - linear index pointing out to what metadata block it is
                    cooperative_groups::memcpy_async(tile, (&localBlockMetaData[0]), (&mainArr[  mainShmem[startOfLocalWorkQ + i]*metaData.mainArrSectionLength + metaData.metaDataOffset])
                        , cuda::aligned_size_t<4>(sizeof(uint32_t) *18));
                }


                sync(cta);
                // now we have metadata linear coordinate and information is it gold or segm pass ...

                //loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                //validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep,  oldRef, blockFpConter, blockFnConter);

                //////on the basis of isAnythingInPadding we will mark  the neighbouring block as to be activated if there is and if such neighbouring block exists
                //auto activeC = coalesced_threads();

                //if (localWorkQueue[i][3] == 1) {//gold
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                //};
                //// marking blocks as full 

                //if (localWorkQueue[i][3] == 1) {//gold
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                //};
                //sync(cta);// all results that should be saved to result list are saved                        

                ////we need to clear isAnythingInPadding to 0
                //clearisAnythingInPadding(isAnythingInPadding);
            }
        }
    }
    sync(cta);
    //     updating global counters
//    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);


}


//
//
//template <typename TKKI>
//inline __global__ void paddingDilatation(ForBoolKernelArgs<TKKI> fbArgs) {
//
//
//
//    thread_block cta = this_thread_block();
//    thread_block_tile<32> tile = tiled_partition<32>(cta);
//
//    char* tensorslice;
//    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
//    unsigned int old = 0;
//
//    // some references using as aliases
//    unsigned int& oldRef = old;
//
//
//
//    // main shared memory spaces 
//    __shared__ uint32_t sourceShared[32][32];
//    __shared__ uint32_t resShared[32][32];
//    // holding data about paddings 
//
//
//    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
//    __shared__ bool isAnythingInPadding[6];
//    //variables needed for all threads
//    __shared__ unsigned int iterationNumb[1];
//    __shared__ unsigned int globalWorkQueueOffset[1];
//    __shared__ unsigned int globalWorkQueueCounter[1];
//    __shared__ unsigned int localWorkQueueCounter[1];
//    __shared__ bool isBlockToBeValidated[1];
//    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters
//
//    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
//    //counters for per block number of results added in this iteration
//    __shared__ unsigned int localFpConter[1];
//    __shared__ unsigned int localFnConter[1];
//
//    __shared__ unsigned int blockFpConter[1];
//    __shared__ unsigned int blockFnConter[1];
//
//    //result list offset - needed to know where to write a result in a result list
//    __shared__ unsigned int resultfpOffset[1];
//    __shared__ unsigned int resultfnOffset[1];
//
//    __shared__ unsigned int worQueueStep[1];
//
//    // we will load here multiple entries from workqueue
//    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
//    //initializations and loading    
//    auto active = coalesced_threads();
//    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
//    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
//    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number
//
//    if (isToBeExecutedOnActive(active, 3)) {
//        localWorkQueueCounter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 4)) {
//        blockFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 5)) {
//        blockFnConter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 6)) {
//        localFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 7)) {
//        localFnConter[0] = 0;
//    };
//
//
//
//    if (isToBeExecutedOnActive(active, 1)) {
//        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
//        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
//        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
//    };
//    sync(cta);
//    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
//    /// load work QueueData into shared memory 
//
//    //TODO change looping so it will access contigous memory
//    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
//        // grid stride loop - sadly most of threads will be idle 
//        ///////////// loading to work queue
//        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);
//
//        sync(cta);// now local work queue is populated 
//
//            //now all of the threads in the block needs to have the same i value so we will increment by 1
//        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
//            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
//
//
//
//                //TODO() remove
//       /*         auto activee = coalesced_threads();
//                if (isToBeExecutedOnActive(activee, 3)) {
//                    printf("\n in padding looping  xMeta %d yMeta %d zMeta %d isGold %d \n"
//                        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
//                };*/
//
//
//
//                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
//                // so we do not need iteration loop 
//
//                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);
//
//                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
//                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, oldRef, blockFpConter, blockFnConter);
//
//                sync(cta);
//            }
//        }
//    }
//    sync(cta);
//    //     updating global counters
//    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);
//
//
//    //KROWA!!!
//    //remember to zero out the global work queue counter
//    //and inccrement iterationNumb[1]
//}