#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




//template <typename TKKI, typename forPipeline >
template <typename TKKI >
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI>& fbArgs, uint32_t* mainArrAPointer,
    uint32_t* mainArrBPointer, MetaDataGPU& metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint32_t* resultListPointerLocal, uint32_t* resultListPointerIterNumb,
    thread_block& cta, thread_block_tile<32>& tile, grid_group& grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6], bool isBlockFull[1], int iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint32_t localBlockMetaData[], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint32_t* metaDataArr, bool iasAnyProcessed[1],
    bool isGoldForLocQueue[localWorkQueLength], bool isBlockToBeValidated[1]
    , cuda::pipeline<cuda::thread_scope_thread>& pipeline, cuda::aligned_size_t<128Ui64>& bigShape
    , cuda::aligned_size_t<128Ui64>& thirdRegShape
) {

    //initial cleaning  and initializations include loading min maxes
    dilBlockInitialClean(tile, isPaddingPass, iterationNumb, localWorkQueueCounter, blockFpConter,
        blockFnConter, localFpConter, localFnConter, isBlockFull, fpFnLocCounter,
        iasAnyProcessed, localTotalLenthOfWorkQueue, globalWorkQueueOffset
        , worQueueStep, minMaxes, localMinMaxes);
    sync(cta);
    /// load work QueueData into shared memory 
    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        loadWorkQueue(mainShmem, workQueue, isGoldForLocQueue, bigloop, worQueueStep);

        //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
////##### pipeline Step 0









        //if (i + 1 <= worQueueStep[0]) {
        //    if (tile.thread_rank() < 20 && tile.meta_group_rank() == 0) {

        //        localBlockMetaData[20 * (i & 1) + tile.thread_rank()] =
        //            metaDataArr[(mainShmem[startOfLocalWorkQ + i + 1])
        //            * metaData.metaDataSectionLength + tile.thread_rank()];
        //    };
        //}



        sync(cta);

        pipeline.producer_acquire();

        loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 0, 0);

        pipeline.producer_commit();

        //loading main data for first dilatation
        pipeline.producer_acquire();
        cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
            mainShmem[startOfLocalWorkQ] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[0])],
            bigShape, pipeline);
        pipeline.producer_commit();

        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
                    iasAnyProcessed[0] = true;
                }




                pipeline.producer_acquire();

                 if (mainShmem[startOfLocalWorkQ + i] < (metaData.totalMetaLength - 1)) {
                    cooperative_groups::memcpy_async(tile, (&mainShmem[begSMallRegShmemB]),
                        &getSourceReduced(fbArgs, iterationNumb)[
                            (mainShmem[startOfLocalWorkQ + i] + 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                                + tile.meta_group_rank() * 32], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                        cuda::aligned_size_t<4>(sizeof(uint32_t))
                                );
                }

                //load data of interst form block to the left
                if (mainShmem[startOfLocalWorkQ + i] > 0) {
                    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemA + tile.meta_group_rank()]),
                        &getSourceReduced(fbArgs, iterationNumb)[
                            (mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                                //we look for indicies 31,63... up to metaData.mainArrXLength
                                + (tile.meta_group_rank() * 32) + 31]
                        , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);

                }
                pipeline.producer_commit();



                // we need to do the cleaning after previous block .. compute first we load data about calculated linear index meta and information is it gold iteration ...

                //compute - now we have data in source shmem about this block and left and right padding and we need to process it 
                pipeline.consumer_wait();
                // first we perform up and down dilatations inside the block
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
                //TODO remove
                pipeline.consumer_release();


                ////////#### pipeline step 2)  load block from top and process center that is in source shmem; and both smallRegShmems
                               //load for next step - so we load block to the top
                pipeline.producer_acquire();

                if (localBlockMetaData[13] < isGoldOffset) {
                    pipeline.producer_acquire();
                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[13] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);

                }
                pipeline.producer_commit();

                //compute - now we have data in source shmem about this block and left and right padding and we need to process it 
                pipeline.consumer_wait();
                // first we perform up and down dilatations inside the block
            //if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0) {
            //    printf("source shmem linLocalInd %d  linMeta %d \n",(threadIdx.x + threadIdx.y * 32), mainShmem[startOfLocalWorkQ + ii] );
            //}
                 //we also do the left and right dilatations
                ////left
                dilatateHelperForTransverse((threadIdx.x == 0),
                    2, (-1), (0), mainShmem, isAnythingInPadding
                    , 0, threadIdx.y
                    , 15, begSMallRegShmemA, localBlockMetaData);

                //right
                dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
                    3, (1), (0), mainShmem, isAnythingInPadding
                    , 0, threadIdx.y
                    , 16, begSMallRegShmemB, localBlockMetaData);


                pipeline.consumer_release();
                ////////#### pipeline step 3) we load bottom, anterior and posterior and we process top
                                      //load anterior and posterior and bottom
                pipeline.producer_acquire();
                //block to anterior 
                if (localBlockMetaData[17] < isGoldOffset && tile.meta_group_rank() == 0) {

                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemA], &getSourceReduced(fbArgs, iterationNumb)[
                        (localBlockMetaData[17]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        thirdRegShape, pipeline);

                }
                // block to posterior
                if (localBlockMetaData[18] < isGoldOffset && tile.meta_group_rank() == 1) {
                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemB], &getSourceReduced(fbArgs, iterationNumb)[
                        (localBlockMetaData[18]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                            + (blockDim.y - 1) * 32// we need last 32 length entry of the posterior block 
                    ], thirdRegShape, pipeline);

                }

                //bottom  block
                if (localBlockMetaData[14] < isGoldOffset) {
                    cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[
                            localBlockMetaData[14] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                                , pipeline);
                }
                pipeline.producer_commit();



                //    compute - now we have data in source shmem about block to the top
                pipeline.consumer_wait();
                dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
                    , 31// represent a uint32 number that has a bit of intrest in this block set and all others 0 here first bit is set
                    , 0
                    , begfirstRegShmem);
                pipeline.consumer_release();
                ////////#### pipeline step 5) if block is to be validated we load reference data and we process bottom, left and right
                                //load reference data if block is to be validated otherwise if it is not the last step in the loop we load data for next loop
                pipeline.producer_acquire();
                if (localBlockMetaData[((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                     > localBlockMetaData[((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate            //now we load data from reference arrays 
                        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                            &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
                            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                            , pipeline);

                }
  
                //    compute - now we have data in source shmem about block to the bottom, left and right

                pipeline.producer_commit();

                pipeline.consumer_wait();
                //bottom
                dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
                    , 0// represent a uint32 number that has a bit of intrest in this block set and all others 0 here last bit is set
                    , 31
                    , begSecRegShmem);
                //posterior
                dilatateHelperForTransverse((threadIdx.y == 0), 5
                    , (0), (-1), mainShmem, isAnythingInPadding
                    , 0, threadIdx.x // we add offset depending on y dimension
                    , 18, begSMallRegShmemB, localBlockMetaData);
                //anterior
                dilatateHelperForTransverse((threadIdx.y == (fbArgs.dbYLength - 1)), 4
                    , (0), (1), mainShmem, isAnythingInPadding
                    , 0, threadIdx.x
                    , 17, begSMallRegShmemA, localBlockMetaData);


                // setting information about is block full


                if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] != UINT32_MAX) {
                    isBlockFull[0] = false;
                }
                pipeline.consumer_release();


                ////////#### pipeline step 6) if block is to be validated we process the res and reference data and start loading data for begining of the next loop
               sync(cta);
                ////load data for next iteration
                if (i + 1 <= worQueueStep[0]) {
                    pipeline.producer_acquire();
                    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
                        mainShmem[startOfLocalWorkQ + i+1] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i+1])],
                        bigShape, pipeline);
                    pipeline.producer_commit();
                
                }


                // now all of the data is processed we need to save it into global memory
                // TODO try to use mempcy async here
                //if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]>0) {
                getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                    + threadIdx.x + threadIdx.y * 32]
                    = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
                //}


                if (localBlockMetaData[((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                        > localBlockMetaData[((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



                    //we now look for bits prasent in both reference arrays and current one
                    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);


                    // now we look through bits and when some is set we call it a result 
                    #pragma unroll
                    for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                        //if any bit here is set it means it should be added to result list 
                        if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                            //first we add to the resList
                            //TODO consider first passing it into shared memory and then async mempcy ...
                            //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                            unsigned int old = 0;
                            ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                            if (isGoldForLocQueue[i]) {
                                old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[5] + localBlockMetaData[3];
                            }
                            else {
                                old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[6] + localBlockMetaData[4];
                            };
                            //   add results to global memory    
                            //we add one gere jjust to distinguish it from empty result
                            resultListPointerMeta[old] = uint32_t(mainShmem[startOfLocalWorkQ + i] +(isGoldOffset * isGoldForLocQueue[i])+1);
                            resultListPointerLocal[old] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x) );
                            resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);

                         /*   printf("rrrrresult i %d  meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d  xLoc %d yLoc %d zLoc %d \n"
                                ,i
                                ,mainShmem[startOfLocalWorkQ + i]
                                , isGoldForLocQueue[i]
                                , old
                                , localFpConter[0]
                                , localFnConter[0]
                                , localBlockMetaData[ 5]
                                , localBlockMetaData[6]
                                , uint32_t(mainShmem[startOfLocalWorkQ + i] + isGoldOffset * isGoldForLocQueue[i])
                                , uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x))
                                , threadIdx.x
                                , threadIdx.y
                                , bitPos
                            );*/

                        }

                    };
              sync(cta);
                }
                    //loading metadaa for next loop 
                    if (i + 1 <= worQueueStep[0]) {
                        if (tile.thread_rank() < 20 && tile.meta_group_rank() == 2) {
                            //if (tile.thread_rank() == 0) {
                            //    printf("loading metdata for %d  in i %d \n"
                            //    , mainShmem[startOfLocalWorkQ + i + 1]
                            //     ,i
                            //    );
                            //}
                             localBlockMetaData[tile.thread_rank()] = 
                                metaDataArr[(mainShmem[startOfLocalWorkQ + i + 1])
                                    * metaData.metaDataSectionLength + tile.thread_rank()];
                        };
                    }

                    //finilizing
                    afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i,
                        metaData, tile, localFpConter, localFnConter
                        , blockFpConter, blockFnConter
                        , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue);


                sync(cta);


            }
        }
    }

    //here we are after all of the blocks planned to be processed by this block are
    sync(cta);

    //updating local counters of last local block (normally it is done at the bagining of the next block)
    //but we need to check weather any block was processed at all
    if (iasAnyProcessed[0]) {
        afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, 1,
            metaData, tile, localFpConter, localFnConter
            , blockFpConter, blockFnConter
            , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue);
    }

    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        if (blockFpConter[0] > 0) {
            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
        }
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        if (blockFnConter[0] > 0) {
            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
        }
    };
    // in first thread block we zero work queue counter
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        if (blockIdx.x == 0) {
            minMaxes[9] = 0;
        }
    };


}
