#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;




template <typename TKKI>
inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArr, MetaDataGPU metaData
    , unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* resultListPointerMeta, uint16_t* resultListPointerLocal, uint16_t* resultListPointerIterNumb,
    thread_block cta, thread_block_tile<32> tile, grid_group grid, uint32_t mainShmem[lengthOfMainShmem]
    , bool isAnythingInPadding[6]  , bool isBlockFull[1], uint32_t iterationNumb[1], unsigned int globalWorkQueueOffset[1],
    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
    unsigned int localFnConter[1], unsigned int blockFpConter[1],
    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
    , uint32_t localBlockMetaData[19], unsigned int fpFnLocCounter[1]
    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
    , uint32_t* origArrs, uint16_t* metaDataArr
) {
    auto pipeline = cuda::make_pipeline();
    auto bigShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength));
    auto thirdRegShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (32));


    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0  && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] =true;
    };
    


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }

    sync(cta);
    /// load work QueueData into shared memory 
    for (uint16_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        
        cooperative_groups::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop]), cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]));
        sync(cta);
        //now all of the threads in the block needs to have the same i value so we will increment by 1
        // we are preloading to the pipeline block metaData
        ////##### pipeline Step 0
        pipeline.producer_acquire();

        cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&metaDataArr[(mainShmem[startOfLocalWorkQ] - UINT16_MAX * (mainShmem[startOfLocalWorkQ] >= UINT16_MAX)) * metaData.metaDataSectionLength])
            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 18), pipeline);

        //cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&mainArr[(mainShmem[startOfLocalWorkQ] - UINT16_MAX * (mainShmem[startOfLocalWorkQ] >= UINT16_MAX)) * metaData.mainArrSectionLength + metaData.metaDataOffset])
        //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 18), pipeline);

        pipeline.producer_commit();
        
        for (uint16_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                 ///#### pipeline step 1) now we load data for next step (to sourceshmem) and process data loaded in previous step
                    pipeline.producer_acquire();

                    cuda::memcpy_async(cta, (&mainShmem[0]), (&mainArr[getIndexForSourceShmem(metaData, mainShmem, iterationNumb,i )]) , bigShape, pipeline);
                    //cuda::memcpy_async(cta, (&mainShmem[32]), (&mainArr[32]) , bigShape, pipeline);
                    pipeline.producer_commit();

        //        ////compute first we load data about calculated linear index meta and information is it gold iteration ...
        //            pipeline.consumer_wait();
        //                if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
        //                    isGold[0] = uint32_t(mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX);
        //                    if (isGold[0]) {
        //                        //removing info about wheather it is gold or not pass so we will be able to use it as linear metadata index
        //                        currLinIndM[0] = mainShmem[startOfLocalWorkQ + i] - UINT16_MAX;
        //                    }
        //                };
        //            pipeline.consumer_release();


        //        ////////#### pipeline step 2) 
        //        //load for next step - so we load posterior of anterior block and left of block to the right given they exist
        //            //anterior
        //            pipeline.producer_acquire();
        //                if (localBlockMetaData[17]<UINT32_MAX) {
        //                    //cooperative_groups::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
        //                    //    (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData,17 )])
        //                    //    , bigShape, pipeline);

        //                    cuda::memcpy_async(cta, (&mainShmem[0]), (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 17)]), bigShape, pipeline);


        //                }
        //                //left
        //                if (localBlockMetaData[16] < UINT32_MAX) {
        //                    cuda::memcpy_async(cta, (&mainShmem[0]), (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 16)]), thirdRegShape, pipeline);

        //                    //cooperative_groups::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
        //                    //    (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData, 16)])
        //                    //    , thirdRegShape, pipeline);
        //                }
        //            pipeline.producer_commit();
        //        //compute - now we have data in source shmem about this block only so what can be done is to dilatate the source shmem data up and down and save data in res shmem - additionally saving data about is anything in to or bottom bits
        //            pipeline.consumer_wait();
        //                // first we perform up and down dilatations

        //                mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = bitDilatate(mainShmem[threadIdx.x + threadIdx.y * 32]);
        //                //we also need to set shmem paddings on the basis of first and last bits ...
        //                //top            0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
        //                if (isBitAt(mainShmem[threadIdx.x + threadIdx.y * 32], 0)) {
        //                    // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
        //                    isAnythingInPadding[0] = true;
        //                };
        //                //bottom
        //                if (isBitAt(mainShmem[threadIdx.x + threadIdx.y * 32], (fbArgs.dbZLength - 1))) {
        //                    isAnythingInPadding[1] = true;
        //                };
        //            pipeline.consumer_release();



        //        ////// pipeline step 3) 




        //        ///########## last step loading for next iteration if it is present
        //        if (i + 1<= worQueueStep[0]) {
        //            pipeline.producer_acquire();
        //            cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&mainArr[(mainShmem[startOfLocalWorkQ+1+i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ+i+1] >= UINT16_MAX)) * metaData.mainArrSectionLength + metaData.metaDataOffset])
        //                , cuda::aligned_size_t<4>(sizeof(uint32_t) * 18), pipeline);
        //            pipeline.producer_commit();
        //        }






                sync(cta);
                // now we have metadata linear coordinate and information is it gold or segm pass ...

                //loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);

                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
                //validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
                //    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep,  oldRef, blockFpConter, blockFnConter);

                //////on the basis of isAnythingInPadding we will mark  the neighbouring block as to be activated if there is and if such neighbouring block exists
                //auto activeC = coalesced_threads();

                //if (localWorkQueue[i][3] == 1) {//gold
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedGold, isAnythingInPadding, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    setNextBlocksActivity(tensorslice, localWorkQueue, i, fbArgs.metaData.isToBeActivatedSegm, isAnythingInPadding, activeC);
                //};
                //// marking blocks as full 

                //if (localWorkQueue[i][3] == 1) {//gold
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullGold, activeC);
                //};
                //if (localWorkQueue[i][3] == 0) {//segm
                //    markIsBlockFull(tensorslice, localWorkQueue, i, isBlockFull, fbArgs.metaData.isFullSegm, activeC);
                //};
                //sync(cta);// all results that should be saved to result list are saved                        

                ////we need to clear isAnythingInPadding to 0
                //clearisAnythingInPadding(isAnythingInPadding);
            }
        }
    }
    sync(cta);
    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
         atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
    };
    // in first thread block we zero work queue counter
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        if (blockIdx.x==0) {
            minMaxes[9] = 0;
        }
    };





}
//
//
//template <typename TKKI>
//inline __global__ void paddingDilatation(ForBoolKernelArgs<TKKI> fbArgs) {
//
//
//
//    thread_block cta = this_thread_block();
//    thread_block_tile<32> tile = tiled_partition<32>(cta);
//
//    char* tensorslice;
//    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
//    unsigned int old = 0;
//
//    // some references using as aliases
//    unsigned int& oldRef = old;
//
//
//
//    // main shared memory spaces 
//    __shared__ uint32_t sourceShared[32][32];
//    __shared__ uint32_t resShared[32][32];
//    // holding data about paddings 
//
//
//    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
//    __shared__ bool isAnythingInPadding[6];
//    //variables needed for all threads
//    __shared__ unsigned int iterationNumb[1];
//    __shared__ unsigned int globalWorkQueueOffset[1];
//    __shared__ unsigned int globalWorkQueueCounter[1];
//    __shared__ unsigned int localWorkQueueCounter[1];
//    __shared__ bool isBlockToBeValidated[1];
//    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters
//
//    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
//    //counters for per block number of results added in this iteration
//    __shared__ unsigned int localFpConter[1];
//    __shared__ unsigned int localFnConter[1];
//
//    __shared__ unsigned int blockFpConter[1];
//    __shared__ unsigned int blockFnConter[1];
//
//    //result list offset - needed to know where to write a result in a result list
//    __shared__ unsigned int resultfpOffset[1];
//    __shared__ unsigned int resultfnOffset[1];
//
//    __shared__ unsigned int worQueueStep[1];
//
//    // we will load here multiple entries from workqueue
//    __shared__ uint16_t localWorkQueue[localWorkQueLength][4];
//    //initializations and loading    
//    auto active = coalesced_threads();
//    if (isToBeExecutedOnActive(active, 0)) { iterationNumb[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[13]; };
//    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
//    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number
//
//    if (isToBeExecutedOnActive(active, 3)) {
//        localWorkQueueCounter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 4)) {
//        blockFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 5)) {
//        blockFnConter[0] = 0;
//    };
//
//    if (isToBeExecutedOnActive(active, 6)) {
//        localFpConter[0] = 0;
//    };
//    if (isToBeExecutedOnActive(active, 7)) {
//        localFnConter[0] = 0;
//    };
//
//
//
//    if (isToBeExecutedOnActive(active, 1)) {
//        localTotalLenthOfWorkQueue[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9];
//        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
//        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
//    };
//    sync(cta);
//    // TODO - use pipelines as described at 201 in https://docs.nvidia.com/cuda/pdf/CUDA_C_Programming_Guide.pdf
//    /// load work QueueData into shared memory 
//
//    //TODO change looping so it will access contigous memory
//    for (uint8_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
//        // grid stride loop - sadly most of threads will be idle 
//        ///////////// loading to work queue
//        loadFromGlobalToLocalWorkQueue(fbArgs, tensorslice, localWorkQueue, bigloop, globalWorkQueueOffset, localTotalLenthOfWorkQueue, worQueueStep);
//
//        sync(cta);// now local work queue is populated 
//
//            //now all of the threads in the block needs to have the same i value so we will increment by 1
//        for (uint8_t i = 0; i < worQueueStep[0]; i += 1) {
//            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
//
//
//
//                //TODO() remove
//       /*         auto activee = coalesced_threads();
//                if (isToBeExecutedOnActive(activee, 3)) {
//                    printf("\n in padding looping  xMeta %d yMeta %d zMeta %d isGold %d \n"
//                        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]);
//                };*/
//
//
//
//                // now we have metadata coordinates we need to start go over associated data block - in order to make it as efficient as possible data block size is set to be the same as datablock size
//                // so we do not need iteration loop 
//
//                loadAndDilatateAndSave(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep);
//
//                ///////////////////////// validation if it is to be validated, also we checked for bing full before dilatations - if it was full at the begining - no point in validation
//                validateAndUpMetaCounter(fbArgs, tensorslice, localWorkQueue, bigloop, sourceShared, resShared, isAnythingInPadding, iterationNumb, isBlockFull, cta, i,
//                    isBlockToBeValidated, localTotalLenthOfWorkQueue, localFpConter, localFnConter, resultfpOffset, resultfnOffset, worQueueStep, oldRef, blockFpConter, blockFnConter);
//
//                sync(cta);
//            }
//        }
//    }
//    sync(cta);
//    //     updating global counters
//    updateGlobalCountersAndClear(fbArgs, tensorslice, blockFpConter, blockFnConter, localWorkQueueCounter, localFpConter, localFnConter);
//
//
//    //KROWA!!!
//    //remember to zero out the global work queue counter
//    //and inccrement iterationNumb[1]
//}