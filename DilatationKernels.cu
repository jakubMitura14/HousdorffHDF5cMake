
#include <hip/hip_runtime.h>
//#include "CPUAllocations.cu"
//#include "MetaData.cu"
// 
//#include "ExceptionManagUtils.cu"
//#include "CooperativeGroupsUtils.cu"
//#include "ForBoolKernel.cu"
//#include "FirstMetaPass.cu"
//#include "MainPassFunctions.cu"
//#include <cooperative_groups.h>
//#include <cooperative_groups/reduce.h>
//#include "UnitTestUtils.cu"
//#include "MetaDataOtherPasses.cu"
//#include <cooperative_groups/memcpy_async.h>
//#include <cuda/pipeline>
//using namespace cooperative_groups;
//
//
//
//
////template <typename TKKI, typename forPipeline >
//template <typename TKKI >
//inline __device__ void mainDilatation(const bool isPaddingPass, ForBoolKernelArgs<TKKI>& fbArgs, uint32_t*& mainArrAPointer,
//    uint32_t*& mainArrBPointer, MetaDataGPU& metaData
//    , unsigned int*& minMaxes, uint32_t*& workQueue
//    , uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb,
//    thread_block& cta, thread_block_tile<32>& tile, grid_group& grid, uint32_t (&mainShmem)[lengthOfMainShmem]
//    , bool(&isAnythingInPadding)[6], bool (&isBlockFull)[1], int(&iterationNumb)[1], unsigned int(&globalWorkQueueOffset)[1]
//    ,unsigned int(&globalWorkQueueCounter)[1]
//    , unsigned int(&localWorkQueueCounter)[1],unsigned int(&localTotalLenthOfWorkQueue)[1]
//    , unsigned int(&localFpConter)[1]
//    ,unsigned int(&localFnConter)[1], unsigned int(&blockFpConter)[1]
//   , unsigned int(&blockFnConter)[1], unsigned int(&resultfpOffset)[1]
//    ,unsigned int(&resultfnOffset)[1], unsigned int(&worQueueStep)[1]
//,unsigned int(&localMinMaxes)[5]
//    , uint32_t(&localBlockMetaData)[40]
//    , unsigned int(&fpFnLocCounter)[1]
//    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
//    , uint32_t*& origArrs, uint32_t*& metaDataArr, bool (&isGoldForLocQueue)[localWorkQueLength]
//     , uint32_t(&lastI)[1]
//    , cuda::pipeline<cuda::thread_scope_block>& pipeline
//) {
//
//
//    //initial cleaning  and initializations include loading min maxes
//    dilBlockInitialClean(tile, isPaddingPass, iterationNumb, localWorkQueueCounter, blockFpConter,
//        blockFnConter, localFpConter, localFnConter, isBlockFull
//        , fpFnLocCounter,
//        localTotalLenthOfWorkQueue, globalWorkQueueOffset
//        , worQueueStep, minMaxes, localMinMaxes, lastI);
//    sync(cta);
//
//    /// load work QueueData into shared memory 
//    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
//        // grid stride loop - sadly most of threads will be idle 
//        /////////// loading to work queue
//        loadWorkQueue(mainShmem, workQueue, isGoldForLocQueue, bigloop, worQueueStep);
//
//        //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
//        ////##### pipeline Step 0
//
//
//
//        sync(cta);
//        //loading metadata
//        pipeline.producer_acquire();
//
//        //cuda::memcpy_async(cta, (&localBlockMetaData[20]),
//        //    (&metaDataArr[(mainShmem[startOfLocalWorkQ ])
//        //        * metaData.metaDataSectionLength])
//        //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
//
//        cuda::memcpy_async(cta, (&localBlockMetaData[0]),
//            (&metaDataArr[(mainShmem[startOfLocalWorkQ])
//                * metaData.metaDataSectionLength])
//            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
//
//        //loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 0, 0);
//
//        pipeline.producer_commit();
//
//
//
//        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
//            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
//            
//                //if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {                      
//                //    printf("\n linMeta beg %d is gold %d is padding pass %d\n ", mainShmem[startOfLocalWorkQ + i], isGoldForLocQueue[i], isPaddingPass);
//                //};
//
//                // if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0 && isGoldForLocQueue[i]==0 ) {
//                //    printf("\n linMeta beg %d is gold %d is padding pass %d\n ", mainShmem[startOfLocalWorkQ + i], isGoldForLocQueue[i], isPaddingPass);
//                //};
//
////////////////// step 0  load main data and final processing of previous block
//               //loading main data for first dilatation
//                //IMPORTANT we need to keep a lot of variables constant here like is Anuthing in padding of fp count .. as the represent processing of previous block  - so do not modify them here ...
//                loadMain( fbArgs  , cta , localBlockMetaData , mainShmem, pipeline, metaDataArr, metaData , i , tile, isGoldForLocQueue, iterationNumb
//                );
//                                
//                pipeline.consumer_wait();
//                afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i-1,
//                        metaData, tile, localFpConter, localFnConter
//                        , blockFpConter, blockFnConter
//                        , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);
//                //needed for after block metadata update
//                if (tile.thread_rank() == 0 && tile.meta_group_rank() == 3) {
//                    lastI[0] = i;
//                }
//
//                pipeline.consumer_release();
//
/////////// step 1 load top and process main data 
//               //load top 
//                loadTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);
//                //process main
//                processMain(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isBlockFull);
/////////// step 2 load bottom and process top 
//                loadBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
//                //process top
//                processTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);                     
///////////// step 3 load right  process bottom  
//                loadRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
//                //process bototm
//                processBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////////// step 4 load left process right  
//               
//                loadLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
//                processRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 5 load anterior process left 
//                loadAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
//                processLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 6 load posterior process anterior 
//                loadPosterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
//                processAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
///////// step 7 
//// 
//                
//            //    sync(cta);
//
//                //load reference if needed or data for next iteration if there is such 
//                //process posterior, save data from res shmem to global memory also we mark weather block is full
//                lastLoad(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, origArrs, worQueueStep);
//                processPosteriorAndSaveResShmem(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, isBlockFull);
//                sync(cta);
//
// //////// step 8 basically in order to complete here anyting the count need to be bigger than counter
//               // loading for next block if block is not to be validated it was already done earlier
//                pipeline.producer_acquire();
//                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
//                  > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
//                    if (i + 1 <= worQueueStep[0]) {
//                        loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 1, i);
//                    }
//                }
//                pipeline.producer_commit();
//                
//
//                //validation - so looking for newly covered voxel for opposite array so new fps or new fns
//                pipeline.consumer_wait();
//
//                validate(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, isBlockFull, localFpConter, localFnConter, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb);
//                /////////
//                pipeline.consumer_release();
//
//              //  sync(cta);
//
//                //pipeline.producer_acquire();
//
//                //pipeline.producer_commit();
//
//                //pipeline.consumer_wait();
//
//                //getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
//                //    + threadIdx.x + threadIdx.y * 32]
//                //    = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
//
//                //pipeline.consumer_release();
//
//           }
//       }
//
//        //here we are after all of the blocks planned to be processed by this block are
//
////updating local counters of last local block (normally it is done at the bagining of the next block)
////but we need to check weather any block was processed at all
//        pipeline.consumer_wait();
//
//        if (lastI[0] != UINT32_MAX) {
//            afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, lastI[0],
//                metaData, tile, localFpConter, localFnConter
//                , blockFpConter, blockFnConter
//                , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);
//        
//         if (tile.thread_rank() == 0 && tile.meta_group_rank() == 3) {// this is how it is encoded wheather it is gold or segm block
//                 lastI[0] = UINT32_MAX;
//            }
//        }
//        pipeline.consumer_release();
//
//    }
//
//
//
//    sync(cta);
//
//    //     updating global counters
//    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
//        if (blockFpConter[0] > 0) {
//            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
//        }
//    };
//    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
//        if (blockFnConter[0] > 0) {
//            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
//        }
//    };
//    // in first thread block we zero work queue counter
//    if (threadIdx.x == 2 && threadIdx.y == 0) {
//        if (blockIdx.x == 0) {
//       
//            minMaxes[9] = 0;
//        }
//    };
//
//
//}
