
#include <hip/hip_runtime.h>
////#include "CPUAllocations.cu"
////#include "MetaData.cu"
//// 
////#include "ExceptionManagUtils.cu"
////#include "CooperativeGroupsUtils.cu"
////#include "ForBoolKernel.cu"
////#include "FirstMetaPass.cu"
////#include "MainPassFunctions.cu"
////#include <cooperative_groups.h>
////#include <cooperative_groups/reduce.h>
////#include "UnitTestUtils.cu"
////#include "MetaDataOtherPasses.cu"
////#include <cooperative_groups/memcpy_async.h>
////#include <cuda/pipeline>
////using namespace cooperative_groups;
////
////
////
////
////template <typename TKKI>
////inline __device__ void mainDilatation(bool isPaddingPass, ForBoolKernelArgs<TKKI> fbArgs, uint32_t* mainArrAPointer,
////    uint32_t* mainArrBPointer, MetaDataGPU metaData
////    , unsigned int* minMaxes, uint32_t* workQueue
////    , uint32_t* resultListPointerMeta, uint32_t* resultListPointerLocal, uint32_t* resultListPointerIterNumb,
////    thread_block cta, thread_block_tile<32> tile, grid_group grid, uint32_t mainShmem[lengthOfMainShmem]
////    , bool isAnythingInPadding[6], bool isBlockFull[1], int iterationNumb[1], unsigned int globalWorkQueueOffset[1],
////    unsigned int globalWorkQueueCounter[1], unsigned int localWorkQueueCounter[1],
////    unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1],
////    unsigned int localFnConter[1], unsigned int blockFpConter[1],
////    unsigned int blockFnConter[1], unsigned int resultfpOffset[1],
////    unsigned int resultfnOffset[1], unsigned int worQueueStep[1],
////    uint32_t isGold[1], uint32_t currLinIndM[1], unsigned int localMinMaxes[5]
////    , uint32_t localBlockMetaData[(i & 1) * 20+20], unsigned int fpFnLocCounter[1]
////    , bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]
////    , uint32_t* origArrs, uint32_t* metaDataArr, uint32_t oldIsGold[1], uint32_t oldLinIndM[1], uint32_t localBlockMetaDataOld[20],
////    bool isGoldForLocQueue[localWorkQueLength], bool isBlockToBeValidated[1]
////) {
////
////
////
////
////    auto pipeline = cuda::make_pipeline();
////    auto bigShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength));
////    auto thirdRegShape = cuda::aligned_size_t<128>(sizeof(uint32_t) * (32));
////
////
////    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
////        iterationNumb[0] += 1;
////    };
////
////    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
////        localWorkQueueCounter[0] = 0;
////    };
////
////    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
////        blockFpConter[0] = 0;
////    };
////    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
////        blockFnConter[0] = 0;
////    };
////    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
////        localFpConter[0] = 0;
////    };
////    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
////        localFnConter[0] = 0;
////    };
////    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
////        isBlockFull[0] = true;
////    };
////    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
////        fpFnLocCounter[0] = 0;
////    };
////
////    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
////        // if it will be still of such value it mean that no block was processed
////        oldLinIndM[0] = isGoldOffset;
////    };
////
////    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
////        localTotalLenthOfWorkQueue[0] = minMaxes[9];
////        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
////        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
////    };
////    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
////0 : global FP count;
////1 : global FN count;
////2 : workQueueCounter
////3 : resultFP globalCounter
////4 : resultFn globalCounter
////*/
////    if (tile.meta_group_rank() == 1) {
////        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
////    }
////
////
////
////
////    sync(cta);
////    /// load work QueueData into shared memory 
////    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
////        // grid stride loop - sadly most of threads will be idle 
////        /////////// loading to work queue
////
////        //cuda::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop])
////        //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]), pipeline);
////
////
////        for (uint16_t ii = 0; ii < worQueueStep[0]; ii++) {
////            mainShmem[startOfLocalWorkQ + ii] = workQueue[bigloop + ii];
////            isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
////            mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];
////
////        }
////
////
////        //to do change into barrier
////
////        //now all of the threads in the block needs to have the same i value so we will increment by 1
////        // we are preloading to the pipeline block metaData
////////##### pipeline Step 0
////
////        pipeline.producer_acquire();
////
////        cuda::memcpy_async(cta, (&localBlockMetaData[(i & 1) * 20+0]),
////            (&metaDataArr[mainShmem[startOfLocalWorkQ] * metaData.metaDataSectionLength])
////            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
////
////        pipeline.producer_commit();
////        sync(cta);
////
////        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
////            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
////                ///#### pipeline step 1) now we load data for next step (to mainly sourceshmem and left-right if apply) and process data loaded in previous step
////
////
////                pipeline.producer_acquire();
////
////                cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
////                    mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
////                    bigShape, pipeline);
////
////
////                if (mainShmem[startOfLocalWorkQ + i] < (metaData.totalMetaLength - 1)) {
////                    cooperative_groups::memcpy_async(tile, (&mainShmem[begSMallRegShmemB]),
////                        &getSourceReduced(fbArgs, iterationNumb)[
////                            (mainShmem[startOfLocalWorkQ + i] + 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
////                                + tile.meta_group_rank() * 32], //we look for indicies 0,32,64... up to metaData.mainArrXLength
////                        cuda::aligned_size_t<4>(sizeof(uint32_t))
////                                );
////
////                }
////
////                //load data of interst form block to the left
////                if (mainShmem[startOfLocalWorkQ + i] > 0) {
////                    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemA + tile.meta_group_rank()]),
////                        &getSourceReduced(fbArgs, iterationNumb)[
////                            (mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
////                                //we look for indicies 31,63... up to metaData.mainArrXLength
////                                + (tile.meta_group_rank() * 32) + 31]
////                        , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
////
////                }
////                pipeline.producer_commit();
////
////
////
////                // we need to do the cleaning after previous block .. compute first we load data about calculated linear index meta and information is it gold iteration ...
////                pipeline.consumer_wait();
////
////
////                afterBlockClean(cta, worQueueStep, localBlockMetaDataOld, mainShmem, i,
////                    metaData, tile, localFpConter, localFnConter
////                    , blockFpConter, blockFnConter
////                    , metaDataArr, oldLinIndM, oldIsGold
////                    , isAnythingInPadding, isBlockFull, isPaddingPass);
////
////
////
////
////
////                pipeline.consumer_release();
////
////                ////////#### pipeline step 2)  load block from top and process center that is in source shmem; and both smallRegShmems
////                               //load for next step - so we load block to the top
////                pipeline.producer_acquire();
////
////                if (localBlockMetaData[(i & 1) * 20+13] < isGoldOffset) {
////
////                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
////                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20+13] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
////                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
////                        , pipeline);
////
////                }
////                pipeline.producer_commit();
////
////                //compute - now we have data in source shmem about this block and left and right padding and we need to process it 
////                pipeline.consumer_wait();
////                // first we perform up and down dilatations inside the block
////            //if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0) {
////            //    printf("source shmem linLocalInd %d  linMeta %d \n",(threadIdx.x + threadIdx.y * 32), mainShmem[startOfLocalWorkQ + ii] );
////            //}
////
////                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
////
////
////                //we also do the left and right dilatations
////                ////left
////                dilatateHelperForTransverse((threadIdx.x == 0),
////                    2, (-1), (0), mainShmem, isAnythingInPadding
////                    , 0, threadIdx.y
////                    , 15, begSMallRegShmemA, localBlockMetaData);
////
////                //right
////                dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
////                    3, (1), (0), mainShmem, isAnythingInPadding
////                    , 0, threadIdx.y
////                    , 16, begSMallRegShmemB, localBlockMetaData);
////                ///////////saving old
////                //additionally we save previous copies of data so refreshing will keep easier
////                if (threadIdx.x < 20 && threadIdx.y == 0) {
////                    localBlockMetaDataOld[tile.thread_rank()] = localBlockMetaData[(i & 1) * 20+tile.thread_rank()];
////                }
////                if (threadIdx.x == 0 && threadIdx.y == 1) {
////                    oldIsGold[0] = isGoldForLocQueue[i];
////                }
////                if (threadIdx.x == 1 && threadIdx.y == 1) {
////                    oldLinIndM[0] = mainShmem[startOfLocalWorkQ + i];
////
////                }
////                if (threadIdx.x == 2 && threadIdx.y == 1) {
////                    isBlockToBeValidated[0] = ((localBlockMetaData[(i & 1) * 20+2 - isGoldForLocQueue[i]]) > localBlockMetaData[(i & 1) * 20+(4 - isGoldForLocQueue[i])]);
////
////                }
////
////
////
////                pipeline.consumer_release();
////                ////////#### pipeline step 3) we load bottom, anterior and posterior and we process top
////                                      //load anterior and posterior and bottom
////                pipeline.producer_acquire();
////                //block to anterior 
////                if (localBlockMetaData[(i & 1) * 20+17] < isGoldOffset && tile.meta_group_rank() == 0) {
////
////                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemA], &getSourceReduced(fbArgs, iterationNumb)[
////                        (localBlockMetaData[(i & 1) * 20+17]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
////                        thirdRegShape, pipeline);
////
////                }
////                // block to posterior
////                if (localBlockMetaData[(i & 1) * 20+18] < isGoldOffset && tile.meta_group_rank() == 1) {
////                    cuda::memcpy_async(tile, &mainShmem[begSMallRegShmemB], &getSourceReduced(fbArgs, iterationNumb)[
////                        (localBlockMetaData[(i & 1) * 20+18]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
////                            + (blockDim.y - 1) * 32// we need last 32 length entry of the posterior block 
////                    ], thirdRegShape, pipeline);
////
////                }
////
////                //bottom  block
////                if (localBlockMetaData[(i & 1) * 20+14] < isGoldOffset) {
////                    cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
////                        &getSourceReduced(fbArgs, iterationNumb)[
////                            localBlockMetaData[(i & 1) * 20+14] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
////                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
////                                , pipeline);
////
////                }
////                pipeline.producer_commit();
////
////
////                //    compute - now we have data in source shmem about block to the top
////                pipeline.consumer_wait();
////                dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
////                    , 31// represent a uint32 number that has a bit of intrest in this block set and all others 0 here first bit is set
////                    , 0
////                    , begfirstRegShmem);
////                pipeline.consumer_release();
////                ////////#### pipeline step 5) if block is to be validated we load reference data and we process bottom, left and right
////                                //load reference data if block is to be validated otherwise if it is not the last step in the loop we load data for next loop
////                pipeline.producer_acquire();
////                if (isBlockToBeValidated[0]) {// so count is bigger than counter so we should validate
////            //now we load data from reference arrays 
////                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
////                        &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
////                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
////                        , pipeline);
////
////                }
////                else {//if we are not validating we immidiately start loading data for next loop
////                    if (i + 1 <= worQueueStep[0]) {
////                        cuda::memcpy_async(cta, (&localBlockMetaData[(i & 1) * 20+0]),
////                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1])
////                                * metaData.metaDataSectionLength])
////                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
////                    }
////                }
////                //    compute - now we have data in source shmem about block to the bottom, left and right
////
////                pipeline.producer_commit();
////                pipeline.consumer_wait();
////                //bottom
////                dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
////                    , 0// represent a uint32 number that has a bit of intrest in this block set and all others 0 here last bit is set
////                    , 31
////                    , begSecRegShmem);
////                //posterior
////                dilatateHelperForTransverse((threadIdx.y == 0), 5
////                    , (0), (-1), mainShmem, isAnythingInPadding
////                    , 0, threadIdx.x // we add offset depending on y dimension
////                    , 18, begSMallRegShmemB, localBlockMetaData);
////                //anterior
////                dilatateHelperForTransverse((threadIdx.y == (fbArgs.dbYLength - 1)), 4
////                    , (0), (1), mainShmem, isAnythingInPadding
////                    , 0, threadIdx.x
////                    , 17, begSMallRegShmemA, localBlockMetaData);
////
////                // now all of the data is processed we need to save it into global memory
////                // TODO try to use mempcy async here
////                 //if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]>0) {
////                getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
////                    + threadIdx.x + threadIdx.y * 32]
////                    = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
////                //}
////                // setting information about is block full
////
////
////                if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] != UINT32_MAX) {
////                    isBlockFull[0] = false;
////                }
////
////
////                pipeline.consumer_release();
////
////
////                //////////#### pipeline step 6) if block is to be validated we process the res and reference data and start loading data for begining of the next loop
////
////
////
////                                                                                    ////load data for next iteration
////                pipeline.producer_acquire();
////                if (localBlockMetaDataOld[((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
////                > localBlockMetaDataOld[((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
////                    if (i + 1 <= worQueueStep[0]) {
////                        cuda::memcpy_async(cta, (&localBlockMetaData[(i & 1) * 20+0]),
////                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1])
////                                * metaData.metaDataSectionLength])
////                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
////                    }
////                }
////                pipeline.producer_commit();
////
////                //process check is there any new result (we have reference in begfirstRegShmem)
////                         //now first we need to check for bits that are true now after dilatation but were not in source we will save it in res shmem becouse we will no longer need it
////                pipeline.consumer_wait();
////                if (isBlockToBeValidated[0]) {// so count is bigger than counter so we should validate
////
////                    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);
////
////
////
////                    //we now look for bits prasent in both reference arrays and current one
////                    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);
////
////
////                    // now we look through bits and when some is set we call it a result 
////#pragma unroll
////                    for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
////                        //if any bit here is set it means it should be added to result list 
////                        if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
////                            //first we add to the resList
////                            //TODO consider first passing it into shared memory and then async mempcy ...
////                            //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
////                            unsigned int old = 0;
////                            ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
////                            if (isGoldForLocQueue[i]) {
////                                old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaDataOld[5];
////                            }
////                            else {
////                                old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaDataOld[6];
////                            };
////                            //   add results to global memory    
////                            resultListPointerMeta[old] = uint32_t(oldLinIndM[0] + isGoldOffset * oldIsGold[0]);
////                            resultListPointerLocal[old] = uint16_t(fbArgs.dbYLength * 32 * bitPos + threadIdx.y * 32 + threadIdx.x);
////                            resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);
////
////                            printf("rrrrresult meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d\n"
////                                , mainShmem[startOfLocalWorkQ + i]
////                                , isGoldForLocQueue[i]
////                                , old
////                                , localFpConter[0]
////                                , localFnConter[0]
////                                , localBlockMetaDataOld[6]
////                                , localBlockMetaDataOld[7]
////                                , uint32_t(oldLinIndM[0] + isGoldOffset * oldIsGold[0])
////                                , (fbArgs.dbYLength * 32 * bitPos + threadIdx.y * 32 + threadIdx.x)
////                            );
////
////                        }
////                    }
////                    pipeline.consumer_release();
////
////                };
////            }
////        }
////    }
////
////    //here we are after all of the blocks planned to be processed by this block are
////    sync(cta);
////
////    //updating local counters of last local block (normally it is done at the bagining of the next block)
////    //but we need to check weather any block was processed at all
////    if (oldLinIndM[0] != isGoldOffset) {
////        afterBlockClean(cta, worQueueStep, localBlockMetaDataOld, mainShmem, 2,//2 is completely arbitrary important it is bigger than 0
////            metaData, tile, localFpConter, localFnConter
////            , blockFpConter, blockFnConter
////            , metaDataArr, oldLinIndM, oldIsGold
////            , isAnythingInPadding, isBlockFull, isPaddingPass);
////    }
////
////    //     updating global counters
////    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
////        if (blockFpConter[0] > 0) {
////            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
////        }
////    };
////    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
////        if (blockFnConter[0] > 0) {
////            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
////        }
////    };
////    // in first thread block we zero work queue counter
////    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
////        if (blockIdx.x == 0) {
////            minMaxes[9] = 0;
////        }
////    };
////
////
////}
//
//
//
//
//template <typename TKKI>
//inline __global__ void testKernel(ForBoolKernelArgs<TKKI> fbArgs, unsigned int* minMaxes, uint32_t* mainArr, MetaDataGPU metaData, uint32_t* workQueue, uint32_t* origArr) {
//    thread_block cta = this_thread_block();
//
//    //work queue !!
//    //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//    //    for (uint32_t ii = blockIdx.x; ii < 7; ii += gridDim.x) {
//    //        if (workQueue[ii] > 0) {
//    //            if (workQueue[ii] > (isGoldOffset-1)) {
//    //                printf("in gold workqueue elment %d  \n", (workQueue[ii] - isGoldOffset));
//    //            }
//    //            else {
//    //                printf("in segm workqueue elment %d  \n", (workQueue[ii]));
//
//    //            }
//
//    //        }
//
//    //    }
//    //}
//    // 
//        //results  !!
//    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        for (uint32_t ii = blockIdx.x; ii < 10; ii += gridDim.x) {
//            if (fbArgs.resultListPointerMeta[ii] > 0) {
//                printf("in TEST kernel  result lin meta %d ii  \n", fbArgs.resultListPointerMeta[ii]);
//
//            }
//
//        }
//    }
//
//
//
//    sync(cta);
//    char* tensorslice;
//
//
//    for (uint32_t linIdexMeta = blockIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += gridDim.x) {
//        //we get from linear index  the coordinates of the metadata block of intrest
//        uint8_t xMeta = linIdexMeta % metaData.metaXLength;
//        uint8_t zMeta = floor((float)(linIdexMeta / (metaData.metaXLength * metaData.MetaYLength)));
//        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength));
//
//        for (uint8_t xLoc = threadIdx.x; xLoc < 32; xLoc += blockDim.x) {
//            uint32_t x = (xMeta + metaData.minX) * 32 + xLoc;//absolute position
//            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
//                uint32_t  y = (yMeta + metaData.minY) * fbArgs.dbYLength + yLoc;//absolute position
//                for (uint8_t zLoc = 0; zLoc < 32; zLoc++) {
//
//                    uint32_t z = (zMeta + metaData.minZ) * fbArgs.dbZLength + zLoc;//absolute position
//                    uint8_t ww = 0;
//                    //uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength)*ww];//
//                    uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * 32) + (metaData.mainArrXLength) * ww];//
//                    //uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength)];
//
//
//
//
//                    //rrrrresult meta 1 isGold 1 old 0 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655351  localInd 24544
//
//                    //if (linIdexMeta== 1 ) {
//                    //    if (  (fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 24544) {
//                    //            printf("in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
//                    //  ,  x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);
//
//                    //    }
//                    //
//                    //}
//                    ////    rrrrresult meta 2 isGold 1 old 1 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655352  localInd 23839
//
//                    //if (linIdexMeta == 2) {
//                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 23839) {
//                    //        printf("in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
//                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);
//
//                    //    }
//
//                    //}
//                    ////    rrrrresult meta 4 isGold 1 old 2 localFpConter 1 localFnConter 0 fpOffset 0 fnOffset 0 linIndUpdated 655354  localInd 767
//
//                    //if (linIdexMeta == 4) {
//                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 767) {
//                    //        printf("in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
//                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);
//
//                    //    }
//
//                    //}
//                    ////    rrrrresult meta 0 isGold 0 old 3 localFpConter 0 localFnConter 1 fpOffset 3 fnOffset 1 linIndUpdated 0  localInd 24575
//
//                    //if (linIdexMeta == 0) {
//                    //    if ((fbArgs.dbYLength * 32 * zLoc + yLoc * 32 + xLoc) == 24575) {
//                    //        printf("in TEST kernel x %d y%d z %d linearLocal %d linIdexMeta  \n"
//                    //            , x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta);
//
//                    //    }
//
//                    //}
//
//
//
//                 //if (x==33 && y==1 && z==71) {
//                 //    printf("in 33 1 71 TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d column %d looking in %d \n"
//                 //        , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
//                 //        , column, linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww);
//                 //}
//
//
//
//
//                    if (isBitAt(column, zLoc) && column > 0) {
//
//
//                        printf("in TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d looking in %d    \n"
//                            , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * 32), linIdexMeta
//                            , column, linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww, fbArgs.dbYLength);
//                    }
//
//                    ww = 1;
//                    // uint32_t column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww];//
//                    column = mainArr[linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww];//
//
//
//                    //if (x == 33 && y == 1 && z == 71) {
//                    //    printf("in 33 1 71 TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d column %d looking in %d \n"
//                    //        , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
//                    //        , column, linIdexMeta * metaData.mainArrSectionLength + (threadIdx.x + threadIdx.y * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww);
//                    //}
//
//                    if (isBitAt(column, zLoc) && column > 0) {
//
//                        printf("in TEST kernel Metax %d yMeta %d zMeta %d x %d y%d z %d linearLocal %d linIdexMeta %d looking in %d   \n"
//                            , xMeta, yMeta, zMeta, x, y, z, (xLoc + yLoc * fbArgs.dbXLength), linIdexMeta
//                            , column, linIdexMeta * metaData.mainArrSectionLength + (xLoc + yLoc * fbArgs.dbXLength) + (metaData.mainArrXLength) * ww, fbArgs.dbYLength);
//                    }
//
//                }
//            }
//        }
//
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    auto count = fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 1];
//        //    if (count > 0) {
//        //        printf("in TEST kernel looking fp count  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d counter %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count, fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 3]);
//        //    }
//        //}
//        //if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
//        //    auto count = fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 2];
//        //    if (count > 0) {
//        //        printf("in TEST kernel looking fn count   xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d counter %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count, fbArgs.metaDataArrPointer[linIdexMeta * metaData.metaDataSectionLength + 4]);
//        //    }
//        //}
//
//
//
//
//
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength+ metaData.metaDataOffset + 7];
//        //    if (count ==1) {
//        //        printf("in TEST kernel looking active gold  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
//        //    }
//        //}
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 9];
//        //    if (count == 1) {
//        //        printf("in TEST kernel looking active segm  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
//        //    }
//        //}
//        ///// testing  calculation of surrounding blocks linear indicies
//        // block 1,1,1
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    if (xMeta==1 && yMeta==1 && zMeta==1) {
//        //        printf("linear indicies from metadata  top %d bottom %d left %d right %d anterior %d posterior %d  linIdexMeta current %d \n    "
//        //            ,mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 13]
//        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 14]
//
//        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 15]
//        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 16]
//
//        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 17]
//        //            , mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 18]
//        //            , linIdexMeta
//        //        );
//        //    }
//        //    if (xMeta ==  1&& yMeta == 1 && zMeta == 0) {
//        //        printf("linear index top linIdexMeta %d \n    ", linIdexMeta);
//        //    
//        //    }
//        //    if (xMeta ==1 && yMeta == 1 && zMeta == 2) {
//        //        printf("linear index bottom linIdexMeta %d \n    ", linIdexMeta);
//
//        //    }
//        //    if (xMeta == 1&& yMeta == 2 && zMeta == 1) {
//        //        printf("linear index anterior linIdexMeta %d \n    ", linIdexMeta);
//
//        //    }
//        //    if (xMeta == 1&& yMeta == 0 && zMeta == 1) {
//        //        printf("linear index posterior linIdexMeta %d \n    ", linIdexMeta);
//
//        //    }
//
//        //    if (xMeta ==2 && yMeta == 1 && zMeta == 1) {
//        //        printf("linear index right linIdexMeta %d \n    ", linIdexMeta);
//
//        //    }
//        //    if (xMeta == 0&& yMeta == 1 && zMeta == 1) {
//        //        printf("linear index left linIdexMeta %d \n    ", linIdexMeta);
//
//        //    }
//
//        //}
//
////// checking weather on edges it shows UINT32_MAX
//        //   if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//
//        //    if (xMeta ==  1&& yMeta == 1 && zMeta == 0) {
//        //        printf("linear index top linIdexMeta %d  and max is %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 13], UINT32_MAX);
//        //    
//        //    }
//        //    if (xMeta ==1 && yMeta == 1 && zMeta == 3) {
//        //        printf("linear index bottom linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 14]);
//
//        //    }
//        //    if (xMeta == 1&& yMeta == 5 && zMeta == 1) {
//        //        printf("linear index anterior linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 17]);
//
//        //    }
//        //    if (xMeta == 1&& yMeta == 0 && zMeta == 1) {
//        //        printf("linear index posterior linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 18]);
//
//        //    }
//
//        //    if (xMeta ==2 && yMeta == 1 && zMeta == 1) {
//        //        printf("linear index right linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 16]);
//
//        //    }
//        //    if (xMeta == 0&& yMeta == 1 && zMeta == 1) {
//        //        printf("linear index left linIdexMeta %d \n    ", mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 15]);
//
//        //    }
//
//        //}
//
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 5];
//        //    if (count >0) {
//        //        printf("in TEST kernel offset fp  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
//        //    }
//        //}
//        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//        //    auto count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + 6];
//        //    if (count > 0) {
//        //        printf("in TEST kernel offset fn  xMeta %d yMeta %d zMeta %d linIdexMeta %d count %d \n"
//        //            , xMeta, yMeta, zMeta, linIdexMeta, count);
//        //    }
//        //}
//
//    }
//
//
//
//
//    //for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x; linIdexMeta < 80; linIdexMeta += blockDim.x * blockDim.y * gridDim.x) {
//
//
//    // /*   if (fbArgs.metaData.resultList[linIdexMeta * 5 + 4] != 131 && fbArgs.metaData.resultList[linIdexMeta * 5] > 0) {
//
//    //        printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d \n ",
//    //            fbArgs.metaData.resultList[linIdexMeta * 5]
//    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 1]
//    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 2]
//    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 3]
//    //            , fbArgs.metaData.resultList[linIdexMeta * 5 + 4]
//    //            , linIdexMeta
//
//
//    //        );
//    //    }
//    //    else {
//    //        printf(" *** ");
//    //        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[17]), 1);
//
//    //    }*/
//    //}
//}
