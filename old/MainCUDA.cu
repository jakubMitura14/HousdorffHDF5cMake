
#include <hip/hip_runtime.h>
//
//
//#include "cuda_runtime.h"
//#include <cmath>
//
//#include "device_launch_parameters.h"
//#include "simpletest.h"
//
//// includes, system
//#include <iostream>     // std::cout
//#include <algorithm>    // std::min
////#include <helper_cuda.h>
//#include <cmath>
////#include "Structs.cu"
//#include <math.h>
////#include "MemoryTransfers.cu"
//#include <cstdint>
//#include <assert.h>
//#include <numeric>
//#include <cooperative_groups.h>
//#include <cooperative_groups/reduce.h>
////#include "BoolKernelTests.cu"
//#include "testAll.cu"
//using namespace cooperative_groups;
//
//#include <iostream>
//#include <string>
//#include <vector>
//#include <H5Cpp.h>
//using namespace H5;
//
//
//const H5std_string FILE_NAME("SDS.h5");
//const H5std_string DATASET_NAME("IntArray");
//const int    NX_SUB = 3;    // hyperslab dimensions
//const int    NY_SUB = 4;
//const int    NX = 7;        // output buffer dimensions
//const int    NY = 7;
//const int    NZ = 3;
//const int    RANK_OUT = 3;
//
//
//
//int main()
//{
//    /*
//   * Output buffer initialization.
//   */
//    int i, j, k;
//    int         data_out[NX][NY][NZ]; /* output buffer */
//    for (j = 0; j < NX; j++)
//    {
//        for (i = 0; i < NY; i++)
//        {
//            for (k = 0; k < NZ; k++)
//                data_out[j][i][k] = 0;
//        }
//    }
//    /*
//     * Try block to detect exceptions raised by any of the calls inside it
//     */
//    try
//    {
//        /*
//         * Turn off the auto-printing when failure occurs so that we can
//         * handle the errors appropriately
//         */
//        Exception::dontPrint();
//        /*
//         * Open the specified file and the specified dataset in the file.
//         */
//        H5File file(FILE_NAME, H5F_ACC_RDONLY);
//        DataSet dataset = file.openDataSet(DATASET_NAME);
//        /*
//         * Get the class of the datatype that is used by the dataset.
//         */
//        H5T_class_t type_class = dataset.getTypeClass();
//        /*
//         * Get class of datatype and print message if it's an integer.
//         */
//        if (type_class == H5T_INTEGER)
//        {
//            //cout << "Data set has INTEGER type" << endl;
//            /*
//         * Get the integer datatype
//             */
//            IntType intype = dataset.getIntType();
//            /*
//             * Get order of datatype and print message if it's a little endian.
//             */
//            H5std_string order_string;
//            H5T_order_t order = intype.getOrder(order_string);
//          //  cout << order_string << endl;
//            /*
//             * Get size of the data element stored in file and print it.
//             */
//            size_t size = intype.getSize();
//           // cout << "Data size is " << size << endl;
//        }
//        /*
//         * Get dataspace of the dataset.
//         */
//        DataSpace dataspace = dataset.getSpace();
//        /*
//         * Get the number of dimensions in the dataspace.
//         */
//        int rank = dataspace.getSimpleExtentNdims();
//        /*
//         * Get the dimension size of each dimension in the dataspace and
//         * display them.
//         */
//        hsize_t dims_out[2];
//        int ndims = dataspace.getSimpleExtentDims(dims_out, NULL);
//       /* cout << "rank " << rank << ", dimensions " <<
//            (unsigned long)(dims_out[0]) << " x " <<
//            (unsigned long)(dims_out[1]) << endl;*/
//        /*
//         * Define hyperslab in the dataset; implicitly giving strike and
//         * block NULL.
//         */
//        hsize_t      offset[2];   // hyperslab offset in the file
//        hsize_t      count[2];    // size of the hyperslab in the file
//        offset[0] = 1;
//        offset[1] = 2;
//        count[0] = NX_SUB;
//        count[1] = NY_SUB;
//        dataspace.selectHyperslab(H5S_SELECT_SET, count, offset);
//        /*
//         * Define the memory dataspace.
//         */
//        hsize_t     dimsm[3];              /* memory space dimensions */
//        dimsm[0] = NX;
//        dimsm[1] = NY;
//        dimsm[2] = NZ;
//        DataSpace memspace(RANK_OUT, dimsm);
//        /*
//         * Define memory hyperslab.
//         */
//        hsize_t      offset_out[3];   // hyperslab offset in memory
//        hsize_t      count_out[3];    // size of the hyperslab in memory
//        offset_out[0] = 3;
//        offset_out[1] = 0;
//        offset_out[2] = 0;
//        count_out[0] = NX_SUB;
//        count_out[1] = NY_SUB;
//        count_out[2] = 1;
//        memspace.selectHyperslab(H5S_SELECT_SET, count_out, offset_out);
//        /*
//         * Read data from hyperslab in the file into the hyperslab in
//         * memory and display the data.
//         */
//        dataset.read(data_out, PredType::NATIVE_INT, memspace, dataspace);
//        //for (j = 0; j < NX; j++)
//        //{
//        //    for (i = 0; i < NY; i++)
//        //    //    cout << data_out[j][i][0] << " ";
//        //  //  cout << endl;
//        //}
//        /*
//         * 0 0 0 0 0 0 0
//         * 0 0 0 0 0 0 0
//         * 0 0 0 0 0 0 0
//         * 3 4 5 6 0 0 0
//         * 4 5 6 7 0 0 0
//         * 5 6 7 8 0 0 0
//         * 0 0 0 0 0 0 0
//         */
//    }  // end of try block
//    // catch failure caused by the H5File operations
//    catch (FileIException error)
//    {
//      //  error.printError();
//        return -1;
//    }
//    // catch failure caused by the DataSet operations
//    catch (DataSetIException error)
//    {
//      //  error.printError();
//        return -1;
//    }
//    // catch failure caused by the DataSpace operations
//    catch (DataSpaceIException error)
//    {
//      //  error.printError();
//        return -1;
//    }
//    // catch failure caused by the DataSpace operations
//    catch (DataTypeIException error)
//    {
//      //  error.printError();
//        return -1;
//    }
//    return 0;  // successfully terminated
//
//
//
//
//
//
////    ///tst
////    int localTotalLenthOfWorkQueue[1];
////    int globalWorkQueueOffset[1];
////    int worQueueStep[1];
////
////
////    localTotalLenthOfWorkQueue[0] =177;
////    int gridDimX = 2;
////    int blockDimX = 32;
////
////    globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDimX))+1;
////    worQueueStep[0] = std::min(30, globalWorkQueueOffset[0]);
////
////    int debugArr[1000];
////
////    for (int j = 0; j < 1000; j++) {
////        debugArr[j]=0;
////    };
////
////    for (int blockIdxX = 0; blockIdxX < gridDimX; blockIdxX++) {
////
////        for (int bigloop = blockIdxX * globalWorkQueueOffset[0]; bigloop < ((blockIdxX+1) * globalWorkQueueOffset[0])
////            ; bigloop += worQueueStep[0]) {
////
////            for (int threadidX = 0; threadidX < blockDimX; threadidX++){
////
////                for (int i = threadidX ; i < worQueueStep[0]; i += blockDimX) {
////
////                    if (((bigloop + i)< localTotalLenthOfWorkQueue[0]) && ((bigloop + i) <  ((blockIdxX + 1) * globalWorkQueueOffset[0]) )) {
////                        printf("%d    blockIdxX %d  bigloop %d  threadidX %d i %d \n", bigloop + i, blockIdxX, bigloop, threadidX, i);
////
////                        debugArr[bigloop + i] += 1;
////
////                    }
////                };
////        };
////    };
////    };
////
////
////    int oo = 0;
////for (int j = 0; j < 1000; j++) {
////    oo += debugArr[j];
////};
////printf("worQueueStep %d  globalWorkQueueOffset %d  \n", worQueueStep[0], globalWorkQueueOffset[0]);
////
////printf("summ %d  localTotalLenthOfWorkQueue %d  \n", oo , localTotalLenthOfWorkQueue[0]);
////for (int j = 0; j < 178; j++) {
////    printf("%d in %d \n ",j, debugArr[j]);
////
////};
//
//
//
//  testMainPasswes();
//
//
//
//
//
//
//
//
//
//
//
//
//
//
//    //uint32_t numb = 0;
//    //int pos1 = 2;
//    //int pos2 = 8;
//    //int pos3 = 22;
//
//    //std::cout << "pre" << std::endl;
//    //std::cout << (numb & (1 << (pos1))) << std::endl;
//    //std::cout << (numb & (1 << (pos2))) << std::endl;
//    //std::cout << (numb & (1 << (pos3))) << std::endl;
//
//
//    //numb |= 1 << pos1;
//    //numb |= 1 << pos2;
//    //numb |= 1 << pos3;
//
//
//    //std::cout << "post" << std::endl;
//    //std::cout << ((numb & (1 << (pos1))) > 0) << std::endl;
//    //std::cout << ((numb & (1 << (pos2))) > 0) << std::endl;
//    //std::cout << ((numb & (1 << (pos3))) > 0) << std::endl;
//
//    //std::cout << ((numb & (1 << (3))) > 0) << std::endl;
//    //std::cout << ((numb & (1 << (7))) > 0) << std::endl;
//    //std::cout << ((numb & (1 << (30))) > 0) << std::endl;
//
//
//
//
//
//
//
//
//    return 0;
//}
//
