
#include <hip/hip_runtime.h>
//#include "FirstMetaPass.cu"
//#include "Structs.cu"
//
//
//
////testing loopMeta function in order to execute test unhash proper function in loopMeta
//#pragma once
//extern "C" inline void testFirstMetaPass() {
//	// threads and blocks for bool kernel
//	const int blocks = 10;
//	const int xThreadDim = 32;
//	const int yThreadDim = 8;
//	const dim3 threads = dim3(xThreadDim, yThreadDim);
//	// threads and blocks for first metadata pass
//	int threadsFirstMetaDataPass = 32;
//	int blocksFirstMetaDataPass = 10;
//
//	//metadata
//	const int metaXLength = 14;
//	const int MetaYLength = 14;
//	const int MetaZLength = 17;
//
//
//	const int totalLength = metaXLength * MetaYLength * MetaZLength;
//	const int loopMetaTimes = floor(totalLength / blocks);
//
//	/*   int*** h_tensor;
//	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/
//
//
//	int*** forDebugArr;
//
//	const int dXLength = metaXLength;
//	const int dYLength = MetaYLength;
//	const int dZLength = MetaZLength;
//
//	//datablock dimensions
//	const int dbXLength = xThreadDim;
//	const int dbYLength = yThreadDim;
//	const int dbZLength = 32;
//
//	const int mainXLength = dbXLength * metaXLength;
//	const int mainYLength = dbYLength * MetaYLength;
//	const int mainZLength = dbZLength * MetaZLength;
//
//
//	//main data arrays
//	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//
//	int*** segmArr;
//	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//	MetaDataCPU metaData;
//	metaData.metaXLength = metaXLength;
//	metaData.MetaYLength = MetaYLength;
//	metaData.MetaZLength = MetaZLength;
//	metaData.totalMetaLength = totalLength;
//	auto fpCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	
//	auto fpCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//
//	auto fpOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//
//
//	auto minMaxesPointer = alloc_tensorToZeros<int>(17, 1, 1);
//	
//
//
//
//	auto fpC = get3dArrCPU(fpCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto fnC = get3dArrCPU(fnCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto minMaxes = get3dArrCPU(minMaxesPointer, 9, 1, 1);
//
//	metaData.fpCount = fpC;
//	metaData.fnCount = fnC;
//	metaData.minMaxes = minMaxes;
//
//	metaData.fpCounter = get3dArrCPU(fpCounterPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fnCounter = get3dArrCPU(fnCounterPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fpOffset = get3dArrCPU(fpOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fnOffset = get3dArrCPU(fnOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
//
//	int workQueueAndRLLength = 200;
//	int workQueueWidth = 4;
//	int resultListWidth = 5;
//	//allocating to semiarbitrrary size 
//	auto workQueuePointer = alloc_tensorToZeros<uint32_t>(workQueueAndRLLength, workQueueWidth, 1);
//	auto resultListPointer = alloc_tensorToZeros<uint32_t>(workQueueAndRLLength, resultListWidth, 1);
//	metaData.workQueue = get3dArrCPU(workQueuePointer, workQueueAndRLLength, workQueueWidth, 1);
//	metaData.resultList = get3dArrCPU(resultListPointer, workQueueAndRLLength, resultListWidth, 1);
//
//
//
//
//	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);
//
//
//
//
//
//
//
//	uint32_t*** reducedGold = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//	uint32_t*** reducedSegm = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//
//
//	// arguments to pass
//	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
//	forFullBoolPrepArgs.metaData = metaData;
//	forFullBoolPrepArgs.numberToLookFor = 2;
//	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
//	forFullBoolPrepArgs.dbXLength = dbXLength;
//	forFullBoolPrepArgs.dbYLength = dbYLength;
//	forFullBoolPrepArgs.dbZLength = dbZLength;
//	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
//	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
//
//	forFullBoolPrepArgs.reducedGold = get3dArrCPU(reducedGold, mainXLength, mainYLength, MetaZLength);
//	forFullBoolPrepArgs.reducedSegm = get3dArrCPU(reducedSegm, mainXLength, mainYLength, MetaZLength);
//
//	forFullBoolPrepArgs.reducedArrsZdim = mainZLength;
//
//	forFullBoolPrepArgs.threads = threads;
//	forFullBoolPrepArgs.blocks = blocks;
//
//	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
//	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;
//
//	//populate segm  and gold Arr
//
//
//	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
//	auto arrSegmObj = forFullBoolPrepArgs.segmArr;
//	//setArrCPU(arrGoldObj,2, 3,4,2);
//	setArrCPU(arrGoldObj, dbXLength + 2, dbYLength + 1, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength + 2, dbYLength + 2, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength + 2 + 1, dbYLength + 2, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength * 2 + 2, dbYLength * 2 + 3, dbZLength * 2 + 4, 2);
//	setArrCPU(arrGoldObj, dbXLength * 2 + 3, dbYLength * 2 + 3, dbZLength * 2 + 4, 2);
//	setArrCPU(arrGoldObj, dbXLength * 3 + 2, dbYLength + 2, dbZLength * 2 + 5, 2);
//	setArrCPU(arrGoldObj, dbXLength * 4 + 9, dbYLength * 2, dbZLength * 2 + 1, 2);
//
//	//setArrCPU(arrGoldObj, dbXLength * 2 + 2, dbYLength * 2 + 3, dbZLength  + 4, 2);
//	//setArrCPU(arrSegmObj, dbXLength * 2 + 3, dbYLength * 2 + 3, dbZLength * 4 + 4, 2);
//	//setArrCPU(arrSegmObj, dbXLength * 3 + 2, dbYLength + 2, dbZLength * 5 + 5, 2);
//	//setArrCPU(arrGoldObj, dbXLength * 4 + 9, dbYLength * 2, dbZLength * 6 + 1, 2);
//
//
//
//
//	setArrCPU(arrSegmObj, dbXLength * 6 + 1, dbYLength * 2, dbZLength * 2 + 1, 2);
//	setArrCPU(arrSegmObj, dbXLength * 6 + 5, dbYLength * 2, dbZLength * 2 + 1, 2);
//	setArrCPU(arrSegmObj, dbXLength * 7 + 4, dbYLength * 2, dbZLength * 2 + 1, 2);
//
//
//	//printf("mainXLength %d mainYLength %d mainZLength %d \n", mainXLength, mainYLength, mainZLength);
//	firstMetaAndBoolRun(forFullBoolPrepArgs);
//
//
//
//
//
//	int i, j, k, value = 0;
//	//for (i = 0; i < mainXLength; i++) {
//	//	for (j = 0; j < mainYLength; j++) {
//	//		for (k = 0; k < MetaZLength; k++) {
//	//			//goldArr[k][j][i] = 1;
//	//			if (reducedGold[k][j][i] > 0) {
//	//				for (int tt = 0; tt < 32; tt++) {
//	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
//	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);
//
//	//					}
//	//				}
//
//
//	//			}
//	//		}
//	//	}
//	//}
//
//	//for (i = 0; i < mainXLength; i++) {
//	//	for (j = 0; j < mainYLength; j++) {
//	//		for (k = 0; k < MetaZLength; k++) {
//	//			//goldArr[k][j][i] = 1;
//	//			if (forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] > 0) {
//	//				for (int tt = 0; tt < 32; tt++) {
//	//					if ((forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] & (1 << (tt)))) {
//	//						printf("found in reduced fn [%d][%d][%d]\n", i, j, k * 32 + tt);
//
//	//					}
//	//				}
//
//
//	//			}
//	//		}
//	//	}
//	//}
//
//
//
//	i, j, k, value = 0;
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (isActive.arrP[k][j][i]) {
//
//					printf("found as Active [%d][%d][%d]\n", i, j, k);
//
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//
//	i, j, k, value = 0;
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (fpC.arrP[k][j][i] > 0) {
//					printf("found Fp %d  [%d][%d][%d]\n", fpC.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (fnC.arrP[k][j][i] > 0) {
//					printf("found Fn %d  [%d][%d][%d]\n", fnC.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//
//
//
//
//
//
//	//i, j, k, value = 0;
//	//for (i = 1; i < 9; i++) {
//	//	for (j = 0; j < 1; j++) {
//	//		for (k = 0; k < 1; k++) {
//	//			//goldArr[k][j][i] = 1;
//
//
//	//			printf("in minMaxes %d  [%d][%d][%d]\n", minMaxes.arrP[k][j][i], i, j, k);
//
//
//
//
//
//	//		}
//	//	}
//
//	//};
//
//
//
//
//
//	for (i = 0; i < mainXLength; i++) {
//		for (j = 0; j < mainYLength; j++) {
//			for (k = 0; k < mainZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (goldArr[k][j][i] > 0) {
//					printf("segmArr[%d][%d][%d] = %d\n", i, j, k, goldArr[k][j][i]);
//				}
//			}
//		}
//	}
//
//
//	printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! firstMetaPass!!!!!!!!!!!!!!!!!!!!!\n\n");
//
//	i, j, k, value = 0;
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (metaData.fpOffset.arrP[k][j][i] > 0) {
//					printf("Offsets Fp %d  [%d][%d][%d]\n", metaData.fpOffset.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (metaData.fnOffset.arrP[k][j][i] > 0) {
//					printf("Offsets Fn %d  [%d][%d][%d]\n", metaData.fnOffset.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//
//
//
//	for (i = 0; i < workQueueAndRLLength; i++) {
//	
//				//goldArr[k][j][i] = 1;
//				if (workQueuePointer[0][0][i] > 0) {
//					printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
//						, 0, 0, i
//						, workQueuePointer[0][0][i]
//						, workQueuePointer[0][1][i]
//						, workQueuePointer[0][2][i]
//						, workQueuePointer[0][3][i]
//						);
//				}
//
//	}
//
//
//
//	//	for (i = 0; i < dXLength; i++) {
//	//	for (j = 0; j < dYLength; j++) {
//	//		for (k = 0; k < dZLength; k++) {
//	//			//goldArr[k][j][i] = 1;
//	//			
//	//				printf("found in forDebugArr %d  [%d][%d][%d]\n", forDebugArr[k][j][i], i, j, k);
//
//
//	//		}
//	//	}
//	//}
//
//
//
//
//
//	free(metaData.minMaxes.arrP);
//	free(metaData.fpCount.arrP);
//	free(metaData.fnCount.arrP);
//	free(metaData.fpCounter.arrP);
//	free(metaData.fnCounter.arrP);
//	free(metaData.fpOffset.arrP);
//	free(metaData.fnOffset.arrP);
//	free(metaData.isActive.arrP);
//	free(metaData.isToBeActivated.arrP);
//	free(workQueuePointer);
//	free(resultListPointer);
//	free(metaData.isFull.arrP);
//
//
//
//
//
//	free(forDebugArr);
//	free(goldArr);
//	free(segmArr);
//	free(reducedSegm);
//	free(reducedGold);
//
//
//
//	//std::cout << longInts[3] << std::endl;
//
//}
//
//
//
//
//
//
//
//
//
//
//
//
//
