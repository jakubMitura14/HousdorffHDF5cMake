#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(ForBoolKernelArgs<TXPI>& fbArgs, int(&iterationNumb)[1]) {


    if ((iterationNumb[0] & 1) == 0) {
      return fbArgs.mainArrAPointer;

    }
    else {       
       return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(ForBoolKernelArgs<TXPPI>& fbArgs, int(&iterationNumb)[1]) {

    if ((iterationNumb[0] & 1) == 0) {
        //printf(" BB ");

      return fbArgs.mainArrBPointer;

    }
    else {       
       // printf(" AA ");

       return fbArgs.mainArrAPointer  ;

    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t& x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}


inline uint32_t isBitAtCPU(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}






//
///*
//given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
//*/
//#pragma once
//inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
//    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
//   // return target;
//}

///////////////////////////////// new functions





/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
inline __device__ void dilatateHelperForTransverse(const bool predicate,
    const uint8_t  paddingPos, const   int8_t  normalXChange, const  int8_t normalYchange
, uint32_t (&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6]
,const uint8_t forBorderYcoord,const  uint8_t forBorderXcoord
,const uint8_t metaDataCoordIndex,const uint32_t targetShmemOffset , uint32_t (&localBlockMetaData)[40], uint32_t& i ) {
    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[(i & 1) * 20+metaDataCoordIndex] < isGoldOffset) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                | mainShmem[targetShmemOffset + forBorderXcoord + forBorderYcoord * 32];

        };
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block


         mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] 
        = mainShmem[begSourceShmem+(threadIdx.x+ normalXChange)+(threadIdx.y+ normalYchange)*32]
             | mainShmem[begResShmem+threadIdx.x+threadIdx.y*32];
    
    }
   

}


#pragma once
inline __device__ void dilatateHelperTopDown( const uint8_t paddingPos, 
uint32_t(&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6], uint32_t(&localBlockMetaData)[40]
,const uint8_t metaDataCoordIndex
,const  uint8_t sourceBit 
, const uint8_t targetBit
, const uint32_t targetShmemOffset, uint32_t& i
) {
       // now we need to load the data from the neigbouring blocks
       //first checking is there anything to look to 
       if (localBlockMetaData[(i & 1) * 20+metaDataCoordIndex]< isGoldOffset) {
           if (isBitAt(mainShmem[begSourceShmem+ threadIdx.x + threadIdx.y * 32], targetBit)) {
                              // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
                              isAnythingInPadding[paddingPos] = true;
           };
           // if in bit of intrest of neighbour block is set

     mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] >> sourceBit) & 1) << targetBit;



           //mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
           //    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
           //    | (mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] & numberWithCorrBitSetInNeigh);

       }   

}


//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint32_t localBlockMetaData[(i & 1) * 20+]
//    , uint32_t mainShmem[], uint32_t i, MetaDataGPU metaData
//) {


//
///*
//constitutes end of pipeline  where we load data for next iteration if such is present
//*/
//template <typename TXPPI>
//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block& cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint32_t localBlockMetaData[(i & 1) * 20+]
//    , uint32_t mainShmem[], uint32_t i, MetaDataGPU metaData, uint32_t* metaDataArr
//) {
//
//    if (i + 1 <= worQueueStep[0]) {
//        cuda::memcpy_async(cta, (&localBlockMetaData[(i & 1) * 20+0]),
//            (&metaDataArr[(mainShmem[startOfLocalWorkQ + i - isGoldOffset * (mainShmem[startOfLocalWorkQ + i] >= isGoldOffset))
//                * metaData.metaDataSectionLength]])
//            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
//    }
//
//
//};

/*
we need to define here the function that will update the metadata result for the given block -
also if it is not padding pass we need to set the neighbouring blocks as to be activated according to the data in shmem
this will also include preparations for next round of iterations through blocks from work queue
isInPipeline - marks is it meant to be executed at the begining of the pipeline or after the pipeline
finilizing operations for last block
*/





inline __device__  void afterBlockClean(thread_block& cta
    , unsigned int(&worQueueStep)[1], uint32_t (&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem],const uint32_t i, MetaDataGPU& metaData
    , thread_block_tile<32>& tile
    , unsigned int(&localFpConter)[1], unsigned int(&localFnConter)[1]
    , unsigned int(&blockFpConter)[1], unsigned int (&blockFnConter)[1]
    , uint32_t*& metaDataArr
    , bool (&isAnythingInPadding)[6],bool (&isBlockFull)[1],const bool isPaddingPass, bool (&isGoldForLocQueue)[localWorkQueLength], uint32_t(&lastI)[1]
   ) {



    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                    //this will be executed only if fp or fn counters are bigger than 0 so not during first pass
        if (localFpConter[0] >= 0) {
            metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 3] += localFpConter[0];
            blockFpConter[0] += localFpConter[0];
            localFpConter[0] = 0;
        }
    };
    if (tile.thread_rank() == 8 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block

        if (localFnConter[0] >= 0) {
            metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 4] += localFnConter[0];

            blockFnConter[0] += localFnConter[0];
            localFnConter[0] = 0;
        }
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        //executed in case of previous block
        if (isBlockFull[0] && i >= 0) {
            //setting data in metadata that block is full
            metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 10 - (isGoldForLocQueue[i] * 2)] = true;
        }
        //resetting
        isBlockFull[0] = true;
    };



    
    //we do it only for non padding pass
    if (tile.thread_rank() < 6 && tile.meta_group_rank() == 1 && !isPaddingPass) {   
        //executed in case of previous block
        if (i>=0) {

          /*  if (isAnythingInPadding[tile.thread_rank()]) {
                printf("info in padding %d linMeta %d \n ", 13 + tile.thread_rank(), mainShmem[startOfLocalWorkQ + i]);

            }*/

            if (localBlockMetaData[(i & 1) * 20+   13+tile.thread_rank()] < isGoldOffset) {
                //printf("info in range %d linMeta %d \n ", 13 + tile.thread_rank(), mainShmem[startOfLocalWorkQ + i]);

                if (isAnythingInPadding[tile.thread_rank()]) {
                    metaDataArr[localBlockMetaData[(i & 1) * 20 + 13 + tile.thread_rank()] * metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]] = 1;
                    //printf("info in padding AND range %d linMeta %d \n ", 13 + tile.thread_rank(), mainShmem[startOfLocalWorkQ + i]);

                }
                
            }
        }
        isAnythingInPadding[0] = false;
    };



}





////////////////// with pipeline ofr barrier

/*
initial cleaning  and initializations of dilatation kernel

*/
inline __device__  void dilBlockInitialClean(thread_block_tile<32>& tile,
    const  bool isPaddingPass, int(&iterationNumb)[1],
    unsigned int(&localWorkQueueCounter)[1], unsigned int(&blockFpConter)[1],
    unsigned int(&blockFnConter)[1], unsigned int(&localFpConter)[1],
    unsigned int(&localFnConter)[1],bool (&isBlockFull)[1], 
    unsigned int(&fpFnLocCounter)[1],
    unsigned int(&localTotalLenthOfWorkQueue)[1], unsigned int(&globalWorkQueueOffset)[1]
    , unsigned int(&worQueueStep)[1], unsigned int*& minMaxes, unsigned int(&localMinMaxes)[5], uint32_t(&lastI)[1])
 {

    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] = true;
    };
    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        lastI[0] = UINT32_MAX;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }
}



/*
load work que from global memory
*/
inline __device__  void loadWorkQueue(uint32_t(&mainShmem)[lengthOfMainShmem], uint32_t*& workQueue
, bool(&isGoldForLocQueue)[localWorkQueLength], uint32_t& bigloop, unsigned int(&worQueueStep)[1]) {

    //to do change into barrier

    //cuda::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]), pipeline);


    for (uint16_t ii = 0; ii < worQueueStep[0]; ii++) {
        mainShmem[startOfLocalWorkQ + ii] = workQueue[bigloop + ii];
        isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
        mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];

    }
}


/*
loads metadata of given block to meta data 
*/
inline __device__  void loadMetaDataToShmem(thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
, uint32_t*& metaDataArr, MetaDataGPU& metaData, const uint8_t toAdd, uint32_t& ii) {
   
    //cuda::memcpy_async(cta, (&localBlockMetaData[(ii&1)*20]),
    //    (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd+ii])
    //        * metaData.metaDataSectionLength])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

    cuda::memcpy_async(cta, (&localBlockMetaData[((ii+1) & 1) * 20]),
        (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd + ii])
            * metaData.metaDataSectionLength])
        , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


}
