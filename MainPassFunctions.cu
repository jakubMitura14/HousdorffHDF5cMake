#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng source array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ array3dWithDimsGPU getSourceReduced(ForBoolKernelArgs<TXPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {       
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }    
    }


}
/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ array3dWithDimsGPU getTargetReduced(ForBoolKernelArgs<TXPPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) != 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {     
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }
    }


}
/*
loading data from appropriate reduce Arr to shared memory 
*/
#pragma once
template <typename TXI>
inline __device__ void loadDataToShmem(ForBoolKernelArgs<TXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU sourceReduced
, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i ) {
      sourceShared[threadIdx.x][ threadIdx.y]  
          = getTensorRow<uint32_t>(tensorslice, sourceReduced, sourceReduced.Ny
              , localWorkQueue[i][1] * fbArgs.dbYLength+ threadIdx.y
              , localWorkQueue[i][2])[localWorkQueue[i][0] *fbArgs.dbXLength+ threadIdx.x];
    //if (sourceShared[threadIdx.x][threadIdx.y] > 0) {
    //    printf("non zero in idX %d idY %d \n ", threadIdx.x, threadIdx.y);
    //}
}

/*
in order to be later able to analyze paddings we will save copy of the currently dilatated array 
(before dilatation) to global memory
*/
//template <typename TPYXI>
//inline __device__ void fromShmemToGlobal(ForBoolKernelArgs<TPYXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU target
//    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
//) {
//    
//    getTensorRow<uint32_t>(tensorslice, target, target.Ny, yMeta * fbArgs.dbYLength + threadIdx.y, zMeta)[xMeta * fbArgs.dbXLength + threadIdx.x]= sourceShared[threadIdx.x][ threadIdx.y];
//}
//


/*
saving dilatated data to global memory
*/
#pragma once
template <typename TXTI>
inline __device__ void saveToDilatationArr(ForBoolKernelArgs<TXTI> fbArgs, char* tensorslice, uint32_t resShared[32][32], array3dWithDimsGPU resDilatated
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
) {
    //if (resShared[threadIdx.x][threadIdx.y]>0) {
    //    printf("non zero in saving  in idX %d idY %d zMeta %d \n ", threadIdx.x, threadIdx.y, localWorkQueue[i][2]);

    //}
    //    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny,localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]; 

    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
    = resShared[threadIdx.x][ threadIdx.y];
}




///*
//checking in metadata weather block need to be validated
//*/
//#pragma once
//inline __device__ void isBlockToBeValidatedd(char* tensorslice, bool isBlockToBeValidated[1], array3dWithDimsGPU sourceReduced
//    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta)
//{
//    isBlockToBeValidated[0] = getTensorRow<bool>(tensorslice, sourceReduced, sourceReduced.Ny, yMeta , zMeta)[xMeta ];
//}
//

/*
marking that block is already full*/
#pragma once
inline __device__ void markIsBlockFull(char* tensorslice
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, bool isBlockFull, array3dWithDimsGPU targetMeta, coalesced_group active)
{
    if (isBlockFull && isToBeExecutedOnActive(active, 8)) {
        
      //  printf("set block as full  %d %d %d " , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]);

        getTensorRow<bool>(tensorslice, targetMeta, targetMeta.Ny, localWorkQueue[i][1], localWorkQueue[i][2])[localWorkQueue[i][0]] = true;
    }
}

/*
set the fp or fn counters of metadata
*/
#pragma once
inline __device__ void updateMetaCounters(char* tensorslice
    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta, uint16_t isGold,   array3dWithDimsGPU targetMeta,unsigned int fpOrFnCount,  coalesced_group active)
{
    if ( isToBeExecutedOnActive(active, 9)) {
        getTensorRow<unsigned int>(tensorslice, targetMeta, targetMeta.Ny, yMeta, zMeta)[xMeta] += fpOrFnCount;
    }
}





/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
*/
#pragma once
template <typename TXTOI>
inline __device__ void dilatateHelper(bool predicate,
    int paddingPos,   int  padingVariedB, int  normalXChange, int normalYchange
, uint32_t sourceShared[32][32], uint32_t resShared[32][32], bool isAnythingInPadding[6]
,bool predicateToLoadOutside, char* tensorslice, ForBoolKernelArgs<TXTOI> fbArgs, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
, unsigned int iterationNumb[1], uint8_t forBorderYcoord, uint8_t forBorderXcoord) {
   


    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (predicateToLoadOutside) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (sourceShared[threadIdx.x][threadIdx.y] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            //printf("looking padding currMetaX %d currMetaY %d currMetaZ %d X %d Y %d padding pos  paddingPos %d value %d  ;  %d  xChange %d   y Change %d  \n"
            //   ,  localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
            //, (localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord
            //    , (localWorkQueue[i][1] + normalYchange)* fbArgs.dbYLength + forBorderYcoord 
            //        , paddingPos,  isAnythingInPadding[paddingPos], sourceShared[threadIdx.x][threadIdx.y]
            //, normalXChange, normalYchange
            //);

            //printf("looking padding xChange %d yChange %d currMetaX %d currMetaY %d currMetaZ %d new X %d new Y %d value %d  \n"
            //    , normalXChange, normalYchange, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
            //, (localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord
            //    , (localWorkQueue[i][1] + normalYchange)* fbArgs.dbYLength + forBorderYcoord 
            //    , getTensorRow<uint32_t>(tensorslice, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb)
            //        , fbArgs.reducedGold.Ny, (localWorkQueue[i][1] + normalYchange) * fbArgs.dbYLength + forBorderYcoord
            //        , localWorkQueue[i][2])[(localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord]);


            resShared[threadIdx.x][threadIdx.y] = 
                resShared[threadIdx.x][threadIdx.y]
                    | getTensorRow<uint32_t>(tensorslice, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb)
                        , fbArgs.reducedGold.Ny, (localWorkQueue[i][1] + normalYchange) * fbArgs.dbYLength + forBorderYcoord
                            , localWorkQueue[i][2])[(localWorkQueue[i][0]+ normalXChange) * fbArgs.dbXLength + forBorderXcoord];
            ;

        }
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or 
        resShared[threadIdx.x][threadIdx.y] = sourceShared[threadIdx.x+ normalXChange][threadIdx.y+ normalYchange] | resShared[threadIdx.x][threadIdx.y];
    
    }
   

}




inline __device__ void clearisAnythingInPadding (bool isAnythingInPadding[6]) {

    auto active = coalesced_threads();
    #pragma unroll
    for (int ii; ii < 6; ii++) {
        if (isToBeExecutedOnActive(active, ii)) { isAnythingInPadding[ii] = 0; };
    };
}

/**
loading some data on single threads to shared memory that can be needed by all blocks 
*/

#pragma once
template <typename TYXI>
inline __device__ void loadSmallVars(ForBoolKernelArgs<TYXI> fbArgs, char* tensorslice
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], bool isBlockToBeValidated[1]
    ,uint16_t xMeta, uint16_t yMeta, uint16_t zMeta,uint16_t isGold, coalesced_group active
    , unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    //is to be validates
    if (isToBeExecutedOnActive(active, 0) && isGold == 1) {
       
        //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
        //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    , xMeta, yMeta, zMeta);


   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);*/

        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);
       // isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 1) && isGold == 0) {
       
    //    printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
    //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //< getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //, xMeta, yMeta, zMeta);



   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
  */      
        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
        //isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    //offsets
    if (isToBeExecutedOnActive(active, 2)) {// && isGold == 1
        resultfpOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpOffset, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
      //  printf("\n resultfpOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfpOffset[0], xMeta, yMeta, zMeta);

    };
    if (isToBeExecutedOnActive(active, 3) ) {//&& isGold == 0
       //printf("\n resultfnOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfnOffset[0], xMeta, yMeta, zMeta);


        resultfnOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnOffset, fbArgs.metaData.fnOffset.Ny, yMeta, zMeta)[xMeta];
    };
    // block counters
    if (isToBeExecutedOnActive(active, 4) && isGold == 1) {
        //auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
        //printf("setting ");

        localFpConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 5) && isGold == 0) {
        localFnConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta];
    };



}



#pragma once
inline __device__ void setNextBlockAsIsToBeActivated(coalesced_group active, char* tensorslice,
    int paddingNumb, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, 
    int xMetaChange, int yMetaChange, int zMetaChange
    ,array3dWithDimsGPU targetArr,bool isAnythingInPadding[6], bool isInRagePred
) {
    //if (isToBeExecutedOnActive(active, paddingNumb)) {
    //    printf("\n setting neighbour of %d %d %d to active- %d %d %d padding numb %d  isAnyInPadding %d\n"
    //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
    //        , localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange
    //        , paddingNumb , isAnythingInPadding[paddingNumb]
    //    );
    //}

    if (isAnythingInPadding[paddingNumb] && isToBeExecutedOnActive(active, paddingNumb) && isInRagePred) {


      //  printf(" \n saving to be actvated  xMeta %d yMeta %d zMeta %d isGold %d \n ", localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange, localWorkQueue[i][3]);


        getTensorRow<bool>(tensorslice, targetArr, targetArr.Ny, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange)[localWorkQueue[i][0] + xMetaChange] = true;
    };

}


#pragma once
inline __device__ void setNextBlocksActivity( char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU targetArr
    , bool isAnythingInPadding[6], coalesced_group active) {
    //0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    //top
    setNextBlockAsIsToBeActivated(active, tensorslice, 0, localWorkQueue, i, 0, 0, -1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]>0);
    //bottom
    setNextBlockAsIsToBeActivated(active, tensorslice, 1, localWorkQueue, i, 0, 0, 1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]<(targetArr.Nz-1));
    //left
    setNextBlockAsIsToBeActivated(active, tensorslice, 2, localWorkQueue, i, -1, 0, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][0]>0);
    //right
    setNextBlockAsIsToBeActivated(active, tensorslice, 3, localWorkQueue, i, 1, 0, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][0] < (targetArr.Nx - 1));
    //anterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 4, localWorkQueue, i, 0, 1, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][1] < (targetArr.Ny - 1));
    //posterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 5, localWorkQueue, i, 0, -1, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][1] > 0);



}

/*
given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
*/
#pragma once
inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
   // return target;
}


/*
now we will  additionally get bottom bit of block above and top of block below given they exist
*/
#pragma once
template <typename TXTIO>
inline __device__ void checkBlockToUpAndBottom (ForBoolKernelArgs<TXTIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU sourceArr, uint32_t resShared[32][32]) {
 
    //looking up
    if (localWorkQueue[i][2] > 0) {//boundary check
     //auto  xx =   getTensorRow<unsigned int>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]

      // printf(" looking up  ");
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , (fbArgs.dbZLength - 1) //sourceBit
            , resShared//target
            , 0//target bit
        );

    };
    //look down 
    if (localWorkQueue[i][2] < (fbArgs.metaData.MetaZLength - 1)) {//boundary check
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] + 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , 0 //sourceBit
            , resShared//target
            , (fbArgs.dbZLength - 1)//target bit
        );

    };


}


template <typename TXYYOI>
inline __device__ void clearShmemBeforeDilatation(ForBoolKernelArgs<TXYYOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    auto activeD = coalesced_threads();
    //resetting
    if (isToBeExecutedOnActive(activeD, 3)) {
        localWorkQueueCounter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 4)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 5)) {
        localFnConter[0] = 0;
    };

}




/*
establish wheather we still need dilatations in both passes
*/
template <typename TXTJIOP>
inline __device__ void checkIsToBeDilatated(ForBoolKernelArgs<TXTJIOP> fbArgs, char* tensorslice, bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]) {
    auto activeE = coalesced_threads();
    if (isToBeExecutedOnActive(activeE, 0)) {
        isGoldPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] * fbArgs.robustnessPercent)
    > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);
   
   //     isGoldPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] 
   // > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);

    }
    if (isToBeExecutedOnActive(activeE, 1)) {
       
        //TODO() remove 
     /*   auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8];
        unsigned int counter = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11];
        printf("\n  setting is to be dilatated   global fn count %d times robustness %f counter %f is to be accepted %d \n",xx
            , ceilf((float)xx * fbArgs.robustnessPercent), counter,  ( ceilf(xx* 0.95)> counter));
        */


       
        isSegmPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8] * fbArgs.robustnessPercent)
            > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);


    //    isSegmPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8]
      //      > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);

    }

}

/*
update global fp and fn counters and resets shared memory values after dilatations*/
template <typename TXTJIOI>
inline __device__ void updateGlobalCountersAndClear(ForBoolKernelArgs<TXTJIOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {
  
    auto activeD = coalesced_threads();
    if (isToBeExecutedOnActive(activeD, 6)) {
        //if (blockFpConter[0]>0) {
        //    printf("\n adding to global fp counter  %d \n", blockFpConter[0]);

        //}
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]), (blockFpConter[0]));
       // blockFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 7)) {
        //if (blockFnConter[0]) {
        //    printf("\n adding to global fn counter  %d \n", blockFnConter[0]);
        //}
       // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]), (blockFnConter[0]));
     //   blockFnConter[0] = 0;
    };

    if (isToBeExecutedOnActive(activeD, 8)) {
        // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9] = 0;
    };
    //resetting
    //if (isToBeExecutedOnActive(activeD, 3)) {
    //    localWorkQueueCounter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 4)) {
    //    localFpConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};

}




///////////////////////////////// bigger functions

/*
load from global to shared memory work queue
*/
#pragma once
template <typename TXTOIO>
inline __device__ void loadFromGlobalToLocalWorkQueue(ForBoolKernelArgs<TXTOIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint8_t bigloop, unsigned int globalWorkQueueOffset[1]
, unsigned int localTotalLenthOfWorkQueue[1], unsigned int worQueueStep[1] ) {
    for (uint8_t i = threadIdx.x + threadIdx.y*blockDim.x ; i < worQueueStep[0]; i += (blockDim.x * blockDim.y)) {
        if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
            //printf("adding to local wor queue  %d  \n", bigloop + i);
            localWorkQueue[i][0] = getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[bigloop + i];
            localWorkQueue[i][1] = getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[bigloop + i];
            localWorkQueue[i][2] = getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[bigloop + i];
            localWorkQueue[i][3] = getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[bigloop + i];

            //printf("\n local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d  i %d workQueLength %d workQueueStep %d globalWorkQueueOffset %d bigloop %d blockIdx.x %d"
            //    , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3],i
            //, localTotalLenthOfWorkQueue[0], worQueueStep[0], globalWorkQueueOffset[0], bigloop,  blockIdx.x);
        }
    }

}



/*
load and dilatates the entries in gold or segm ...
*/
#pragma once
template <typename TXTOIO>
inline __device__ void loadAndDilatateAndSave(ForBoolKernelArgs<TXTOIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint8_t bigloop,
    uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    ,bool isAnythingInPadding[6],    unsigned int iterationNumb[1], bool& isBlockFull, thread_block cta, uint16_t i
    ,bool isBlockToBeValidated[1], unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], unsigned int worQueueStep[1]
) {

    //first we load data to source shmem
    loadDataToShmem(fbArgs, tensorslice, sourceShared, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb), localWorkQueue, i);

    /// ///////////////// dilatations
    // first we perform up and down dilatations
    resShared[threadIdx.x][threadIdx.y] = bitDilatate(sourceShared[threadIdx.x][threadIdx.y]);

    //we also need to set shmem paddings on the basis of first and last bits ...

    //top            0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    if (isBitAt(sourceShared[threadIdx.x][threadIdx.y], 0)) {
        // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
        isAnythingInPadding[0] = true;
    };
    //shmemPaddingsTopBottom[threadIdx.x][threadIdx.y][0]=true; };
//bottom
    if (isBitAt(sourceShared[threadIdx.x][threadIdx.y], (fbArgs.dbZLength - 1))) {
        //shmemPaddingsTopBottom[threadIdx.x][threadIdx.y][1] = true;
        isAnythingInPadding[1] = true;
    };
    //now we will  additionally get bottom bit of block above and top of block below given they exist 
    checkBlockToUpAndBottom(fbArgs, tensorslice, localWorkQueue, i, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb), resShared);

    //we also need to save data into shared memory weather this block is marked to be validated (are there any voxels that can be potentially saved into result queue)
    auto activeC = coalesced_threads();

    loadSmallVars(fbArgs, tensorslice, resultfpOffset, resultfnOffset, isBlockToBeValidated, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]
        , activeC, localFpConter, localFnConter);




    sync(cta);//we loaded and  dilatated up and down - we need also to dilatate anterior, posterior, Hovewer in those cases we need also to check boundary conditions ...


              //TODO() 4 corner threads has too much work and probably couse warp divergence ...- so those that for example have both threadidx and y=0 or max ...
    //we will also immidiately send data to    

    //#left
    dilatateHelper((threadIdx.x == 0), 2, threadIdx.y, (-1), (0), sourceShared, resShared, isAnythingInPadding, localWorkQueue[i][0] > 0,
        tensorslice, fbArgs, localWorkQueue, i, iterationNumb,
        threadIdx.y, (fbArgs.dbXLength - 1));
    ////right
    dilatateHelper((threadIdx.x == (fbArgs.dbXLength - 1)), 3, threadIdx.y, (1), (0), sourceShared, resShared, isAnythingInPadding
        , (localWorkQueue[i][0] < (fbArgs.metaData.metaXLength - 1)), tensorslice, fbArgs, localWorkQueue, i, iterationNumb, threadIdx.y, 0);
    sync(cta);// we are synchronizing just becouse of corners TODO() rethink corners                
    //posterior
    dilatateHelper((threadIdx.y == 0), 5, threadIdx.x, (0), (-1), sourceShared, resShared, isAnythingInPadding, localWorkQueue[i][1] > 0,
        tensorslice, fbArgs, localWorkQueue, i, iterationNumb, (fbArgs.dbYLength - 1), threadIdx.x);
    //anterior
    dilatateHelper((threadIdx.y == (fbArgs.dbYLength - 1)), 4, threadIdx.x, (0), (1), sourceShared, resShared, isAnythingInPadding
        , localWorkQueue[i][1] < (fbArgs.dbYLength - 1), tensorslice, fbArgs, localWorkQueue, i, iterationNumb, 0, threadIdx.x);



    //syncing we now check is block full
    //marking that we have no more space for dilatations
    isBlockFull = (resShared[threadIdx.x][threadIdx.y] == UINT32_MAX);

    isBlockFull = __syncthreads_and(isBlockFull); ;// all dilatations completed 


   //now we need to move the data into global memory - so dilatated arrays to dilatation reduced arrays and paddings to paddings store
    saveToDilatationArr(fbArgs, tensorslice, resShared, getTargetReduced(fbArgs, localWorkQueue, i, iterationNumb), localWorkQueue, i);

}







/*
load and dilatates the entries in gold or segm ...
*/
#pragma once
template <typename TXTOIO>
inline __device__ void validateAndUpMetaCounter(ForBoolKernelArgs<TXTOIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint8_t bigloop,
    uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , bool isAnythingInPadding[6], unsigned int iterationNumb[1], bool isBlockFull, thread_block cta, uint16_t i
    , bool isBlockToBeValidated[1], unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], unsigned int worQueueStep[1],  unsigned int& old
    , unsigned int blockFpConter[1], unsigned int blockFnConter[1]
) {
    if ((isBlockToBeValidated[0] || iterationNumb[0] == 0) && !isBlockFull) {
        //now first we need to check for bits that are true now after dilatation but were not in source we will save it in res shmem becouse we will no longer need it
        resShared[threadIdx.x][threadIdx.y] = ((~sourceShared[threadIdx.x][threadIdx.y]) & resShared[threadIdx.x][threadIdx.y]);
        //now we load appropriate reference array (opposite to source)

        if (localWorkQueue[i][3] == 0) { loadDataToShmem(fbArgs, tensorslice, sourceShared, fbArgs.reducedGoldRef, localWorkQueue, i); };
        if (localWorkQueue[i][3] == 1) { loadDataToShmem(fbArgs, tensorslice, sourceShared, fbArgs.reducedSegmRef, localWorkQueue, i); };

        //we now look for bits prasent in both reference arrays and current one
        resShared[threadIdx.x][threadIdx.y] = ((sourceShared[threadIdx.x][threadIdx.y]) & resShared[threadIdx.x][threadIdx.y]);
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(resShared[threadIdx.x][threadIdx.y], bitPos)) {
                //first we add to the resList
                //TODO consider first passing it into shared memory and then async mempcy ...
                //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)


                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (localWorkQueue[i][3] == 1) { old = atomicAdd(&(localFpConter[0]), 1) + resultfnOffset[0]; };
                if (localWorkQueue[i][3] == 0) { old = atomicAdd(&(localFnConter[0]), 1) + resultfpOffset[0]; };

                //if (localWorkQueue[i][3] == 1) { old = atomicAdd(&(localFpConter[0]), 1) + resultfnOffset[0]; };
                //if (localWorkQueue[i][3] == 0) { old = atomicAdd(&(localFnConter[0]), 1) + resultfpOffset[0]; };

                //if (((localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x)<31
                //    || (localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x)>80
                //    || (localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y)<12
                //    || (localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y)>62)
                //   // ||(localWorkQueue[i][2] * fbArgs.dbZLength + bitPos)!=31    ||  (localWorkQueue[i][2] * fbArgs.dbZLength + bitPos)!=41 
                //    ) {
                //    printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d  fpLocCounter %d  fnLocCounter %d   resultfpOffset %d  resultfnOffset %d  xMeta %d yMeta %d zMeta %d isGold %d \n ",

                //        localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x,
                //        localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y,
                //        localWorkQueue[i][2] * fbArgs.dbZLength + bitPos,
                //        localWorkQueue[i][3],
                //        iterationNumb[0]
                //        , old
                //        , localFpConter[0]
                //        , localFnConter[0]
                //        , resultfnOffset[0]
                //        , resultfpOffset[0]
                //        , localWorkQueue[i][0]
                //        , localWorkQueue[i][1]
                //        , localWorkQueue[i][2]
                //        , localWorkQueue[i][3]

                //    );
                //}
                //else {
                //    printf(" *** ");
                //}
                 
                //TODO remove
                //getTensorRow<int>(tensorslice, fbArgs.forDebugArr, fbArgs.forDebugArr.Ny, 0, 0)[old] += 1;





                fbArgs.metaData.resultList[old*5]= (localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x);
                fbArgs.metaData.resultList[old*5+1]= (localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y);
                fbArgs.metaData.resultList[old*5+2]= (localWorkQueue[i][2] * fbArgs.dbZLength + bitPos);
                fbArgs.metaData.resultList[old*5+3]= (localWorkQueue[i][3]);
                fbArgs.metaData.resultList[old*5+4]= (iterationNumb[0]);

                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 0, 0)[old] = int(localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 1, 0)[old] = int(localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 2, 0)[old] = int(localWorkQueue[i][2] * fbArgs.dbZLength + bitPos);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 3, 0)[old] = int(localWorkQueue[i][3]);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old] = int(iterationNumb[0]);
            
            




    //            if (getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old] !=9) {
    //    printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d  fpLocCounter %d  fnLocCounter %d   resultfpOffset %d  resultfnOffset %d  xMeta %d yMeta %d zMeta %d isGold %d \n ",

    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 0, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 1, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 2, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 3, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old]
    //        , old
    //        , localFpConter[0]
    //        , localFnConter[0]
    //        , resultfnOffset[0]
    //        , resultfpOffset[0]
    //        , localWorkQueue[i][0]
    //        , localWorkQueue[i][1]
    //        , localWorkQueue[i][2]
    //        , localWorkQueue[i][3]

    //    );
    //}
    //else {
    //    printf(" *** ");
    //}
            
            }
        }
        sync(cta);


        coalesced_group activeE = coalesced_threads();
        //update metadata  fp, fn conters
        if (localWorkQueue[i][3] == 1) {//gold
            updateMetaCounters(tensorslice, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3], fbArgs.metaData.fpCounter, localFpConter[0], activeE);
        };
        if (localWorkQueue[i][3] == 0) {//segm
            updateMetaCounters(tensorslice, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3], fbArgs.metaData.fnCounter, localFnConter[0], activeE);
        };
        if (isToBeExecutedOnActive(activeE, 4)) {
            blockFpConter[0] += localFpConter[0];
            localFpConter[0] = 0;
        };
        if (isToBeExecutedOnActive(activeE, 5)) {
            blockFnConter[0] += localFnConter[0];
            localFnConter[0] = 0;

        };

    }
}








