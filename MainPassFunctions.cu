#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng source array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ array3dWithDimsGPU getSourceReduced(ForBoolKernelArgs<TXPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {       
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }    
    }


}
/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ array3dWithDimsGPU getTargetReduced(ForBoolKernelArgs<TXPPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) != 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {     
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }
    }


}
/*
loading data from appropriate reduce Arr to shared memory 
*/
#pragma once
template <typename TXI>
inline __device__ void loadDataToShmem(ForBoolKernelArgs<TXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU sourceReduced
, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i ) {
      sourceShared[threadIdx.x][ threadIdx.y]  
          = getTensorRow<uint32_t>(tensorslice, sourceReduced, sourceReduced.Ny
              , localWorkQueue[i][1] * fbArgs.dbYLength+ threadIdx.y
              , localWorkQueue[i][2])[localWorkQueue[i][0] *fbArgs.dbXLength+ threadIdx.x];
    //if (sourceShared[threadIdx.x][threadIdx.y] > 0) {
    //    printf("non zero in idX %d idY %d \n ", threadIdx.x, threadIdx.y);
    //}
}

/*
in order to be later able to analyze paddings we will save copy of the currently dilatated array 
(before dilatation) to global memory
*/
//template <typename TPYXI>
//inline __device__ void fromShmemToGlobal(ForBoolKernelArgs<TPYXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU target
//    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
//) {
//    
//    getTensorRow<uint32_t>(tensorslice, target, target.Ny, yMeta * fbArgs.dbYLength + threadIdx.y, zMeta)[xMeta * fbArgs.dbXLength + threadIdx.x]= sourceShared[threadIdx.x][ threadIdx.y];
//}
//


/*
saving dilatated data to global memory
*/
#pragma once
template <typename TXTI>
inline __device__ void saveToDilatationArr(ForBoolKernelArgs<TXTI> fbArgs, char* tensorslice, uint32_t resShared[32][32], array3dWithDimsGPU resDilatated
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
) {
    //if (resShared[threadIdx.x][threadIdx.y]>0) {
    //    printf("non zero in saving  in idX %d idY %d zMeta %d \n ", threadIdx.x, threadIdx.y, localWorkQueue[i][2]);

    //}
    //    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny,localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]; 

    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
    = resShared[threadIdx.x][ threadIdx.y];
}




///*
//checking in metadata weather block need to be validated
//*/
//#pragma once
//inline __device__ void isBlockToBeValidatedd(char* tensorslice, bool isBlockToBeValidated[1], array3dWithDimsGPU sourceReduced
//    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta)
//{
//    isBlockToBeValidated[0] = getTensorRow<bool>(tensorslice, sourceReduced, sourceReduced.Ny, yMeta , zMeta)[xMeta ];
//}
//

/*
marking that block is already full*/
#pragma once
inline __device__ void markIsBlockFull(char* tensorslice
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, bool isBlockFull, array3dWithDimsGPU targetMeta, coalesced_group active)
{
    if (isBlockFull && isToBeExecutedOnActive(active, 8)) {
        
      //  printf("set block as full  %d %d %d " , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]);

        getTensorRow<bool>(tensorslice, targetMeta, targetMeta.Ny, localWorkQueue[i][1], localWorkQueue[i][2])[localWorkQueue[i][0]] = true;
    }
}

/*
set the fp or fn counters of metadata
*/
#pragma once
inline __device__ void updateMetaCounters(char* tensorslice
    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta, uint16_t isGold,   array3dWithDimsGPU targetMeta,unsigned int fpOrFnCount,  coalesced_group active)
{
    if ( isToBeExecutedOnActive(active, 9)) {
        getTensorRow<unsigned int>(tensorslice, targetMeta, targetMeta.Ny, yMeta, zMeta)[xMeta] += fpOrFnCount;
    }
}





/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


inline uint32_t isBitAtCPU(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
*/
#pragma once
template <typename TXTOI>
inline __device__ void dilatateHelper(bool predicate,
    int paddingPos,   int  padingVariedB, int  normalXChange, int normalYchange
, uint32_t sourceShared[32][32], uint32_t resShared[32][32], bool isAnythingInPadding[6]
,bool predicateToLoadOutside, char* tensorslice, ForBoolKernelArgs<TXTOI> fbArgs, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
, unsigned int iterationNumb[1], uint8_t forBorderYcoord, uint8_t forBorderXcoord) {
   


    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (predicateToLoadOutside) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (sourceShared[threadIdx.x][threadIdx.y] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            //printf("looking padding currMetaX %d currMetaY %d currMetaZ %d X %d Y %d padding pos  paddingPos %d value %d  ;  %d  xChange %d   y Change %d  \n"
            //   ,  localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
            //, (localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord
            //    , (localWorkQueue[i][1] + normalYchange)* fbArgs.dbYLength + forBorderYcoord 
            //        , paddingPos,  isAnythingInPadding[paddingPos], sourceShared[threadIdx.x][threadIdx.y]
            //, normalXChange, normalYchange
            //);

            //printf("looking padding xChange %d yChange %d currMetaX %d currMetaY %d currMetaZ %d new X %d new Y %d value %d  \n"
            //    , normalXChange, normalYchange, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
            //, (localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord
            //    , (localWorkQueue[i][1] + normalYchange)* fbArgs.dbYLength + forBorderYcoord 
            //    , getTensorRow<uint32_t>(tensorslice, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb)
            //        , fbArgs.reducedGold.Ny, (localWorkQueue[i][1] + normalYchange) * fbArgs.dbYLength + forBorderYcoord
            //        , localWorkQueue[i][2])[(localWorkQueue[i][0] + normalXChange) * fbArgs.dbXLength + forBorderXcoord]);


            resShared[threadIdx.x][threadIdx.y] = 
                resShared[threadIdx.x][threadIdx.y]
                    | getTensorRow<uint32_t>(tensorslice, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb)
                        , fbArgs.reducedGold.Ny, (localWorkQueue[i][1] + normalYchange) * fbArgs.dbYLength + forBorderYcoord
                            , localWorkQueue[i][2])[(localWorkQueue[i][0]+ normalXChange) * fbArgs.dbXLength + forBorderXcoord];
            ;

        }
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or 
        resShared[threadIdx.x][threadIdx.y] = sourceShared[threadIdx.x+ normalXChange][threadIdx.y+ normalYchange] | resShared[threadIdx.x][threadIdx.y];
    
    }
   

}




inline __device__ void clearisAnythingInPadding (bool isAnythingInPadding[6]) {

    auto active = coalesced_threads();
    #pragma unroll
    for (int ii; ii < 6; ii++) {
        if (isToBeExecutedOnActive(active, ii)) { isAnythingInPadding[ii] = 0; };
    };
}

/**
loading some data on single threads to shared memory that can be needed by all blocks 
*/

#pragma once
template <typename TYXI>
inline __device__ void loadSmallVars(ForBoolKernelArgs<TYXI> fbArgs, char* tensorslice
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], bool isBlockToBeValidated[1]
    ,uint16_t xMeta, uint16_t yMeta, uint16_t zMeta,uint16_t isGold, coalesced_group active
    , unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    //is to be validates
    if (isToBeExecutedOnActive(active, 0) && isGold == 1) {
       
        //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
        //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    , xMeta, yMeta, zMeta);


   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);*/

        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);
       // isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 1) && isGold == 0) {
       
    //    printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
    //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //< getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //, xMeta, yMeta, zMeta);



   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
  */      
        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
        //isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    //offsets
    if (isToBeExecutedOnActive(active, 2)) {// && isGold == 1
        resultfpOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpOffset, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
      //  printf("\n resultfpOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfpOffset[0], xMeta, yMeta, zMeta);

    };
    if (isToBeExecutedOnActive(active, 3) ) {//&& isGold == 0
       //printf("\n resultfnOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfnOffset[0], xMeta, yMeta, zMeta);


        resultfnOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnOffset, fbArgs.metaData.fnOffset.Ny, yMeta, zMeta)[xMeta];
    };
    // block counters
    if (isToBeExecutedOnActive(active, 4) && isGold == 1) {
        //auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
        //printf("setting ");

        localFpConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 5) && isGold == 0) {
        localFnConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta];
    };



}



#pragma once
inline __device__ void setNextBlockAsIsToBeActivated(coalesced_group active, char* tensorslice,
    int paddingNumb, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, 
    int xMetaChange, int yMetaChange, int zMetaChange
    ,array3dWithDimsGPU targetArr,bool isAnythingInPadding[6], bool isInRagePred
) {
    //if (isToBeExecutedOnActive(active, paddingNumb)) {
    //    printf("\n setting neighbour of %d %d %d to active- %d %d %d padding numb %d  isAnyInPadding %d\n"
    //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
    //        , localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange
    //        , paddingNumb , isAnythingInPadding[paddingNumb]
    //    );
    //}

    if (isAnythingInPadding[paddingNumb] && isToBeExecutedOnActive(active, paddingNumb) && isInRagePred) {


      //  printf(" \n saving to be actvated  xMeta %d yMeta %d zMeta %d isGold %d \n ", localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange, localWorkQueue[i][3]);


        getTensorRow<bool>(tensorslice, targetArr, targetArr.Ny, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange)[localWorkQueue[i][0] + xMetaChange] = true;
    };

}


#pragma once
inline __device__ void setNextBlocksActivity( char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU targetArr
    , bool isAnythingInPadding[6], coalesced_group active) {
    //0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    //top
    setNextBlockAsIsToBeActivated(active, tensorslice, 0, localWorkQueue, i, 0, 0, -1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]>0);
    //bottom
    setNextBlockAsIsToBeActivated(active, tensorslice, 1, localWorkQueue, i, 0, 0, 1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]<(targetArr.Nz-1));
    //left
    setNextBlockAsIsToBeActivated(active, tensorslice, 2, localWorkQueue, i, -1, 0, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][0]>0);
    //right
    setNextBlockAsIsToBeActivated(active, tensorslice, 3, localWorkQueue, i, 1, 0, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][0] < (targetArr.Nx - 1));
    //anterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 4, localWorkQueue, i, 0, 1, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][1] < (targetArr.Ny - 1));
    //posterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 5, localWorkQueue, i, 0, -1, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][1] > 0);



}

/*
given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
*/
#pragma once
inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
   // return target;
}


/*
now we will  additionally get bottom bit of block above and top of block below given they exist
*/
#pragma once
template <typename TXTIO>
inline __device__ void checkBlockToUpAndBottom (ForBoolKernelArgs<TXTIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU sourceArr, uint32_t resShared[32][32]) {
 
    //looking up
    if (localWorkQueue[i][2] > 0) {//boundary check
     //auto  xx =   getTensorRow<unsigned int>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]

      // printf(" looking up  ");
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , (fbArgs.dbZLength - 1) //sourceBit
            , resShared//target
            , 0//target bit
        );

    };
    //look down 
    if (localWorkQueue[i][2] < (fbArgs.metaData.MetaZLength - 1)) {//boundary check
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] + 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , 0 //sourceBit
            , resShared//target
            , (fbArgs.dbZLength - 1)//target bit
        );

    };


}


template <typename TXYYOI>
inline __device__ void clearShmemBeforeDilatation(ForBoolKernelArgs<TXYYOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    auto activeD = coalesced_threads();
    //resetting
    if (isToBeExecutedOnActive(activeD, 3)) {
        localWorkQueueCounter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 4)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 5)) {
        localFnConter[0] = 0;
    };

}




/*
establish wheather we still need dilatations in both passes
*/
template <typename TXTJIOP>
inline __device__ void checkIsToBeDilatated(ForBoolKernelArgs<TXTJIOP> fbArgs, char* tensorslice, bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]) {
    auto activeE = coalesced_threads();
    if (isToBeExecutedOnActive(activeE, 0)) {
        isGoldPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] * fbArgs.robustnessPercent)
    > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);
   
   //     isGoldPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] 
   // > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);

    }
    if (isToBeExecutedOnActive(activeE, 1)) {
       
        //TODO() remove 
     /*   auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8];
        unsigned int counter = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11];
        printf("\n  setting is to be dilatated   global fn count %d times robustness %f counter %f is to be accepted %d \n",xx
            , ceilf((float)xx * fbArgs.robustnessPercent), counter,  ( ceilf(xx* 0.95)> counter));
        */


       
        isSegmPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8] * fbArgs.robustnessPercent)
            > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);


    //    isSegmPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8]
      //      > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);

    }

}

/*
update global fp and fn counters and resets shared memory values after dilatations*/
template <typename TXTJIOI>
inline __device__ void updateGlobalCountersAndClear(ForBoolKernelArgs<TXTJIOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {
  
    auto activeD = coalesced_threads();
    if (isToBeExecutedOnActive(activeD, 6)) {
        //if (blockFpConter[0]>0) {
        //    printf("\n adding to global fp counter  %d \n", blockFpConter[0]);

        //}
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]), (blockFpConter[0]));
       // blockFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 7)) {
        //if (blockFnConter[0]) {
        //    printf("\n adding to global fn counter  %d \n", blockFnConter[0]);
        //}
       // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]), (blockFnConter[0]));
     //   blockFnConter[0] = 0;
    };

    if (isToBeExecutedOnActive(activeD, 8)) {
        // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9] = 0;
    };
    //resetting
    //if (isToBeExecutedOnActive(activeD, 3)) {
    //    localWorkQueueCounter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 4)) {
    //    localFpConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};

}




///////////////////////////////// bigger functions









