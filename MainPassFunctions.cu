#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(ForBoolKernelArgs<TXPI> fbArgs, uint32_t iterationNumb[1]) {


    if ((iterationNumb[0] & 1) == 0) {
      return fbArgs.mainArrAPointer;
    }
    else {       
       return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(ForBoolKernelArgs<TXPPI> fbArgs, uint32_t iterationNumb[1]) {

    if ((iterationNumb[0] & 1) == 0) {
      return fbArgs.mainArrBPointer;
    }
    else {       
       return fbArgs.mainArrAPointer  ;
    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


inline uint32_t isBitAtCPU(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}






#pragma once
inline __device__ void setNextBlockAsIsToBeActivated(coalesced_group active, char* tensorslice,
    int paddingNumb, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, 
    int xMetaChange, int yMetaChange, int zMetaChange
    ,array3dWithDimsGPU targetArr,bool isAnythingInPadding[6], bool isInRagePred
) {
    //if (isToBeExecutedOnActive(active, paddingNumb)) {
    //    printf("\n setting neighbour of %d %d %d to active- %d %d %d padding numb %d  isAnyInPadding %d\n"
    //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
    //        , localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange
    //        , paddingNumb , isAnythingInPadding[paddingNumb]
    //    );
    //}

    if (isAnythingInPadding[paddingNumb] && isToBeExecutedOnActive(active, paddingNumb) && isInRagePred) {


      //  printf(" \n saving to be actvated  xMeta %d yMeta %d zMeta %d isGold %d \n ", localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange, localWorkQueue[i][3]);


        getTensorRow<bool>(tensorslice, targetArr, targetArr.Ny, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange)[localWorkQueue[i][0] + xMetaChange] = true;
    };

}


#pragma once
inline __device__ void setNextBlocksActivity( char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU targetArr
    , bool isAnythingInPadding[6], coalesced_group active) {
    //0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    //top
    setNextBlockAsIsToBeActivated(active, tensorslice, 0, localWorkQueue, i, 0, 0, -1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]>0);
    //bottom
    setNextBlockAsIsToBeActivated(active, tensorslice, 1, localWorkQueue, i, 0, 0, 1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]<(targetArr.Nz-1));
    //left
    setNextBlockAsIsToBeActivated(active, tensorslice, 2, localWorkQueue, i, -1, 0, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][0]>0);
    //right
    setNextBlockAsIsToBeActivated(active, tensorslice, 3, localWorkQueue, i, 1, 0, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][0] < (targetArr.Nx - 1));
    //anterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 4, localWorkQueue, i, 0, 1, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][1] < (targetArr.Ny - 1));
    //posterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 5, localWorkQueue, i, 0, -1, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][1] > 0);



}

/*
given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
*/
#pragma once
inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
   // return target;
}

///////////////////////////////// new functions


/*
calculate index in main shmem where array that is source for this dilatation round is present
*/
#pragma once
inline __device__ uint16_t getIndexForSourceShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    ,  uint16_t i, bool isGold){
    return  metaData.mainArrXLength * 
    ((1 -isGold)// here calculating offset depending on what iteration and is gold;
        + (mainShmem[startOfLocalWorkQ + i] - (UINT16_MAX * (isGold))) * metaData.mainArrSectionLength )  ;// offset depending on linear index of metadata block of intrest

}
#pragma once
inline __device__ uint16_t getFullIndexForSourceShmemTotal(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint16_t i, bool isGold) {
    return  (( (mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * isGold) >0)* (-32)) // we check weather there is anything to the left - not on left border if so we load left 32 entries
        + getIndexForSourceShmem(metaData, mainShmem,  i, isGold);
}




/*
getting index where we should put first load - so data about this block and if apply block to the left and right
*/
#pragma once
inline __device__ uint16_t getIndexOfShmemToFirstLoad(uint32_t mainShmem[lengthOfMainShmem], uint16_t i, bool isGold) {
    return  (((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX 
        * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX)) > 0)* (-32)) + begSourceShmem;
}

/*
calculating where to put the data from res shmem - so data after dilatation back to global memory
*/
#pragma once
inline __device__ uint16_t getLengthOfShmemToFirstLoad(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint16_t i, bool isGold) {
    return    (metaData.mainArrXLength + 32 * (((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (isGold)) > 0)
        + ((mainShmem[startOfLocalWorkQ + i] - UINT16_MAX * (isGold)) < (metaData.totalMetaLength - 1))));// offset depending on linear index of this block
}




/*
calculate index in main shmem where array that is source for this dilatation round is present in the neighboutring block ...
*/
#pragma once
inline __device__ uint16_t getIndexForNeighbourForShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint32_t isGold[1], uint16_t currLinIndM[1], uint16_t localBlockMetaData[19],  size_t inMetaIndex) {
       return  metaData.mainArrXLength * 
    ((1 - (isGold[1]) )// here calculating offset depending on what iteration and is gold;
        + (localBlockMetaData[inMetaIndex]) * metaData.mainArrSectionLength )  ;// offset depending on linear index of metadata block of intrest
}

/*
calculating where to put the data from res shmem - so data after dilatation back to global memory
*/
inline __device__ uint16_t getIndexForSaveResShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint32_t isGold[1], uint16_t currLinIndM[1], uint16_t localBlockMetaData[19]) {
    return  metaData.mainArrXLength * (isGold[1])// here calculating offset depending on what iteration and is gold;
        + (currLinIndM[0] * metaData.mainArrSectionLength);// offset depending on linear index of this block
}



/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
inline __device__ void dilatateHelperForTransverse(bool predicate,
    uint8_t paddingPos,    int8_t  normalXChange, int8_t normalYchange
, uint32_t mainShmem[], bool isAnythingInPadding[6]
,uint8_t forBorderYcoord, uint8_t forBorderXcoord
,uint8_t metaDataCoordIndex, uint16_t targetShmemOffset , uint16_t localBlockMetaData[20]) {
    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[metaDataCoordIndex] < UINT16_MAX) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[threadIdx.x + threadIdx.y * 32] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                | mainShmem[targetShmemOffset + forBorderXcoord + forBorderYcoord * 32];

        };
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block


         mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] 
        = mainShmem[(threadIdx.x+ normalXChange)+(threadIdx.y+ normalYchange)*32] | mainShmem[begResShmem+threadIdx.x+threadIdx.y*32];
    
    }
   

}


#pragma once
inline __device__ void dilatateHelperTopDown( uint8_t paddingPos, 
uint32_t* mainShmem, bool isAnythingInPadding[6], uint16_t localBlockMetaData[20]
,uint8_t metaDataCoordIndex
, uint32_t numberbitOfIntrestInBlock // represent a uint32 number that has a bit of intrest in this block set and all others 0 
, uint32_t numberWithCorrBitSetInNeigh// represent a uint32 number that has a bit of intrest in neighbouring block set and all others 0 
, uint16_t targetShmemOffset
) {
       // now we need to load the data from the neigbouring blocks
       //first checking is there anything to look to 
       if (localBlockMetaData[metaDataCoordIndex]< UINT16_MAX) {
           //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
           if (mainShmem[threadIdx.x + threadIdx.y * 32] & numberbitOfIntrestInBlock) {
                              // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
                              isAnythingInPadding[0] = true;
           };
           mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
               mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
               | (mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] & numberWithCorrBitSetInNeigh);

       }   

}


//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint16_t localBlockMetaData[]
//    , uint32_t mainShmem[], uint16_t i, MetaDataGPU metaData
//) {


//
///*
//constitutes end of pipeline  where we load data for next iteration if such is present
//*/
//template <typename TXPPI>
//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block& cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint16_t localBlockMetaData[]
//    , uint32_t mainShmem[], uint16_t i, MetaDataGPU metaData, uint16_t* metaDataArr
//) {
//
//    if (i + 1 <= worQueueStep[0]) {
//        cuda::memcpy_async(cta, (&localBlockMetaData[0]),
//            (&metaDataArr[(mainShmem[startOfLocalWorkQ + i - UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX))
//                * metaData.metaDataSectionLength]])
//            , cuda::aligned_size_t<4>(sizeof(uint16_t) * 20), pipeline);
//    }
//
//
//};

/*
we need to define here the function that will update the metadata result for the given block -
also if it is not padding pass we need to set the neighbouring blocks as to be activated according to the data in shmem
this will also include preparations for next round of iterations through blocks from work queue
isInPipeline - marks is it meant to be executed at the begining of the pipeline or after the pipeline
finilizing operations for last block
*/





inline __device__  void afterBlockClean(thread_block cta
    , unsigned int worQueueStep[1], uint16_t localBlockMetaDataOld[6]
    , uint32_t mainShmem[], uint16_t i, MetaDataGPU metaData
    , thread_block_tile<32> tile
    , unsigned int localFpConter[1], unsigned int localFnConter[1]
    , unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , uint16_t* metaDataArr, uint16_t oldLinIndM[1], uint32_t oldIsGold[1]
    , bool isAnythingInPadding[6],bool isBlockFull[1], bool isPaddingPass) {



    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block
                    //this will be executed only if fp or fn counters are bigger than 0 so not during first pass
        if (localFpConter[0] > 0) {
            metaDataArr[oldLinIndM[0] * metaData.metaDataSectionLength + 3] += localFpConter[0];
            blockFpConter[0] += localFpConter[0];
            localFpConter[0] = 0;
        }
    };
    if (tile.thread_rank() == 8 && tile.meta_group_rank() == 0) {// this is how it is encoded wheather it is gold or segm block

        if (localFnConter[0] > 0) {
            metaDataArr[oldLinIndM[0] * metaData.metaDataSectionLength + 4] += localFnConter[0];

            blockFnConter[0] += localFnConter[0];
            localFnConter[0] = 0;
        }
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        //executed in case of previous block
        if (isBlockFull[0] && i > 0) {
            //setting data in metadata that block is full
            metaDataArr[oldLinIndM[0] * metaData.metaDataSectionLength + 10 - (oldIsGold[0] * 2)] = true;
        }
        //resetting
        isBlockFull[0] = true;
    };


    //we do it only for non padding pass
    if (tile.thread_rank() < 6 && tile.meta_group_rank() == 1 && !isPaddingPass) {   
        //executed in case of previous block
        if (i>0) {
            if (localBlockMetaDataOld[tile.thread_rank()] < UINT16_MAX) {
                metaDataArr[localBlockMetaDataOld[tile.thread_rank()] * metaData.metaDataSectionLength + 12 - oldIsGold[0]] = isAnythingInPadding[tile.thread_rank()];
            }
        }

        isAnythingInPadding[0] = false;
    };



}






