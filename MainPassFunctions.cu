#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(ForBoolKernelArgs<TXPI> fbArgs, uint32_t iterationNumb[1]) {


    if ((iterationNumb[0] & 1) == 0) {
      return fbArgs.mainArrAPointer;
    }
    else {       
       return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(ForBoolKernelArgs<TXPI> fbArgs, uint32_t iterationNumb[1]) {

    if ((iterationNumb[0] & 1) == 0) {
      return fbArgs.mainArrBPointer;
    }
    else {       
       return fbArgs.mainArrAPointer  ;
    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


inline uint32_t isBitAtCPU(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}






#pragma once
inline __device__ void setNextBlockAsIsToBeActivated(coalesced_group active, char* tensorslice,
    int paddingNumb, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, 
    int xMetaChange, int yMetaChange, int zMetaChange
    ,array3dWithDimsGPU targetArr,bool isAnythingInPadding[6], bool isInRagePred
) {
    //if (isToBeExecutedOnActive(active, paddingNumb)) {
    //    printf("\n setting neighbour of %d %d %d to active- %d %d %d padding numb %d  isAnyInPadding %d\n"
    //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
    //        , localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange
    //        , paddingNumb , isAnythingInPadding[paddingNumb]
    //    );
    //}

    if (isAnythingInPadding[paddingNumb] && isToBeExecutedOnActive(active, paddingNumb) && isInRagePred) {


      //  printf(" \n saving to be actvated  xMeta %d yMeta %d zMeta %d isGold %d \n ", localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange, localWorkQueue[i][3]);


        getTensorRow<bool>(tensorslice, targetArr, targetArr.Ny, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange)[localWorkQueue[i][0] + xMetaChange] = true;
    };

}


#pragma once
inline __device__ void setNextBlocksActivity( char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU targetArr
    , bool isAnythingInPadding[6], coalesced_group active) {
    //0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    //top
    setNextBlockAsIsToBeActivated(active, tensorslice, 0, localWorkQueue, i, 0, 0, -1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]>0);
    //bottom
    setNextBlockAsIsToBeActivated(active, tensorslice, 1, localWorkQueue, i, 0, 0, 1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]<(targetArr.Nz-1));
    //left
    setNextBlockAsIsToBeActivated(active, tensorslice, 2, localWorkQueue, i, -1, 0, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][0]>0);
    //right
    setNextBlockAsIsToBeActivated(active, tensorslice, 3, localWorkQueue, i, 1, 0, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][0] < (targetArr.Nx - 1));
    //anterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 4, localWorkQueue, i, 0, 1, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][1] < (targetArr.Ny - 1));
    //posterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 5, localWorkQueue, i, 0, -1, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][1] > 0);



}

/*
given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
*/
#pragma once
inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
   // return target;
}

///////////////////////////////// new functions


/*
calculate index in main shmem where array that is source for this dilatation round is present
*/
inline __device__ uint16_t getIndexForSourceShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint16_t i){
    return  metaData.mainArrXLength * 
    (1 - (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX))// here calculating offset depending on what iteration and is gold;
        + (mainShmem[startOfLocalWorkQ + i] - (UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX))) * metaData.mainArrSectionLength   ;// offset depending on linear index of metadata block of intrest

}


/*
calculate index in main shmem where array that is source for this dilatation round is present in the neighboutring block ...
*/
inline __device__ uint16_t getIndexForNeighbourForShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint32_t isGold[1], uint16_t currLinIndM[1], uint16_t localBlockMetaData[19],  size_t inMetaIndex) {
       return  metaData.mainArrXLength * 
    ((1 - (isGold[1]) )// here calculating offset depending on what iteration and is gold;
        + (localBlockMetaData[inMetaIndex]) * metaData.mainArrSectionLength )  ;// offset depending on linear index of metadata block of intrest
}


/*
calculating where to put the data from res shmem - so data after dilatation back to global memory
*/
inline __device__ uint16_t getIndexForSaveResShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint32_t isGold[1], uint16_t currLinIndM[1], uint16_t localBlockMetaData[19]) {
    return  metaData.mainArrXLength *
        (1 - (isGold[1]) * 2))// here calculating offset depending on what iteration and is gold;
            + (currLinIndM[0] * metaData.mainArrSectionLength);// offset depending on linear index of this block
}

/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
inline __device__ void dilatateHelperForTransverse(bool predicate,
    uint8_t paddingPos,    uint8_t  normalXChange, uint8_t normalYchange
, uint32_t mainShmem[], bool isAnythingInPadding[6]
,uint8_t forBorderYcoord, uint8_t forBorderXcoord
,uint8_t metaDataCoordIndex, uint16_t targetShmemOffset , uint16_t localBlockMetaData[20]) {
    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[metaDataCoordIndex]< UINT16_MAX) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[threadIdx.x+threadIdx.y*32] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                | mainShmem[targetShmemOffset + forBorderXcoord + forBorderYcoord * 32];

        }
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
        mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] 
        = mainShmem[(threadIdx.x+ normalXChange)+(threadIdx.y+ normalYchange)*32] | mainShmem[begResShmem+threadIdx.x+threadIdx.y*32];
    
    }
   

}


#pragma once
template <typename TXTOI>
inline __device__ void dilatateHelperTopDown( uint8_t paddingPos, 
, uint32_t mainShmem[], bool isAnythingInPadding[6], localBlockMetaData
,uint8_t metaDataCoordIndex
, uint32_t numberbitOfIntrestInBlock // represent a uint32 number that has a bit of intrest in this block set and all others 0 
, uint32_t numberWithCorrBitSetInNeigh// represent a uint32 number that has a bit of intrest in neighbouring block set and all others 0 
, uint16_t targetShmemOffset
) {
       // now we need to load the data from the neigbouring blocks
       //first checking is there anything to look to 
       if (localBlockMetaData[metaDataCoordIndex]< UINT16_MAX) {
           //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
           if (mainShmem[threadIdx.x + threadIdx.y * 32] & numberbitOfIntrestInBlock) {
                              // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
                              isAnythingInPadding[0] = true;
           };
           mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = 
               mainShmem[begResShmem+threadIdx.x+threadIdx.y*32]
                   | (mainShmem[targetShmemOffset+forBorderXcoord+forBorderYcoord*32] & numberWithCorrBitSetInNeigh )

       }   

}
//
//
//
///*
//in pipeline defined to load data for next step and simultaneously process the previous step data  
//used for left,right,anterior,posterior dilatations
//*/
//inline __device__  void loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
//localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
//, metaData,mainArr, //pointers to arrays with data
////now some variables needed to load data  
//    uint8_t metaDataCoordIndexToLoad // where is the index describing linear index of the neighbour in direction of intrest
//    ,uint16_t targetShmemOffset //offset defined in shared memory used to load data into 
//    , shape // shape and alignment of data in load - inludes length of data
////now variables needed for dilatations
//    uint8_t metaDataCoordIndexToProcess // where is the index describing linear index of the neighbour in direction of intrest
//    ,uint16_t sourceShmemOffset //offset defined in shared memory used to process  data from 
//,bool predicate // defining when our thread is a corner case and need to load data from outside of the block
//,uint8_t paddingPos,// needed to know wheather block in given direction should be marked as to be activated
//uint8_t  normalXChange, uint8_t normalYchange
//, uint8_t forBorderYcoord, uint8_t forBorderXcoord
//
//){
//
//krowa rethink weather pipeline.producer_acquire() and commit should not be inside the if statements for border cases
//
//               pipeline.producer_acquire();
//                       if (localBlockMetaData[metaDataCoordIndexToLoad]<UINT16_MAX) {
//                           cooperative_groups::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
//                              (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData,metaDataCoordIndexToLoad )]) 
//                              , shape, pipeline);
//
//                       }
//                     
//               pipeline.producer_commit();
//               //compute 
//                    //if we want to do left riaght, anterior , posterior dilatations
//                  dilatateHelperForTransverse(predicate), paddingPos, normalXChange, normalYchange, mainShmem
//                     , isAnythingInPadding,  iterationNumb,forBorderYcoord, forBorderXcoord,metaDataCoordIndexToProcess,sourceShmemOffset );
//  
//                     
//                     
//              
//}
//
//
/*
constitutes end of pipeline  where we load data for next iteration if such is present
*/
inline __device__  void lastLoad(cta//some needed CUDA objects
worQueueStep, localBlockMetaData, mainArr, mainShmem, i, metaData
){
              if (i + 1<= worQueueStep[0]) {
                  cuda::memcpy_async(cta, (&localBlockMetaData[0]), (&mainArr[(mainShmem[startOfLocalWorkQ+1+i] - UINT16_MAX * (mainShmem[startOfLocalWorkQ+i+1] >= UINT16_MAX)) 
                  * metaData.mainArrSectionLength + metaData.metaDataOffset])
                      , cuda::aligned_size_t<4>(sizeof(uint32_t) * 18), pipeline);
              }
}

/// we need to define here the function that will update the metadata result for the given block - also if it is not padding pass we need to set the neighbouring blocks as to be activated according to the data in shmem


  if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
      blockFpConter[0]+=localFpConter[0]
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0]+=localFnConter[0] ;
        localFnConter[0]=0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
          add info about increase fp or fn count to metadata block and to block variable in thread block






