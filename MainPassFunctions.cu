#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;



/*
gettinng source array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing 
wheather the iteration number is odd or even
*/
template <typename TXPI>
inline __device__ array3dWithDimsGPU getSourceReduced(ForBoolKernelArgs<TXPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {       
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }    
    }


}
/*
gettinng target array for dilatations
*/
template <typename TXPPI>
inline __device__ array3dWithDimsGPU getTargetReduced(ForBoolKernelArgs<TXPPI> fbArgs
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, unsigned int iterationNumb[1]) {


    if ((iterationNumb[0] & 1) != 0) {
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGoldPrev;
        }
        else {
            return fbArgs.reducedSegmPrev;
        }
    }
    else {     
        if (localWorkQueue[i][3] == 1) {
            return fbArgs.reducedGold;
        }
        else {
            return fbArgs.reducedSegm;
        }
    }


}
/*
loading data from appropriate reduce Arr to shared memory 
*/
#pragma once
template <typename TXI>
inline __device__ void loadDataToShmem(ForBoolKernelArgs<TXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU sourceReduced
, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i ) {
      sourceShared[threadIdx.x][ threadIdx.y]  
          = getTensorRow<uint32_t>(tensorslice, sourceReduced, sourceReduced.Ny
              , localWorkQueue[i][1] * fbArgs.dbYLength+ threadIdx.y
              , localWorkQueue[i][2])[localWorkQueue[i][0] *fbArgs.dbXLength+ threadIdx.x];
    //if (sourceShared[threadIdx.x][threadIdx.y] > 0) {
    //    printf("non zero in idX %d idY %d \n ", threadIdx.x, threadIdx.y);
    //}
}

/*
in order to be later able to analyze paddings we will save copy of the currently dilatated array 
(before dilatation) to global memory
*/
//template <typename TPYXI>
//inline __device__ void fromShmemToGlobal(ForBoolKernelArgs<TPYXI> fbArgs, char* tensorslice, uint32_t sourceShared[32][32], array3dWithDimsGPU target
//    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
//) {
//    
//    getTensorRow<uint32_t>(tensorslice, target, target.Ny, yMeta * fbArgs.dbYLength + threadIdx.y, zMeta)[xMeta * fbArgs.dbXLength + threadIdx.x]= sourceShared[threadIdx.x][ threadIdx.y];
//}
//


/*
saving dilatated data to global memory
*/
#pragma once
template <typename TXTI>
inline __device__ void saveToDilatationArr(ForBoolKernelArgs<TXTI> fbArgs, char* tensorslice, uint32_t resShared[32][32], array3dWithDimsGPU resDilatated
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i
) {
    //if (resShared[threadIdx.x][threadIdx.y]>0) {
    //    printf("non zero in saving  in idX %d idY %d zMeta %d \n ", threadIdx.x, threadIdx.y, localWorkQueue[i][2]);

    //}
    //    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny,localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]; 

    getTensorRow<uint32_t>(tensorslice, resDilatated, resDilatated.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2])[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
    = resShared[threadIdx.x][ threadIdx.y];
}




///*
//checking in metadata weather block need to be validated
//*/
//#pragma once
//inline __device__ void isBlockToBeValidatedd(char* tensorslice, bool isBlockToBeValidated[1], array3dWithDimsGPU sourceReduced
//    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta)
//{
//    isBlockToBeValidated[0] = getTensorRow<bool>(tensorslice, sourceReduced, sourceReduced.Ny, yMeta , zMeta)[xMeta ];
//}
//

/*
marking that block is already full*/
#pragma once
inline __device__ void markIsBlockFull(char* tensorslice
    , uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, bool isBlockFull, array3dWithDimsGPU targetMeta, coalesced_group active)
{
    if (isBlockFull && isToBeExecutedOnActive(active, 8)) {
        
      //  printf("set block as full  %d %d %d " , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]);

        getTensorRow<bool>(tensorslice, targetMeta, targetMeta.Ny, localWorkQueue[i][1], localWorkQueue[i][2])[localWorkQueue[i][0]] = true;
    }
}

/*
set the fp or fn counters of metadata
*/
#pragma once
inline __device__ void updateMetaCounters(char* tensorslice
    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta, uint16_t isGold,   array3dWithDimsGPU targetMeta,unsigned int fpOrFnCount,  coalesced_group active)
{
    if ( isToBeExecutedOnActive(active, 9)) {
        getTensorRow<unsigned int>(tensorslice, targetMeta, targetMeta.Ny, yMeta, zMeta)[xMeta] += fpOrFnCount;
    }
}





/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0 
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}


inline uint32_t isBitAtCPU(uint32_t numb, int pos) {
    return (numb & (1 << (pos)));
}






inline __device__ void clearisAnythingInPadding (bool isAnythingInPadding[6]) {

    auto active = coalesced_threads();
    #pragma unroll
    for (int ii; ii < 6; ii++) {
        if (isToBeExecutedOnActive(active, ii)) { isAnythingInPadding[ii] = 0; };
    };
}

/**
loading some data on single threads to shared memory that can be needed by all blocks 
*/

#pragma once
template <typename TYXI>
inline __device__ void loadSmallVars(ForBoolKernelArgs<TYXI> fbArgs, char* tensorslice
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], bool isBlockToBeValidated[1]
    ,uint16_t xMeta, uint16_t yMeta, uint16_t zMeta,uint16_t isGold, coalesced_group active
    , unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    //is to be validates
    if (isToBeExecutedOnActive(active, 0) && isGold == 1) {
       
        //printf("\n isToBeValidatedFp %d count %d counter %d     %d xMeta %d yMeta %d zMeta %d \n  ",
        //    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]
        //    , getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
        //    , xMeta, yMeta, zMeta);


   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.isToBeValidatedFp.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);*/

        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta]);
       // isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFp, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 1) && isGold == 0) {
       
    //    printf("\n isToBeValidated Fn  %d count %d counter %d     xMeta %d yMeta %d zMeta %d   \n  ",
    //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //< getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]
    //, getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
    //, xMeta, yMeta, zMeta);



   /*     getTensorRow<bool>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.isToBeValidatedFn.Ny, yMeta, zMeta)[xMeta]
            = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
                < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
  */      
        isBlockToBeValidated[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta]
            < getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta]);
        //isBlockToBeValidated[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.isToBeValidatedFn, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
    };
    //offsets
    if (isToBeExecutedOnActive(active, 2)) {// && isGold == 1
        resultfpOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpOffset, fbArgs.metaData.fpOffset.Ny, yMeta, zMeta)[xMeta];
      //  printf("\n resultfpOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfpOffset[0], xMeta, yMeta, zMeta);

    };
    if (isToBeExecutedOnActive(active, 3) ) {//&& isGold == 0
       //printf("\n resultfnOffset[0] %d xMeta %d yMeta %d  zMeta %d \n ", resultfnOffset[0], xMeta, yMeta, zMeta);


        resultfnOffset[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnOffset, fbArgs.metaData.fnOffset.Ny, yMeta, zMeta)[xMeta];
    };
    // block counters
    if (isToBeExecutedOnActive(active, 4) && isGold == 1) {
        //auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
        //printf("setting ");

        localFpConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCounter, fbArgs.metaData.fpCounter.Ny, yMeta, zMeta)[xMeta];
    };
    if (isToBeExecutedOnActive(active, 5) && isGold == 0) {
        localFnConter[0] = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCounter, fbArgs.metaData.fnCounter.Ny, yMeta, zMeta)[xMeta];
    };



}



#pragma once
inline __device__ void setNextBlockAsIsToBeActivated(coalesced_group active, char* tensorslice,
    int paddingNumb, uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, 
    int xMetaChange, int yMetaChange, int zMetaChange
    ,array3dWithDimsGPU targetArr,bool isAnythingInPadding[6], bool isInRagePred
) {
    //if (isToBeExecutedOnActive(active, paddingNumb)) {
    //    printf("\n setting neighbour of %d %d %d to active- %d %d %d padding numb %d  isAnyInPadding %d\n"
    //        , localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2]
    //        , localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange
    //        , paddingNumb , isAnythingInPadding[paddingNumb]
    //    );
    //}

    if (isAnythingInPadding[paddingNumb] && isToBeExecutedOnActive(active, paddingNumb) && isInRagePred) {


      //  printf(" \n saving to be actvated  xMeta %d yMeta %d zMeta %d isGold %d \n ", localWorkQueue[i][0] + xMetaChange, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange, localWorkQueue[i][3]);


        getTensorRow<bool>(tensorslice, targetArr, targetArr.Ny, localWorkQueue[i][1] + yMetaChange, localWorkQueue[i][2] + zMetaChange)[localWorkQueue[i][0] + xMetaChange] = true;
    };

}


#pragma once
inline __device__ void setNextBlocksActivity( char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU targetArr
    , bool isAnythingInPadding[6], coalesced_group active) {
    //0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    //top
    setNextBlockAsIsToBeActivated(active, tensorslice, 0, localWorkQueue, i, 0, 0, -1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]>0);
    //bottom
    setNextBlockAsIsToBeActivated(active, tensorslice, 1, localWorkQueue, i, 0, 0, 1, targetArr, isAnythingInPadding
    , localWorkQueue[i][2]<(targetArr.Nz-1));
    //left
    setNextBlockAsIsToBeActivated(active, tensorslice, 2, localWorkQueue, i, -1, 0, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][0]>0);
    //right
    setNextBlockAsIsToBeActivated(active, tensorslice, 3, localWorkQueue, i, 1, 0, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][0] < (targetArr.Nx - 1));
    //anterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 4, localWorkQueue, i, 0, 1, 0, targetArr, isAnythingInPadding
        , localWorkQueue[i][1] < (targetArr.Ny - 1));
    //posterior
    setNextBlockAsIsToBeActivated(active, tensorslice, 5, localWorkQueue, i, 0, -1, 0, targetArr, isAnythingInPadding
    , localWorkQueue[i][1] > 0);



}

/*
given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
*/
#pragma once
inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
   // return target;
}


/*
now we will  additionally get bottom bit of block above and top of block below given they exist
*/
#pragma once
template <typename TXTIO>
inline __device__ void checkBlockToUpAndBottom (ForBoolKernelArgs<TXTIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint16_t i, array3dWithDimsGPU sourceArr, uint32_t resShared[32][32]) {
 
    //looking up
    if (localWorkQueue[i][2] > 0) {//boundary check
     //auto  xx =   getTensorRow<unsigned int>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]

      // printf(" looking up  ");
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] - 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , (fbArgs.dbZLength - 1) //sourceBit
            , resShared//target
            , 0//target bit
        );

    };
    //look down 
    if (localWorkQueue[i][2] < (fbArgs.metaData.MetaZLength - 1)) {//boundary check
        //source
        setBitTo(getTensorRow<uint32_t>(tensorslice, sourceArr, sourceArr.Ny, localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y, localWorkQueue[i][2] + 1)[localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x]
            , 0 //sourceBit
            , resShared//target
            , (fbArgs.dbZLength - 1)//target bit
        );

    };


}


template <typename TXYYOI>
inline __device__ void clearShmemBeforeDilatation(ForBoolKernelArgs<TXYYOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {

    auto activeD = coalesced_threads();
    //resetting
    if (isToBeExecutedOnActive(activeD, 3)) {
        localWorkQueueCounter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 4)) {
        localFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 5)) {
        localFnConter[0] = 0;
    };

}




/*
establish wheather we still need dilatations in both passes
*/
template <typename TXTJIOP>
inline __device__ void checkIsToBeDilatated(ForBoolKernelArgs<TXTJIOP> fbArgs, char* tensorslice, bool isGoldPassToContinue[1], bool isSegmPassToContinue[1]) {
    auto activeE = coalesced_threads();
    if (isToBeExecutedOnActive(activeE, 0)) {
        isGoldPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] * fbArgs.robustnessPercent)
    > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);
   
   //     isGoldPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7] 
   // > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]);

    }
    if (isToBeExecutedOnActive(activeE, 1)) {
       
        //TODO() remove 
     /*   auto xx = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8];
        unsigned int counter = getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11];
        printf("\n  setting is to be dilatated   global fn count %d times robustness %f counter %f is to be accepted %d \n",xx
            , ceilf((float)xx * fbArgs.robustnessPercent), counter,  ( ceilf(xx* 0.95)> counter));
        */


       
        isSegmPassToContinue[0] = (ceilf(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8] * fbArgs.robustnessPercent)
            > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);


    //    isSegmPassToContinue[0] = (getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8]
      //      > getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);

    }

}

/*
update global fp and fn counters and resets shared memory values after dilatations*/
template <typename TXTJIOI>
inline __device__ void updateGlobalCountersAndClear(ForBoolKernelArgs<TXTJIOI> fbArgs, char* tensorslice, unsigned int blockFpConter[1], unsigned int blockFnConter[1]
    , unsigned int localWorkQueueCounter[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
) {
  
    auto activeD = coalesced_threads();
    if (isToBeExecutedOnActive(activeD, 6)) {
        //if (blockFpConter[0]>0) {
        //    printf("\n adding to global fp counter  %d \n", blockFpConter[0]);

        //}
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[10]), (blockFpConter[0]));
       // blockFpConter[0] = 0;
    };
    if (isToBeExecutedOnActive(activeD, 7)) {
        //if (blockFnConter[0]) {
        //    printf("\n adding to global fn counter  %d \n", blockFnConter[0]);
        //}
       // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]), (blockFnConter[0]));
     //   blockFnConter[0] = 0;
    };

    if (isToBeExecutedOnActive(activeD, 8)) {
        // printf("\n  block fn counter %d curr value %d \n", blockFnConter[0], getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[11]);
        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9] = 0;
    };
    //resetting
    //if (isToBeExecutedOnActive(activeD, 3)) {
    //    localWorkQueueCounter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 4)) {
    //    localFpConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};
    //if (isToBeExecutedOnActive(activeD, 5)) {
    //    localFnConter[0] = 0;
    //};

}

///////////////////////////////// new functions


/*
calculate index in main shmem where array that is source for this dilatation round is present
*/
inline __device__ uint16_t getIndexForSourceShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint16_t i){
    return  metaData.mainArrXLength * 
    ((1 - ((mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX))) + (( (iterationNumb[0] & 1)) * 2))// here calculating offset depending on what iteration and is gold;
        + (mainShmem[startOfLocalWorkQ + i] - (UINT16_MAX * (mainShmem[startOfLocalWorkQ + i] >= UINT16_MAX))) * metaData.mainArrSectionLength   ;// offset depending on linear index of metadata block of intrest

}


/*
calculate index in main shmem where array that is source for this dilatation round is present in the neighboutring block ...
*/
inline __device__ uint16_t getIndexForNeighbourForShmem(MetaDataGPU metaData, uint32_t mainShmem[lengthOfMainShmem]
    , uint32_t iterationNumb[1], uint32_t isGold[1], uint32_t currLinIndM[1], uint32_t localBlockMetaData[19],  size_t inMetaIndex) {
       return  metaData.mainArrXLength * 
    ((1 - (isGold[1]) + (( (iterationNumb[0] & 1)) * 2))// here calculating offset depending on what iteration and is gold;
        + (localBlockMetaData[inMetaIndex]) * metaData.mainArrSectionLength   ;// offset depending on linear index of metadata block of intrest
}


/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
inline __device__ void dilatateHelperForTransverse(bool predicate,
    uint8_t paddingPos,    uint8_t  normalXChange, uint8_t normalYchange
, uint32_t mainShmem[], bool isAnythingInPadding[6], pipeline
,uint8_t forBorderYcoord, uint8_t forBorderXcoord
,uint8_t metaDataCoordIndex, uint16_t targetShmemOffset   ) {
   

 pipeline.consumer_wait();

    // so we first check for corner cases 
    if (predicate) {
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[metaDataCoordIndex]< UINT16_MAX) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[threadIdx.x+threadIdx.y*32] > 0) {
                isAnythingInPadding[paddingPos] = true;
            };
            mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = 
                mainShmem[begResShmem+threadIdx.x+threadIdx.y*32]
                    | mainShmem[targetShmemOffset+forBorderXcoord+forBorderYcoord*32]

        }
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
        mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] 
        = mainShmem[(threadIdx.x+ normalXChange)+(threadIdx.y+ normalYchange)*32] | mainShmem[begResShmem+threadIdx.x+threadIdx.y*32];
    
    }
   
              pipeline.consumer_release();

}


#pragma once
template <typename TXTOI>
inline __device__ void dilatateHelperTopDown( uint8_t paddingPos, 
, uint32_t mainShmem[], bool isAnythingInPadding[6], pipeline
,uint8_t metaDataCoordIndex
, uint32_t numberbitOfIntrestInBlock // represent a uint32 number that has a bit of intrest in this block set and all others 0 
, uint32_t numberWithCorrBitSetInNeigh// represent a uint32 number that has a bit of intrest in neighbouring block set and all others 0 
, uint16_t targetShmemOffset
) {
        pipeline.consumer_wait();
        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[metaDataCoordIndex]< UINT16_MAX) {
            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[threadIdx.x + threadIdx.y * 32] & numberbitOfIntrestInBlock) {
                               // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
                               isAnythingInPadding[0] = true;
            };
            mainShmem[begResShmem+threadIdx.x+threadIdx.y*32] = 
                mainShmem[begResShmem+threadIdx.x+threadIdx.y*32]
                    | (mainShmem[targetShmemOffset+forBorderXcoord+forBorderYcoord*32] & numberWithCorrBitSetInNeigh )

        }   
         pipeline.consumer_release();

}



/*
in pipeline defined to load data for next step and simultaneously process the previous step data  
used for left,right,anterior,posterior dilatations
*/
inline __device__  void loadNextAndProcessPreviousSides(pipeline,cta//some needed CUDA objects
localBlockMetaData,mainShmem,iterationNumb,isGold, currLinIndM// shared memory arrays used block wide
, metaData,mainArr, //pointers to arrays with data
//now some variables needed to load data  
    uint8_t metaDataCoordIndexToLoad // where is the index describing linear index of the neighbour in direction of intrest
    ,uint16_t targetShmemOffset //offset defined in shared memory used to load data into 
    , shape // shape and alignment of data in load - inludes length of data
//now variables needed for dilatations
    uint8_t metaDataCoordIndexToProcess // where is the index describing linear index of the neighbour in direction of intrest
    ,uint16_t sourceShmemOffset //offset defined in shared memory used to process  data from 
,bool predicate // defining when our thread is a corner case and need to load data from outside of the block
,uint8_t paddingPos,// needed to know wheather block in given direction should be marked as to be activated
uint8_t  normalXChange, uint8_t normalYchange
, uint8_t forBorderYcoord, uint8_t forBorderXcoord

){
               pipeline.producer_acquire();
                       if (localBlockMetaData[metaDataCoordIndexToLoad]<UINT16_MAX) {
                           cooperative_groups::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                              (&mainArr[getIndexForNeighbourForShmem(metaData, mainShmem, iterationNumb, isGold, currLinIndM, localBlockMetaData,metaDataCoordIndexToLoad )]) 
                              , shape, pipeline);

                       }
                     
               pipeline.producer_commit();
               //compute 
               pipeline.consumer_wait();
                    //if we want to do left riaght, anterior , posterior dilatations
                  dilatateHelperForTransverse(predicate), paddingPos, normalXChange, normalYchange, mainShmem
                     , isAnythingInPadding,  iterationNumb,forBorderYcoord, forBorderXcoord,metaDataCoordIndexToProcess,sourceShmemOffset );
  
                     
                     
              
}












