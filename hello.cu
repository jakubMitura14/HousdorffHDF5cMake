#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>


#include "MetaData.cu"
#include "ExceptionManagUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"

#include <cuda/annotated_ptr>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include <cooperative_groups/memcpy_async.h>
using namespace cooperative_groups;



//#include <torch/extension.h>
//#include <iostream>


//#include "hdf5Manag.cu"
#include <iostream>
#include <string>
#include <vector>
#define H5_BUILT_AS_DYNAMIC_LIB 1
#include <H5Cpp.h>


//#include "xlsxwriter.h
//#include "torch/torch.h"
//#include <torch/extension.h>
//#include <iostream>



/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing
wheather the iteration number is odd or even
*/
#pragma once
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(const ForBoolKernelArgs<TXPI>& fbArgs, const int(&iterationNumb)[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        return fbArgs.mainArrAPointer;

    }
    else {
        return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
#pragma once
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(const ForBoolKernelArgs<TXPPI>& fbArgs, const  int(&iterationNumb)[1]) {

    if ((iterationNumb[0] & 1) == 0) {
        //printf(" BB ");

        return fbArgs.mainArrBPointer;

    }
    else {
        // printf(" AA ");

        return fbArgs.mainArrAPointer;

    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(const uint32_t& x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0
*/
#pragma once
inline __device__ uint32_t isBitAt(const uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}

#pragma once
inline uint32_t isBitAtCPU(const uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}







/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/




template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {



    thread_block cta = cooperative_groups::this_thread_block();

    grid_group grid = cooperative_groups::this_grid();

    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    // __shared__ uint32_t mainShmem[lengthOfMainShmem];
    __shared__ uint32_t mainShmem[lengthOfMainShmem];
    cuda::associate_access_property(&mainShmem, cuda::access_property::shared{});



    constexpr size_t stages_count = 2; // Pipeline stages number

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;

    //cuda::pipeline<cuda::thread_scope_thread>  pipeline = cuda::make_pipeline(cta, &shared_state);
    cuda::pipeline<cuda::thread_scope_block>  pipeline = cuda::make_pipeline(cta, &shared_state);



    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[2];

    __shared__ uint32_t lastI[1];


    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];


    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[40];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX
 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior
    */


    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (threadIdx.x == 9 && threadIdx.y == 0) { iterationNumb[0] = -1; };
    if (threadIdx.x == 11 && threadIdx.y == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (threadIdx.x == 12 && threadIdx.y == 0) {
        isSegmPassToContinue[0] = true;

    };


    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number
    sync(cta);

    do {

        for (uint8_t isPaddingPass = 0; isPaddingPass < 2; isPaddingPass++) {


            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /// dilataions

    //initial cleaning  and initializations include loading min maxes
            if (threadIdx.x == 7 && threadIdx.y == 0 && !isPaddingPass) {
                iterationNumb[0] += 1;
            };

            if (threadIdx.x == 6 && threadIdx.y == 0) {
                localWorkQueueCounter[0] = 0;
            };

            if (threadIdx.x == 1 && threadIdx.y == 0) {
                blockFpConter[0] = 0;
            };
            if (threadIdx.x == 2 && threadIdx.y == 0) {
                blockFnConter[0] = 0;
            };
            if (threadIdx.x == 3 && threadIdx.y == 0) {
                localFpConter[0] = 0;
            };
            if (threadIdx.x == 4 && threadIdx.y == 0) {
                localFnConter[0] = 0;
            };
            if (threadIdx.x == 9 && threadIdx.y == 0) {
                isBlockFull[0] = true;
            };
            if (threadIdx.x == 9 && threadIdx.y == 1) {
                isBlockFull[1] = true;
            };

            if (threadIdx.x == 10 && threadIdx.y == 0) {
                fpFnLocCounter[0] = 0;
            };


            if (threadIdx.x == 10 && threadIdx.y == 2) {// this is how it is encoded wheather it is gold or segm block

                lastI[0] = UINT32_MAX;
            };


            if (threadIdx.x == 0 && threadIdx.y == 0) {
                localTotalLenthOfWorkQueue[0] = fbArgs.minMaxes[9];
                globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
                worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
            };

            if (threadIdx.y == 1) {
                cooperative_groups::memcpy_async(cta, (&localMinMaxes[0]), (&fbArgs.minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
            }

            sync(cta);

            /// load work QueueData into shared memory 
            for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {

                //grid stride loop - sadly most of threads will be idle 
               ///////// loading to work queue
                if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                    for (uint16_t ii = cta.thread_rank(); ii < worQueueStep[0]; ii += cta.size()) {

                        mainShmem[startOfLocalWorkQ + ii] = fbArgs.workQueuePointer[bigloop + ii];
                        isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
                        mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];


                    }

                }
                //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
                ////##### pipeline Step 0

                sync(cta);




                //loading metadata
                pipeline.producer_acquire();
                if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                    cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                        (&fbArgs.metaDataArrPointer[mainShmem[startOfLocalWorkQ] * fbArgs.metaData.metaDataSectionLength])
                        , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                }
                pipeline.producer_commit();


                sync(cta);

                for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {




                    if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {



                        pipeline.producer_acquire();
                        cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
                            mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                            cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength), pipeline);
                        pipeline.producer_commit();

                        //just so pipeline will work well
                        pipeline.consumer_wait();



                        pipeline.consumer_release();
                        sync(cta);

                        ///////// step 1 load top and process main data 
                                        //load top 
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
                                * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process main
                        pipeline.consumer_wait();
                        //marking weather block is already full and no more dilatations are possible 
                        if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) < 32) {
                            isBlockFull[i & 1] = false;
                        }
                        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
                        pipeline.consumer_release();

                        ///////// step 2 load bottom and process top 
                                        //load bottom
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14]
                                * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process top
                        pipeline.consumer_wait();


                        if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
                            if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], 0)) {
                                // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
                                isAnythingInPadding[0] = true;
                            };
                            // if in bit of intrest of neighbour block is set
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32] >> 31) & 1) << 0;
                        }

                        pipeline.consumer_release();
                        sync(cta);

                        /////////// step 3 load right  process bottom  
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 16] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process bototm
                        pipeline.consumer_wait();


                        if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
                            if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], 31)) {
                                isAnythingInPadding[1] = true;
                            };
                            // if in bit of intrest of neighbour block is set
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] >> 0) & 1) << 31;
                        }



                        /*  dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
                              , 0, 31
                              , begSecRegShmem, i);*/

                        pipeline.consumer_release();
                        /////////// step 4 load left process right  
                                        //load left 
                        pipeline.producer_acquire();
                        if (mainShmem[startOfLocalWorkQ + i] > 0) {
                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i] - 1) * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process right
                        pipeline.consumer_wait();

                        if (threadIdx.x == (fbArgs.dbXLength - 1)) {
                            // now we need to load the data from the neigbouring blocks
                            //first checking is there anything to look to 
                            if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
                                //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
                                if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                                    isAnythingInPadding[3] = true;

                                };
                                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                                    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                    | mainShmem[begfirstRegShmem + (threadIdx.y * 32)];

                            };
                        }
                        else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                = mainShmem[begSourceShmem + (threadIdx.x + 1) + (threadIdx.y) * 32]
                                | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                        }

                        pipeline.consumer_release();
                        sync(cta);
                        /////// step 5 load anterior process left 
                                        //load anterior
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {

                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process left 
                        pipeline.consumer_wait();

                        // so we first check for corner cases 
                        if (threadIdx.x == 0) {
                            // now we need to load the data from the neigbouring blocks
                            //first checking is there anything to look to 
                            if (localBlockMetaData[(i & 1) * 20 + 15] < isGoldOffset) {
                                //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
                                if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                                    isAnythingInPadding[2] = true;

                                };
                                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                                    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                    | mainShmem[begSecRegShmem + 31 + threadIdx.y * 32];

                            };
                        }
                        else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                = mainShmem[begSourceShmem + (threadIdx.x - 1) + (threadIdx.y) * 32]
                                | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                        }


                        pipeline.consumer_release();
                        sync(cta);

                        /////// step 6 load posterior process anterior 
                                        //load posterior
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {


                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();

                        //process anterior
                        pipeline.consumer_wait();

                        // so we first check for corner cases 
                        if (threadIdx.y == (fbArgs.dbYLength - 1)) {
                            // now we need to load the data from the neigbouring blocks
                            //first checking is there anything to look to 
                            if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {
                                //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
                                if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                                    isAnythingInPadding[4] = true;

                                };
                                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                                    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                    | mainShmem[begfirstRegShmem + threadIdx.x];

                            };
                        }
                        else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                = mainShmem[begSourceShmem + (threadIdx.x) + (threadIdx.y + 1) * 32]
                                | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                        }


                        pipeline.consumer_release();
                        sync(cta);

                        /////// step 7 
                                       //load reference if needed or data for next iteration if there is such 
                                        //process posterior, save data from res shmem to global memory also we mark weather block is full
                        pipeline.producer_acquire();

                        //if block should be validated we load data for validation
                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                        > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &fbArgs.origArrsPointer[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);

                        }
                        else {//if we are not validating we immidiately start loading data for next loop
                            if (i + 1 < worQueueStep[0]) {
                                cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                                    (&fbArgs.metaDataArrPointer[(mainShmem[startOfLocalWorkQ + 1 + i])
                                        * fbArgs.metaData.metaDataSectionLength])
                                    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


                            }
                        }


                        pipeline.producer_commit();

                        //processPosteriorAndSaveResShmem

                        pipeline.consumer_wait();
                        //dilatate posterior 


                        // so we first check for corner cases 
                        if (threadIdx.y == 0) {
                            // now we need to load the data from the neigbouring blocks
                            //first checking is there anything to look to 
                            if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {
                                //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
                                if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                                    isAnythingInPadding[5] = true;

                                };
                                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                                    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                    | mainShmem[begSecRegShmem + threadIdx.x + (fbArgs.dbYLength - 1) * 32];

                            };
                        }
                        else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block
                            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                                = mainShmem[begSourceShmem + (threadIdx.x) + (threadIdx.y - 1) * 32]
                                | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                        }

                        //now all data should be properly dilatated we save it to global memory
                        //try save target reduced via mempcy async ...


                        //cuda::memcpy_async(cta,
                        //    &getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])]
                        //    , (&mainShmem[begResShmem]),
                        //    cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                        //    , pipeline);



                        getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                            + threadIdx.x + threadIdx.y * 32]
                            = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];





                        pipeline.consumer_release();

                        sync(cta);

                        //////// step 8 basically in order to complete here anyting the count need to be bigger than counter
                                                      // loading for next block if block is not to be validated it was already done earlier
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                            > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                            if (i + 1 < worQueueStep[0]) {


                                cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                                    (&fbArgs.metaDataArrPointer[(mainShmem[startOfLocalWorkQ + 1 + i])
                                        * fbArgs.metaData.metaDataSectionLength])
                                    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                            }
                        }
                        pipeline.producer_commit();




                        sync(cta);

                        //validation - so looking for newly covered voxel for opposite array so new fps or new fns
                        pipeline.consumer_wait();

                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                            > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                                        // now we look through bits and when some is set we call it a result 
#pragma unroll
                            for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                                //if any bit here is set it means it should be added to result list 
                                if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    ) {

                                    //just re
                                    mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = 0;
                                    ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                                    if (isGoldForLocQueue[i]) {
                                        mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3]);
                                        //TODO remove
                                        //atomicAdd_block(&(blockFpConter[0]), 1);

                                    }
                                    else {

                                        mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4]);

                                        //TODO remove
                                        //atomicAdd_block(&(blockFnConter[0]), 1);

                                        //    printf("local fn counter add \n");

                                    };
                                    //   add results to global memory    
                                    //we add one gere jjust to distinguish it from empty result
                                    fbArgs.resultListPointerMeta[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                                    fbArgs.resultListPointerLocal[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                                    fbArgs.resultListPointerIterNumb[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(iterationNumb[0]);




                                }

                            };

                        }
                        /////////
                        pipeline.consumer_release();

                        /// /// cleaning 

                        sync(cta);

                        if (threadIdx.x == 9 && threadIdx.y == 2) {// this is how it is encoded wheather it is gold or segm block

         //executed in case of previous block
                            if (isBlockFull[i & 1] && i >= 0) {
                                //setting data in metadata that block is full
                                fbArgs.metaDataArrPointer[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.metaDataSectionLength + 10 - (isGoldForLocQueue[i] * 2)] = true;
                            }
                            //resetting for some reason  block 0 gets as full even if it should not ...
                            isBlockFull[i & 1] = true;// mainShmem[startOfLocalWorkQ + i]>0;//!isPaddingPass;
                        };




                        //we do it only for non padding pass
                        if (threadIdx.x < 6 && threadIdx.y == 1 && !isPaddingPass) {
                            //executed in case of previous block
                            if (i >= 0) {

                                if (localBlockMetaData[(i & 1) * 20 + 13 + threadIdx.x] < isGoldOffset) {

                                    if (isAnythingInPadding[threadIdx.x]) {
                                        // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
                                        fbArgs.metaDataArrPointer[localBlockMetaData[(i & 1) * 20 + 13 + threadIdx.x] * fbArgs.metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]] = 1;
                                    }

                                }
                            }
                            isAnythingInPadding[threadIdx.x] = false;
                        };






                        if (threadIdx.x == 7 && threadIdx.y == 0) {
                            //this will be executed only if fp or fn counters are bigger than 0 so not during first pass
                            if (localFpConter[0] > 0) {
                                fbArgs.metaDataArrPointer[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.metaDataSectionLength + 3] += localFpConter[0];

                                blockFpConter[0] += localFpConter[0];
                                localFpConter[0] = 0;
                            }


                        };
                        if (threadIdx.x == 8 && threadIdx.y == 0) {

                            if (localFnConter[0] > 0) {
                                fbArgs.metaDataArrPointer[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.metaDataSectionLength + 4] += localFnConter[0];

                                blockFnConter[0] += localFnConter[0];
                                localFnConter[0] = 0;
                            }
                        };

                        sync(cta);

                    }
                }

                //here we are after all of the blocks planned to be processed by this block are

                // just for pipeline to work
                pipeline.consumer_wait();



                pipeline.consumer_release();

            }

            sync(cta);

            //     updating global counters
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                if (blockFpConter[0] > 0) {
                    atomicAdd(&(fbArgs.minMaxes[10]), (blockFpConter[0]));
                }
            };
            if (threadIdx.x == 1 && threadIdx.y == 0) {
                if (blockFnConter[0] > 0) {
                    //if (blockFnConter[0]>10) {
                    //    printf("Fn %d  ", blockFnConter[0]);
                    //}
                    atomicAdd(&(fbArgs.minMaxes[11]), (blockFnConter[0]));
                }
            };
            grid.sync();

            // in first thread block we zero work queue counter
            if (threadIdx.x == 2 && threadIdx.y == 0) {
                if (blockIdx.x == 0) {

                    fbArgs.minMaxes[9] = 0;
                }
            };

            grid.sync();
            /////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/// metadata pass










            // preparation loads
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                fpFnLocCounter[0] = 0;
            }
            if (threadIdx.x == 1 && threadIdx.y == 0) {
                localWorkQueueCounter[0] = 0;
            }
            if (threadIdx.x == 2 && threadIdx.y == 0) {
                localWorkQueueCounter[0] = 0;
            }
            if (threadIdx.x == 3 && threadIdx.y == 0) {
                localWorkQueueCounter[0] = 0;

            }

            if (threadIdx.x == 0 && threadIdx.y == 1) {

                isGoldPassToContinue[0]
                    = ((fbArgs.minMaxes[7] * fbArgs.robustnessPercent) > fbArgs.minMaxes[10]);

            };

            if (threadIdx.x == 0 && threadIdx.y == 1) {

                isSegmPassToContinue[0]
                    = ((fbArgs.minMaxes[8] * fbArgs.robustnessPercent) > fbArgs.minMaxes[11]);
            };


            __syncthreads();

            /////////////////////////////////

            for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x
                ; linIdexMeta <= fbArgs.metaData.totalMetaLength
                ; linIdexMeta += (blockDim.x * blockDim.y * gridDim.x)
                ) {


                if (isPaddingPass == 0) {

                    //goldpass
                    if (isGoldPassToContinue[0] && fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 11]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 7]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 8]) {

                        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);
                        //setting to be activated to 0 
                        fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 11] = 0;
                        //setting active to 1
                        fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 7] = 1;


                    };

                }
                //contrary to number it is when we are not in padding pass
                else {
                    //gold pass
                    if (isGoldPassToContinue[0] && fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 7]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 8]) {

                        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta + (isGoldOffset);

                    };

                }
            }

            __syncthreads();

            if (localWorkQueueCounter[0] > 0) {
                if (threadIdx.x == 0 && threadIdx.y == 0) {
                    globalWorkQueueCounter[0] = atomicAdd(&(fbArgs.minMaxes[9]), (localWorkQueueCounter[0]));


                }
                __syncthreads();
                for (uint32_t linI = threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y) {
                    fbArgs.workQueuePointer[globalWorkQueueCounter[0] + linI] = mainShmem[linI];
                }
                __syncthreads();

            }

            __syncthreads();

            if (threadIdx.x == 0 && threadIdx.y == 0) {

                localWorkQueueCounter[0] = 0;
            }
            __syncthreads();

            for (uint32_t linIdexMeta = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x
                ; linIdexMeta <= fbArgs.metaData.totalMetaLength
                ; linIdexMeta += (blockDim.x * blockDim.y * gridDim.x)
                ) {


                if (isPaddingPass == 0) {

                    //segm pass
                    if ((isSegmPassToContinue[0] && fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 12]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 9]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 10])) {



                        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;

                        //setting to be activated to 0 
                        fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 12] = 0;
                        //setting active to 1
                        fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 9] = 1;

                    }

                }
                //contrary to number it is when we are not in padding pass
                else {
                    //segm pass
                    if ((isSegmPassToContinue[0] && fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 9]
                        && !fbArgs.metaDataArrPointer[linIdexMeta * fbArgs.metaData.metaDataSectionLength + 10])) {



                        mainShmem[atomicAdd_block(&localWorkQueueCounter[0], 1)] = linIdexMeta;
                    }

                }
            }
            __syncthreads();

            if (localWorkQueueCounter[0] > 0) {
                if (threadIdx.x == 0 && threadIdx.y == 0) {
                    globalWorkQueueCounter[0] = atomicAdd(&(fbArgs.minMaxes[9]), (localWorkQueueCounter[0]));


                }
                __syncthreads();
                for (uint32_t linI = threadIdx.y * blockDim.x + threadIdx.x; linI < localWorkQueueCounter[0]; linI += blockDim.x * blockDim.y) {
                    fbArgs.workQueuePointer[globalWorkQueueCounter[0] + linI] = mainShmem[linI];

                }

            }



            grid.sync();
        }



    } while (isGoldPassToContinue[0] || isSegmPassToContinue[0]);


    //setting global iteration number to local one 
    if (blockIdx.x == 0) {
        if (threadIdx.x == 2 && threadIdx.y == 0) {
            fbArgs.metaData.minMaxes[13] = iterationNumb[0];
        }
    }
}



/*
get data from occupancy calculator API used to get optimal number of thread blocks and threads per thread block
*/
template <typename T>
inline occupancyCalcData getOccupancy() {

    occupancyCalcData res;

    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<T>,
        0);
    res.warpsNumbForMinMax = blockSize / 32;
    res.blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    res.warpsNumbForboolPrepareKernel = blockSize / 32;
    res.blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    res.theadsForFirstMetaPass = blockSize;
    res.blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)mainPassKernel<T>,
        0);
    res.warpsNumbForMainPass = blockSize / 32;
    res.blockForMainPass = minGridSize;

    // res.blockForMainPass = 5;
     //res.blockForMainPass = 136;
     //res.warpsNumbForMainPass = 8;

    printf("warpsNumbForMainPass %d blockForMainPass %d  ", res.warpsNumbForMainPass, res.blockForMainPass);
    return res;
}













/*
TODO consider representing as a CUDA graph
executing Algorithm as CUDA graph  based on official documentation and
https://codingbyexample.com/2020/09/25/cuda-graph-usage/
*/
#pragma once
template <typename T>
ForBoolKernelArgs<T> executeHausdoff(ForFullBoolPrepArgs<T>& fFArgs, const int WIDTH, const int HEIGHT, const int DEPTH, occupancyCalcData& occData,
    hipStream_t stream, bool resToSave = false) {

    // For Graph
    //hipStream_t streamForGraph;
    //hipGraph_t graph;
    //std::vector<hipGraphNode_t> nodeDependencies;
    //hipGraphNode_t memcpyNode, kernelNode;
    //hipKernelNodeParams kernelNodeParams = { 0 };
    //  cudaMemcpyParams memcpyParams = { 0 };



    ForBoolKernelArgs<T> fbArgs = getArgsForKernel<T>(fFArgs, occData.warpsNumbForMainPass, occData.blockForMainPass, WIDTH, HEIGHT, DEPTH, stream);

    //checkCuda(hipDeviceSynchronize(), "a1");

    //getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > ( minMaxes);
    getMinMaxes << <occData.blockSizeForMinMax, dim3(32, occData.warpsNumbForMinMax) >> > (fbArgs, fbArgs.minMaxes, fbArgs.goldArr.arrP, fbArgs.segmArr.arrP, fbArgs.metaData);

    //checkCuda(hipDeviceSynchronize(), "a1b");

    fbArgs.metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs, stream);

    //checkCuda(hipDeviceSynchronize(), "a2b");

    boolPrepareKernel << <occData.blockSizeFoboolPrepareKernel, dim3(32, occData.warpsNumbForboolPrepareKernel) >> > (
        fbArgs, fbArgs.metaData, fbArgs.origArrsPointer, fbArgs.metaDataArrPointer, fbArgs.goldArr.arrP, fbArgs.segmArr.arrP, fbArgs.minMaxes);

    //checkCuda(hipDeviceSynchronize(), "a3");

    int fpPlusFn = allocateMemoryAfterBoolKernel(fbArgs, fFArgs, stream);

    //checkCuda(hipDeviceSynchronize(), "a4");


    firstMetaPrepareKernel << <occData.blockForFirstMetaPass, occData.theadsForFirstMetaPass >> > (fbArgs, fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.origArrsPointer, fbArgs.metaDataArrPointer);

    //checkCuda(hipDeviceSynchronize(), "a5");

    void* kernel_args[] = { &fbArgs };
    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), occData.blockForMainPass, dim3(32, occData.warpsNumbForMainPass), kernel_args);

    //checkCuda(hipDeviceSynchronize(), "a6");

    if (resToSave) {
        copyResultstoCPU(fbArgs, fFArgs, stream);

    }
    hipFreeAsync(fbArgs.resultListPointerMeta, stream);
    hipFreeAsync(fbArgs.resultListPointerLocal, stream);
    hipFreeAsync(fbArgs.resultListPointerIterNumb, stream);
    hipFreeAsync(fbArgs.workQueuePointer, stream);
    hipFreeAsync(fbArgs.origArrsPointer, stream);
    hipFreeAsync(fbArgs.metaDataArrPointer, stream);
    hipFreeAsync(fbArgs.mainArrAPointer, stream);
    hipFreeAsync(fbArgs.mainArrBPointer, stream);

    return fbArgs;

}



#pragma once
template <typename T>
ForBoolKernelArgs<T> mainKernelsRun(ForFullBoolPrepArgs<T>& fFArgs, const int WIDTH, const int HEIGHT, const int DEPTH, hipStream_t stream, bool resToSave = false
) {

    //hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;

    occupancyCalcData occData = getOccupancy<T>();

    //pointers ...
    ForBoolKernelArgs<T> fbArgs = executeHausdoff(fFArgs, WIDTH, HEIGHT, DEPTH, occData, resToSave, stream);

    checkCuda(hipDeviceSynchronize(), "last ");

    /////////// error handling 
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    return fbArgs;
}








void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {

    H5::H5File file(FILE_NAME, H5F_ACC_RDONLY);
    H5::DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    H5::DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();

    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    printf("Datasize: %d \n ", dims[0]); // this is the correct number of values

    // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    H5::DataSpace memspace(1, dimsm);
    data = (bool*)calloc(dims[0], sizeof(bool));
    dset.read(data, H5::PredType::NATIVE_HBOOL, memspace, dspace);
    file.close();

}

template<typename T>
T FindMax(T* arr, size_t n)
{
    int max = arr[0];

    for (size_t j = 0; j < n; ++j) {
        if (arr[j] > max) {
            max = arr[j];
        }
    }
    return max;
}






void benchmarkMitura(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH, hipStream_t stream1) {



    bool resultToCopy = true;
    //// some preparations and configuring
    MetaDataCPU metaData;
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = true;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(onlyBladderBoolFlat, WIDTH, HEIGHT, DEPTH);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);

    occupancyCalcData occData = getOccupancy<bool>();

    //pointers ...

    //function invocation
    auto begin = std::chrono::high_resolution_clock::now();
    hipDeviceSynchronize();

    ForBoolKernelArgs<bool> fbArgs = executeHausdoff(forFullBoolPrepArgs, WIDTH, HEIGHT, DEPTH, occData, stream1, resultToCopy);

    // ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, WIDTH, HEIGHT, DEPTH);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    checkCuda(hipDeviceSynchronize(), "a7a");


    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;
    checkCuda(hipDeviceSynchronize(), "a7b");


    size_t sizeMinMax = sizeof(unsigned int) * 20;
    hipMemcpy(minMaxesCPU, fbArgs.metaData.minMaxes, sizeMinMax, hipMemcpyDeviceToHost);
    checkCuda(hipDeviceSynchronize(), "a7c");

    printf("HD: %d \n", minMaxesCPU[13]);
    printf("debug sum : %d \n", minMaxesCPU[15]);


    printf("max iter numb %d  \n", FindMax(forFullBoolPrepArgs.resultListPointerIterNumb, (minMaxesCPU[7] + minMaxesCPU[8] + 50)));


    checkCuda(hipDeviceSynchronize(), "a8");

    if (resultToCopy) {
        free(forFullBoolPrepArgs.resultListPointerMeta);
        free(forFullBoolPrepArgs.resultListPointerLocalCPU);
        free(forFullBoolPrepArgs.resultListPointerIterNumb);
    }

    checkCuda(hipDeviceSynchronize(), "a9");

    // printf("debug sum : %d \n", minMaxesCPU[15]);


     // freeee
    free(onlyBladderBoolFlat);
    free(onlyLungsBoolFlat);


    checkCuda(hipDeviceSynchronize(), "a10");


}







typedef unsigned char uchar;
typedef unsigned int uint;
#pragma once
class Volume {

private:
    bool* volume;
    int width, height, depth;
    int getLinearIndex(int x, int y, int z);
public:
    bool getVoxelValue(int x, int y, int z);
    bool getPixelValue(int x, int y);
    uint getWidth();
    uint getHeight();
    uint getDepth();
    bool* getVolume();
    void setVoxelValue(bool value, int x, int y, int z);
    void setPixelValue(bool value, int x, int y);
    Volume(int width, int height, int depth);
    Volume(int width, int height);
    void dispose();

};




#define CUDA_DEVICE_INDEX 0 //setting the index of your CUDA device

#define IS_3D 1 //setting this to 0 would grant a very slightly improvement on the performance if working with images only
#define CHEBYSHEV 0 //if not set to 1, then this algorithm would use an Euclidean-like metric, it is just an approximation. 
//It can be changed according to the structuring element
#pragma once
class HausdorffDistance {

private:
    void print(hipError_t error, char* msg);

public:
    int computeDistance(Volume* img1, Volume* img2);

};


inline Volume::Volume(const int width, const int height, const int depth) {
    this->width = width; this->height = height; this->depth = depth;
    volume = (bool*)calloc(width * height * depth, sizeof(bool));
}

#pragma once
inline Volume::Volume(const int width, const int height) {
    this->width = width; this->height = height; this->depth = 1;
    volume = (bool*)calloc(width * height * depth, sizeof(bool));
}
#pragma once
inline int Volume::getLinearIndex(const int x, const int y, const int z) {
    const int a = 1, b = width, c = (width) * (height);
    return a * x + b * y + c * z;
}

inline uint Volume::getWidth() { return this->width; }
inline uint Volume::getHeight() { return this->height; }
inline uint Volume::getDepth() { return this->depth; }
inline bool* Volume::getVolume() { return this->volume; }
inline bool Volume::getPixelValue(int x, int y) { return this->volume[getLinearIndex(x, y, 0)]; }
#pragma once
inline bool Volume::getVoxelValue(int x, int y, int z) {
    return volume[getLinearIndex(x, y, z)];
}
#pragma once
inline void Volume::setPixelValue(bool value, const int x, const int y) {
    volume[getLinearIndex(x, y, 0)] = value;
}
#pragma once
inline void Volume::setVoxelValue(bool value, const int x, const int y, const int z) {
    volume[getLinearIndex(x, y, z)] = value;
}
#pragma once
inline void Volume::dispose() {
    free(volume);
}

typedef unsigned char uchar;
typedef unsigned int uint;

#pragma once
__device__ int finished; //global variable that contains a boolean which indicates when to stop the kernel processing
#pragma once
__constant__ __device__ int WIDTH, HEIGHT, DEPTH; //constant variables that contain the size of the volume


#pragma once
__global__ void dilate(const bool* IMG1, const bool* IMG2, const bool* img1Read, const bool* img2Read,
    bool* img1Write, bool* img2Write) {

    const int id = blockDim.x * blockIdx.x + threadIdx.x;
#if !IS_3D
    const int x = id % WIDTH, y = id / WIDTH;
#else
    const int x = id % WIDTH, y = (id / WIDTH) % HEIGHT, z = (id / WIDTH) / HEIGHT;
#endif

    if (id < WIDTH * HEIGHT * DEPTH) {


        if (img1Read[id]) {
            if (x + 1 < WIDTH) img1Write[id + 1] = true;
            if (x - 1 >= 0) img1Write[id - 1] = true;
            if (y + 1 < HEIGHT) img1Write[id + WIDTH] = true;
            if (y - 1 >= 0) img1Write[id - WIDTH] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH) img1Write[id + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0) img1Write[id - WIDTH * HEIGHT] = true;
#endif

#if CHEBYSHEV
            //diagonals
            if (x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1] = true;
            if (x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1] = true;
            if (x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1] = true;
            if (x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 - WIDTH * HEIGHT] = true;
#endif
#endif
        }


        if (img2Read[id]) {
            if (x + 1 < WIDTH) img2Write[id + 1] = true;
            if (x - 1 >= 0) img2Write[id - 1] = true;
            if (y + 1 < HEIGHT) img2Write[id + WIDTH] = true;
            if (y - 1 >= 0) img2Write[id - WIDTH] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH) img2Write[id + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0) img2Write[id - WIDTH * HEIGHT] = true;
#endif

#if CHEBYSHEV
            //diagonals
            if (x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1] = true;
            if (x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1] = true;
            if (x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1] = true;
            if (x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 - WIDTH * HEIGHT] = true;
#endif
#endif
        }


        //this is an atomic and computed to the finished global variable, if image 1 contains all of image 2 and image 2 contains all pixels of
        //image 1 then finished is true
        atomicAnd(&finished, (img2Read[id] || !IMG1[id]) && (img1Read[id] || !IMG2[id]));
    }
}

#pragma once
int HausdorffDistance::computeDistance(Volume* img1, Volume* img2) {

    const int height = (*img1).getHeight(), width = (*img1).getWidth(), depth = (*img1).getDepth();

    size_t size = width * height * depth * sizeof(bool);

    //getting details of your CUDA device
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //device index = 0, you can change it if you have more CUDA devices
    const int threadsPerBlock = props.maxThreadsPerBlock / 2;
    const int blocksPerGrid = (height * width * depth + threadsPerBlock - 1) / threadsPerBlock;


    //copying the dimensions to the GPU
    hipMemcpyToSymbolAsync(HIP_SYMBOL(WIDTH), &width, sizeof(width),0);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(HEIGHT), &height, sizeof(height),0);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(DEPTH), &depth, sizeof(depth),0);


    //allocating the input images on the GPU
    bool* d_img1, * d_img2;
    hipMalloc(&d_img1, size);
    hipMalloc(&d_img2, size);


    //copying the data to the allocated memory on the GPU
    hipMemcpyAsync(d_img1, (*img1).getVolume(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_img2, (*img2).getVolume(), size, hipMemcpyHostToDevice);


    //allocating the images that will be the processing ones
    bool* d_img1Write, * d_img1Read, * d_img2Write, * d_img2Read;
    hipMalloc(&d_img1Write, size); hipMalloc(&d_img1Read, size);
    hipMalloc(&d_img2Write, size); hipMalloc(&d_img2Read, size);


    //cloning the input images to these two image versions (write and read)
    hipMemcpyAsync(d_img1Read, d_img1, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img2Read, d_img2, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img1Write, d_img1, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img2Write, d_img2, size, hipMemcpyDeviceToDevice);



    //required variables to compute the distance
    int h_finished = false, t = true;
    int distance = -1;

    //where the magic happens
    while (!h_finished) {
        //reset the bool variable that verifies if the processing ended
        hipMemcpyToSymbol(HIP_SYMBOL(finished), &t, sizeof(h_finished));


        //lauching the verify kernel, which verifies if the processing finished
        dilate << < blocksPerGrid, threadsPerBlock >> > (d_img1, d_img2, d_img1Read, d_img2Read, d_img1Write, d_img2Write);

        //hipDeviceSynchronize();

        //updating the imgRead (cloning imgWrite to imgRead)
        hipMemcpy(d_img1Read, d_img1Write, size, hipMemcpyDeviceToDevice);
        hipMemcpy(d_img2Read, d_img2Write, size, hipMemcpyDeviceToDevice);

        //copying the result back to host memory
        hipMemcpyFromSymbol(&h_finished, HIP_SYMBOL(finished), sizeof(h_finished));


        //incrementing the distance at each iteration
        distance++;
    }


    //freeing memory
    hipFree(d_img1); hipFree(d_img2);
    hipFree(d_img1Write); hipFree(d_img1Read);
    hipFree(d_img2Write); hipFree(d_img2Read);

    //resetting device
   // hipDeviceReset();

    //print(hipGetLastError(), "processing CUDA. Something may be wrong with your CUDA device.");

    return distance;

}
#pragma once
inline void HausdorffDistance::print(hipError_t error, char* msg) {
    if (error != hipSuccess)
    {
        printf("Error on %s ", msg);
        fprintf(stderr, "Error code: %s!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}



/*
benchmark for original code from  https://github.com/Oyatsumi/HausdorffDistanceComparison
*/
void benchmarkOliviera(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT
    , const int DEPTH) {
    Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    for (int x = 0; x < WIDTH; x++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int z = 0; z < DEPTH; z++) {
                img1.setVoxelValue(onlyLungsBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
                img2.setVoxelValue(onlyBladderBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
            }
        }
    }

    auto begin = std::chrono::high_resolution_clock::now();
    HausdorffDistance* hd = new HausdorffDistance();
    hipDeviceSynchronize();

    int dist = (*hd).computeDistance(&img1, &img2);
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;

    printf("HD: %d \n", dist);

    //freeing memory
    img1.dispose(); img2.dispose();

    //Datasize: 216530944
   //Datasize : 216530944
    //Total elapsed time : 2.62191s
    //HD : 234

}

void loadHDF(hipStream_t stream) {



    const int WIDTH = 512;
    const int HEIGHT = 512;
    //    const int DEPTH = 536;

    int DEPTH = 826;

    const H5std_string FILE_NAMEonlyLungsBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");
    const H5std_string FILE_NAMEonlyBladderBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");

    const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlat");
    //const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlat, DATASET_NAMEonlyLungsBoolFlat, onlyLungsBoolFlat);



    const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlat");
    //const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlat, onlyBladderBoolFlat);
    //onlyBladderBoolFlat = (bool*)calloc(WIDTH* HEIGHT* DEPTH, sizeof(bool));

    //onlyBladderBoolFlat[0] = true;

    //benchmarkOliviera(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);//125 
    //benchmarkMitura(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);//124 or 259
    benchmarkMitura(onlyLungsBoolFlat, onlyBladderBoolFlat, WIDTH, HEIGHT, DEPTH, stream);//124 or 259


    //DEPTH = 536;
    //const H5std_string FILE_NAMEonlyLungsBoolFlatB("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");

    //const H5std_string DATASET_NAMEonlyLungsBoolFlatB("onlyLungsBoolFlatB");
    //// create a vector the same size as the dataset
    //bool* onlyLungsBoolFlatB;
    //loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlatB, DATASET_NAMEonlyLungsBoolFlatB, onlyLungsBoolFlatB);


    //const H5std_string DATASET_NAMEonlyBladderBoolFlatB("onlyBladderBoolFlatB");
    //// create a vector the same size as the dataset
    //bool* onlyBladderBoolFlatB;
    //loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlatB, onlyBladderBoolFlatB);

    ////benchmarkOliviera(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH);//125 
    //benchmarkMitura(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH);//124 or 259

}



void loadHDFB(hipStream_t stream) {



    const int WIDTH = 512;
    const int HEIGHT = 512;
    const int  DEPTH = 536;

    const H5std_string FILE_NAMEonlyLungsBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");
    const H5std_string FILE_NAMEonlyBladderBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");



    const H5std_string FILE_NAMEonlyLungsBoolFlatB("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");

    const H5std_string DATASET_NAMEonlyLungsBoolFlatB("onlyLungsBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlatB;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlatB, DATASET_NAMEonlyLungsBoolFlatB, onlyLungsBoolFlatB);


    const H5std_string DATASET_NAMEonlyBladderBoolFlatB("onlyBladderBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlatB;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlatB, onlyBladderBoolFlatB);

    // benchmarkOliviera(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH);//125 
    benchmarkMitura(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH, stream);//124 or 259

}




void setCPU(bool* arr, int x, int y, int z, int xDim, int yDim) {
    arr[x + y * xDim + z * xDim * yDim] = true;
}

/*
void testAll() {



    //hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;

    const int WIDTH = 512;
    const int HEIGHT = 512;
    //    const int DEPTH = 536;

    int DEPTH = 826;

    bool resultToCopy = true;
    //// some preparations and configuring
    MetaDataCPU metaData;
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    bool* arrA = (bool*)calloc(WIDTH * HEIGHT * DEPTH, sizeof(bool));
    bool* arrB = (bool*)calloc(WIDTH * HEIGHT * DEPTH, sizeof(bool));

    //for (int i = 0; i < 500; i++) {
    //    setCPU(arrB, i, i, 000, WIDTH, HEIGHT);
    //}


    for (int i = 0; i < 500;i++) {
        for (int j = 0; j < 500; j++) {
            setCPU(arrA, i, j, 0, WIDTH, HEIGHT);
        }
    }
    //setCPU(arrA, 2, 2, 0, WIDTH, HEIGHT);

   // setCPU(arrA, 0, 0, 300, WIDTH, HEIGHT);


    //for (int i = 0; i < 500; i++) {
    //    setCPU(arrB, i, i, 600, WIDTH, HEIGHT);
    //}

    //setCPU(arrB, 30, 30, 605, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 36, 606, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 39, 607, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 12, 608, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 33, 609, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 66, 610, WIDTH, HEIGHT);
    setCPU(arrB, 0, 0, 500, WIDTH, HEIGHT);





    setCPU(arrB, 2, 88, 800, WIDTH, HEIGHT);
    setCPU(arrB, 99, 7, 801, WIDTH, HEIGHT);
    setCPU(arrB, 45, 77, 802, WIDTH, HEIGHT);
    //setCPU(arrB, 30, 332, 612, WIDTH, HEIGHT);



    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = true;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(arrA, WIDTH, HEIGHT, DEPTH);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(arrB, WIDTH, HEIGHT, DEPTH);

    ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, WIDTH, HEIGHT, DEPTH, resultToCopy);

    size_t sizeMinMax = sizeof(unsigned int) * 20;
    hipMemcpy(minMaxesCPU, fbArgs.metaData.minMaxes, sizeMinMax, hipMemcpyDeviceToHost);

    printf("HD: %d \n", minMaxesCPU[13]);

    free(arrA);
    free(arrB);

    printf("max iter numb %d  \n", FindMax(forFullBoolPrepArgs.resultListPointerIterNumb, (minMaxesCPU[7] + minMaxesCPU[8] + 50)));



    if (resultToCopy) {
        free(forFullBoolPrepArgs.resultListPointerMeta);
        free(forFullBoolPrepArgs.resultListPointerLocalCPU);
        free(forFullBoolPrepArgs.resultListPointerIterNumb);
    }

    checkCuda(hipDeviceSynchronize(), "last ");

    /////////// error handling
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


}
*/


int main(void) {


    hipStream_t stream1;
    hipStreamCreate(&stream1);

    hipStream_t stream2;
    hipStreamCreate(&stream2);

    for (int i = 0; i < 10; i++) {
        loadHDF(stream1);
    }

    for (int i = 0; i < 10; i++) {
        loadHDFB(stream2);
    }

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    //  testAll();


    return 0;  // successfully terminated
}

