#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

// includes, system
#include <iostream>     // std::cout
#include <algorithm>    // std::min
//#include <hip/hip_runtime_api.h>
#include <cmath>
//#include "Structs.cu"
#include <math.h>
//#include "MemoryTransfers.cu"
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

//#include "BoolKernelTests.cu"
#include "testAll.cu"
#include "CooperativeGroupsUtils.cu"
using namespace cooperative_groups;

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif



using std::cout;
using std::endl;

#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif
using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
using namespace H5;
const H5std_string FILE_NAME("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
const H5std_string DATASET_NAME("onlyLungs");
//const int    NX_SUB = 3;    // hyperslab dimensions
//const int    NY_SUB = 4;
//const int    NX = 7;        // output buffer dimensions
//const int    NY = 7;
//const int    NZ = 3;
//const int    RANK_OUT = 3;



__device__ void computeA(uint32_t* global_out, uint32_t const* shared_in) {
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < 32; linIdexMeta += blockDim.x * gridDim.x) {
        
     //   printf("  ***  ");
       global_out[linIdexMeta] = shared_in[linIdexMeta] +1;   }

};

__device__ void computeB(uint32_t* global_out, uint32_t const* shared_in) {
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < 32; linIdexMeta += blockDim.x * gridDim.x) {

        //   printf("  ***  ");
        global_out[linIdexMeta] = shared_in[linIdexMeta] + 2;
    }

};

__device__ void computeC(uint32_t* global_out, uint32_t const* shared_in) {
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < 32; linIdexMeta += blockDim.x * gridDim.x) {

        //   printf("  ***  ");
        global_out[linIdexMeta] = shared_in[linIdexMeta] + 3;
    }

};


__global__ void with_staging(uint32_t* global_out, uint32_t* global_inA,  uint32_t* global_inB,  uint32_t* global_inC) {
    auto grid = cooperative_groups::this_grid();
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    constexpr size_t stages_count = 2; // Pipeline with two stages
   
                                       
    bool isBlockFull = true;// usefull to establish do we have block completely filled and no more dilatations possible
    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-4468 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t shmem[100];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ unsigned int localBlockMetaData[19];


   cuda::pipeline<cuda::thread_scope_thread> pipeline = cuda::make_pipeline();

    size_t shared_offset[stages_count] = { 0, block.size() }; // Offsets to each

    // Initialize first pipeline stage by submitting a `memcpy_async` to fetch a whole batch for the block 


   // cuda::memcpy_async(block, &shmem[0], &global_in[0], cuda::aligned_size_t <alignof(uint32_t)>(sizeof(uint32_t) * 32), pipeline);
    //pipeline.producer_commit();

    // get first data into pipeline so from global in to first half in shmem
    pipeline.producer_acquire();
    cuda::memcpy_async(block, &shmem[0], &global_inA[0], cuda::aligned_size_t<4>(sizeof(uint32_t) * 32), pipeline);
    pipeline.producer_commit();

    
    // loadIntoShmem(pipeline, block, shmem, global_inA,  0, 0, 32);

    // Pipelined copy/compute:
    for (size_t batch = 1; batch < 3; ++batch) {
        //here we load data for compute step that will be in next loop iteration
        pipeline.producer_acquire();
        cuda::memcpy_async(block, &shmem[(batch & 1) *32], &global_inA[batch*32], cuda::aligned_size_t <alignof(uint32_t)>(sizeof(uint32_t) * 32), pipeline);
        pipeline.producer_commit();

        //so here we wait for previous data load - in case it is fist loop we wait for data that was scheduled before loop started
        pipeline.consumer_wait();
        computeA(&global_out[(batch-1)*32] , &shmem[((batch-1) & 1) * 32]);
        // Collectively release the stage resources
        pipeline.consumer_release();
    }
    // Compute the data fetch by the last iteration

    pipeline.consumer_wait();
    computeA(&global_out[2 * 32] , &shmem[(2 & 1) * 32]);
    pipeline.consumer_release();

    }







int main(void){
    //creating test data for pipeline concept
    uint32_t* globalInGPUA;
    uint32_t* globalInGPUB;
    uint32_t* globalInGPUC;


    uint32_t* globalOutGPU;
    size_t sizeC = (320 * sizeof(uint32_t));
    uint32_t* globalInCPUA = (uint32_t*)calloc(320 , sizeof(uint32_t));
    uint32_t* globalInCPUB = (uint32_t*)calloc(320 , sizeof(uint32_t));
    uint32_t* globalInCPUC = (uint32_t*)calloc(320 , sizeof(uint32_t));

    //populating to ones
    for (int i = 0; i < 96; i++) {
        globalInCPUA[i] = 10;
    };

    //populating to ones
    for (int i = 0; i < 96; i++) {
        globalInCPUB[i] = 100;
    };


    //populating to ones
    for (int i = 0; i < 96; i++) {
        globalInCPUC[i] = 1000;
    };

    uint32_t* globalOUTCPU = (uint32_t*)calloc(320, sizeof(uint32_t));


    //hipMallocAsync(&mainArr, sizeB, 0);
    hipMalloc(&globalInGPUA, sizeC);
    hipMemcpy(globalInGPUA, globalInCPUA, sizeC, hipMemcpyHostToDevice);

    hipMalloc(&globalInGPUB, sizeC);
    hipMemcpy(globalInGPUB, globalInCPUB, sizeC, hipMemcpyHostToDevice);

    hipMalloc(&globalInGPUC, sizeC);
    hipMemcpy(globalInGPUC, globalInCPUC, sizeC, hipMemcpyHostToDevice);


    hipMalloc(&globalOutGPU, sizeC);
    hipMemcpy(globalOutGPU, globalOUTCPU, sizeC, hipMemcpyHostToDevice);

    with_staging << <1,32 >> > (globalOutGPU, globalInGPUA, globalInGPUB, globalInGPUC);


    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
    
    hipMemcpy(globalOUTCPU, globalOutGPU, sizeC, hipMemcpyDeviceToHost);

    for (int i = 0; i < 96; i++) {
       printf("val %d in %d \n", globalOUTCPU[i],i);
    };


    //workqueue








  




   // testMainPasswes();


        ///*
        // * Open the specified file and the specified dataset in the file.
        // */
        //H5File file(FILE_NAME, H5F_ACC_RDONLY);
        //DataSet dataset = file.openDataSet(DATASET_NAME);
        ///*
        // * Get the class of the datatype that is used by the dataset.
        // */
        //H5T_class_t type_class = dataset.getTypeClass();
        //DataSpace dataspace = dataset.getSpace();
        //int rank = dataspace.getSimpleExtentNdims();
        ///*
        // * Get the dimension size of each dimension in the dataspace and
        // * display them.
        // */
        //hsize_t dims_out[3];
        //int ndims = dataspace.getSimpleExtentDims(dims_out, NULL);
        //cout << "rank " << rank << ", dimensions " <<
        //    (unsigned long)(dims_out[0]) << " x " <<
        //    (unsigned long)(dims_out[1]) << 
        //    (unsigned long)(dims_out[2]) << endl;


        ///*
        // * Get class of datatype and print message if it's an integer.
        // */
        //if (type_class == H5T_INTEGER)
        //{
        //    cout << "Data set has INTEGER type" << endl;
        //    /*
        // * Get the integer datatype
        //     */
        //    IntType intype = dataset.getIntType();
        //    /*
        //     * Get order of datatype and print message if it's a little endian.
        //     */
        //    H5std_string order_string;
        //    H5T_order_t order = intype.getOrder(order_string);
        //    cout << order_string << endl;
        //    /*
        //     * Get size of the data element stored in file and print it.
        //     */
        //    size_t size = intype.getSize();
        //    cout << "Data size is " << size << endl;
        //}








        //hsize_t memdim = dims_out[0] * dims_out[1] * dims_out[2];;

        //std::vector<float> data_out(memdim);






        //use the same layout for file and memory
        //dataset.read(data_out.data(), PredType::NATIVE_INT64, dataspace, dataspace);


        ///*
        // * Define hyperslab in the dataset; implicitly giving strike and
        // * block NULL.
        // */
        //hsize_t      offset[2];   // hyperslab offset in the file
        //hsize_t      count[2];    // size of the hyperslab in the file
        //offset[0] = 1;
        //offset[1] = 2;
        //count[0] = NX_SUB;
        //count[1] = NY_SUB;
        //dataspace.selectHyperslab(H5S_SELECT_SET, count, offset);
        ///*
        // * Define the memory dataspace.
        // */
        //hsize_t     dimsm[3];              /* memory space dimensions */
        //dimsm[0] = NX;
        //dimsm[1] = NY;
        //dimsm[2] = NZ;
        //DataSpace memspace(RANK_OUT, dimsm);
        ///*
        // * Define memory hyperslab.
        // */
        //hsize_t      offset_out[3];   // hyperslab offset in memory
        //hsize_t      count_out[3];    // size of the hyperslab in memory
        //offset_out[0] = 3;
        //offset_out[1] = 0;
        //offset_out[2] = 0;
        //count_out[0] = NX_SUB;
        //count_out[1] = NY_SUB;
        //count_out[2] = 1;
        //memspace.selectHyperslab(H5S_SELECT_SET, count_out, offset_out);
        ///*
        // * Read data from hyperslab in the file into the hyperslab in
        // * memory and display the data.
        // */
        //dataset.read(data_out, PredType::NATIVE_INT, memspace, dataspace);
        //for (j = 0; j < NX; j++)
        //{
        //    for (i = 0; i < NY; i++)
        //        cout << data_out[j][i][0] << " ";
        //    cout << endl;
        //}
        /*
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 3 4 5 6 0 0 0
         * 4 5 6 7 0 0 0
         * 5 6 7 8 0 0 0
         * 0 0 0 0 0 0 0
         */
 



    return 0;  // successfully terminated
}