#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

// includes, system
#include <iostream>     // std::cout
#include <algorithm>    // std::min
//#include <hip/hip_runtime_api.h>
#include <cmath>
//#include "Structs.cu"
#include <math.h>
//#include "MemoryTransfers.cu"
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

//#include "BoolKernelTests.cu"
#include "testAll.cu"
#include "CooperativeGroupsUtils.cu"
using namespace cooperative_groups;

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif



using std::cout;
using std::endl;

#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif
using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
using namespace H5;
const H5std_string FILE_NAME("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
const H5std_string DATASET_NAME("onlyLungs");
//const int    NX_SUB = 3;    // hyperslab dimensions
//const int    NY_SUB = 4;
//const int    NX = 7;        // output buffer dimensions
//const int    NY = 7;
//const int    NZ = 3;
//const int    RANK_OUT = 3;





//  pipeline_producer_commit(pipeline, barrier);


__global__ void with_staging(uint32_t* global_out, uint32_t* global_inA, uint32_t* globalOutGPUB, float* globalDummyGPU) {
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(block);

    __shared__ uint32_t shmem[200];
    __shared__ uint32_t currBatch[1];
    __shared__ float dummmy[1];

    cuda::pipeline<cuda::thread_scope_thread> pipeline = cuda::make_pipeline();

    // for simplicity ignored Initializing first pipeline stage of submitting `memcpy_async` 
    //pipeline.producer_acquire();
    //...
    //pipeline.producer_commit();
    
    
    for (size_t batch = 1; batch < 10; ++batch) {
        ///////step 1
        // load
        pipeline.producer_acquire();
            if (tile.meta_group_rank() == 0) {
                cuda::memcpy_async(tile, &shmem[0], &global_inA[batch * 64], cuda::aligned_size_t<64>(sizeof(uint32_t) * 16), pipeline);
                pipeline.producer_commit();
            }
            if (tile.meta_group_rank() == 1) {
                cuda::memcpy_async(tile, &shmem[16], &global_inA[batch * 64 +16], cuda::aligned_size_t<64>(sizeof(uint32_t) * 16), pipeline);
                pipeline.producer_commit();
            }

        //compute data loaded in step 2 of previous iteration
        cuda::pipeline_consumer_wait_prior<0>(pipeline);
         
        //this works correctly
        if (tile.meta_group_rank() == 0) {
            global_out[batch * 64 + 32 + tile.thread_rank()] = shmem[32 + tile.thread_rank()];
        }

        if (tile.thread_rank() == batch && tile.meta_group_rank() == 0) {
            float w = 326;
            for (int j = 0; j < 5000; j++) {
                w += w / j;
            };
            globalDummyGPU[0] += w;
            currBatch[0] = batch;
        };

        pipeline.consumer_release();
        ///// step 2 
        //load
        pipeline.producer_acquire();
        if (tile.meta_group_rank() == 0) {
            cuda::memcpy_async(tile, &shmem[32], &global_inA[(batch +1)* 64+32], cuda::aligned_size_t<64>(sizeof(uint32_t) * 16), pipeline);
            pipeline.producer_commit();
        }
        if (tile.meta_group_rank() == 1) {
            cuda::memcpy_async(tile, &shmem[32 + 16], &global_inA[(batch +1)* 64 +32+ 16], cuda::aligned_size_t<64>(sizeof(uint32_t) * 16), pipeline);
            pipeline.producer_commit();
        }
        //compute data loaded in  step 1
        cuda::pipeline_consumer_wait_prior<0>(pipeline);
        
        //this works correctly
        if (tile.meta_group_rank() == 0) {
           global_out[batch * 64 + tile.thread_rank()] = shmem[tile.thread_rank()];//correct
        }

        if (tile.thread_rank() == (batch+1) && tile.meta_group_rank() == 1) {
            float w = 326;
            for (int j = 0; j < 5000; j++) {
                w += w / j;
            };
            globalOutGPUB[batch]=currBatch[0];
            globalDummyGPU[0] += w;
        };

        pipeline.consumer_release();

    }
    //  for simplicity ignored Computing the data fetch by the last iteration
    //cuda::pipeline_consumer_wait_prior<0>(pipeline);
    ////last computatons .. here omitted
    //pipeline.consumer_release();

    }


/*
results 

val 68 in 1
val 2 in 2
val 196 in 3
val 4 in 4
val 324 in 5
val 6 in 6
val 8 in 7
val 9 in 8
val 580 in 9


*/




int main(void){



    testMainPasswes();
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
////    creating test data for pipeline concept
//    uint32_t* globalInGPUA;
//    int sizeOfArr = 6400;
//    int sizeOfArrB = 20;
//    uint32_t* globalOutGPU;
//    uint32_t* globalOutCPU;
//    float* globalDummyGPU;
//    uint32_t* globalOutGPUB;
//    size_t sizeC = (sizeOfArr * sizeof(uint32_t));
//    size_t sizeD = (sizeOfArrB * sizeof(uint32_t));
//    size_t sizeE = (sizeOfArrB * sizeof(float));
//    uint32_t* globalInCPUA = (uint32_t*)calloc(sizeOfArr, sizeof(uint32_t));
//
//
//    //populating with data 
//    for (int i = 0; i < sizeOfArr; i++) {
//        globalInCPUA[i] = i;
//    };
//
//
//    uint32_t* globalOUTCPU = (uint32_t*)calloc(sizeOfArr, sizeof(uint32_t));
//    uint32_t* globalOUTCPB = (uint32_t*)calloc(sizeOfArrB, sizeof(uint32_t));
//
//
//    //hipMallocAsync(&mainArr, sizeB, 0);
//    hipMalloc(&globalInGPUA, sizeC);
//    hipMemcpy(globalInGPUA, globalInCPUA, sizeC, hipMemcpyHostToDevice);
//
//    hipMalloc(&globalOutGPU, sizeC);
//    hipMemcpy(globalOutGPU, globalOUTCPU, sizeC, hipMemcpyHostToDevice);
//
//
//
//    float* globalDummyCPU = (float*)calloc(sizeOfArrB, sizeof(float));
//    hipMalloc(&globalDummyGPU, sizeE);
//
//    hipMalloc(&globalOutGPUB, sizeD);
//    hipMemcpy(globalOutGPUB, globalOUTCPB, sizeD, hipMemcpyHostToDevice);
//
//    with_staging << <1,64 >> > (globalOutGPU, globalInGPUA, globalOutGPUB, globalDummyGPU);
//
//    //this works correctly
//    hipDeviceSynchronize();    
//
//
//
//
//    hipMemcpy(globalOUTCPU, globalOutGPU, sizeC, hipMemcpyDeviceToHost);
//    for (int i = 130; i < 500; i++) {
//        if (globalOUTCPU[i]!= i) {
//            printf("val %d in %d \n", globalOUTCPU[i], i);
//        }
//    };
//
//    hipMemcpy(globalOUTCPB, globalOutGPUB, sizeD, hipMemcpyDeviceToHost);
//
//    for (int i = 0; i < 10; i++) {
//            printf("val %d in %d \n", globalOUTCPB[i], i);
//      
//    };
//    hipMemcpy(globalDummyCPU, globalDummyGPU, sizeE, hipMemcpyDeviceToHost);
//    printf("duppy %f \n", globalDummyCPU[0]);
//
//
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//

    //workqueue








  




   // testMainPasswes();


        ///*
        // * Open the specified file and the specified dataset in the file.
        // */
        //H5File file(FILE_NAME, H5F_ACC_RDONLY);
        //DataSet dataset = file.openDataSet(DATASET_NAME);
        ///*
        // * Get the class of the datatype that is used by the dataset.
        // */
        //H5T_class_t type_class = dataset.getTypeClass();
        //DataSpace dataspace = dataset.getSpace();
        //int rank = dataspace.getSimpleExtentNdims();
        ///*
        // * Get the dimension size of each dimension in the dataspace and
        // * display them.
        // */
        //hsize_t dims_out[3];
        //int ndims = dataspace.getSimpleExtentDims(dims_out, NULL);
        //cout << "rank " << rank << ", dimensions " <<
        //    (unsigned long)(dims_out[0]) << " x " <<
        //    (unsigned long)(dims_out[1]) << 
        //    (unsigned long)(dims_out[2]) << endl;


        ///*
        // * Get class of datatype and print message if it's an integer.
        // */
        //if (type_class == H5T_INTEGER)
        //{
        //    cout << "Data set has INTEGER type" << endl;
        //    /*
        // * Get the integer datatype
        //     */
        //    IntType intype = dataset.getIntType();
        //    /*
        //     * Get order of datatype and print message if it's a little endian.
        //     */
        //    H5std_string order_string;
        //    H5T_order_t order = intype.getOrder(order_string);
        //    cout << order_string << endl;
        //    /*
        //     * Get size of the data element stored in file and print it.
        //     */
        //    size_t size = intype.getSize();
        //    cout << "Data size is " << size << endl;
        //}








        //hsize_t memdim = dims_out[0] * dims_out[1] * dims_out[2];;

        //std::vector<float> data_out(memdim);






        //use the same layout for file and memory
        //dataset.read(data_out.data(), PredType::NATIVE_INT64, dataspace, dataspace);


        ///*
        // * Define hyperslab in the dataset; implicitly giving strike and
        // * block NULL.
        // */
        //hsize_t      offset[2];   // hyperslab offset in the file
        //hsize_t      count[2];    // size of the hyperslab in the file
        //offset[0] = 1;
        //offset[1] = 2;
        //count[0] = NX_SUB;
        //count[1] = NY_SUB;
        //dataspace.selectHyperslab(H5S_SELECT_SET, count, offset);
        ///*
        // * Define the memory dataspace.
        // */
        //hsize_t     dimsm[3];              /* memory space dimensions */
        //dimsm[0] = NX;
        //dimsm[1] = NY;
        //dimsm[2] = NZ;
        //DataSpace memspace(RANK_OUT, dimsm);
        ///*
        // * Define memory hyperslab.
        // */
        //hsize_t      offset_out[3];   // hyperslab offset in memory
        //hsize_t      count_out[3];    // size of the hyperslab in memory
        //offset_out[0] = 3;
        //offset_out[1] = 0;
        //offset_out[2] = 0;
        //count_out[0] = NX_SUB;
        //count_out[1] = NY_SUB;
        //count_out[2] = 1;
        //memspace.selectHyperslab(H5S_SELECT_SET, count_out, offset_out);
        ///*
        // * Read data from hyperslab in the file into the hyperslab in
        // * memory and display the data.
        // */
        //dataset.read(data_out, PredType::NATIVE_INT, memspace, dataspace);
        //for (j = 0; j < NX; j++)
        //{
        //    for (i = 0; i < NY; i++)
        //        cout << data_out[j][i][0] << " ";
        //    cout << endl;
        //}
        /*
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 3 4 5 6 0 0 0
         * 4 5 6 7 0 0 0
         * 5 6 7 8 0 0 0
         * 0 0 0 0 0 0 0
         */
 



    return 0;  // successfully terminated
}