#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

// includes, system
#include <iostream>     // std::cout
#include <algorithm>    // std::min
//#include <hip/hip_runtime_api.h>
#include <cmath>
//#include "Structs.cu"
#include <math.h>
//#include "MemoryTransfers.cu"
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

//#include "BoolKernelTests.cu"
#include "testAll.cu"
#include "forBench/hdf5Manag.cu"
#include "CooperativeGroupsUtils.cu"
using namespace cooperative_groups;

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>

#include <H5Cpp.h>
using namespace H5;






int main(void){

  //  const int WIDTH = atoi(argv[1]), HEIGHT = WIDTH, DEPTH = 1;
 //   Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    loadHDF();
  


    return 0;  // successfully terminated
}