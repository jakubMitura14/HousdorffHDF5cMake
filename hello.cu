#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include "MainPassFunctions.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "UnitTestUtils.cu"
#include "MetaDataOtherPasses.cu"
#include "DilatationKernels.cu"
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include "BiggerMainFunctions.cu"
#include <cooperative_groups/memcpy_async.h>
#include "testAll.cu"
using namespace cooperative_groups;

using namespace cooperative_groups;

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;
using std::cout;
using std::endl;
#include <string>
#include "forBench/Volume.h"
#include "forBench/HausdorffDistance.cuh"
#include "forBench/HausdorffDistance.cu"





//using std::cout;
//using std::endl;
//#include <string>
//#include "Volume.cuh"
//#include "HausdorffDistance.cuh"
//#include "HausdorffDistance.cu"
//
//#include <iostream>
//#include <string>
//#include <vector>




/*
becouse we need a lot of the additional memory spaces to minimize memory consumption allocations will be postponed after first kernel run enabling
*/
#pragma once
template <typename ZZR>
inline int allocateMemoryAfterBoolKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs,
    uint32_t*& resultListPointerMeta
    , uint32_t*& resultListPointerLocal
    , uint32_t*& resultListPointerIterNumb,
    uint32_t*& origArrsPointer,
    uint32_t*& mainArrAPointer,
    uint32_t*& mainArrBPointer, MetaDataGPU& metaData, array3dWithDimsGPU<ZZR>& goldArr, array3dWithDimsGPU<ZZR>& segmArr) {

    //free no longer needed arrays
    hipFreeAsync(goldArr.arrP, 0);
    hipFreeAsync(segmArr.arrP, 0);

    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, gpuArgs.metaData.minMaxes, size, hipMemcpyDeviceToHost);

    unsigned int fpPlusFn = cpuArgs.metaData.minMaxes[7] + cpuArgs.metaData.minMaxes[8];


    size = sizeof(uint32_t) * (fpPlusFn + 50);


    hipMallocAsync(&resultListPointerLocal, size, 0);
    hipMallocAsync(&resultListPointerIterNumb, size, 0);
    hipMallocAsync(&resultListPointerMeta, size, 0);


    auto xRange = metaData.metaXLength;
    auto yRange = metaData.MetaYLength;
    auto zRange = metaData.MetaZLength;




    size_t sizeB = metaData.totalMetaLength * metaData.mainArrSectionLength * sizeof(uint32_t);

    //printf("size of reduced main arr %d total meta len %d mainArrSectionLen %d  \n", sizeB, metaData.totalMetaLength, metaData.mainArrSectionLength);

    hipMallocAsync(&mainArrAPointer, sizeB, 0);
    hipMemcpyAsync(mainArrAPointer, origArrsPointer, sizeB, hipMemcpyDeviceToDevice, 0);


    hipMallocAsync(&mainArrBPointer, sizeB, 0);
    hipMemcpyAsync(mainArrBPointer, origArrsPointer, sizeB, hipMemcpyDeviceToDevice, 0);

    //just in order set it to 0
    uint32_t* resultListPointerMetaCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    hipMemcpyAsync(resultListPointerMeta, resultListPointerMetaCPU, size, hipMemcpyHostToDevice, 0);
    free(resultListPointerMetaCPU);




    return fpPlusFn;
};




#pragma once
template <typename ZZR>
inline MetaDataGPU allocateMemoryAfterMinMaxesKernel(ForBoolKernelArgs<ZZR>& gpuArgs, ForFullBoolPrepArgs<ZZR>& cpuArgs,
    uint32_t*& workQueue, unsigned int* minMaxes, MetaDataGPU& metaData, uint32_t*& origArr
    , uint32_t*& metaDataArr) {
    ////reduced arrays


    //copy on cpu
    size_t size = sizeof(unsigned int) * 20;
    hipMemcpy(cpuArgs.metaData.minMaxes, minMaxes, size, hipMemcpyDeviceToHost);

    //read an modify
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    //7)global FP count; 8)global FN count
    unsigned int xRange = cpuArgs.metaData.minMaxes[1] - cpuArgs.metaData.minMaxes[2] + 1;
    unsigned int yRange = cpuArgs.metaData.minMaxes[3] - cpuArgs.metaData.minMaxes[4] + 1;
    unsigned int zRange = cpuArgs.metaData.minMaxes[5] - cpuArgs.metaData.minMaxes[6] + 1;
    unsigned int totalMetaLength = (xRange) * (yRange) * (zRange);
    /* printf("in allocateMemoryAfterMinMaxesKernel totalMetaLength  %d  xRange %d yRange %d zRange %d \n"
         , totalMetaLength
         , (xRange)
         , (yRange)
         , (zRange));*/

         //updating size informations
    metaData.metaXLength = xRange;
    metaData.MetaYLength = yRange;
    metaData.MetaZLength = zRange;
    metaData.totalMetaLength = totalMetaLength;

    //cpuArgs.metaData.metaXLength = xRange;
    //cpuArgs.metaData.MetaYLength = yRange;
    //cpuArgs.metaData.MetaZLength = zRange;
    //cpuArgs.metaData.totalMetaLength = totalMetaLength;
    //saving min maxes
    metaData.maxX = cpuArgs.metaData.minMaxes[1];
    metaData.minX = cpuArgs.metaData.minMaxes[2];
    metaData.maxY = cpuArgs.metaData.minMaxes[3];
    metaData.minY = cpuArgs.metaData.minMaxes[4];
    metaData.maxZ = cpuArgs.metaData.minMaxes[5];
    metaData.minZ = cpuArgs.metaData.minMaxes[6];





    //int i = 1;
    //printf("maxX %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);
    //i = 2;
    //printf("minX %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);
    //i = 3;
    //printf("maxY %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);
    //i = 4;
    //printf("minY %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);
    //i = 5;
    //printf("maxZ %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);
    //i = 6;
    //printf("minZ %d  [%d]\n", cpuArgs.metaData.minMaxes[i], i);

  /*  int ii = 7;
    printf("global FP count %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);
    ii = 8;
    printf("global FN count %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);
    ii = 9;
    printf("workQueueCounter %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);
    ii = 10;
    printf("resultFP globalCounter %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);
    ii = 11;
    printf("resultFn globalCounter %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);
    ii = 12;
    printf("global offset counter %d  [%d]\n", cpuArgs.metaData.minMaxes[ii], ii);*/










    //allocating needed memory
    // main array
    unsigned int mainArrXLength = gpuArgs.dbXLength * gpuArgs.dbYLength;
    unsigned int mainArrSectionLength = (mainArrXLength * 2);
    metaData.mainArrXLength = mainArrXLength;
    metaData.mainArrSectionLength = mainArrSectionLength;

    size_t sizeB = totalMetaLength * mainArrSectionLength * sizeof(uint32_t);


    //hipMallocAsync(&mainArr, sizeB, 0);
    size_t sizeorigArr = totalMetaLength * (mainArrXLength * 2) * sizeof(uint32_t);
    hipMallocAsync(&origArr, sizeorigArr, 0);
    size_t sizemetaDataArr = totalMetaLength * (20) * sizeof(uint32_t) + 100;
    hipMallocAsync(&metaDataArr, sizemetaDataArr, 0);


    size_t sizeC = (totalMetaLength * 2 * sizeof(uint32_t) + 50);
    //hipMallocAsync(&workQueue, size, 0);
    hipMallocAsync(&workQueue, sizeC, 0);
    // printf("in allocateMemoryAfterMinMaxesKernel workQueu size  %d isGold constant value %d  \n", totalMetaLength * 2  + 50, isGoldOffset );

    return metaData;
};





/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing
wheather the iteration number is odd or even
*/
#pragma once
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(ForBoolKernelArgs<TXPI>& fbArgs, int(&iterationNumb)[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        return fbArgs.mainArrAPointer;

    }
    else {
        return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
#pragma once
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(ForBoolKernelArgs<TXPPI>& fbArgs, int(&iterationNumb)[1]) {

    if ((iterationNumb[0] & 1) == 0) {
        //printf(" BB ");

        return fbArgs.mainArrBPointer;

    }
    else {
        // printf(" AA ");

        return fbArgs.mainArrAPointer;

    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t& x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}

#pragma once
inline uint32_t isBitAtCPU(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}






//
///*
//given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
//*/
//#pragma once
//inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
//    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
//   // return target;
//}

///////////////////////////////// new functions





/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
template <typename TXPI>
inline __device__ void dilatateHelperForTransverse(ForBoolKernelArgs<TXPI>& fbArgs, const bool predicate,
    const uint8_t  paddingPos, const   int8_t  normalXChange, const  int8_t normalYchange
    , uint32_t(&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6]
    , const uint8_t forBorderYcoord, const  uint8_t forBorderXcoord
    , const uint8_t metaDataCoordIndex, const uint32_t targetShmemOffset, uint32_t(&localBlockMetaData)[40], uint32_t& i
    , bool(&isGoldForLocQueue)[localWorkQueLength]) {



    //if (paddingPos == 3 && mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0 && isGoldForLocQueue[i] == 0 ) {
    //if ( mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0 && isGoldForLocQueue[i] == 1 ) {
    //
    //    printf("something in loaded from right idX %d idY %d  paddingPos %d \n", threadIdx.x, threadIdx.y , paddingPos );
    //}


    // so we first check for corner cases 
    if (predicate) {


        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[(i & 1) * 20 + metaDataCoordIndex] < isGoldOffset) {

            //if (paddingPos == 2 && isGoldForLocQueue[i] == 0) {
            //    printf("b padding begining  in processs left  \n"
            //    );

            //}

            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                isAnythingInPadding[paddingPos] = true;

                //if (paddingPos == 3 && isGoldForLocQueue[i] == 0) {
                //    printf("c padding begining  in processs right  \n"
                //    );

                //}

            };



            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                | mainShmem[targetShmemOffset + forBorderXcoord + forBorderYcoord * 32];

        };
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block


        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
            = mainShmem[begSourceShmem + (threadIdx.x + normalXChange) + (threadIdx.y + normalYchange) * 32]
            | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

    }


}


#pragma once
inline __device__ void dilatateHelperTopDown(const uint8_t paddingPos,
    uint32_t(&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6], uint32_t(&localBlockMetaData)[40]
    , const uint8_t metaDataCoordIndex
    , const  uint8_t sourceBit
    , const uint8_t targetBit
    , const uint32_t targetShmemOffset, uint32_t& i
) {
    // now we need to load the data from the neigbouring blocks
    //first checking is there anything to look to 
    if (localBlockMetaData[(i & 1) * 20 + metaDataCoordIndex] < isGoldOffset) {
        if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], targetBit)) {
            // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
            isAnythingInPadding[paddingPos] = true;



        };
        // if in bit of intrest of neighbour block is set
        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] >> sourceBit) & 1) << targetBit;
        //if (paddingPos==0) {               

        //    //mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] >> sourceBit) & 1) << targetBit;
        //    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] & uint32_t(1)));
        //}
        //else {
        //  
        //   // mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] >> sourceBit) & 1) << targetBit;
        //      mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] & uint32_t(2147483648)));
        //
        //}


        //mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
        //    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
        //    | (mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] & numberWithCorrBitSetInNeigh);

    }

}


//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint32_t localBlockMetaData[(i & 1) * 20+]
//    , uint32_t mainShmem[], uint32_t i, MetaDataGPU metaData
//) {


//
///*
//constitutes end of pipeline  where we load data for next iteration if such is present
//*/
//template <typename TXPPI>
//inline __device__  void lastLoad(ForBoolKernelArgs<TXPPI> fbArgs, thread_block& cta//some needed CUDA objects
//    , unsigned int worQueueStep[1], uint32_t localBlockMetaData[(i & 1) * 20+]
//    , uint32_t mainShmem[], uint32_t i, MetaDataGPU metaData, uint32_t* metaDataArr
//) {
//
//    if (i + 1 <= worQueueStep[0]) {
//        cuda::memcpy_async(cta, (&localBlockMetaData[(i & 1) * 20+0]),
//            (&metaDataArr[(mainShmem[startOfLocalWorkQ + i - isGoldOffset * (mainShmem[startOfLocalWorkQ + i] >= isGoldOffset))
//                * metaData.metaDataSectionLength]])
//            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);
//    }
//
//
//};

/*
we need to define here the function that will update the metadata result for the given block -
also if it is not padding pass we need to set the neighbouring blocks as to be activated according to the data in shmem
this will also include preparations for next round of iterations through blocks from work queue
isInPipeline - marks is it meant to be executed at the begining of the pipeline or after the pipeline
finilizing operations for last block
*/




#pragma once
inline __device__  void afterBlockClean(thread_block& cta
    , unsigned int(&worQueueStep)[1], uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], const uint32_t i, MetaDataGPU& metaData
    , thread_block_tile<32>& tile
    , unsigned int(&localFpConter)[1], unsigned int(&localFnConter)[1]
    , unsigned int(&blockFpConter)[1], unsigned int(&blockFnConter)[1]
    , uint32_t*& metaDataArr
    , bool(&isAnythingInPadding)[6], bool(&isBlockFull)[1], const bool isPaddingPass, bool(&isGoldForLocQueue)[localWorkQueLength], uint32_t(&lastI)[1]
) {



    if (threadIdx.x == 7 && threadIdx.y == 0) {// this is how it is encoded wheather it is gold or segm block
                    //this will be executed only if fp or fn counters are bigger than 0 so not during first pass
        if (localFpConter[0] >= 0) {
            metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 3] += localFpConter[0];
            blockFpConter[0] += localFpConter[0];
            localFpConter[0] = 0;
        }
    };
    if (threadIdx.x == 8 && threadIdx.y == 3) {

        if (localFnConter[0] >= 0) {
            metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 4] += localFnConter[0];

            blockFnConter[0] += localFnConter[0];
            localFnConter[0] = 0;
        }
    };
    if (threadIdx.x == 9 && threadIdx.y == 2) {// this is how it is encoded wheather it is gold or segm block

        //executed in case of previous block
        if (isBlockFull[0] && i > 0) {
            //setting data in metadata that block is full
           // metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 10 - (isGoldForLocQueue[i] * 2)] = true;
        }
        //resetting for some reason  block 0 gets as full even if it should not ...
        isBlockFull[0] = true;// mainShmem[startOfLocalWorkQ + i]>0;//!isPaddingPass;
    };




    //we do it only for non padding pass
    if (threadIdx.x < 6 && threadIdx.y == 1 && !isPaddingPass) {
        //executed in case of previous block
        if (i >= 0) {
            auto metadataTarget = localBlockMetaData[(i & 1) * 20 + 13 + threadIdx.x];

            if (metadataTarget < isGoldOffset) {

                if (isAnythingInPadding[threadIdx.x]) {
                    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,

                    //if (threadIdx.x == 4   ) {
                    //    printf(" padding in end  processs anterior  at the end of linMeta %d  isGold %d \n"
                    //    , metadataTarget
                    //        , isGoldForLocQueue[i]
                    //    );

                    //}


                    //if (threadIdx.x == 5) {
                    //    printf(" padding in end  processs posterior  at the end of linMeta %d  isGold %d \n"
                    //        , metadataTarget
                    //        , isGoldForLocQueue[i]
                    //    );

                    //}



                   // printf( "in setting paddings metadata target %d  full index %d  \n", metadataTarget, metadataTarget * metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]);
 /*                   if (metadataTarget>0 && metadataTarget < metaData.totalMetaLength) {
                        metaDataArr[localBlockMetaData[(i & 1) * 20 + 13 + threadIdx.x] * metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]] = 1;
                    }*/
                    //if (metadataTarget > 0 && metadataTarget < metaData.totalMetaLength) {
                    metaDataArr[metadataTarget * metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]] = 1;
                    //}


                }

            }
        }
        isAnythingInPadding[tile.thread_rank()] = false;
    };
    //if (tile.thread_rank() == 0 && tile.meta_group_rank() == 3) {// this is how it is encoded wheather it is gold or segm block

    //    if (i >= 0) {
    //        lastI[0] = UINT32_MAX;
    //    };
    //}

}





////////////////// with pipeline ofr barrier

/*
initial cleaning  and initializations of dilatation kernel

*/
#pragma once
inline __device__  void dilBlockInitialClean(thread_block_tile<32>& tile,
    const  bool isPaddingPass, int(&iterationNumb)[1],
    unsigned int(&localWorkQueueCounter)[1], unsigned int(&blockFpConter)[1],
    unsigned int(&blockFnConter)[1], unsigned int(&localFpConter)[1],
    unsigned int(&localFnConter)[1], bool(&isBlockFull)[1],
    unsigned int(&fpFnLocCounter)[1],
    unsigned int(&localTotalLenthOfWorkQueue)[1], unsigned int(&globalWorkQueueOffset)[1]
    , unsigned int(&worQueueStep)[1], unsigned int*& minMaxes, unsigned int(&localMinMaxes)[5], uint32_t(&lastI)[1])
{

    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] = true;
    };
    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        lastI[0] = UINT32_MAX;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };
    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
0 : global FP count;
1 : global FN count;
2 : workQueueCounter
3 : resultFP globalCounter
4 : resultFn globalCounter
*/
    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }
}



/*
load work que from global memory
*/
#pragma once
inline __device__  void loadWorkQueue(thread_block& cta, uint32_t(&mainShmem)[lengthOfMainShmem], uint32_t*& workQueue
    , bool(&isGoldForLocQueue)[localWorkQueLength], uint32_t& bigloop, unsigned int(&worQueueStep)[1]) {

    //to do change into barrier

    //cuda::memcpy_async(cta, (&mainShmem[startOfLocalWorkQ]), (&workQueue[bigloop])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * worQueueStep[0]), pipeline);

    for (uint16_t ii = cta.thread_rank(); ii < worQueueStep[0]; ii += cta.size()) {
        mainShmem[startOfLocalWorkQ + ii] = workQueue[bigloop + ii];
        isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
        mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];

    }
}


/*
loads metadata of given block to meta data
*/
#pragma once
inline __device__  void loadMetaDataToShmem(thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, const uint8_t toAdd, uint32_t& ii) {

    //cuda::memcpy_async(cta, (&localBlockMetaData[(ii&1)*20]),
    //    (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd+ii])
    //        * metaData.metaDataSectionLength])
    //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

    cuda::memcpy_async(cta, (&localBlockMetaData[((ii + 1) & 1) * 20]),
        (&metaDataArr[(mainShmem[startOfLocalWorkQ + toAdd + ii])
            * metaData.metaDataSectionLength])
        , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


}





////////////////////MAin
/*
loading data about this block to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    //auto inMainLineMeta = mainShmem[startOfLocalWorkQ + i] ;
    //auto inMainFullIndex = inMainLineMeta * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i]);
    //printf("inMain load full index %d \n ", inMainFullIndex);

    //cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[inMainFullIndex],
    //    cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    //pipeline.producer_commit();


    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block
*/
#pragma once
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isBlockFull)[1]) {

    pipeline.consumer_wait();
    //if ((((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]))  > 0)
//    || mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]==0
//    ) {
   // isBlockFull[0] = false;
    //    }
    //if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32])<32) {
    //
    //    isBlockFull[0] = false;
    //}


    //if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0 && isGoldForLocQueue[i] == 1) {

    //    printf("something in loaded  in main load idX %d idY %d  \n", threadIdx.x, threadIdx.y);
    //}


    //if (getSourceReduced(fbArgs, iterationNumb)[
     //   mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])+ threadIdx.x + threadIdx.y * 32] > 0 && isGoldForLocQueue[i] == 0) {
    //if (isGoldForLocQueue[i] == 1) {
    //    if ( threadIdx.x + threadIdx.y * 32 ==0 ) {
    //        printf("in lin meta  %d looking for non zero  looking for index starting  %d  mainArrSection %d \n"
    //            , mainShmem[startOfLocalWorkQ + i]
    //            ,4* metaData.mainArrSectionLength + threadIdx.x + threadIdx.y * 32
    //        , metaData.mainArrSectionLength
    //        , );
    //    }
    //    //printf("aaain main load idX %d idY vall  %d \n", threadIdx.x, threadIdx.y, fbArgs.mainArrBPointer[4 * metaData.mainArrSectionLength + threadIdx.x + threadIdx.y * 32]);

    //    for (int ii = 0; ii < 6; ii++) {
    //        //if (fbArgs.mainArrBPointer[ii * metaData.mainArrSectionLength + metaData.mainArrXLength + threadIdx.x + threadIdx.y * 32] > 0) {
    //        if (fbArgs.mainArrBPointer[ii * metaData.mainArrSectionLength + threadIdx.x + threadIdx.y * 32] > 0) {

    //            printf("something in traditionally loaded  in main load idX %d idY %d ii %d \n", threadIdx.x, threadIdx.y, ii);
    //        }
    //    }

    //}


    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
    //marking weather block is already full and no more dilatations are possible 


    pipeline.consumer_release();


}

////////////////TOP
/*
loading data about block above to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
#pragma once
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem, i);

    pipeline.consumer_release();

}

/////BOTTOM
#pragma once
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14]
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}
#pragma once
template <typename TXPI>
inline __device__  void processBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
        , 0, 31
        , begSecRegShmem, i);

    pipeline.consumer_release();

}






///////////// right
#pragma once
template <typename TXPI>
inline __device__  void loadRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.x == (fbArgs.dbXLength - 1)),
        3, (1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 0
        , 16, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);

    pipeline.consumer_release();
}



///////////// left
#pragma once
template <typename TXPI>
inline __device__  void loadLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (mainShmem[startOfLocalWorkQ + i] > 0) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.x == 0),
        2, (-1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 31
        , 15, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);

    pipeline.consumer_release();
}

///////////// anterior
#pragma once
template <typename TXPI>
inline __device__  void loadAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {

        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}

#pragma once
template <typename TXPI>
inline __device__  void processAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs, (threadIdx.y == (fbArgs.dbYLength - 1)), 4
        , (0), (1), mainShmem, isAnythingInPadding
        , 0, threadIdx.x
        , 17, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);
    pipeline.consumer_release();
}

///////////// posterior
#pragma once
template <typename TXPI>
inline __device__  void loadPosterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {


        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}





//////////// last load 

/*
load reference if needed or data for next iteration if there is such
*/
#pragma once
template <typename TXPI>
inline __device__  void lastLoad(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , uint32_t*& origArrs, unsigned int(&worQueueStep)[1]) {

    pipeline.producer_acquire();

    //if block should be validated we load data for validation
    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);

    }
    else {//if we are not validating we immidiately start loading data for next loop
        if (i + 1 < worQueueStep[0]) {
            //auto metaDataIndex = mainShmem[startOfLocalWorkQ + 1 + i];
            //printf(" metaDataIndex  to copy metadata for next %d \n", metaDataIndex);
            //cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
            //    (&metaDataArr[metaDataIndex * metaData.metaDataSectionLength])
            //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

            cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1 + i])
                    * metaData.metaDataSectionLength])
                , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


        }
    }


    pipeline.producer_commit();
}
#pragma once
template <typename TXPI>
inline __device__  void processPosteriorAndSaveResShmem(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta
    , uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6],
    bool(&isBlockFull)[1]) {

    pipeline.consumer_wait();
    //dilatate posterior 
    dilatateHelperForTransverse(fbArgs, (threadIdx.y == 0), 5
        , (0), (-1), mainShmem, isAnythingInPadding
        , fbArgs.dbYLength - 1, threadIdx.x // we add offset depending on y dimension
        , 18, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);
    //now all data should be properly dilatated we save it to global memory
    //try save target reduced via mempcy async ...

    getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
        + threadIdx.x + threadIdx.y * 32]
        = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

    //TODO remove 
    //if (blockIdx.x == 0) {
    //    for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
    //        if (threadIdx.x == 0 && threadIdx.y == 0) {

    //            //if any bit here is set it means it should be added to result list 
    //            if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
    //                if (mainShmem[startOfLocalWorkQ + i] * 32 + bitPos>200) {
    //                printf("bit set loc %d isGold %d \n", mainShmem[startOfLocalWorkQ + i] * 32 + bitPos, isGoldForLocQueue[i]);
    //            }
    //             }

    //        }
    //    }
    //}



    //if (!(localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    //> localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)])) {// so count is bigger than counter so we should validate
    //    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = 0;
    //    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 0;
    //}


    pipeline.consumer_release();
}


//////////// validation
#pragma once
template <typename TXPI>
inline __device__  void validate(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , bool(&isBlockFull)[1]
    , unsigned int(&localFpConter)[1], unsigned int(&localFnConter)[1]
    , uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb

) {

    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
        > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
            //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 
            //    ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) 
            //        & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



            //we now look for bits prasent in both reference arrays and current one
           // mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

            // now we look through bits and when some is set we call it a result 
#pragma unroll
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                ) {
                // if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                     //first we add to the resList
                     //TODO consider first passing it into shared memory and then async mempcy ...
                     //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                unsigned int old = 0;
                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (isGoldForLocQueue[i]) {
                    old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3];
                }
                else {
                    old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4];
                    //    printf("local fn counter add \n");

                };
                //   add results to global memory    
                //we add one gere jjust to distinguish it from empty result
                resultListPointerMeta[old] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                resultListPointerLocal[old] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);

                //printf("rrrrresult i %d  meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d  xLoc %d yLoc %d zLoc %d \n"
                //    ,i
                //    ,mainShmem[startOfLocalWorkQ + i]
                //    , isGoldForLocQueue[i]
                //    , old
                //    , localFpConter[0]
                //    , localFnConter[0]
                //    , localBlockMetaData[(i & 1) * 20+ 5]
                //    , localBlockMetaData[(i & 1) * 20+6]
                //    , uint32_t(mainShmem[startOfLocalWorkQ + i] + isGoldOffset * isGoldForLocQueue[i])
                //    , uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x))
                //    , threadIdx.x
                //    , threadIdx.y
                //    , bitPos
                //);


                //printf("\n rrrrresult meta %d isGold %d old %d  xLoc %d yLoc %d zLoc %d iterNumbb %d \n"
                //    , mainShmem[startOfLocalWorkQ + i]
                //    , isGoldForLocQueue[i]
                //    , old
                //    , threadIdx.x
                //    , threadIdx.y
                //    , bitPos
                //    , iterationNumb[0]
                //);


            }

        };
        //mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = 0;
        //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 0;

    }
}










//template <typename TKKI, typename forPipeline >
#pragma once
template <typename TKKI >
inline __device__ void mainDilatation(const bool isPaddingPass, ForBoolKernelArgs<TKKI>& fbArgs, uint32_t*& mainArrAPointer,
    uint32_t*& mainArrBPointer, MetaDataGPU& metaData
    , unsigned int*& minMaxes, uint32_t*& workQueue
    , uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb,
    thread_block& cta, thread_block_tile<32>& tile, grid_group& grid, uint32_t(&mainShmem)[lengthOfMainShmem]
    , bool(&isAnythingInPadding)[6], bool(&isBlockFull)[1], int(&iterationNumb)[1], unsigned int(&globalWorkQueueOffset)[1]
    , unsigned int(&globalWorkQueueCounter)[1]
    , unsigned int(&localWorkQueueCounter)[1], unsigned int(&localTotalLenthOfWorkQueue)[1]
    , unsigned int(&localFpConter)[1]
    , unsigned int(&localFnConter)[1], unsigned int(&blockFpConter)[1]
    , unsigned int(&blockFnConter)[1], unsigned int(&resultfpOffset)[1]
    , unsigned int(&resultfnOffset)[1], unsigned int(&worQueueStep)[1]
    , unsigned int(&localMinMaxes)[5]
    , uint32_t(&localBlockMetaData)[40]
    , unsigned int(&fpFnLocCounter)[1]
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , uint32_t*& origArrs, uint32_t*& metaDataArr, bool(&isGoldForLocQueue)[localWorkQueLength]
    , uint32_t(&lastI)[1]
    , cuda::pipeline<cuda::thread_scope_block>& pipeline
) {


    //initial cleaning  and initializations include loading min maxes
    dilBlockInitialClean(tile, isPaddingPass, iterationNumb, localWorkQueueCounter, blockFpConter,
        blockFnConter, localFpConter, localFnConter, isBlockFull
        , fpFnLocCounter,
        localTotalLenthOfWorkQueue, globalWorkQueueOffset
        , worQueueStep, minMaxes, localMinMaxes, lastI);
    sync(cta);

    /// load work QueueData into shared memory 
    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

            loadWorkQueue(cta, mainShmem, workQueue, isGoldForLocQueue, bigloop, worQueueStep);
        }
        //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
        ////##### pipeline Step 0

        sync(cta);
        ////TODO(remove) krowa
        //if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
        //    if (cta.thread_rank()< worQueueStep[0]) {
        //        printf("work que just after load %d index %d global index %d zerooLoc %d localTotalLenthOfWorkQueue[0] %d really in wor q 0 %d \n"
        //            , mainShmem[startOfLocalWorkQ+ cta.thread_rank()]
        //            , cta.thread_rank()
        //            , bigloop + cta.thread_rank()
        //            , mainShmem[startOfLocalWorkQ]
        //            , localTotalLenthOfWorkQueue[0]
        //            , workQueue[bigloop] - isGoldOffset * isGoldForLocQueue[0] );
        //    }
        //}
        //sync(cta);




        //loading metadata
        pipeline.producer_acquire();
        if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

            auto FirstindexToLoadFromWQ = mainShmem[startOfLocalWorkQ];

            cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                (&metaDataArr[FirstindexToLoadFromWQ * metaData.metaDataSectionLength])
                , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


            //cuda::memcpy_async(cta, (&localBlockMetaData[0]),
            //    (&metaDataArr[(mainShmem[startOfLocalWorkQ])
            //        * metaData.metaDataSectionLength])
            //    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

            //loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 0, 0);

        }
        pipeline.producer_commit();



        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                //if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {                      
                //    printf("\n linMeta beg %d is gold %d is padding pass %d\n ", mainShmem[startOfLocalWorkQ + i], isGoldForLocQueue[i], isPaddingPass);
                //};

                // if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0 && isGoldForLocQueue[i]==0 ) {
                //    printf("\n linMeta beg %d is gold %d is padding pass %d\n ", mainShmem[startOfLocalWorkQ + i], isGoldForLocQueue[i], isPaddingPass);
                //};

//////////////// step 0  load main data and final processing of previous block
               //loading main data for first dilatation
                //IMPORTANT we need to keep a lot of variables constant here like is Anuthing in padding of fp count .. as the represent processing of previous block  - so do not modify them here ...
                loadMain(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb
                );

                pipeline.consumer_wait();
                afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i - 1,
                    metaData, tile, localFpConter, localFnConter
                    , blockFpConter, blockFnConter
                    , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);
                //needed for after block metadata update
                if (tile.thread_rank() == 0 && tile.meta_group_rank() == 3) {
                    lastI[0] = i;
                }

                pipeline.consumer_release();

                ///////// step 1 load top and process main data 
                               //load top 
                loadTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);
                //process main
                processMain(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isBlockFull);
                ///////// step 2 load bottom and process top 
                loadBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                //process top
                processTop(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                /////////// step 3 load right  process bottom  
                loadRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                //process bototm
                processBottom(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                /////////// step 4 load left process right  

                loadLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processRight(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                /////// step 5 load anterior process left 
                loadAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                /////// step 6 load posterior process anterior 
                loadPosterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                processAnterior(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding);
                /////// step 7 
                // 

                            //    sync(cta);

                                //load reference if needed or data for next iteration if there is such 
                                //process posterior, save data from res shmem to global memory also we mark weather block is full
                lastLoad(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, origArrs, worQueueStep);
                processPosteriorAndSaveResShmem(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, isBlockFull);
                sync(cta);

                //////// step 8 basically in order to complete here anyting the count need to be bigger than counter
                              // loading for next block if block is not to be validated it was already done earlier
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
            > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                    if (i + 1 < worQueueStep[0]) {


                        cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1 + i])
                                * metaData.metaDataSectionLength])
                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                    }
                }
                pipeline.producer_commit();


                //validation - so looking for newly covered voxel for opposite array so new fps or new fns
                pipeline.consumer_wait();

                validate(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb, isAnythingInPadding, isBlockFull, localFpConter, localFnConter, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb);
                /////////
                pipeline.consumer_release();

                //  sync(cta);

                  //pipeline.producer_acquire();

                  //pipeline.producer_commit();

                  //pipeline.consumer_wait();

                  //getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                  //    + threadIdx.x + threadIdx.y * 32]
                  //    = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

                  //pipeline.consumer_release();

            }
        }

        //here we are after all of the blocks planned to be processed by this block are

//updating local counters of last local block (normally it is done at the bagining of the next block)
//but we need to check weather any block was processed at all
        pipeline.consumer_wait();

        if (lastI[0] < UINT32_MAX) {
            afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, lastI[0],
                metaData, tile, localFpConter, localFnConter
                , blockFpConter, blockFnConter
                , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);

        }
        pipeline.consumer_release();

    }



    sync(cta);

    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        if (blockFpConter[0] > 0) {
            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
        }
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        if (blockFnConter[0] > 0) {
            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
        }
    };
    // in first thread block we zero work queue counter
    if (threadIdx.x == 2 && threadIdx.y == 0) {
        if (blockIdx.x == 0) {

            minMaxes[9] = 0;
        }
    };


}





/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/





/*
we need to
Data
- shared memory
    -for uploaded data from reduced arrays
    -for dilatation results
    -for result paddings
0) load data about what metadata blocks should be analyzed from work queue
1) load data from given reduced arr into shared memory
2) perform bit  dilatations in 6 directions
    and save to result to result shared memory - additionally dilatations into its own shared memory
3) given the block is to be validated (in case it is first main pass - all needs to be) we check  if
    - if there is set bit (voxel) in res shmem but not in source shmem
        - we establish is there anything of intrest in the primary given array of other type (so for gold we check segm and for segm gold - but original ones)
        - if so we add this to the result list in a spot we established from offsets of metadata
            - we set metadata's fp and fn result counters - so later we will be able to establish wheather block should be validated at all
            - we also increment local counters of fp and fn - those will be used for later
4) we save data from result shmem into reduced arrays and from paddings into padding store (both in global memory)

*/






template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {

    //inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs, uint32_t * mainArr, MetaDataGPU metaData
    //    , unsigned int* minMaxes, uint32_t * workQueue
    //    , uint32_t * resultListPointerMeta, uint32_t * resultListPointerLocal, uint32_t * resultListPointerIterNumb, uint32_t * origArrs, uint32_t * metaDataArr) {

    //if (threadIdx.x == 0 && threadIdx.y == 0) {
    //    printf("in metadataPass totalMetaLength  %d   \n", fbArgs.metaData.totalMetaLength);

    //};

    thread_block cta = cooperative_groups::this_thread_block();

    thread_block_tile<32> tile = tiled_partition<32>(cta);
    grid_group grid = cooperative_groups::this_grid();

    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t mainShmem[lengthOfMainShmem];



    constexpr size_t stages_count = 2; // Pipeline stages number

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;

    //cuda::pipeline<cuda::thread_scope_thread>  pipeline = cuda::make_pipeline(cta, &shared_state);
    cuda::pipeline<cuda::thread_scope_block>  pipeline = cuda::make_pipeline(cta, &shared_state);



    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[1];

    __shared__ uint32_t lastI[1];


    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];


    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[40];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX

 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior

    */


    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) { iterationNumb[0] = -1; };
    if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        isSegmPassToContinue[0] = true;

        if (blockIdx.x == 0) {
            printf("maxX % d minX % d maxY % d  minY % d maxZ % d minZ % d global FP count % d global FN count % d  total meta len %d \n"
                , fbArgs.minMaxes[1]
                , fbArgs.minMaxes[2]
                , fbArgs.minMaxes[3]
                , fbArgs.minMaxes[4]
                , fbArgs.minMaxes[5]
                , fbArgs.minMaxes[6]
                , fbArgs.minMaxes[7]
                , fbArgs.minMaxes[8]
                , fbArgs.metaData.totalMetaLength

            );
        }

    };


    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number



   // for (int t = 0; t < 3; t++) {
    do {
        //if (threadIdx.x == 2 && threadIdx.y == 0) {
        //    if (blockIdx.x == 0) {
        //        printf("************  iter nuumb %d \n", iterationNumb[0]);
        //        //  fbArgs.metaData.minMaxes[13] = iterationNumb[0];
        //    }
        //};

        mainDilatation(false, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
            , fbArgs.workQueuePointer
            , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
            , cta, tile, grid, mainShmem
            , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset
            , globalWorkQueueCounter
            , localWorkQueueCounter
            , localTotalLenthOfWorkQueue
            , localFpConter
            , localFnConter, blockFpConter
            , blockFnConter
            , resultfpOffset
            , resultfnOffset, worQueueStep, localMinMaxes
            , localBlockMetaData, fpFnLocCounter
            , isGoldPassToContinue, isSegmPassToContinue
            , fbArgs.origArrsPointer
            , fbArgs.metaDataArrPointer, isGoldForLocQueue
            , lastI, pipeline

        );

        grid.sync();
        /*  if (blockIdx.x == 0) {
              if (threadIdx.x == 2 && threadIdx.y == 0) {
                  printf("b iter nuumb %d \n", iterationNumb[0]);
              }
          }*/
          ///////////// loading work queue for padding dilatations
        metadataPass(fbArgs, true, 11, 7, 8,
            12, 9, 10
            , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
            , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
            , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
            , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);




        //////////// padding dilatations
        grid.sync();
        //if (blockIdx.x == 0) {
        //    if (threadIdx.x == 2 && threadIdx.y == 0) {
        //        printf("c iter nuumb %d \n", iterationNumb[0]);
        //    }
        //}
        mainDilatation(true, fbArgs, fbArgs.mainArrAPointer, fbArgs.mainArrBPointer, fbArgs.metaData, fbArgs.minMaxes
            , fbArgs.workQueuePointer
            , fbArgs.resultListPointerMeta, fbArgs.resultListPointerLocal, fbArgs.resultListPointerIterNumb
            , cta, tile, grid, mainShmem
            , isAnythingInPadding, isBlockFull, iterationNumb, globalWorkQueueOffset
            , globalWorkQueueCounter
            , localWorkQueueCounter
            , localTotalLenthOfWorkQueue
            , localFpConter
            , localFnConter, blockFpConter
            , blockFnConter
            , resultfpOffset
            , resultfnOffset, worQueueStep, localMinMaxes
            , localBlockMetaData, fpFnLocCounter
            , isGoldPassToContinue, isSegmPassToContinue
            , fbArgs.origArrsPointer
            , fbArgs.metaDataArrPointer, isGoldForLocQueue
            , lastI, pipeline

        );


        grid.sync();
        /*  if (blockIdx.x == 0) {
              if (threadIdx.x == 2 && threadIdx.y == 0) {
                  printf("d iter nuumb %d \n", iterationNumb[0]);
              }
          }*/
          ////////////////////////main metadata pass
        metadataPass(fbArgs, false, 7, 8, 8,
            9, 10, 8
            , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
            , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
            , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
            , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);
        grid.sync();

    } while (isGoldPassToContinue[0] || isSegmPassToContinue[0]);
    //}
    //grid.sync();

    ////for final result
    //if (threadIdx.x == 2 && threadIdx.y == 0) {
    //    if (blockIdx.x == 0) {

    //      //  fbArgs.metaData.minMaxes[13] = iterationNumb[0];
    //    }
    //};


    //grid.sync();


    //if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
    //    printf("  isGoldPassToContinue %d isSegmPassToContinue %d \n ", isGoldPassToContinue[0], isSegmPassToContinue[0]);
    //};

//  }// end while

  //setting global iteration number to local one 
    if (blockIdx.x == 0) {
        if (threadIdx.x == 2 && threadIdx.y == 0) {
            fbArgs.metaData.minMaxes[13] = iterationNumb[0];
        }
    }
}





#pragma once
template <typename T>
ForBoolKernelArgs<T> mainKernelsRun(ForFullBoolPrepArgs<T> fFArgs, uint32_t*& reducedResCPU
    , uint32_t*& resultListPointerMetaCPU
    , uint32_t*& resultListPointerLocalCPU
    , uint32_t*& resultListPointerIterNumbCPU
    , uint32_t*& metaDataArrPointerCPU
    , uint32_t*& workQueuePointerCPU
    , uint32_t*& origArrsCPU
    , const int WIDTH, const int HEIGHT, const int DEPTH
) {

    //hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;
    int device = 0;
    unsigned int cpuIterNumb = -1;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);
    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<T>,
        0);
    int warpsNumbForMinMax = blockSize / 32;
    int blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    int warpsNumbForboolPrepareKernel = blockSize / 32;
    int blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    int theadsForFirstMetaPass = blockSize;
    int blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)mainPassKernel<T>,
        0);
    int warpsNumbForMainPass = blockSize / 32;
    int blockForMainPass = minGridSize;
    printf("warpsNumbForMainPass %d blockForMainPass %d  ", warpsNumbForMainPass, blockForMainPass);


    // warpsNumbForMainPass = 5;
    //blockForMainPass = 1;
    blockSizeForMinMax = 1;





    //pointers ...
    uint32_t* resultListPointerMeta;
    uint32_t* resultListPointerLocal;
    uint32_t* resultListPointerIterNumb;

    uint32_t* origArrsPointer;
    uint32_t* mainArrAPointer;
    uint32_t* mainArrBPointer;
    uint32_t* metaDataArrPointer;

    uint32_t* workQueuePointer;



    //main arrays allocations
    T* goldArrPointer;
    T* segmArrPointer;
    //size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);
    size_t sizeMainArr = (sizeof(T) * WIDTH * HEIGHT * DEPTH);

    hipMallocAsync(&goldArrPointer, sizeMainArr, 0);
    hipMallocAsync(&segmArrPointer, sizeMainArr, 0);

    hipMemcpyAsync(goldArrPointer, fFArgs.goldArr.arrP, sizeMainArr, hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(segmArrPointer, fFArgs.segmArr.arrP, sizeMainArr, hipMemcpyHostToDevice, 0);


    array3dWithDimsGPU<T> goldArr;
    array3dWithDimsGPU<T> segmArr;

    goldArr.arrP = goldArrPointer;
    goldArr.Nx = WIDTH;
    goldArr.Ny = HEIGHT;
    goldArr.Nz = DEPTH;



    segmArr.arrP = segmArrPointer;
    segmArr.Nx = WIDTH;
    segmArr.Ny = HEIGHT;
    segmArr.Nz = DEPTH;
    checkCuda(hipDeviceSynchronize(), "a0a");

    unsigned int* minMaxes;
    size_t sizeminMaxes = sizeof(unsigned int) * 20;
    hipMallocAsync(&minMaxes, sizeminMaxes, 0);




    checkCuda(hipDeviceSynchronize(), "a0b");
    ForBoolKernelArgs<T> fbArgs = getArgsForKernel<T>(fFArgs, goldArrPointer, segmArrPointer, minMaxes, warpsNumbForMainPass, blockForMainPass, WIDTH, HEIGHT, DEPTH);
    fbArgs.metaData.minMaxes = minMaxes;
    fbArgs.minMaxes = minMaxes;


    fbArgs.goldArr = goldArr;
    fbArgs.segmArr = segmArr;


    ////preparation kernel

    // initialize, then launch

    checkCuda(hipDeviceSynchronize(), "a1");


    //getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > ( minMaxes);
    getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > (fbArgs, minMaxes, goldArrPointer, segmArrPointer, fbArgs.metaData);

    checkCuda(hipDeviceSynchronize(), "a1b");


    checkCuda(hipDeviceSynchronize(), "a2a");

    fbArgs.metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs, workQueuePointer, minMaxes, fbArgs.metaData, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a2b");

    boolPrepareKernel << <blockSizeFoboolPrepareKernel, dim3(32, warpsNumbForboolPrepareKernel) >> > (fbArgs, fbArgs.metaData, origArrsPointer, metaDataArrPointer, goldArrPointer, segmArrPointer, minMaxes);
    //  //uint32_t* origArrs, uint32_t* metaDataArr     metaDataArr[linIdexMeta * metaData.metaDataSectionLength     metaDataOffset

    checkCuda(hipDeviceSynchronize(), "a3");



    int fpPlusFn = allocateMemoryAfterBoolKernel(fbArgs, fFArgs, resultListPointerMeta, resultListPointerLocal, resultListPointerIterNumb, origArrsPointer, mainArrAPointer, mainArrBPointer, fbArgs.metaData, goldArr, segmArr);




    checkCuda(hipDeviceSynchronize(), "a4");

    //hipFreeAsync(goldArrPointer, 0);
    //hipFreeAsync(segmArrPointer, 0);

    firstMetaPrepareKernel << <blockForFirstMetaPass, theadsForFirstMetaPass >> > (fbArgs, fbArgs.metaData, minMaxes, workQueuePointer, origArrsPointer, metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a5");
    //void* kernel_args[] = { &fbArgs, mainArrPointer,&metaData,minMaxes, workQueuePointer,resultListPointerMeta,resultListPointerLocal, resultListPointerIterNumb };



    //fbArgs.goldArr = goldArr;
    //fbArgs.segmArr = segmArr;
    //fbArgs.metaData = metaData;

    fbArgs.resultListPointerMeta = resultListPointerMeta;
    fbArgs.resultListPointerLocal = resultListPointerLocal;
    fbArgs.resultListPointerIterNumb = resultListPointerIterNumb;

    fbArgs.origArrsPointer = origArrsPointer;
    fbArgs.mainArrAPointer = mainArrAPointer;
    fbArgs.mainArrBPointer = mainArrBPointer;


    fbArgs.metaDataArrPointer = metaDataArrPointer;
    fbArgs.workQueuePointer = workQueuePointer;
    fbArgs.minMaxes = minMaxes;
    void* kernel_args[] = { &fbArgs };


    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), blockForMainPass, dim3(32, warpsNumbForMainPass), kernel_args);



    checkCuda(hipDeviceSynchronize(), "a6");


    auto metaData = fbArgs.metaData;
    size_t sizeMinnMax = sizeof(unsigned int) * 20;

    hipMemcpy(fFArgs.metaData.minMaxes, minMaxes, sizeMinnMax, hipMemcpyDeviceToHost);

    //copy to CPU
    size_t sizeCPU = metaData.totalMetaLength * fbArgs.metaData.mainArrSectionLength * sizeof(uint32_t);
    reducedResCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(reducedResCPU, mainArrAPointer, sizeCPU, hipMemcpyDeviceToHost);

    origArrsCPU = (uint32_t*)calloc(metaData.totalMetaLength * metaData.mainArrSectionLength, sizeof(uint32_t));
    hipMemcpy(origArrsCPU, origArrsPointer, sizeCPU, hipMemcpyDeviceToHost);


    size_t sizeRes = sizeof(uint32_t) * (fpPlusFn + 50);


    resultListPointerMetaCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    resultListPointerLocalCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    resultListPointerIterNumbCPU = (uint32_t*)calloc(fpPlusFn + 50, sizeof(uint32_t));
    hipMemcpy(resultListPointerMetaCPU, resultListPointerMeta, sizeRes, hipMemcpyDeviceToHost);

    hipMemcpy(resultListPointerLocalCPU, resultListPointerLocal, sizeRes, hipMemcpyDeviceToHost);

    hipMemcpy(resultListPointerIterNumbCPU, resultListPointerIterNumb, sizeRes, hipMemcpyDeviceToHost);

    size_t sizemetaDataArr = metaData.totalMetaLength * (20) * sizeof(uint32_t);
    metaDataArrPointerCPU = (uint32_t*)calloc(metaData.totalMetaLength * (20), sizeof(uint32_t));
    hipMemcpy(metaDataArrPointerCPU, metaDataArrPointer, sizemetaDataArr, hipMemcpyDeviceToHost);

    size_t sizeC = (metaData.totalMetaLength * sizeof(uint32_t));

    workQueuePointerCPU = (uint32_t*)calloc(metaData.totalMetaLength, sizeof(uint32_t));
    hipMemcpy(workQueuePointerCPU, workQueuePointer, sizeC, hipMemcpyDeviceToHost);



    checkCuda(hipDeviceSynchronize(), "a7");






    //  //hipLaunchCooperativeKernel((void*)mainPassKernel<int>, deviceProp.multiProcessorCount, fFArgs.threadsMainPass, fbArgs);




    //  ////copyDeviceToHost3d(goldArr, fFArgs.goldArr);
    //  ////copyDeviceToHost3d(segmArr, fFArgs.segmArr);
    //  //// getting arrays allocated on  cpu to 


    //  //copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);

    //  //// printForDebug(fbArgs, fFArgs, resultListPointer, mainArrPointer, workQueuePointer, metaData);


    //  checkCuda(hipDeviceSynchronize(), "just after copy device to host");
    //  //hipGetLastError();

    //hipFreeAsync(goldArrPointer, 0);
    //hipFreeAsync(segmArrPointer, 0);


    hipFreeAsync(resultListPointerMeta, 0);
    hipFreeAsync(resultListPointerLocal, 0);
    hipFreeAsync(resultListPointerIterNumb, 0);
    hipFreeAsync(workQueuePointer, 0);
    hipFreeAsync(origArrsPointer, 0);
    hipFreeAsync(metaDataArrPointer, 0);
    hipFreeAsync(mainArrAPointer, 0);
    hipFreeAsync(mainArrBPointer, 0);



    checkCuda(hipDeviceSynchronize(), "last ");

    /////////// error handling 
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    ForBoolKernelArgs<T> res;
    return res;
    // return fbArgs;
}








inline void setArrCPUB(bool* arrCPU, int x, int y, int z, int  Nx, int Ny) {

    arrCPU[x + y * Nx + z * Nx * Ny] = true;
};



//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
    // threads and blocks for bool kernel
    const int blocks = 17;
    const int xThreadDim = 32;
    const int yThreadDim = 12;
    const dim3 threads = dim3(xThreadDim, yThreadDim);
    // threads and blocks for first metadata pass
    int threadsFirstMetaDataPass = 32;
    int blocksFirstMetaDataPass = 10;



    //datablock dimensions
    int dbXLength = xThreadDim;
    int dbYLength = 5;
    int dbZLength = 32;



    //threads and blocks for main pass 
    dim3 threadsMainPass = dim3(dbXLength, dbYLength);
    int blocksMainPass = 7;
    //threads and blocks for padding pass 
    dim3 threadsPaddingPass = dim3(32, 11);
    int blocksPaddingPass = 13;
    //threads and blocks for non first metadata passes 
    int threadsOtherMetaDataPasses = 32;
    int blocksOtherMetaDataPasses = 7;


    int minMaxesLength = 20;



    //metadata
    const int metaXLength = 5;//8
    const int MetaYLength = 10;//30
    const int MetaZLength = 30;//8


    const int totalLength = metaXLength * MetaYLength * MetaZLength;

    /*   int*** h_tensor;
       h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

    int i, j, k, value = 0;

    const int mainXLength = dbXLength * metaXLength;
    const int mainYLength = 1200;//dbYLength * MetaYLength;
    const int mainZLength = dbZLength * MetaZLength;


    //main data arrays
    bool* goldArr = alloc_tensorToZeros<bool>(mainXLength, mainYLength, mainZLength);

    bool* segmArr = alloc_tensorToZeros<bool>(mainXLength, mainYLength, mainZLength);
    MetaDataCPU metaData;
    metaData.metaXLength = metaXLength;
    metaData.MetaYLength = MetaYLength;
    metaData.MetaZLength = MetaZLength;
    metaData.totalMetaLength = totalLength;


    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    int workQueueAndRLLength = 200;
    int workQueueWidth = 4;
    int resultListWidth = 5;
    //allocating to semiarbitrrary size 
    auto workQueuePointer = alloc_tensorToZeros<uint32_t>(workQueueAndRLLength, workQueueWidth, 1);




    // arguments to pass
    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = 2;
    forFullBoolPrepArgs.dbXLength = dbXLength;
    forFullBoolPrepArgs.dbYLength = dbYLength;
    forFullBoolPrepArgs.dbZLength = dbZLength;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
    forFullBoolPrepArgs.threads = threads;
    forFullBoolPrepArgs.blocks = blocks;

    forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
    forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

    forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
    forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

    forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
    forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

    forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
    forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

    //populate segm  and gold Arr


    auto arrGoldObj = forFullBoolPrepArgs.goldArr;
    auto arrSegmObj = forFullBoolPrepArgs.segmArr;




    for (int i = 0; i < 1; i++) {
        setArrCPUB(segmArr, i, i, 0, mainXLength, mainYLength);//
    }

    for (int i = 0; i < 1; i++) {
        setArrCPUB(goldArr, i, i, 900, mainXLength, mainYLength);//
    }
    /* int x = 0;
     int y = 900;
     int z = 0;
     int Nx = 5 * 32;
     goldArr[x + y * Nx] = true;*/


     //900 - 720
     //800 - 640

    // goldArr[ 300 * 32 ] = true;


    // goldArr[300 * 32] = true;

     //int lenn = 900;
     //goldArr[0] = true;
     //segmArr[lenn] = true;
     //goldArr[lenn] = true;
     //segmArr[lenn] = true;
     //segmArr[49*32] = true;



     //int plane = mainXLength * mainYLength;

     //for (int y = 0; y < mainXLength * (mainYLength / 2); y++) {
     //    goldArr[y] = true;

     //}

     ////segmArr[plane+1] = true;

     //int offset = plane * 3 * dbZLength;
     ////for (int y = offset; y < offset + mainXLength * (mainYLength / 2); y++) {
     ////	segmArr[y] = true;

     ////}



     ////
     //offset = mainXLength * mainYLength * mainZLength - (plane * 4);
     //for (int y = offset; y < offset + mainXLength * (mainYLength / 2); y++) {
     //    segmArr[y] = true;

     //}




     //int pointsNumber = 0;
     //int& pointsNumberRef = pointsNumber;
     //forTestPointStruct allPointsA[] = {
     //	// meta 2,2,2 only gold points not in result after 2 dilataions
     //getTestPoint(
     //2,2,2//x,y,z
     //,true//isGold
     //,0,0,0//xMeta,yMeta,Zmeta
     //,dbXLength,dbYLength,dbZLength,pointsNumberRef)
     //};

     /*
     maxX 2  [1]
 minX 1  [2]
 maxY 1  [3]
 minY 0  [4]
 maxZ 5  [5]
 minZ 2  [6]
     */


    printf("\n aaa \n");

    uint32_t* resultListPointerMetaCPU;
    uint32_t* resultListPointerLocalCPU;
    uint32_t* resultListPointerIterNumbCPU;
    uint32_t* metaDataArrPointerCPU;
    uint32_t* workQueuePointerCPU;

    uint32_t* reducedResCPU;
    uint32_t* origArrsCPU;



    ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
        , resultListPointerLocalCPU, resultListPointerIterNumbCPU
        , metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU, mainXLength, mainYLength, mainZLength
    );

    //for (int outer = 0; outer< ceil(lenn/ int(32)); outer++ ) {
    //	for (int u = 0; u < 32; u++) {
    //		int coord = outer * 32 + u;


    //		//3printf("set %d in %d \n  ", (reducedResCPU[u] >0), u);
    //	}
    //}







    //for (uint32_t linIdexMeta = 0; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
    //	//we get from linear index  the coordinates of the metadata block of intrest
    //	uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
    //	uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
    //	uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));

    //	for (int locPos = 0; locPos < 32 * fbArgs.dbYLength; locPos++) {
    //		auto col = reducedResCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
    //		if (col > 0) {
    //			for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
    //				int x = locPos % 32 + xMeta * fbArgs.dbXLength;
    //				int y = int(floor((float)(locPos / 32)) + yMeta * fbArgs.dbYLength);
    //				int z = bitPos + zMeta * fbArgs.dbZLength;

    //				if (y==0 && z==0) {
    //					if (isBitAtCPU(col, bitPos)) {
    //						printf("point gold set at x %d y %d z %d  \n"
    //							, locPos % 32 + xMeta * fbArgs.dbXLength
    //							, int(floor((float)(locPos / 32)) + yMeta * fbArgs.dbYLength)
    //							, bitPos + zMeta * fbArgs.dbZLength
    //						);
    //					}
    //				}
    //			}
    //		}
    //	}


    //	//for (int locPos = 32 * fbArgs.dbYLength; locPos < 32 * 2 * fbArgs.dbYLength; locPos++) {
    //	//	auto col = reducedResCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
    //	//	if (col > 0) {
    //	//		for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
    //	//			if (isBitAtCPU(col, bitPos)) {
    //	//				int locPosB = locPos - 32 * fbArgs.dbYLength;
    //	//				int x = locPosB % 32 + xMeta * fbArgs.dbXLength;
    //	//				int y = int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength);
    //	//				int z = bitPos + zMeta * fbArgs.dbZLength;
    //	//				if (y == 0 && z == 0) {

    //	//					printf("point segm  set at x %d y %d z %d  \n"
    //	//						, locPosB % 32 + xMeta * fbArgs.dbXLength
    //	//						, int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength)
    //	//						, bitPos + zMeta * fbArgs.dbZLength
    //	//					);
    //	//				}
    //	//			}
    //	//		}
    //	//	}
    //	//}


    //}















    //testDilatations(fbArgs, allPointsA, );






    //printFromReduced(fbArgs, reducedResCPU);
    //printIsBlockActiveEtc(fbArgs, metaDataArrPointerCPU, fbArgs.metaData);


    //for (int wQi = 0; wQi < minMaxesCPU[9]; wQi ++ ) {
    //	printf("in work q %d  \n ", workQueuePointerCPU[wQi] - isGoldOffset * (workQueuePointerCPU[wQi] >= isGoldOffset) );
    //}

    //for (int wQi = 0; wQi < 700; wQi++) {
    //	if (metaDataArrPointerCPU[wQi]==1) {
    //		printf("\n in metadaArr i %d  \n ", wQi);
    //	}
    //}

    //info in padding AND range 14 linMeta 2 new block adress 30   inMetadataArrIndex 612
    //	info in padding AND range 15 linMeta 2 new block adress 1   inMetadataArrIndex 32
    //	info in padding AND range 14 linMeta 0 new block adress 28   inMetadataArrIndex 571

//printf(" for cpu results ranges xMeta %d yMeta %d zMeta %d ", fbArgs.metaData.metaXLength, fbArgs.metaData.MetaYLength, fbArgs.metaData.MetaZLength);


    unsigned int xRange = minMaxesCPU[1] - minMaxesCPU[2] + 1;
    unsigned int yRange = minMaxesCPU[3] - minMaxesCPU[4] + 1;
    unsigned int zRange = minMaxesCPU[5] - minMaxesCPU[6] + 1;

    printf("before results xRange %d yRange %d zRange %d \n", xRange, yRange, zRange);
    dbYLength = 12;
    for (int i = 0; i < 5; i++) {
        if (resultListPointerLocalCPU[i] > 0 || resultListPointerMetaCPU[i] > 0) {
            uint32_t linIdexMeta = resultListPointerMetaCPU[i] - (isGoldOffset * (resultListPointerMetaCPU[i] >= isGoldOffset)) - 1;
            uint32_t xMeta = linIdexMeta % xRange;
            uint32_t zMeta = uint32_t(floor((float)(linIdexMeta / (xRange * yRange))));
            uint32_t yMeta = uint32_t(floor((float)((linIdexMeta - ((zMeta * xRange * yRange) + xMeta)) / xRange)));

            uint32_t linLocal = resultListPointerLocalCPU[i];
            uint32_t xLoc = linLocal % 32;
            uint32_t zLoc = uint32_t(floor((float)(linLocal / (32 * dbYLength))));
            uint32_t yLoc = uint32_t(floor((float)((linLocal - ((zLoc * 32 * dbYLength) + xLoc)) / 32)));


            uint32_t x = xMeta * 32 + xLoc;
            uint32_t y = yMeta * dbYLength + yLoc;
            uint32_t z = zMeta * 32 + zLoc;
            uint32_t iterNumb = resultListPointerIterNumbCPU[i];

            printf("resullt linIdexMeta %d x %d y %d z %d  xMeta %d yMeta %d zMeta %d xLoc %d yLoc %d zLoc %d linLocal %d  iterNumb %d \n"
                , linIdexMeta
                , x, y, z
                , xMeta, yMeta, zMeta
                , xLoc, yLoc, zLoc
                , linLocal
                , iterNumb


            );



        }
    }





    printf("\n **************************************** \n");

    i = 1;
    printf("maxX %d  [%d]\n", minMaxesCPU[i], i);
    i = 2;
    printf("minX %d  [%d]\n", minMaxesCPU[i], i);
    i = 3;
    printf("maxY %d  [%d]\n", minMaxesCPU[i], i);
    i = 4;
    printf("minY %d  [%d]\n", minMaxesCPU[i], i);
    i = 5;
    printf("maxZ %d  [%d]\n", minMaxesCPU[i], i);
    i = 6;
    printf("minZ %d  [%d]\n", minMaxesCPU[i], i);

    int ii = 7;
    printf("global FP count %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 8;
    printf("global FN count %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 9;
    printf("workQueueCounter %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 10;
    printf("resultFP globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 11;
    printf("resultFn globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 12;
    printf("global offset counter %d  [%d]\n", minMaxesCPU[ii], ii);

    ii = 13;
    printf("globalIterationNumb %d  [%d]\n", minMaxesCPU[ii], ii);
    ii = 17;
    printf("suum debug %d  [%d]\n", minMaxesCPU[ii], ii);





    //i, j, k, value = 0;
    //i = 31;
    //j = 12;
    //for (k = 0; k < MetaZLength; k++) {
    //	goldArr[k][j][i] = 1;
    //	if (reducedGold[k][j][i] > 0) {
    //		for (int tt = 0; tt < 32; tt++) {
    //			if ((reducedGold[k][j][i] & (1 << (tt)))) {
    //				printf("found in reduced fp  [%d]\n", k * 32 + tt);

    //			}
    //		}

    //	}
    //}


    //		i, j, k, value = 0;
    //for (i = 0; i < mainXLength; i++) {
    //	for (j = 0; j < mainYLength; j++) {
    //		for (k = 0; k < MetaZLength; k++) {
    //			//goldArr[k][j][i] = 1;
    //			if (reducedGold[k][j][i] > 0) {
    //				for (int tt = 0; tt < 32; tt++) {
    //					if ((reducedGold[k][j][i] & (1 << (tt)))) {
    //						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);

    //					}
    //				}

    //			}
    //		}
    //	}
    //}






    //minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]

    //int sumDebug = 0;
    //for (int ji = 0; ji < 8000; ji++) {
    //	if (forDebugArr[0][0][ji]==1) {
    //		sumDebug += forDebugArr[0][0][ji];
    //		//printf("for debug %d i %d \n", forDebugArr[0][0][ji],ji);
    //	}
    //}
    //printf("\n sumDebug %d \n", sumDebug);


//
//
//	//	for (int ji = 0; ji < minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]; ji++) {
//		for (int ji = 0; ji < 10; ji++) {
//    if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//   	 int x = forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji];
//	 int y = forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji];
//	 int z = forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji];
//	 int isGold = forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji];
//	 int iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji];
//
//	 //uint32_t x = forFullBoolPrepArgs.metaData.resultList.arrP[ji][0][0];
//	 //uint32_t y = forFullBoolPrepArgs.metaData.resultList.arrP[ji][1][0];
//	 //uint32_t z = forFullBoolPrepArgs.metaData.resultList.arrP[ji][2][0];
//	 //uint32_t isGold = forFullBoolPrepArgs.metaData.resultList.arrP[ji][3][0];
//	 //uint32_t iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[ji][4][0];
//
//
//   	 if (iternumb!=9) {
//   		 printf("result  in point  %d %d %d isGold %d iteration %d \n "
//   			 , x
//   			 , y
//   			 , z
//   			 , isGold
//   			 , iternumb);
//   	 }
//   	 else {
//   		 printf("**");
//   	 }
//
//    }
//}






     //for (int i = 0; i < workQueueAndRLLength; i++) {

        // if (workQueuePointer[0][2][i] > 0) {
        //	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
        //		 , 0, 0, i
        //		 , workQueuePointer[0][0][i]
        //		 , workQueuePointer[0][1][i]
        //		 , workQueuePointer[0][2][i]
        //		 , workQueuePointer[0][3][i]
        //	 );
        // }

     //}






    printf("cleaaning");



    free(goldArr);
    free(segmArr);


    free(resultListPointerMetaCPU);
    free(resultListPointerLocalCPU);
    free(resultListPointerIterNumbCPU);
    free(metaDataArrPointerCPU);
    free(workQueuePointerCPU);

    free(reducedResCPU);
    free(origArrsCPU);



}










void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {
    /*
     * Open the specified file and the specified dataset in the file.
     */
    H5File file(FILE_NAME, H5F_ACC_RDONLY);
    DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();


    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    cout << "Datasize: " << dims[0] << endl; // this is the correct number of values

    // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    DataSpace memspace(1, dimsm);



    data = (bool*)calloc(dims[0], sizeof(bool));




    dset.read(data, PredType::NATIVE_HBOOL, memspace, dspace);


    //int sum = 0;
    //for (int i = 0; i < dims[0]; i++) {
    //    sum += data[i];
    //}
    //printf("suuum %d \n  ", sum);


    file.close();

}



/*
benchmark for original code from  https://github.com/Oyatsumi/HausdorffDistanceComparison
*/
void benchmarkOliviera(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {
    Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    for (int x = 0; x < WIDTH; x++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int z = 0; z < DEPTH; z++) {
                img1.setVoxelValue(onlyLungsBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
                img2.setVoxelValue(onlyBladderBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
            }
        }
    }



    auto begin = std::chrono::high_resolution_clock::now();


    HausdorffDistance* hd = new HausdorffDistance();
    int dist = (*hd).computeDistance(&img1, &img2);



    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;

    printf("HD: %d \n", dist);

    //freeing memory
    img1.dispose(); img2.dispose();

    //Datasize: 216530944
    //Datasize : 216530944
    //Total elapsed time : 2.62191s
    //HD : 234 or 274 


    //reversed args
         //Total elapsed time : 1.44947s
     //    HD : 146

}





void benchmarkMitura(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {

    //// some preparations and configuring
    MetaDataCPU metaData;
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = true;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(onlyBladderBoolFlat, WIDTH, HEIGHT, DEPTH);
   // forFullBoolPrepArgs.goldArr = get3dArrCPU(onlyBladderBoolFlat, WIDTH, DEPTH, HEIGHT);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);
   // forFullBoolPrepArgs.segmArr = get3dArrCPU(onlyLungsBoolFlat, WIDTH, DEPTH, HEIGHT);
    /// for debugging
    uint32_t* resultListPointerMetaCPU;
    uint32_t* resultListPointerLocalCPU;
    uint32_t* resultListPointerIterNumbCPU;
    uint32_t* metaDataArrPointerCPU;
    uint32_t* workQueuePointerCPU;
    uint32_t* reducedResCPU;
    uint32_t* origArrsCPU;


    //function invocation
    auto begin = std::chrono::high_resolution_clock::now();

    ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
        , resultListPointerLocalCPU, resultListPointerIterNumbCPU
        , metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU, WIDTH, HEIGHT, DEPTH
    );


    //ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
    //    , resultListPointerLocalCPU, resultListPointerIterNumbCPU
    //    , metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU, WIDTH,  DEPTH, HEIGHT
    //);


    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;


    size_t sizeMinMax = sizeof(unsigned int) * 20;
    hipMemcpy(minMaxesCPU, fbArgs.metaData.minMaxes, sizeMinMax, hipMemcpyDeviceToHost);

    printf("HD: %d \n", minMaxesCPU[13]);


    // freeee
    free(onlyBladderBoolFlat);
    free(onlyLungsBoolFlat);


    free(resultListPointerMetaCPU);
    free(resultListPointerLocalCPU);
    free(resultListPointerIterNumbCPU);
    free(metaDataArrPointerCPU);
    free(workQueuePointerCPU);

    free(reducedResCPU);
    free(origArrsCPU);

}



void loadHDF() {
    const int WIDTH = 512;
    const int HEIGHT = 512;
    const int DEPTH = 826;





	//main data arrays
	//bool* onlyBladderBoolFlat = alloc_tensorToZeros<bool>(WIDTH, HEIGHT, DEPTH);

	//bool* onlyLungsBoolFlat = alloc_tensorToZeros<bool>(WIDTH, HEIGHT, DEPTH);

 //   onlyBladderBoolFlat[0] = true;
 //   onlyLungsBoolFlat[500] = true;
    const H5std_string FILE_NAMEonlyLungsBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlat, DATASET_NAMEonlyLungsBoolFlat, onlyLungsBoolFlat);

    const H5std_string FILE_NAMEonlyBladderBoolFlat("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
    const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlat");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlat, onlyBladderBoolFlat);

  //   benchmarkOliviera(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);
   //  benchmarkOliviera(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, DEPTH, HEIGHT);
    benchmarkMitura(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH,  DEPTH, HEIGHT);




}



int main(void) {

    //  const int WIDTH = atoi(argv[1]), HEIGHT = WIDTH, DEPTH = 1;
   //   Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);
   // testMainPasswes();
    loadHDF();



    return 0;  // successfully terminated
}