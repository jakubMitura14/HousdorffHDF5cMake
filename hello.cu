#include "hip/hip_runtime.h"



#include "hip/hip_runtime.h"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>


#include "MetaData.cu"
#include "ExceptionManagUtils.cu"
#include "ForBoolKernel.cu"
#include "FirstMetaPass.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "MetaDataOtherPasses.cu"
#include "MinMaxesKernel.cu"
#include "MainKernelMetaHelpers.cu"
#include <cooperative_groups/memcpy_async.h>
using namespace cooperative_groups;



//#include "hdf5Manag.cu"
#include <iostream>
#include <string>
#include <vector>
#define H5_BUILT_AS_DYNAMIC_LIB 1
#include <H5Cpp.h>






/*
gettinng  array for dilatations
basically arrays will alternate between iterations once one will be source other target then they will switch - we will decide upon knowing
wheather the iteration number is odd or even
*/
#pragma once
template <typename TXPI>
inline __device__ uint32_t* getSourceReduced(ForBoolKernelArgs<TXPI>& fbArgs, int(&iterationNumb)[1]) {


    if ((iterationNumb[0] & 1) == 0) {
        return fbArgs.mainArrAPointer;

    }
    else {
        return fbArgs.mainArrBPointer;
    }


}


/*
gettinng target array for dilatations
*/
#pragma once
template <typename TXPPI>
inline __device__ uint32_t* getTargetReduced(ForBoolKernelArgs<TXPPI>& fbArgs, int(&iterationNumb)[1]) {

    if ((iterationNumb[0] & 1) == 0) {
        //printf(" BB ");

        return fbArgs.mainArrBPointer;

    }
    else {
        // printf(" AA ");

        return fbArgs.mainArrAPointer;

    }

}


/*
dilatation up and down - using bitwise operators
*/
#pragma once
inline __device__ uint32_t bitDilatate(uint32_t& x) {
    return ((x) >> 1) | (x) | ((x) << 1);
}

/*
return 1 if at given position of given number bit is set otherwise 0
*/
#pragma once
inline __device__ uint32_t isBitAt(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}

#pragma once
inline uint32_t isBitAtCPU(uint32_t& numb, const int pos) {
    return (numb & (1 << (pos)));
}






//
///*
//given source and target uint32 it will check the bit of intrest  of source and set the target to bit of target intrest
//*/
//#pragma once
//inline __device__ void setBitTo(uint32_t source, uint8_t sourceBit, uint32_t resShared[32][32], uint8_t targetBit) {   
//    resShared[threadIdx.x][threadIdx.y] |= ((source >> sourceBit) & 1) << targetBit;
//   // return target;
//}

///////////////////////////////// new functions





/*
to iterate over the threads and given their position - checking edge cases do appropriate dilatations ...
works only for anterior - posterior lateral an medial dilatations
predicate - indicates what we consider border case here
paddingPos = integer marking which padding we are currently talking about(top ? bottom ? anterior ? ...)
padingVariedA, padingVariedB - eithr bitPos threadid X or Y depending what will be changing in this case

normalXChange, normalYchange - indicating which wntries we are intrested in if we are not at the boundary so how much to add to xand y thread position
metaDataCoordIndex - index where in the metadata of this block th linear index of neihjbouring block is present
targetShmemOffset - offset where loaded data needed for dilatation of outside of the block is present for example defining  register shmem one or 2 ...
*/
#pragma once
template <typename TXPI>
inline __device__ void dilatateHelperForTransverse(ForBoolKernelArgs<TXPI>& fbArgs, const bool predicate,
    const uint8_t  paddingPos, const   int8_t  normalXChange, const  int8_t normalYchange
    , uint32_t(&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6]
    , const uint8_t forBorderYcoord, const  uint8_t forBorderXcoord
    , const uint8_t metaDataCoordIndex, const uint32_t targetShmemOffset, uint32_t(&localBlockMetaData)[40], uint32_t& i
    , bool(&isGoldForLocQueue)[localWorkQueLength]) {



    //if (paddingPos == 3 && mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0 && isGoldForLocQueue[i] == 0 ) {
    //if ( mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]>0 && isGoldForLocQueue[i] == 1 ) {
    //
    //    printf("something in loaded from right idX %d idY %d  paddingPos %d \n", threadIdx.x, threadIdx.y , paddingPos );
    //}


    // so we first check for corner cases 
    if (predicate) {


        // now we need to load the data from the neigbouring blocks
        //first checking is there anything to look to 
        if (localBlockMetaData[(i & 1) * 20 + metaDataCoordIndex] < isGoldOffset) {


            //now we load - we already done earlier up and down so now we are considering only anterior, posterior , left , right possibilities
            if (mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] > 0) {
                isAnythingInPadding[paddingPos] = true;

            };



            mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] =
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
                | mainShmem[targetShmemOffset + forBorderXcoord + forBorderYcoord * 32];

        };
    }
    else {//given we are not in corner case we need just to do the dilatation using biwise or with the data inside the block


        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]
            = mainShmem[begSourceShmem + (threadIdx.x + normalXChange) + (threadIdx.y + normalYchange) * 32]
            | mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

    }


}


#pragma once
inline __device__ void dilatateHelperTopDown(const uint8_t paddingPos,
    uint32_t(&mainShmem)[lengthOfMainShmem], bool(&isAnythingInPadding)[6], uint32_t(&localBlockMetaData)[40]
    , const uint8_t metaDataCoordIndex
    , const  uint8_t sourceBit
    , const uint8_t targetBit
    , const uint32_t targetShmemOffset, uint32_t& i
) {
    // now we need to load the data from the neigbouring blocks
    //first checking is there anything to look to 
    if (localBlockMetaData[(i & 1) * 20 + metaDataCoordIndex] < isGoldOffset) {
        if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], targetBit)) {
            // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
            isAnythingInPadding[paddingPos] = true;
        };
        // if in bit of intrest of neighbour block is set
        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] |= ((mainShmem[targetShmemOffset + threadIdx.x + threadIdx.y * 32] >> sourceBit) & 1) << targetBit;
    }

}



/*
we need to define here the function that will update the metadata result for the given block -
also if it is not padding pass we need to set the neighbouring blocks as to be activated according to the data in shmem
this will also include preparations for next round of iterations through blocks from work queue
isInPipeline - marks is it meant to be executed at the begining of the pipeline or after the pipeline
finilizing operations for last block
*/
#pragma once
inline __device__  void afterBlockClean(thread_block& cta
    , unsigned int(&worQueueStep)[1], uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], const uint32_t i, MetaDataGPU& metaData
    , thread_block_tile<32>& tile
    , unsigned int(&localFpConter)[1], unsigned int(&localFnConter)[1]
    , unsigned int(&blockFpConter)[1], unsigned int(&blockFnConter)[1]
    , uint32_t*& metaDataArr
    , bool(&isAnythingInPadding)[6], bool(&isBlockFull)[2], const bool isPaddingPass, bool(&isGoldForLocQueue)[localWorkQueLength], uint32_t(&lastI)[1]
) {

    if (i < UINT32_MAX) {

        if (threadIdx.x == 7 && threadIdx.y == 0) {// this is how it is encoded wheather it is gold or segm block
                       //this will be executed only if fp or fn counters are bigger than 0 so not during first pass
            if (localFpConter[0] >= 0) {
                metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 3] += localFpConter[0];
                blockFpConter[0] += localFpConter[0];
                localFpConter[0] = 0;
            }
        };
        if (threadIdx.x == 8 && threadIdx.y == 3) {

            if (localFnConter[0] >= 0) {
                metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 4] += localFnConter[0];

                blockFnConter[0] += localFnConter[0];
                localFnConter[0] = 0;
            }
        };
        if (threadIdx.x == 9 && threadIdx.y == 2) {// this is how it is encoded wheather it is gold or segm block

            //executed in case of previous block
            if (isBlockFull[i & 1] && i >= 0) {
                //setting data in metadata that block is full
                metaDataArr[mainShmem[startOfLocalWorkQ + i] * metaData.metaDataSectionLength + 10 - (isGoldForLocQueue[i] * 2)] = true;
            }
            //resetting for some reason  block 0 gets as full even if it should not ...
            isBlockFull[i & 1] = true;// mainShmem[startOfLocalWorkQ + i]>0;//!isPaddingPass;
        };




        //we do it only for non padding pass
        if (threadIdx.x < 6 && threadIdx.y == 1 && !isPaddingPass) {
            //executed in case of previous block
            if (i >= 0) {
                auto metadataTarget = localBlockMetaData[(i & 1) * 20 + 13 + threadIdx.x];

                if (metadataTarget < isGoldOffset) {

                    if (isAnythingInPadding[threadIdx.x]) {
                        // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
                        metaDataArr[metadataTarget * metaData.metaDataSectionLength + 12 - isGoldForLocQueue[i]] = 1;
                    }

                }
            }
            isAnythingInPadding[tile.thread_rank()] = false;
        };

    };


}













//template <typename TKKI, typename forPipeline >
#pragma once
template <typename TKKI >
inline __device__ void mainDilatation(const bool isPaddingPass, ForBoolKernelArgs<TKKI>& fbArgs, uint32_t*& mainArrAPointer,
    uint32_t*& mainArrBPointer, MetaDataGPU& metaData
    , unsigned int*& minMaxes, uint32_t*& workQueue
    , uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb,
    thread_block& cta, thread_block_tile<32>& tile, grid_group& grid, uint32_t(&mainShmem)[lengthOfMainShmem]
    , bool(&isAnythingInPadding)[6], bool(&isBlockFull)[2], int(&iterationNumb)[1], unsigned int(&globalWorkQueueOffset)[1]
    , unsigned int(&globalWorkQueueCounter)[1]
    , unsigned int(&localWorkQueueCounter)[1], unsigned int(&localTotalLenthOfWorkQueue)[1]
    , unsigned int(&localFpConter)[1]
    , unsigned int(&localFnConter)[1], unsigned int(&blockFpConter)[1]
    , unsigned int(&blockFnConter)[1], unsigned int(&resultfpOffset)[1]
    , unsigned int(&resultfnOffset)[1], unsigned int(&worQueueStep)[1]
    , unsigned int(&localMinMaxes)[5]
    , uint32_t(&localBlockMetaData)[40]
    , unsigned int(&fpFnLocCounter)[1]
    , bool(&isGoldPassToContinue)[1], bool(&isSegmPassToContinue)[1]
    , uint32_t*& origArrs, uint32_t*& metaDataArr, bool(&isGoldForLocQueue)[localWorkQueLength]
    , uint32_t(&lastI)[1]
    , cuda::pipeline<cuda::thread_scope_block>& pipeline
) {


    //initial cleaning  and initializations include loading min maxes
    if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
        iterationNumb[0] += 1;
    };

    if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
        localWorkQueueCounter[0] = 0;
    };

    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        blockFpConter[0] = 0;
    };
    if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
        blockFnConter[0] = 0;
    };
    if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
        localFpConter[0] = 0;
    };
    if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
        localFnConter[0] = 0;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
        isBlockFull[0] = true;
    };
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 1) {
        isBlockFull[1] = true;
    };

    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
        fpFnLocCounter[0] = 0;
    };


    if (tile.thread_rank() == 10 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

        lastI[0] = UINT32_MAX;
    };


    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        localTotalLenthOfWorkQueue[0] = minMaxes[9];
        globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
        worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
    };

    if (tile.meta_group_rank() == 1) {
        cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
    }

    sync(cta);

    /// load work QueueData into shared memory 
    for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
        // grid stride loop - sadly most of threads will be idle 
        /////////// loading to work queue
        if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

            for (uint16_t ii = cta.thread_rank(); ii < worQueueStep[0]; ii += cta.size()) {
                mainShmem[startOfLocalWorkQ + ii] = workQueue[bigloop + ii];
                isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
                mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];

            }

        }
        //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
        ////##### pipeline Step 0

        sync(cta);




        //loading metadata
        pipeline.producer_acquire();
        if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

            cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                (&metaDataArr[mainShmem[startOfLocalWorkQ] * metaData.metaDataSectionLength])
                , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

        }
        pipeline.producer_commit();



        for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
            if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                //////////////// step 0  load main data and final processing of previous block
                              //loading main data for first dilatation
                               //IMPORTANT we need to keep a lot of variables constant here like is Anuthing in padding of fp count .. as the represent processing of previous block  - so do not modify them here ...

                pipeline.producer_acquire();
                cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
                    mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                    cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
                pipeline.producer_commit();

                pipeline.consumer_wait();


                afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i - 1,
                    metaData, tile, localFpConter, localFnConter
                    , blockFpConter, blockFnConter
                    , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);


                //needed for after block metadata update
                if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
                    lastI[0] = i;
                }

                pipeline.consumer_release();

                ///////// step 1 load top and process main data 
                                //load top 
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
                        * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();
                //process main
                pipeline.consumer_wait();
                //marking weather block is already full and no more dilatations are possible 
                if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) < 32) {
                    isBlockFull[i & 1] = false;
                }
                mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
                pipeline.consumer_release();

                ///////// step 2 load bottom and process top 
                                //load bottom
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
                    cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14]
                        * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();
                //process top
                pipeline.consumer_wait();

                dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
                    , 31, 0
                    , begfirstRegShmem, i);

                pipeline.consumer_release();

                /////////// step 3 load right  process bottom  
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();
                //process bototm
                pipeline.consumer_wait();

                dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
                    , 0, 31
                    , begSecRegShmem, i);

                pipeline.consumer_release();
                /////////// step 4 load left process right  
                                //load left 
                pipeline.producer_acquire();
                if (mainShmem[startOfLocalWorkQ + i] > 0) {
                    cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();
                //process right
                pipeline.consumer_wait();

                dilatateHelperForTransverse(fbArgs, (threadIdx.x == (fbArgs.dbXLength - 1)),
                    3, (1), (0), mainShmem, isAnythingInPadding
                    , threadIdx.y, 0
                    , 16, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);

                pipeline.consumer_release();

                /////// step 5 load anterior process left 
                                //load anterior
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {

                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();
                //process left 
                pipeline.consumer_wait();

                dilatateHelperForTransverse(fbArgs, (threadIdx.x == 0),
                    2, (-1), (0), mainShmem, isAnythingInPadding
                    , threadIdx.y, 31
                    , 15, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);

                pipeline.consumer_release();
                /////// step 6 load posterior process anterior 
                                //load posterior
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {


                    cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                        &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);
                }
                pipeline.producer_commit();

                //process anterior
                pipeline.consumer_wait();

                dilatateHelperForTransverse(fbArgs, (threadIdx.y == (fbArgs.dbYLength - 1)), 4
                    , (0), (1), mainShmem, isAnythingInPadding
                    , 0, threadIdx.x
                    , 17, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);
                pipeline.consumer_release();

                /////// step 7 
                               //load reference if needed or data for next iteration if there is such 
                                //process posterior, save data from res shmem to global memory also we mark weather block is full
                pipeline.producer_acquire();

                //if block should be validated we load data for validation
                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                    cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                        &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
                        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                        , pipeline);

                }
                else {//if we are not validating we immidiately start loading data for next loop
                    if (i + 1 < worQueueStep[0]) {
                        cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1 + i])
                                * metaData.metaDataSectionLength])
                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


                    }
                }


                pipeline.producer_commit();

                //processPosteriorAndSaveResShmem

                pipeline.consumer_wait();
                //dilatate posterior 
                dilatateHelperForTransverse(fbArgs, (threadIdx.y == 0), 5
                    , (0), (-1), mainShmem, isAnythingInPadding
                    , fbArgs.dbYLength - 1, threadIdx.x // we add offset depending on y dimension
                    , 18, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);
                //now all data should be properly dilatated we save it to global memory
                //try save target reduced via mempcy async ...

                getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                    + threadIdx.x + threadIdx.y * 32]
                    = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];



                pipeline.consumer_release();

                sync(cta);

                //////// step 8 basically in order to complete here anyting the count need to be bigger than counter
                                              // loading for next block if block is not to be validated it was already done earlier
                pipeline.producer_acquire();
                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                    if (i + 1 < worQueueStep[0]) {


                        cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                            (&metaDataArr[(mainShmem[startOfLocalWorkQ + 1 + i])
                                * metaData.metaDataSectionLength])
                            , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                    }
                }
                pipeline.producer_commit();


                //validation - so looking for newly covered voxel for opposite array so new fps or new fns
                pipeline.consumer_wait();

                if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                        //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 
                        //    ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) 
                        //        & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



                        //we now look for bits prasent in both reference arrays and current one
                       // mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

                        // now we look through bits and when some is set we call it a result 
#pragma unroll
                    for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                        //if any bit here is set it means it should be added to result list 
                        if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                            && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                            && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                            ) {

                            //just re
                            mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = 0;
                            ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                            if (isGoldForLocQueue[i]) {
                                mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3]);
                            }
                            else {
                                mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4]);
                                //    printf("local fn counter add \n");

                            };
                            //   add results to global memory    
                            //we add one gere jjust to distinguish it from empty result
                            resultListPointerMeta[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                            resultListPointerLocal[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                            resultListPointerIterNumb[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(iterationNumb[0]);



                        }

                    };

                }
                /////////
                pipeline.consumer_release();
                sync(cta);


            }
        }

        //here we are after all of the blocks planned to be processed by this block are

//updating local counters of last local block (normally it is done at the bagining of the next block)
//but we need to check weather any block was processed at all
        pipeline.consumer_wait();

        afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, lastI[0],
            metaData, tile, localFpConter, localFnConter
            , blockFpConter, blockFnConter
            , metaDataArr, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);


        pipeline.consumer_release();

    }

    sync(cta);

    //     updating global counters
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
        if (blockFpConter[0] > 0) {
            atomicAdd(&(minMaxes[10]), (blockFpConter[0]));
        }
    };
    if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
        if (blockFnConter[0] > 0) {
            atomicAdd(&(minMaxes[11]), (blockFnConter[0]));
        }
    };
    // in first thread block we zero work queue counter
    if (threadIdx.x == 2 && threadIdx.y == 0) {
        if (blockIdx.x == 0) {

            minMaxes[9] = 0;
        }
    };


}





/*
5)Main block
    a) we define the work queue iteration - so we divide complete work queue into parts  and each thread block analyzes its own part - one data block at a textLinesFromStrings
    b) we load values of data block into shared memory  and immidiately do the bit wise up and down dilatations, and mark booleans needed to establish is the datablock full
    c) synthreads - left,right, anterior,posterior dilatations...
    d) add the dilatated info into dilatation array and padding info from dilatation to global memory
    e) if block is to be validated we check is there is in the point of currently coverd voxel some voxel in other mas if so we add it to the result list and increment local reult counter
    f) syncgrid()
6)analyze padding
    we iterate over work queue as in 5
    a) we load into shared memory information from padding from blocks all around the block of intrest checking for boundary conditions
    b) we save data of dilatated voxels into dilatation array making sure to synchronize appropriately in the thread block
    c) we analyze the positive entries given the block is to be validated  so we check is such entry is already in dilatation mask if not is it in other mask if first no and second yes we add to the result
    d) also given any positive entry we set block as to be activated simple sum reduction should be sufficient
    e) sync grid
*/




template <typename TKKI>
inline __global__ void mainPassKernel(ForBoolKernelArgs<TKKI> fbArgs) {



    thread_block cta = cooperative_groups::this_thread_block();

    thread_block_tile<32> tile = tiled_partition<32>(cta);
    grid_group grid = cooperative_groups::this_grid();

    /*
    * according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes
    main shared memory spaces
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-  4127: small 32 length resgister 3 space
    4128-4500 (372 length) : place for local work queue in dilatation kernels
    */
    __shared__ uint32_t mainShmem[lengthOfMainShmem];



    constexpr size_t stages_count = 2; // Pipeline stages number

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;

    //cuda::pipeline<cuda::thread_scope_thread>  pipeline = cuda::make_pipeline(cta, &shared_state);
    cuda::pipeline<cuda::thread_scope_block>  pipeline = cuda::make_pipeline(cta, &shared_state);



    //usefull for iterating through local work queue
    __shared__ bool isGoldForLocQueue[localWorkQueLength];
    // holding data about paddings 


    // holding data weather we have anything in padding 0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior,
    __shared__ bool isAnythingInPadding[6];

    __shared__ bool isBlockFull[2];

    __shared__ uint32_t lastI[1];


    //variables needed for all threads
    __shared__ int iterationNumb[1];
    __shared__ unsigned int globalWorkQueueOffset[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    __shared__ unsigned int localWorkQueueCounter[1];
    // keeping data wheather gold or segmentation pass should continue - on the basis of global counters

    __shared__ unsigned int localTotalLenthOfWorkQueue[1];
    //counters for per block number of results added in this iteration
    __shared__ unsigned int localFpConter[1];
    __shared__ unsigned int localFnConter[1];

    __shared__ unsigned int blockFpConter[1];
    __shared__ unsigned int blockFnConter[1];

    __shared__ unsigned int fpFnLocCounter[1];

    //result list offset - needed to know where to write a result in a result list
    __shared__ unsigned int resultfpOffset[1];
    __shared__ unsigned int resultfnOffset[1];

    __shared__ unsigned int worQueueStep[1];


    /* will be used to store all of the minMaxes varibles from global memory (from 7 to 11)
    0 : global FP count;
    1 : global FN count;
    2 : workQueueCounter
    3 : resultFP globalCounter
    4 : resultFn globalCounter
    */
    __shared__ unsigned int localMinMaxes[5];

    /* will be used to store all of block metadata
  nothing at  0 index
 1 :fpCount
 2 :fnCount
 3 :fpCounter
 4 :fnCounter
 5 :fpOffset
 6 :fnOffset
 7 :isActiveGold
 8 :isFullGold
 9 :isActiveSegm
 10 :isFullSegm
 11 :isToBeActivatedGold
 12 :isToBeActivatedSegm
 12 :isToBeActivatedSegm
//now linear indexes of the blocks in all sides - if there is no block in given direction it will equal UINT32_MAX
 13 : top
 14 : bottom
 15 : left
 16 : right
 17 : anterior
 18 : posterior
    */

    __shared__ uint32_t localBlockMetaData[40];

    /*
 //now linear indexes of the previous block in all sides - if there is no block in given direction it will equal UINT32_MAX

 0 : top
 1 : bottom
 2 : left
 3 : right
 4 : anterior
 5 : posterior

    */


    /////used mainly in meta passes

//    __shared__ unsigned int fpFnLocCounter[1];
    __shared__ bool isGoldPassToContinue[1];
    __shared__ bool isSegmPassToContinue[1];





    //initializations and loading    
    if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) { iterationNumb[0] = -1; };
    if (tile.thread_rank() == 11 && tile.meta_group_rank() == 0) {
        isGoldPassToContinue[0] = true;
    };
    if (tile.thread_rank() == 12 && tile.meta_group_rank() == 0) {
        isSegmPassToContinue[0] = true;

    };


    //here we caclulate the offset for given block depending on length of the workqueue and number of the  available blocks in a grid
    // - this will give us number of work queue items per block - we will calculate offset on the basis of the block number



   // for (int t = 0; t < 3; t++) {
    do {

        //for (bool isPaddingPass = false; isPaddingPass; isPaddingPass = true) {
        for (uint8_t isPaddingPass = 0; isPaddingPass < 2; isPaddingPass++) {
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /////////////////////////****************************************************************************************************************  
            /// dilataions

    //initial cleaning  and initializations include loading min maxes
            if (tile.thread_rank() == 7 && tile.meta_group_rank() == 0 && !isPaddingPass) {
                iterationNumb[0] += 1;
            };

            if (tile.thread_rank() == 6 && tile.meta_group_rank() == 0) {
                localWorkQueueCounter[0] = 0;
            };

            if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
                blockFpConter[0] = 0;
            };
            if (tile.thread_rank() == 2 && tile.meta_group_rank() == 0) {
                blockFnConter[0] = 0;
            };
            if (tile.thread_rank() == 3 && tile.meta_group_rank() == 0) {
                localFpConter[0] = 0;
            };
            if (tile.thread_rank() == 4 && tile.meta_group_rank() == 0) {
                localFnConter[0] = 0;
            };
            if (tile.thread_rank() == 9 && tile.meta_group_rank() == 0) {
                isBlockFull[0] = true;
            };
            if (tile.thread_rank() == 9 && tile.meta_group_rank() == 1) {
                isBlockFull[1] = true;
            };

            if (tile.thread_rank() == 10 && tile.meta_group_rank() == 0) {
                fpFnLocCounter[0] = 0;
            };


            if (tile.thread_rank() == 10 && tile.meta_group_rank() == 2) {// this is how it is encoded wheather it is gold or segm block

                lastI[0] = UINT32_MAX;
            };


            if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
                localTotalLenthOfWorkQueue[0] = fbArgs.minMaxes[9];
                globalWorkQueueOffset[0] = floor((float)(localTotalLenthOfWorkQueue[0] / gridDim.x)) + 1;
                worQueueStep[0] = min(localWorkQueLength, globalWorkQueueOffset[0]);
            };

            if (tile.meta_group_rank() == 1) {
                cooperative_groups::memcpy_async(tile, (&localMinMaxes[0]), (&fbArgs.minMaxes[7]), cuda::aligned_size_t<4>(sizeof(unsigned int) * 5));
            }

            sync(cta);

            /// load work QueueData into shared memory 
            for (uint32_t bigloop = blockIdx.x * globalWorkQueueOffset[0]; bigloop < ((blockIdx.x + 1) * globalWorkQueueOffset[0]); bigloop += worQueueStep[0]) {
                // grid stride loop - sadly most of threads will be idle 
                /////////// loading to work queue
                if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                    for (uint16_t ii = cta.thread_rank(); ii < worQueueStep[0]; ii += cta.size()) {
                        mainShmem[startOfLocalWorkQ + ii] = fbArgs.workQueuePointer[bigloop + ii];
                        isGoldForLocQueue[ii] = (mainShmem[startOfLocalWorkQ + ii] >= isGoldOffset);
                        mainShmem[startOfLocalWorkQ + ii] = mainShmem[startOfLocalWorkQ + ii] - isGoldOffset * isGoldForLocQueue[ii];

                    }

                }
                //now all of the threads in the block needs to have the same i value so we will increment by 1 we are preloading to the pipeline block metaData
                ////##### pipeline Step 0

                sync(cta);




                //loading metadata
                pipeline.producer_acquire();
                if (((bigloop) < localTotalLenthOfWorkQueue[0]) && ((bigloop) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {

                    cuda::memcpy_async(cta, (&localBlockMetaData[0]),
                        (&fbArgs.metaDataArrPointer[mainShmem[startOfLocalWorkQ] * fbArgs.metaData.metaDataSectionLength])
                        , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                }
                pipeline.producer_commit();



                for (uint32_t i = 0; i < worQueueStep[0]; i += 1) {
                    if (((bigloop + i) < localTotalLenthOfWorkQueue[0]) && ((bigloop + i) < ((blockIdx.x + 1) * globalWorkQueueOffset[0]))) {
                        //////////////// step 0  load main data and final processing of previous block
                                      //loading main data for first dilatation
                                       //IMPORTANT we need to keep a lot of variables constant here like is Anuthing in padding of fp count .. as the represent processing of previous block  - so do not modify them here ...

                        pipeline.producer_acquire();
                        cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
                            mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                            cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength), pipeline);
                        pipeline.producer_commit();

                        pipeline.consumer_wait();


                        afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, i - 1,
                            fbArgs.metaData, tile, localFpConter, localFnConter
                            , blockFpConter, blockFnConter
                            , fbArgs.metaDataArrPointer, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);


                        //needed for after block metadata update
                        if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
                            lastI[0] = i;
                        }

                        pipeline.consumer_release();

                        ///////// step 1 load top and process main data 
                                        //load top 
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 13] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
                                * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process main
                        pipeline.consumer_wait();
                        //marking weather block is already full and no more dilatations are possible 
                        if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) < 32) {
                            isBlockFull[i & 1] = false;
                        }
                        mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
                        pipeline.consumer_release();

                        ///////// step 2 load bottom and process top 
                                        //load bottom
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 14] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14]
                                * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process top
                        pipeline.consumer_wait();

                        dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
                            , 31, 0
                            , begfirstRegShmem, i);

                        pipeline.consumer_release();

                        /////////// step 3 load right  process bottom  
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 16] < isGoldOffset) {
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 16] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process bototm
                        pipeline.consumer_wait();

                        dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
                            , 0, 31
                            , begSecRegShmem, i);

                        pipeline.consumer_release();
                        /////////// step 4 load left process right  
                                        //load left 
                        pipeline.producer_acquire();
                        if (mainShmem[startOfLocalWorkQ + i] > 0) {
                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i] - 1) * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process right
                        pipeline.consumer_wait();

                        dilatateHelperForTransverse(fbArgs, (threadIdx.x == (fbArgs.dbXLength - 1)),
                            3, (1), (0), mainShmem, isAnythingInPadding
                            , threadIdx.y, 0
                            , 16, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);

                        pipeline.consumer_release();

                        /////// step 5 load anterior process left 
                                        //load anterior
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 17] < isGoldOffset) {

                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();
                        //process left 
                        pipeline.consumer_wait();

                        dilatateHelperForTransverse(fbArgs, (threadIdx.x == 0),
                            2, (-1), (0), mainShmem, isAnythingInPadding
                            , threadIdx.y, 31
                            , 15, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);

                        pipeline.consumer_release();
                        /////// step 6 load posterior process anterior 
                                        //load posterior
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + 18] < isGoldOffset) {


                            cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
                                &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);
                        }
                        pipeline.producer_commit();

                        //process anterior
                        pipeline.consumer_wait();

                        dilatateHelperForTransverse(fbArgs, (threadIdx.y == (fbArgs.dbYLength - 1)), 4
                            , (0), (1), mainShmem, isAnythingInPadding
                            , 0, threadIdx.x
                            , 17, begfirstRegShmem, localBlockMetaData, i, isGoldForLocQueue);
                        pipeline.consumer_release();

                        /////// step 7 
                                       //load reference if needed or data for next iteration if there is such 
                                        //process posterior, save data from res shmem to global memory also we mark weather block is full
                        pipeline.producer_acquire();

                        //if block should be validated we load data for validation
                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                        > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                            cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
                                &fbArgs.origArrsPointer[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
                                cuda::aligned_size_t<128>(sizeof(uint32_t) * fbArgs.metaData.mainArrXLength)
                                , pipeline);

                        }
                        else {//if we are not validating we immidiately start loading data for next loop
                            if (i + 1 < worQueueStep[0]) {
                                cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                                    (&fbArgs.metaDataArrPointer[(mainShmem[startOfLocalWorkQ + 1 + i])
                                        * fbArgs.metaData.metaDataSectionLength])
                                    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);


                            }
                        }


                        pipeline.producer_commit();

                        //processPosteriorAndSaveResShmem

                        pipeline.consumer_wait();
                        //dilatate posterior 
                        dilatateHelperForTransverse(fbArgs, (threadIdx.y == 0), 5
                            , (0), (-1), mainShmem, isAnythingInPadding
                            , fbArgs.dbYLength - 1, threadIdx.x // we add offset depending on y dimension
                            , 18, begSecRegShmem, localBlockMetaData, i, isGoldForLocQueue);
                        //now all data should be properly dilatated we save it to global memory
                        //try save target reduced via mempcy async ...

                        getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * fbArgs.metaData.mainArrSectionLength + fbArgs.metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                            + threadIdx.x + threadIdx.y * 32]
                            = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];



                        pipeline.consumer_release();

                        sync(cta);

                        //////// step 8 basically in order to complete here anyting the count need to be bigger than counter
                                                      // loading for next block if block is not to be validated it was already done earlier
                        pipeline.producer_acquire();
                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                            > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                            if (i + 1 < worQueueStep[0]) {


                                cuda::memcpy_async(cta, (&localBlockMetaData[((i + 1) & 1) * 20]),
                                    (&fbArgs.metaDataArrPointer[(mainShmem[startOfLocalWorkQ + 1 + i])
                                        * fbArgs.metaData.metaDataSectionLength])
                                    , cuda::aligned_size_t<4>(sizeof(uint32_t) * 20), pipeline);

                            }
                        }
                        pipeline.producer_commit();


                        //validation - so looking for newly covered voxel for opposite array so new fps or new fns
                        pipeline.consumer_wait();

                        if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                            > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
                                //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 
                                //    ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) 
                                //        & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



                                //we now look for bits prasent in both reference arrays and current one
                               // mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

                                // now we look through bits and when some is set we call it a result 
#pragma unroll
                            for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
                                //if any bit here is set it means it should be added to result list 
                                if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                                    ) {

                                    //just re
                                    mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = 0;
                                    ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                                    if (isGoldForLocQueue[i]) {
                                        mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3]);
                                    }
                                    else {
                                        mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32] = uint32_t(atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4]);
                                        //    printf("local fn counter add \n");

                                    };
                                    //   add results to global memory    
                                    //we add one gere jjust to distinguish it from empty result
                                    fbArgs.resultListPointerMeta[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                                    fbArgs.resultListPointerLocal[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                                    fbArgs.resultListPointerIterNumb[mainShmem[begSecRegShmem + threadIdx.x + threadIdx.y * 32]] = uint32_t(iterationNumb[0]);



                                }

                            };

                        }
                        /////////
                        pipeline.consumer_release();
                        sync(cta);


                    }
                }

                //here we are after all of the blocks planned to be processed by this block are

        //updating local counters of last local block (normally it is done at the bagining of the next block)
        //but we need to check weather any block was processed at all
                sync(cta);
                pipeline.consumer_wait();

                afterBlockClean(cta, worQueueStep, localBlockMetaData, mainShmem, lastI[0],
                    fbArgs.metaData, tile, localFpConter, localFnConter
                    , blockFpConter, blockFnConter
                    , fbArgs.metaDataArrPointer, isAnythingInPadding, isBlockFull, isPaddingPass, isGoldForLocQueue, lastI);


                pipeline.consumer_release();

            }

            sync(cta);

            //     updating global counters
            if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
                if (blockFpConter[0] > 0) {
                    atomicAdd(&(fbArgs.minMaxes[10]), (blockFpConter[0]));
                }
            };
            if (tile.thread_rank() == 1 && tile.meta_group_rank() == 0) {
                if (blockFnConter[0] > 0) {
                    atomicAdd(&(fbArgs.minMaxes[11]), (blockFnConter[0]));
                }
            };
            // in first thread block we zero work queue counter
            if (threadIdx.x == 2 && threadIdx.y == 0) {
                if (blockIdx.x == 0) {

                    fbArgs.minMaxes[9] = 0;
                }
            };

            grid.sync();
            /////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/////////////////////////****************************************************************************************************************  
/// metadata pass
            metadataPass(fbArgs, !isPaddingPass
                , mainShmem, globalWorkQueueOffset, globalWorkQueueCounter
                , localWorkQueueCounter, localTotalLenthOfWorkQueue, localMinMaxes
                , fpFnLocCounter, isGoldPassToContinue, isSegmPassToContinue, cta, tile
                , fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.metaDataArrPointer);




            grid.sync();
        }

    } while (isGoldPassToContinue[0] || isSegmPassToContinue[0]);


    //setting global iteration number to local one 
    if (blockIdx.x == 0) {
        if (threadIdx.x == 2 && threadIdx.y == 0) {
            fbArgs.metaData.minMaxes[13] = iterationNumb[0];
        }
    }
}



/*
get data from occupancy calculator API used to get optimal number of thread blocks and threads per thread block
*/
template <typename T>
inline occupancyCalcData getOccupancy() {

    occupancyCalcData res;

    int blockSize; // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)getMinMaxes<T>,
        0);
    res.warpsNumbForMinMax = blockSize / 32;
    res.blockSizeForMinMax = minGridSize;

    // for min maxes kernel 
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    res.warpsNumbForboolPrepareKernel = blockSize / 32;
    res.blockSizeFoboolPrepareKernel = minGridSize;
    // for first meta pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)boolPrepareKernel<T>,
        0);
    res.theadsForFirstMetaPass = blockSize;
    res.blockForFirstMetaPass = minGridSize;
    //for main pass kernel
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)mainPassKernel<T>,
        0);
    res.warpsNumbForMainPass = blockSize / 32;
    res.blockForMainPass = minGridSize;

    printf("warpsNumbForMainPass %d blockForMainPass %d  ", res.warpsNumbForMainPass, res.blockForMainPass);
    return res;
}













/*
TODO consider representing as a CUDA graph
executing Algorithm as CUDA graph  based on official documentation and
https://codingbyexample.com/2020/09/25/cuda-graph-usage/
*/
#pragma once
template <typename T>
ForBoolKernelArgs<T> executeHausdoffGraph(ForFullBoolPrepArgs<T> fFArgs, const int WIDTH, const int HEIGHT, const int DEPTH, occupancyCalcData occData) {

    // For Graph
    hipStream_t streamForGraph;
    hipGraph_t graph;
    std::vector<hipGraphNode_t> nodeDependencies;
    hipGraphNode_t memcpyNode, kernelNode;
    hipKernelNodeParams kernelNodeParams = { 0 };
    //  cudaMemcpyParams memcpyParams = { 0 };



    ForBoolKernelArgs<T> fbArgs = getArgsForKernel<T>(fFArgs, occData.warpsNumbForMainPass, occData.blockForMainPass, WIDTH, HEIGHT, DEPTH);

    checkCuda(hipDeviceSynchronize(), "a1");

    //getMinMaxes << <blockSizeForMinMax, dim3(32, warpsNumbForMinMax) >> > ( minMaxes);
    getMinMaxes << <occData.blockSizeForMinMax, dim3(32, occData.warpsNumbForMinMax) >> > (fbArgs, fbArgs.minMaxes, fbArgs.goldArr.arrP, fbArgs.segmArr.arrP, fbArgs.metaData);

    checkCuda(hipDeviceSynchronize(), "a1b");

    fbArgs.metaData = allocateMemoryAfterMinMaxesKernel(fbArgs, fFArgs);

    checkCuda(hipDeviceSynchronize(), "a2b");

    boolPrepareKernel << <occData.blockSizeFoboolPrepareKernel, dim3(32, occData.warpsNumbForboolPrepareKernel) >> > (
        fbArgs, fbArgs.metaData, fbArgs.origArrsPointer, fbArgs.metaDataArrPointer, fbArgs.goldArr.arrP, fbArgs.segmArr.arrP, fbArgs.minMaxes);

    checkCuda(hipDeviceSynchronize(), "a3");

    int fpPlusFn = allocateMemoryAfterBoolKernel(fbArgs, fFArgs);

    checkCuda(hipDeviceSynchronize(), "a4");


    firstMetaPrepareKernel << <occData.blockForFirstMetaPass, occData.theadsForFirstMetaPass >> > (fbArgs, fbArgs.metaData, fbArgs.minMaxes, fbArgs.workQueuePointer, fbArgs.origArrsPointer, fbArgs.metaDataArrPointer);

    checkCuda(hipDeviceSynchronize(), "a5");

    void* kernel_args[] = { &fbArgs };
    hipLaunchCooperativeKernel((void*)(mainPassKernel<int>), occData.blockForMainPass, dim3(32, occData.warpsNumbForMainPass), kernel_args);


    hipFreeAsync(fbArgs.resultListPointerMeta, 0);
    hipFreeAsync(fbArgs.resultListPointerLocal, 0);
    hipFreeAsync(fbArgs.resultListPointerIterNumb, 0);
    hipFreeAsync(fbArgs.workQueuePointer, 0);
    hipFreeAsync(fbArgs.origArrsPointer, 0);
    hipFreeAsync(fbArgs.metaDataArrPointer, 0);
    hipFreeAsync(fbArgs.mainArrAPointer, 0);
    hipFreeAsync(fbArgs.mainArrBPointer, 0);

    return fbArgs;

}



#pragma once
template <typename T>
ForBoolKernelArgs<T> mainKernelsRun(ForFullBoolPrepArgs<T> fFArgs, const int WIDTH, const int HEIGHT, const int DEPTH
) {

    //hipDeviceReset();
    hipError_t syncErr;
    hipError_t asyncErr;

    occupancyCalcData occData = getOccupancy<T>();

    //pointers ...


    ForBoolKernelArgs<T> fbArgs = executeHausdoffGraph(fFArgs, WIDTH, HEIGHT, DEPTH, occData);




    checkCuda(hipDeviceSynchronize(), "last ");

    /////////// error handling 
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));


    hipDeviceReset();

    return fbArgs;
}





inline void setArrCPUB(bool* arrCPU, int x, int y, int z, int  Nx, int Ny) {

    arrCPU[x + y * Nx + z * Nx * Ny] = true;
};






void loadHDFIntoBoolArr(H5std_string FILE_NAME, H5std_string DATASET_NAME, bool*& data) {




    H5::H5File file(FILE_NAME, H5F_ACC_RDONLY);
    H5::DataSet dset = file.openDataSet(DATASET_NAME);
    /*
     * Get the class of the datatype that is used by the dataset.
     */
    H5T_class_t type_class = dset.getTypeClass();
    H5::DataSpace dspace = dset.getSpace();
    int rank = dspace.getSimpleExtentNdims();


    hsize_t dims[2];
    rank = dspace.getSimpleExtentDims(dims, NULL); // rank = 1
    printf("Datasize: %d \n ", dims[0]); // this is the correct number of values

     // Define the memory dataspace
    hsize_t dimsm[1];
    dimsm[0] = dims[0];
    H5::DataSpace memspace(1, dimsm);

    data = (bool*)calloc(dims[0], sizeof(bool));

    dset.read(data, H5::PredType::NATIVE_HBOOL, memspace, dspace);

    file.close();

}





void benchmarkMitura(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT, const int DEPTH) {

    //// some preparations and configuring
    MetaDataCPU metaData;
    size_t size = sizeof(unsigned int) * 20;
    unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
    metaData.minMaxes = minMaxesCPU;

    ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
    forFullBoolPrepArgs.metaData = metaData;
    forFullBoolPrepArgs.numberToLookFor = true;
    forFullBoolPrepArgs.goldArr = get3dArrCPU(onlyBladderBoolFlat, WIDTH, HEIGHT, DEPTH);
    forFullBoolPrepArgs.segmArr = get3dArrCPU(onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);

    occupancyCalcData occData = getOccupancy<bool>();

    //pointers ...

    //function invocation
    auto begin = std::chrono::high_resolution_clock::now();

    // ForBoolKernelArgs<bool> fbArgs = executeHausdoff(forFullBoolPrepArgs, WIDTH, HEIGHT, DEPTH, occData);

    ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, WIDTH, HEIGHT, DEPTH);
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;


    size_t sizeMinMax = sizeof(unsigned int) * 20;
    hipMemcpy(minMaxesCPU, fbArgs.metaData.minMaxes, sizeMinMax, hipMemcpyDeviceToHost);

    printf("HD: %d \n", minMaxesCPU[13]);


    // freeee
    free(onlyBladderBoolFlat);
    free(onlyLungsBoolFlat);




}







typedef unsigned char uchar;
typedef unsigned int uint;
#pragma once
class Volume {

private:
    bool* volume;
    int width, height, depth;
    int getLinearIndex(int x, int y, int z);
public:
    bool getVoxelValue(int x, int y, int z);
    bool getPixelValue(int x, int y);
    uint getWidth();
    uint getHeight();
    uint getDepth();
    bool* getVolume();
    void setVoxelValue(bool value, int x, int y, int z);
    void setPixelValue(bool value, int x, int y);
    Volume(int width, int height, int depth);
    Volume(int width, int height);
    void dispose();

};




#define CUDA_DEVICE_INDEX 0 //setting the index of your CUDA device

#define IS_3D 1 //setting this to 0 would grant a very slightly improvement on the performance if working with images only
#define CHEBYSHEV 0 //if not set to 1, then this algorithm would use an Euclidean-like metric, it is just an approximation. 
//It can be changed according to the structuring element
#pragma once
class HausdorffDistance {

private:
    void print(hipError_t error, char* msg);

public:
    int computeDistance(Volume* img1, Volume* img2);

};


inline Volume::Volume(const int width, const int height, const int depth) {
    this->width = width; this->height = height; this->depth = depth;
    volume = (bool*)calloc(width * height * depth, sizeof(bool));
}

#pragma once
inline Volume::Volume(const int width, const int height) {
    this->width = width; this->height = height; this->depth = 1;
    volume = (bool*)calloc(width * height * depth, sizeof(bool));
}
#pragma once
inline int Volume::getLinearIndex(const int x, const int y, const int z) {
    const int a = 1, b = width, c = (width) * (height);
    return a * x + b * y + c * z;
}

inline uint Volume::getWidth() { return this->width; }
inline uint Volume::getHeight() { return this->height; }
inline uint Volume::getDepth() { return this->depth; }
inline bool* Volume::getVolume() { return this->volume; }
inline bool Volume::getPixelValue(int x, int y) { return this->volume[getLinearIndex(x, y, 0)]; }
#pragma once
inline bool Volume::getVoxelValue(int x, int y, int z) {
    return volume[getLinearIndex(x, y, z)];
}
#pragma once
inline void Volume::setPixelValue(bool value, const int x, const int y) {
    volume[getLinearIndex(x, y, 0)] = value;
}
#pragma once
inline void Volume::setVoxelValue(bool value, const int x, const int y, const int z) {
    volume[getLinearIndex(x, y, z)] = value;
}
#pragma once
inline void Volume::dispose() {
    free(volume);
}

typedef unsigned char uchar;
typedef unsigned int uint;

#pragma once
__device__ int finished; //global variable that contains a boolean which indicates when to stop the kernel processing
#pragma once
__constant__ __device__ int WIDTH, HEIGHT, DEPTH; //constant variables that contain the size of the volume


#pragma once
__global__ void dilate(const bool* IMG1, const bool* IMG2, const bool* img1Read, const bool* img2Read,
    bool* img1Write, bool* img2Write) {

    const int id = blockDim.x * blockIdx.x + threadIdx.x;
#if !IS_3D
    const int x = id % WIDTH, y = id / WIDTH;
#else
    const int x = id % WIDTH, y = (id / WIDTH) % HEIGHT, z = (id / WIDTH) / HEIGHT;
#endif

    if (id < WIDTH * HEIGHT * DEPTH) {


        if (img1Read[id]) {
            if (x + 1 < WIDTH) img1Write[id + 1] = true;
            if (x - 1 >= 0) img1Write[id - 1] = true;
            if (y + 1 < HEIGHT) img1Write[id + WIDTH] = true;
            if (y - 1 >= 0) img1Write[id - WIDTH] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH) img1Write[id + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0) img1Write[id - WIDTH * HEIGHT] = true;
#endif

#if CHEBYSHEV
            //diagonals
            if (x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1] = true;
            if (x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1] = true;
            if (x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1] = true;
            if (x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img1Write[id - WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img1Write[id - WIDTH - 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img1Write[id + WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img1Write[id + WIDTH - 1 - WIDTH * HEIGHT] = true;
#endif
#endif
        }


        if (img2Read[id]) {
            if (x + 1 < WIDTH) img2Write[id + 1] = true;
            if (x - 1 >= 0) img2Write[id - 1] = true;
            if (y + 1 < HEIGHT) img2Write[id + WIDTH] = true;
            if (y - 1 >= 0) img2Write[id - WIDTH] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH) img2Write[id + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0) img2Write[id - WIDTH * HEIGHT] = true;
#endif

#if CHEBYSHEV
            //diagonals
            if (x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1] = true;
            if (x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1] = true;
            if (x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1] = true;
            if (x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1] = true;
#if IS_3D //if working with 3d volumes, then the 3D part
            if (z + 1 < DEPTH && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 + WIDTH * HEIGHT] = true;
            if (z + 1 < DEPTH && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 + WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y - 1 >= 0) img2Write[id - WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y - 1 >= 0) img2Write[id - WIDTH - 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x + 1 < WIDTH && y + 1 < HEIGHT) img2Write[id + WIDTH + 1 - WIDTH * HEIGHT] = true;
            if (z - 1 >= 0 && x - 1 >= 0 && y + 1 < HEIGHT) img2Write[id + WIDTH - 1 - WIDTH * HEIGHT] = true;
#endif
#endif
        }


        //this is an atomic and computed to the finished global variable, if image 1 contains all of image 2 and image 2 contains all pixels of
        //image 1 then finished is true
        atomicAnd(&finished, (img2Read[id] || !IMG1[id]) && (img1Read[id] || !IMG2[id]));
    }
}

#pragma once
int HausdorffDistance::computeDistance(Volume* img1, Volume* img2) {

    const int height = (*img1).getHeight(), width = (*img1).getWidth(), depth = (*img1).getDepth();

    size_t size = width * height * depth * sizeof(bool);

    //getting details of your CUDA device
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, CUDA_DEVICE_INDEX); //device index = 0, you can change it if you have more CUDA devices
    const int threadsPerBlock = props.maxThreadsPerBlock / 2;
    const int blocksPerGrid = (height * width * depth + threadsPerBlock - 1) / threadsPerBlock;


    //copying the dimensions to the GPU
    hipMemcpyToSymbolAsync(HIP_SYMBOL(WIDTH), &width, sizeof(width));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(HEIGHT), &height, sizeof(height));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(DEPTH), &depth, sizeof(depth));


    //allocating the input images on the GPU
    bool* d_img1, * d_img2;
    hipMalloc(&d_img1, size);
    hipMalloc(&d_img2, size);


    //copying the data to the allocated memory on the GPU
    hipMemcpyAsync(d_img1, (*img1).getVolume(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_img2, (*img2).getVolume(), size, hipMemcpyHostToDevice);


    //allocating the images that will be the processing ones
    bool* d_img1Write, * d_img1Read, * d_img2Write, * d_img2Read;
    hipMalloc(&d_img1Write, size); hipMalloc(&d_img1Read, size);
    hipMalloc(&d_img2Write, size); hipMalloc(&d_img2Read, size);


    //cloning the input images to these two image versions (write and read)
    hipMemcpyAsync(d_img1Read, d_img1, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img2Read, d_img2, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img1Write, d_img1, size, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(d_img2Write, d_img2, size, hipMemcpyDeviceToDevice);



    //required variables to compute the distance
    int h_finished = false, t = true;
    int distance = -1;

    //where the magic happens
    while (!h_finished) {
        //reset the bool variable that verifies if the processing ended
        hipMemcpyToSymbol(HIP_SYMBOL(finished), &t, sizeof(h_finished));


        //lauching the verify kernel, which verifies if the processing finished
        dilate << < blocksPerGrid, threadsPerBlock >> > (d_img1, d_img2, d_img1Read, d_img2Read, d_img1Write, d_img2Write);

        //hipDeviceSynchronize();

        //updating the imgRead (cloning imgWrite to imgRead)
        hipMemcpy(d_img1Read, d_img1Write, size, hipMemcpyDeviceToDevice);
        hipMemcpy(d_img2Read, d_img2Write, size, hipMemcpyDeviceToDevice);



        //copying the result back to host memory
        hipMemcpyFromSymbol(&h_finished, HIP_SYMBOL(finished), sizeof(h_finished));


        //incrementing the distance at each iteration
        distance++;
    }


    //freeing memory
    hipFree(d_img1); hipFree(d_img2);
    hipFree(d_img1Write); hipFree(d_img1Read);
    hipFree(d_img2Write); hipFree(d_img2Read);

    //resetting device
    hipDeviceReset();

    print(hipGetLastError(), "processing CUDA. Something may be wrong with your CUDA device.");

    return distance;

}
#pragma once
inline void HausdorffDistance::print(hipError_t error, char* msg) {
    if (error != hipSuccess)
    {
        printf("Error on %s ", msg);
        fprintf(stderr, "Error code: %s!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}



/*
benchmark for original code from  https://github.com/Oyatsumi/HausdorffDistanceComparison
*/
void benchmarkOliviera(bool* onlyBladderBoolFlat, bool* onlyLungsBoolFlat, const int WIDTH, const int HEIGHT
    , const int DEPTH) {
    Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);

    for (int x = 0; x < WIDTH; x++) {
        for (int y = 0; y < HEIGHT; y++) {
            for (int z = 0; z < DEPTH; z++) {
                img1.setVoxelValue(onlyLungsBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
                img2.setVoxelValue(onlyBladderBoolFlat[x + y * WIDTH + z * WIDTH * HEIGHT], x, y, z);
            }
        }
    }

    auto begin = std::chrono::high_resolution_clock::now();
    HausdorffDistance* hd = new HausdorffDistance();
    int dist = (*hd).computeDistance(&img1, &img2);
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "Total elapsed time: ";
    std::cout << (double)(std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / (double)1000000000) << "s" << std::endl;

    printf("HD: %d \n", dist);

    //freeing memory
    img1.dispose(); img2.dispose();

    //Datasize: 216530944
   //Datasize : 216530944
    //Total elapsed time : 2.62191s
    //HD : 234

}

void loadHDF() {
    const int WIDTH = 512;
    const int HEIGHT = 512;
    //    const int DEPTH = 536;

    int DEPTH = 826;

    const H5std_string FILE_NAMEonlyLungsBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");

    const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlat");
    //const H5std_string DATASET_NAMEonlyLungsBoolFlat("onlyLungsBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlat, DATASET_NAMEonlyLungsBoolFlat, onlyLungsBoolFlat);

    const H5std_string FILE_NAMEonlyBladderBoolFlat("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");


    const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlat");
    //const H5std_string DATASET_NAMEonlyBladderBoolFlat("onlyBladderBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlat;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlat, onlyBladderBoolFlat);
    //onlyBladderBoolFlat = (bool*)calloc(WIDTH* HEIGHT* DEPTH, sizeof(bool));

    //onlyBladderBoolFlat[0] = true;

    // benchmarkOliviera(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);//125 

    benchmarkMitura(onlyBladderBoolFlat, onlyLungsBoolFlat, WIDTH, HEIGHT, DEPTH);//124 or 259


    DEPTH = 536;
    const H5std_string FILE_NAMEonlyLungsBoolFlatB("D:\\dataSets\\forMainHDF5\\forHausdorffTests.hdf5");

    const H5std_string DATASET_NAMEonlyLungsBoolFlatB("onlyLungsBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyLungsBoolFlatB;
    loadHDFIntoBoolArr(FILE_NAMEonlyLungsBoolFlatB, DATASET_NAMEonlyLungsBoolFlatB, onlyLungsBoolFlatB);


    const H5std_string DATASET_NAMEonlyBladderBoolFlatB("onlyBladderBoolFlatB");
    // create a vector the same size as the dataset
    bool* onlyBladderBoolFlatB;
    loadHDFIntoBoolArr(FILE_NAMEonlyBladderBoolFlat, DATASET_NAMEonlyBladderBoolFlatB, onlyBladderBoolFlatB);

    //benchmarkOliviera(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH);//125 
    benchmarkMitura(onlyBladderBoolFlatB, onlyLungsBoolFlatB, WIDTH, HEIGHT, DEPTH);//124 or 259

}











int main(void) {

    //  const int WIDTH = atoi(argv[1]), HEIGHT = WIDTH, DEPTH = 1;
   //   Volume img1 = Volume(WIDTH, HEIGHT, DEPTH), img2 = Volume(WIDTH, HEIGHT, DEPTH);
   // testMainPasswes();
    loadHDF();



    return 0;  // successfully terminated
}



