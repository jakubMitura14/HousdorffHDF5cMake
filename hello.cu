#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>

#include ""

// includes, system
#include <iostream>     // std::cout
#include <algorithm>    // std::min
//#include <hip/hip_runtime_api.h>
#include <cmath>
//#include "Structs.cu"
#include <math.h>
//#include "MemoryTransfers.cu"
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

//#include "BoolKernelTests.cu"
#include "testAll.cu"
using namespace cooperative_groups;

#include <iostream>
#include <string>
#include <vector>
#include <H5Cpp.h>
using namespace H5;


#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif



using std::cout;
using std::endl;

#ifdef OLD_HEADER_FILENAME
#include <iostream.h>
#else
#include <iostream>
#endif
using std::cout;
using std::endl;
#include <string>
#include "H5Cpp.h"
using namespace H5;
const H5std_string FILE_NAME("C:\\Users\\1\\PycharmProjects\\pythonProject3\\mytestfile.hdf5");
const H5std_string DATASET_NAME("onlyLungs");
//const int    NX_SUB = 3;    // hyperslab dimensions
//const int    NY_SUB = 4;
//const int    NX = 7;        // output buffer dimensions
//const int    NY = 7;
//const int    NZ = 3;
//const int    RANK_OUT = 3;



__device__ void compute(int* global_out, int const* shared_in) {



};
__global__ void with_staging(int* global_out, int const* global_in, size_t size,
    size_t batch_sz) {
    auto grid = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();
    assert(size == batch_sz * grid.size()); // Assume input size fits batch_sz *  grid_size
        constexpr size_t stages_count = 2; // Pipeline with two stages
        // Two batches must fit in shared memory:
    extern __shared__ int shared[]; // stages_count * block.size() * sizeof(int)     bytes
        size_t shared_offset[stages_count] = { 0, block.size() }; // Offsets to each    batch
        // Allocate shared storage for a two-stage cuda::pipeline:
        __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
        > shared_state;
    auto pipeline = cuda::make_pipeline(block, &shared_state);
    // Each thread processes `batch_sz` elements.
    // Compute offset of the batch `batch` of this thread block in global memory:
    auto block_batch = [&](size_t batch) -> int {
        return block.group_index().x * block.size() + grid.size() * batch;
    };
    // Initialize first pipeline stage by submitting a `memcpy_async` to fetch a
    //whole batch for the block :
    if (batch_sz == 0) return;
    pipeline.producer_acquire();
    cuda::memcpy_async(block, shared + shared_offset[0], global_in +   block_batch(0), sizeof(int) * block.size(), pipeline);
    pipeline.producer_commit();
    // Pipelined copy/compute:
    for (size_t batch = 1; batch < batch_sz; ++batch) {
        // Stage indices for the compute and copy stages:
        size_t compute_stage_idx = (batch - 1) % 2;
        size_t copy_stage_idx = batch % 2;
        size_t global_idx = block_batch(batch);
        // Collectively acquire the pipeline head stage from all producer threads:
        pipeline.producer_acquire();
        // Submit async copies to the pipeline's head stage to be
        // computed in the next loop iteration
        cuda::memcpy_async(block, shared + shared_offset[copy_stage_idx], global_in     + global_idx, sizeof(int) * block.size(), pipeline);
        // Collectively commit (advance) the pipeline's head stage
        pipeline.producer_commit();
        // Collectively wait for the operations commited to the
        // previous `compute` stage to complete:
        pipeline.consumer_wait();
        // Computation overlapped with the memcpy_async of the "copy" stage:
        compute(global_out + global_idx, shared + shared_offset[compute_stage_idx]);
        // Collectively release the stage resources
        pipeline.consumer_release();
    }
    // Compute the data fetch by the last iteration
    pipeline.consumer_wait();
    compute(global_out + block_batch(batch_sz - 1), shared + shared_offset[(batch_sz -
        1) % 2]);
    pipeline.consumer_release();
}






int main(void){

  




   // testMainPasswes();


        ///*
        // * Open the specified file and the specified dataset in the file.
        // */
        //H5File file(FILE_NAME, H5F_ACC_RDONLY);
        //DataSet dataset = file.openDataSet(DATASET_NAME);
        ///*
        // * Get the class of the datatype that is used by the dataset.
        // */
        //H5T_class_t type_class = dataset.getTypeClass();
        //DataSpace dataspace = dataset.getSpace();
        //int rank = dataspace.getSimpleExtentNdims();
        ///*
        // * Get the dimension size of each dimension in the dataspace and
        // * display them.
        // */
        //hsize_t dims_out[3];
        //int ndims = dataspace.getSimpleExtentDims(dims_out, NULL);
        //cout << "rank " << rank << ", dimensions " <<
        //    (unsigned long)(dims_out[0]) << " x " <<
        //    (unsigned long)(dims_out[1]) << 
        //    (unsigned long)(dims_out[2]) << endl;


        ///*
        // * Get class of datatype and print message if it's an integer.
        // */
        //if (type_class == H5T_INTEGER)
        //{
        //    cout << "Data set has INTEGER type" << endl;
        //    /*
        // * Get the integer datatype
        //     */
        //    IntType intype = dataset.getIntType();
        //    /*
        //     * Get order of datatype and print message if it's a little endian.
        //     */
        //    H5std_string order_string;
        //    H5T_order_t order = intype.getOrder(order_string);
        //    cout << order_string << endl;
        //    /*
        //     * Get size of the data element stored in file and print it.
        //     */
        //    size_t size = intype.getSize();
        //    cout << "Data size is " << size << endl;
        //}








        //hsize_t memdim = dims_out[0] * dims_out[1] * dims_out[2];;

        //std::vector<float> data_out(memdim);






        //use the same layout for file and memory
        //dataset.read(data_out.data(), PredType::NATIVE_INT64, dataspace, dataspace);


        ///*
        // * Define hyperslab in the dataset; implicitly giving strike and
        // * block NULL.
        // */
        //hsize_t      offset[2];   // hyperslab offset in the file
        //hsize_t      count[2];    // size of the hyperslab in the file
        //offset[0] = 1;
        //offset[1] = 2;
        //count[0] = NX_SUB;
        //count[1] = NY_SUB;
        //dataspace.selectHyperslab(H5S_SELECT_SET, count, offset);
        ///*
        // * Define the memory dataspace.
        // */
        //hsize_t     dimsm[3];              /* memory space dimensions */
        //dimsm[0] = NX;
        //dimsm[1] = NY;
        //dimsm[2] = NZ;
        //DataSpace memspace(RANK_OUT, dimsm);
        ///*
        // * Define memory hyperslab.
        // */
        //hsize_t      offset_out[3];   // hyperslab offset in memory
        //hsize_t      count_out[3];    // size of the hyperslab in memory
        //offset_out[0] = 3;
        //offset_out[1] = 0;
        //offset_out[2] = 0;
        //count_out[0] = NX_SUB;
        //count_out[1] = NY_SUB;
        //count_out[2] = 1;
        //memspace.selectHyperslab(H5S_SELECT_SET, count_out, offset_out);
        ///*
        // * Read data from hyperslab in the file into the hyperslab in
        // * memory and display the data.
        // */
        //dataset.read(data_out, PredType::NATIVE_INT, memspace, dataspace);
        //for (j = 0; j < NX; j++)
        //{
        //    for (i = 0; i < NY; i++)
        //        cout << data_out[j][i][0] << " ";
        //    cout << endl;
        //}
        /*
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 0 0 0 0 0 0 0
         * 3 4 5 6 0 0 0
         * 4 5 6 7 0 0 0
         * 5 6 7 8 0 0 0
         * 0 0 0 0 0 0 0
         */
 



    return 0;  // successfully terminated
}