#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>

using namespace cooperative_groups;

/*
given appropriate hipPitchedPtr and ForFullBoolPrepArgs will return ForBoolKernelArgs
*/
#pragma once
template <typename TCC>
inline ForBoolKernelArgs<TCC> getArgsForKernel(ForFullBoolPrepArgs<TCC> mainFunArgs, array3dWithDimsGPU forDebugArr
    , array3dWithDimsGPU goldArr
    , array3dWithDimsGPU segmArr
    ,unsigned int*& minMaxes
    ,int warpsNumbForMainPass,int blockForMainPass
) {

    ForBoolKernelArgs<TCC> res;
    res.metaData = allocateMetaDataOnGPU(mainFunArgs.metaData, minMaxes);
    res.forDebugArr = forDebugArr;
    res.goldArr = goldArr;
    res.segmArr = segmArr;

    res.numberToLookFor = mainFunArgs.numberToLookFor;
    res.dbXLength = 32;
    res.dbYLength = warpsNumbForMainPass;
    res.dbZLength = 32;



    return res;
}


/*
setting the linear index of metadata blocks that are in given direction if there is no such (out of range) we will save it as UINT32_MAX
*/
template <typename TCC>
__device__ inline void setNeighbourBlocks(ForBoolKernelArgs<TCC> fbArgs,uint8_t idX, uint8_t inArrIndex, bool predicate, uint32_t toAdd
    , uint32_t linIdexMeta , MetaDataGPU metaData, uint32_t localBlockMetaData[20]) {

    if ((threadIdx.x == idX) && (threadIdx.y == 0)) {
        if (predicate) {
            localBlockMetaData[inArrIndex] = (linIdexMeta + toAdd);
        }
        else {
            localBlockMetaData[inArrIndex] = isGoldOffset;
        }
    };
}





/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYU>
__device__ void metaDataIter(ForBoolKernelArgs<TYU> fbArgs
    , MetaDataGPU metaData, uint32_t* origArrs, uint32_t* metaDataArr) {

    ////////////some initializations
    bool goldBool = false;
    bool segmBool = false;
    bool isNotEmpty = false;
  
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);
    uint32_t sumFp = 0;
    uint32_t sumFn = 0;
   
    auto pipeline = cuda::make_pipeline();


    //shared memory

    //TODO() make it dynamically sized 
    __shared__ uint32_t sharedForGold[1024];
    __shared__ uint32_t sharedForSegm[1024];


    //for storing fp and fn sums to later accumulate it to global values
    __shared__ uint32_t fpSFnS[2];
    __shared__ uint32_t localBlockMetaData[20];

    __shared__ bool anyInGold[1];
    __shared__ bool anyInSegm[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ int minMaxesInShmem[7];



    if ((threadIdx.x == 1) && (threadIdx.y == 1)) { fpSFnS[0] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 1)) { fpSFnS[1] = 0; };
    if ((threadIdx.x == 3) && (threadIdx.y == 1)) { anyInGold[1] = false; };
    if ((threadIdx.x == 4) && (threadIdx.y == 1)) { anyInSegm[1] = false; };

    //we need to load also min and maxes of metdata 
    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
    if (cta.thread_rank() == 0) {
        init(&barrier, cta.size()); // Friend function initializes barrier
    }


    sync(cta);

    /////////////////////////


    //main metadata iteration
    for (uint32_t linIdexMeta = blockIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (metaData.metaXLength * metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength));
        //reset
        isNotEmpty = false;
        sumFp = 0;
        sumFn = 0;
        anyInGold[0] = false;
        anyInSegm[0] = false;
        //iterating over data block
        sync(cta);
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint32_t x = (xMeta+ metaData.minX)* fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint32_t  y = (yMeta+ metaData.minY) * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                    // resetting 
                    sharedForGold[xLoc + yLoc * fbArgs.dbXLength] = 0;
                    sharedForSegm[xLoc + yLoc * fbArgs.dbXLength] = 0;
        

                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint32_t z = (zMeta+ metaData.minZ)* fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nx) {
                            char* tensorslice;

                            //first array gold
                            goldBool = (getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);

                            // now segmentation  array
                            segmBool = (getTensorRow<TYU>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);
                            // setting bits
                            sharedForGold[xLoc+yLoc* fbArgs.dbXLength] |= goldBool << zLoc;
                            sharedForSegm[xLoc+yLoc* fbArgs.dbXLength] |= segmBool << zLoc;
                            // setting value of local boolean marking that any of the entries was evaluated to true in either of arrays
                            isNotEmpty = (isNotEmpty || (goldBool || segmBool));
                            sumFp += (!goldBool && segmBool);
                            sumFn += (goldBool && !segmBool);
                            if (goldBool)  anyInGold[0] = true;
                            if (segmBool)  anyInSegm[0] = true;
                           
                            //if (goldBool) {
                            //    printf("in kernel  gold x %d y %d z %d linearLocal %d linIdexMeta %d\n", x, y, z, xLoc + yLoc * fbArgs.dbXLength, linIdexMeta);
                            //}

                            //if (segmBool) {
                            //    printf("in kernel  segm  x %d y %d z %d linearLocal %d linIdexMeta %d\n", x, y, z, xLoc + yLoc * fbArgs.dbXLength, linIdexMeta);
                            //}


                        }
                    }
                }

                //if (sharedForGold[xLoc + yLoc * fbArgs.dbXLength] > 0) {
                //    printf("in kernel Metax %d yMeta %d zMeta %d linearLocal %d linIdexMeta %d column %d \n"
                //        , xMeta, yMeta, zMeta,  xLoc + yLoc * fbArgs.dbXLength, linIdexMeta
                //    , sharedForGold[xLoc + yLoc * fbArgs.dbXLength]);
                //}


            }
        }
        //reset local metadata
        if ((threadIdx.x <20) && (threadIdx.y == 0)) {
            localBlockMetaData[threadIdx.x]=0;
        }
        

    
        isNotEmpty = __syncthreads_or(isNotEmpty);
        //exporting to global memory
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint32_t x = (xMeta + metaData.minX) * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint32_t  y = (yMeta + metaData.minY) * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {
                    origArrs[linIdexMeta * metaData.mainArrSectionLength + yLoc * 32 + xLoc] = sharedForGold[yLoc * 32 + xLoc];
                    origArrs[linIdexMeta * metaData.mainArrSectionLength + yLoc * 32 + xLoc + metaData.mainArrXLength] = sharedForSegm[yLoc * 32 + xLoc];


                }
            }
        }

     //   sync(cta);

        //copy data to global memory from shmem

        //mainArr[linIdexMeta * metaData.mainArrSectionLength + threadIdx.x + threadIdx.y * metaData.metaXLength] = sharedForGold[threadIdx.x + threadIdx.y * metaData.metaXLength];
        //cooperative_groups::memcpy_async(cta, (mainArr), (sharedForGold), (sizeof(uint32_t) *2) );
       

        //cuda::memcpy_async(cta, (&origArrs[linIdexMeta * metaData.mainArrSectionLength]) , (sharedForGold), sizeof(uint32_t) * cta.size(), barrier);
        //barrier.arrive_and_wait(); // Waits for all copies to complete

    
       // cuda::memcpy_async(cta, (&origArrs[linIdexMeta * metaData.mainArrSectionLength]), (sharedForGoldB), (sizeof(uint32_t) * blockDim.x * blockDim.y), barrier);
       //barrier.arrive_and_wait(); // Waits for all copies to complete

       //cuda::memcpy_async(cta, (&origArrs[linIdexMeta * metaData.mainArrSectionLength + metaData.mainArrXLength]), (sharedForSegmB), (sizeof(uint32_t) * blockDim.x * blockDim.y), barrier);
       //barrier.arrive_and_wait(); // Waits for all copies to complete

       //cuda::memcpy_async(cta, (&mainArr[linIdexMeta * metaData.mainArrSectionLength ]), (sharedForGoldB), (sizeof(uint32_t) * blockDim.x * blockDim.y), barrier);
       // barrier.arrive_and_wait(); // Waits for all copies to complete

       // cuda::memcpy_async(cta, (&mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.mainArrXLength*1]), (sharedForSegmB), (sizeof(uint32_t) * blockDim.x * blockDim.y) , barrier);
       // barrier.arrive_and_wait(); // Waits for all copies to complete

       // cuda::memcpy_async(cta, (&mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.mainArrXLength*2]), (sharedForGoldB), (sizeof(uint32_t) * blockDim.x * blockDim.y), barrier);
       // barrier.arrive_and_wait(); // Waits for all copies to complete

       // cuda::memcpy_async(cta, (&mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.mainArrXLength*3]), (sharedForSegmB), (sizeof(uint32_t) * blockDim.x * blockDim.y), barrier);
       // barrier.arrive_and_wait(); // Waits for all copies to complete

       sync(cta);



        /////adding the block and total number of the Fp's and Fn's 
        sumFp = reduce(tile, sumFp, plus<uint32_t>());
        sumFn = reduce(tile, sumFn, plus<uint32_t>());
        //reusing shared memory and adding accumulated values from tiles
        if (tile.thread_rank() == 0) {
            sharedForGold[tile.meta_group_rank()] = sumFp;
            sharedForSegm[tile.meta_group_rank()] = sumFn;
        }
        sync(cta);//waiting so shared memory will be loaded evrywhere
        //on single thread we do last sum reduction
        auto active = coalesced_threads();

        //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        //    printf("xMeta %d yMeta %d zMeta %d \n", xMeta, yMeta, zMeta);
        //}

        if ((threadIdx.x == 0) && (threadIdx.y == 0) && isNotEmpty) {
            sharedForGold[33] = 0;//reset
            for (int i = 0; i < tile.meta_group_size(); i += 1) {
                sharedForGold[33] += sharedForGold[i];
 /*               if (sharedForGold[i]>0) {
                    printf("adding sharedForGold[i] %d in gold \n ", sharedForGold[i]);
                }*/

            };
            fpSFnS[0] += sharedForGold[33];// will be needed later for global set
            //metaDataArr[linIdexMeta * metaData.metaDataSectionLength + 1] = sharedForGold[33];
            localBlockMetaData[1] = sharedForGold[33];

           // getTensorRow<unsigned int>(tensorslice, metaData.fpCount, metaData.fpCount.Ny, yMeta, zMeta)[xMeta] = sharedForGold[1][0];
        }
       // if (isToBeExecutedOnActive(active, 1) && isNotEmpty) {
        if ((threadIdx.x == 0) && (threadIdx.y == 1) && isNotEmpty) {


            sharedForSegm[33] = 0;//reset
            for (int i = 0; i < tile.meta_group_size(); i += 1) {
                sharedForSegm[33] += sharedForSegm[i];
            };
            fpSFnS[1] += sharedForSegm[33];// will be needed later for global set
            //setting metadata
            localBlockMetaData[2] = sharedForSegm[33];


           // getTensorRow<unsigned int>(tensorslice, metaData.fnCount, metaData.fnCount.Ny, yMeta, zMeta)[xMeta] = sharedForSegm[1][0];

        }

        //marking as active 
//FP pass
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && isNotEmpty && anyInGold[0]) { 
            localBlockMetaData[7] = 1;
           // printf("in bool kernel mark fp as sctive linIdexMeta %d in index  %d \n  ", linIdexMeta);

        };
        //FN pass
        if ((threadIdx.x == 1) && (threadIdx.y == 0) && isNotEmpty && anyInSegm[0]) {
            //printf("in bool kernel mark fn as sctive linIdexMeta %d in index  %d \n  ", linIdexMeta);
            localBlockMetaData[9] = 1;

        };


        //after we streamed over all block we save also information about indicies of the surrounding blocks - given they are in range if not UINT32_MAX will be saved 
        //top



        setNeighbourBlocks(fbArgs, 3, 13, (zMeta > 0), (-(metaData.metaXLength * metaData.MetaYLength)), linIdexMeta, metaData, localBlockMetaData);//top
        setNeighbourBlocks(fbArgs, 4, 14, (zMeta < (metaData.MetaZLength - 1)), (metaData.metaXLength* metaData.MetaYLength), linIdexMeta, metaData, localBlockMetaData);//bottom

        setNeighbourBlocks(fbArgs, 6 ,15, (xMeta > 0), (-1), linIdexMeta, metaData, localBlockMetaData);//left
        setNeighbourBlocks(fbArgs, 7, 16, (xMeta < (metaData.metaXLength - 1)), 1, linIdexMeta, metaData, localBlockMetaData);//right

        setNeighbourBlocks(fbArgs, 8, 17, (yMeta < (metaData.MetaYLength - 1)), metaData.metaXLength, linIdexMeta, metaData, localBlockMetaData);//anterior
        setNeighbourBlocks(fbArgs, 9, 18, (yMeta > 0), (-metaData.metaXLength), linIdexMeta, metaData, localBlockMetaData);//posterior

  if ((threadIdx.x <20) && (threadIdx.y == 0)) {
metaDataArr[linIdexMeta * metaData.metaDataSectionLength+ threadIdx.x]= localBlockMetaData[threadIdx.x];
    };

        sync(cta); // just to reduce the warp divergence
        
        // copy metadata to global memory

        //cuda::memcpy_async(cta, &metaDataArr[linIdexMeta * metaData.metaDataSectionLength], (&localBlockMetaData[0]), (sizeof(uint32_t) * 20), barrier);
       // barrier.arrive_and_wait(); // Waits for all copies to complete

    }
    sync(cta);


    //setting global fp and fn
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
      /*  printf("metaData.totalMetaLength %d metaData.mainArrSectionLength %d metaData.metaXLength %d \n"
            , metaData.totalMetaLength, metaData.mainArrSectionLength, metaData.metaXLength);*/

        atomicAdd(&(metaData.minMaxes[7]), fpSFnS[0]);
    };

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
          atomicAdd(&(metaData.minMaxes[8]), fpSFnS[1]);

    };
   



}



/*
collecting all needed functions for GPU execution to prepare data from calculating Housedorff distance
*/
#pragma once
template <typename TYO>
__global__ void boolPrepareKernel(ForBoolKernelArgs<TYO> fbArgs
    , MetaDataGPU metaData, uint32_t* origArrs, uint32_t* metaDataArr) {
    metaDataIter(fbArgs,  metaData, origArrs, metaDataArr);
}

