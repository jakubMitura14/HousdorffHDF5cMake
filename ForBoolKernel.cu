#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;

/*
given appropriate hipPitchedPtr and ForFullBoolPrepArgs will return ForBoolKernelArgs
*/
#pragma once
template <typename TCC>
inline ForBoolKernelArgs<TCC> getArgsForKernel(ForFullBoolPrepArgs<int> mainFunArgs, array3dWithDimsGPU forDebugArr
    , array3dWithDimsGPU goldArr
    , array3dWithDimsGPU segmArr
) {

    ForBoolKernelArgs<TCC> res;
    MetaDataGPU resMeta;



    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 1, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 2, 0, 0, 1000, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 3, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 4, 0, 0, 1000, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 5, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 6, 0, 0, 1000, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 7, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 8, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 9, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 10, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 11, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 12, 0, 0, 1, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 13, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 14, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 15, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 16, 0, 0, 0, false);
    setArrCPU<unsigned int>(metaDataCPU.minMaxes, 17, 0, 0, 0, false);


    resMeta.minMaxes = allocate3dInGPU(metaDataCPU.minMaxes);

    res.metaData = resMeta;
    res.forDebugArr = forDebugArr;
    res.goldArr = goldArr;
    res.segmArr = segmArr;
    //allocate the reduced arrays
    //res.reducedGold = reducedGold;
    //res.reducedSegm = reducedSegm;
    //res.reducedGoldPrev = reducedGoldPrev;
    //res.reducedSegmPrev = reducedSegmPrev;

    //res.reducedGoldRef = reducedGoldRef;
    //res.reducedSegmRef = reducedSegmRef;

    res.numberToLookFor = mainFunArgs.numberToLookFor;
    res.dbXLength = mainFunArgs.dbXLength;
    res.dbYLength = mainFunArgs.dbYLength;
    res.dbZLength = mainFunArgs.dbZLength;



    return res;
}







/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYU>
__device__ void metaDataIter(ForBoolKernelArgs<TYU> fbArgs) {

    ////////////some initializations
    bool goldBool = false;
    bool segmBool = false;
    bool isNotEmpty = false;
    bool& goldBoolRef = goldBool;
    bool& segmBoolRef = segmBool;
    bool& isNotEmptyRef = isNotEmpty;
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);

    char* tensorslice;
    uint16_t sumFp = 0;
    uint16_t sumFn = 0;

    //shared memory
    
    //TODO() make it dynamically sized 
    __shared__ uint32_t sharedForGold[32][32];
    __shared__ uint32_t sharedForSegm[32][32];
    //for storing fp and fn sums to later accumulate it to global values
    __shared__ uint32_t fpSFnS[2];

    __shared__ bool anyInGold[1];
    __shared__ bool anyInSegm[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ int minMaxesInShmem[7];
    if ((threadIdx.x == 1) && (threadIdx.y == 1)) { fpSFnS[0] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 1)) { fpSFnS[1] = 0; };
    if ((threadIdx.x == 3) && (threadIdx.y == 1)) { anyInGold[1] = false; };
    if ((threadIdx.x == 4) && (threadIdx.y == 1)) { anyInSegm[1] = false; };

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //iterating over data block
        //now we need to iterate over the data in the data block voxel by voxel
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint16_t x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint16_t y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                    // resetting 
                    sharedForGold[xLoc][yLoc] = 0;
                    sharedForSegm[xLoc][yLoc] = 0;
                    isNotEmpty = false;
                    sumFp = 0;
                    sumFn = 0;
                    anyInGold[0] = false;
                    anyInSegm[0] = false;

                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint16_t z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nx) {
                            //first array gold
                            uint8_t& zLocRef = zLoc; uint8_t& yLocRef = yLoc; uint8_t& xLocRef = xLoc;

                            goldBool = (getTensorRow<TYI>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);

                            // now segmentation  array
                            segmBool = (getTensorRow<TYI>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);
                            // setting bits
                             //setting  bits for reduced representation 
                            sharedForGold[xLoc][yLoc] |= goldBool << zLoc;
                            sharedForSegm[xLoc][yLoc] |= segmBool << zLoc;
                            // setting value of local boolean marking that any of the entries was evaluated to true in either of arrays
                            isNotEmpty = (isNotEmpty || (goldBool || segmBool));
                            sumFp += (!goldBool && segmBool);
                            sumFn += (goldBool && !segmBool);
                            if (goldBool)  anyInGold[0] = true;
                            if (segmBool)  anyInSegm[0] = true;
                        }
                    }
                    ////after we streamed over all z layers we need to save it into reduced representation arrays
                    //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGold, fbArgs.reducedGold.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                    //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegm, fbArgs.reducedSegm.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];
                    //// TODO() establish is it faster that way or better at the end do mempcy async
                    //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGoldRef, fbArgs.reducedGoldRef.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                    //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegmRef, fbArgs.reducedSegmRef.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];

                    getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGoldPrev, fbArgs.reducedGoldPrev.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                    getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegmPrev, fbArgs.reducedSegmPrev.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];
                    //we establish wheather this block is not empty if it is not - we will mark it as active
                    isNotEmpty = __syncthreads_or(isNotEmpty);


                    /////adding the block and total number of the Fp's and Fn's 
                    sumFp = reduce(tile, sumFp, plus<uint16_t>());
                    sumFn = reduce(tile, sumFn, plus<uint16_t>());
                    //reusing shared memory and adding accumulated values from tiles
                    if (tile.thread_rank() == 0) {
                        sharedForGold[0][tile.meta_group_rank()] = sumFp;
                        sharedForSegm[0][tile.meta_group_rank()] = sumFn;
                    }
                    sync(cta);//waiting so shared memory will be loaded evrywhere
                    //on single thread we do last sum reduction
                    auto active = coalesced_threads();
                    //gold
                    if ((threadIdx.x == 0) && (threadIdx.y == 0) && isNotEmpty) {
                        //if (isToBeExecutedOnActive(active, 0) && isNotEmpty) {
                        sharedForGold[1][0] = 0;//reset
                        for (int i = 0; i < tile.meta_group_size(); i += 1) {
                            sharedForGold[1][0] += sharedForGold[0][i];
                        };
                        fpSFnS[0] += sharedForGold[1][0];// will be needed later for global set
                        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta] = sharedForGold[1][0];
                    }
                    //segm
                   // if (isToBeExecutedOnActive(active, 1) && isNotEmpty) {
                    if ((threadIdx.x == 0) && (threadIdx.y == 1) && isNotEmpty) {
                        sharedForSegm[1][0] = 0;//reset
                        for (uint8_t i = 0; i < tile.meta_group_size(); i += 1) {
                            sharedForSegm[1][0] += sharedForSegm[0][i];
                        };
                        fpSFnS[1] += sharedForSegm[1][0];// will be needed later for global set
                        //setting metadata
                        getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta] = sharedForSegm[1][0];

                    }
                    /////////////////// setting min and maxes

                    //marking as active 
    //FP pass
                    if (isToBeExecutedOnActive(active, 8) && isNotEmpty && anyInGold[0]) {  //&& anyInGold[0]
                        getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta] = true;

                    };
                    //FN pass
                    if (isToBeExecutedOnActive(active, 9) && isNotEmpty && anyInSegm[0]) { // && anyInSegm[0]
                        getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta] = true;

                    };




                    sync(cta); // just to reduce the warp divergence




                }
            }

        }

    }
    sync(cta);
    ////// completing reductions of fp and fns



    auto active = coalesced_threads();

   
//setting global fp and fn
    if (isToBeExecutedOnActive(active, 0)) {
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7]), fpSFnS[0]);
    };

    if (isToBeExecutedOnActive(active, 1)) {
             atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8]), fpSFnS[1]);
    
};





}



/*
collecting all needed functions for GPU execution to prepare data from calculating Housedorff distance
*/
#pragma once
template <typename TYO>
__global__ void boolPrepareKernel(ForBoolKernelArgs<TYO> fbArgs) {
    metaDataIter(fbArgs);
}


//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
