#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;

/*
given appropriate hipPitchedPtr and ForFullBoolPrepArgs will return ForBoolKernelArgs
*/
#pragma once
template <typename TCC>
inline ForBoolKernelArgs<TCC> getArgsForKernel(ForFullBoolPrepArgs<int> mainFunArgs, array3dWithDimsGPU forDebugArr
    , array3dWithDimsGPU goldArr
    , array3dWithDimsGPU segmArr
    //, array3dWithDimsGPU reducedGold
    //, array3dWithDimsGPU reducedSegm
    //, array3dWithDimsGPU reducedGoldRef
    //, array3dWithDimsGPU reducedSegmRef
    //, array3dWithDimsGPU reducedGoldPrev
    //, array3dWithDimsGPU reducedSegmPrev

) {

    ForBoolKernelArgs<TCC> res;
    res.metaData = allocateMetaDataOnGPU(mainFunArgs.metaData);
    res.forDebugArr = forDebugArr;
    res.goldArr = goldArr;
    res.segmArr = segmArr;
    //allocate the reduced arrays
    //res.reducedGold = reducedGold;
    //res.reducedSegm = reducedSegm;
    //res.reducedGoldPrev = reducedGoldPrev;
    //res.reducedSegmPrev = reducedSegmPrev;

    //res.reducedGoldRef = reducedGoldRef;
    //res.reducedSegmRef = reducedSegmRef;

    res.numberToLookFor = mainFunArgs.numberToLookFor;
    res.dbXLength = mainFunArgs.dbXLength;
    res.dbYLength = mainFunArgs.dbYLength;
    res.dbZLength = mainFunArgs.dbZLength;



    return res;
}





// helper functions and utilities to work with CUDA from https://github.com/NVIDIA/cuda-samples


/*
iteration over data block given metadata coordinates
*/
#pragma once
template <typename TPI>
__device__ void fillReduCedArr(ForBoolKernelArgs<TPI> fbArgs,
    int minMaxesInShmem[6], uint16_t& sumFp, uint16_t& sumFn,
    bool& isNotEmpty, uint8_t xMeta, uint8_t yMeta, uint8_t zMeta, char* tensorslice
    , uint16_t& z, uint16_t& y, uint16_t& x, bool& goldBool, bool& segmBool
    , uint32_t sharedForGold[32][32], uint32_t sharedForSegm[32][32]
    , uint8_t& xLoc, uint8_t& yLoc, uint8_t& zLoc
    , bool anyInGold[1], bool anyInSegm[1]) {

    //setting  bits for reduced representation 
    sharedForGold[xLoc][yLoc] |= goldBool << zLoc;
    sharedForSegm[xLoc][yLoc] |= segmBool << zLoc;
    // setting value of local boolean marking that any of the entries was evaluated to true in either of arrays
    isNotEmpty = (isNotEmpty || (goldBool || segmBool));
    sumFp += (!goldBool && segmBool);
    sumFn += (goldBool && !segmBool);
    if (goldBool)  anyInGold[0] = true;
    if (segmBool)  anyInSegm[0] = true;


    //if (goldBool && !segmBool) {
    //    printf("nnnnnnnnnnnn  fn x %d y %d z %d    xMeta [%d] yMeta [%d] zMeta [%d]  \n", x, y, z, xMeta, yMeta, zMeta);
    //}
    //if (!goldBool && segmBool) {
    //    printf("pppppppp  fp x %d y %d z %d    xMeta [%d] yMeta [%d] zMeta [%d]  \n", x, y, z, xMeta, yMeta, zMeta);
    //}

}


/*
iteration over data block given metadata coordinates
*/

#pragma once
template <typename TYI>
__device__ void dataBlockIter(ForBoolKernelArgs<TYI> fbArgs, thread_block cta, thread_block_tile<32> tile,
    int minMaxesInShmem[6], uint16_t& sumFp, uint16_t& sumFn,
    bool& isNotEmpty, uint8_t xMeta, uint8_t yMeta, uint8_t zMeta, char* tensorslice
    , uint16_t& z, uint16_t& y, uint16_t& x, bool& goldBool, bool& segmBool
    , uint32_t sharedForGold[32][32], uint32_t sharedForSegm[32][32], uint32_t fpSFnS[2]
    , bool anyInGold[1], bool anyInSegm[1]
) {
    //now we need to iterate over the data in the data block voxel by voxel
    for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
        x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
        for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
            y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
            if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                // resetting 
                sharedForGold[xLoc][yLoc] = 0;
                sharedForSegm[xLoc][yLoc] = 0;
                isNotEmpty = false;
                sumFp = 0;
                sumFn = 0;
                anyInGold[0] = false;
                anyInSegm[0] = false;

                for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                    z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                    if (z < fbArgs.goldArr.Nx) {
                        //first array gold
                        uint8_t& zLocRef = zLoc; uint8_t& yLocRef = yLoc; uint8_t& xLocRef = xLoc;

                        goldBool = (getTensorRow<TYI>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);

                        // now segmentation  array
                        segmBool = (getTensorRow<TYI>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);
                        // setting bits
                        fillReduCedArr(fbArgs, minMaxesInShmem, sumFp, sumFn, isNotEmpty, xMeta, yMeta, zMeta
                            , tensorslice, z, y, x, goldBool, segmBool, sharedForGold, sharedForSegm
                            , xLocRef, yLocRef, zLocRef, anyInGold, anyInSegm);
                    }
                }
                //after we streamed over all z layers we need to save it into reduced representation arrays
                getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGold, fbArgs.reducedGold.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegm, fbArgs.reducedSegm.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];
                // TODO() establish is it faster that way or better at the end do mempcy async
                //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGoldRef, fbArgs.reducedGoldRef.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegmRef, fbArgs.reducedSegmRef.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];

                //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedGoldPrev, fbArgs.reducedGoldPrev.Ny, y, zMeta)[x] = sharedForGold[xLoc][yLoc];
                //getTensorRow<uint32_t>(tensorslice, fbArgs.reducedSegmPrev, fbArgs.reducedSegmPrev.Ny, y, zMeta)[x] = sharedForSegm[xLoc][yLoc];
                //we establish wheather this block is not empty if it is not - we will mark it as active
                isNotEmpty = __syncthreads_or(isNotEmpty);


                /////adding the block and total number of the Fp's and Fn's 
                sumFp = reduce(tile, sumFp, plus<uint16_t>());
                sumFn = reduce(tile, sumFn, plus<uint16_t>());
                //reusing shared memory and adding accumulated values from tiles
                if (tile.thread_rank() == 0) {
                    sharedForGold[0][tile.meta_group_rank()] = sumFp;
                    sharedForSegm[0][tile.meta_group_rank()] = sumFn;
                }
                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction
                auto active = coalesced_threads();
                //gold
                if ((threadIdx.x == 0) && (threadIdx.y == 0) && isNotEmpty) {
                    //if (isToBeExecutedOnActive(active, 0) && isNotEmpty) {
                    sharedForGold[1][0] = 0;//reset
                    for (int i = 0; i < tile.meta_group_size(); i += 1) {
                        sharedForGold[1][0] += sharedForGold[0][i];
                    };
                    fpSFnS[0] += sharedForGold[1][0];// will be needed later for global set
                   // printf("adding fps %d  xMeta [%d] yMeta [%d] zMeta [%d]  \n", sharedForGold[1][0], xMeta, yMeta, zMeta);

               //     printf("locMeta x %d y %d z %d fp %d \n ", xMeta, yMeta, zMeta, sharedForGold[1][0]);

                    //setting metadata
                    //printf("\n in bool kernel fp count  x %d y %d z %d fp %d \n ", xMeta, yMeta, zMeta, sharedForGold[1][0]);

                    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpCount, fbArgs.metaData.fpCount.Ny, yMeta, zMeta)[xMeta] = sharedForGold[1][0];
                }
                //segm
               // if (isToBeExecutedOnActive(active, 1) && isNotEmpty) {
                if ((threadIdx.x == 0) && (threadIdx.y == 1) && isNotEmpty) {
                    sharedForSegm[1][0] = 0;//reset
                    for (int i = 0; i < tile.meta_group_size(); i += 1) {
                        sharedForSegm[1][0] += sharedForSegm[0][i];
                    };
                    fpSFnS[1] += sharedForSegm[1][0];// will be needed later for global set
                    //setting metadata

                   // printf("\n in bool kernel fn count  x %d y %d z %d fn %d \n " , xMeta, yMeta, zMeta, sharedForSegm[1][0]);
                    getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnCount, fbArgs.metaData.fnCount.Ny, yMeta, zMeta)[xMeta] = sharedForSegm[1][0];

                }
                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                if (isToBeExecutedOnActive(active, 2) && isNotEmpty) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && isNotEmpty) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && isNotEmpty) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && isNotEmpty) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && isNotEmpty) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && isNotEmpty) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

                //marking as active 
//FP pass
                if (isToBeExecutedOnActive(active, 8) && isNotEmpty && anyInGold[0]) {  //&& anyInGold[0]
                 //   printf("\n set activeee in gold xMeta %d yMeta %d  zMeta %d \n",xMeta,yMeta,zMeta);
                    getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveGold, fbArgs.metaData.isActiveGold.Ny, yMeta, zMeta)[xMeta] = true;

                };
                //FN pass
                if (isToBeExecutedOnActive(active, 9) && isNotEmpty && anyInSegm[0]) { // && anyInSegm[0]
                 //   printf("\n set activeee in segm xMeta %d yMeta %d  zMeta %d \n", xMeta, yMeta, zMeta);
                    getTensorRow<bool>(tensorslice, fbArgs.metaData.isActiveSegm, fbArgs.metaData.isActiveSegm.Ny, yMeta, zMeta)[xMeta] = true;

                };




                sync(cta); // just to reduce the warp divergence




            }
        }

    }
}


///*					if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
//                        printf("dataBlockIter xMeta [%d] yMeta [%d] zMeta [%d]  \n",  xMeta, yMeta, zMeta);
//                        }*/
//
//
//                        /*					if (goldBool) {
//                                                printf("goldBool x %d y %d z %d    xMeta [%d] yMeta [%d] zMeta [%d]  \n", x, y, z, xMeta, yMeta, zMeta);
//                                            }*/
//
//                                            //now segmentation array
//
//
//                                            //tensorrow = getTensorRow(tensorslice, fbArgs.segmArr, fbArgs.mainArrYLength, y, z);
//                                            //segmBool = (tensorrow[x] == fbArgs.numberToLookFor);
//                                            //
//                                            //
//                                            //fillReduCedArr(fbArgs, xMeta, yMeta, zMeta, tensorslice, tensorrow, z, y, x
//                                            //	, zLocRef, yLocRef, xLocRef, goldBool, segmBool);
//                //if (x==1 && y==2 && z==3) {
//                //    atomicAdd(&tensorrow[x], 1);
//                //}
//                                            //debug
//
//                //printf("dataBlockIter x %d y %d z %d  curr %d ||   xLoc %d yLoc %d zLoc %d xMeta [%d] yMeta [%d] zMeta [%d]  dbX %d dbY %d  dbZ %d  \n", x, y, z, tensorrow[x], xLoc, yLoc, zLoc, xMeta, yMeta, zMeta, fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength);
//
//
//      /*              if (tensorrow[x] > 0) {
//                        printf("dataBlockIter x %d y %d z %d  xLoc %d yLoc %d zLoc %d xMeta [%d] yMeta [%d] zMeta [%d]  dbX %d dbY %d  dbZ %d  \n", x, y, z, xLoc, yLoc, zLoc, xMeta, yMeta, zMeta, fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength);
//                    }*/
//
//                    //tensorrow = getTensorRow(tensorslice, fbArgs.goldArr, fbArgs.mainArrYLength,  y,  z);
//                    //tensorrow[x] += 1;
//
//                    //debug
//                    //tensorslice = ((char*)fbArgs.forDebugArr.ptr) + z * fbArgs.forDebugArr.pitch * fbArgs.dYLength;
//                    //tensorrow = (int*)(tensorslice + y * fbArgs.forDebugArr.pitch);
//                    //tensorrow[x] += 1;
//
//                    //printf("dataBlockIter %d tensorrow[x]    xMeta [%d] yMeta [%d] zMeta [%d]  \n", tensorrow[x], xMeta, yMeta, zMeta);
//
//                    //array segmentation output




/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYU>
__device__ void metaDataIter(ForBoolKernelArgs<TYU> fbArgs) {

    ////////////some initializations
    bool goldBool = false;
    bool segmBool = false;
    bool isNotEmpty = false;
    bool& goldBoolRef = goldBool;
    bool& segmBoolRef = segmBool;
    bool& isNotEmptyRef = isNotEmpty;
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);

    uint16_t z;	 uint16_t x;    uint16_t y;	 uint16_t sumFpOrig;    uint16_t sumFnOrig;
    uint16_t& refZ = z; uint16_t& refX = x; uint16_t& refY = y; uint16_t& sumFp = sumFpOrig; uint16_t& sumFn = sumFnOrig;


    uint8_t xMeta; uint8_t zMeta; uint8_t yMeta;
    char* tensorslice;


    //shared memory

    //TODO() make it dynamically sized 
    __shared__ uint32_t sharedForGold[32][32];
    __shared__ uint32_t sharedForSegm[32][32];
    //for storing fp and fn sums to later accumulate it to global values
    __shared__ uint32_t fpSFnS[2];

    __shared__ bool anyInGold[1];
    __shared__ bool anyInSegm[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ int minMaxesInShmem[7];

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) { minMaxesInShmem[1] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 0)) { minMaxesInShmem[2] = 1000; };

    if ((threadIdx.x == 3) && (threadIdx.y == 0)) { minMaxesInShmem[3] = 0; };
    if ((threadIdx.x == 4) && (threadIdx.y == 0)) { minMaxesInShmem[4] = 1000; };

    if ((threadIdx.x == 5) && (threadIdx.y == 0)) { minMaxesInShmem[5] = 0; };
    if ((threadIdx.x == 0) && (threadIdx.y == 1)) { minMaxesInShmem[6] = 1000; };
    if ((threadIdx.x == 1) && (threadIdx.y == 1)) { fpSFnS[0] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 1)) { fpSFnS[1] = 0; };
    if ((threadIdx.x == 3) && (threadIdx.y == 1)) { anyInGold[1] = false; };
    if ((threadIdx.x == 4) && (threadIdx.y == 1)) { anyInSegm[1] = false; };

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //iterating over data block
        dataBlockIter(fbArgs, cta, tile, minMaxesInShmem, sumFp, sumFn, isNotEmptyRef
            , xMeta, yMeta, zMeta, tensorslice, refZ, refY, refX
            , goldBoolRef, segmBoolRef, sharedForGold, sharedForSegm, fpSFnS, anyInGold, anyInSegm);


        /*
           //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
           //	printf("dataBlockIter xMeta [%d] yMeta [%d] zMeta [%d]  \n",  xMeta, yMeta, zMeta);

           //}
           //unhash for debugging
           /// checking are all covered
           ////debugging loop
           //char* tensorslice;    int* tensorrow;
           ////tensorslice = ((char*)fbArgs.forDebugArr.ptr) + zMeta * fbArgs.forDebugArr.pitch * fbArgs.MetaYLength;
           ////tensorrow = (int*)(tensorslice + yMeta * fbArgs.forDebugArr.pitch);
           //////tensorrow[xMeta] += 1;
           ////if ((threadIdx.x == 0 )&& (threadIdx.y == 0)) {
           ////    atomicAdd(&tensorrow[xMeta], 1);
           ////}
           //// checking
           //tensorslice = ((char*)fbArgs.forDebugArr.ptr) + 0 * fbArgs.forDebugArr.pitch * fbArgs.MetaYLength;
           //tensorrow = (int*)(tensorslice + 0 * fbArgs.forDebugArr.pitch);
           ////tensorrow[xMeta] += 1;
           //if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
           //    atomicAdd(&tensorrow[0], 1);
           //}


           //printf("metaCudaTensor[%d][%d][%d] = %d\n", i, j, k, tensorrow[i]);
   */

    }
    sync(cta);
    ////// completing reductions of fp and fns


    /// setting min maxes 
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ

    auto active = coalesced_threads();
    //int prev;
    //if (g.thread_rank() == 0) {
    //    prev = atomicAdd(p, g.num_threads());
    //}
    //active.thread_rank()
    //    active.num_threads

    if (isToBeExecutedOnActive(active, 0)) {
        //printf("in minMaxes internal  %d \n", minMaxesInShmem[0]);
        //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, fbArgs.metaData.minMaxes.Ny, 0, 0)[0] = 61;
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[1]), minMaxesInShmem[1]);
    };

    if (isToBeExecutedOnActive(active, 1)) {

        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[2]), minMaxesInShmem[2]);
    };

    if (isToBeExecutedOnActive(active, 2)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[3]), minMaxesInShmem[3]);
    };

    if (isToBeExecutedOnActive(active, 3)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[4]), minMaxesInShmem[4]);
    };



    if (isToBeExecutedOnActive(active, 4)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[5]), minMaxesInShmem[5]);
    };

    if (isToBeExecutedOnActive(active, 5)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[6]), minMaxesInShmem[6]);
    };

    //setting global fp and fn
    if (isToBeExecutedOnActive(active, 6)) {
        //printf("internal last fp  %d \n", fpSFnS[0]);
        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[7]), fpSFnS[0]);
    };

    if (isToBeExecutedOnActive(active, 7)) {
        //if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
       //if (active.thread_rank() == 7 && active.meta_group_rank() == 0) {
       //     printf("internal last fn  %d idX %d  idY %d tile meta size %d \n", fpSFnS[1], threadIdx.x, threadIdx.y, tile.meta_group_size());

        atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[8]), fpSFnS[1]);

    };





}



/*
collecting all needed functions for GPU execution to prepare data from calculating Housedorff distance
*/
#pragma once
template <typename TYO>
__global__ void boolPrepareKernel(ForBoolKernelArgs<TYO> fbArgs) {
    metaDataIter(fbArgs);
}


//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
