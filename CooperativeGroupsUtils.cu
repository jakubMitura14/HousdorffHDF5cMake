#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;

/*
We will  here return true if the thread is at the moment active - Hovewer sometimes we have couple things to do so we will add additional numb
untlin numb will not execute number of available threads in a group it will be executed on this thread in other case we will use some already used thread ...
*/

#pragma once
inline __device__ bool isToBeExecutedOnActive(coalesced_group group, int numb, int metaNumb=0) {
    return ((threadIdx.x == numb) && (threadIdx.y == metaNumb));
    //if ((numb < group.num_threads()) && (threadIdx.x == numb) && (threadIdx.y == metaNumb) ) {
    //    return true;
    //}
    //else {// defoult is first thread in group
    //    if (threadIdx.y == 0 && (threadIdx.y == metaNumb)) {
    //        return true;
    //    }
    //};
    //return false;


}
