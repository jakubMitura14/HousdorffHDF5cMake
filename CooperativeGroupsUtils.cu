#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;

/*
We will  here return true if the thread is at the moment active - Hovewer sometimes we have couple things to do so we will add additional numb
untlin numb will not execute number of available threads in a group it will be executed on this thread in other case we will use some already used thread ...
*/

#pragma once
inline __device__ bool isToBeExecutedOnActive(coalesced_group group, int numb, int metaNumb=0) {
    return ((threadIdx.x == numb) && (threadIdx.y == metaNumb));
    //if ((numb < group.num_threads()) && (threadIdx.x == numb) && (threadIdx.y == metaNumb) ) {
    //    return true;
    //}
    //else {// defoult is first thread in group
    //    if (threadIdx.y == 0 && (threadIdx.y == metaNumb)) {
    //        return true;
    //    }
    //};
    //return false;


}


/*
copy asynchronously into shared memopry using pipeline interface - works only for uint32_t
pipeline: pipeline object
block: thread block in cooperative groups definition
mainShmem : shared memory to which we load data
globalIn : global memory from which we take data
alignSize: defines what is the smallest aligned byte lenghth - for best performance should be 128 so for example 32 uint32_t
shmemStart : where in shared memory we have starting point for our load
globalStart : where in global memory we start load
length : how many uint32_t we want to copy from global to shmem
*/
//#pragma once
//inline __device__ void  loadIntoShmem(cuda::pipeline<cuda::thread_scope_thread> pipeline, thread_block block,
//                                        uint32_t* mainShmem, uint32_t* globalIn 
//                                   ,int shmemStart, int globalStart, int length ) {
//    
//    pipeline.producer_acquire();
//    cuda::memcpy_async(block, &mainShmem[shmemStart], &globalIn[globalStart], cuda::aligned_size_t<4>(sizeof(uint32_t) * length), pipeline);
//    pipeline.producer_commit();
//  
//}
