#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;





// helper functions and utilities to work with CUDA from https://github.com/NVIDIA/cuda-samples



/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYU>
__device__ void metaDataIterB(ForBoolKernelArgs<TYU> fbArgs) {

    ////////////some initializations
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);
     

    char* tensorslice;


    //shared memory
    bool isNotEmpty = false;
    __shared__ bool anyInGold[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ uint8_t minMaxesInShmem[8];
    __shared__ uint8_t minXInShmem[1];
    //__shared__ bool [2000];

    if ((threadIdx.x == 1)) { minMaxesInShmem[1] = 0; };
    if ((threadIdx.x == 2)) { minMaxesInShmem[2] = 1000; };

    if ((threadIdx.x == 3)) { minMaxesInShmem[3] = 0; };
    if ((threadIdx.x == 4) ) { minMaxesInShmem[4] = 1000; };

    if ((threadIdx.x == 5) ) { minMaxesInShmem[5] = 0; };
    if ((threadIdx.x == 0)) { minMaxesInShmem[6] = 1000; };

    //if ((threadIdx.x == 3) && (threadIdx.y == 1)) { anyInGold[1] = false; };

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //if (threadIdx.x == 0) { anyInGold[0] = false; };
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
         isNotEmpty = __syncthreads_or(isNotEmpty);;
        if (threadIdx.x == 0) {
            printf("linIdexMeta %d xMeta %d yMeta %d zMeta %d metaXLength %d MetaYLength %d totalMetaLength %d anyInGold[0] %d gold Nx %d nY %d nZ %d segm Nx %d Ny %d Nz %d    \n "
                , linIdexMeta, xMeta, yMeta, zMeta, fbArgs.metaData.metaXLength, fbArgs.metaData.MetaYLength, fbArgs.metaData.totalMetaLength, isNotEmpty
            , fbArgs.goldArr.Nx, fbArgs.goldArr.Ny, fbArgs.goldArr.Nz , fbArgs.segmArr.Nx, fbArgs.segmArr.Ny, fbArgs.segmArr.Nz
            );
        }
        sync(cta);
        isNotEmpty = false;
        //iterating over data block
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint16_t x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint16_t  y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                //   //  resetting 


                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint16_t z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nx) {
                            //first array gold
                            // setting bits

                            if ((getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor)
                                || (getTensorRow<TYU>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor)) {
                                isNotEmpty = true;
                            }

                            if (threadIdx.x == 0 && isNotEmpty) {
                                printf("x %d y%d z %d linIdexMeta %d xMeta %d yMeta %d zMeta %d metaXLength %d MetaYLength %d totalMetaLength %d anyInGold[0] \n "
                                    , x, y, z, linIdexMeta, xMeta, yMeta, zMeta, fbArgs.metaData.metaXLength, fbArgs.metaData.MetaYLength, fbArgs.metaData.totalMetaLength, isNotEmpty
                                    , fbArgs.goldArr.Nx, fbArgs.goldArr.Ny, fbArgs.goldArr.Nz, fbArgs.segmArr.Nx, fbArgs.segmArr.Ny, fbArgs.segmArr.Nz
                                );
                            }


                        }

                    }

                }
            }
        }



      //  isNotEmpty = __syncthreads_or(isNotEmpty);



        ///////////////// setting min and maxes
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
       // if (((threadIdx.x == 0)) && anyInGold[0]) { minXInShmem[0] = xMeta; }
                //if (((threadIdx.x == 0)) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                //if (((threadIdx.x == 1)) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                //if (((threadIdx.x == 2)) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                //if (((threadIdx.x == 3) ) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                //if (((threadIdx.x == 4) ) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                //if (((threadIdx.x == 5) ) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

                //if (((threadIdx.x == 6)) && anyInGold[0]) { anyInGold[0] = false; };

                //if (anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                //if ( anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                //if (anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                //if (anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                //if (anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                //if (anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };
                //if (isNotEmpty) { isNotEmpty = false; };
                //sync(cta);
 }
    sync(cta);


    auto active = coalesced_threads();
    if (isToBeExecutedOnActive(active, 0)) {
        //printf("in minMaxes internal  %d \n", minMaxesInShmem[0]);
        //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, fbArgs.metaData.minMaxes.Ny, 0, 0)[0] = 61;
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[1]), minMaxesInShmem[1]);
    };

    if (isToBeExecutedOnActive(active, 1)) {

        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[2]), minMaxesInShmem[2]);
    };

    if (isToBeExecutedOnActive(active, 2)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[3]), minMaxesInShmem[3]);
    };

    if (isToBeExecutedOnActive(active, 3)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[4]), minMaxesInShmem[4]);
    };



    if (isToBeExecutedOnActive(active, 4)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[5]), minMaxesInShmem[5]);
    };

    if (isToBeExecutedOnActive(active, 5)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[6]), minMaxesInShmem[6]);
    };






}



/*
collecting all needed functions for GPU execution to prepare data from calculating Housedorff distance
*/
#pragma once
template <typename TYO>
__global__ void getMinMaxes(ForBoolKernelArgs<TYO> fbArgs) {
    metaDataIterB(fbArgs);
}


//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
