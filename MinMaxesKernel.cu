#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"

#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;





// helper functions and utilities to work with CUDA from https://github.com/NVIDIA/cuda-samples



/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYO>
__global__ void getMinMaxes(ForBoolKernelArgs<TYO> fbArgs
    , unsigned int* minMaxes
    , TYO* goldArr, TYO* segmArr, MetaDataGPU metaData
) {

    // __global__ void getMinMaxes(unsigned int* minMaxes) {
     ////////////some initializations
    thread_block cta = this_thread_block();
    //thread_block_tile<32> tile = tiled_partition<32>(cta);



    //shared memory

    __shared__ bool anyInGold[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ unsigned int minMaxesInShmem[7];

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) { minMaxesInShmem[1] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 0)) { minMaxesInShmem[2] = 1000; };

    if ((threadIdx.x == 3) && (threadIdx.y == 0)) { minMaxesInShmem[3] = 0; };
    if ((threadIdx.x == 4) && (threadIdx.y == 0)) { minMaxesInShmem[4] = 1000; };

    if ((threadIdx.x == 5) && (threadIdx.y == 0)) { minMaxesInShmem[5] = 0; };
    if ((threadIdx.x == 6) && (threadIdx.y == 0)) { minMaxesInShmem[6] = 1000; };

    if ((threadIdx.x == 7) && (threadIdx.y == 0)) { anyInGold[1] = false; };


    //if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
    //    //printf("in minMaxes beg  totalMetaLength  %d Nx %d Ny %d Nz %d \n"
    //    //    , fbArgs.metaData.totalMetaLength
    //    //    , fbArgs.goldArr.Nx
    //    //    , fbArgs.goldArr.Ny
    //    //    , fbArgs.goldArr.Nz
    //    //
    //    //);

    //    if (blockIdx.x == 0) {
    //        printf(" dims meta in min maxes  kernel Meta X %d MetaY %d metaZ %d dbXSize %d dbYsize %d dbZsize %d minX %d minY %d minZ \n "
    //            , metaData.metaXLength, metaData.MetaYLength, metaData.MetaZLength
    //            , fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength
    //            , metaData.minX, metaData.minY, metaData.minZ
    //        );

    //}

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        int  xMeta = linIdexMeta % metaData.metaXLength;
        int   zMeta = int(floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * metaData.MetaYLength))));
        int   yMeta = int(floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength)));
        //iterating over data block
        for (uint8_t xLoc = threadIdx.x; xLoc < 32; xLoc += blockDim.x) {
            uint32_t x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint32_t  y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
                //if (y == 0) {
                //    printf("x %d  in min maxes \n ", x);

                //}
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nx) {

                    // resetting 


                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint32_t z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nz) {
                            //first array gold
                            //uint8_t& zLocRef = zLoc; uint8_t& yLocRef = yLoc; uint8_t& xLocRef = xLoc;

                            // setting bits
                            bool goldBool = goldArr[x + y * fbArgs.goldArr.Nx + z * fbArgs.goldArr.Nx * fbArgs.goldArr.Ny] == fbArgs.numberToLookFor;  // (getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);
                            bool segmBool = segmArr[x + y * fbArgs.goldArr.Nx + z * fbArgs.goldArr.Nx * fbArgs.goldArr.Ny] == fbArgs.numberToLookFor;
                            if (goldBool || segmBool) {
                                anyInGold[0] = true;
                                //printf(" \n in min maxes dims meta in min maxes   x %d y%d z%d xMeta %d yMeta %d zMeta %d  kernel Meta X %d MetaY %d metaZ %d dbXSize %d dbYsize %d dbZsize %d minX %d minY %d minZ %d linIdexMeta %d counted %d  \n "
                                //    ,x,y,z,
                                //    xMeta,yMeta,zMeta
                                //    , metaData.metaXLength, metaData.MetaYLength, metaData.MetaZLength
                                //    , fbArgs.dbXLength, fbArgs.dbYLength, fbArgs.dbZLength
                                //    , metaData.minX, metaData.minY, metaData.minZ
                                //    , linIdexMeta
                                //    , int(floor((float)((linIdexMeta - ((zMeta * metaData.metaXLength * metaData.MetaYLength) + xMeta)) / metaData.metaXLength)))
                                //);

                            }



                        }

                    }
                }

                //  __syncthreads();
                  //waiting so shared memory will be loaded evrywhere
                  //on single thread we do last sum reduction

                  /////////////////// setting min and maxes
  //    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();
                sync(cta);
                active.sync();

                if ((threadIdx.x == 0) && (threadIdx.y == 0) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if ((threadIdx.x == 1) && (threadIdx.y == 0) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if ((threadIdx.x == 2) && (threadIdx.y == 0) && anyInGold[0]) {

                    minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]);

                    //if (minMaxesInShmem[3] > 0) {
                    //    printf(" prim minMaxesInShmem maxY %d meta %d \n ", minMaxesInShmem[3], yMeta);
                    //}

                };
                if ((threadIdx.x == 3) && (threadIdx.y == 0) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if ((threadIdx.x == 4) && (threadIdx.y == 0) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if ((threadIdx.x == 5) && (threadIdx.y == 0) && anyInGold[0]) {
                    minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]);
                    // printf("local fifth %d  \n", minMaxesInShmem[6]);
                };
                // active.sync();
                sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);

    auto active = coalesced_threads();

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        //  printf("\n in minMaxes internal  %d \n", minMaxesInShmem[1]);
       //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, fbArgs.metaData.minMaxes.Ny, 0, 0)[0] = 61;
        atomicMax(&minMaxes[1], minMaxesInShmem[1]);
        //atomicMax(&minMaxes[1], 2);
       // minMaxes[1] = 0;
    };

    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {

        atomicMin(&minMaxes[2], minMaxesInShmem[2]);
    };

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        atomicMax(&minMaxes[3], minMaxesInShmem[3]);
        //  printf(" minMaxesInShmem maxY %d \n ", minMaxes[3]);

    };

    if ((threadIdx.x == 2) && (threadIdx.y == 0)) {
        atomicMin(&minMaxes[4], minMaxesInShmem[4]);
        //   printf(" minMaxesInShmem minY %d \n ", minMaxes[4]);

    };



    if (threadIdx.x == 3 && threadIdx.y == 0) {
        atomicMax(&minMaxes[5], minMaxesInShmem[5]);
        //  printf(" minMaxesInShmem  %d \n ", minMaxes[5]);
    };

    if (threadIdx.x == 4 && threadIdx.y == 0) {
        atomicMin(&minMaxes[6], minMaxesInShmem[6]);
        // printf(" minMaxesInShmem  %d \n ", minMaxes[6]);

    };





}





//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
