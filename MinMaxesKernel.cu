#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;





// helper functions and utilities to work with CUDA from https://github.com/NVIDIA/cuda-samples



/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYU>
__device__ void metaDataIterB(ForBoolKernelArgs<TYU> fbArgs) {

    ////////////some initializations
    thread_block cta = this_thread_block();
    thread_block_tile<32> tile = tiled_partition<32>(cta);


    char* tensorslice;


    //shared memory
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD

=======
    
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======
    
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
    __shared__ bool anyInGold[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ int minMaxesInShmem[7];

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) { minMaxesInShmem[1] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 0)) { minMaxesInShmem[2] = 1000; };

    if ((threadIdx.x == 3) && (threadIdx.y == 0)) { minMaxesInShmem[3] = 0; };
    if ((threadIdx.x == 4) && (threadIdx.y == 0)) { minMaxesInShmem[4] = 1000; };

    if ((threadIdx.x == 5) && (threadIdx.y == 0)) { minMaxesInShmem[5] = 0; };
    if ((threadIdx.x == 0) && (threadIdx.y == 1)) { minMaxesInShmem[6] = 1000; };

    if ((threadIdx.x == 3) && (threadIdx.y == 1)) { anyInGold[1] = false; };

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //iterating over data block
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint16_t x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint16_t  y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                    // resetting 


                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint16_t z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nx) {
                            //first array gold
                            uint8_t& zLocRef = zLoc; uint8_t& yLocRef = yLoc; uint8_t& xLocRef = xLoc;
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD

                            // setting bits
                            if ((getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor) || (getTensorRow<TYU>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor)) {
                                anyInGold[0] = true;
                            }
                        }

                    }
                }
<<<<<<< HEAD

                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

=======

                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
                sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);
<<<<<<< HEAD

=======
   
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======

=======

>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
                            // setting bits
                            if ((getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor) || (getTensorRow<TYU>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor)) {
                                anyInGold[0] = true;
                            }
                        }

                    }
                }
<<<<<<< HEAD

                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

                sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);
   
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======

                            // setting bits
                            if ((getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor) || (getTensorRow<TYU>(tensorslice, fbArgs.segmArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor)) {
                                anyInGold[0] = true;
                            }
                        }

                    }
                }

                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

                sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);
   
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
=======

                sync(cta);//waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]); };

                sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);
   
>>>>>>> parent of ebdf6ce (up not working min maxes for some reason)
    auto active = coalesced_threads();
    if (isToBeExecutedOnActive(active, 0)) {
        //printf("in minMaxes internal  %d \n", minMaxesInShmem[0]);
        //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, fbArgs.metaData.minMaxes.Ny, 0, 0)[0] = 61;
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[1]), minMaxesInShmem[1]);
    };

    if (isToBeExecutedOnActive(active, 1)) {

        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[2]), minMaxesInShmem[2]);
    };

    if (isToBeExecutedOnActive(active, 2)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[3]), minMaxesInShmem[3]);
    };

    if (isToBeExecutedOnActive(active, 3)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[4]), minMaxesInShmem[4]);
    };



    if (isToBeExecutedOnActive(active, 4)) {
        atomicMax(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[5]), minMaxesInShmem[5]);
    };

    if (isToBeExecutedOnActive(active, 5)) {
        atomicMin(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[6]), minMaxesInShmem[6]);
    };






}



/*
collecting all needed functions for GPU execution to prepare data from calculating Housedorff distance
*/
#pragma once
template <typename TYO>
__global__ void getMinMaxes(ForBoolKernelArgs<TYO> fbArgs) {
    metaDataIterB(fbArgs);
}


//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
