#include "hip/hip_runtime.h"


#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;





// helper functions and utilities to work with CUDA from https://github.com/NVIDIA/cuda-samples



/*
iteration over metadata - becouse metadata may be small and to maximize occupancy we use linear index and then clalculate xMeta,ymeta,zMeta from this linear index ...
*/
#pragma once
template <typename TYO>
__global__ void getMinMaxes(ForBoolKernelArgs<TYO> fbArgs
    , unsigned int* minMaxes
    , TYO* goldArr, TYO* segmArr
) {

   // __global__ void getMinMaxes(unsigned int* minMaxes) {
    ////////////some initializations
    thread_block cta = this_thread_block();
    //thread_block_tile<32> tile = tiled_partition<32>(cta);



    //shared memory

    __shared__ bool anyInGold[1];
    //__shared__ uint32_t reduction_s[32];
    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
    __shared__ unsigned int minMaxesInShmem[7];

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) { minMaxesInShmem[1] = 0; };
    if ((threadIdx.x == 2) && (threadIdx.y == 0)) { minMaxesInShmem[2] = 1000; };

    if ((threadIdx.x == 3) && (threadIdx.y == 0)) { minMaxesInShmem[3] = 0; };
    if ((threadIdx.x == 4) && (threadIdx.y == 0)) { minMaxesInShmem[4] = 1000; };

    if ((threadIdx.x == 5) && (threadIdx.y == 0)) { minMaxesInShmem[5] = 0; };
    if ((threadIdx.x == 6) && (threadIdx.y == 0)) { minMaxesInShmem[6] = 1000; };

    if ((threadIdx.x == 7) && (threadIdx.y == 0)) { anyInGold[1] = false; };

    __syncthreads();

    /////////////////////////


    //main metadata iteration
    for (auto linIdexMeta = blockIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //iterating over data block
        for (uint8_t xLoc = threadIdx.x; xLoc < fbArgs.dbXLength; xLoc += blockDim.x) {
            uint32_t x = xMeta * fbArgs.dbXLength + xLoc;//absolute position
            for (uint8_t yLoc = threadIdx.y; yLoc < fbArgs.dbYLength; yLoc += blockDim.y) {
                uint32_t  y = yMeta * fbArgs.dbYLength + yLoc;//absolute position
                if (y < fbArgs.goldArr.Ny && x < fbArgs.goldArr.Nz) {

                    // resetting 


                    for (uint8_t zLoc = 0; zLoc < fbArgs.dbZLength; zLoc++) {
                        uint32_t z = zMeta * fbArgs.dbZLength + zLoc;//absolute position
                        if (z < fbArgs.goldArr.Nx) {
                            //first array gold
                            uint8_t& zLocRef = zLoc; uint8_t& yLocRef = yLoc; uint8_t& xLocRef = xLoc;

                            // setting bits
                            bool goldBool = goldArr[x + y * fbArgs.goldArr.Nx + z * fbArgs.goldArr.Nx * fbArgs.goldArr.Ny] == fbArgs.numberToLookFor;  // (getTensorRow<TYU>(tensorslice, fbArgs.goldArr, fbArgs.goldArr.Ny, y, z)[x] == fbArgs.numberToLookFor);
                            bool segmBool = segmArr[x + y * fbArgs.goldArr.Nx + z * fbArgs.goldArr.Nx * fbArgs.goldArr.Ny] == fbArgs.numberToLookFor;
                             if (goldBool || segmBool) {
                                anyInGold[0] = true;
                            //    printf("seen as true  xMeta %d yMeta %d  zMeta %d \n", xMeta, yMeta,zMeta);

                            }
                        }

                    }
                }

              //  __syncthreads();
                //waiting so shared memory will be loaded evrywhere
                //on single thread we do last sum reduction

                /////////////////// setting min and maxes
//    //1)maxX 2)minX 3)maxY 4) minY 5) maxZ 6) minZ
                auto active = coalesced_threads();
                sync(cta);
                active.sync();

                if (isToBeExecutedOnActive(active, 2) && anyInGold[0]) { minMaxesInShmem[1] = max(xMeta, minMaxesInShmem[1]); };
                if (isToBeExecutedOnActive(active, 3) && anyInGold[0]) { minMaxesInShmem[2] = min(xMeta, minMaxesInShmem[2]); };

                if (isToBeExecutedOnActive(active, 4) && anyInGold[0]) { minMaxesInShmem[3] = max(yMeta, minMaxesInShmem[3]); };
                if (isToBeExecutedOnActive(active, 5) && anyInGold[0]) { minMaxesInShmem[4] = min(yMeta, minMaxesInShmem[4]); };

                if (isToBeExecutedOnActive(active, 6) && anyInGold[0]) { minMaxesInShmem[5] = max(zMeta, minMaxesInShmem[5]); };
                if (isToBeExecutedOnActive(active, 7) && anyInGold[0]) { minMaxesInShmem[6] = min(zMeta, minMaxesInShmem[6]);
               // printf("local fifth %d  \n", minMaxesInShmem[6]);
                };
                active.sync();
               // sync(cta); // just to reduce the warp divergence
                anyInGold[0] = false;




            }
        }

    }
    sync(cta);

    auto active = coalesced_threads();

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        //printf("in minMaxes internal  %d \n", minMaxesInShmem[0]);
        //getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, fbArgs.metaData.minMaxes.Ny, 0, 0)[0] = 61;
        atomicMax(&minMaxes[1], minMaxesInShmem[1]);
        //atomicMax(&minMaxes[1], 2);
       // minMaxes[1] = 0;
    };

    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {

        atomicMin(&minMaxes[2], minMaxesInShmem[2]);
    };

    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        atomicMax(&minMaxes[3], minMaxesInShmem[3]);
    };

    if ((threadIdx.x == 2) && (threadIdx.y == 0)) {
        atomicMin(&minMaxes[4], minMaxesInShmem[4]);
    };



    if (threadIdx.x == 3 && threadIdx.y == 0) {
       atomicMax(&minMaxes[5], minMaxesInShmem[5]);
        //printf(" minMaxesInShmem  %d \n ", minMaxes[5]);
    };

    if (threadIdx.x == 4 && threadIdx.y == 0) {
        atomicMin(&minMaxes[6], minMaxesInShmem[6]);
    };





}





//
//#pragma once
//extern "C" inline bool boolPrepare(ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//
//    boolPrepareKernel <<< fFArgs.blocks, fFArgs.threads >>> (fbArgs);
//
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
