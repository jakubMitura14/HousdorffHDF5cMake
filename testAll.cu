#include "hip/hip_runtime.h"
#include "MainPassesKernels.cu"
//#include "Structs.cu"
#include "UnitTestUtils.cu"







//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
	// threads and blocks for bool kernel
	const int blocks = 17;
	const int xThreadDim = 32;
	const int yThreadDim = 12;
	const dim3 threads = dim3(xThreadDim, yThreadDim);
	// threads and blocks for first metadata pass
	int threadsFirstMetaDataPass = 32;
	int blocksFirstMetaDataPass = 10;



	//datablock dimensions
	const int dbXLength = xThreadDim;
	const int dbYLength = yThreadDim;
	const int dbZLength = 32;



	//threads and blocks for main pass 
	dim3 threadsMainPass = dim3(dbXLength, dbYLength);
	int blocksMainPass = 7;
	//threads and blocks for padding pass 
	dim3 threadsPaddingPass = dim3(32, 11);
	int blocksPaddingPass = 13;
	//threads and blocks for non first metadata passes 
	int threadsOtherMetaDataPasses = 32;
	int blocksOtherMetaDataPasses = 7;


	int minMaxesLength = 17;



	//metadata
	const int metaXLength = 13;
	const int MetaYLength = 13;
	const int MetaZLength = 13;


	const int totalLength = metaXLength * MetaYLength * MetaZLength;
	const int loopMetaTimes = floor(totalLength / blocks);

	/*   int*** h_tensor;
	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

	int i, j, k, value = 0;
	int*** forDebugArr;

	const int dXLength = metaXLength;
	const int dYLength = MetaYLength;
	const int dZLength = MetaZLength;


	const int mainXLength = dbXLength * metaXLength;
	const int mainYLength = dbYLength * MetaYLength;
	const int mainZLength = dbZLength * MetaZLength;


	//main data arrays
	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);

	int*** segmArr;
	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
	MetaDataCPU metaData;
	metaData.metaXLength = metaXLength;
	metaData.MetaYLength = MetaYLength;
	metaData.MetaZLength = MetaZLength;
	metaData.totalMetaLength = totalLength;
	auto fpCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);

	auto fpCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);

	auto fpOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);


	auto minMaxesPointer = alloc_tensorToZeros<unsigned int>(minMaxesLength, 1, 1);

	auto isActiveGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isFullGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isActiveSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isFullSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);

	auto isToBeActivatedGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isToBeActivatedSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);



	auto isToBeValidatedFpPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isToBeValidatedFnPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);



	auto fpC = get3dArrCPU(fpCPointer, metaXLength, MetaYLength, MetaZLength);
	auto fnC = get3dArrCPU(fnCPointer, metaXLength, MetaYLength, MetaZLength);
	auto minMaxes = get3dArrCPU(minMaxesPointer, minMaxesLength, 1, 1);

	auto isToBeValidatedFp = get3dArrCPU(isToBeValidatedFpPointer, metaXLength, MetaYLength, MetaZLength);
	auto isToBeValidatedFn = get3dArrCPU(isToBeValidatedFnPointer, metaXLength, MetaYLength, MetaZLength);

	metaData.fpCount = fpC;
	metaData.fnCount = fnC;
	metaData.minMaxes = minMaxes;

	metaData.fpCounter = get3dArrCPU(fpCounterPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.fnCounter = get3dArrCPU(fnCounterPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.fpOffset = get3dArrCPU(fpOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.fnOffset = get3dArrCPU(fnOffsetPointer, metaXLength, MetaYLength, MetaZLength);;

	metaData.isActiveGold = get3dArrCPU(isActiveGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.isFullGold = get3dArrCPU(isFullGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.isActiveSegm = get3dArrCPU(isActiveSegmPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.isFullSegm = get3dArrCPU(isFullSegmPointer, metaXLength, MetaYLength, MetaZLength);;

	metaData.isToBeActivatedGold = get3dArrCPU(isToBeActivatedGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	metaData.isToBeActivatedSegm = get3dArrCPU(isToBeActivatedSegmPointer, metaXLength, MetaYLength, MetaZLength);;


	metaData.isToBeValidatedFp = isToBeValidatedFp;
	metaData.isToBeValidatedFn = isToBeValidatedFn;


	//int paddingStoreX = metaXLength * 32;
	//int paddingStoreY = MetaYLength * 32;
	//int paddingStoreZ = MetaZLength;

	//auto paddingsStoreGoldPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);
	//auto paddingsStoreSegmPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);

	int workQueueAndRLLength = 200;
	int workQueueWidth = 4;
	int resultListWidth = 5;
	//allocating to semiarbitrrary size 
	auto workQueuePointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, workQueueWidth, 1);
	auto resultListPointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, resultListWidth, 1);
	metaData.workQueue = get3dArrCPU(workQueuePointer, workQueueAndRLLength, workQueueWidth, 1);
	metaData.resultList = get3dArrCPU(resultListPointer, workQueueAndRLLength, resultListWidth, 1);


	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);

	uint32_t*** reducedGold = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegm = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	uint32_t*** reducedGoldRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegmRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	uint32_t*** reducedGoldPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegmPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	// arguments to pass
	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
	forFullBoolPrepArgs.metaData = metaData;
	forFullBoolPrepArgs.numberToLookFor = 2;
	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
	forFullBoolPrepArgs.dbXLength = dbXLength;
	forFullBoolPrepArgs.dbYLength = dbYLength;
	forFullBoolPrepArgs.dbZLength = dbZLength;
	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);

	forFullBoolPrepArgs.reducedGold = get3dArrCPU(reducedGold, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegm = get3dArrCPU(reducedSegm, mainXLength, mainYLength, MetaZLength);

	forFullBoolPrepArgs.reducedGoldRef = get3dArrCPU(reducedGoldRef, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegmRef = get3dArrCPU(reducedSegmRef, mainXLength, mainYLength, MetaZLength);

	forFullBoolPrepArgs.reducedGoldPrev = get3dArrCPU(reducedGoldPrevPointer, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegmPrev = get3dArrCPU(reducedSegmPrevPointer, mainXLength, mainYLength, MetaZLength);


	forFullBoolPrepArgs.threads = threads;
	forFullBoolPrepArgs.blocks = blocks;

	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

	//populate segm  and gold Arr


	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
	auto arrSegmObj = forFullBoolPrepArgs.segmArr;

	// 2 planes with distance 7 relative to each other
	for (int x = 10; x < 50; x++) {
		for (int y = 10; y < 50; y++) {

			setArrCPU(arrGoldObj,8, x, y, 2);

			setArrCPU(arrSegmObj,19, x, y, 2);

		}
	
	}


	mainKernelsRun(forFullBoolPrepArgs);


		 int ii = 7;
	 	printf("global FP count %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
		ii = 8;
	 	printf("global FN count %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
		ii = 9;
	 	printf("workQueueCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
		ii = 10;
	 	printf("resultFP globalCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
		ii = 11;
	 	printf("resultFn globalCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
		ii = 12;
		printf("global offset counter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);

		ii  = 13;
	 	printf("globalIterationNumb %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);

	 //for (int i = 0; i < workQueueAndRLLength; i++) {

		// if (workQueuePointer[0][2][i] > 0) {
		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
		//		 , 0, 0, i
		//		 , workQueuePointer[0][0][i]
		//		 , workQueuePointer[0][1][i]
		//		 , workQueuePointer[0][2][i]
		//		 , workQueuePointer[0][3][i]
		//	 );
		// }

	 //}

	 for (int ji = 0; ji < 5; ji++) {
		 if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
			 printf("result  in point  %d %d %d isGold %d iteration %d \n ", forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji]
				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]
				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji]
				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji]
				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji]);
		 }
	 }




	printf("cleaaning");

	free(isToBeValidatedFpPointer);
	free(isToBeValidatedFnPointer);
	free(metaData.minMaxes.arrP);
	free(metaData.fpCount.arrP);
	free(metaData.fnCount.arrP);
	free(metaData.fpCounter.arrP);
	free(metaData.fnCounter.arrP);
	free(metaData.fpOffset.arrP);
	free(metaData.fnOffset.arrP);

	free(metaData.isActiveGold.arrP);
	free(metaData.isFullGold.arrP);

	free(metaData.isActiveSegm.arrP);
	free(metaData.isFullSegm.arrP);

	free(workQueuePointer);
	free(resultListPointer);

	free(isToBeActivatedGoldPointer);
	free(isToBeActivatedSegmPointer);


	free(forDebugArr);
	free(goldArr);
	free(segmArr);
	free(reducedSegm);
	free(reducedGold);
	free(reducedGoldPrevPointer);
	free(reducedSegmPrevPointer);
	free(reducedGoldRef);
	free(reducedSegmRef);



}













