#include "hip/hip_runtime.h"
#include "MainPassesKernels.cu"
//#include "Structs.cu"
#include "UnitTestUtils.cu"







//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
	// threads and blocks for bool kernel
	const int blocks = 17;
	const int xThreadDim = 32;
	const int yThreadDim = 12;
	const dim3 threads = dim3(xThreadDim, yThreadDim);
	// threads and blocks for first metadata pass
	int threadsFirstMetaDataPass = 32;
	int blocksFirstMetaDataPass = 10;



	//datablock dimensions
	const int dbXLength = xThreadDim;
	const int dbYLength = yThreadDim;
	const int dbZLength = 32;



	//threads and blocks for main pass 
	dim3 threadsMainPass = dim3(dbXLength, dbYLength);
	int blocksMainPass = 7;
	//threads and blocks for padding pass 
	dim3 threadsPaddingPass = dim3(32, 11);
	int blocksPaddingPass = 13;
	//threads and blocks for non first metadata passes 
	int threadsOtherMetaDataPasses = 32;
	int blocksOtherMetaDataPasses = 7;


	int minMaxesLength = 17;



	//metadata
	const int metaXLength = 6;
	const int MetaYLength = 6;
	const int MetaZLength = 10;


	const int totalLength = metaXLength * MetaYLength * MetaZLength;
	const int loopMetaTimes = floor(totalLength / blocks);

	/*   int*** h_tensor;
	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

	int i, j, k, value = 0;
	int*** forDebugArr;

	const int dXLength = 8;
	const int dYLength = 1;
	const int dZLength = 1;


	const int mainXLength = dbXLength * metaXLength;
	const int mainYLength = dbYLength * MetaYLength;
	const int mainZLength = dbZLength * MetaZLength;


	//main data arrays
	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);

	int*** segmArr;
	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
	MetaDataCPU metaData;
	metaData.metaXLength = metaXLength;
	metaData.MetaYLength = MetaYLength;
	metaData.MetaZLength = MetaZLength;
	metaData.totalMetaLength = totalLength;
	auto fpCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);

	auto fpCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);

	auto fpOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
	auto fnOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);


	auto minMaxesPointer = alloc_tensorToZeros<unsigned int>(minMaxesLength, 1, 1);

	auto isActiveGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isFullGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isActiveSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isFullSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);

	auto isToBeActivatedGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isToBeActivatedSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);



	auto isToBeValidatedFpPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
	auto isToBeValidatedFnPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);



	auto fpC = get3dArrCPU(fpCPointer, metaXLength, MetaYLength, MetaZLength);
	auto fnC = get3dArrCPU(fnCPointer, metaXLength, MetaYLength, MetaZLength);
	auto minMaxes = get3dArrCPU(minMaxesPointer, minMaxesLength, 1, 1);

	auto isToBeValidatedFp = get3dArrCPU(isToBeValidatedFpPointer, metaXLength, MetaYLength, MetaZLength);
	auto isToBeValidatedFn = get3dArrCPU(isToBeValidatedFnPointer, metaXLength, MetaYLength, MetaZLength);


	size_t size = sizeof(unsigned int) * 20;
	unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
	metaData.minMaxes = minMaxesCPU;

	//metaData.fpCount = fpC;
	//metaData.fnCount = fnC;
	////metaData.minMaxes = minMaxes;

	//metaData.fpCounter = get3dArrCPU(fpCounterPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.fnCounter = get3dArrCPU(fnCounterPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.fpOffset = get3dArrCPU(fpOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.fnOffset = get3dArrCPU(fnOffsetPointer, metaXLength, MetaYLength, MetaZLength);;

	//metaData.isActiveGold = get3dArrCPU(isActiveGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.isFullGold = get3dArrCPU(isFullGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.isActiveSegm = get3dArrCPU(isActiveSegmPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.isFullSegm = get3dArrCPU(isFullSegmPointer, metaXLength, MetaYLength, MetaZLength);;

	//metaData.isToBeActivatedGold = get3dArrCPU(isToBeActivatedGoldPointer, metaXLength, MetaYLength, MetaZLength);;
	//metaData.isToBeActivatedSegm = get3dArrCPU(isToBeActivatedSegmPointer, metaXLength, MetaYLength, MetaZLength);;


	//metaData.isToBeValidatedFp = isToBeValidatedFp;
	//metaData.isToBeValidatedFn = isToBeValidatedFn;


	//int paddingStoreX = metaXLength * 32;
	//int paddingStoreY = MetaYLength * 32;
	//int paddingStoreZ = MetaZLength;

	//auto paddingsStoreGoldPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);
	//auto paddingsStoreSegmPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);

	int workQueueAndRLLength = 200;
	int workQueueWidth = 4;
	int resultListWidth = 5;
	//allocating to semiarbitrrary size 
	auto workQueuePointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, workQueueWidth, 1);
	//auto resultListPointer = alloc_tensorToZeros<int>(workQueueAndRLLength, resultListWidth, 1);
	metaData.workQueue = get3dArrCPU(workQueuePointer, workQueueAndRLLength, workQueueWidth, 1);
	//metaData.resultList = get3dArrCPU(resultListPointer, workQueueAndRLLength, resultListWidth, 1);


	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);

	uint32_t*** reducedGold = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegm = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	uint32_t*** reducedGoldRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegmRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	uint32_t*** reducedGoldPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
	uint32_t*** reducedSegmPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);

	// arguments to pass
	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
	forFullBoolPrepArgs.metaData = metaData;
	forFullBoolPrepArgs.numberToLookFor = 2;
	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
	forFullBoolPrepArgs.dbXLength = dbXLength;
	forFullBoolPrepArgs.dbYLength = dbYLength;
	forFullBoolPrepArgs.dbZLength = dbZLength;
	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);

	forFullBoolPrepArgs.reducedGold = get3dArrCPU(reducedGold, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegm = get3dArrCPU(reducedSegm, mainXLength, mainYLength, MetaZLength);

	forFullBoolPrepArgs.reducedGoldRef = get3dArrCPU(reducedGoldRef, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegmRef = get3dArrCPU(reducedSegmRef, mainXLength, mainYLength, MetaZLength);

	forFullBoolPrepArgs.reducedGoldPrev = get3dArrCPU(reducedGoldPrevPointer, mainXLength, mainYLength, MetaZLength);
	forFullBoolPrepArgs.reducedSegmPrev = get3dArrCPU(reducedSegmPrevPointer, mainXLength, mainYLength, MetaZLength);


	forFullBoolPrepArgs.threads = threads;
	forFullBoolPrepArgs.blocks = blocks;

	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

	//populate segm  and gold Arr


	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
	auto arrSegmObj = forFullBoolPrepArgs.segmArr;

	// 2 planes with distance 7 relative to each other
	//for (int x = 33; x < 71; x++) {
	//	for (int y = 1; y < 40; y++) {

	//		setArrCPU(arrGoldObj, x, y,70 ,2);

	//		setArrCPU(arrSegmObj,x, y,100 , 2);

	//	}
	//
	//}

	setArrCPU(arrGoldObj, 33, 1, 71, 2);
	setArrCPU(arrGoldObj, 75, 20, 70, 2);

	setArrCPU(arrGoldObj, 33, 1, 110, 2);
	setArrCPU(arrGoldObj, 75, 20, 161, 2);

	/*
	maxX 2  [1]
minX 1  [2]
maxY 1  [3]
minY 0  [4]
maxZ 5  [5]
minZ 2  [6]
	*/


	printf("\n aaa \n");
	mainKernelsRun(forFullBoolPrepArgs);



	i = 1;
	printf("maxX %d  [%d]\n", minMaxesCPU[i], i);
	i = 2;
	printf("minX %d  [%d]\n", minMaxesCPU[i], i);
	i = 3;
	printf("maxY %d  [%d]\n", minMaxesCPU[i], i);
	i = 4;
	printf("minY %d  [%d]\n", minMaxesCPU[i], i);
	i = 5;
	printf("maxZ %d  [%d]\n", minMaxesCPU[i], i);
	i = 6;
	printf("minZ %d  [%d]\n", minMaxesCPU[i], i);

	int ii = 7;
	printf("global FP count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 8;
	printf("global FN count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 9;
	printf("workQueueCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 10;
	printf("resultFP globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 11;
	printf("resultFn globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 12;
	printf("global offset counter %d  [%d]\n", minMaxesCPU[ii], ii);

	ii = 13;
	printf("globalIterationNumb %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 17;
	printf("suum debug %d  [%d]\n", minMaxesCPU[ii], ii);





	i, j, k, value = 0;
	i = 31;
	j = 12;
	for (k = 0; k < MetaZLength; k++) {
		//goldArr[k][j][i] = 1;
		if (reducedGold[k][j][i] > 0) {
			for (int tt = 0; tt < 32; tt++) {
				if ((reducedGold[k][j][i] & (1 << (tt)))) {
					printf("found in reduced fp  [%d]\n", k * 32 + tt);

				}
			}

		}
	}


	//		i, j, k, value = 0;
	//for (i = 0; i < mainXLength; i++) {
	//	for (j = 0; j < mainYLength; j++) {
	//		for (k = 0; k < MetaZLength; k++) {
	//			//goldArr[k][j][i] = 1;
	//			if (reducedGold[k][j][i] > 0) {
	//				for (int tt = 0; tt < 32; tt++) {
	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);

	//					}
	//				}

	//			}
	//		}
	//	}
	//}






	//minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]

	//int sumDebug = 0;
	//for (int ji = 0; ji < 8000; ji++) {
	//	if (forDebugArr[0][0][ji]==1) {
	//		sumDebug += forDebugArr[0][0][ji];
	//		//printf("for debug %d i %d \n", forDebugArr[0][0][ji],ji);
	//	}
	//}
	//printf("\n sumDebug %d \n", sumDebug);


//
//
//	//	for (int ji = 0; ji < minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]; ji++) {
//		for (int ji = 0; ji < 10; ji++) {
//    if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//   	 int x = forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji];
//	 int y = forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji];
//	 int z = forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji];
//	 int isGold = forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji];
//	 int iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji];
//
//	 //uint16_t x = forFullBoolPrepArgs.metaData.resultList.arrP[ji][0][0];
//	 //uint16_t y = forFullBoolPrepArgs.metaData.resultList.arrP[ji][1][0];
//	 //uint16_t z = forFullBoolPrepArgs.metaData.resultList.arrP[ji][2][0];
//	 //uint16_t isGold = forFullBoolPrepArgs.metaData.resultList.arrP[ji][3][0];
//	 //uint16_t iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[ji][4][0];
//
//
//   	 if (iternumb!=9) {
//   		 printf("result  in point  %d %d %d isGold %d iteration %d \n "
//   			 , x
//   			 , y
//   			 , z
//   			 , isGold
//   			 , iternumb);
//   	 }
//   	 else {
//   		 printf("**");
//   	 }
//
//    }
//}





	 //for (int i = 0; i < workQueueAndRLLength; i++) {

		// if (workQueuePointer[0][2][i] > 0) {
		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
		//		 , 0, 0, i
		//		 , workQueuePointer[0][0][i]
		//		 , workQueuePointer[0][1][i]
		//		 , workQueuePointer[0][2][i]
		//		 , workQueuePointer[0][3][i]
		//	 );
		// }

	 //}






	printf("cleaaning");

	//free(isToBeValidatedFpPointer);
	//free(isToBeValidatedFnPointer);
	/*free(metaData.minMaxes);
	free(metaData.fpCount.arrP);
	free(metaData.fnCount.arrP);
	free(metaData.fpCounter.arrP);
	free(metaData.fnCounter.arrP);
	free(metaData.fpOffset.arrP);
	free(metaData.fnOffset.arrP);

	free(metaData.isActiveGold.arrP);
	free(metaData.isFullGold.arrP);

	free(metaData.isActiveSegm.arrP);
	free(metaData.isFullSegm.arrP);*/

	free(workQueuePointer);
	//	free(resultListPointer);

	//free(isToBeActivatedGoldPointer);
	//free(isToBeActivatedSegmPointer);


	free(forDebugArr);
	free(goldArr);
	free(segmArr);
	free(reducedSegm);
	free(reducedGold);
	free(reducedGoldPrevPointer);
	free(reducedSegmPrevPointer);
	free(reducedGoldRef);
	free(reducedSegmRef);



}













