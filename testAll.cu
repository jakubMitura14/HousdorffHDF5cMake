#include "hip/hip_runtime.h"
#include "MainPassesKernels.cu"
//#include "Structs.cu"
#include "UnitTestUtils.cu"
#include "testData.cu"







//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
	// threads and blocks for bool kernel
	const int blocks = 17;
	const int xThreadDim = 32;
	const int yThreadDim = 12;
	const dim3 threads = dim3(xThreadDim, yThreadDim);
	// threads and blocks for first metadata pass
	int threadsFirstMetaDataPass = 32;
	int blocksFirstMetaDataPass = 10;



	//datablock dimensions
	const int dbXLength = xThreadDim;
	const int dbYLength = yThreadDim;
	const int dbZLength = 32;



	//threads and blocks for main pass 
	dim3 threadsMainPass = dim3(dbXLength, dbYLength);
	int blocksMainPass = 7;
	//threads and blocks for padding pass 
	dim3 threadsPaddingPass = dim3(32, 11);
	int blocksPaddingPass = 13;
	//threads and blocks for non first metadata passes 
	int threadsOtherMetaDataPasses = 32;
	int blocksOtherMetaDataPasses = 7;


	int minMaxesLength = 17;



	//metadata
	const int metaXLength = 6;
	const int MetaYLength = 6;
	const int MetaZLength = 10;


	const int totalLength = metaXLength * MetaYLength * MetaZLength;
	const int loopMetaTimes = floor(totalLength / blocks);

	/*   int*** h_tensor;
	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

	int i, j, k, value = 0;
	int*** forDebugArr;

	const int dXLength = 8;
	const int dYLength = 1;
	const int dZLength = 1;


	const int mainXLength = dbXLength * metaXLength;
	const int mainYLength = dbYLength * MetaYLength;
	const int mainZLength = dbZLength * MetaZLength;


	//main data arrays
	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);

	int*** segmArr;
	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
	MetaDataCPU metaData;
	metaData.metaXLength = metaXLength;
	metaData.MetaYLength = MetaYLength;
	metaData.MetaZLength = MetaZLength;
	metaData.totalMetaLength = totalLength;


	size_t size = sizeof(unsigned int) * 20;
	unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
	metaData.minMaxes = minMaxesCPU;

	int workQueueAndRLLength = 200;
	int workQueueWidth = 4;
	int resultListWidth = 5;
	//allocating to semiarbitrrary size 
	auto workQueuePointer = alloc_tensorToZeros<uint32_t>(workQueueAndRLLength, workQueueWidth, 1);


	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);


	// arguments to pass
	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
	forFullBoolPrepArgs.metaData = metaData;
	forFullBoolPrepArgs.numberToLookFor = 2;
	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
	forFullBoolPrepArgs.dbXLength = dbXLength;
	forFullBoolPrepArgs.dbYLength = dbYLength;
	forFullBoolPrepArgs.dbZLength = dbZLength;
	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.threads = threads;
	forFullBoolPrepArgs.blocks = blocks;

	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

	//populate segm  and gold Arr


	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
	auto arrSegmObj = forFullBoolPrepArgs.segmArr;




	//setArrCPU(arrGoldObj, 0, 0, 0, 2);//
	//setArrCPU(arrGoldObj, 8, 8, 6, 2);//

	////setArrCPU(arrSegmObj, 8, 8, 5, 2);//
	//
	//
	//
	//
	//setArrCPU(arrGoldObj, 32, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 31, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 19, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 20, 31, 2);//

	////setArrCPU(arrSegmObj, 38, 38, 35, 2);//
	////setArrCPU(arrGoldObj, 38, 38, 36, 2);//
	////setArrCPU(arrSegmObj, 38, 38, 37, 2);//





	setArrCPU(arrGoldObj, 0, 0, 0, 2);//
//	setArrCPU(arrSegmObj, 0, 0, 0, 2);//

	setArrCPU(arrGoldObj, 39, 5, 5, 2);//

	setArrCPU(arrGoldObj, 31, 5, 5, 2);//

	setArrCPU(arrGoldObj, 32, 7, 7, 2);//


	//setArrCPU(arrGoldObj, 3, 3, 32, 2);//

	//setArrCPU(arrGoldObj, 35, 38, 31, 2);//
	//setArrCPU(arrGoldObj, 35, 38, 39, 2);//

	//setArrCPU(arrSegmObj, 16, 16, 16, 2);//
	//setArrCPU(arrGoldObj, 38, 38, 36, 2);//

	//setArrCPU(arrSegmObj, 8, 8, 5, 2);//




	//setArrCPU(arrGoldObj, 32, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 31, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 19, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 20, 31, 2);//






	/*
	maxX 2  [1]
minX 1  [2]
maxY 1  [3]
minY 0  [4]
maxZ 5  [5]
minZ 2  [6]
	*/


	printf("\n aaa \n");

	uint32_t* resultListPointerMetaCPU;
	uint32_t* resultListPointerLocalCPU;
	uint32_t* resultListPointerIterNumbCPU;
	uint32_t* metaDataArrPointerCPU;
	uint32_t* workQueuePointerCPU;

	uint32_t* reducedResCPU;
	uint32_t* origArrsCPU;
	ForBoolKernelArgs<int> fbArgs= mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
		, resultListPointerLocalCPU, resultListPointerIterNumbCPU
		, metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU
		);

	printFromReduced(fbArgs, reducedResCPU);


	for (int i = 0; i < 5;i++) {
		if (resultListPointerLocalCPU[i]>0 || resultListPointerMetaCPU[i]>0) {
			uint32_t linIdexMeta = resultListPointerMetaCPU[i] - (isGoldOffset * (resultListPointerMetaCPU[i] >= isGoldOffset))-1;
			uint32_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
			uint32_t zMeta = uint32_t(floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength))));
			uint32_t yMeta = uint32_t(floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength)));
			
			uint32_t linLocal = resultListPointerLocalCPU[i];
			uint32_t xLoc = linLocal % fbArgs.dbXLength;
			uint32_t zLoc = uint32_t(floor((float)(linLocal / (32 * fbArgs.dbYLength))));
			uint32_t yLoc = uint32_t(floor((float)((linLocal - ((zLoc * 32 * fbArgs.dbYLength) + xLoc)) / 32)));


			uint32_t x = xMeta * 32 + xLoc;
			uint32_t y= yMeta * fbArgs.dbYLength + yLoc;
			uint32_t z = zMeta * 32 + zLoc;

			printf("resullt linIdexMeta %d x %d y %d z %d  xMeta %d yMeta %d zMeta %d xLoc %d yLoc %d zLoc %d linLocal %d   \n"
				,linIdexMeta
				,x,y,z
				,xMeta,yMeta, zMeta
				,xLoc,yLoc,zLoc
				, linLocal


			);


		
		}
	}





	printf("\n **************************************** \n");

	i = 1;
	printf("maxX %d  [%d]\n", minMaxesCPU[i], i);
	i = 2;
	printf("minX %d  [%d]\n", minMaxesCPU[i], i);
	i = 3;
	printf("maxY %d  [%d]\n", minMaxesCPU[i], i);
	i = 4;
	printf("minY %d  [%d]\n", minMaxesCPU[i], i);
	i = 5;
	printf("maxZ %d  [%d]\n", minMaxesCPU[i], i);
	i = 6;
	printf("minZ %d  [%d]\n", minMaxesCPU[i], i);

	int ii = 7;
	printf("global FP count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 8;
	printf("global FN count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 9;
	printf("workQueueCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 10;
	printf("resultFP globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 11;
	printf("resultFn globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 12;
	printf("global offset counter %d  [%d]\n", minMaxesCPU[ii], ii);

	ii = 13;
	printf("globalIterationNumb %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 17;
	printf("suum debug %d  [%d]\n", minMaxesCPU[ii], ii);





	//i, j, k, value = 0;
	//i = 31;
	//j = 12;
	//for (k = 0; k < MetaZLength; k++) {
	//	goldArr[k][j][i] = 1;
	//	if (reducedGold[k][j][i] > 0) {
	//		for (int tt = 0; tt < 32; tt++) {
	//			if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//				printf("found in reduced fp  [%d]\n", k * 32 + tt);

	//			}
	//		}

	//	}
	//}


	//		i, j, k, value = 0;
	//for (i = 0; i < mainXLength; i++) {
	//	for (j = 0; j < mainYLength; j++) {
	//		for (k = 0; k < MetaZLength; k++) {
	//			//goldArr[k][j][i] = 1;
	//			if (reducedGold[k][j][i] > 0) {
	//				for (int tt = 0; tt < 32; tt++) {
	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);

	//					}
	//				}

	//			}
	//		}
	//	}
	//}






	//minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]

	//int sumDebug = 0;
	//for (int ji = 0; ji < 8000; ji++) {
	//	if (forDebugArr[0][0][ji]==1) {
	//		sumDebug += forDebugArr[0][0][ji];
	//		//printf("for debug %d i %d \n", forDebugArr[0][0][ji],ji);
	//	}
	//}
	//printf("\n sumDebug %d \n", sumDebug);


//
//
//	//	for (int ji = 0; ji < minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]; ji++) {
//		for (int ji = 0; ji < 10; ji++) {
//    if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//   	 int x = forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji];
//	 int y = forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji];
//	 int z = forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji];
//	 int isGold = forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji];
//	 int iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji];
//
//	 //uint32_t x = forFullBoolPrepArgs.metaData.resultList.arrP[ji][0][0];
//	 //uint32_t y = forFullBoolPrepArgs.metaData.resultList.arrP[ji][1][0];
//	 //uint32_t z = forFullBoolPrepArgs.metaData.resultList.arrP[ji][2][0];
//	 //uint32_t isGold = forFullBoolPrepArgs.metaData.resultList.arrP[ji][3][0];
//	 //uint32_t iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[ji][4][0];
//
//
//   	 if (iternumb!=9) {
//   		 printf("result  in point  %d %d %d isGold %d iteration %d \n "
//   			 , x
//   			 , y
//   			 , z
//   			 , isGold
//   			 , iternumb);
//   	 }
//   	 else {
//   		 printf("**");
//   	 }
//
//    }
//}





	 //for (int i = 0; i < workQueueAndRLLength; i++) {

		// if (workQueuePointer[0][2][i] > 0) {
		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
		//		 , 0, 0, i
		//		 , workQueuePointer[0][0][i]
		//		 , workQueuePointer[0][1][i]
		//		 , workQueuePointer[0][2][i]
		//		 , workQueuePointer[0][3][i]
		//	 );
		// }

	 //}






	printf("cleaaning");

	//free(isToBeValidatedFpPointer);
	//free(isToBeValidatedFnPointer);
	/*free(metaData.minMaxes);
	free(metaData.fpCount.arrP);
	free(metaData.fnCount.arrP);
	free(metaData.fpCounter.arrP);
	free(metaData.fnCounter.arrP);
	free(metaData.fpOffset.arrP);
	free(metaData.fnOffset.arrP);

	free(metaData.isActiveGold.arrP);
	free(metaData.isFullGold.arrP);

	free(metaData.isActiveSegm.arrP);
	free(metaData.isFullSegm.arrP);*/

	free(workQueuePointer);
	//	free(resultListPointer);

	//free(isToBeActivatedGoldPointer);
	//free(isToBeActivatedSegmPointer);


	free(forDebugArr);
	free(goldArr);
	free(segmArr);


	free(resultListPointerMetaCPU);
	free(resultListPointerLocalCPU);
	free(resultListPointerIterNumbCPU);
	free(metaDataArrPointerCPU);
	free(workQueuePointerCPU);

	free(reducedResCPU);
	free(origArrsCPU);

}













