#include "hip/hip_runtime.h"
#include "MainPassesKernels.cu"
//#include "Structs.cu"
#include "UnitTestUtils.cu"







//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
	// threads and blocks for bool kernel
	const int blocks = 17;
	const int xThreadDim = 32;
	const int yThreadDim = 12;
	const dim3 threads = dim3(xThreadDim, yThreadDim);
	// threads and blocks for first metadata pass
	int threadsFirstMetaDataPass = 32;
	int blocksFirstMetaDataPass = 10;



	//datablock dimensions
	const int dbXLength = xThreadDim;
	const int dbYLength = yThreadDim;
	const int dbZLength = 32;



	//threads and blocks for main pass 
	dim3 threadsMainPass = dim3(dbXLength, dbYLength);
	int blocksMainPass = 7;
	//threads and blocks for padding pass 
	dim3 threadsPaddingPass = dim3(32, 11);
	int blocksPaddingPass = 13;
	//threads and blocks for non first metadata passes 
	int threadsOtherMetaDataPasses = 32;
	int blocksOtherMetaDataPasses = 7;


	int minMaxesLength = 17;



	//metadata
	const int metaXLength = 6;
	const int MetaYLength = 6;
	const int MetaZLength = 10;


	const int totalLength = metaXLength * MetaYLength * MetaZLength;
	const int loopMetaTimes = floor(totalLength / blocks);

	/*   int*** h_tensor;
	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

	int i, j, k, value = 0;
	int*** forDebugArr;

	const int dXLength = 8;
	const int dYLength = 1;
	const int dZLength = 1;


	const int mainXLength = dbXLength * metaXLength;
	const int mainYLength = dbYLength * MetaYLength;
	const int mainZLength = dbZLength * MetaZLength;


	//main data arrays
	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);

	int*** segmArr;
	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
	MetaDataCPU metaData;
	metaData.metaXLength = metaXLength;
	metaData.MetaYLength = MetaYLength;
	metaData.MetaZLength = MetaZLength;
	metaData.totalMetaLength = totalLength;


	size_t size = sizeof(unsigned int) * 20;
	unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
	metaData.minMaxes = minMaxesCPU;

	int workQueueAndRLLength = 200;
	int workQueueWidth = 4;
	int resultListWidth = 5;
	//allocating to semiarbitrrary size 
	auto workQueuePointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, workQueueWidth, 1);


	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);


	// arguments to pass
	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
	forFullBoolPrepArgs.metaData = metaData;
	forFullBoolPrepArgs.numberToLookFor = 2;
	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
	forFullBoolPrepArgs.dbXLength = dbXLength;
	forFullBoolPrepArgs.dbYLength = dbYLength;
	forFullBoolPrepArgs.dbZLength = dbZLength;
	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.threads = threads;
	forFullBoolPrepArgs.blocks = blocks;

	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

	//populate segm  and gold Arr


	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
	auto arrSegmObj = forFullBoolPrepArgs.segmArr;

	// 2 planes with distance 7 relative to each other
	//for (int x = 33; x < 71; x++) {
	//	for (int y = 1; y < 40; y++) {

	//		setArrCPU(arrGoldObj, x, y,70 ,2);

	//		setArrCPU(arrSegmObj,x, y,100 , 2);

	//	}
	//
	//}

	//setArrCPU(arrGoldObj, 33, 1, 71, 2);
	////setArrCPU(arrGoldObj, 33, 1, 99, 2);
	//setArrCPU(arrGoldObj, 75, 20, 70, 2);//
	//setArrCPU(arrGoldObj, 100, 70, 70, 2);//
	//setArrCPU(arrGoldObj, 99, 70, 70, 2);//



	//setArrCPU(arrSegmObj, 33, 3, 110, 2);//
	setArrCPU(arrSegmObj, 0, 0, 0, 2);//


	//setArrCPU(arrSegmObj, 75, 20, 161, 2);//

	//setArrCPU(arrSegmObj, 75, 21, 161, 2);//




	/*
	maxX 2  [1]
minX 1  [2]
maxY 1  [3]
minY 0  [4]
maxZ 5  [5]
minZ 2  [6]
	*/


	printf("\n aaa \n");
	mainKernelsRun(forFullBoolPrepArgs);


	printf("\n **************************************** \n");

	i = 1;
	printf("maxX %d  [%d]\n", minMaxesCPU[i], i);
	i = 2;
	printf("minX %d  [%d]\n", minMaxesCPU[i], i);
	i = 3;
	printf("maxY %d  [%d]\n", minMaxesCPU[i], i);
	i = 4;
	printf("minY %d  [%d]\n", minMaxesCPU[i], i);
	i = 5;
	printf("maxZ %d  [%d]\n", minMaxesCPU[i], i);
	i = 6;
	printf("minZ %d  [%d]\n", minMaxesCPU[i], i);

	int ii = 7;
	printf("global FP count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 8;
	printf("global FN count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 9;
	printf("workQueueCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 10;
	printf("resultFP globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 11;
	printf("resultFn globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 12;
	printf("global offset counter %d  [%d]\n", minMaxesCPU[ii], ii);

	ii = 13;
	printf("globalIterationNumb %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 17;
	printf("suum debug %d  [%d]\n", minMaxesCPU[ii], ii);





	//i, j, k, value = 0;
	//i = 31;
	//j = 12;
	//for (k = 0; k < MetaZLength; k++) {
	//	goldArr[k][j][i] = 1;
	//	if (reducedGold[k][j][i] > 0) {
	//		for (int tt = 0; tt < 32; tt++) {
	//			if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//				printf("found in reduced fp  [%d]\n", k * 32 + tt);

	//			}
	//		}

	//	}
	//}


	//		i, j, k, value = 0;
	//for (i = 0; i < mainXLength; i++) {
	//	for (j = 0; j < mainYLength; j++) {
	//		for (k = 0; k < MetaZLength; k++) {
	//			//goldArr[k][j][i] = 1;
	//			if (reducedGold[k][j][i] > 0) {
	//				for (int tt = 0; tt < 32; tt++) {
	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);

	//					}
	//				}

	//			}
	//		}
	//	}
	//}






	//minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]

	//int sumDebug = 0;
	//for (int ji = 0; ji < 8000; ji++) {
	//	if (forDebugArr[0][0][ji]==1) {
	//		sumDebug += forDebugArr[0][0][ji];
	//		//printf("for debug %d i %d \n", forDebugArr[0][0][ji],ji);
	//	}
	//}
	//printf("\n sumDebug %d \n", sumDebug);


//
//
//	//	for (int ji = 0; ji < minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]; ji++) {
//		for (int ji = 0; ji < 10; ji++) {
//    if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//   	 int x = forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji];
//	 int y = forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji];
//	 int z = forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji];
//	 int isGold = forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji];
//	 int iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji];
//
//	 //uint16_t x = forFullBoolPrepArgs.metaData.resultList.arrP[ji][0][0];
//	 //uint16_t y = forFullBoolPrepArgs.metaData.resultList.arrP[ji][1][0];
//	 //uint16_t z = forFullBoolPrepArgs.metaData.resultList.arrP[ji][2][0];
//	 //uint16_t isGold = forFullBoolPrepArgs.metaData.resultList.arrP[ji][3][0];
//	 //uint16_t iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[ji][4][0];
//
//
//   	 if (iternumb!=9) {
//   		 printf("result  in point  %d %d %d isGold %d iteration %d \n "
//   			 , x
//   			 , y
//   			 , z
//   			 , isGold
//   			 , iternumb);
//   	 }
//   	 else {
//   		 printf("**");
//   	 }
//
//    }
//}





	 //for (int i = 0; i < workQueueAndRLLength; i++) {

		// if (workQueuePointer[0][2][i] > 0) {
		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
		//		 , 0, 0, i
		//		 , workQueuePointer[0][0][i]
		//		 , workQueuePointer[0][1][i]
		//		 , workQueuePointer[0][2][i]
		//		 , workQueuePointer[0][3][i]
		//	 );
		// }

	 //}






	printf("cleaaning");

	//free(isToBeValidatedFpPointer);
	//free(isToBeValidatedFnPointer);
	/*free(metaData.minMaxes);
	free(metaData.fpCount.arrP);
	free(metaData.fnCount.arrP);
	free(metaData.fpCounter.arrP);
	free(metaData.fnCounter.arrP);
	free(metaData.fpOffset.arrP);
	free(metaData.fnOffset.arrP);

	free(metaData.isActiveGold.arrP);
	free(metaData.isFullGold.arrP);

	free(metaData.isActiveSegm.arrP);
	free(metaData.isFullSegm.arrP);*/

	free(workQueuePointer);
	//	free(resultListPointer);

	//free(isToBeActivatedGoldPointer);
	//free(isToBeActivatedSegmPointer);


	free(forDebugArr);
	free(goldArr);
	free(segmArr);




}













