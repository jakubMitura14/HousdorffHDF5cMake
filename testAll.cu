#include "hip/hip_runtime.h"
#include "MainPassesKernels.cu"
//#include "Structs.cu"
#include "UnitTestUtils.cu"
#include "testData.cu"







//testing loopMeta function in order to execute test unhash proper function in loopMeta
#pragma once
extern "C" inline void testMainPasswes() {
	// threads and blocks for bool kernel
	const int blocks = 17;
	const int xThreadDim = 32;
	const int yThreadDim = 12;
	const dim3 threads = dim3(xThreadDim, yThreadDim);
	// threads and blocks for first metadata pass
	int threadsFirstMetaDataPass = 32;
	int blocksFirstMetaDataPass = 10;



	//datablock dimensions
	const int dbXLength = xThreadDim;
	const int dbYLength = yThreadDim;
	const int dbZLength = 32;



	//threads and blocks for main pass 
	dim3 threadsMainPass = dim3(dbXLength, dbYLength);
	int blocksMainPass = 7;
	//threads and blocks for padding pass 
	dim3 threadsPaddingPass = dim3(32, 11);
	int blocksPaddingPass = 13;
	//threads and blocks for non first metadata passes 
	int threadsOtherMetaDataPasses = 32;
	int blocksOtherMetaDataPasses = 7;


	int minMaxesLength = 20;



	//metadata
	const int metaXLength = 8;//8
	const int MetaYLength = 20;//30
	const int MetaZLength = 8;//8


	const int totalLength = metaXLength * MetaYLength * MetaZLength;
	const int loopMetaTimes = floor(totalLength / blocks);

	/*   int*** h_tensor;
	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/

	int i, j, k, value = 0;
	int*** forDebugArr;

	const int dXLength = 8;
	const int dYLength = 1;
	const int dZLength = 1;


	const int mainXLength = dbXLength * metaXLength;
	const int mainYLength = dbYLength * MetaYLength;
	const int mainZLength = dbZLength * MetaZLength;


	//main data arrays
	bool* goldArr = alloc_tensorToZeros<bool>(mainXLength, mainYLength, mainZLength);

	bool* segmArr = alloc_tensorToZeros<bool>(mainXLength, mainYLength, mainZLength);
	MetaDataCPU metaData;
	metaData.metaXLength = metaXLength;
	metaData.MetaYLength = MetaYLength;
	metaData.MetaZLength = MetaZLength;
	metaData.totalMetaLength = totalLength;


	size_t size = sizeof(unsigned int) * 20;
	unsigned int* minMaxesCPU = (unsigned int*)malloc(size);
	metaData.minMaxes = minMaxesCPU;

	int workQueueAndRLLength = 200;
	int workQueueWidth = 4;
	int resultListWidth = 5;
	//allocating to semiarbitrrary size 
	auto workQueuePointer = alloc_tensorToZeros<uint32_t>(workQueueAndRLLength, workQueueWidth, 1);




	// arguments to pass
	ForFullBoolPrepArgs<bool> forFullBoolPrepArgs;
	forFullBoolPrepArgs.metaData = metaData;
	forFullBoolPrepArgs.numberToLookFor = 2;
	forFullBoolPrepArgs.dbXLength = dbXLength;
	forFullBoolPrepArgs.dbYLength = dbYLength;
	forFullBoolPrepArgs.dbZLength = dbZLength;
	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
	forFullBoolPrepArgs.threads = threads;
	forFullBoolPrepArgs.blocks = blocks;

	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;

	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;

	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;

	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;

	//populate segm  and gold Arr


	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
	auto arrSegmObj = forFullBoolPrepArgs.segmArr;




	//setArrCPU(arrGoldObj, 0, 0, 0, 2);//
	//setArrCPU(arrGoldObj, 8, 8, 6, 2);//

	////setArrCPU(arrSegmObj, 8, 8, 5, 2);//
	//
	//
	//
	//
	//setArrCPU(arrGoldObj, 32, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 31, 20, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 19, 32, 2);//
	//setArrCPU(arrSegmObj, 32, 20, 31, 2);//

	////setArrCPU(arrSegmObj, 38, 38, 35, 2);//
	////setArrCPU(arrGoldObj, 38, 38, 36, 2);//
	////setArrCPU(arrSegmObj, 38, 38, 37, 2);//
	// goldArr[100]=2 ;

	//segmArr[0]=2;



   //setArrCPU(arrGoldObj, 0, 0, 200, 2);//

	//setArrCPU(arrGoldObj, 90, 0, 0, 2);//
	//setArrCPU(arrSegmObj, 0, 0, 0, 2);//
	//setArrCPU(arrGoldObj, 90, 0, 0, 2);//
	//setArrCPU(arrSegmObj, 0, 0, 200, 2);//


	//setArrCPU(arrGoldObj, 90, 1, 0, 2);//
	//setArrCPU(arrGoldObj, 0, 8, 200, 2);//

	//setArrCPU(arrGoldObj, 90, 9, 0, 2);//
	//setArrCPU(arrGoldObj, 0, 19, 200, 2);//

	//setArrCPU(arrGoldObj, 90, 20, 0, 2);//
	//setArrCPU(arrGoldObj, 0, 2, 200, 2);//

	//setArrCPU(arrGoldObj, 90, 8, 0, 2);//
	//setArrCPU(arrGoldObj, 0, 7, 200, 2);//



	setArrCPU(arrGoldObj, 90, 0, 0, true);//
	setArrCPU(arrSegmObj, 0, 0, 0, true);//
	setArrCPU(arrGoldObj, 90, 0, 0, true);//
	setArrCPU(arrSegmObj, 0, 0, 200, true);//


	setArrCPU(arrGoldObj, 90, 1, 0, true);//
	setArrCPU(arrGoldObj, 0, 8, 200, true);//

	setArrCPU(arrGoldObj, 90, 9, 0, true);//
	setArrCPU(arrGoldObj, 0, 19, 200, true);//

	setArrCPU(arrGoldObj, 90, 20, 0, true);//
	setArrCPU(arrGoldObj, 0, 2, 200, true);//

	setArrCPU(arrGoldObj, 90, 8, 0, true);//
	setArrCPU(arrGoldObj, 0, 7, 200, true);//




	int pointsNumber = 0;
	int& pointsNumberRef = pointsNumber;
	forTestPointStruct allPointsA[] = {
		// meta 2,2,2 only gold points not in result after 2 dilataions
	getTestPoint(
	2,2,2//x,y,z
	,true//isGold
	,0,0,0//xMeta,yMeta,Zmeta
	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
	};

	/*
	maxX 2  [1]
minX 1  [2]
maxY 1  [3]
minY 0  [4]
maxZ 5  [5]
minZ 2  [6]
	*/


	printf("\n aaa \n");

	uint32_t* resultListPointerMetaCPU;
	uint32_t* resultListPointerLocalCPU;
	uint32_t* resultListPointerIterNumbCPU;
	uint32_t* metaDataArrPointerCPU;
	uint32_t* workQueuePointerCPU;

	uint32_t* reducedResCPU;
	uint32_t* origArrsCPU;
	
	
	
	ForBoolKernelArgs<bool> fbArgs = mainKernelsRun(forFullBoolPrepArgs, reducedResCPU, resultListPointerMetaCPU
		, resultListPointerLocalCPU, resultListPointerIterNumbCPU
		, metaDataArrPointerCPU, workQueuePointerCPU, origArrsCPU, mainXLength, mainYLength, mainZLength
	);

	//for (uint32_t linIdexMeta = 0; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += 1) {
	//	//we get from linear index  the coordinates of the metadata block of intrest
	//	uint8_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
	//	uint8_t zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
	//	uint8_t yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));

	//	for (int locPos = 32 * fbArgs.dbYLength; locPos < 32 * 2 * fbArgs.dbYLength; locPos++) {
	//		auto col = reducedResCPU[linIdexMeta * fbArgs.metaData.mainArrSectionLength + locPos];
	//		if (col > 0) {
	//			for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
	//				if (isBitAtCPU(col, bitPos)) {
	//					int locPosB = locPos - 32 * fbArgs.dbYLength;
	//					if (bitPos + zMeta * fbArgs.dbZLength>190) {
	//						printf("point segm  set at x %d y %d z %d  \n"
	//							, locPosB % 32 + xMeta * fbArgs.dbXLength
	//							, int(floor((float)(locPosB / 32)) + yMeta * fbArgs.dbYLength)
	//							, bitPos + zMeta * fbArgs.dbZLength
	//						);
	//					}
	//				}
	//			}
	//		}
	//	}
	//}


	//testDilatations(fbArgs, allPointsA, );






	//printFromReduced(fbArgs, reducedResCPU);
	//printIsBlockActiveEtc(fbArgs, metaDataArrPointerCPU, fbArgs.metaData);


	//for (int wQi = 0; wQi < minMaxesCPU[9]; wQi ++ ) {
	//	printf("in work q %d  \n ", workQueuePointerCPU[wQi] - isGoldOffset * (workQueuePointerCPU[wQi] >= isGoldOffset) );
	//}

	//for (int wQi = 0; wQi < 700; wQi++) {
	//	if (metaDataArrPointerCPU[wQi]==1) {
	//		printf("\n in metadaArr i %d  \n ", wQi);
	//	}
	//}

	//info in padding AND range 14 linMeta 2 new block adress 30   inMetadataArrIndex 612
	//	info in padding AND range 15 linMeta 2 new block adress 1   inMetadataArrIndex 32
	//	info in padding AND range 14 linMeta 0 new block adress 28   inMetadataArrIndex 571


	for (int i = 0; i < 5;i++) {
		if (resultListPointerLocalCPU[i]>0 || resultListPointerMetaCPU[i]>0) {
			uint32_t linIdexMeta = resultListPointerMetaCPU[i] - (isGoldOffset * (resultListPointerMetaCPU[i] >= isGoldOffset))-1;
			uint32_t xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
			uint32_t zMeta = uint32_t(floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength))));
			uint32_t yMeta = uint32_t(floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength)));
			
			uint32_t linLocal = resultListPointerLocalCPU[i];
			uint32_t xLoc = linLocal % fbArgs.dbXLength;
			uint32_t zLoc = uint32_t(floor((float)(linLocal / (32 * fbArgs.dbYLength))));
			uint32_t yLoc = uint32_t(floor((float)((linLocal - ((zLoc * 32 * fbArgs.dbYLength) + xLoc)) / 32)));


			uint32_t x = xMeta * 32 + xLoc;
			uint32_t y= yMeta * fbArgs.dbYLength + yLoc;
			uint32_t z = zMeta * 32 + zLoc;
			uint32_t iterNumb  = resultListPointerIterNumbCPU[i];

			printf("resullt linIdexMeta %d x %d y %d z %d  xMeta %d yMeta %d zMeta %d xLoc %d yLoc %d zLoc %d linLocal %d  iterNumb %d \n"
				,linIdexMeta
				,x,y,z
				,xMeta,yMeta, zMeta
				,xLoc,yLoc,zLoc
				, linLocal
				, iterNumb


			);


		
		}
	}





	printf("\n **************************************** \n");

	i = 1;
	printf("maxX %d  [%d]\n", minMaxesCPU[i], i);
	i = 2;
	printf("minX %d  [%d]\n", minMaxesCPU[i], i);
	i = 3;
	printf("maxY %d  [%d]\n", minMaxesCPU[i], i);
	i = 4;
	printf("minY %d  [%d]\n", minMaxesCPU[i], i);
	i = 5;
	printf("maxZ %d  [%d]\n", minMaxesCPU[i], i);
	i = 6;
	printf("minZ %d  [%d]\n", minMaxesCPU[i], i);

	int ii = 7;
	printf("global FP count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 8;
	printf("global FN count %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 9;
	printf("workQueueCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 10;
	printf("resultFP globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 11;
	printf("resultFn globalCounter %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 12;
	printf("global offset counter %d  [%d]\n", minMaxesCPU[ii], ii);

	ii = 13;
	printf("globalIterationNumb %d  [%d]\n", minMaxesCPU[ii], ii);
	ii = 17;
	printf("suum debug %d  [%d]\n", minMaxesCPU[ii], ii);





	//i, j, k, value = 0;
	//i = 31;
	//j = 12;
	//for (k = 0; k < MetaZLength; k++) {
	//	goldArr[k][j][i] = 1;
	//	if (reducedGold[k][j][i] > 0) {
	//		for (int tt = 0; tt < 32; tt++) {
	//			if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//				printf("found in reduced fp  [%d]\n", k * 32 + tt);

	//			}
	//		}

	//	}
	//}


	//		i, j, k, value = 0;
	//for (i = 0; i < mainXLength; i++) {
	//	for (j = 0; j < mainYLength; j++) {
	//		for (k = 0; k < MetaZLength; k++) {
	//			//goldArr[k][j][i] = 1;
	//			if (reducedGold[k][j][i] > 0) {
	//				for (int tt = 0; tt < 32; tt++) {
	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);

	//					}
	//				}

	//			}
	//		}
	//	}
	//}






	//minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]

	//int sumDebug = 0;
	//for (int ji = 0; ji < 8000; ji++) {
	//	if (forDebugArr[0][0][ji]==1) {
	//		sumDebug += forDebugArr[0][0][ji];
	//		//printf("for debug %d i %d \n", forDebugArr[0][0][ji],ji);
	//	}
	//}
	//printf("\n sumDebug %d \n", sumDebug);


//
//
//	//	for (int ji = 0; ji < minMaxes.arrP[0][0][10] + minMaxes.arrP[0][0][11]; ji++) {
//		for (int ji = 0; ji < 10; ji++) {
//    if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//   	 int x = forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji];
//	 int y = forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji];
//	 int z = forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji];
//	 int isGold = forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji];
//	 int iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji];
//
//	 //uint32_t x = forFullBoolPrepArgs.metaData.resultList.arrP[ji][0][0];
//	 //uint32_t y = forFullBoolPrepArgs.metaData.resultList.arrP[ji][1][0];
//	 //uint32_t z = forFullBoolPrepArgs.metaData.resultList.arrP[ji][2][0];
//	 //uint32_t isGold = forFullBoolPrepArgs.metaData.resultList.arrP[ji][3][0];
//	 //uint32_t iternumb = forFullBoolPrepArgs.metaData.resultList.arrP[ji][4][0];
//
//
//   	 if (iternumb!=9) {
//   		 printf("result  in point  %d %d %d isGold %d iteration %d \n "
//   			 , x
//   			 , y
//   			 , z
//   			 , isGold
//   			 , iternumb);
//   	 }
//   	 else {
//   		 printf("**");
//   	 }
//
//    }
//}


	



	 //for (int i = 0; i < workQueueAndRLLength; i++) {

		// if (workQueuePointer[0][2][i] > 0) {
		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
		//		 , 0, 0, i
		//		 , workQueuePointer[0][0][i]
		//		 , workQueuePointer[0][1][i]
		//		 , workQueuePointer[0][2][i]
		//		 , workQueuePointer[0][3][i]
		//	 );
		// }

	 //}






	printf("cleaaning");



	free(goldArr);
	free(segmArr);


	free(resultListPointerMetaCPU);
	free(resultListPointerLocalCPU);
	free(resultListPointerIterNumbCPU);
	free(metaDataArrPointerCPU);
	free(workQueuePointerCPU);

	free(reducedResCPU);
	free(origArrsCPU);



}













