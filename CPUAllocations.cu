#include "hip/hip_runtime.h"
#include <cstdint>
#include "Structs.cu"

/*
* from https://stackoverflow.com/questions/23310520/using-cudamemcpy3d-to-transfer-pointer
*/
#pragma once
template <typename T>
inline T*** alloc_tensor(int Nx, int Ny, int Nz) {
    int i, j;
    T*** tensor;

    tensor = (T***)malloc((size_t)(Nz * sizeof(T**)));
    tensor[0] = (T**)malloc((size_t)(Nz * Ny * sizeof(T*)));
    tensor[0][0] = (T*)malloc((size_t)(Nz * Ny * Nx * sizeof(T)));

    for (j = 1; j < Ny; j++)
        tensor[0][j] = tensor[0][j - 1] + Nx;
    for (i = 1; i < Nz; i++) {
        tensor[i] = tensor[i - 1] + Ny;
        tensor[i][0] = tensor[i - 1][0] + Ny * Nx;
        for (j = 1; j < Ny; j++)
            tensor[i][j] = tensor[i][j - 1] + Nx;
    }

    return tensor;

    /*    int i, j;
    T*** tensor;

    tensor = (T***)malloc((size_t)(Nx * sizeof(T**)));
    tensor[0] = (T**)malloc((size_t)(Nx * Ny * sizeof(T*)));
    tensor[0][0] = (T*)malloc((size_t)(Nx * Ny * Nz * sizeof(T)));

    for (j = 1; j < Ny; j++)
        tensor[0][j] = tensor[0][j - 1] + Nz;
    for (i = 1; i < Nx; i++) {
        tensor[i] = tensor[i - 1] + Ny;
        tensor[i][0] = tensor[i - 1][0] + Ny * Nz;
        for (j = 1; j < Ny; j++)
            tensor[i][j] = tensor[i][j - 1] + Nz;
    }

    return tensor;*/
}



/*
* from https://stackoverflow.com/questions/23310520/using-cudamemcpy3d-to-transfer-pointer
*/
#pragma once
template <typename TC>
inline TC*** alloc_tensorToZeros(int Nx, int Ny, int Nz) {
    int i, j;
    TC*** tensor;



    tensor = (TC***)calloc(Nz, sizeof(TC**));
    tensor[0] = (TC**)calloc(Nz * Ny, sizeof(TC*));
    tensor[0][0] = (TC*)calloc(Nz * Ny * Nx, sizeof(TC));


    //tensor = (TC***)calloc((size_t)(Nz , sizeof(TC**)));
    //tensor[0] = (TC**)calloc((size_t)(Nz * Ny , sizeof(TC*)));
    //tensor[0][0] = (TC*)calloc((size_t)(Nz * Ny * Nx , sizeof(TC)));

    for (j = 1; j < Ny; j++)
        tensor[0][j] = tensor[0][j - 1] + Nx;
    for (i = 1; i < Nz; i++) {
        tensor[i] = tensor[i - 1] + Ny;
        tensor[i][0] = tensor[i - 1][0] + Ny * Nx;
        for (j = 1; j < Ny; j++)
            tensor[i][j] = tensor[i][j - 1] + Nx;
    }

    return tensor;
    //int i, j;
    //TC*** tensor;

    //tensor = (TC***)calloc(Nx, sizeof(TC**));
    //tensor[0] = (TC**)calloc(Nx * Ny, sizeof(TC*));
    //tensor[0][0] = (TC*)calloc(Nx * Ny * Nz, sizeof(TC));

    //for (j = 1; j < Ny; j++)
    //    tensor[0][j] = tensor[0][j - 1] + Nz;
    //for (i = 1; i < Nx; i++) {
    //    tensor[i] = tensor[i - 1] + Ny;
    //    tensor[i][0] = tensor[i - 1][0] + Ny * Nz;
    //    for (j = 1; j < Ny; j++)
    //        tensor[i][j] = tensor[i][j - 1] + Nz;
    //}

    //return tensor;
}



#pragma once
template <typename EEY>
array3dWithDimsCPU<EEY>  get3dArrCPU(EEY*** arrP, int Nx, int Ny, int Nz) {
    array3dWithDimsCPU<EEY> res;
    res.Nx = Nx;
    res.Ny = Ny;
    res.Nz = Nz;
    res.arrP = arrP;

    return res;
}
