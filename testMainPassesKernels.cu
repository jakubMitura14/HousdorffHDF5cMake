
#include <hip/hip_runtime.h>
//#include "MainPassesKernels.cu"
////#include "Structs.cu"
//#include "UnitTestUtils.cu"
//
//
//
//
//
//
//
////testing loopMeta function in order to execute test unhash proper function in loopMeta
//#pragma once
//extern "C" inline void testMainPasswes() {
//	// threads and blocks for bool kernel
//	const int blocks = 17;
//	const int xThreadDim = 32;
//	const int yThreadDim = 12;
//	const dim3 threads = dim3(xThreadDim, yThreadDim);
//	// threads and blocks for first metadata pass
//	int threadsFirstMetaDataPass = 32;
//	int blocksFirstMetaDataPass = 10;
//	
//	
//
//	//datablock dimensions
//	const int dbXLength = xThreadDim;
//	const int dbYLength = yThreadDim;
//	const int dbZLength = 32;
//	
//	
//	
//	//threads and blocks for main pass 
//	dim3 threadsMainPass= dim3(dbXLength, dbYLength);
//	int blocksMainPass =7;
//	//threads and blocks for padding pass 
//	dim3 threadsPaddingPass = dim3(32, 11);
//	int blocksPaddingPass=13;
//	//threads and blocks for non first metadata passes 
//	int threadsOtherMetaDataPasses=32;
//	int blocksOtherMetaDataPasses=7;
//
//
//	int minMaxesLength = 17;
//
//
//
//	//metadata
//	const int metaXLength = 13;
//	const int MetaYLength = 13;
//	const int MetaZLength = 13;
//
//
//	const int totalLength = metaXLength * MetaYLength * MetaZLength;
//	const int loopMetaTimes = floor(totalLength / blocks);
//
//	/*   int*** h_tensor;
//	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/
//
//	int i, j, k, value = 0;
//	int*** forDebugArr;
//
//	const int dXLength = metaXLength;
//	const int dYLength = MetaYLength;
//	const int dZLength = MetaZLength;
//
//
//	const int mainXLength = dbXLength * metaXLength;
//	const int mainYLength = dbYLength * MetaYLength;
//	const int mainZLength = dbZLength * MetaZLength;
//
//
//	//main data arrays
//	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//
//	int*** segmArr;
//	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//	MetaDataCPU metaData;
//	metaData.metaXLength = metaXLength;
//	metaData.MetaYLength = MetaYLength;
//	metaData.MetaZLength = MetaZLength;
//	metaData.totalMetaLength = totalLength;
//	auto fpCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//
//	auto fpCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnCounterPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//
//	auto fpOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnOffsetPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//
//
//	auto minMaxesPointer = alloc_tensorToZeros<unsigned int>(minMaxesLength, 1, 1);
//
//	auto isActiveGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//	auto isFullGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//	auto isActiveSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//	auto isFullSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//
//	auto isToBeActivatedGoldPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//	auto isToBeActivatedSegmPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//
//
//
//	auto isToBeValidatedFpPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//	auto isToBeValidatedFnPointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//
//
//
//	auto fpC = get3dArrCPU(fpCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto fnC = get3dArrCPU(fnCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto minMaxes = get3dArrCPU(minMaxesPointer, minMaxesLength, 1, 1);
//
//	auto isToBeValidatedFp = get3dArrCPU(isToBeValidatedFpPointer, metaXLength, MetaYLength, MetaZLength);
//	auto isToBeValidatedFn = get3dArrCPU(isToBeValidatedFnPointer, metaXLength, MetaYLength, MetaZLength);
//
//	metaData.fpCount = fpC;
//	metaData.fnCount = fnC;
//	metaData.minMaxes = minMaxes;
//
//	metaData.fpCounter = get3dArrCPU(fpCounterPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fnCounter = get3dArrCPU(fnCounterPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fpOffset = get3dArrCPU(fpOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.fnOffset = get3dArrCPU(fnOffsetPointer, metaXLength, MetaYLength, MetaZLength);;
//
//	metaData.isActiveGold = get3dArrCPU(isActiveGoldPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.isFullGold = get3dArrCPU(isFullGoldPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.isActiveSegm = get3dArrCPU(isActiveSegmPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.isFullSegm = get3dArrCPU(isFullSegmPointer, metaXLength, MetaYLength, MetaZLength);;
//
//	metaData.isToBeActivatedGold = get3dArrCPU(isToBeActivatedGoldPointer, metaXLength, MetaYLength, MetaZLength);;
//	metaData.isToBeActivatedSegm = get3dArrCPU(isToBeActivatedSegmPointer, metaXLength, MetaYLength, MetaZLength);;
//
//
//	metaData.isToBeValidatedFp = isToBeValidatedFp;
//	metaData.isToBeValidatedFn = isToBeValidatedFn;
//
//
//	//int paddingStoreX = metaXLength * 32;
//	//int paddingStoreY = MetaYLength * 32;
//	//int paddingStoreZ = MetaZLength;
//
//	//auto paddingsStoreGoldPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);
//	//auto paddingsStoreSegmPointer = alloc_tensorToZeros<uint8_t>(paddingStoreX, paddingStoreY, paddingStoreZ);
//
//	int workQueueAndRLLength = 200;
//	int workQueueWidth = 4;
//	int resultListWidth = 5;
//	//allocating to semiarbitrrary size 
//	auto workQueuePointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, workQueueWidth, 1);
//	auto resultListPointer = alloc_tensorToZeros<uint16_t>(workQueueAndRLLength, resultListWidth, 1);
//	metaData.workQueue = get3dArrCPU(workQueuePointer, workQueueAndRLLength, workQueueWidth, 1);
//	metaData.resultList = get3dArrCPU(resultListPointer, workQueueAndRLLength, resultListWidth, 1);
//	
//
//	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);
//
//	uint32_t*** reducedGold = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//	uint32_t*** reducedSegm = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//
//	uint32_t*** reducedGoldRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//	uint32_t*** reducedSegmRef = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//
//	uint32_t*** reducedGoldPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//	uint32_t*** reducedSegmPrevPointer = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//
//	// arguments to pass
//	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
//	forFullBoolPrepArgs.metaData = metaData;
//	forFullBoolPrepArgs.numberToLookFor = 2;
//	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
//	forFullBoolPrepArgs.dbXLength = dbXLength;
//	forFullBoolPrepArgs.dbYLength = dbYLength;
//	forFullBoolPrepArgs.dbZLength = dbZLength;
//	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
//	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
//
//	forFullBoolPrepArgs.reducedGold = get3dArrCPU(reducedGold, mainXLength, mainYLength, MetaZLength);
//	forFullBoolPrepArgs.reducedSegm = get3dArrCPU(reducedSegm, mainXLength, mainYLength, MetaZLength);
//
//	forFullBoolPrepArgs.reducedGoldRef = get3dArrCPU(reducedGoldRef, mainXLength, mainYLength, MetaZLength);
//	forFullBoolPrepArgs.reducedSegmRef = get3dArrCPU(reducedSegmRef, mainXLength, mainYLength, MetaZLength);
//
//	forFullBoolPrepArgs.reducedGoldPrev = get3dArrCPU(reducedGoldPrevPointer, mainXLength, mainYLength, MetaZLength);
//	forFullBoolPrepArgs.reducedSegmPrev = get3dArrCPU(reducedSegmPrevPointer, mainXLength, mainYLength, MetaZLength);
//
//
//	forFullBoolPrepArgs.threads = threads;
//	forFullBoolPrepArgs.blocks = blocks;
//
//	forFullBoolPrepArgs.threadsFirstMetaDataPass = threadsFirstMetaDataPass;
//	forFullBoolPrepArgs.blocksFirstMetaDataPass = blocksFirstMetaDataPass;
//
//	forFullBoolPrepArgs.threadsMainPass = threadsMainPass;
//	forFullBoolPrepArgs.blocksMainPass = blocksMainPass;
//
//	forFullBoolPrepArgs.threadsPaddingPass = threadsPaddingPass;
//	forFullBoolPrepArgs.blocksPaddingPass = blocksPaddingPass;
//
//	forFullBoolPrepArgs.threadsOtherMetaDataPasses = threadsOtherMetaDataPasses;
//	forFullBoolPrepArgs.blocksOtherMetaDataPasses = blocksOtherMetaDataPasses;
//
//	//populate segm  and gold Arr
//
//
//	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
//	auto arrSegmObj = forFullBoolPrepArgs.segmArr;
//
//
//
//	//printf("mainXLength %d mainYLength %d mainZLength %d \n", mainXLength, mainYLength, mainZLength);
//
//
//
//
//	//assert(("There are five lights", 2 + 2 == 5));
////
////	int i, j, k, value = 0;
////	for (i = 0; i < mainXLength; i++) {
////		for (j = 0; j < mainYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (reducedGold[k][j][i] > 0) {
////					for (int tt = 0; tt < 32; tt++) {
////						if ((reducedGold[k][j][i] & (1 << (tt)))) {
////							printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);
////
////						}
////					}
////
////				}
////			}
////		}
////	}
////
////	for (i = 0; i < mainXLength; i++) {
////		for (j = 0; j < mainYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] > 0) {
////					for (int tt = 0; tt < 32; tt++) {
////						if ((forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] & (1 << (tt)))) {
////							printf("found in reduced fn [%d][%d][%d]\n", i, j, k * 32 + tt);
////						}
////					}
////				}
////			}
////		}
////	}
////
////	i, j, k, value = 0;
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (metaData.isActiveGold.arrP[k][j][i]) {
////					printf("found as Active in gold  [%d][%d][%d]\n", i, j, k);
////				}
////			}
////		}
////	};
////
////	i, j, k, value = 0;
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (metaData.isActiveSegm.arrP[k][j][i]) {
////					printf("found as Active in segm  [%d][%d][%d]\n", i, j, k);
////				}
////			}
////		}
////	};
////
////	i, j, k, value = 0;
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (fpC.arrP[k][j][i] > 0) {
////					printf("found Fp %d  [%d][%d][%d]\n", fpC.arrP[k][j][i], i, j, k);
////				}
////			}
////		}
////	};
////
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (fnC.arrP[k][j][i] > 0) {
////					printf("found Fn %d  [%d][%d][%d]\n", fnC.arrP[k][j][i], i, j, k);
////				}
////			}
////		}
////	};
////
////	i = 1;
////	printf("maxX %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 2;
////	printf("minX %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 3;
////	printf("maxY %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 4;
////	printf("minY %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 5;
////	printf("maxZ %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 6;
////	printf("minZ %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 7;
////	printf("global FP count %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 8;
////	printf("global FN count %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 9;
////	printf("workQueueCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 10;
////	printf("resultFP globalCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 11;
////	printf("resultFn globalCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 12;
////	printf("global FPandFn offset %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////	i = 13;
////	printf("globalIterationNumb %d  [%d]\n", minMaxes.arrP[0][0][i], i);
////
////	for (i = 0; i < mainXLength; i++) {
////		for (j = 0; j < mainYLength; j++) {
////			for (k = 0; k < mainZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (goldArr[k][j][i] > 0) {
////					printf("segmArr[%d][%d][%d] = %d\n", i, j, k, goldArr[k][j][i]);
////				}
////			}
////		}
////	}
////
////
////	printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! firstMetaPass!!!!!!!!!!!!!!!!!!!!!\n\n");
////
////	i, j, k, value = 0;
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (metaData.fpOffset.arrP[k][j][i] > 0) {
////					printf("Offsets Fp %d  [%d][%d][%d]\n", metaData.fpOffset.arrP[k][j][i], i, j, k);
////				}
////			}
////		}
////    };
////
////
////	for (i = 0; i < metaXLength; i++) {
////		for (j = 0; j < MetaYLength; j++) {
////			for (k = 0; k < MetaZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				if (metaData.fnOffset.arrP[k][j][i] > 0) {
////					printf("Offsets Fn %d  [%d][%d][%d]\n", metaData.fnOffset.arrP[k][j][i], i, j, k);
////				}
////			}
////		}
////	};
////
////
////
////
////
////	for (i = 0; i < workQueueAndRLLength; i++) {
////
////		goldArr[k][j][i] = 1;
////		if (workQueuePointer[0][0][i] > 0) {
////			printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
////				, 0, 0, i
////				, workQueuePointer[0][0][i]
////				, workQueuePointer[0][1][i]
////				, workQueuePointer[0][2][i]
////				, workQueuePointer[0][3][i]
////			);
////		}
////
////	}
////
////	printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! main pass kernel !!!!!!!!!!!!!!!!!!!!!\n\n");
////	/*
////	need to test up, down , left , right dilatations given it will not get over the edge of data block
////	check is block correctly set as full
////	check do the results are added to the res list
////	check weather fp and fn counters are updated correctly
////	check is prev reducesd are set corrctly
////	
////	*/
////for (i = 0; i < mainXLength; i++) {
////	for (j = 0; j < mainYLength; j++) {
////		for (k = 0; k < MetaZLength; k++) {
////			 k = 5;
////		if (reducedSegm[k][j][i] > 0) {
////			for (int tt = 0; tt < 32; tt++) {
////				if ((reducedSegm[k][j][i] & (1 << (tt)))) {
////					printf("found in reduced segm  [%d][%d][%d]\n", i, j, k * 32 + tt);
////				}
////			}
////		}
////		}
////	}
////}
////
////
////
////
////for (i = 0; i < mainXLength; i++) {
////	for (j = 0; j < mainYLength; j++) {
////		for (k = 0; k < MetaZLength; k++) {
////			if (forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] > 0) {
////				for (int tt = 0; tt < 32; tt++) {
////					if ((forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] & (1 << (tt)))) {
////						printf("found in reduced fn [%d][%d][%d]\n", i, j, k * 32 + tt);
////
////					}
////				}
////
////
////			}
////		}
////	}
////}
////
////
////
////
////
////		for (i = 0; i < dXLength; i++) {
////		for (j = 0; j < dYLength; j++) {
////			for (k = 0; k < dZLength; k++) {
////				//goldArr[k][j][i] = 1;
////				
////					printf("found in forDebugArr %d  [%d][%d][%d]\n", forDebugArr[k][j][i], i, j, k);
////
////
////			}
////		}
////	}
//int pointsNumber = 0;
//int metasNumber =0;
//
//int& pointsNumberRef = pointsNumber;
//int& metasNumberRef = metasNumber;
//	printf("teeests");
//	/////////////
//	/////define Test points 
//	forTestPointStruct allPointsA[]={
//
//
//
//	// inside the block
//	// meta 2,2,2 only gold points not in result after 2 dilataions
//	getTestPoint(
//	2,5,8//x,y,z
//	,true//isGold
//	,2,2,2//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
//	,getTestPoint(
//	3,3,9//x,y,z
//	,true//isGold
//	,2,2,2//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
//	,getTestPoint(
//	1,5,3//x,y,z
//	,true//isGold
//	,2,2,2//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
//	// block 0 corner 0 
//	,getTestPoint(
//	0,0,1//x,y,z
//	,true//isGold
//	,0,0,0//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//	//lpwer right corner	
//	,getTestPoint(
//	dbXLength-2,dbYLength-2,dbZLength-2//x,y,z
//	,true//isGold
//	,metaXLength-1,MetaYLength-1,MetaZLength-1//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
//	// block 0 corner 0 
//	,getTestPoint(
//	0,0,0//x,y,z
//	,false//isGold
//	,0,0,0//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//	//lpwer right corner	
//	,getTestPoint(
//	dbXLength-2,dbYLength-2,dbZLength-2//x,y,z
//	,false//isGold
//	,metaXLength-1,MetaYLength-1,MetaZLength-1//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)
//	//// some overlapping  voxels - should lead to dilatation but not add to fp or fn 	
//	,getTestPoint(
//	5,6,7//x,y,z
//	,false//isGold
//	,3,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,true
//	)
//	,getTestPoint(
//	9,11,7//x,y,z
//	,false//isGold
//	,3,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,true)	,
//		
//
//
//	//now some points that should be covered by first dilatation		
//	getTestPoint(
//	9,11,7//x,y,z
//	,false//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)	
//
//	,getTestPoint(
//	9,11,8//x,y,z
//	,true//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//		
//		
//	,getTestPoint(
//	9,3,7//x,y,z
//	,false//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//	,getTestPoint(
//	9,2,7//x,y,z
//	,true//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//		
//	,getTestPoint(
//	9,5,7//x,y,z
//	,false//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//	,getTestPoint(
//	9,6,7//x,y,z
//	,true//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//		
//	,getTestPoint(
//	2,3,7//x,y,z
//	,false//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef,false, true)
//	,getTestPoint(
//	3,3,7//x,y,z
//	,true//isGold
//	,7,4,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef, false, true)
//		
//		
//		
//	//now some points that should be covered by second dilatation		
//	,getTestPoint(
//	9,11,7//x,y,z
//	,false//isGold
//	,7,2,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef, false, false, true)
//	,getTestPoint(
//	9,11,9//x,y,z
//	,true//isGold
//	,7,2,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef, false, false, true)
//		
//		
//	,getTestPoint(
//	9,3,7//x,y,z
//	,false//isGold
//	,7,2,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef, false, false, true)
//	,getTestPoint(
//	9,1,7//x,y,z
//	,true//isGold
//	,7,2,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef, false, false, true)	,
//		
//		
//		
//	/*now specifically we will get some points on the borders  to establish if they dilatate properly
//
//
//
//
//
//	//top*/
//        getTestPoint(
//	2,2,0//x,y,z
//	,false//isGold
//	,0,0,2//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef),	
//
//
//			////top*/
//			//getTestPoint(
//			//	2, 2, 1//x,y,z
//			//	, true//isGold
//			//	, 0, 0, 2//xMeta,yMeta,Zmeta
//			//	, dbXLength, dbYLength, dbZLength, pointsNumberRef),
//	
//
//			////getTestPoint(
//			////	2, 2, 9//x,y,z
//			////	, false//isGold
//			////	, 0, 0, 2//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef),
//			//getTestPoint(
//			//	2, 2, 15//x,y,z
//			//	, true//isGold
//			//	, 0, 0, 2//xMeta,yMeta,Zmeta
//			//	, dbXLength, dbYLength, dbZLength, pointsNumberRef),
//
//			//getTestPoint(
//			//	2, 2, 19//x,y,z
//			//	, true//isGold
//			//	, 0, 0, 2//xMeta,yMeta,Zmeta
//			//	, dbXLength, dbYLength, dbZLength, pointsNumberRef),
//
//
//	//bottom	
//        getTestPoint(
//	2,2,dbZLength-1//x,y,z
//	,false//isGold
//	,0,0,4//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	,
//		
//	//left	
//        getTestPoint(
//	0,2,2//x,y,z
//	,false//isGold
//	,8,0,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	,		
//		
//	//right	
//        getTestPoint(
//	dbXLength-1,3,7//x,y,z
//	,false//isGold
//	,0,0,8//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//		
//		
//	//anterior	
//        ,getTestPoint(
//	9,dbYLength-1,7//x,y,z
//	,false//isGold
//	,0,0,10//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//		
//	//posterior	
//        ,getTestPoint(
//	9,0,7//x,y,z
//	,false//isGold
//	,2,2,4//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//		
//
//
//	
//
//		
//			////top
//			//getTestPoint(
//			//	2, 2, 1//x,y,z
//			//	, true//isGold
//			//	, 0, 0, 5//xMeta,yMeta,Zmeta
//			//	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//
//
//			//////bottom	
//			////, getTestPoint(
//			////	2, 9, 7//x,y,z
//			////	, true//isGold
//			////	, 0, 0, 11//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//			//////left	
//			////, getTestPoint(
//			////	2, 2, 2//x,y,z
//			////	, true//isGold
//			////	, 0, 0, 11//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//			//////right	
//			////, getTestPoint(
//			////	2, 3, 7//x,y,z
//			////	, true//isGold
//			////	, 0, 0, 11//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//
//			//////anterior	
//			////, getTestPoint(
//			////	9, 2, 7//x,y,z
//			////	, true//isGold
//			////	, 0, 1, 7//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//			//////posterior	
//			////, getTestPoint(
//			////	9, 2, 7//x,y,z
//			////	, true//isGold
//			////	, 2, 1, 7//xMeta,yMeta,Zmeta
//			////	, dbXLength, dbYLength, dbZLength, pointsNumberRef)
//
//
//
//
//		
//	//left up anterior corner	
//        ,getTestPoint(
//	0,dbYLength-1,0//x,y,z
//	,false//isGold
//	,2,2,6//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//		
//		
//	//right up anterior corner	
//        ,getTestPoint(
//	dbXLength-1,dbYLength-1,0//x,y,z
//	,false//isGold
//	,2,2,8//xMeta,yMeta,Zmeta
// 	,dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//		
// 	//left down anterior corner	
// //        ,getTestPoint(
// //	0,dbYLength-1,dbZLength-1//x,y,z
// //	,false//isGold
// //	,2,2,10//xMeta,yMeta,Zmeta
// //	dbXLength,dbYLength,dbZLength,pointsNumberRef)			
//	//	
// //	//right dow anterior  corner	
// //        ,getTestPoint(
// //	dbXLength-1,dbYLength-1,dbZLength-1//x,y,z
// //	,false//isGold
// //	,4,4,2//xMeta,yMeta,Zmeta
// //	dbXLength,dbYLength,dbZLength,pointsNumberRef)			
//	//	
//
//	//	
//	//	
//	//	
// //	//left up posterior corner	
// //        ,getTestPoint(
// //	0,0,0//x,y,z
// //	,false//isGold
// //	,4,4,4//xMeta,yMeta,Zmeta
// //	dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//	//	
//	//	
// //	//right up posterior corner	
// //        ,getTestPoint(
// //	dbXLength-1,0,0//x,y,z
// //	,false//isGold
// //	,7,2,6//xMeta,yMeta,Zmeta
// //	dbXLength,dbYLength,dbZLength,pointsNumberRef)	
//	//	
// //	//left down posterior corner	
// //        ,getTestPoint(
// //	0,0,dbZLength-1//x,y,z
// //	,false//isGold
// //	,7,2,6//xMeta,yMeta,Zmeta
// //	dbXLength,dbYLength,dbZLength,pointsNumberRef)			
//	//	
//	//right dow posterior  corner	
//        ,getTestPoint(
//	dbXLength-1,0,dbZLength-1//x,y,z
//	,false//isGold
//	,4,4,4//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)			
//				
//	//rshould be activated aafter two dilatations
//        ,getTestPoint(
//	1,1,1//x,y,z
//	,false//isGold
//	,4,4,8//xMeta,yMeta,Zmeta
//	,dbXLength,dbYLength,dbZLength,pointsNumberRef)			
//
//
//
//
//
//		
//		};// list holding most points
//	
//
//
//
//
//	
//	//now we neeed additionally to supply some block that will be full
//	// 5,5,5 full at start	
//		for (i = 0; i < dbXLength; i++) {
//			for (j = 0; j < dbYLength; j++) {
//				for (k = 0; k < dbZLength; k++) {
//					setArrCPU(arrSegmObj, dbXLength * 5 + i, dbYLength * 5 + j, dbZLength * 5 + k, 2,false);
//				}
//			}
//		};
//
//	// 5,7,7 full after one dil
//for (i = 1; i < dbXLength; i++) {
//	for (j = 0; j < dbYLength; j++) {
//		for (k = 0; k < dbZLength; k++) {
//			setArrCPU(arrSegmObj, dbXLength * 5 + i, dbYLength * 7 + j, dbZLength * 7 + k, 2, false);
//		}
//	}
//};
//
//			// 5,7,10 full after two dil
//for (i = 2; i < dbXLength - 2; i++) {
//	for (j = 0; j < dbYLength; j++) {
//		for (k = 0; k < dbZLength; k++) {
//			setArrCPU(arrSegmObj, dbXLength * 5 + i, dbYLength * 7 + j, dbZLength * 10 + k, 2, false);
//		}
//	}
//};
//		
//
//		
//
//
//
//
//		
//
//
//		forTestMetaDataStruct a0 = getMetdataTestStruct(metasNumberRef, 0, 0, 0, 1, 1);
//		a0.fpConterAfterOneDil = 1;
//		a0.fnConterAfterOneDil = 1;
//		a0.isToBeValidatedFpAfterOneIter = false;
//
//
//	forTestMetaDataStruct a1 = getMetdataTestStruct(metasNumberRef,3, 4, 6,2);
//	a1.isToBeValidatedFpAfterOneIter = false;
//	a1.isToBeValidatedFpAfterTwoIter = false;
//	a1.isToBeValidatedFnAfterOneIter = false;
//	a1.isToBeValidatedFnAfterTwoIter = false;
//
//	forTestMetaDataStruct a2 = getMetdataTestStruct(metasNumberRef,7, 4, 6);
//	a2.fnCount = 4;
//	a2.fpCount = 4;
//	a2.fpConterAfterOneDil = 4;
//	a2.fnConterAfterOneDil = 4;
//
//	a2.fpConterAfterTwoDil = 4;
//	a2.fnConterAfterTwoDil = 4;
//
//	a2.isToBeValidatedFpAfterOneIter = false;
//	a2.isToBeValidatedFpAfterTwoIter = false;
//	a2.isToBeValidatedFnAfterOneIter = false;
//	a2.isToBeValidatedFnAfterTwoIter = false;
//
//	forTestMetaDataStruct a3 = getMetdataTestStruct(metasNumberRef,7, 2, 6);
//	a3.fnCount = 2;
//	a3.fpCount = 2;
//	a3.fpConterAfterOneDil = 0;
//	a3.fnConterAfterOneDil = 0;
//
//	a3.fpConterAfterTwoDil = 2;
//	a3.fnConterAfterTwoDil = 2;
//
//	a3.isToBeValidatedFpAfterOneIter = true;
//	a3.isToBeValidatedFpAfterTwoIter = true;
//	a3.isToBeValidatedFnAfterOneIter = false;
//	a3.isToBeValidatedFnAfterTwoIter = false;
//
//	forTestMetaDataStruct full1 = getMetdataTestStruct(metasNumberRef,5, 7, 7, (dbXLength - 1) * dbYLength * dbZLength);
//	full1.isToBeFullAfterOneIter = true;
//	forTestMetaDataStruct full2 = getMetdataTestStruct(metasNumberRef,5, 5, 5, (dbXLength )* dbYLength * dbZLength);
//	full2.isToBeFullAfterOneIter = true;
//	forTestMetaDataStruct full3 = getMetdataTestStruct(metasNumberRef,5, 7, 10, (dbXLength - 4) * dbYLength * dbZLength);
//	full3.isToBeFullAfterTwoIter = true;
//	
//	
//
//	forTestMetaDataStruct allMetas[] = {
//	getMetdataTestStruct(metasNumberRef,2,2,2,  0, 3)
//		,a0
//
//		
//		//	,getMetdataTestStruct(metasNumberRef,metaXLength - 1,MetaYLength - 1,MetaZLength - 1 , 1,1)
//		,a1// should not be validated at all
//		,a2//now some points that should be covered by second dilatation after one dilatation no need to validate it
//		,a3, //now some points that should be covered by second dilatation after one dilatation no need to validate it
//
//		getMetdataTestStruct(metasNumberRef,0,0,2, 1)
//		,getMetdataTestStruct(metasNumberRef,0,0,1,0,0,false,true)//just marking it get activated	
//		,getMetdataTestStruct(metasNumberRef,0,0,4, 1)
//		,getMetdataTestStruct(metasNumberRef,0,0,5,0,0,false,true)//just marking it get activated	
//		,getMetdataTestStruct(metasNumberRef,8,0,6, 1)
//		,getMetdataTestStruct(metasNumberRef,7,0,6,0,0,false,true)//just marking it get activated	
//
//		,getMetdataTestStruct(metasNumberRef,0,0,8, 1)
//		,getMetdataTestStruct(metasNumberRef,1,0,8,0,0,false,true)//just marking it get activated	
//		,getMetdataTestStruct(metasNumberRef,0,0,10, 1)
//		,getMetdataTestStruct(metasNumberRef,0,1,10 ,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,2,2,4, 1)
//		,getMetdataTestStruct(metasNumberRef,2,1,4,0,0,false,true)//just marking it get activated			
//		
//		,getMetdataTestStruct(metasNumberRef,2,2,6, 1)
//		,getMetdataTestStruct(metasNumberRef,1,2,6,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,2,2,5,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,2,3,6,0,0,false,true)//just marking it get activated		
//
//				//right dow posterior  corner	
//		,getMetdataTestStruct(metasNumberRef,4,4,4, 1)
//		,getMetdataTestStruct(metasNumberRef,5,4,4,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,4,4,5,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,4,3,4,0,0,false,true)//just marking it get activated		
//
//		,getMetdataTestStruct(metasNumberRef,4,4,8, 1)
//		,getMetdataTestStruct(metasNumberRef,3,4,8,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,4,3,8,0,0,false,true)//just marking it get activated			
//		,getMetdataTestStruct(metasNumberRef,4,4,7,0,0,false,true)//just marking it get activated		
//		,getMetdataTestStruct(metasNumberRef,9,9,9,0,0)//some ampty block
//
//
//
//		
//	,full1, full2, full3
//	};
//	
//
//
//
//	 /// <summary>
//	 /// setting points 
//	 /// </summary>
//	 for (int i = 0; i < pointsNumber; i++) {
//		 forTestPointStruct currPoint = allPointsA[i];
//			 if (currPoint.isGold) {
//				 setArrCPU(arrGoldObj, currPoint.x, currPoint.y, currPoint.z, 2);
//			 }
//			 else {
//				 setArrCPU(arrSegmObj, currPoint.x, currPoint.y, currPoint.z, 2);
//
//		 };
//
//	 }
//
//	 //setArrCPU(arrGoldObj, 671, 263, 735, 2);
//	 //setArrCPU(arrSegmObj, 671, 263, 735, 2);
//
//
//
//	//mainKernelsRun(forFullBoolPrepArgs);
//	//printf("\n aaaaaaaaaaaaaaaaaaaaa\n ");
//
//	//i = 9;
//	//printf("workQueueCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//
//
//
//
//
//
//
//
//
//
//
//	 mainKernelsTestRun(forFullBoolPrepArgs, allPointsA, allMetas,pointsNumber, metasNumber);
//
//
//	 int ii = 7;
//	 	printf("global FP count %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//		ii = 8;
//	 	printf("global FN count %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//		ii = 9;
//	 	printf("workQueueCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//		ii = 10;
//	 	printf("resultFP globalCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//		ii = 11;
//	 	printf("resultFn globalCounter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//		ii = 12;
//		printf("global offset counter %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//
//		ii  = 13;
//	 	printf("globalIterationNumb %d  [%d]\n", minMaxes.arrP[0][0][ii], ii);
//
//	 //for (int i = 0; i < workQueueAndRLLength; i++) {
//
//		// if (workQueuePointer[0][2][i] > 0) {
//		//	 printf("work queue [%d][%d][%d] = [%d][%d][%d][%d]\n"
//		//		 , 0, 0, i
//		//		 , workQueuePointer[0][0][i]
//		//		 , workQueuePointer[0][1][i]
//		//		 , workQueuePointer[0][2][i]
//		//		 , workQueuePointer[0][3][i]
//		//	 );
//		// }
//
//	 //}
//
//	 for (int ji = 0; ji < 30; ji++) {
//		 if (forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji] + forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]  > 0) {
//			 printf("result  in point  %d %d %d isGold %d iteration %d \n ", forFullBoolPrepArgs.metaData.resultList.arrP[0][0][ji]
//				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][1][ji]
//				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][2][ji]
//				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][3][ji]
//				 , forFullBoolPrepArgs.metaData.resultList.arrP[0][4][ji]);
//		 }
//	 }
//
//
//
//	// 	i, j, k, value = 0;
// //for (i = 0; i < metaXLength; i++) {
// //	for (j = 0; j < MetaYLength; j++) {
// //		for (k = 0; k < MetaZLength; k++) {
// //			//goldArr[k][j][i] = 1;
// //			if (metaData.isToBeValidatedFp.arrP[k][j][i]) {
// //				printf("found as to be validated fp  [%d][%d][%d]\n", i, j, k);
// //			}
// //		}
// //	}
// //};
//
// //i, j, k, value = 0;
// //for (i = 0; i < metaXLength; i++) {
// //	for (j = 0; j < MetaYLength; j++) {
// //		for (k = 0; k < MetaZLength; k++) {
// //			//goldArr[k][j][i] = 1;
// //			if (metaData.isToBeValidatedFn.arrP[k][j][i]) {
// //				printf("found as  to be validated fn  [%d][%d][%d]\n", i, j, k);
// //			}
// //		}
// //	}
// //};
//
//
//
//
//
//
//
//	 //for (i = 0; i < mainXLength; i++) {
// 	//	for (j = 0; j < mainYLength; j++) {
// 	//		for (k = 0; k < MetaZLength; k++) {
// 	//			//goldArr[k][j][i] = 1;
// 	//			if (reducedGold[k][j][i] > 0) {
// 	//				for (int tt = 0; tt < 32; tt++) {
// 	//					if ((reducedGold[k][j][i] & (1 << (tt)))) {
// 	//						printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);
// 
// 	//					}
// 	//				}
// 
// 	//			}
// 	//		}
// 	//	}
// 	//}
//
//
//	 //for (i = 0; i < mainXLength; i++) {
//		// for (j = 0; j < mainYLength; j++) {
//		//	 for (k = 0; k < MetaZLength; k++) {
//		//		 //goldArr[k][j][i] = 1;
//		//		 if (reducedSegm[k][j][i] > 0) {
//		//			 for (int tt = 0; tt < 32; tt++) {
//		//				 if ((reducedSegm[k][j][i] & (1 << (tt)))) {
//		//					 printf("found in reduced fn  [%d][%d][%d]\n", i, j, k * 32 + tt);
//
//		//				 }
//		//			 }
//
//		//		 }
//		//	 }
//		// }
//	 //}
//
//
//	 
//	 /*	i = 1;
//	 	printf("maxX %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 2;
//	 	printf("minX %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 3;
//	 	printf("maxY %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 4;
//	 	printf("minY %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 5;
//	 	printf("maxZ %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 6;
//	 	printf("minZ %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 7;
//	 	printf("global FP count %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 8;
//	 	printf("global FN count %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 9;
//	 	printf("workQueueCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 10;
//	 	printf("resultFP globalCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 11;
//	 	printf("resultFn globalCounter %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 12;
//	 	printf("global FPandFn offset %d  [%d]\n", minMaxes.arrP[0][0][i], i);
//	 	i = 13;
//	 	printf("globalIterationNumb %d  [%d]\n", minMaxes.arrP[0][0][i], i);*/
//
//
//	
//	//	setArrCPU(arrSegmObj, dbXLength * 5 + 2, dbYLength * 5+2, dbZLength * 5 + 2, 2);
//
//	//setArrCPU(arrGoldObj, dbXLength  + 2, dbYLength  + 3, dbZLength  + 4, 2);
//	
//	
//	/////define metadata
//	
//	
//	
//	
//	
//	
//	
//	
//	
//	printf("cleaaning");
//
//	free(isToBeValidatedFpPointer);
//	free(isToBeValidatedFnPointer);
//	free(metaData.minMaxes.arrP);
//	free(metaData.fpCount.arrP);
//	free(metaData.fnCount.arrP);
//	free(metaData.fpCounter.arrP);
//	free(metaData.fnCounter.arrP);
//	free(metaData.fpOffset.arrP);
//	free(metaData.fnOffset.arrP);
//
//	free(metaData.isActiveGold.arrP);
//	free(metaData.isFullGold.arrP);
//
//	free(metaData.isActiveSegm.arrP);
//	free(metaData.isFullSegm.arrP);
//
//	free(workQueuePointer);
//	free(resultListPointer);
//
//	free(isToBeActivatedGoldPointer);
//	free(isToBeActivatedSegmPointer);
//
//
//	free(forDebugArr);
//	free(goldArr);
//	free(segmArr);
//	free(reducedSegm);
//	free(reducedGold);
//	free(reducedGoldPrevPointer);
//	free(reducedSegmPrevPointer);
//	free(reducedGoldRef);
//	free(reducedSegmRef);
//
//
//
//}
//
//
//
//
//
//
//
//
//
//
//
//
//
