
#include <hip/hip_runtime.h>
//#include "ForBoolKernel.cu"
//#include "Structs.cu"
//
//
//
////testing loopMeta function in order to execute test unhash proper function in loopMeta
//#pragma once
//extern "C" inline void testDataTransfer() {
//	const int blocks = 1;
//	const int xThreadDim = 32;
//	const int yThreadDim = 11;
//	const dim3 threads = dim3(xThreadDim, yThreadDim);
//
//	//metadata
//	const int metaXLength =10;
//	const int MetaYLength = 10;
//	const int MetaZLength = 10;
//
//
//	const int totalLength = metaXLength * MetaYLength * MetaZLength;
//	const int loopMetaTimes = floor(totalLength / blocks);
//
//	/*   int*** h_tensor;
//	   h_tensor = alloc_tensorToZeros<int>(metaXLength, MetaYLength, MetaZLength);*/
//
//
//	int*** forDebugArr;
//
//	const int dXLength = 4;
//	const int dYLength = 4;
//	const int dZLength = 4;
//
//	//datablock dimensions
//	const int dbXLength = xThreadDim;
//	const int dbYLength = yThreadDim;
//	const int dbZLength = 32;
//
//	const int mainXLength = dbXLength * metaXLength;
//	const int mainYLength = dbYLength * MetaYLength;
//	const int mainZLength = dbZLength * MetaZLength;
//
//
//	//main data arrays
//	int*** goldArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//
//	int*** segmArr;
//	segmArr = alloc_tensorToZeros<int>(mainXLength, mainYLength, mainZLength);
//	MetaDataCPU metaData;
//	metaData.metaXLength = metaXLength;
//	metaData.MetaYLength = MetaYLength;
//	metaData.MetaZLength = MetaZLength;
//	metaData.totalMetaLength = totalLength;
//	auto fpCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto fnCPointer = alloc_tensorToZeros<unsigned int>(metaXLength, MetaYLength, MetaZLength);
//	auto minMaxesPointer = alloc_tensorToZeros<int>(7, 1, 1);
//	auto isActivePointer = alloc_tensorToZeros<bool>(metaXLength, MetaYLength, MetaZLength);
//
//	auto fpC = get3dArrCPU(fpCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto fnC = get3dArrCPU(fnCPointer, metaXLength, MetaYLength, MetaZLength);
//	auto minMaxes = get3dArrCPU(minMaxesPointer, 9, 1, 1);
//	auto isActive = get3dArrCPU(isActivePointer, metaXLength, MetaYLength, MetaZLength);
//
//	metaData.fpCount = fpC;
//	metaData.fnCount = fnC;
//	metaData.minMaxes = minMaxes;
//	forDebugArr = alloc_tensorToZeros<int>(dXLength, dYLength, dZLength);
//
//
//	uint32_t*** reducedGold = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//	uint32_t*** reducedSegm = alloc_tensorToZeros<uint32_t>(mainXLength, mainYLength, mainZLength);
//
//
//	// arguments to pass
//	ForFullBoolPrepArgs<int> forFullBoolPrepArgs;
//	forFullBoolPrepArgs.metaData = metaData;
//	forFullBoolPrepArgs.numberToLookFor = 2;
//	forFullBoolPrepArgs.forDebugArr = get3dArrCPU(forDebugArr, dXLength, dYLength, dZLength);
//	forFullBoolPrepArgs.dbXLength = dbXLength;
//	forFullBoolPrepArgs.dbYLength = dbYLength;
//	forFullBoolPrepArgs.dbZLength = dbZLength;
//	forFullBoolPrepArgs.goldArr = get3dArrCPU(goldArr, mainXLength, mainYLength, mainZLength);
//	forFullBoolPrepArgs.segmArr = get3dArrCPU(segmArr, mainXLength, mainYLength, mainZLength);
//
//	forFullBoolPrepArgs.reducedGold = get3dArrCPU(reducedGold, mainXLength, mainYLength, MetaZLength);
//	forFullBoolPrepArgs.reducedSegm = get3dArrCPU(reducedSegm, mainXLength, mainYLength, MetaZLength);
//
//	forFullBoolPrepArgs.reducedArrsZdim = mainZLength;
//
//	forFullBoolPrepArgs.threads = threads;
//	forFullBoolPrepArgs.blocks = blocks;
//
//	//populate segm  and gold Arr
//
//
//	auto arrGoldObj = forFullBoolPrepArgs.goldArr;
//	auto arrSegmObj = forFullBoolPrepArgs.segmArr;
//	//setArrCPU(arrGoldObj,2, 3,4,2);
//	setArrCPU(arrGoldObj, dbXLength + 2, dbYLength + 1, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength + 2, dbYLength + 2, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength + 2+1, dbYLength + 2, dbZLength * 3 + 1, 2);
//	setArrCPU(arrGoldObj, dbXLength * 2 + 2, dbYLength * 2 + 3, dbZLength * 2 + 4, 2);
//	setArrCPU(arrGoldObj, dbXLength * 2 + 3, dbYLength * 2 + 3, dbZLength * 2 + 4, 2);
//	setArrCPU(arrGoldObj, dbXLength * 3 + 2, dbYLength + 2, dbZLength * 2 + 5, 2);
//	setArrCPU(arrGoldObj, dbXLength * 4 + 9, dbYLength * 2, dbZLength * 2 + 1, 2);
//	
//	
//	setArrCPU(arrSegmObj, dbXLength * 6 + 1, dbYLength * 2, dbZLength * 2 + 1, 2);
//	setArrCPU(arrSegmObj, dbXLength * 6 + 5, dbYLength * 2, dbZLength * 2 + 1, 2);
//	setArrCPU(arrSegmObj, dbXLength * 7 + 4, dbYLength * 2, dbZLength * 2 + 1, 2);
//
//
//	//printf("mainXLength %d mainYLength %d mainZLength %d \n", mainXLength, mainYLength, mainZLength);
//	boolPrepare(forFullBoolPrepArgs);
//
//
//	//int i, j, k, value = 0;
//	//for (i = 0; i < mainXLength; i++) {
//	//	for (j = 0; j < mainYLength; j++) {
//	//		for (k = 0; k < mainZLength; k++) {
//	//			//goldArr[k][j][i] = 1;
//	//			if (goldArr[k][j][i] > 0){
//	//				printf("segmArr[%d][%d][%d] = %d\n", i, j, k, goldArr[k][j][i]);
//	//		}
//	//		}
//	//	}
//	//}
//
//
//
//	int i, j, k, value = 0;
//	for (i = 0; i < mainXLength; i++) {
//		for (j = 0; j < mainYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (reducedGold[k][j][i] > 0) {
//					for (int tt = 0; tt < 32; tt++) {
//						if ((reducedGold[k][j][i] & (1 << (tt)))) {
//							printf("found in reduced fp  [%d][%d][%d]\n", i, j, k * 32 + tt);
//
//						}
//					}
//
//
//				}
//			}
//		}
//	}
//
//	for (i = 0; i < mainXLength; i++) {
//		for (j = 0; j < mainYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] > 0) {
//					for (int tt = 0; tt < 32; tt++) {
//						if ((forFullBoolPrepArgs.reducedSegm.arrP[k][j][i] & (1 << (tt)))) {
//							printf("found in reduced fn [%d][%d][%d]\n", i, j, k * 32 + tt);
//
//						}
//					}
//
//
//				}
//			}
//		}
//	}
//
//
//
//	i, j, k, value = 0;
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (isActive.arrP[k][j][i]) {
//
//					printf("found as Active [%d][%d][%d]\n", i, j, k);
//
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//
//	i, j, k, value = 0;
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (fpC.arrP[k][j][i]>0) {
//					printf("found Fp %d  [%d][%d][%d]\n", fpC.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//	for (i = 0; i < metaXLength; i++) {
//		for (j = 0; j < MetaYLength; j++) {
//			for (k = 0; k < MetaZLength; k++) {
//				//goldArr[k][j][i] = 1;
//				if (fnC.arrP[k][j][i]>0) {
//					printf("found Fn %d  [%d][%d][%d]\n", fnC.arrP[k][j][i], i, j, k);
//
//				}
//
//
//			}
//		}
//
//	};
//
//
//
//
//	i, j, k, value = 0;
//	for (i = 1; i < 9; i++) {
//		for (j = 0; j < 1; j++) {
//			for (k = 0; k < 1; k++) {
//				//goldArr[k][j][i] = 1;
//
//
//				printf("in minMaxes %d  [%d][%d][%d]\n", minMaxes.arrP[k][j][i], i, j, k);
//
//
//
//
//
//			}
//		}
//
//	};
//
//
//
//
//
//
//	//int i, j, k, value = 0;
//	//for (i = 0; i < mainXLength; i++) {
//	//	for (j = 0; j < mainYLength; j++) {
//	//		for (k = 0; k < mainZLength; k++) {
//	//			//goldArr[k][j][i] = 1;
//	//			if (goldArr[k][j][i] > 0) {
//	//				printf("segmArr[%d][%d][%d] = %d\n", i, j, k, goldArr[k][j][i]);
//	//			}
//	//		}
//	//	}
//	//}
//
//
//
//
//
//	free(metaData.fpCount.arrP);
//	free(metaData.fnCount.arrP);
//	free(metaData.isActive.arrP);
//	free(forDebugArr);
//	free(goldArr);
//	free(segmArr);
//	free(reducedSegm);
//	free(reducedGold);
//
//
//
//	//std::cout << longInts[3] << std::endl;
//
//}
//
//
//
//
//
//
//
//
//
//
//
//
//
