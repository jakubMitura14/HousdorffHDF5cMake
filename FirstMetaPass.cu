#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;


/*
    a) we define offsets in the result list to have the results organized and avoid overwiting
    b) if metadata block is active we add it in the work queue
*/


/*
we add here to appropriate queue data  about metadata of blocks of intrest
minMaxesPos- marks in minmaxes the postion of global offset counter -12) global FP offset 13) global FnOffset
offsetMetadataArr- arrays from metadata holding data about result list offsets it can be either fbArgs.metaData.fpOffset or fbArgs.metaData.fnOffset
*/


#pragma once
__device__ inline void addToQueue( uint32_t linIdexMeta, uint8_t isGold
    , unsigned int fpFnLocCounter[1], uint32_t localWorkQueue[1600], uint32_t localOffsetQueue[1600], unsigned int localWorkQueueCounter[1]
    , uint8_t countIndexNumb, uint8_t isActiveIndexNumb, uint8_t offsetIndexNumb
    , uint32_t* metaDataArr, MetaDataGPU metaData, unsigned int* minMaxes,uint32_t* workQueue) {

    unsigned int count = metaDataArr[linIdexMeta * metaData.metaDataSectionLength + countIndexNumb];
        //given fp is non zero we need to  add this to local queue
        if (metaDataArr[linIdexMeta * metaData.metaDataSectionLength + isActiveIndexNumb]==1) {

           // printf("adding to local in first meta pass linIdexMeta %d isGold %d isActiveIndexNumb %d \n  ", linIdexMeta, isGold, isActiveIndexNumb);

            count = atomicAdd_block(&fpFnLocCounter[0], count);
            unsigned int  old = atomicAdd_block(&localWorkQueueCounter[0], 1);
            //we check weather we still have space in shared memory
            if (old < 1590) {// so we still have space in shared memory
                // will be equal or above isGoldOffset  if it is gold pass
                localWorkQueue[old] = linIdexMeta+(isGoldOffset * isGold);
                localOffsetQueue[old] = uint32_t(count);
                     }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(minMaxes[9]), old);
                //workQueue
                workQueue[old] = linIdexMeta + (isGoldOffset * isGold);
                //and offset 
                metaDataArr[linIdexMeta * metaData.metaDataSectionLength + offsetIndexNumb] = atomicAdd(&(minMaxes[12]), count);
            };
     }
}


#pragma once
template <typename PYO>
__global__ void firstMetaPrepareKernel(ForBoolKernelArgs<PYO> fbArgs
    , MetaDataGPU metaData, unsigned int* minMaxes, uint32_t* workQueue
    , uint32_t* origArrs, uint32_t* metaDataArr) {

    //////initializations
    thread_block cta = this_thread_block();
     char* tensorslice;// needed for iterations over 3d arrays
    //local offset counters  for fp and fn's
    __shared__ unsigned int fpFnLocCounter[1];
    // used to store the start position in global memory for whole block
    __shared__ unsigned int globalOffsetForBlock[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    //used as local work queue counter
    __shared__ unsigned int localWorkQueueCounter[1];     
    //according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes so it will give us 1600 length of shared memory
    //so here we will store locally the calculated offsets and coordinates of meta data block of intrest marking also wheather we are  talking about gold or segmentation pass (fp or fn )
    __shared__ uint32_t localWorkQueue[1600];
    __shared__ uint32_t localOffsetQueue[1600];
    if ((threadIdx.x == 0)) {
        fpFnLocCounter[0] = 0;
    }
    if ((threadIdx.x == 1)) {
        localWorkQueueCounter[0] = 0;
    }
    if ((threadIdx.x == 2)) {
        globalWorkQueueCounter[0] = 0;
    }
    if ((threadIdx.x == 3)) {
        globalOffsetForBlock[0] = 0;
    }
    sync(cta);


    // classical grid stride loop - in case of unlikely event we will run out of space we will empty it prematurly
    //main metadata iteration
    for (uint32_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += blockDim.x * gridDim.x) {
         
       // if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
          //  printf("in first meta pass linIdexMeta %d blockIdx.x %d blockDim.x %d metaData.totalMetaLength %d threadIdx.x %d \n  ", linIdexMeta, blockIdx.x, blockDim.x, metaData.totalMetaLength, threadIdx.x );
        //}
        
        //goldpass
        addToQueue( linIdexMeta, 0
            , fpFnLocCounter, localWorkQueue, localOffsetQueue, localWorkQueueCounter
            , 1, 9, 6
            , metaDataArr, metaData, minMaxes, workQueue);
          //segmPass  
        addToQueue( linIdexMeta, 1
            , fpFnLocCounter, localWorkQueue, localOffsetQueue, localWorkQueueCounter
            , 2, 7, 5
            , metaDataArr, metaData, minMaxes, workQueue);
    
        
        
 /*       addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fpOffset, fbArgs.metaData.fpCount, 0, fbArgs.metaData.isActiveSegm, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);
        addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fnOffset, fbArgs.metaData.fnCount, 1, fbArgs.metaData.isActiveGold, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);*/
        }
    sync(cta);
    if ((threadIdx.x == 0) ) {
        globalOffsetForBlock[0] = atomicAdd(&(minMaxes[12]), (fpFnLocCounter[0]));

       /* if (fpFnLocCounter[0]>0) {
            printf("\n in meta first pass global offset %d  locCounter %d \n  ", globalOffsetForBlock[0], fpFnLocCounter[0]);
        }*/
    };
    if ((threadIdx.x == 1) ) {
        if (localWorkQueueCounter[0]>0) {
            globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
         }
    }
    sync(cta);

    //exporting to global work queue
    //cooperative_groups::memcpy_async(cta, (&workQueue[globalWorkQueueCounter[0]]), (localWorkQueue), (sizeof(uint32_t) * localWorkQueueCounter[0]));

    
    //setting offsets
    for (uint32_t i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {
        workQueue[globalWorkQueueCounter[0] +i]=localWorkQueue[i]; 

   /*     printf("FFIrst meta pass lin meta to Work Q %d is gold %d to spot %d  \n "
    , localWorkQueue[i] - isGoldOffset*(localWorkQueue[i] >= isGoldOffset)
        , (localWorkQueue[i] >= isGoldOffset), globalWorkQueueCounter[0] + i);*/

        //FP pass
        if (localWorkQueue[i]>= isGoldOffset) {
            metaDataArr[(localWorkQueue[i] - isGoldOffset) * metaData.metaDataSectionLength + 5] = localOffsetQueue[i] + globalOffsetForBlock[0];
            //printf("fp offset lin meta %d total offset  %d  global part %d local part %d \n "
            //    , localWorkQueue[i] - isGoldOffset
            //    , localOffsetQueue[i] + globalOffsetForBlock[0] 
            //, globalOffsetForBlock[0]
            //, localOffsetQueue[i]);

        }
        //FN pass
        else {
            metaDataArr[(localWorkQueue[i]) * metaData.metaDataSectionLength + 6] = localOffsetQueue[i] + globalOffsetForBlock[0];
            //printf("fn offset lin meta %d total offset  %d  global part %d local part %d \n "
            //    , localWorkQueue[i] 
            //    , localOffsetQueue[i] + globalOffsetForBlock[0]
            //    , globalOffsetForBlock[0]
            //    , localOffsetQueue[i]);
        
        };

        //sync(cta);

        
    }

           

    };







