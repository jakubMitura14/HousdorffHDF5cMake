#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;


/*
    a) we define offsets in the result list to have the results organized and avoid overwiting
    b) if metadata block is active we add it in the work queue
*/


/*
we add here to appropriate queue data  about metadata of blocks of intrest
minMaxesPos- marks in minmaxes the postion of global offset counter -12) global FP offset 13) global FnOffset
offsetMetadataArr- arrays from metadata holding data about result list offsets it can be either fbArgs.metaData.fpOffset or fbArgs.metaData.fnOffset
*/


#pragma once
__device__ inline void addToQueue( uint16_t linIdexMeta, uint8_t isGold
    , unsigned int fpFnLocCounter[1], uint32_t localWorkQueue[1600], uint32_t localOffsetQueue[1600], unsigned int localWorkQueueCounter[1]
    , uint8_t countIndexNumb, uint8_t isActiveIndexNumb, uint8_t offsetIndexNumb
    , uint32_t* mainArr, MetaDataGPU metaData, unsigned int* minMaxes,uint32_t* workQueue) {

    unsigned int count = mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + countIndexNumb];
        //given fp is non zero we need to  add this to local queue
        if (mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + isActiveIndexNumb]==1) {

           // printf("in first meta pass linIdexMeta %d isGold %d \n  ", linIdexMeta, isGold);

            count = atomicAdd_block(&fpFnLocCounter[0], count);
            unsigned int  old = atomicAdd_block(&localWorkQueueCounter[0], 1);
            //we check weather we still have space in shared memory
            if (old < 1590) {// so we still have space in shared memory
                // will be equal or above UINT16_MAX if it is gold pass
                localWorkQueue[old] = uint32_t(linIdexMeta+(UINT16_MAX* isGold));
                localOffsetQueue[old] = uint32_t(count);
                     }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(minMaxes[9]), old);
                //workQueue
                workQueue[old] = uint32_t(linIdexMeta + (UINT16_MAX * isGold));
                //and offset 
                mainArr[linIdexMeta * metaData.mainArrSectionLength + metaData.metaDataOffset + offsetIndexNumb] = atomicAdd(&(minMaxes[12]), count);
            };
     }
}


#pragma once
template <typename PYO>
__global__ void firstMetaPrepareKernel(ForBoolKernelArgs<PYO> fbArgs
    , uint32_t* mainArr, MetaDataGPU metaData, unsigned int* minMaxes, uint32_t* workQueue) {

    //////initializations
    thread_block cta = this_thread_block();
     char* tensorslice;// needed for iterations over 3d arrays
    //local offset counters  for fp and fn's
    __shared__ unsigned int fpFnLocCounter[1];
    // used to store the start position in global memory for whole block
    __shared__ unsigned int globalOffsetForBlock[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    //used as local work queue counter
    __shared__ unsigned int localWorkQueueCounter[1];     
    //according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes so it will give us 1600 length of shared memory
    //so here we will store locally the calculated offsets and coordinates of meta data block of intrest marking also wheather we are  talking about gold or segmentation pass (fp or fn )
    __shared__ uint32_t localWorkQueue[1600];
    __shared__ uint32_t localOffsetQueue[1600];
    if ((threadIdx.x == 0)) {
        fpFnLocCounter[0] = 0;
    }
    if ((threadIdx.x == 1)) {
        localWorkQueueCounter[0] = 0;
    }
    if ((threadIdx.x == 2)) {
        globalWorkQueueCounter[0] = 0;
    }
    if ((threadIdx.x == 3)) {
        globalOffsetForBlock[0] = 0;
    }
    sync(cta);


    // classical grid stride loop - in case of unlikely event we will run out of space we will empty it prematurly
    //main metadata iteration
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < metaData.totalMetaLength; linIdexMeta += blockDim.x * gridDim.x) {
         
       // if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
         //   printf("in first meta pass linIdexMeta %d \n  ", linIdexMeta);
        //}
        
        //goldpass
        addToQueue( linIdexMeta, 0
            , fpFnLocCounter, localWorkQueue, localOffsetQueue, localWorkQueueCounter
            , 1, 9, 6
            , mainArr, metaData, minMaxes, workQueue);
          //segmPass  
        addToQueue( linIdexMeta, 1
            , fpFnLocCounter, localWorkQueue, localOffsetQueue, localWorkQueueCounter
            , 2, 7, 5
            , mainArr, metaData, minMaxes, workQueue);
    
        
        
 /*       addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fpOffset, fbArgs.metaData.fpCount, 0, fbArgs.metaData.isActiveSegm, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);
        addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fnOffset, fbArgs.metaData.fnCount, 1, fbArgs.metaData.isActiveGold, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);*/
        }
    sync(cta);
    if ((threadIdx.x == 0) ) {
        globalOffsetForBlock[0] = atomicAdd(&(minMaxes[12]), (fpFnLocCounter[0]))- fpFnLocCounter[0];

       /* if (fpFnLocCounter[0]>0) {
            printf("\n in meta first pass global offset %d  locCounter %d \n  ", globalOffsetForBlock[0], fpFnLocCounter[0]);
        }*/
    };
    if ((threadIdx.x == 1) ) {
        if (localWorkQueueCounter[0]>0) {
            globalWorkQueueCounter[0] = atomicAdd(&(minMaxes[9]), (localWorkQueueCounter[0]));
         }
    }
    sync(cta);

    //exporting to global work queue
    cooperative_groups::memcpy_async(cta, (&workQueue[globalWorkQueueCounter[0]]), (localWorkQueue), (sizeof(uint32_t) * localWorkQueueCounter[0]));
    //setting offsets
    for (uint16_t i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {
       // 
       //// printf("addTo %d global Queue xMeta [%d] yMeta [%d] zMeta [%d] isGold %d \n", globalWorkQueueCounter[0] + i, localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);
       // //TODO() instead of copying memory manually better would be to use mempcyasync ...
       //// printf("\n saving to local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d   ", localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);

       // getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[globalWorkQueueCounter[0]+i] = localWorkAndOffsetQueue[i][0];
       // getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][1];
       // getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][2];
       // getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][3];
       // //and offset 
        
        //FP pass
        if (localWorkQueue[i]>= UINT16_MAX) {
            mainArr[(localWorkQueue[i]- UINT16_MAX) * metaData.mainArrSectionLength + metaData.metaDataOffset + 5] = localOffsetQueue[i] + globalOffsetForBlock[0];
        }
        //FN pass
        else {
            mainArr[(localWorkQueue[i]) * metaData.mainArrSectionLength + metaData.metaDataOffset + 6] = localOffsetQueue[i] + globalOffsetForBlock[0];

        };

        sync(cta);

    }

           

    };







