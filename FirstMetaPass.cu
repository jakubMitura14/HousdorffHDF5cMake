#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "ForBoolKernel.cu"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;


/*
    a) we define offsets in the result list to have the results organized and avoid overwiting
    b) if metadata block is active we add it in the work queue
*/


/*
we add here to appropriate queue data  about metadata of blocks of intrest
minMaxesPos- marks in minmaxes the postion of global offset counter -12) global FP offset 13) global FnOffset
offsetMetadataArr- arrays from metadata holding data about result list offsets it can be either fbArgs.metaData.fpOffset or fbArgs.metaData.fnOffset
*/


#pragma once
template <typename PYO>
__device__ void addToQueue(ForBoolKernelArgs<PYO> fbArgs, unsigned int& old, unsigned int& count, char* tensorslice
    , uint16_t xMeta, uint16_t yMeta, uint16_t zMeta, array3dWithDimsGPU& offsetMetadataArr, array3dWithDimsGPU& countMetadataArr
    , uint16_t isGold, array3dWithDimsGPU& isActiveArr, unsigned int fpFnLocCounter[1], uint16_t localWorkAndOffsetQueue[1600][5], unsigned int localWorkQueueCounter[1]
) {

    count = getTensorRow<unsigned int>(tensorslice, countMetadataArr, countMetadataArr.Ny, yMeta, zMeta)[xMeta];
        //given fp is non zero we need to  add this to local queue
        if (getTensorRow<bool>(tensorslice, isActiveArr, isActiveArr.Ny, yMeta, zMeta)[xMeta]) {
            //we need to establish where to put the entry in the local queue
            //if (count>0) {
            //    printf("\n in add queue count %d xMeta %d yMeta %d zMeta %d \n", count, xMeta, yMeta, zMeta);
            //}
            count = atomicAdd(&fpFnLocCounter[0], count);
            //printf("\n in add queue fpFnLocCounter %d xMeta %d yMeta %d zMeta %d \n", fpFnLocCounter[0], xMeta, yMeta, zMeta);

            old = atomicAdd(&localWorkQueueCounter[0], 1);
            //we check weather we still have space in shared memory
            if (old < 1590) {// so we still have space in shared memory
                localWorkAndOffsetQueue[old][0] = xMeta;
                localWorkAndOffsetQueue[old][1] = yMeta;
                localWorkAndOffsetQueue[old][2] = zMeta;
                localWorkAndOffsetQueue[old][3] = isGold;// marking it is about gold pass - FP
                localWorkAndOffsetQueue[old][4] = count;// marking local offset - this will need to be incremented later by global and local value
            }
            else {// so we do not have any space more in the sared memory  - it is unlikely so we will just in this case save immidiately to global memory
                old = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), old);
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[old] = xMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[old] = yMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[old] = zMeta;
                getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[old] = isGold;
                //and offset 
                getTensorRow<unsigned int>(tensorslice, offsetMetadataArr, offsetMetadataArr.Ny, yMeta, zMeta)[xMeta]
                    = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[12]), count);
            };


        }
}







#pragma once
template <typename PYO>
__global__ void firstMetaPrepareKernel(ForBoolKernelArgs<PYO> fbArgs) {

    //////initializations
    thread_block cta = this_thread_block();
     char* tensorslice;// needed for iterations over 3d arrays
     unsigned int old = 0;// local variable
     unsigned int count = 0;// local variable
     uint16_t xMeta=0;
     uint16_t yMeta=0;
     uint16_t zMeta=0;
    //local offset counters  for fp and fn's
    __shared__ unsigned int fpFnLocCounter[1];
    // used to store the start position in global memory for whole block
    __shared__ unsigned int globalOffsetForBlock[1];
    __shared__ unsigned int globalWorkQueueCounter[1];
    //used as local work queue counter
    __shared__ unsigned int localWorkQueueCounter[1];     
    //according to https://forums.developer.nvidia.com/t/find-the-limit-of-shared-memory-that-can-be-used-per-block/48556 it is good to keep shared memory below 16kb kilo bytes so it will give us 1600 length of shared memory
    //so here we will store locally the calculated offsets and coordinates of meta data block of intrest marking also wheather we are  talking about gold or segmentation pass (fp or fn )
    __shared__ uint16_t localWorkAndOffsetQueue[1600][5];
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        fpFnLocCounter[0] = 0;
    }
    sync(cta);


    // classical grid stride loop - in case of unlikely event we will run out of space we will empty it prematurly
    //main metadata iteration
    for (uint16_t linIdexMeta = blockIdx.x * blockDim.x + threadIdx.x; linIdexMeta < fbArgs.metaData.totalMetaLength; linIdexMeta += blockDim.x * gridDim.x) {
        //we get from linear index  the coordinates of the metadata block of intrest
        xMeta = linIdexMeta % fbArgs.metaData.metaXLength;
        zMeta = floor((float)(linIdexMeta / (fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength)));
        yMeta = floor((float)((linIdexMeta - ((zMeta * fbArgs.metaData.metaXLength * fbArgs.metaData.MetaYLength) + xMeta)) / fbArgs.metaData.metaXLength));
        //we define offsets in the result list to have the results organizedand avoid overwiting

        //TODO remove only debugging    
        //getTensorRow<unsigned int>(tensorslice, fbArgs.forDebugArr, fbArgs.forDebugArr.Ny, yMeta, zMeta)[xMeta] += 1;

        ////gold pass
        //addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fpOffset, fbArgs.metaData.fpCount, 1,fbArgs.metaData.isActiveGold,  fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);
        ////segmPass
        //addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fnOffset, fbArgs.metaData.fnCount, 0,fbArgs.metaData.isActiveSegm,  fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);
        
        addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fpOffset, fbArgs.metaData.fpCount, 0, fbArgs.metaData.isActiveSegm, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);
        addToQueue(fbArgs, old, count, tensorslice, xMeta, yMeta, zMeta, fbArgs.metaData.fnOffset, fbArgs.metaData.fnCount, 1, fbArgs.metaData.isActiveGold, fpFnLocCounter, localWorkAndOffsetQueue, localWorkQueueCounter);


        }
    sync(cta);
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        globalOffsetForBlock[0] = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[12]), (fpFnLocCounter[0]));
       /* if (fpFnLocCounter[0]>0) {
            printf("\n in meta first pass global offset %d  locCounter %d \n  ", globalOffsetForBlock[0], fpFnLocCounter[0]);
        }*/
    };
    if ((threadIdx.x == 1) && (threadIdx.y == 0)) {
        if (localWorkQueueCounter[0]>0) {
            globalWorkQueueCounter[0] = atomicAdd(&(getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.minMaxes, 1, 0, 0)[9]), (localWorkQueueCounter[0]));

         }
    }
    sync(cta);
    //grid stride loop for pushing value from local memory to global 


    for (uint16_t i = threadIdx.x; i < localWorkQueueCounter[0]; i += blockDim.x) {
        
       // printf("addTo %d global Queue xMeta [%d] yMeta [%d] zMeta [%d] isGold %d \n", globalWorkQueueCounter[0] + i, localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);
        //TODO() instead of copying memory manually better would be to use mempcyasync ...
       // printf("\n saving to local work queue xMeta %d  yMeta %d  zMeta %d  isGold %d   ", localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2], localWorkAndOffsetQueue[i][3]);

        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 0, 0)[globalWorkQueueCounter[0]+i] = localWorkAndOffsetQueue[i][0];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 1, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][1];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 2, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][2];
        getTensorRow<uint16_t>(tensorslice, fbArgs.metaData.workQueue, fbArgs.metaData.workQueue.Ny, 3, 0)[globalWorkQueueCounter[0] + i] = localWorkAndOffsetQueue[i][3];
        //and offset 
        
        //FP pass
        if (localWorkAndOffsetQueue[i][3] == 1) {

          /*  printf("\n in meta first pass saving  offset %d  locCounter  %d xMeta %d yMeta %d zMeta %d \n  ", globalOffsetForBlock[0], fpFnLocCounter[0]
                , localWorkAndOffsetQueue[i][0], localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][3]);*/

            getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fpOffset, fbArgs.metaData.fpOffset.Ny, localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2])[localWorkAndOffsetQueue[i][0]]
                = localWorkAndOffsetQueue[i][4] + globalOffsetForBlock[0];

        }
        //FN pass
        else {
            getTensorRow<unsigned int>(tensorslice, fbArgs.metaData.fnOffset, fbArgs.metaData.fnOffset.Ny, localWorkAndOffsetQueue[i][1], localWorkAndOffsetQueue[i][2])[localWorkAndOffsetQueue[i][0]]
                = localWorkAndOffsetQueue[i][4] + globalOffsetForBlock[0];
        };




    }

           

    };



    //for (uint8_t xMeta = threadIdx.x; xMeta < krowa; xMeta += blockDim.x) {
    //    for (uint8_t yMeta = threadIdx.y; yMeta < krowa; yMeta += blockDim.y) {
    //            for (uint8_t zMeta = 0; zMeta < krowa; zMeta++) {






    //            }
    //         
    //            sync(cta); // just to reduce the warp divergence

    //                       
    //    }

    //}









//
//
//
//#pragma once
//extern "C" inline bool firstMetaAndBoolRun (ForFullBoolPrepArgs<int> fFArgs) {
//
//
//    hipError_t syncErr;
//    hipError_t asyncErr;
//
//
//
//
//
//    //for debugging
//    array3dWithDimsGPU forDebug = allocate3dInGPU(fFArgs.forDebugArr);
//    //main arrays allocations
//    array3dWithDimsGPU goldArr = allocate3dInGPU(fFArgs.goldArr);
//
//    array3dWithDimsGPU segmArr = allocate3dInGPU(fFArgs.segmArr);
//    ////reduced arrays
//    array3dWithDimsGPU reducedGold = allocate3dInGPU(fFArgs.reducedGold);
//    array3dWithDimsGPU reducedSegm = allocate3dInGPU(fFArgs.reducedSegm);
//
//
//
//    array3dWithDimsGPU paddingsStore = allocate3dInGPU(fFArgs.paddingsStore);
//
//
//
//
//
//
//    ForBoolKernelArgs<int> fbArgs = getArgsForKernel<int>(fFArgs, forDebug, goldArr, segmArr, reducedGold, reducedSegm, paddingsStore);
//
//    //preparation kernel
//    boolPrepareKernel << <fFArgs.blocks, fFArgs.threads >> > (fbArgs);
//    //sync
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//    
//    //here threads one dimensionsonal !!
//    //TODO() reallocate memory - make reduced arrs and metadata smaller - allocate work queue, padding store, result list ...
//
//
//    firstMetaPrepareKernel << <fFArgs.blocksFirstMetaDataPass, fFArgs.threadsFirstMetaDataPass >> > (fbArgs);
//    //sync
//    checkCuda(hipDeviceSynchronize(), "just after boolPrepareKernel");
//
//
//
//
//    //deviceTohost
//
//    copyDeviceToHost3d(forDebug, fFArgs.forDebugArr);
//
//
//    copyDeviceToHost3d(goldArr, fFArgs.goldArr);
//    copyDeviceToHost3d(segmArr, fFArgs.segmArr);
//
//    copyDeviceToHost3d(reducedGold, fFArgs.reducedGold);
//    copyDeviceToHost3d(reducedSegm, fFArgs.reducedSegm);
//
//
//    copyMetaDataToCPU(fFArgs.metaData, fbArgs.metaData);
//
//
//
//    checkCuda(hipDeviceSynchronize(), "just after copy device to host");
//    //hipGetLastError();
//
//    hipFree(forDebug.arrPStr.ptr);
//    hipFree(goldArr.arrPStr.ptr);
//    hipFree(segmArr.arrPStr.ptr);
//    hipFree(reducedGold.arrPStr.ptr);
//    hipFree(reducedSegm.arrPStr.ptr);
//
//
//    freeMetaDataGPU(fbArgs.metaData);
//
//
//    /*
// * Catch errors for both the kernel launch above and any
// * errors that occur during the asynchronous `doubleElements`
// * kernel execution.
// */
//
//    syncErr = hipGetLastError();
//    asyncErr = hipDeviceSynchronize();
//
//    /*
//     * Print errors should they exist.
//     */
//
//    if (syncErr != hipSuccess) printf("Error in syncErr: %s\n", hipGetErrorString(syncErr));
//    if (asyncErr != hipSuccess) printf("Error in asyncErr: %s\n", hipGetErrorString(asyncErr));
//
//
//
//    return true;
//}
