#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/////////// loading functions

template <typename TXPI>
inline __device__  void loadRightLeft(ForBoolKernelArgs<TXPI> fbArgs, thread_block& cta, uint32_t localBlockMetaData[]
    , uint32_t mainShmem[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t* metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool isGoldForLocQueue[localWorkQueLength], int iterationNumb[1]) {

    if (mainShmem[startOfLocalWorkQ + i] < (metaData.totalMetaLength - 1)) {
        cooperative_groups::memcpy_async(tile, (&mainShmem[begSMallRegShmemB]),
            &getSourceReduced(fbArgs, iterationNumb)[
                (mainShmem[startOfLocalWorkQ + i] + 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                    + tile.meta_group_rank() * 32], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<4>(sizeof(uint32_t))
                    );
    }
}




////////////////////MAin
/*
loading data about this block to shmem
*/
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block 
*/
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.consumer_wait();

    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);

    pipeline.consumer_release();


}



////////////////TOP
/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    if (localBlockMetaData[13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[13] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem);

    pipeline.consumer_release();

}

/////BOTTOM
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.producer_acquire();
    if (localBlockMetaData[14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[
                localBlockMetaData[14] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                    , pipeline);
    }
    pipeline.producer_commit();

}


