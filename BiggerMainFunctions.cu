#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/////////// loading functions

template <typename TXPI>

inline __device__  void loadRightLeft(ForBoolKernelArgs<TXPI> fbArgs, thread_block& cta, uint32_t localBlockMetaData[]
    , uint32_t mainShmem[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t* metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool isGoldForLocQueue[localWorkQueLength], int iterationNumb[1]) {

    if (mainShmem[startOfLocalWorkQ + i] < (metaData.totalMetaLength - 1)) {
        cooperative_groups::memcpy_async(tile, (&mainShmem[begSMallRegShmemB]),
            &getSourceReduced(fbArgs, iterationNumb)[
                (mainShmem[startOfLocalWorkQ + i] + 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
                    + tile.meta_group_rank() * 32], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<4>(sizeof(uint32_t))
                    );
    }
}


//if (mainShmem[startOfLocalWorkQ + i] > 0) {
//    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemA + tile.meta_group_rank()]),
//        &getSourceReduced(fbArgs, iterationNumb)[
//            (mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
//                //we look for indicies 31,63... up to metaData.mainArrXLength
//                + (tile.meta_group_rank() * 32) + 31]
//        , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
//
//}


//
//
//pipeline.producer_acquire();
////load data of interst form block to the right
//loadRightLeft(fbArgs, cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, i, tile, isGoldForLocQueue, iterationNumb);
//
//
//////load data of interst form block to the left
////if (mainShmem[startOfLocalWorkQ + i] > 0) {
////    cuda::memcpy_async(tile, (&mainShmem[begSMallRegShmemA + tile.meta_group_rank()]),
////        &getSourceReduced(fbArgs, iterationNumb)[
////            (mainShmem[startOfLocalWorkQ + i] - 1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
////                //we look for indicies 31,63... up to metaData.mainArrXLength
////                + (tile.meta_group_rank() * 32) + 31]
////        , cuda::aligned_size_t<4>(sizeof(uint32_t)), pipeline);
//
////}
//pipeline.producer_commit();