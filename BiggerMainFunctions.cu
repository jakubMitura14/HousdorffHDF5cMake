#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/*
load and dilatates the entries in gold or segm ...
all operations are on the single data block represented by single entry in metadata 

1) load data into pipeline head source shmem from either gold or segmentation array - and either ref or prev depending on wheather iteration number is odd or not ...
2) a) compute dilatations of souce and save to rs shmem; 
   b) also mark up and bottom of is anythink in padding
   d) simultaneously pipeline should load the  data from the block above (if it exist) to register shmem  one
3) dilatate from block above save to resshmem and simultaneously load data from block below and save to register shmem two
commit so we will have register shmem one free
4) dilatate from below  and simulatenously using 4 tiles we load into register one the padding info required for dilatations - anterior, posterior, left, right
5) we dilatate anterior, posterior, left, right we need to have registers 2 cleared  to use and load to it the data from reduced gold or segm (originals) if it is to be validated - if not we skip this
   b) mark is block full - if it is all of the resshmem entries are equal UINT32_MAX
6) if it was to be validated we compare resshmem to loaded data and write down results
7) save data from resshmem to global memory
8) a) in case of non padding pass we use the data from is anything in padding  to activate neighbouring blocks
   b) save the updated values of block metadata back to global memory
*/
#pragma once
template <typename TXTOIO>
inline __device__ void loadAndDilatateAndSave(ForBoolKernelArgs<TXTOIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint8_t bigloop,
    uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , bool isAnythingInPadding[6], unsigned int iterationNumb[1], bool& isBlockFull, thread_block cta, uint16_t i
    , bool isBlockToBeValidated[1], unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], unsigned int worQueueStep[1]
    , uint32_t* mainArr, MetaDataGPU metaData , unsigned int* minMaxes, uint32_t* workQueue, unsigned int localMinMaxes[5], unsigned int localBlockMetaData[19]
    , uint32_t mainShmem[4468], uint32_t isGold[1]
    , cuda::barrier<cuda::thread_scope::thread_scope_block> barrier
) {

    /*
     main shared memory spaces reference 
    0-1023 : sourceShmem
    1024-2047 : resShmem
    2048-3071 : first register space
    3072-4095 : second register space
    4096-4468 (372 length) : place for local work queue in dilatation kernels
    */
    //we use isGold[0] and iteration number to establish what we need to load
    //(iterationNumb[0] & 1) will evaluate to 1 for odd iteration rest of calculation will lead to correct list for given combination
    
    //loading sourceshmem
    cooperative_groups::memcpy_async(cta, (&mainShmem[0]), (&mainArr[ metaData.mainArrXLength*( 1+ (1-isGold[0]) +  ((1+ (iterationNumb[0] & 1))*2 ) ) ])
    , cuda::aligned_size_t<128>(sizeof(uint32_t) * (metaData.mainArrXLength) ));
    //now to registers we load also 
    


    /// ///////////////// dilatations
    // first we perform up and down dilatations
    resShared[threadIdx.x][threadIdx.y] = bitDilatate(sourceShared[threadIdx.x][threadIdx.y]);

    //we also need to set shmem paddings on the basis of first and last bits ...

    //top            0)top  1)bottom, 2)left 3)right, 4)anterior, 5)posterior, 
    if (isBitAt(sourceShared[threadIdx.x][threadIdx.y], 0)) {
        // printf("setting padding top val %d \n ", isAnythingInPadding[0]);
        isAnythingInPadding[0] = true;
    };
    //shmemPaddingsTopBottom[threadIdx.x][threadIdx.y][0]=true; };
//bottom
    if (isBitAt(sourceShared[threadIdx.x][threadIdx.y], (fbArgs.dbZLength - 1))) {
        //shmemPaddingsTopBottom[threadIdx.x][threadIdx.y][1] = true;
        isAnythingInPadding[1] = true;
    };
    //now we will  additionally get bottom bit of block above and top of block below given they exist 
    checkBlockToUpAndBottom(fbArgs, tensorslice, localWorkQueue, i, getSourceReduced(fbArgs, localWorkQueue, i, iterationNumb), resShared);

    //we also need to save data into shared memory weather this block is marked to be validated (are there any voxels that can be potentially saved into result queue)
    auto activeC = coalesced_threads();

    loadSmallVars(fbArgs, tensorslice, resultfpOffset, resultfnOffset, isBlockToBeValidated, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3]
        , activeC, localFpConter, localFnConter);




    sync(cta);//we loaded and  dilatated up and down - we need also to dilatate anterior, posterior, Hovewer in those cases we need also to check boundary conditions ...


              //TODO() 4 corner threads has too much work and probably couse warp divergence ...- so those that for example have both threadidx and y=0 or max ...
    //we will also immidiately send data to    

    //krowa so we can use 8 tiles 4 will check for the is anythink in padding and 4 will load from neighbours ...

    //#left
    dilatateHelper((threadIdx.x == 0), 2, threadIdx.y, (-1), (0), sourceShared, resShared, isAnythingInPadding, localWorkQueue[i][0] > 0,
        tensorslice, fbArgs, localWorkQueue, i, iterationNumb,
        threadIdx.y, (fbArgs.dbXLength - 1));
    ////right
    dilatateHelper((threadIdx.x == (fbArgs.dbXLength - 1)), 3, threadIdx.y, (1), (0), sourceShared, resShared, isAnythingInPadding
        , (localWorkQueue[i][0] < (fbArgs.metaData.metaXLength - 1)), tensorslice, fbArgs, localWorkQueue, i, iterationNumb, threadIdx.y, 0);
    sync(cta);// we are synchronizing just becouse of corners TODO() rethink corners                
    //posterior
    dilatateHelper((threadIdx.y == 0), 5, threadIdx.x, (0), (-1), sourceShared, resShared, isAnythingInPadding, localWorkQueue[i][1] > 0,
        tensorslice, fbArgs, localWorkQueue, i, iterationNumb, (fbArgs.dbYLength - 1), threadIdx.x);
    //anterior
    dilatateHelper((threadIdx.y == (fbArgs.dbYLength - 1)), 4, threadIdx.x, (0), (1), sourceShared, resShared, isAnythingInPadding
        , localWorkQueue[i][1] < (fbArgs.dbYLength - 1), tensorslice, fbArgs, localWorkQueue, i, iterationNumb, 0, threadIdx.x);



    //syncing we now check is block full
    //marking that we have no more space for dilatations
    isBlockFull = (resShared[threadIdx.x][threadIdx.y] == UINT32_MAX);

    isBlockFull = __syncthreads_and(isBlockFull); ;// all dilatations completed 


   //now we need to move the data into global memory - so dilatated arrays to dilatation reduced arrays and paddings to paddings store
    saveToDilatationArr(fbArgs, tensorslice, resShared, getTargetReduced(fbArgs, localWorkQueue, i, iterationNumb), localWorkQueue, i);

}







/*
load and dilatates the entries in gold or segm ...
*/
#pragma once
template <typename TXTOIO>
inline __device__ void validateAndUpMetaCounter(ForBoolKernelArgs<TXTOIO> fbArgs, char* tensorslice,
    uint16_t localWorkQueue[localWorkQueLength][4], uint8_t bigloop,
    uint32_t sourceShared[32][32], uint32_t resShared[32][32]
    , bool isAnythingInPadding[6], unsigned int iterationNumb[1], bool isBlockFull, thread_block cta, uint16_t i
    , bool isBlockToBeValidated[1], unsigned int localTotalLenthOfWorkQueue[1], unsigned int localFpConter[1], unsigned int localFnConter[1]
    , unsigned int resultfpOffset[1], unsigned int resultfnOffset[1], unsigned int worQueueStep[1], unsigned int& old
    , unsigned int blockFpConter[1], unsigned int blockFnConter[1]
) {
    if ((isBlockToBeValidated[0] || iterationNumb[0] == 0) && !isBlockFull) {
        //now first we need to check for bits that are true now after dilatation but were not in source we will save it in res shmem becouse we will no longer need it
        resShared[threadIdx.x][threadIdx.y] = ((~sourceShared[threadIdx.x][threadIdx.y]) & resShared[threadIdx.x][threadIdx.y]);
        //now we load appropriate reference array (opposite to source)

        if (localWorkQueue[i][3] == 0) { loadDataToShmem(fbArgs, tensorslice, sourceShared, fbArgs.reducedGoldRef, localWorkQueue, i); };
        if (localWorkQueue[i][3] == 1) { loadDataToShmem(fbArgs, tensorslice, sourceShared, fbArgs.reducedSegmRef, localWorkQueue, i); };

        //we now look for bits prasent in both reference arrays and current one
        resShared[threadIdx.x][threadIdx.y] = ((sourceShared[threadIdx.x][threadIdx.y]) & resShared[threadIdx.x][threadIdx.y]);
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(resShared[threadIdx.x][threadIdx.y], bitPos)) {
                //first we add to the resList
                //TODO consider first passing it into shared memory and then async mempcy ...
                //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)


                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (localWorkQueue[i][3] == 1) { old = atomicAdd(&(localFpConter[0]), 1) + resultfnOffset[0]; };
                if (localWorkQueue[i][3] == 0) { old = atomicAdd(&(localFnConter[0]), 1) + resultfpOffset[0]; };




                fbArgs.metaData.resultList[old * 5] = (localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x);
                fbArgs.metaData.resultList[old * 5 + 1] = (localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y);
                fbArgs.metaData.resultList[old * 5 + 2] = (localWorkQueue[i][2] * fbArgs.dbZLength + bitPos);
                fbArgs.metaData.resultList[old * 5 + 3] = (localWorkQueue[i][3]);
                fbArgs.metaData.resultList[old * 5 + 4] = (iterationNumb[0]);

                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 0, 0)[old] = int(localWorkQueue[i][0] * fbArgs.dbXLength + threadIdx.x);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 1, 0)[old] = int(localWorkQueue[i][1] * fbArgs.dbYLength + threadIdx.y);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 2, 0)[old] = int(localWorkQueue[i][2] * fbArgs.dbZLength + bitPos);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 3, 0)[old] = int(localWorkQueue[i][3]);
                //getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old] = int(iterationNumb[0]);






    //            if (getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old] !=9) {
    //    printf("\n in kernel saving result x %d y %d z %d isGold %d iteration %d spotToUpdate %d  fpLocCounter %d  fnLocCounter %d   resultfpOffset %d  resultfnOffset %d  xMeta %d yMeta %d zMeta %d isGold %d \n ",

    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 0, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 1, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 2, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 3, 0)[old],
    //        getTensorRow<int>(tensorslice, fbArgs.metaData.resultList, fbArgs.metaData.resultList.Ny, 4, 0)[old]
    //        , old
    //        , localFpConter[0]
    //        , localFnConter[0]
    //        , resultfnOffset[0]
    //        , resultfpOffset[0]
    //        , localWorkQueue[i][0]
    //        , localWorkQueue[i][1]
    //        , localWorkQueue[i][2]
    //        , localWorkQueue[i][3]

    //    );
    //}
    //else {
    //    printf(" *** ");
    //}

            }
        }
        sync(cta);


        coalesced_group activeE = coalesced_threads();
        //update metadata  fp, fn conters
        if (localWorkQueue[i][3] == 1) {//gold
            updateMetaCounters(tensorslice, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3], fbArgs.metaData.fpCounter, localFpConter[0], activeE);
        };
        if (localWorkQueue[i][3] == 0) {//segm
            updateMetaCounters(tensorslice, localWorkQueue[i][0], localWorkQueue[i][1], localWorkQueue[i][2], localWorkQueue[i][3], fbArgs.metaData.fnCounter, localFnConter[0], activeE);
        };
        if (isToBeExecutedOnActive(activeE, 4)) {
            blockFpConter[0] += localFpConter[0];
            localFpConter[0] = 0;
        };
        if (isToBeExecutedOnActive(activeE, 5)) {
            blockFnConter[0] += localFnConter[0];
            localFnConter[0] = 0;

        };

    }
}


