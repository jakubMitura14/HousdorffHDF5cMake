#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/////////// loading functions





////////////////////MAin
/*
loading data about this block to shmem
*/
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block 
*/
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.consumer_wait();

    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);

    pipeline.consumer_release();


}

////////////////TOP
/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20+13] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem,i);

    pipeline.consumer_release();

}

/////BOTTOM
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[
                localBlockMetaData[(i & 1) * 20+14] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                    , pipeline);
    }
    pipeline.producer_commit();

}

template <typename TXPI>
inline __device__  void processBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
        , 0, 31
        , begSecRegShmem,i);

    pipeline.consumer_release();

}






///////////// right
template <typename TXPI>
inline __device__  void loadRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {



    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+16] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20+16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
        3, (1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 0
        , 16, begfirstRegShmem, localBlockMetaData,i);

    pipeline.consumer_release();
}



///////////// left
template <typename TXPI>
inline __device__  void loadLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {



    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+15] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20+15] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse((threadIdx.x == 0),
        2, (-1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 31
        , 15, begSecRegShmem, localBlockMetaData,i);

    pipeline.consumer_release();
}

///////////// anterior
template <typename TXPI>
inline __device__  void loadAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+17] < isGoldOffset && tile.meta_group_rank() == 0) {

        cuda::memcpy_async(tile, &mainShmem[begfirstRegShmem], &getSourceReduced(fbArgs, iterationNumb)[
            (localBlockMetaData[(i & 1) * 20+17]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * (32)), pipeline);

    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();
    dilatateHelperForTransverse((threadIdx.y == (fbArgs.dbYLength - 1)), 4
        , (0), (1), mainShmem, isAnythingInPadding
        , 0, threadIdx.x
        , 17, begfirstRegShmem, localBlockMetaData,i);
    pipeline.consumer_release();
}

///////////// posterior
template <typename TXPI>
inline __device__  void loadPosterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+18] < isGoldOffset && tile.meta_group_rank() == 0) {

        cuda::memcpy_async(tile, &mainShmem[begSecRegShmem], &getSourceReduced(fbArgs, iterationNumb)[
            (localBlockMetaData[(i & 1) * 20+18]) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * (32)), pipeline);

    }
    pipeline.producer_commit();
}





//////////// last load 

/*
load reference if needed or data for next iteration if there is such
*/
template <typename TXPI>
inline __device__  void lastLoad(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding, uint32_t*& origArrs, unsigned int*& worQueueStep) {

    pipeline.producer_acquire();
      
    //if block should be validated we load data for validation
    if (localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                   > localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
      
    }
    else {//if we are not validating we immidiately start loading data for next loop
        if (i + 1 <= worQueueStep[0]) {
            loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 1, i);
        }
    }


    pipeline.producer_commit();
}

template <typename TXPI>
inline __device__  void processPosteriorAndSaveResShmem(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding, bool*& isBlockFull) {

    pipeline.consumer_wait();
    //dilatate posterior 
    dilatateHelperForTransverse((threadIdx.y == 0), 5
        , (0), (-1), mainShmem, isAnythingInPadding
        , 0, threadIdx.x // we add offset depending on y dimension
        , 18, begSecRegShmem, localBlockMetaData,i);
    //now all data should be properly dilatated we save it to global memory
    getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
        + threadIdx.x + threadIdx.y * 32]
        = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];
    //marking weather block is already full and no more dilatations are possible 
    if (mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] != UINT32_MAX) {
        isBlockFull[(i&1)] = false;
    }
    pipeline.consumer_release();
}


//////////// validation

template <typename TXPI>
inline __device__  void validate(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding, bool*& isBlockFull
, unsigned int*& localFpConter, unsigned int*& localFnConter
, uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb

) {

    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



        //we now look for bits prasent in both reference arrays and current one
        mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

        // now we look through bits and when some is set we call it a result 
        #pragma unroll
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                //first we add to the resList
                //TODO consider first passing it into shared memory and then async mempcy ...
                //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                unsigned int old = 0;
                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (isGoldForLocQueue[i]) {
                    old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 4];
                }
                else {
                    old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 3];
                };
                //   add results to global memory    
                //we add one gere jjust to distinguish it from empty result
                resultListPointerMeta[old] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                resultListPointerLocal[old] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);

                /*   printf("rrrrresult i %d  meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d  xLoc %d yLoc %d zLoc %d \n"
                       ,i
                       ,mainShmem[startOfLocalWorkQ + i]
                       , isGoldForLocQueue[i]
                       , old
                       , localFpConter[0]
                       , localFnConter[0]
                       , localBlockMetaData[(i & 1) * 20+ 5]
                       , localBlockMetaData[(i & 1) * 20+6]
                       , uint32_t(mainShmem[startOfLocalWorkQ + i] + isGoldOffset * isGoldForLocQueue[i])
                       , uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x))
                       , threadIdx.x
                       , threadIdx.y
                       , bitPos
                   );*/

            }

        };
    }
}
