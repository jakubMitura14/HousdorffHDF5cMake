#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/////////// loading functions





////////////////////MAin
/*
loading data about this block to shmem
*/
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block 
*/
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.consumer_wait();

    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);

    pipeline.consumer_release();


}



////////////////TOP
/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb) {

    pipeline.producer_acquire();
    if (localBlockMetaData[13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[13] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem);

    pipeline.consumer_release();

}

/////BOTTOM
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.producer_acquire();
    if (localBlockMetaData[14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[
                localBlockMetaData[14] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], //we look for indicies 0,32,64... up to metaData.mainArrXLength
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
                    , pipeline);
    }
    pipeline.producer_commit();

}

template <typename TXPI>
inline __device__  void processBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
        , 0, 31
        , begSecRegShmem);

    pipeline.consumer_release();

}






///////////// right
template <typename TXPI>
inline __device__  void loadRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {



    pipeline.producer_acquire();
    if (localBlockMetaData[16] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t*& localBlockMetaData
    , uint32_t*& mainShmem, cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool*& isGoldForLocQueue, int*& iterationNumb, bool*& isAnythingInPadding) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse((threadIdx.x == (fbArgs.dbXLength - 1)),
        3, (1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y,0
        , 16, begSMallRegShmemB, localBlockMetaData);

    pipeline.consumer_release();



}




