#include "hip/hip_runtime.h"
#pragma once


#include "hip/hip_runtime.h"
#include "CPUAllocations.cu"
#include "MetaData.cu"
#include "IterationUtils.cu"
#include "ExceptionManagUtils.cu"
#include "CooperativeGroupsUtils.cu"
#include "MainPassFunctions.cu"
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

using namespace cooperative_groups;

/////////// loading functions





////////////////////MAin
/*
loading data about this block to shmem
*/
template <typename TXPI>
inline __device__  void loadMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    cuda::memcpy_async(cta, &mainShmem[begSourceShmem], &getSourceReduced(fbArgs, iterationNumb)[
        mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
        cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength), pipeline);
    pipeline.producer_commit();


}

/*
process data about this block 
*/
template <typename TXPI>
inline __device__  void processMain(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isBlockFull)[1]) {

    pipeline.consumer_wait();
    //if ((((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]))  > 0)
//    || mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]==0
//    ) {
   // isBlockFull[0] = false;
    //    }
    //if (__popc(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32])<32) {
    //
    //    isBlockFull[0] = false;
    //}


    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = bitDilatate(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]);
    //marking weather block is already full and no more dilatations are possible 


    pipeline.consumer_release();


}

////////////////TOP
/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void loadTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+13] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 13]
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}


/*
loading data about block above to shmem
*/
template <typename TXPI>
inline __device__  void processTop(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6] ) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(0, mainShmem, isAnythingInPadding, localBlockMetaData, 13
        , 31, 0
        , begfirstRegShmem,i);

    pipeline.consumer_release();

}

/////BOTTOM
template <typename TXPI>
inline __device__  void loadBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+14] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 14] 
            * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();

}

template <typename TXPI>
inline __device__  void processBottom(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperTopDown(1, mainShmem, isAnythingInPadding, localBlockMetaData, 14
        , 0, 31
        , begSecRegShmem,i);

    pipeline.consumer_release();

}






///////////// right
template <typename TXPI>
inline __device__  void loadRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+16] < isGoldOffset) {
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20+16] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])], 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processRight(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs,(threadIdx.x == (fbArgs.dbXLength - 1)),
        3, (1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 0
        , 16, begfirstRegShmem, localBlockMetaData,i);

    pipeline.consumer_release();
}



///////////// left
template <typename TXPI>
inline __device__  void loadLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {



    pipeline.producer_acquire();
    if (mainShmem[startOfLocalWorkQ + i]>0) {
        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[(mainShmem[startOfLocalWorkQ + i]-1) * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processLeft(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {


    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs,(threadIdx.x == 0),
        2, (-1), (0), mainShmem, isAnythingInPadding
        , threadIdx.y, 31
        , 15, begSecRegShmem, localBlockMetaData,i);

    pipeline.consumer_release();
}

///////////// anterior
template <typename TXPI>
inline __device__  void loadAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+17] < isGoldOffset ) {

        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 17] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}


template <typename TXPI>
inline __device__  void processAnterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.consumer_wait();

    dilatateHelperForTransverse(fbArgs,(threadIdx.y == (fbArgs.dbYLength - 1)), 4
        , (0), (1), mainShmem, isAnythingInPadding
        , 0, threadIdx.x
        , 17, begfirstRegShmem, localBlockMetaData, i);
    pipeline.consumer_release();
}

///////////// posterior
template <typename TXPI>
inline __device__  void loadPosterior(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]) {

    pipeline.producer_acquire();
    if (localBlockMetaData[(i & 1) * 20+18] < isGoldOffset) {


        cuda::memcpy_async(cta, (&mainShmem[begSecRegShmem]),
            &getSourceReduced(fbArgs, iterationNumb)[localBlockMetaData[(i & 1) * 20 + 18] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])],
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
    }
    pipeline.producer_commit();
}





//////////// last load 

/*
load reference if needed or data for next iteration if there is such
*/
template <typename TXPI>
inline __device__  void lastLoad(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , uint32_t*& origArrs, unsigned int (&worQueueStep)[1]) {

    pipeline.producer_acquire();
      
    //if block should be validated we load data for validation
    if (localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
                   > localBlockMetaData[(i & 1) * 20+((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        cuda::memcpy_async(cta, (&mainShmem[begfirstRegShmem]),
            &origArrs[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (isGoldForLocQueue[i])], //we look for 
            cuda::aligned_size_t<128>(sizeof(uint32_t) * metaData.mainArrXLength)
            , pipeline);
      
    }
    else {//if we are not validating we immidiately start loading data for next loop
        if (i + 1 <= worQueueStep[0]) {
            loadMetaDataToShmem(cta, localBlockMetaData, mainShmem, pipeline, metaDataArr, metaData, 1, i);
        }
    }


    pipeline.producer_commit();
}

template <typename TXPI>
inline __device__  void processPosteriorAndSaveResShmem(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta
    , uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6],
    bool(&isBlockFull)[1]) {

    pipeline.consumer_wait();
    //dilatate posterior 
    dilatateHelperForTransverse(fbArgs, (threadIdx.y == 0), 5
        , (0), (-1), mainShmem, isAnythingInPadding
        , fbArgs.dbYLength - 1, threadIdx.x // we add offset depending on y dimension
        , 18, begSecRegShmem, localBlockMetaData, i);
    //now all data should be properly dilatated we save it to global memory
    //try save target reduced via mempcy async ...

    getTargetReduced(fbArgs, iterationNumb)[mainShmem[startOfLocalWorkQ + i] * metaData.mainArrSectionLength + metaData.mainArrXLength * (1 - isGoldForLocQueue[i])
        + threadIdx.x + threadIdx.y * 32]
        = mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32];

    //TODO remove 

    //for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
    //    if (threadIdx.x == 0 && threadIdx.y == 0) {

    //    //if any bit here is set it means it should be added to result list 
    //    if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
    //        //if (mainShmem[startOfLocalWorkQ + i] * 32 + bitPos>130) {
    //            printf("bit set loc %d isGold %d \n", mainShmem[startOfLocalWorkQ + i] * 32 + bitPos, isGoldForLocQueue[i]);
    //        //}
    //    }
    //    
    //    }
    //}
    
    
    
    //if (!(localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    //> localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)])) {// so count is bigger than counter so we should validate
    //    mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = 0;
    //    mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 0;
    //}


    pipeline.consumer_release();
}


//////////// validation

template <typename TXPI>
inline __device__  void validate(ForBoolKernelArgs<TXPI>& fbArgs, thread_block& cta, uint32_t(&localBlockMetaData)[40]
    , uint32_t(&mainShmem)[lengthOfMainShmem], cuda::pipeline<cuda::thread_scope_block>& pipeline
    , uint32_t*& metaDataArr, MetaDataGPU& metaData, uint32_t& i, thread_block_tile<32>& tile
    , bool(&isGoldForLocQueue)[localWorkQueLength], int(&iterationNumb)[1], bool(&isAnythingInPadding)[6]
    , bool(&isBlockFull)[1]
, unsigned int (&localFpConter)[1], unsigned int (&localFnConter)[1]
, uint32_t*& resultListPointerMeta, uint32_t*& resultListPointerLocal, uint32_t*& resultListPointerIterNumb

) {

    if (localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 1)] //fp for gold and fn count for not gold
    > localBlockMetaData[(i & 1) * 20 + ((1 - isGoldForLocQueue[i]) + 3)]) {// so count is bigger than counter so we should validate
        //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 
        //    ((~mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) 
        //        & mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32]);



        //we now look for bits prasent in both reference arrays and current one
       // mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = ((mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32]) & mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32]);

        // now we look through bits and when some is set we call it a result 
        #pragma unroll
        for (uint8_t bitPos = 0; bitPos < 32; bitPos++) {
            //if any bit here is set it means it should be added to result list 
            if (isBitAt(mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && !isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                && isBitAt(mainShmem[begfirstRegShmem + threadIdx.x + threadIdx.y * 32], bitPos)
                ) {
           // if (isBitAt(mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32], bitPos)) {
                //first we add to the resList
                //TODO consider first passing it into shared memory and then async mempcy ...
                //we use offset plus number of results already added (we got earlier count from global memory now we just atomically add locally)
                unsigned int old = 0;
                ////// IMPORTANT for some reason in order to make it work resultfnOffset and resultfnOffset swith places
                if (isGoldForLocQueue[i]) {
                    old = atomicAdd_block(&(localFpConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 6] + localBlockMetaData[(i & 1) * 20 + 3];
                }
                else {
                    old = atomicAdd_block(&(localFnConter[0]), 1) + localBlockMetaData[(i & 1) * 20 + 5] + localBlockMetaData[(i & 1) * 20 + 4];
                //    printf("local fn counter add \n");

                };
                //   add results to global memory    
                //we add one gere jjust to distinguish it from empty result
                resultListPointerMeta[old] = uint32_t(mainShmem[startOfLocalWorkQ + i] + (isGoldOffset * isGoldForLocQueue[i]) + 1);
                resultListPointerLocal[old] = uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x));
                resultListPointerIterNumb[old] = uint32_t(iterationNumb[0]);

                   //printf("rrrrresult i %d  meta %d isGold %d old %d localFpConter %d localFnConter %d fpOffset %d fnOffset %d linIndUpdated %d  localInd %d  xLoc %d yLoc %d zLoc %d \n"
                   //    ,i
                   //    ,mainShmem[startOfLocalWorkQ + i]
                   //    , isGoldForLocQueue[i]
                   //    , old
                   //    , localFpConter[0]
                   //    , localFnConter[0]
                   //    , localBlockMetaData[(i & 1) * 20+ 5]
                   //    , localBlockMetaData[(i & 1) * 20+6]
                   //    , uint32_t(mainShmem[startOfLocalWorkQ + i] + isGoldOffset * isGoldForLocQueue[i])
                   //    , uint32_t((fbArgs.dbYLength * 32 * bitPos) + (threadIdx.y * 32) + (threadIdx.x))
                   //    , threadIdx.x
                   //    , threadIdx.y
                   //    , bitPos
                   //);


                   printf("\n rrrrresult meta %d isGold %d old %d  xLoc %d yLoc %d zLoc %d iterNumbb %d \n"
                       , mainShmem[startOfLocalWorkQ + i]
                       , isGoldForLocQueue[i]
                       , old
                       , threadIdx.x
                       , threadIdx.y
                       , bitPos
                       , iterationNumb[0]
                   );


            }

        };
        //mainShmem[begResShmem + threadIdx.x + threadIdx.y * 32] = 0;
        //mainShmem[begSourceShmem + threadIdx.x + threadIdx.y * 32] = 0;

    }
}
