#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "Structs.cu"
#include <cstdint>
#include <assert.h>
/*
given reference to array  it will return reference to tensor row
tensorslice - holds referenco to slice
tensor - reference to struct representing 3d tensor
YLength - max in Y dimension of array we iterate through
y,z - coordinates of row of intrest in tensor
*/
#pragma once
template <typename UIO>
inline __device__ UIO* getTensorRow(char* tensorslice, array3dWithDimsGPU tensor, int YLength, int y, int z) {
    tensorslice = ((char*)tensor.arrPStr.ptr) + z * tensor.arrPStr.pitch * YLength;
    return (UIO*)(tensorslice + y * tensor.arrPStr.pitch);
}

#pragma once
template <typename UHO>
inline __device__ UHO* getTensorRowSimple(char* tensorslice, hipPitchedPtr tensor, int YLength, int y, int z) {
    tensorslice = ((char*)tensor.ptr) + z * tensor.pitch * YLength;
    return (UHO*)(tensorslice + y * tensor.pitch);
}



