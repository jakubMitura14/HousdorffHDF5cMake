#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>
#include ""
#include <cstdint>
#include <assert.h>
#include <numeric>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
using namespace cooperative_groups;





__global__ void testPipeline(uint32_t* global_out, uint32_t* global_inA, uint32_t* globalOutGPUB, float* globalDummyGPU) {


}
